#include "hip/hip_runtime.h"
#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"
#include "cuda_accel_IN.h"

int    cuMedianBuffSz = -1;

void CPU_Norm_Spread(cuFFdotBatch* batch, int norm_type, fcomplexcu* fft)
{
  nvtxRangePush("CPU_Norm_Spread");

  int harm = 0;

  FOLD // Normalise, spread and copy raw input fft data to pinned memory  .
  {
    for (int stack = 0; stack < batch->noStacks; stack++)
    {
      cuFfdotStack* cStack = &batch->stacks[stack];

      int sz = 0;

#ifdef TIMING // Timing  .
      struct timeval start, end;
      gettimeofday(&start, NULL);
#endif

      for (int si = 0; si < cStack->noInStack; si++)
      {
        for (int step = 0; step < batch->noSteps; step++)
        {
          rVals* rVal = &batch->rValues[step][harm];

          if ( rVal->numdata )
          {
            if ( norm_type== 0 )  // Normal normalise  .
            {
              double norm;    /// The normalising factor
              int start = rVal->lobin < 0 ? -rVal->lobin : 0 ;
              int end   = rVal->lobin + rVal->numdata >= batch->SrchSz->searchRHigh ? rVal->lobin + rVal->numdata - batch->SrchSz->searchRHigh : rVal->numdata ;

              FOLD // Calculate and store powers  .
              {
                nvtxRangePush("Powers");
                for (int ii = 0; ii < rVal->numdata; ii++)
                {
                  if ( rVal->lobin+ii < 0 || rVal->lobin+ii  >= batch->SrchSz->searchRHigh ) // Zero Pad
                  {
                    batch->normPowers[ii] = 0;
                  }
                  else
                  {
                    batch->normPowers[ii] = POWERCU(fft[rVal->lobin+ii].r, fft[rVal->lobin+ii].i);
                  }
                }
                nvtxRangePop();
              }

              FOLD // Calculate normalisation factor from median  .
              {
                nvtxRangePush("Median");
                if ( batch->flags & CU_NORM_EQUIV )
                {
                  norm = 1.0 / sqrt(median(batch->normPowers, (rVal->numdata)) / log(2.0));        /// NOTE: This is the same method as CPU version
                }
                else
                {
                  norm = 1.0 / sqrt(median(&batch->normPowers[start], (end-start)) / log(2.0));    /// NOTE: This is a slightly better method (in my opinion)
                }
                nvtxRangePop();
              }

              FOLD // Normalise and spread  .
              {
                nvtxRangePush("Write");
                for (int ii = 0; ( ii < rVal->numdata ) && ( (ii*ACCEL_NUMBETWEEN) < cStack->strideCmplx ); ii++)
                {
                  if ( rVal->lobin+ii < 0  || rVal->lobin+ii  >= batch->SrchSz->searchRHigh )  // Zero Pad
                  {
                    cStack->h_iData[sz + ii * ACCEL_NUMBETWEEN].r = 0;
                    cStack->h_iData[sz + ii * ACCEL_NUMBETWEEN].i = 0;
                  }
                  else
                  {
                    if ( ii * ACCEL_NUMBETWEEN > cStack->strideCmplx )
                    {
                      fprintf(stderr, "ERROR: nice_numdata is greater that width.\n");
                      exit(EXIT_FAILURE);
                    }

                    cStack->h_iData[sz + ii * ACCEL_NUMBETWEEN].r = fft[rVal->lobin + ii].r * norm;
                    cStack->h_iData[sz + ii * ACCEL_NUMBETWEEN].i = fft[rVal->lobin + ii].i * norm;
                  }
                }
                nvtxRangePop();
              }
            }
            else                  // or double-tophat normalisation
            {
              int nice_numdata = next2_to_n_cu(rVal->numdata);  // for FFTs

              if ( nice_numdata > cStack->width )
              {
                fprintf(stderr, "ERROR: nice_numdata is greater that width.\n");
                //exit(EXIT_FAILURE);
              }

              // Do the actual copy
              //memcpy(batch->h_powers, &fft[lobin], numdata * sizeof(fcomplexcu) );

              //  new-style running double-tophat local-power normalization
              float *loc_powers;

              //powers = gen_fvect(nice_numdata);
              for (int ii = 0; ii< nice_numdata; ii++)
              {
                batch->normPowers[ii] = POWERCU(fft[rVal->lobin+ii].r, fft[rVal->lobin+ii].i);
              }
              loc_powers = corr_loc_pow(batch->normPowers, nice_numdata);

              //memcpy(&batch->h_iData[sz], &fft[lobin], nice_numdata * sizeof(fcomplexcu) );

              for (int ii = 0; ii < rVal->numdata; ii++)
              {
                float norm = invsqrt(loc_powers[ii]);

                batch->h_iData[sz + ii * ACCEL_NUMBETWEEN].r = fft[rVal->lobin+ ii].r* norm;
                batch->h_iData[sz + ii * ACCEL_NUMBETWEEN].i = fft[rVal->lobin+ ii].i* norm;
              }

              vect_free(loc_powers);  // I hate doing this!!!
            }
          }

          sz += cStack->strideCmplx;
        }
        harm++;
      }

#ifdef TIMING // Timing  .
      gettimeofday(&end, NULL);

      float v1 =  ((end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec))*1e-3  ;
      batch->normTime[stack] += v1;
#endif
    }
  }

  nvtxRangePop();
}

/** Calculate the r bin values for this batch of steps and store them in planes->rInput
 *
 * This calculates r-low and halfwidth
 *
 * @param batch the batch to work with
 * @param searchRLow an array of the step r-low values
 * @param searchRHi an array of the step r-high values
 */
void setStackRVals(cuFFdotBatch* batch, double* searchRLow, double* searchRHi)
{
#ifdef STPMSG
  printf("\tSet Stack R-Vals\n");
#endif

  int       hibin;
  int       binoffset;  // The extra bins to add onto the start of the data
  double    drlo, drhi;

  int lobin;      /// The first bin to copy from the the input fft ( serachR scaled - halfwidth )
  int numdata;    /// The number of input fft points to read
  int numrs;      /// The number of good bins in the plane ( expanded units )

  for (int harm = 0; harm < batch->noHarms; harm++)
  {
    cuHarmInfo* cHInfo      = &batch->hInfos[harm];       // The current harmonic we are working on
    binoffset               = batch->planes[harm].kerStart / ACCEL_NUMBETWEEN; // This aligns all the planes so the all the "usable" parts start at the same offset in the stack

    for (int step = 0; step < batch->noSteps; step++)
    {
      rVals* rVal           = &batch->rValues[step][harm];

      if ( searchRLow[step] == searchRHi[step] )
      {
        rVal->drlo          = 0;
        rVal->lobin         = 0;
        rVal->numrs         = 0;
        rVal->numdata       = 0;
        rVal->expBin        = 0;
        rVal->step          = -1; // Invalid step!
      }
      else
      {
        drlo                = calc_required_r_gpu(cHInfo->harmFrac, searchRLow[step]);
        drhi                = calc_required_r_gpu(cHInfo->harmFrac, searchRHi[step] );

        lobin               = (int) floor(drlo) - binoffset;
        hibin               = (int) ceil(drhi)  + binoffset;

        numdata             = hibin - lobin + 1;
        numrs               = (int) ((ceil(drhi) - floor(drlo)) * ACCEL_RDR + DBLCORRECT) + 1;

        if ( harm == 0 )
          numrs             = batch->accelLen;
        else if ( numrs % ACCEL_RDR )
          numrs             = (numrs / ACCEL_RDR + 1) * ACCEL_RDR;

        rVal->drlo          = drlo;
        rVal->lobin         = lobin;
        rVal->numrs         = numrs;
        rVal->numdata       = numdata;
        rVal->expBin        = (lobin+binoffset)*ACCEL_RDR;

        int noEls           = numrs + 2*binoffset*ACCEL_RDR;

        if  ( noEls > cHInfo->width )
        {
          fprintf(stderr, "ERROR: Number of elements in step greater than width of the plane! harm: %i\n", harm);
          exit(EXIT_FAILURE);
        }
      }
    }
  }
}

/** Initialise input data for a f-∂f plane(s)  ready for convolution  .
 * This:
 *  Normalises the chunk of input data
 *  Spreads it (interbinning)
 *  FFT it ready for convolution
 *
 * @param batch the batch to work with
 * @param norm_type   The type of normalisation to perform
 */
void initInput(cuFFdotBatch* batch, int norm_type )
{
#ifdef TIMING // Timing variables  .
  struct timeval start, end;
#endif

  if ( batch->rValues[0][0].numrs ) // This is real data ie this isn't just a call to finish off asynchronous work
  {
    nvtxRangePush("Input");

    fcomplexcu* fft = (fcomplexcu*)batch->sInf->sSpec->fftInf.fft;

#ifdef STPMSG
    printf("\tInput\n");
#endif

    FOLD  // Normalise and spread and copy to device memory  .
    {
      if ( batch->flags & CU_NORM_CPU  ) // Copy chunks of FFT data and normalise and spread using the CPU  .
      {
#ifdef STPMSG
        printf("\t\tCPU normalisation\n");
#endif

        FOLD // Blocking synchronisation, Make sure the previous thread has complete reading from page locked memory
        {
          nvtxRangePush("EventSynch");
          CUDA_SAFE_CALL(hipGetLastError(), "Before Synchronising");
          CUDA_SAFE_CALL(hipEventSynchronize(batch->iDataCpyComp), "At a blocking synchronisation. This is probably a error in one of the previous asynchronous CUDA calls.");
          nvtxRangePop();
        }

        FOLD // Zero pinned host memory  .
        {
          nvtxRangePush("Zero");
          memset(batch->h_iData, 0, batch->inpDataSize);
          nvtxRangePop();
        }

        CPU_Norm_Spread(batch, norm_type, fft);

        if ( batch->flags & CU_INPT_FFT_CPU ) // CPU FFT  .
        {
#ifdef STPMSG
          printf("\t\tCPU FFT Input\n");
#endif

#pragma omp critical
          FOLD
          {
            for (int stack = 0; stack < batch->noStacks; stack++)
            {
              cuFfdotStack* cStack = &batch->stacks[stack];

#ifdef TIMING // Timing  .
              gettimeofday(&start, NULL);
#endif

              nvtxRangePush("CPU FFT");
              fftwf_execute_dft(cStack->inpPlanFFTW, (fftwf_complex*)cStack->h_iData, (fftwf_complex*)cStack->h_iData);
              nvtxRangePop();

#ifdef TIMING // Timing  .
              gettimeofday(&end, NULL);

              float v1 =  ((end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec))*1e-3  ;
              //printf("Input FFT stack %02i  %15.2f \n", stack, v1);
              batch->InpFFTTime[stack] += v1;
#endif

            }
          }
        }

        FOLD // Synchronisation  .
        {
          // Wait for per stack multiplications to finish
          for (int ss = 0; ss < batch->noStacks; ss++)
          {
            cuFfdotStack* cStack = &batch->stacks[ss];
            CUDA_SAFE_CALL(hipStreamWaitEvent(batch->inpStream, cStack->multComp, 0), "Waiting for GPU to be ready to copy data to device.\n");
          }

          // Wait for batch multiplications to finish
          CUDA_SAFE_CALL(hipStreamWaitEvent(batch->inpStream, batch->multComp, 0), "Waiting for GPU to be ready to copy data to device.\n");

#ifdef TIMING
          hipEventRecord(batch->iDataCpyInit, batch->inpStream);
#endif

        }

        FOLD // Copy pinned memory to device  .
        {
#ifdef STPMSG
          printf("\t\tCopy to device\n");
#endif
          CUDA_SAFE_CALL(hipMemcpyAsync(batch->d_iData, batch->h_iData, batch->inpDataSize, hipMemcpyHostToDevice, batch->inpStream), "Failed to copy input data to device");
          CUDA_SAFE_CALL(hipGetLastError(), "Preparing the input data.");
        }

        FOLD // Synchronisation  .
        {
          hipEventRecord(batch->normComp,      batch->inpStream);
          hipEventRecord(batch->iDataCpyComp,  batch->inpStream);

          if ( batch->flags & CU_INPT_FFT_CPU )
          {
            for (int ss = 0; ss < batch->noStacks; ss++)
            {
              cuFfdotStack* cStack = &batch->stacks[ss];
              hipEventRecord(cStack->prepComp, batch->inpStream);
            }
          }
        }
      }
      else                              // Copy chunks of FFT data and normalise and spread using the GPU  .
      {
#ifdef STPMSG
        printf("\t\tGPU normalisation\n");
#endif

        FOLD // Synchronisation  .
        {
          // Make sure the previous thread has complete reading from page locked memory
          nvtxRangePush("EventSynch");
          CUDA_SAFE_CALL(hipEventSynchronize(batch->iDataCpyComp), "At a blocking synchronisation. This is probably a error in one of the previous asynchronous CUDA calls.");
          nvtxRangePop();
        }

        FOLD // Zero pinned host memory  .
        {
          nvtxRangePush("Zero");
          memset(batch->h_iData, 0, batch->inpDataSize);
          nvtxRangePop();
        }

        FOLD // Copy fft data to device  .
        {
          FOLD // Write fft data segments to contiguous page locked memory  .
          {
            int harm  = 0;
            int sz    = 0;

            for ( int stack = 0; stack< batch->noStacks; stack++)  // Loop over stack
            {
              cuFfdotStack* cStack = &batch->stacks[stack];

              for ( int plane = 0; plane < cStack->noInStack; plane++)
              {
                for (int step = 0; step < batch->noSteps; step++)
                {
                  rVals* rVal = &batch->rValues[step][harm];

                  if ( rVal->numdata )
                  {
                    int start = 0;
                    if ( rVal->lobin < 0 )
                      start = -rVal->lobin;

                    // Do the actual copy
                    memcpy(&batch->h_iData[sz+start], &fft[rVal->lobin+start], (rVal->numdata-start) * sizeof(fcomplexcu));
                  }
                  sz += cStack->strideCmplx;
                }
                harm++;
              }
            }
          }

          FOLD // Synchronisation  .
          {
            // Wait for per stack multiplications to finish
            for (int ss = 0; ss< batch->noStacks; ss++)
            {
              cuFfdotStack* cStack = &batch->stacks[ss];
              CUDA_SAFE_CALL(hipStreamWaitEvent(batch->inpStream, cStack->multComp, 0), "Waiting for GPU to be ready to copy data to device\n");
            }

            // Wait for batch multiplication to finish
            CUDA_SAFE_CALL(hipStreamWaitEvent(batch->inpStream, batch->multComp, 0), "Waiting for GPU to be ready to copy data to device\n");

#ifdef TIMING  // Timing  .
            hipEventRecord(batch->iDataCpyInit, batch->inpStream);
#endif
          }

          FOLD // Copy to device  .
          {
#ifdef STPMSG
            printf("\t\tCopy to device\n");
#endif
            CUDA_SAFE_CALL(hipMemcpyAsync(batch->d_iData, batch->h_iData, batch->inpDataSize, hipMemcpyHostToDevice, batch->inpStream), "Failed to copy data to device");

            // Synchronisation
            hipEventRecord(batch->iDataCpyComp, batch->inpStream);

            CUDA_SAFE_CALL(hipGetLastError(), "Copying a section of input FTD data to the device.");
          }
        }

        FOLD // Normalise and spread on GPU  .
        {
#ifdef STPMSG
          printf("\t\tNormalise on device\n");
#endif

#ifdef SYNCHRONOUS
          cuFfdotStack* pStack = NULL;  // Previous stack
#endif

          for ( int stack = 0; stack < batch->noStacks; stack++)  // Loop over stacks  .
          {
            cuFfdotStack* cStack = &batch->stacks[stack];

            FOLD // Synchronisation  .
            {
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->inptStream, batch->iDataCpyComp, 0), "Waiting for GPU to be ready to copy data to device\n");

#ifdef SYNCHRONOUS
              // Wait for previous FFT to complete
              if ( pStack != NULL )
                hipStreamWaitEvent(cStack->inptStream, pStack->normComp, 0);
#endif

#ifdef TIMING
              hipEventRecord(cStack->normInit, cStack->inptStream);
#endif
            }

            FOLD // Call the kernel to normalise and spread the input data  .
            {
              normAndSpread_f(cStack->inptStream, batch, stack );
            }

            FOLD // Synchronisation  .
            {
              hipEventRecord(cStack->normComp, cStack->inptStream);

#ifdef SYNCHRONOUS
              pStack = cStack;
#endif
            }
          }

#ifdef SYNCHRONOUS // Wait for the last stack to complete normalisation  .
          cuFfdotStack* lStack = &batch->stacks[batch->noStacks -1];
          hipStreamWaitEvent(lStack->inptStream, lStack->normComp, 0);
          hipEventRecord(batch->normComp, lStack->inptStream);
#endif
        }
      }
    }

    FOLD  // FFT the input on the GPU data  .
    {
      if ( !(batch->flags & CU_INPT_FFT_CPU) )
      {
#ifdef STPMSG
        printf("\t\tGPU FFT\n");
#endif

#ifdef SYNCHRONOUS
        cuFfdotStack* pStack = NULL;  // Previous stack
#endif

        for (int stackIdx = 0; stackIdx < batch->noStacks; stackIdx++)
        {
          cuFfdotStack* cStack = &batch->stacks[stackIdx];

          CUDA_SAFE_CALL(hipGetLastError(), "Before input fft.");

          FOLD // Synchronisation  .
          {
            hipStreamWaitEvent(cStack->fftIStream, cStack->normComp,     0);
            hipStreamWaitEvent(cStack->fftIStream, batch->normComp,      0);
            hipStreamWaitEvent(cStack->fftIStream, batch->iDataCpyComp,  0);

#ifdef SYNCHRONOUS
            // Wait for the search to complete before FFT'ing the next set of input
            hipStreamWaitEvent(cStack->fftIStream, batch->searchComp, 0);

            // Wait for previous FFT to complete
            if ( pStack != NULL )
              hipStreamWaitEvent(cStack->fftIStream, pStack->prepComp, 0);

            // Wait for all GPU normalisations to complete
            for (int stack2Idx = 0; stack2Idx < batch->noStacks; stack2Idx++)
            {
              cuFfdotStack* stack2 = &batch->stacks[stackIdx];
              hipStreamWaitEvent(cStack->fftIStream, stack2->normComp, 0);
            }
#endif
          }

          FOLD // Do the FFT on the GPU  .
          {
#pragma omp critical
            FOLD // Kernel
            {
#ifdef TIMING // Event .
              hipEventRecord(cStack->inpFFTinit, cStack->fftIStream);
#endif

              CUFFT_SAFE_CALL(hipfftSetStream(cStack->inpPlan, cStack->fftIStream),"Failed associating a CUFFT plan with FFT input stream\n");
              CUFFT_SAFE_CALL(hipfftExecC2C(cStack->inpPlan, (hipfftComplex *) cStack->d_iData, (hipfftComplex *) cStack->d_iData, HIPFFT_FORWARD),"Failed to execute input CUFFT plan.");

              CUDA_SAFE_CALL(hipGetLastError(), "FFT'ing the input data.");
            }
          }

          FOLD // Synchronisation  .
          {
            hipEventRecord(cStack->prepComp, cStack->fftIStream);

#ifdef SYNCHRONOUS
            pStack = cStack;
#endif

          }
        }
      }
    }

    nvtxRangePop();
  }
}
