#include "hip/hip_runtime.h"
#include "cuda_accel_CV.h"

/** Kernel for testing best possible performance - Just write to ffdot plain - 1 thread per complex value
 *
 * @param kernels
 * @param inpData
 * @param ffdot
 * @param width
 * @param height
 * @param stride
 * @param noSteps
 * @param kerHeight
 */
__global__ void convolveffdot00_k(const fcomplexcu* __restrict__ kernels, const fcomplexcu* __restrict__ inpData, fcomplexcu* __restrict__ ffdot, const int width, const int height, const int stride, const int noSteps, const int noPlns, int kerHeight )
{
  const int ix = blockIdx.x * CNV_DIMX + threadIdx.x;
  const int iy = blockIdx.y * CNV_DIMY + threadIdx.y;

  fcomplexcu ker;                                 /// kernel data
  uint nHeight = height * noSteps;

  ker.i = 0;
  ker.r = 0;

  if (ix < width && iy < nHeight)
  {
    // Calculate flat index
    const int idx = iy * stride + ix;

    ffdot[idx] = ker;
  }
}

__global__ void convolveffdot01_k(const __restrict__ fcomplexcu* kernels, const __restrict__ fcomplexcu* inpData, __restrict__ fcomplexcu* ffdot, const int width, const int height, const int stride, const int noSteps, const int noPlns, int kerHeight )
{
  const int bidx = threadIdx.y * CNV_DIMX + threadIdx.x;          /// Block ID - flat index
  const int tid  = blockIdx.x  * CNV_DIMX * CNV_DIMY + bidx;      /// Global thread ID - flat index ie column index of stack

  fcomplexcu ker;                                 /// kernel data

  if ( tid < width )  // Valid thread  .
  {
    int idx;                                      /// flat index of output plain

    FOLD  // Stride, kernel, input data & output data  .
    {
      kernels += tid;
      ffdot   += tid;
      inpData += tid;
    }

    FOLD // Read input data  .
    {
      for (int step = 0; step < noSteps; step++)
      {
        for (int pln = 0; pln < noPlns; pln++)                // Loop through the plains  .
        {
          fcomplexcu ipd        = inpData[ (int)(pln*noSteps*stride + step*stride) ];

          if ( ipd.r < 0 && ipd.r > 0 )  // TMP
          {
            printf("ipd \n");
          }
        }
      }
    }

    FOLD // Read kernel  .
    {
      for (int k = 0; k < kerHeight; k++ )
      {
        idx   = k * stride;
        ker   = kernels[idx];

        if ( ker.r < 0 && ker.r > 0 )  // TMP
        {
          printf("ker \n");
        }
      }
    }

    ker.i = 0;
    ker.r = 0;

    uint nHeight = height * noSteps;

    FOLD // Write data to plains  .
    {
      for (int y = 0; y < nHeight; y++ )
      {
        idx  = y * stride;

        FOLD // Write  .
        {
          ffdot[idx] = ker;
        }
      }
    }
  }
}

/** Kernel for testing best possible performance - Just write to ffdot plain - Each thread loops down over column  .
 *
 * @param kernels
 * @param inpData
 * @param ffdot
 * @param width
 * @param height
 * @param stride
 * @param noSteps
 * @param kerHeight
 */
__host__  void convolveffdot00_f(hipStream_t cnvlStream, cuFFdotBatch* batch, uint stack)
{
  dim3 dimGrid, dimBlock;

  cuFfdotStack* cStack = &batch->stacks[stack];

  dimBlock.x = CNV_DIMX;
  dimBlock.y = CNV_DIMY;

  if (0)
  {
    dimGrid.x = ceil(cStack->width                    / (float) ( CNV_DIMX ));
    dimGrid.y = ceil(cStack->height*batch->noSteps    / (float) ( CNV_DIMX ));

    convolveffdot00_k<<<dimGrid, dimBlock, 0, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->height, cStack->strideCmplx, batch->noSteps, cStack->noInStack, cStack->kerHeigth);
  }
  else
  {
    dimGrid.x = ceil(cStack->width / (float) ( CNV_DIMX * CNV_DIMY ));
    dimGrid.y = 1;

    convolveffdot01_k<<<dimGrid, dimBlock, 0, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->height, cStack->strideCmplx, batch->noSteps, cStack->noInStack, cStack->kerHeigth);
  }

}
