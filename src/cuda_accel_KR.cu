#include "hip/hip_runtime.h"
#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"
#include "cuda_response.h"

#define KR_DIM_X	16
#define KR_DIM_Y	16

template<typename readT, typename writeT>
__global__ void typeChangeKer(readT* read, writeT* write, size_t stride, size_t height)
{

  const int bidx = threadIdx.y * KR_DIM_X + threadIdx.x;          /// Block ID - flat index
  const int tid  = blockIdx.x  * KR_DIM_X * KR_DIM_Y + bidx;      /// Global thread ID - flat index ie column index of stack
  size_t offset;

  if ( tid < stride )  // Valid thread  .
  {
    read  += tid;
    write += tid;

    for ( int h = 0; h < height; h++ )
    {
      offset = h * stride;

      write[offset] = read[offset];
    }
  }
}

/** Create the convolution kernel for a f-∂f plane  .
 *
 *  This is "copied" from gen_z_response in respocen.c
 *
 * @param response
 * @param maxZ
 * @param fftlen
 */
template<typename genT, typename storeT>
__global__ void init_kernels(storeT* response, double zStart, double zEnd, int noZ, int width, int half_width, int rSteps)
{
  int cx, cy;							/// The x and y index of this thread in the array
  int rx = -1;							/// The x index of the value in the kernel

  // Calculate the 2D index of this thread
  cx = blockDim.x * blockIdx.x + threadIdx.x;			/// use BLOCKSIZE rather (its constant)
  cy = blockDim.y * blockIdx.y + threadIdx.y;			/// use BLOCKSIZE rather (its constant)

  if ( cy < noZ && cx < width )
  {
    // In bound

    // Calculate the z value for the row
    genT z;
    if ( noZ == 0 )
      z = zStart + (zEnd-zStart)/(genT)(noZ-1)*cy;		/// The Fourier Frequency derivative
    else
      z = zStart;

    if      ( half_width == 0  )				// Standard low accuracy half width
    {
      half_width    = cu_z_resp_halfwidth_low<genT>(z);
    }
    else if ( half_width == 1  )				// Use high accuracy kernels
    {
      half_width    = cu_z_resp_halfwidth_high<genT>(z);
    }
    else							// Only used for debug purposes
    {
      // Use the actual halfwidth value for all rows

      //int hw2       = MAX(0.6*z, 16*1);
      //half_width    = MIN( half_width, hw2 ) ;
      //half_width    = cu_z_resp_halfwidth((double) z);
      //half_width    = cu_z_resp_halfwidth((double) z);
    }

    int noResp	= half_width * rSteps;				// The number of response variables per side
    genT offset	= 0;						// The distance of the response value from 0 (negative to the leaf)

    // Calculate the kernel index for this thread (centred on zero inverted and wrapped)
    if		( cx < noResp )
    {
      // Beginning of array ( left half of responce values mirrored about zero)
      offset = -1 * cx / (genT)rSteps;
      rx = 1;
    }
    else if	(cx >= width - noResp )
    {
      // End of array ( right half of responce values mirrored about zero)
      offset = ( width - cx ) / (genT)rSteps;
      rx = 1;
    }

    // The complex response
    genT real = 0.0;
    genT imag = 0.0;

    FOLD // Calculate the response value  .
    {
      if (rx != -1)
      {
	calc_response_off<genT> (offset, z, &real, &imag);
      }
    }

    response[cy * width + cx].x = real;
    response[cy * width + cx].y = imag;
  }
}

/** Create one GPU kernel. One kernel the size of the largest plane  .
 *
 * @param kernel
 * @return
 */
int createStackKernel(cuFfdotStack* cStack)
{
  dim3 dimBlock, dimGrid;

  dimBlock.x          = KR_DIM_X;  // in my experience 16 is almost always best (half warp)
  dimBlock.y          = KR_DIM_Y;  // in my experience 16 is almost always best (half warp)

  // Set up grid
  dimGrid.x = ceil(  cStack->width     / ( float ) dimBlock.x );
  dimGrid.y = ceil ( cStack->kerHeigth / ( float ) dimBlock.y );

  int halfWidth;

  if ( cStack->flags & FLAG_KER_MAX )
  {
    // Use one halfwidth for the entire kernel
    halfWidth = cStack->harmInf->kerStart / 2.0;
  }
  else
  {
    if ( cStack->flags & FLAG_KER_HIGH )
    {
      // high accuracy
       halfWidth = 1;
    }
    else
    {
      // Standard "low" accuracy
      halfWidth = 0;
    }
  }

  FOLD // Call the CUDA kernels  .
  {
    if      ( (cStack->flags & FLAG_KER_DOUBFFT) || (cStack->flags & FLAG_DOUBLE) )
    {
      init_kernels<double, double2><<<dimGrid, dimBlock, 0, cStack->initStream>>>((double2*)cStack->kernels->d_kerData, cStack->harmInf->zStart, cStack->harmInf->zEnd, cStack->harmInf->noZ, cStack->width, halfWidth, cStack->harmInf->noResPerBin);
    }
    else if ( cStack->flags & FLAG_KER_DOUBGEN )
    {
      init_kernels<double, float2><<<dimGrid, dimBlock, 0, cStack->initStream>>>((float2*)cStack->kernels->d_kerData, cStack->harmInf->zStart, cStack->harmInf->zEnd, cStack->harmInf->noZ, cStack->width, halfWidth, cStack->harmInf->noResPerBin);
    }
    else
    {
      init_kernels<float, float2><<<dimGrid, dimBlock, 0, cStack->initStream>>>((float2*)cStack->kernels->d_kerData, cStack->harmInf->zStart, cStack->harmInf->zEnd, cStack->harmInf->noZ, cStack->width, halfWidth, cStack->harmInf->noResPerBin);
    }

    // Run message
    CUDA_SAFE_CALL(hipGetLastError(), "At kernel launch");
  }

  return 0;
}


int copyKerDoubleToFloat(cuKernel* doubleKer, cuKernel* floatKer, hipStream_t stream) //   cuFfdotStack* cStack, float* d_orrKer)
{
  dim3 dimBlock, dimGrid;

  dimBlock.x     = KR_DIM_X;  // in my experience 16 is almost always best (half warp)
  dimBlock.y     = KR_DIM_Y;  // in my experience 16 is almost always best (half warp)

  size_t width   = doubleKer->stride * 2 ; // Stride is in complex valuses

  // Set up grid
  dimGrid.x = ceil(  width / ( float ) ( dimBlock.x * dimBlock.y ) );
  dimGrid.y = 1;

  typeChangeKer<double, float><<<dimGrid, dimBlock, 0, stream>>>((double*)doubleKer->d_kerData, (float*)floatKer->d_kerData, width, doubleKer->harmInf->noZ );

  return 0;
}

void createBatchKernels(cuFFdotBatch* batch, void* buffer)
{
  cuKernel doubleKres[MAX_STACKS];
  char msg[1024];

  infoMSG(4,4,"Initialise the multiplication kernels.\n");

  // Run message
  CUDA_SAFE_CALL(hipGetLastError(), "Before creating GPU kernels");

  FOLD // Allocate temporary memory for kernel wanting double precision FFT's  .
  {
    for (int i = 0; i < MAX_STACKS; i++)
    {
      doubleKres[i].d_kerData = NULL;
    }

    if ( (batch->flags & FLAG_KER_DOUBFFT) && !(batch->flags & FLAG_DOUBLE) )
    {
      for (int i = 0; i < batch->noStacks; i++)
      {
	infoMSG(4,6,"Stack %i\n",i);

	cuFfdotStack* cStack = &batch->stacks[i];

	size_t kerSz = cStack->kernels->stride * cStack->kernels->harmInf->noZ * sizeof(double2);

	memcpy(&doubleKres[i], cStack->kernels, sizeof(cuKernel));

	CUDA_SAFE_CALL(hipMalloc((void**)&doubleKres[i].d_kerData, kerSz), "Failed to allocate temporary device memory for kernel stack."); // This is temporary double memory it will be freed at the end of this function
      }
    }
  }

  FOLD // Calculate the response values  .
  {
    infoMSG(4,5,"Calculate the response values\n");

    NV_RANGE_PUSH("Calc stack response");

    for (int i = 0; i < batch->noStacks; i++)
    {
      cuFfdotStack* cStack = &batch->stacks[i];

      // Call the CUDA kernels
      createStackKernel(cStack);
    }

    NV_RANGE_POP();
  }

  FOLD // FFT the kernels  .
  {
    infoMSG(4,5,"FFT the  response values\n");

    NV_RANGE_PUSH("FFT kernels");

    for (int i = 0; i < batch->noStacks; i++)
    {
      infoMSG(4,6,"Stack %i\n",i);

      cuFfdotStack* cStack = &batch->stacks[i];

      if ( (batch->flags & FLAG_KER_DOUBFFT) || (batch->flags & FLAG_DOUBLE) )
      {
	FOLD // Create the plan  .
	{
	  infoMSG(4,6,"Create plan\n");

	  sprintf(msg,"Plan %i",i);
	  NV_RANGE_PUSH(msg);

	  int n[]             = {cStack->width};
	  int inembed[]       = {cStack->strideCmplx* sizeof(double2)};
	  int istride         = 1;
	  int idist           = cStack->strideCmplx;
	  int onembed[]       = {cStack->strideCmplx* sizeof(double2)};
	  int ostride         = 1;
	  int odist           = cStack->strideCmplx;
	  int height          = cStack->kerHeigth;

	  // Normal plans
	  CUFFT_SAFE_CALL(hipfftPlanMany(&cStack->plnPlan,  1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, height), "Creating plan for FFT'ing the kernel.");
	  CUDA_SAFE_CALL(hipGetLastError(), "Creating FFT plans for the stacks.");

	  NV_RANGE_POP();
	}

	FOLD // Call the plan  .
	{
	  infoMSG(4,6,"Call the plan\n");

	  sprintf(msg,"Call %i",i);
	  NV_RANGE_PUSH(msg);

	  CUFFT_SAFE_CALL(hipfftSetStream(cStack->plnPlan, cStack->initStream),  "Error associating a CUFFT plan with multStream.");
	  CUFFT_SAFE_CALL(hipfftExecZ2Z(cStack->plnPlan, (hipfftDoubleComplex *)doubleKres[i].d_kerData, (hipfftDoubleComplex *) doubleKres[i].d_kerData, HIPFFT_FORWARD), "FFT'ing the kernel data. [hipfftExecC2C]");
	  CUDA_SAFE_CALL(hipGetLastError(), "FFT'ing the multiplication kernels.");

	  NV_RANGE_POP();
	}

	FOLD // Destroy the plan  .
	{
	  infoMSG(4,6,"Destroy the plan\n");

	  sprintf(msg,"Dest %i",i);
	  NV_RANGE_PUSH(msg);

	  CUFFT_SAFE_CALL(hipfftDestroy(cStack->plnPlan), "Destroying plan for complex data of stack. [hipfftDestroy]");
	  CUDA_SAFE_CALL(hipGetLastError(), "Destroying the plan.");

	  NV_RANGE_POP();
	}
      }
      else
      {
	FOLD // Create the plan  .
	{
	  infoMSG(4,6,"Create plan\n");

	  sprintf(msg,"Plan %i",i);
	  NV_RANGE_PUSH(msg);
	  size_t workSize;

	  int n[]             = {cStack->width};
	  int inembed[]       = {cStack->strideCmplx* sizeof(fcomplexcu)};
	  int istride         = 1;
	  int idist           = cStack->strideCmplx;
	  int onembed[]       = {cStack->strideCmplx* sizeof(fcomplexcu)};
	  int ostride         = 1;
	  int odist           = cStack->strideCmplx;
	  int height          = cStack->kerHeigth;

          // Normal plans
	  if (buffer)
	  {
	    // use pre allocated memory
	    CUFFT_SAFE_CALL( hipfftCreate(&cStack->plnPlan), "hipfftCreate");
	    CUFFT_SAFE_CALL( hipfftSetAutoAllocation(cStack->plnPlan, 0), "hipfftSetAutoAllocation");
	    CUFFT_SAFE_CALL( hipfftSetWorkArea(cStack->plnPlan, buffer), "hipfftSetWorkArea" ); // Assighn pre allocated memory
	    CUFFT_SAFE_CALL( hipfftMakePlanMany(cStack->plnPlan,  1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, height, &workSize), "hipfftMakePlanMany" );
	  }
	  else
	  {
	    CUFFT_SAFE_CALL(hipfftPlanMany(&cStack->plnPlan,  1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, height), "Creating plan for FFT'ing the kernel.");
	    CUDA_SAFE_CALL(hipGetLastError(), "Creating FFT plans for the stacks.");
	  }
	  
	  NV_RANGE_POP();
	}

	FOLD // Call the plan  .
	{
	  infoMSG(4,6,"Call the plan\n");

	  sprintf(msg,"Call %i",i);
	  NV_RANGE_PUSH(msg);

	  CUFFT_SAFE_CALL(hipfftSetStream(cStack->plnPlan, cStack->initStream),  "Error associating a CUFFT plan with multStream.");
	  CUFFT_SAFE_CALL(hipfftExecC2C(cStack->plnPlan, (hipfftComplex *) cStack->kernels->d_kerData, (hipfftComplex *) cStack->kernels->d_kerData, HIPFFT_FORWARD), "FFT'ing the kernel data. [hipfftExecC2C]");
	  CUDA_SAFE_CALL(hipGetLastError(), "FFT'ing the multiplication kernels.");

	  NV_RANGE_POP();
	}

	FOLD // Destroy the plan  .
	{
	  if (!buffer)
	  {
	    infoMSG(4,6,"Destroy the plan\n");

	    sprintf(msg,"Dest %i",i);
	    NV_RANGE_PUSH(msg);

	    CUFFT_SAFE_CALL(hipfftDestroy(cStack->plnPlan), "Destroying plan for complex data of stack. [hipfftDestroy]");
	    CUDA_SAFE_CALL(hipGetLastError(), "Destroying the plan.");

	    NV_RANGE_POP();
	  }
	}
      }
    }

    CUDA_SAFE_CALL(hipGetLastError(), "FFT'ing the multiplication kernels.");

    NV_RANGE_POP();
  }

  FOLD // Copy double FFT'ed data back to the float kernel  .
  {
    if ( (batch->flags & FLAG_KER_DOUBFFT) && !(batch->flags & FLAG_DOUBLE) )
    {
      for (int i = 0; i < batch->noStacks; i++)
      {
	cuFfdotStack* cStack = &batch->stacks[i];

	copyKerDoubleToFloat( &doubleKres[i], cStack->kernels, cStack->initStream );
      }
    }

    FOLD // Free temporary memory for kernel  .
    {
      if ( (batch->flags & FLAG_KER_DOUBFFT) && !(batch->flags & FLAG_DOUBLE) )
      {
	for (int i = 0; i < batch->noStacks; i++)
	{
	  cudaFreeNull( doubleKres[i].d_kerData );			// Free the temporary double data
	}
      }
    }
  }


}
