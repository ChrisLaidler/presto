#include "hip/hip_runtime.h"
#include "cuda_accel_MU.h"

/** Multiplication kernel - Multiply a stack with a kernel - multi-step - Loop ( Pln - Y - step )  .
 * Each thread loops down a column of the plane
 * Reads the input and multiplies it with the kernel and writes result to plane
 */
template<int64_t FLAGS, int noSteps>
__global__ void mult22_k(const __restrict__ fcomplexcu*  kernels, const __restrict__ fcomplexcu*  inpData, __restrict__ fcomplexcu* ffdot, const int width, const int stride, int noPlns, const int firstPlane )
{
  const int bidx = threadIdx.y * CNV_DIMX + threadIdx.x;          /// Block ID - flat index
  const int tid  = blockIdx.x  * CNV_DIMX * CNV_DIMY + bidx;      /// Global thread ID - flat index ie column index of stack

  if ( tid < width )  // Valid thread  .
  {
    int idx;                                      /// flat index of output plane
    int pHeight = 0;                              /// Height of previous data in the stack
    fcomplexcu ker;                               /// kernel data

    FOLD  // Stride, kernel, input data & output data  .
    {
      kernels += tid;
      ffdot   += tid;
      inpData += tid;
    }

    __restrict__ fcomplexcu inpDat[noSteps];                  // Set of input data for this thread/column

    for (int pln = 0; pln < noPlns; pln++)                    // Loop through the planes  .
    {
      const int plnStrd       = pln*stride*noSteps;
      const int plnHeight     = HEIGHT_HARM[firstPlane + pln];
      const int kerYOffset    = (HEIGHT_HARM[firstPlane] - plnHeight)/2;
      const int ns2           = plnHeight * stride;

      FOLD // Read input data for this plane  .
      {
        for (int step = 0; step < noSteps; step++)
        {
          fcomplexcu inp      = inpData[ (int)(plnStrd + step*stride) ];
          inp.r               /= (float) width;
          inp.i               /= (float) width;
          inpDat[step]        = inp;
        }
      }

      short   lDepth  = ceilf(plnHeight/(float)gridDim.y);
      short   y0      = lDepth*blockIdx.y;
      short   y1      = MIN(y0+lDepth, plnHeight);

      for (int planeY = y0; planeY < y1; planeY++)      // Loop over the individual plane  .
      {
        FOLD // Read the kernel value  .
        {
          ker   = kernels[(kerYOffset+planeY)*stride];
        }

        int off1;

        FOLD // Calculate partial offset  .
        {
          if      ( FLAGS & FLAG_ITLV_ROW )
          {
            off1  = pHeight + planeY*noSteps*stride;
          }
          else
          {
            off1  = pHeight + planeY*stride;
          }
        }

        for ( int step = 0; step < noSteps; ++step )          // Loop over steps .
        {
          FOLD // Calculate indices  .
          {
            if      ( FLAGS & FLAG_ITLV_ROW )
            {
              idx  = off1 + step * stride;
            }
            else
            {
              idx  = off1 + step * ns2;
            }
          }

          FOLD // Multiply  .
          {
            fcomplexcu ipd = inpDat[step];
            fcomplexcu out;

#if CORRECT_MULT
              // This is the "correct" version
              out.r = (ipd.r * ker.r - ipd.i * ker.i);
              out.i = (ipd.r * ker.i + ipd.i * ker.r);
#else
              // This is the version accelsearch uses, ( added for comparison )
              out.r = (ipd.r * ker.r + ipd.i * ker.i);
              out.i = (ipd.i * ker.r - ipd.r * ker.i);
#endif
            ffdot[idx] = out;
          }
        }
      }

      pHeight += plnHeight * noSteps * stride;
    }
  }
}

template<int64_t FLAGS>
__host__  void mult22_s(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t multStream, cuFFdotBatch* batch, cuFfdotStack* cStack)
{
  int offset            = cStack->startIdx;

  switch (batch->noSteps)
  {
    case 1:
    {
      mult22_k<FLAGS,1><<<dimGrid, dimBlock, i1, multStream>>>((fcomplexcu*)cStack->d_kerData , cStack->d_iData, (fcomplexcu*)cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 2:
    {
      mult22_k<FLAGS,2><<<dimGrid, dimBlock, i1, multStream>>>((fcomplexcu*)cStack->d_kerData , cStack->d_iData, (fcomplexcu*)cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 3:
    {
      mult22_k<FLAGS,3><<<dimGrid, dimBlock, i1, multStream>>>((fcomplexcu*)cStack->d_kerData , cStack->d_iData, (fcomplexcu*)cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 4:
    {
      mult22_k<FLAGS,4><<<dimGrid, dimBlock, i1, multStream>>>((fcomplexcu*)cStack->d_kerData , cStack->d_iData, (fcomplexcu*)cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 5:
    {
      mult22_k<FLAGS,5><<<dimGrid, dimBlock, i1, multStream>>>((fcomplexcu*)cStack->d_kerData , cStack->d_iData, (fcomplexcu*)cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 6:
    {
      mult22_k<FLAGS,6><<<dimGrid, dimBlock, i1, multStream>>>((fcomplexcu*)cStack->d_kerData , cStack->d_iData, (fcomplexcu*)cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 7:
    {
      mult22_k<FLAGS,7><<<dimGrid, dimBlock, i1, multStream>>>((fcomplexcu*)cStack->d_kerData , cStack->d_iData, (fcomplexcu*)cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 8:
    {
      mult22_k<FLAGS,8><<<dimGrid, dimBlock, i1, multStream>>>((fcomplexcu*)cStack->d_kerData , cStack->d_iData, (fcomplexcu*)cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    default:
    {
      fprintf(stderr, "ERROR: mult22 has not been templated for %i steps\n", batch->noSteps);
      exit(EXIT_FAILURE);
    }
  }
}

__host__  void mult22(hipStream_t multStream, cuFFdotBatch* batch, cuFfdotStack* cStack)
{
  dim3 dimGrid, dimBlock;

  dimBlock.x = CNV_DIMX;
  dimBlock.y = CNV_DIMY;

  dimGrid.x = ceil(cStack->width / (float) ( CNV_DIMX * CNV_DIMY ));
  dimGrid.y = cStack->mulSlices;

  if      ( batch->flags & FLAG_ITLV_ROW )
    mult22_s<FLAG_ITLV_ROW>(dimGrid, dimBlock, 0, multStream, batch, cStack);
  else
    mult22_s<0>(dimGrid, dimBlock, 0, multStream, batch, cStack);
}
