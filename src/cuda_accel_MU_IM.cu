#include "hip/hip_runtime.h"
#include "cuda_accel_MU.h"

/** Kernel to copy powers from complex plane to in-mem plane  .
 *
 * One thread per column
 *
 */
template<typename T>
__global__ void cpyPowers_ker( T* dst, size_t  dpitch, T*  src, size_t  spitch, size_t  width, size_t  height)
{
  int ix = blockIdx.x * CPY_WIDTH + threadIdx.x ;

  for ( int iy = 0 ; iy < height; iy++)
  {
    if ( ix < width && iy < height)
    {
      dst[iy*dpitch + ix] = src[iy*spitch +ix];
    }
  }
}

/** Kernel to copy powers from complex plane to in-mem plane  .
 *
 * One thread per column
 */
template<typename T>
__global__ void cpyCmplx_ker( T* dst, size_t  dpitch, fcomplexcu* src, size_t  spitch, size_t  width, size_t  height)
{
  int ix = blockIdx.x * CPY_WIDTH + threadIdx.x ;

  const int buffLen = 4;

  float buff[buffLen];

  if ( ix < width )
  {
    int iy;

    FOLD // All iterations with no height check
    {
      for ( iy = 0 ; iy < height - buffLen ; iy+=buffLen)
      {
	for ( int by = 0 ; by < buffLen; by++)
	{
	  int gy = iy + by;

	  buff[by]          = getPowerAsFloat(src, gy*spitch + ix);
	}

	for ( int by = 0 ; by < buffLen; by++)
	{
	  int gy = iy + by;

	  set(dst, gy*dpitch + ix, buff[by]);
	}
      }
    }

    FOLD // One last iteration with height checks
    {
      for ( int by = 0 ; by < buffLen; by++)
      {
	int gy = iy + by;

	if ( gy < height)
	{
	  buff[by]          = getPowerAsFloat(src, gy*spitch + ix);
	}
      }

      for ( int by = 0 ; by < buffLen; by++)
      {
	int gy = iy + by;

	if ( gy < height)
	{
	  set(dst, gy*dpitch + ix, buff[by]);
	}
      }
    }
  }
}

/** Function to call the kernel to copy powers from powers plane to in-mem plane  .
 */
template<typename T>
void cpyPowers( T* dst, size_t  dpitch, T* src, size_t  spitch, size_t  width, size_t  height, hipStream_t  stream)
{
  dim3 dimBlock, dimGrid;

  dimBlock.x  = CPY_WIDTH;
  dimBlock.y  = 1 ;

  float ww    = width  / (float)dimBlock.x ;

  dimGrid.x   = ceil(ww);
  dimGrid.y   = 1 ;

  cpyPowers_ker<T><<<dimGrid,  dimBlock, 0, stream >>>(dst, dpitch, src, spitch, width, height);
}

/** Function to call the kernel to copy powers from powers plane to in-mem plane  .
 */
template<typename T>
void cpyCmplx( T* dst, size_t  dpitch, fcomplexcu* src, size_t  spitch, size_t  width, size_t  height, hipStream_t  stream)
{
  dim3 dimBlock, dimGrid;

  dimBlock.x  = CPY_WIDTH;
  dimBlock.y  = 1 ;

  float ww    = width  / (float)dimBlock.x ;

  dimGrid.x   = ceil(ww);
  dimGrid.y   = 1 ;

  cpyCmplx_ker<T><<<dimGrid,  dimBlock, 0, stream >>>(dst, dpitch, src, spitch, width, height);
}

/** Copy results of iFFT from powers plane to the inmem plane using 2D async memory copy
 *
 * This is done using one appropriately strided 2d memory copy for each step of a stack
 *
 */
template<typename Tin, typename Tout>
void copyIFFTtoPln( cuFFdotBatch* batch, cuFfdotStack* cStack)
{
  Tout*   dst;
  Tin*    src;
  size_t  dpitch;
  size_t  spitch;
  size_t  width;
  size_t  height;

  int inSz  = 1;
  int outSz = 1;

  inSz  = sizeof(Tin);
  outSz = sizeof(Tout);

  dpitch  = batch->cuSrch->inmemStride * outSz;
  height  = cStack->height;
  spitch  = cStack->stridePower * inSz;

  // Error check
  if (cStack->noInStack > 1 )
  {
    fprintf(stderr,"ERROR: %s cannot handle stacks with more than one plane.\n", __FUNCTION__);
    exit(EXIT_FAILURE);
  }

  if ( batch->flags & FLAG_CUFFT_CB_INMEM )
  {
    // Copying was done by the callback directly
    infoMSG(5,5,"break - Copy done by callback");
    return;
  }

  for ( int step = 0; step < batch->noSteps; step++ )
  {
    rVals* rVal	= &(*batch->rAraays)[batch->rActive][step][0];

    if ( rVal->numrs )
    {
      width	= rVal->numrs;					// Width is dependent on the number of good values
      MINN( width, batch->cuSrch->inmemStride - rVal->step * batch->accelLen -1 );	// Clamp to plane

      // Check
      size_t  end = rVal->step * batch->accelLen + width ;
      if ( end >= batch->cuSrch->inmemStride )
      {
	fprintf(stderr,"ERROR: Data exceeds plane.\n");
	exit(EXIT_FAILURE);
      }

      width	*= outSz;
      dst	= ((Tout*)batch->cuSrch->d_planeFull)    + rVal->step * batch->accelLen;

      if      ( batch->flags & FLAG_ITLV_ROW )
      {
	src	= ((Tin*)cStack->d_planePowr)  + cStack->stridePower*step + cStack->harmInf->kerStart;
	spitch	= cStack->stridePower*batch->noSteps*inSz;
      }
#ifdef WITH_ITLV_PLN
      else
      {
	src	= ((Tin*)cStack->d_planePowr)  + cStack->stridePower*height*step + cStack->harmInf->kerStart;
      }
#else
      else
      {
	fprintf(stderr, "ERROR: functionality disabled in %s.\n", __FUNCTION__);
	exit(EXIT_FAILURE);
      }
#endif

      CUDA_SAFE_CALL(hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToDevice, batch->srchStream ), "Calling hipMemcpy2DAsync after IFFT.");
    }
  }
}

/** Copy results of the iFFT from powers plane to the inmem plane using a kernel  .
 *
 */
void cmplxToPln( cuFFdotBatch* batch, cuFfdotStack* cStack)
{
  fcomplexcu*   src;

  size_t        dpitch;
  size_t        spitch;
  size_t        width;
  size_t        height;

  dpitch  = batch->cuSrch->inmemStride;
  width   = batch->accelLen;
  height  = cStack->height;
  spitch  = cStack->strideCmplx;

  // Error check
  if (cStack->noInStack > 1 )
  {
    fprintf(stderr,"ERROR: %s cannot handle stacks with more than one plane.\n", __FUNCTION__);
    exit(EXIT_FAILURE);
  }

  if ( batch->flags & FLAG_CUFFT_CB_INMEM )
  {
    // Copying was done by the callback directly
    return;
  }

  for ( int step = 0; step < batch->noSteps; step++ )
  {
    rVals* rVal = &(*batch->rAraays)[batch->rActive][step][0];

    if ( rVal->numrs ) // Valid step
    {
      FOLD // Calculate striding info
      {
	// Source data location
	if ( batch->flags & FLAG_ITLV_ROW )
	{
	  src     = ((fcomplexcu*)cStack->d_planePowr)  + cStack->strideCmplx*step + cStack->harmInf->kerStart;
	  spitch  = cStack->strideCmplx*batch->noSteps;
	}
#ifdef WITH_ITLV_PLN
	else
	{
	  src     = ((fcomplexcu*)cStack->d_planePowr)  + cStack->strideCmplx*height*step + cStack->harmInf->kerStart;
	}
#else
	else
	{
	  fprintf(stderr, "ERROR: functionality disabled in %s.\n", __FUNCTION__);
	  exit(EXIT_FAILURE);
	}
#endif
      }

      if ( batch->flags & FLAG_POW_HALF )
      {
#ifdef	WITH_HALF_RESCISION_POWERS
#if	CUDA_VERSION >= 7050
	// Each Step has its own start location in the inmem plane
	half *dst = ((half*)batch->cuSrch->d_planeFull)        + rVal->step * batch->accelLen;

	// Call kernel
	cpyCmplx<half>(dst, dpitch, src, spitch,  width,  height, batch->srchStream );
#else	// CUDA_VERSION
	fprintf(stderr,"ERROR: Half precision can only be used with CUDA 7.5 or later!\n");
	exit(EXIT_FAILURE);
#endif	// CUDA_VERSION
#else	// WITH_HALF_RESCISION_POWERS
	EXIT_DIRECTIVE("WITH_HALF_RESCISION_POWERS");
#endif	// WITH_HALF_RESCISION_POWERS
      }
      else
      {
#ifdef	WITH_SINGLE_RESCISION_POWERS
	// Each Step has its own start location in the inmem plane
	float *dst  = ((float*)batch->cuSrch->d_planeFull)        + rVal->step * batch->accelLen;

	// Call kernel
	cpyCmplx<float>(dst, dpitch, src, spitch,  width,  height, batch->srchStream );

#else	// WITH_SINGLE_RESCISION_POWERS
	EXIT_DIRECTIVE("WITH_SINGLE_RESCISION_POWERS");
#endif	// WITH_SINGLE_RESCISION_POWERS
      }
    }
  }
}

/** Copy the complex plane to the in-memory plane  .
 *
 */
void copyToInMemPln(cuFFdotBatch* batch)
{
  PROF // Profiling  .
  {
    if ( (batch->flags & FLAG_PROF) )
    {
      if ( (*batch->rAraays)[batch->rActive+1][0][0].numrs )
      {
	infoMSG(5,5,"Time previous components");

	for (int stack = 0; stack < batch->noStacks; stack++)
	{
	  cuFfdotStack* cStack = &batch->stacks[stack];
	  timeEvents( cStack->ifftMemInit, cStack->ifftMemComp, &batch->compTime[NO_STKS*COMP_GEN_D2D + stack ],  "Copy to full plane");
	}
      }
    }
  }

  if ( (*batch->rAraays)[batch->rActive][0][0].numrs )
  {
    if ( batch->flags & FLAG_SS_INMEM )
    {
      infoMSG(2,2,"Copy powers to in-mem plane - Iteration %3i.", (*batch->rAraays)[batch->rActive][0][0].iteration);

      cuFfdotStack* cStack = batch->stacks;

      PROF // Profiling  .
      {
	NV_RANGE_PUSH("CPY2IM");
      }

      if ( batch->flags & FLAG_CUFFT_CB_INMEM )
      {
	// Copying was done by the callback directly
	return;
      }

      // Error check
      if (batch->noStacks > 1 )
      {
	fprintf(stderr,"ERROR: %s cannot handle a family with more than one plane.\n", __FUNCTION__);
	exit(EXIT_FAILURE);
      }

      FOLD // Copy back data  .
      {
	FOLD // Synchronisation  .
	{
	  infoMSG(5,5,"Synchronise stream %s on %s.\n", "srchStream", "ifftComp");

	  CUDA_SAFE_CALL(hipStreamWaitEvent(batch->srchStream, cStack->ifftComp,    0), "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plane so search must be compete
	}

	PROF // Profiling  .
	{
	  if ( batch->flags & FLAG_PROF )
	  {
	    infoMSG(5,5,"Event %s in %s.\n", "ifftMemInit", "srchStream");

	    CUDA_SAFE_CALL(hipEventRecord(cStack->ifftMemInit, batch->srchStream),"Recording event: ifftMemInit");
	  }
	}

	FOLD // Copy memory on the device  .
	{
	  if ( batch->flags & FLAG_CUFFT_CB_POW )
	  {
	    infoMSG(4,4,"2D async D2D memory copy");

	    // Copy memory using a 2D async memory copy
	    if ( batch->flags & FLAG_POW_HALF )
	    {
#ifdef	WITH_HALF_RESCISION_POWERS
#if 	CUDA_VERSION >= 7050
	      copyIFFTtoPln<half,half>( batch, cStack );
#else
	      fprintf(stderr,"ERROR: Half precision can only be used with CUDA 7.5 or later!\n");
	      exit(EXIT_FAILURE);
#endif
#else	// WITH_HALF_RESCISION_POWERS
	      EXIT_DIRECTIVE("WITH_HALF_RESCISION_POWERS");
#endif	// WITH_HALF_RESCISION_POWERS
	    }
	    else
	    {
#ifdef	WITH_SINGLE_RESCISION_POWERS
	      copyIFFTtoPln<float, float>( batch, cStack );
#else	// WITH_SINGLE_RESCISION_POWERS
	      EXIT_DIRECTIVE("WITH_SINGLE_RESCISION_POWERS");
#endif	// WITH_SINGLE_RESCISION_POWERS
	    }
	  }
	  else
	  {
	    infoMSG(4,4,"Kernel memory copy\n");

	    // Use kernel to copy powers from powers plane to the inmem plane
	    cmplxToPln( batch, cStack );
	  }

	  CUDA_SAFE_CALL(hipGetLastError(), "At IFFT - copyToInMemPln");
	}

	FOLD // Synchronisation  .
	{
	  infoMSG(5,5,"Event %s in %s.\n", "ifftMemComp", "srchStream");

	  CUDA_SAFE_CALL(hipEventRecord(cStack->ifftMemComp, batch->srchStream),"Recording event: ifftMemComp");
	}
      }

      PROF // Profiling  .
      {
	NV_RANGE_POP(); // CPY2IM
      }
    }
  }
}
