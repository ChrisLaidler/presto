#include "hip/hip_runtime.h"
#include "cuda_accel_CV.h"

/** Convolution kernel - Convolve a stack with a kernel - multi-step - Loop ( Y - Pln - step )  .
 * Each thread loops down a column of the plain
 * Reads the input and convolves it with the kernel and writes result to plain
 */
template<int FLAGS, int noSteps, int noPlns>
__global__ void convolveffdot42_k(const __restrict__ fcomplexcu* kernels, const __restrict__ fcomplexcu* inpData, __restrict__ fcomplexcu* ffdot, const int width, const int stride, const int firstPlain )
{
  const int bidx = threadIdx.y * CNV_DIMX + threadIdx.x;          /// Block ID - flat index
  const int tid  = blockIdx.x  * CNV_DIMX * CNV_DIMY + bidx;      /// Global thread ID - flat index ie column index of stack

  if ( tid < width )  // Valid thread  .
  {

    FOLD  // Stride, kernel, input data & output data  .
    {
      kernels += tid;
      ffdot   += tid;
      inpData += tid;
    }

    const int kerHeight = HEIGHT_FAM_ORDER[firstPlain];       // The size of the kernel

    /*
    fcomplexcu inpDat[noSteps][noPlns];                       // Set of input data for this thread/column
    FOLD // Read all input data  .
    {
      for (int step = 0; step < noSteps; step++)
      {
        for (int pln = 0; pln < noPlns; pln++)                // Loop through the plains  .
        {
          fcomplexcu inp        = inpData[ (int)(pln*noSteps*stride + step*stride) ];
          inp.r                 /= (float) width ;
          inp.i                 /= (float) width ;
          inpDat[step][pln]     = inp ;
        }
      }
    }
    */

    fcomplexcu inpDat[noPlns][noSteps];                       // Set of input data for this thread/column
    FOLD // Read all input data  .
    {
      for (int step = 0; step < noSteps; step++)
      {
        for (int pln = 0; pln < noPlns; pln++)                // Loop through the plains  .
        {
          fcomplexcu ipd        = inpData[ (int)(pln*noSteps*stride + step*stride) ];
          ipd.r                 /= (float) width;
          ipd.i                 /= (float) width;
          inpDat[pln][step]     = ipd;
        }
      }
    }

    for (int y = 0; y < kerHeight; y++)                       // Loop through the kernel .
    {
      fcomplexcu ker;                                         // kernel data
      FOLD // Read the kernel value  .
      {
        ker   = kernels[y*stride];
      }

      int pHeight = 0;                                        // Height of previous data in the stack

      for (int pln = 0; pln < noPlns; pln++)                  // Loop through the plains  .
      {
        const int plnHeight     = HEIGHT_FAM_ORDER[firstPlain + pln];
        const int kerYOffset    = (kerHeight - plnHeight)/2;
        const int plainY        = y - kerYOffset;
        const int ns2           = plnHeight * stride;

        if( plainY >= 0 && plainY < plnHeight )
        {
          int off1;
          FOLD // Calculate partial offset  .
          {
            if      ( FLAGS & FLAG_ITLV_ROW )
            {
              off1  = pHeight + plainY*noSteps*stride;
            }
            else if ( FLAGS & FLAG_ITLV_PLN )
            {
              off1  = pHeight + plainY*stride;
            }
          }

          for ( int step = 0; step < noSteps; ++step )        // Loop over steps .
          {
            int idx;
            FOLD // Calculate indices  .
            {
              if      ( FLAGS & FLAG_ITLV_ROW )
              {
                idx  = off1 + step * stride;
              }
              else if ( FLAGS & FLAG_ITLV_PLN )
              {
                idx  = off1 + step * ns2;
              }
            }

            FOLD // Convolve  .
            {
              //ffdot[idx].r = (inpDat[step][pln].r * ker.r + inpDat[step][pln].i * ker.i);
              //ffdot[idx].i = (inpDat[step][pln].i * ker.r - inpDat[step][pln].r * ker.i);

              //ffdot[idx].r = (inpDat[step].r * ker.r + inpDat[step].i * ker.i);
              //ffdot[idx].i = (inpDat[step].i * ker.r - inpDat[step].r * ker.i);

              //fcomplexcu inp = sInputPtr[(pln*noSteps + step)*CNV_DIMX * CNV_DIMY];
              //ffdot[idx].r = (inp.r * ker.r + inp.i * ker.i);
              //ffdot[idx].i = (inp.i * ker.r - inp.r * ker.i);

              fcomplexcu ipd = inpDat[pln][step];
              fcomplexcu vv;
              vv.r = (ipd.r * ker.r + ipd.i * ker.i);
              vv.i = (ipd.i * ker.r - ipd.r * ker.i);
              ffdot[idx] = vv;
            }
          }
        }
        pHeight += plnHeight * noSteps * stride;
      }
    }
  }
}

template<int FLAGS, int noSteps>
__host__  void convolveffdot42_p(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t cnvlStream, cuFFdotBatch* batch, uint stack)
{
  cuFfdotStack* cStack  = &batch->stacks[stack];
  int offset            = cStack->startIdx;

  switch (cStack->noInStack)
  {
    case 1:
    {
      convolveffdot42_k<FLAGS,noSteps,1><<<dimGrid, dimBlock, i1, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->strideCmplx, offset);
      break;
    }
    case 2:
    {
      convolveffdot42_k<FLAGS,noSteps,2><<<dimGrid, dimBlock, i1, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->strideCmplx, offset);
      break;
    }
    case 3:
    {
      convolveffdot42_k<FLAGS,noSteps,3><<<dimGrid, dimBlock, i1, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->strideCmplx, offset);
      break;
    }
    case 4:
    {
      convolveffdot42_k<FLAGS,noSteps,4><<<dimGrid, dimBlock, i1, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->strideCmplx, offset);
      break;
    }
    case 5:
    {
      convolveffdot42_k<FLAGS,noSteps,5><<<dimGrid, dimBlock, i1, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->strideCmplx, offset);
      break;
    }
    case 6:
    {
      convolveffdot42_k<FLAGS,noSteps,6><<<dimGrid, dimBlock, i1, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->strideCmplx, offset);
      break;
    }
    case 7:
    {
      convolveffdot42_k<FLAGS,noSteps,7><<<dimGrid, dimBlock, i1, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->strideCmplx, offset);
      break;
    }
    case 8:
    {
      convolveffdot42_k<FLAGS,noSteps,8><<<dimGrid, dimBlock, i1, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->strideCmplx, offset);
      break;
    }
    case 9:
    {
      convolveffdot42_k<FLAGS,noSteps,9><<<dimGrid, dimBlock, i1, cnvlStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_plainData, cStack->width, cStack->strideCmplx, offset);
      break;
    }
    default:
    {
      fprintf(stderr, "ERROR: convolveffdot42 has not been templated for %i plains in a stack.\n",cStack->noInStack);
      exit(EXIT_FAILURE);
    }
  }
}

template<int FLAGS>
__host__  void convolveffdot42_s(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t cnvlStream, cuFFdotBatch* batch, uint stack)
{

  switch (batch->noSteps)
  {
    case 1:
    {
      convolveffdot42_p<FLAGS,1>(dimGrid, dimBlock, i1, cnvlStream, batch, stack);
      break;
    }
    case 2:
    {
      convolveffdot42_p<FLAGS,2>(dimGrid, dimBlock, i1, cnvlStream, batch, stack);
      break;
    }
    case 3:
    {
      convolveffdot42_p<FLAGS,3>(dimGrid, dimBlock, i1, cnvlStream, batch, stack);
      break;
    }
    case 4:
    {
      convolveffdot42_p<FLAGS,4>(dimGrid, dimBlock, i1, cnvlStream, batch, stack);
      break;
    }
    case 5:
    {
      convolveffdot42_p<FLAGS,5>(dimGrid, dimBlock, i1, cnvlStream, batch, stack);
      break;
    }
    case 6:
    {
      convolveffdot42_p<FLAGS,6>(dimGrid, dimBlock, i1, cnvlStream, batch, stack);
      break;
    }
    case 7:
    {
      convolveffdot42_p<FLAGS,7>(dimGrid, dimBlock, i1, cnvlStream, batch, stack);
      break;
    }
    case 8:
    {
      convolveffdot42_p<FLAGS,8>(dimGrid, dimBlock, i1, cnvlStream, batch, stack);
      break;
    }
    default:
    {
      fprintf(stderr, "ERROR: convolveffdot42 has not been templated for %lu steps\n", batch->noSteps);
      exit(EXIT_FAILURE);
    }
  }
}

__host__  void convolveffdot42_f(hipStream_t cnvlStream, cuFFdotBatch* batch, uint stack)
{
  dim3 dimGrid, dimBlock;

  cuFfdotStack* cStack = &batch->stacks[stack];

  dimBlock.x = CNV_DIMX;
  dimBlock.y = CNV_DIMY;

  dimGrid.x = ceil(cStack->width / (float) ( CNV_DIMX * CNV_DIMY ));
  dimGrid.y = 1;

  if      ( batch->flag & FLAG_ITLV_ROW )
    convolveffdot42_s<FLAG_ITLV_ROW>(dimGrid, dimBlock, 0, cnvlStream, batch, stack);
  else if ( batch->flag & FLAG_ITLV_PLN )
    convolveffdot42_s<FLAG_ITLV_PLN>(dimGrid, dimBlock, 0, cnvlStream, batch, stack);
  else
  {
    fprintf(stderr, "ERROR: convolveffdot42 has not been templated for layout.\n");
    exit(EXIT_FAILURE);
  }
}
