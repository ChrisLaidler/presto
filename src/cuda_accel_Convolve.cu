#include "hip/hip_runtime.h"
#include "cuda_accel_CV.h"

__device__ hipfftCallbackLoadC d_loadCallbackPtr     = CB_ConvolveInput;
__device__ hipfftCallbackStoreC d_storeCallbackPtr   = CB_PowerOut;

__device__ __constant__ int           HEIGHT_FAM_ORDER[MAX_HARM_NO];        ///< Plain height in stage order
__device__ __constant__ int           STRIDE_FAM_ORDER[MAX_HARM_NO];        ///< Plain stride in stage order
__device__ __constant__ fcomplexcu*   KERNEL_FAM_ORDER[MAX_HARM_NO];        ///< Kernel pointer in stage order

__device__ hipfftComplex CB_ConvolveInput( void *dataIn, size_t offset, void *callerInfo, void *sharedPtr)
{

  fftCnvlvInfo *inf = (fftCnvlvInfo*)callerInfo;

  const int strd = inf->stride * inf->noSteps;

  size_t grow = offset / strd;
  size_t col  = offset % inf->stride;
  size_t step = ( offset % strd ) / inf->stride ;
  size_t pln  = 0;

  for ( int i = 0; i < inf->noPlains; i++ )
  {
    if ( grow >= inf->top[i] )
    {
      pln = i;
    }
  }

  size_t row  = grow - inf->top[pln];

  hipfftComplex ker = ((hipfftComplex*)inf->d_kernel[pln])[row*inf->stride + col ];
  hipfftComplex inp = ((hipfftComplex*)inf->d_idata[pln])[step*inf->stride + col ];

  hipfftComplex out;
  out.x = ( inp.x * ker.x + inp.y * ker.y ) / inf->width;
  out.y = ( inp.y * ker.x - inp.x * ker.y ) / inf->width;

  return out;

  //return ((hipfftComplex*)dataIn)[offset];
}

__device__ void CB_PowerOut( void *dataIn, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr)
{
  float power = element.x*element.x + element.y*element.y ;
  ((float*)callerInfo)[offset] = power;
}

void copyCUFFT_LD_CB(cuFFdotBatch* batch)
{
  CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_ldCallbackPtr,  HIP_SYMBOL(d_loadCallbackPtr),  sizeof(hipfftCallbackLoadC)),   "");
  CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_stCallbackPtr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(hipfftCallbackStoreC)),  "");
}

int setConstVals_Fam_Order( cuFFdotBatch* batch )
{
  FOLD // Set other constant values
  {
    void *dcoeffs;

    int           height[MAX_HARM_NO];
    int           stride[MAX_HARM_NO];
    fcomplexcu*   kerPnt[MAX_HARM_NO];
    for (int i = 0; i < batch->noHarms; i++)
    {
      height[i] = batch->hInfos[i].height;
      stride[i] = batch->hInfos[i].width;
      kerPnt[i] = batch->kernels[i].d_kerData;

      if (batch->hInfos[i].width != batch->hInfos[i].inpStride )
      {
        fprintf(stderr,"ERROR: Width is not the same as stride, using width this may case errors in the convolution.\n");
      }
    }

    for (int i = batch->noHarms; i < MAX_HARM_NO; i++) // Zero the rest
    {
      height[i] = 0;
      stride[i] = 0;
    }

    hipGetSymbolAddress((void **)&dcoeffs, HEIGHT_FAM_ORDER);
    CUDA_SAFE_CALL(hipMemcpy(dcoeffs, &height, MAX_HARM_NO * sizeof(int), hipMemcpyHostToDevice),      "Copying stages to device");

    hipGetSymbolAddress((void **)&dcoeffs, STRIDE_FAM_ORDER);
    CUDA_SAFE_CALL(hipMemcpy(dcoeffs, &stride, MAX_HARM_NO * sizeof(int), hipMemcpyHostToDevice),      "Copying stages to device");

    hipGetSymbolAddress((void **)&dcoeffs, KERNEL_FAM_ORDER);
    CUDA_SAFE_CALL(hipMemcpy(dcoeffs, &kerPnt, MAX_HARM_NO * sizeof(fcomplexcu*), hipMemcpyHostToDevice),      "Copying stages to device");
  }

  CUDA_SAFE_CALL(hipGetLastError(), "Error preparing the constant memory values for the convolutions.");

  return 1;
}

/** Convolve and inverse FFT the complex f-∂f plain using FFT callback
 * @param plains
 */
void convolveBatchCUFFT(cuFFdotBatch* batch )
{
  // Convolve this entire stack in one block
  for (int ss = 0; ss< batch->noStacks; ss++)
  {
    cuFfdotStack* cStack = &batch->stacks[ss];

    // Synchronisation
    CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->fftPStream, cStack->prepComp, 0),    "Waiting for GPU to be ready to copy data to device.");  // Need input data
    CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->fftPStream, batch->searchComp, 0),  "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete

    // Do the FFT
#pragma omp critical
    FOLD
    {
      if ( batch->flag & FLAG_CUFFTCB_OUT )
      {
        //CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&h_storeCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&cStack->d_plainPowers ),"");
      }

      CUFFT_SAFE_CALL(hipfftSetStream(cStack->plnPlan, cStack->fftPStream),  "Error associating a CUFFT plan with cnvlStream.");
      CUFFT_SAFE_CALL(hipfftExecC2C(cStack->plnPlan, (hipfftComplex *) cStack->d_plainData, (hipfftComplex *) cStack->d_plainData, HIPFFT_BACKWARD),"Error executing CUFFT plan.");
    }

    // Synchronise
    hipEventRecord(cStack->plnComp, cStack->fftPStream);
  }
}

/** Convolve and inverse FFT the complex f-∂f plain
 * This assumes the input data is ready and on the device
 * This creates a complex f-∂f plain
 */
void convolveBatch(cuFFdotBatch* batch)
{
  //hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  if ( batch->haveInput )
  {
    nvtxRangePush("Convolve & FFT");
#ifdef STPMSG
    printf("\tConvolve & FFT\n");
#endif

    dim3 dimBlock, dimGrid;

    if ( batch->flag & FLAG_CUFFTCB_INP )  	// Do the convolution using a CUFFT callback  .
    {
#ifdef STPMSG
    printf("\t\tConvolve with CUFFT\n");
#endif
      convolveBatchCUFFT( batch );
    }
    else                                    // Do the convolution and FFT separately  .
    {
      FOLD // Convolve  .
      {
#ifdef STPMSG
    printf("\t\tConvolve\n");
#endif

        dimBlock.x = CNV_DIMX;   // in my experience 16 is almost always best (half warp)
        dimBlock.y = CNV_DIMY;   // in my experience 16 is almost always best (half warp)

        // In my testing I found convolving each plain separately works fastest so it is the "default"
        if      ( batch->flag & FLAG_CNV_FAM ) // Do the convolutions one family at a time  .
        {
          dimGrid.x = ceil(batch->hInfos[0].width / (float) ( CNV_DIMX * CNV_DIMY ));
          dimGrid.y = 1;

          FOLD // Synchronisation  .
          {
            for (int ss = 0; ss < batch->noStacks; ss++) // Synchronise input data preparation for all stacks
            {
              cuFfdotStack* cStack = &batch->stacks[ss];
              CUDA_SAFE_CALL(hipStreamWaitEvent(batch->convStream, cStack->prepComp,0),     "Waiting for GPU to be ready to copy data to device.");    // Need input data
            }

            CUDA_SAFE_CALL(hipStreamWaitEvent(batch->convStream, batch->searchComp, 0),      "Waiting for GPU to be ready to copy data to device.");   // This will overwrite the f-fdot plain so search must be compete
          }

          FOLD // Timing event  .
          {
#ifdef TIMING
          CUDA_SAFE_CALL(hipEventRecord(batch->convInit, batch->convStream),"Recording event: convInit");
#endif
          }

          FOLD // call kernel  .
          {
            convolveffdot5_f(dimGrid, dimBlock, 0, batch->convStream, batch);

            // Run message
            CUDA_SAFE_CALL(hipGetLastError(), "Error at kernel launch");
          }

          FOLD // Synchronisation  .
          {
            CUDA_SAFE_CALL(hipEventRecord(batch->convComp, batch->convStream),"Recording event: convComp");
          }

        }
        else if ( batch->flag & FLAG_CNV_STK ) // Do the convolutions one stack  at a time  .
        {
#ifdef SYNCHRONOUS
      cuFfdotStack* pStack = NULL;
#endif

          // Convolve this entire stack in one block
          for (int ss = 0; ss < batch->noStacks; ss++)
          {
            cuFfdotStack* cStack = &batch->stacks[ss];

            iHarmList hlist;
            cHarmList plainsDat;
            cHarmList kerDat;
            iHarmList zUp;
            iHarmList zDn;

            for (int i = 0; i < cStack->noInStack; i++)     // Loop over plains to determine where they start
            {
              hlist.val[i]      =  cStack->harmInf[i].height;
              plainsDat.val[i]  =  cStack->plains[i].d_plainData;
              kerDat.val[i]     =  cStack->kernels[i].d_kerData;

              zUp.val[i]        =  cStack->zUp[i];
              zDn.val[i]        =  cStack->zDn[i];
            }

            dimGrid.x = ceil(cStack->width / (float) ( CNV_DIMX * CNV_DIMY ));
            dimGrid.y = 1;

            FOLD // Synchronisation  .
            {
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->cnvlStream, cStack->prepComp,0),    "Waiting for GPU to be ready to copy data to device.");  // Need input data
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->cnvlStream, batch->searchComp, 0),  "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete

#ifdef SYNCHRONOUS
              // Wait for all the input FFT's to complete
              for (int ss = 0; ss< batch->noStacks; ss++)
              {
                cuFfdotStack* cStack2 = &batch->stacks[ss];
                hipStreamWaitEvent(cStack->cnvlStream, cStack2->prepComp, 0);
              }

              // Wait for the previous convolution to complete
              if ( pStack != NULL )
                hipStreamWaitEvent(cStack->cnvlStream, pStack->convComp, 0);
#endif
            }

            FOLD // Timing event  .
            {
#ifdef TIMING
              CUDA_SAFE_CALL(hipEventRecord(cStack->convInit, cStack->cnvlStream),"Recording event: convInit");
#endif
            }

            FOLD // call kernel(s)  .
            {
              if ( batch->flag & FLAG_RAND_2 )
              {
                convolveffdot00_f(dimGrid, dimBlock, 0, cStack->cnvlStream, batch, ss);
              }
              else
              {
                if ( batch->flag & FLAG_CNV_OVLP )
                {
                  // NOTE: convolveffdot41 seams faster and has been adapted for multi-step
                  convolveffdot71_f(dimGrid, dimBlock, 0, cStack->cnvlStream, cStack->d_kerData, cStack->d_iData, plainsDat, cStack->width, cStack->inpStride, hlist, cStack->height, cStack->kerDatTex, zUp, zDn, batch->noSteps, cStack->noInStack, batch->flag );
                  //convolveffdot72_f(dimGrid, dimBlock, 0, cStack->cnvlStream, batch, ss);
                }
                else
                {
                  if( batch->flag & FLAG_RAND_1 )
                  {
                    convolveffdot43_f(dimGrid, dimBlock, 0, cStack->cnvlStream, batch, ss);
                  }
                  else
                  {
                    convolveffdot41_f(dimGrid, dimBlock, 0, cStack->cnvlStream, cStack->d_kerData, cStack->d_iData, cStack->d_plainData, cStack->width, cStack->inpStride, hlist, cStack->height, kerDat, cStack->kerDatTex, batch->noSteps, cStack->noInStack, batch->flag );
                  }
                }
              }

              // Run message
              CUDA_SAFE_CALL(hipGetLastError(), "Error at kernel launch (convolveffdot7)");
            }

            FOLD // Synchronisation  .
            {
              hipEventRecord(cStack->convComp, cStack->cnvlStream);

#ifdef SYNCHRONOUS
              pStack = cStack;
#endif
            }
          }
        }
        else if ( batch->flag & FLAG_CNV_PLN ) // Do the convolutions one plain  at a time  .
        {
          // NOTE: The use of FLAG_CNV_1KER in this section will be handled because we are using the "kernels" pointers to the complex data
#ifdef SYNCHRONOUS
      cuFfdotStack* pStack = NULL;
#endif

          //for (int ss = plains->noStacks-1; ss >= 0; ss-- )
          for (int ss = 0; ss< batch->noStacks; ss++)              // Loop through Stacks
          {
            cuFfdotStack* cStack = &batch->stacks[ss];
            fcomplexcu* d_plainData;    // The complex f-∂f plain data
            fcomplexcu* d_iData;        // The complex input array

            FOLD // Synchronisation  .
            {
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->cnvlStream, cStack->prepComp,0),    "Waiting for GPU to be ready to copy data to device.");  // Need input data
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->cnvlStream, batch->searchComp, 0),  "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete

#ifdef SYNCHRONOUS
              // Wait for all the input FFT's to complete
              for (int ss = 0; ss< batch->noStacks; ss++)
              {
                cuFfdotStack* cStack2 = &batch->stacks[ss];
                hipStreamWaitEvent(cStack->cnvlStream, cStack2->prepComp, 0);
              }

              // Wait for the previous convolution to complete
              if ( pStack != NULL )
                hipStreamWaitEvent(cStack->cnvlStream, pStack->convComp, 0);
#endif
            }

            FOLD // Timing event  .
            {
#ifdef TIMING
              CUDA_SAFE_CALL(hipEventRecord(cStack->convInit, cStack->cnvlStream),"Recording event: convInit");
#endif
            }

            FOLD // call kernel(s)  .
            {
              for (int si = 0; si< cStack->noInStack; si++)         // Loop through plains in stack
              {
                cuHarmInfo* cHInfo    = &cStack->harmInf[si];       // The current harmonic we are working on
                cuFFdot*    cPlain    = &cStack->plains[si];        // The current f-∂f plain

                dimGrid.x = ceil(cHInfo->width / (float) ( CNV_DIMX * CNV_DIMY ));
                dimGrid.y = 1;

                for (int sti = 0; sti < batch->noSteps; sti++)       // Loop through Steps
                {
                  d_iData       = cPlain->d_iData + cHInfo->inpStride * sti;

                  if      ( batch->flag & FLAG_STP_ROW )
                  {
                    fprintf(stderr,"ERROR: Cannot do single plain convolutions with row interleave multi step stacks.\n");
                    exit(EXIT_FAILURE);
                  }
                  else if ( batch->flag & FLAG_STP_PLN )
                    d_plainData = cPlain->d_plainData + sti * cHInfo->height * cHInfo->inpStride;   // Shift by plain height
                  else if ( batch->flag & FLAG_STP_STK )
                    d_plainData = cPlain->d_plainData + sti * cStack->height * cHInfo->inpStride;   // Shift by stack height
                  else
                    d_plainData   = cPlain->d_plainData;  // If nothing is specified just use plain data

                  if ( batch->flag & FLAG_CNV_TEX )
                    convolveffdot36<<<dimGrid, dimBlock, 0, cStack->cnvlStream>>>(d_plainData, cHInfo->width, cHInfo->inpStride, cHInfo->height, d_iData, cPlain->kernel->kerDatTex);
                  else
                    convolveffdot31<<<dimGrid, dimBlock, 0, cStack->cnvlStream>>>(d_plainData, cHInfo->width, cHInfo->inpStride, cHInfo->height, d_iData, cPlain->kernel->d_kerData);

                  // Run message
                  CUDA_SAFE_CALL(hipGetLastError(), "Error at convolution kernel launch");
                }
              }
            }

            FOLD // Synchronisation  .
            {
              hipEventRecord(cStack->convComp, cStack->cnvlStream);

#ifdef SYNCHRONOUS
              pStack = cStack;
#endif
            }
          }

        }
        else
        {
          fprintf(stderr, "ERROR: convolveBatch not templated for this type of convolution.\n");
        }

        if ( DBG_PLN01 ) // Print debug info  .
        {
          for (int ss = 0; ss < batch->noStacks; ss++)
          {
            cuFfdotStack* cStack = &batch->stacks[ss];
            CUDA_SAFE_CALL(hipStreamSynchronize(cStack->cnvlStream),"");
          }

          for (int ss = 0; ss < batch->noHarms; ss++) // Print
          {
            cuFFdot* cPlain     = &batch->plains[batch->pIdx[ss]];
            printf("\nGPU Convolved h:%i   f: %f\n",ss,cPlain->harmInf->harmFrac);
            printData_cu(batch, batch->flag, batch->pIdx[ss], 10, 1);
            CUDA_SAFE_CALL(hipStreamSynchronize(0),"");
          }
        }
      }

      FOLD // Inverse FFT the  f-∂f plain  .
      {

#ifdef STPMSG
    printf("\t\tInverse FFT\n");
#endif

#ifdef SYNCHRONOUS
      cuFfdotStack* pStack = NULL;
#endif

        // Copy fft data to device
        //for (int ss = plains->noStacks-1; ss >= 0; ss-- )
        for (int ss = 0; ss< batch->noStacks; ss++)
        {
          cuFfdotStack* cStack = &batch->stacks[ss];

          FOLD // Synchronisation  .
          {
            hipStreamWaitEvent(cStack->fftPStream, cStack->convComp, 0);
            hipStreamWaitEvent(cStack->fftPStream, batch->convComp,  0);

#ifdef SYNCHRONOUS
            // Wait for all the convolutions to complete
            for (int ss = 0; ss< batch->noStacks; ss++)
            {
              cuFfdotStack* cStack2 = &batch->stacks[ss];
              hipStreamWaitEvent(cStack->fftPStream, cStack2->convComp, 0);
            }

            // Wait for the previous fft to complete
            if ( pStack != NULL )
              hipStreamWaitEvent(cStack->fftPStream, pStack->plnComp, 0);
#endif
          }

          FOLD // Call the inverse CUFFT  .
          {
#pragma omp critical
            {
              FOLD // Timing  .
              {
#ifdef TIMING
                hipEventRecord(cStack->invFFTinit, cStack->fftPStream);
#endif
              }

              if ( batch->flag & FLAG_CUFFTCB_OUT ) // Set the CUFFT callback to calculate and store powers  .
              {
                //hipfftCallbackLoadC hostCopyOfCallbackPtr;
                //CUDA_SAFE_CALL(hipMemcpyFromSymbol( &hostCopyOfCallbackPtr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(hostCopyOfCallbackPtr)),  "");
                //CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&hostCopyOfCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&cStack->d_plainPowers ),"");

                CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&batch->h_stCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&cStack->d_plainPowers ),"");
              }

              CUFFT_SAFE_CALL(hipfftSetStream(cStack->plnPlan, cStack->fftPStream),  "Error associating a CUFFT plan with cnvlStream.");
              CUFFT_SAFE_CALL(hipfftExecC2C(cStack->plnPlan, (hipfftComplex *) cStack->d_plainData, (hipfftComplex *) cStack->d_plainData, HIPFFT_BACKWARD),"Error executing CUFFT plan.");

              FOLD // Synchronisation  .
              {
                hipEventRecord(cStack->plnComp, cStack->fftPStream);

#ifdef SYNCHRONOUS
                pStack = cStack;
#endif
              }
            }
          }
        }
      }
    }

    batch->haveInput    = 0;
    batch->haveConvData = 1;

    nvtxRangePop();
  }

  // Set the r-values and width for the next iteration when we will be doing the actual Add and Search
  cycleRlists(batch);
}


