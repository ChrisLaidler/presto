#include "hip/hip_runtime.h"
#include "cuda_accel_SS.h"

#define SS34_X           8                    // X Thread Block
#define SS34_Y           8                     // Y Thread Block
#define SS34BS           (SS34_X*SS34_Y)

__device__ const int stride_c[] = {4096, 2048, 4096, 1024, 4096, 4096, 2048, 1024, 4096, 4096, 4096, 4096, 2048, 2048, 1024, 512 };
__device__ const int height_c[] = {301, 151, 227, 77, 263, 189, 113, 39, 283, 245, 207, 169, 133, 95, 57, 19 };
__device__ const int hwidth_c[] = {360, 164, 264, 88, 314, 212, 124, 58, 338, 290, 236, 186, 144, 106, 72, 42 };



/** Sum and Search - loop down - column max - multi-step - step outer .
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base          Used in CU_OUTP_DEVICE
 * @param noSteps
 */
template<uint FLAGS, const int noStages, const int noHarms, const int cunkSize, const int noSteps>
__global__ void add_and_searchCU34_k(const uint width, __restrict__ candPZs* d_cands, tHarmList texs, fsHarmList powersArr, cHarmList cmplxArr )
{
  const int tid   = threadIdx.y * SS34_X  +  threadIdx.x;   /// Block index
  const int gid   = blockIdx.x  * SS34BS  +  tid;           /// Global thread id (ie column) 0 is the first 'good' column

  if ( tid < width )
  {
    const int zeroHeight  = HEIGHT_STAGE[0];
    const int oStride     = STRIDE_STAGE[0];                /// The stride of the output data

    int       inds      [noHarms];
    int       len       [noHarms];
    int       inds1     [noHarms];

    //int       stride    [noHarms];

    // Candidates
    //float     candPow   [noStages];
    //int       candZ     [noStages];
    //float     candPow   [noStages][noSteps];
    //int       candZ     [noStages][noSteps];

    // Powers
    //float     powers    [noHarms][noSteps];
    //int       idxP      [noHarms];

    __shared__  float powersSM[noSteps][cunkSize][SS34BS];

    FOLD // Prep - Initialise the x indices & set candidates to 0 .
    {
      FOLD // Calculate the x indices or create a pointer offset by the correct amount  .
      {
        for ( int harm = 0; harm < noHarms; harm++ )                // loop over harmonic  .
        {
          // NOTE: the indexing below assume each plain starts on a multiple of noHarms
          int   ix    = roundf( gid*FRAC_STAGE[harm] ) + HWIDTH_STAGE[harm] ;
          int   ix0   = roundf( blockIdx.x*SS34BS*FRAC_STAGE[harm] ) + HWIDTH_STAGE[harm] ;
          int   ix1   = roundf( ((blockIdx.x+1)*SS34BS-1)*FRAC_STAGE[harm] ) + HWIDTH_STAGE[harm] ;
          len[harm]   = ix1 - ix0 + 1;
          inds[harm]  = ix0 + tid;
          inds1[harm] = ix - ix0;

//          if ( FLAGS & FLAG_ITLV_PLN )
//          {
//            stride[harm] = STRIDE_STAGE[harm] ;
//          }
//          else
//          {
//            stride[harm] = noSteps*STRIDE_STAGE[harm] ;
//          }
        }
      }

      FOLD  // Set the local and return candidate powers to zero  .
      {
        for ( int stage = 0; stage < noStages; stage++ )
        {
          for ( int step = 0; step < noSteps; step++)                 // Loop over steps  .
          {
            //candPow [stage][step] = POWERCUT_STAGE[stage];
            d_cands[step*noStages*oStride + stage*oStride + gid ].value   = 0 ;
          }
        }
      }
    }

    FOLD //
    {
      float P = 0;

      for( int y = 0; y < zeroHeight ; y += cunkSize )            // loop over chunks  .
      {

        for ( int stage = 0 ; stage < noStages; stage++)          // Loop over stages  .
        {
          int start = STAGE[stage][0] ;
          int end   = STAGE[stage][1] ;

//          FOLD // Initialise powers for each section column to 0  .
//          {
//            //#pragma unroll
//            for ( int step = 0; step < noSteps; step++)                 // Loop over steps .
//            {
//              //#pragma unroll
//              for( int yPlus = 0; yPlus < cunkSize ; yPlus++ )          // Loop over powers .
//              {
//                powers[step][yPlus] = 0;
//              }
//            }
//          }

          for ( int harm = start; harm <= end; harm++ )           // Loop over harmonics (batch) in this stage  .
          {
            //int xx      = inds1[harm];
            //int iy0     = YINDS[ zeroHeight*harm + y ];

            //__syncthreads();

            FOLD // Read into SM  .
            {
              if ( tid < len[harm] )
              {
                int iy0       = YINDS[ zeroHeight*harm + y ];
                int end       = MIN(y+cunkSize, zeroHeight );
                int iy1       = YINDS[ zeroHeight*harm + end-1 ];

                int ix1       = inds[harm] ;
                int ix2       = ix1;

                int yy        = 0;

                for ( int yPlus = iy0; yPlus <= iy1; yy++, yPlus++)
                {
                  for ( int step = 0; step < noSteps; step++)         // Loop over steps  .
                  {
                    int iy2;

                    float pp;

                    FOLD // Calculate index  .
                    {
                      if        ( FLAGS & FLAG_ITLV_PLN )
                      {
                        iy2                   = ( yPlus + step * HEIGHT_STAGE[harm] ) * STRIDE_STAGE[harm];
                      }
                      else
                      {
                        ix2                   = ix1   + step *  STRIDE_STAGE[harm] ;
                        iy2                   = yPlus * noSteps * STRIDE_STAGE[harm] ;
                      }
                    }

                    FOLD // Accumulate powers  .
                    {
                      if      ( FLAGS & FLAG_MUL_CB_OUT )
                      {
                        //powers[harm][step]    = powersArr[harm][ iy2 + ix2 ];

                        pp                      = powersArr[harm][ iy2 + ix2 ];
                        powersSM[step][yy][tid] += pp ;

                        P += pp;
                      }
                      else
                      {
                        fcomplexcu cmpc         = cmplxArr[harm][ iy2 + ix2 ];
                        pp                      = cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                        powersSM[step][yy][tid] += pp ;

                        P += pp;
                      }
                    }
                  }
                }
              }
            }

            __syncthreads();

//            Fout // Accumulate  .
//            {
//              for ( int yPlus = y; yPlus < cunkSize; yPlus++)
//              {
//                int trm     = y + yPlus ;                           ///< True Y index in plain
//                int yy      = YINDS[ zeroHeight*harm + trm ] - iy0;
//
//                for ( int step = 0; step < noSteps; step++)         // Loop over steps  .
//                {
//                  powers[step][yPlus]  += powersSM[step][yy][xx];
//                }
//              }
//            }
          }

//          Fout // Search set of powers  .
//          {
//            for ( int step = 0; step < noSteps; step++)           // Loop over steps  .
//            {
//              for( int yPlus = 0; yPlus < cunkSize ; yPlus++ )     // Loop over section  .
//              {
//                if  (  powers[step][yPlus] > POWERCUT_STAGE[stage] )
//                {
//                  if ( powers[step][yPlus] > candPow [stage][step] )
//                  {
//                    if ( y + yPlus < zeroHeight )
//                    {
//                      // This is our new max!
//                      candPow [stage][step]  = powers[step][yPlus];
//                      candZ   [stage][step]  = y+yPlus;
//                    }
//                  }
//                }
//              }
//            }
//          }
        }
      }

      if ( P < 0 )
      {
        printf("P %f\n", P);
      }
    }

  }
}

template<uint FLAGS, int noStages, const int noHarms, const int cunkSize>
__host__ void add_and_searchCU34_q(dim3 dimGrid, dim3 dimBlock, hipStream_t stream, cuFFdotBatch* batch )
{
  const int noSteps = batch->noSteps ;

  for ( int step = 0; step < noSteps; step++)
  {
    long long firstBin  = (*batch->rConvld)[step][0].expBin ;

    for (int i = 0; i < noHarms; i++)
    {
      int idx =  batch->stageIdx[i];

      long long binb      = (*batch->rConvld)[step][idx].expBin ;

      if ( firstBin * h_FRAC_STAGE[i] != binb )
      {
        fprintf(stderr,"ERROR, in function %s, R values are not properly aligned! Each step should start on a multiple of (2 x No Harms).\n", __FUNCTION__ );
        fprintf(stderr,"%f != %f.\n", firstBin * h_FRAC_STAGE[i], (float)binb );
        exit(EXIT_FAILURE);
      }
    }
  }

  tHarmList   texs;
  fsHarmList  powers;
  cHarmList   cmplx;

  for (int i = 0; i < noHarms; i++)
  {
    int idx         = batch->stageIdx[i];
    texs.val[i]     = batch->plains[idx].datTex;
    powers.val[i]   = batch->plains[idx].d_plainPowers;
    cmplx.val[i]    = batch->plains[idx].d_plainData;
  }

  switch (noSteps)
  {
    case 1:
    {
      add_and_searchCU34_k<FLAGS,noStages,noHarms,cunkSize,1><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candPZs*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 2:
    {
      add_and_searchCU34_k<FLAGS,noStages,noHarms,cunkSize,2><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candPZs*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 3:
    {
      add_and_searchCU34_k<FLAGS,noStages,noHarms,cunkSize,3><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candPZs*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 4:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU34_k<FLAGS),noStages,noHarms,cunkSize,4>, hipFuncCachePreferL1);
      add_and_searchCU34_k<FLAGS,noStages,noHarms,cunkSize,4><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candPZs*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 5:
    {
      add_and_searchCU34_k<FLAGS,noStages,noHarms,cunkSize,5><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candPZs*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 6:
    {
      add_and_searchCU34_k<FLAGS,noStages,noHarms,cunkSize,6><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candPZs*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 7:
    {
      add_and_searchCU34_k<FLAGS,noStages,noHarms,cunkSize,7><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candPZs*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 8:
    {
      add_and_searchCU34_k<FLAGS,noStages,noHarms,cunkSize,8><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candPZs*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    default:
      fprintf(stderr, "ERROR: add_and_searchCU311 has not been templated for %i steps\n", noSteps);
      exit(EXIT_FAILURE);
  }
}

template<uint FLAGS, int noStages, const int noHarms>
__host__ void add_and_searchCU34_c(dim3 dimGrid, dim3 dimBlock, hipStream_t stream, cuFFdotBatch* batch )
{
    switch (globalInt01)
    {
//      case 1:
//      {
//        add_and_searchCU34_q<FLAGS,noStages,noHarms,1>(dimGrid, dimBlock, stream, batch);
//        break;
//      }
//      case 2:
//      {
//        add_and_searchCU34_q<FLAGS,noStages,noHarms,2>(dimGrid, dimBlock, stream, batch);
//        break;
//      }
//      //    case 3:
//      //    {
//      //      add_and_searchCU34_q<FLAGS,noStages,noHarms,3>(dimGrid, dimBlock, stream, batch);
//      //      break;
//      //    }
      case 4:
      {
        add_and_searchCU34_q<FLAGS,noStages,noHarms,4>(dimGrid, dimBlock, stream, batch);
        break;
      }
//      //    case 5:
//      //    {
//      //      add_and_searchCU34_q<FLAGS,noStages,noHarms,5>(dimGrid, dimBlock, stream, batch);
//      //      break;
//      //    }
      case 6:
      {
        add_and_searchCU34_q<FLAGS,noStages,noHarms,6>(dimGrid, dimBlock, stream, batch);
        break;
      }
//      //    case 7:
//      //    {
//      //      add_and_searchCU34_q<FLAGS,noStages,noHarms,7>(dimGrid, dimBlock, stream, batch);
//      //      break;
//      //    }
      case 8:
      {
        add_and_searchCU34_q<FLAGS,noStages,noHarms,8>(dimGrid, dimBlock, stream, batch);
        break;
      }
//      //    case 9:
//      //    {
//      //      add_and_searchCU34_q<FLAGS,noStages,noHarms,9>(dimGrid, dimBlock, stream, batch);
//      //      break;
//      //    }
      case 10:
      {
        add_and_searchCU34_q<FLAGS,noStages,noHarms,10>(dimGrid, dimBlock, stream, batch);
        break;
      }
      case 12:
      {
        add_and_searchCU34_q<FLAGS,noStages,noHarms,12>(dimGrid, dimBlock, stream, batch);
        break;
      }
      case 14:
      {
        add_and_searchCU34_q<FLAGS,noStages,noHarms,14>(dimGrid, dimBlock, stream, batch);
        break;
      }
      case 16:
      {
        add_and_searchCU34_q<FLAGS,noStages,noHarms,16>(dimGrid, dimBlock, stream, batch);
        break;
      }
//      case 18:
//      {
//        add_and_searchCU34_q<FLAGS,noStages,noHarms,18>(dimGrid, dimBlock, stream, batch);
//        break;
//      }
//      case 20:
//      {
//        add_and_searchCU34_q<FLAGS,noStages,noHarms,20>(dimGrid, dimBlock, stream, batch);
//        break;
//      }
//      case 24:
//      {
//        add_and_searchCU34_q<FLAGS,noStages,noHarms,24>(dimGrid, dimBlock, stream, batch);
//        break;
//      }
      default:
        fprintf(stderr, "ERROR: %s has not been templated for %i chunk size.\n", __FUNCTION__, globalInt01);
        exit(EXIT_FAILURE);
    }

  //add_and_searchCU34_q<FLAGS,noStages,noHarms,24>(dimGrid, dimBlock, stream, batch);
}

template<uint FLAGS >
__host__ void add_and_searchCU34_p(dim3 dimGrid, dim3 dimBlock, hipStream_t stream, cuFFdotBatch* batch )
{
  const int noStages = batch->noHarmStages;

  switch (noStages)
  {
    //    case 1:
    //    {
    //      add_and_searchCU34_c<FLAGS,1,1>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    //    case 2:
    //    {
    //      add_and_searchCU34_c<FLAGS,2,2>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    //    case 3:
    //    {
    //      add_and_searchCU34_c<FLAGS,3,4>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    //    case 4:
    //    {
    //      add_and_searchCU34_c<FLAGS,4,8>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    case 5:
    {
      add_and_searchCU34_c<FLAGS,5,16>(dimGrid, dimBlock, stream, batch);
      break;
    }
    default:
      fprintf(stderr, "ERROR: %s has not been templated for %i stages\n", __FUNCTION__, noStages);
      exit(EXIT_FAILURE);
  }
}

__host__ void add_and_searchCU34(hipStream_t stream, cuFFdotBatch* batch )
{
  const uint FLAGS = batch->flag;
  dim3 dimBlock, dimGrid;

  dimBlock.x  = SS34_X ;
  dimBlock.y  = SS34_Y ;

  float bw    = SS34BS ;
  float ww    = batch->accelLen / ( bw );

  dimGrid.x   = ceil(ww);
  dimGrid.y   = 1;

  if        ( FLAGS & FLAG_MUL_CB_OUT )
  {
    if      ( FLAGS & FLAG_ITLV_ROW )
      add_and_searchCU34_p<FLAG_MUL_CB_OUT | FLAG_ITLV_ROW> (dimGrid, dimBlock, stream, batch);
    else if ( FLAGS & FLAG_ITLV_PLN )
      add_and_searchCU34_p<FLAG_MUL_CB_OUT | FLAG_ITLV_PLN>  (dimGrid, dimBlock, stream, batch);
    else
    {
      fprintf(stderr, "ERROR: %s has not been templated for flag combination. \n", __FUNCTION__ );
      exit(EXIT_FAILURE);
    }
  }
  else
  {
    if      ( FLAGS & FLAG_ITLV_ROW )
      add_and_searchCU34_p<FLAG_ITLV_ROW> (dimGrid, dimBlock, stream, batch);
    else if ( FLAGS & FLAG_ITLV_PLN )
      add_and_searchCU34_p<FLAG_ITLV_PLN> (dimGrid, dimBlock, stream, batch);
    else
    {
      fprintf(stderr, "ERROR: %s has not been templated for flag combination. \n", __FUNCTION__ );
      exit(EXIT_FAILURE);
    }
  }

  int tmp = 0;
}

