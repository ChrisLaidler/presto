#include "hip/hip_runtime.h"
/**
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation version 3.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

/*
#include <hipcub/hipcub.hpp>
#include <nvToolsExt.h>
#include <nvToolsExtCudaRt.h>

#include "cuda_accel_utils.h"
#include "cuda_utils.h"

extern "C"
{
#define __float128 long double
#include "accel.h"
}
 */

#include <hipfft/hipfft.h>
#include <algorithm>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <nvToolsExt.h>
#include <nvToolsExtCudaRt.h>

extern "C"
{
//#define __float128 long double
//#include "accel.h"
}

#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"

__device__ __constant__ int        YINDS[MAX_YINDS];
__device__ __constant__ float      POWERCUT[MAX_HARM_NO];
__device__ __constant__ long long  NUMINDEP[MAX_HARM_NO];

/** Return x such that 2**x = n
 *
 * @param n
 * @return
 */
inline int twon_to_index(int n)
{
  int x = 0;

  while (n > 1)
  {
    n >>= 1;
    x++;
  }
  return x;
}

template<uint FLAGS>
__device__ inline int getY(int plainY, const int noSteps,  const int step, const int plainHeight = 0 /*, const int stackHeight = 0*/ )
{
  // Calculate y indice from interleave method
  if      ( FLAGS & FLAG_STP_ROW )
  {
    return plainY * noSteps + step;
  }
  else if ( FLAGS & FLAG_STP_PLN )
  {
    return plainY + plainHeight*step;
  }
  /*
  else if ( FLAGS & FLAG_STP_STK )
  {
    return plainY + stackHeight*step;
  }
   */
  else
    return 0;
}

template<uint FLAGS>
__device__ inline float getPower(const int ix, const int iy, hipTextureObject_t tex, fcomplexcu* base, const int stride)
{
  if  ( (FLAGS & FLAG_PLN_TEX ) )
  {
    const float2 cmpf = tex2D < float2 > (tex, ix, iy);
    return (cmpf.x * cmpf.x + cmpf.y * cmpf.y);
  }
  else
  {
    const fcomplexcu cmpc  = base[iy*stride+ix];
    return (cmpc.r * cmpc.r + cmpc.i * cmpc.i);
  }
}

template<int n>
__host__ __device__ void cdfgam_d(double x, double *p, double* q)
{
  if      ( n == 1 )
  {
    *q = exp(-x);
  }
  else if ( n == 2 )
  {
    *q = exp(-x)*( x + 1.0 );
  }
  else if ( n == 4 )
  {
    *q = exp(-x)*( x*(x*(x/6.0 + 0.5) + 1.0 ) + 1.0 );
  }
  else if ( n == 8 )
  {
    *q = exp(-x)*( x*(x*(x*(x*(x*(x*(x/5040.0 + 1.0/720.0 ) + 1.0/120.0 ) + 1.0/24.0 ) + 1.0/6.0 ) + 0.5 ) + 1.0 ) + 1.0 );
  }
  else if ( n == 16 )
  {
    *q = exp(-x)*( x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x/1.307674368e12 +  1.0/8.71782912e10 ) \
        + 1.0/6227020800.0 )+ 1.0/479001600.0 ) \
        + 1.0/39916800.0 )+ 1.0/3628800.0 )     \
        + 1.0/362880.0 ) + 1.0/40320.0 )        \
        + 1.0/5040.0 ) + 1.0/720.0 ) + 1.0/120.0 ) + 1.0/24.0 ) + 1.0/6.0 ) + 0.5 ) + 1.0 )  + 1.0 );
  }
  else
  {
    *q = 1.0 + x ;
    double numerator    = x;
    double denominator  = 1.0;

#pragma unroll
    for ( int i = 2 ; i < n ; i ++ )
    {
      denominator *= i;
      numerator   *= x;
      *q += numerator/denominator;
    }

  }
  *p = 1-*q;
}

/** Inverse normal CDF - ie calculate σ from p and/or q
 * We include p and q because if p is close to 1 or -1 , q can hold more precision
 */
__host__ __device__ double incdf (double p, double q )
{
  double a[] = {              \
      -3.969683028665376e+01, \
      2.209460984245205e+02,  \
      -2.759285104469687e+02, \
      1.383577518672690e+02,  \
      -3.066479806614716e+01, \
      2.506628277459239e+00   };

  double b[] = {              \
      -5.447609879822406e+01, \
      1.615858368580409e+02,  \
      -1.556989798598866e+02, \
      6.680131188771972e+01,  \
      -1.328068155288572e+01  };

  double c[] = {              \
      -7.784894002430293e-03, \
      -3.223964580411365e-01, \
      -2.400758277161838e+00, \
      -2.549732539343734e+00, \
      4.374664141464968e+00, \
      2.938163982698783e+00 };

  double d[] = {            \
      7.784695709041462e-03, \
      3.224671290700398e-01, \
      2.445134137142996e+00, \
      3.754408661907416e+00 };

  double l, ll, x, e, u;
  double sighn = 1.0;

  // More precision in q so use it
  if ( p > 0.99 || p < -0.99 )
  {
    if ( q < 1.0 )
    {
      sighn = -1.0;
      double tmp = p;
      p = q;
      q = tmp;
    }
  }

  // Make an initial estimate for x
  // The algorithm taken from: http://home.online.no/~pjacklam/notes/invnorm/#The_algorithm
  if ( 0.02425 <= p && p <= 0.97575 )
  {
    l    =  p - 0.5;
    ll   = l*l;
    x    = (((((a[1]*ll+a[2])*ll+a[3])*ll+a[4])*ll+a[5])*ll+a[6])*l / (((((b[1]*ll+b[2])*ll+b[3])*ll+b[4])*ll+b[5])*ll+1.0);
  }
  else
  {
    if ( 0.02425 > p )
    {
      l = sqrt(-2.0*log(p));
    }
    else if ( 0.97575 < p )
    {
      l = sqrt(-2.0*log( 1.0 - p ));
    }
    x = (((((c[1]*l+c[2])*l+c[3])*l+c[4])*l+c[5])*l+c[6]) / ((((d[1]*l+d[2])*l+d[3])*l+d[4])*l+1.0);

    if ( 0.97575 < p )
    {
      x *= -1.0;
    }
  }

  // Now do a Newton Raphson recursion to refine the answer.
  // Using erfc and exp to calculate  f(x) = Φ(x)-p  and  f'(x) = Φ'(x)
  double f = 0.5 * erfc(-x/1.414213562373095048801688724209) - p ;
  double xOld = x;
  for ( int i = 0; i < 5 ; i++ ) // Note: only doing 5 recursions this could be pushed up
  {
    u = 0.398942*exp(-x*x/2.0);
    x = x - f / u ;

    f = 0.5 * erfc(-x/1.414213562373095048801688724209) - p;
    e = f / p;

    if ( fabs(e) < 1e-15 || ( x == xOld ) )
      break ;

    xOld = x;
  }

  return sighn*x;
}

__host__ __device__ double candidate_sigma_cu(double poww, int numharm, long long numindep)
{
  int n = numharm;
  if ( poww > 100)
  {
    double c[] = { \
        -7.784894002430293e-03, \
        -3.223964580411365e-01, \
        -2.400758277161838e+00, \
        -2.549732539343734e+00, \
        4.374664141464968e+00,  \
        2.938163982698783e+00 };

    double d[] = { \
        7.784695709041462e-03, \
        3.224671290700398e-01, \
        2.445134137142996e+00, \
        3.754408661907416e+00 };

    double logQ, logP;
    if      ( n == 1 )
    {
      logQ = -poww;
    }
    else if ( n == 2 )
    {
      logQ = -poww+log( poww + 1.0 );
    }
    else if ( n == 4 )
    {
      logQ = -poww + log( poww*(poww*(poww/6.0 + 0.5) + 1.0 ) + 1.0 );
    }
    else if ( n == 8 )
    {
      logQ = -poww + log( poww*(poww*(poww*(poww*(poww*(poww*(poww/5040.0 + 1.0/720.0 ) + 1.0/120.0 ) + 1.0/24.0 ) + 1.0/6.0 ) + 0.5 ) + 1.0 ) + 1.0 );
    }
    else if ( n == 16 )
    {
      logQ = -poww + log( poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww/1.307674368e12 +  1.0/8.71782912e10 ) \
          + 1.0/6227020800.0 )+ 1.0/479001600.0 ) \
          + 1.0/39916800.0 )+ 1.0/3628800.0 ) \
          + 1.0/362880.0 ) + 1.0/40320.0 ) \
          + 1.0/5040.0 ) + 1.0/720.0 ) + 1.0/120.0 ) + 1.0/24.0 ) + 1.0/6.0 ) + 0.5 ) + 1.0 )  + 1.0 );
    }

    //logP = log(1-exp(logQ));

    logQ += log( (double)numindep );

    double l = sqrt(-2.0*logQ);

    //double x = -1.0 * (((((c[1]*l+c[2])*l+c[3])*l+c[4])*l+c[5])*l+c[6]) / ((((d[1]*l+d[2])*l+d[3])*l+d[4])*l+1.0);
    double x = l - ( 2.515517 + l * (0.802853 + l * 0.010328) ) / ( 1.0 + l * (1.432788 + l * (0.189269 + l * 0.001308)) ) ;

    //return logQ;
    return x;
  }
  else
  {
    double gpu_p, gpu_q, sigc, pp ;

    if(numharm==1)
      cdfgam_d<1>(poww, &gpu_p, &gpu_q );
    else if(numharm==2)
      cdfgam_d<2>(poww, &gpu_p, &gpu_q );
    else if(numharm==4)
      cdfgam_d<4>(poww, &gpu_p, &gpu_q );
    else if(numharm==8)
      cdfgam_d<8>(poww, &gpu_p, &gpu_q );
    else if(numharm==16)
      cdfgam_d<16>(poww, &gpu_p, &gpu_q );

    if (gpu_p == 1.0)
      gpu_q *= numindep;
    else
    {
      gpu_q = 1.0 - pow(gpu_p, (double)numindep);
      //pp = pow((1.0-gpu_q),1.0/(double)numindep);
    }
    gpu_p = 1.0 - gpu_q;

    sigc = incdf(gpu_p, gpu_q);

    //return gpu_q;
    return sigc;
  }
}

/*
__device__ void search(const float powers[SS_Y_NUM][SS_X_NUM], int stage, cand* d_candList, cand* candListsm, int *count, float rlo, float zlo)
{
  int sx;     // Shared memory x location
  int sy;     // Shared memory y location

  // Specialize BlockScan for a 1D block of 128 threads on type int
  //typedef hipcub::BlockScan<int, SS_X, hipcub::BLOCK_SCAN_RAKING, SS_Y> BlockScan;

  // Allocate shared memory for BlockScan
  //__shared__ typename BlockScan::TempStorage temp_storage;

  //const uint tid = threadIdx.y * SS_X + threadIdx.x;

  const float powcut = POWERCUT[stage];
  float mypwer;

  //__shared__ uint ff;

  __shared__ unsigned long long int sem;
  //  __shared__ uint ff2;
  //__shared__ int mutex;
  //__shared__ int cnts[SS_X*SS_Y];

  //if (tid == 0)
  {
    //*count = 0;
    //ff2 = 0;
    //sem = 0;
  }

  //int start = 0;
  //int spin = 0;
  //cand lCands[SS_Y_TILES*SS_X_TILES];

  __syncthreads();

  // Read data from fundamental into shared memory
#ifndef DEBUG
#pragma unroll
#endif
  for (int y = 0; y < SS_Y_TILES; y++)
  {
    sy = y * SS_Y + threadIdx.y;

#ifndef DEBUG
#pragma unroll
#endif
    for (int x = 0; x < SS_X_TILES; x++)
    {
      sx = x * SS_X + threadIdx.x;
      if (sx > SS_X_OVERLAP && sy > SS_Y_OVERLAP && sx < SS_X_NUM - SS_X_OVERLAP && sy < SS_Y_NUM - SS_Y_OVERLAP)
      {
        mypwer = powers[sy][sx];
        int lmax = true;

        if (mypwer > powcut)
        {
          //if (lmax == true)
          {
            //float power =
            //float pow = powers[y][x];
            //float sig = candidate_sigma(pow, numharm, numindep);
            //float rr = (ffdot->rlo + jj * (double) ACCEL_DR) / (double) numharm;
            //float zz = (ffdot->zlo + ii * (double) ACCEL_DZ) / (double) numharm;
            //printf("Found a canidate! Power: %.3f  sig: %.4f \n",pow, sig);
            //          ff++;

            //while ( atomicCAS( &mutex, 0, 1 ) );

            //ff2++;
            //mutex = 0;
            //atomicExch( &mutex, 0 );

            //cnts[tid]++;
            //volatile
            //volatile
            //__syncthreads();
            //while (atomicCAS((unsigned long long int*) &sem, 0, tid + 1) != 0)
            //{
            //}

            //printf("Assuire sem %i \n", tid );

            //acquire_semaphore(&g_canSem);
            // candList[ff2].power =  mypwer;
            //candList[ff2].r = x;
            //candList[ff2].z = y;

            //__syncthreads();

            if (1)
            {
              if (0)
              {
                volatile bool done = false;

                //atomicAdd(&g_canCount, 1);
                // Global synch and add
                int cnt = 0;
                while (!done)
                {
                  volatile int prev = atomicCAS((unsigned long long int*) &g_canSem, SEMFREE, 1);
                  if (prev == SEMFREE )  // || cnt > 5000)
                  {
                    if (g_canCount < MAX_CANDS_PER_BLOCK)
                    {
                      d_candList[g_canCount].power = mypwer;
                      d_candList[g_canCount].r = (rlo + sx * (double) ACCEL_DR);
                      d_candList[g_canCount].z = (zlo + sy * (double) ACCEL_DZ);
                      d_candList[g_canCount].numharm = stage;
                      g_canCount++;
                    }
                    else
                    {
                      printf("/rCUDA Candidate list to short, not recording candidate!  lost: %i candidates.\n", g_canCount-MAX_CANDS_PER_BLOCK);
                      //g_canCount++;
                    }
                    //printf("Found a canidate! Power: %.3f  %i  \n", mypwer, g_canCount);
                    //ff2++;
                    done = true;
                    g_canSem = SEMFREE;
                  }
                  //cnt++;
                  //__syncthreads();
                }
              }
              else
              {
                //int cnt = 0;
                // Shared synch and add
                //while (!done && cnt < 5000)
                {
                  //volatile int prev = atomicCAS((unsigned long long int*) &sem, 0, 1);
                  //prev = 0;
                  //if (prev == 0)
                  //if ( (*count) < SS_X_TILES*SS_Y_TILES*5 )
                  if ( true )
                  {
                    atomicAdd((unsigned long long int*) &g_canCount, 1);

                    candListsm[*count].power = mypwer;
                    candListsm[*count].r = (rlo + sx * (double) ACCEL_DR);
                    candListsm[*count].z = (zlo + sy * (double) ACCEL_DZ);
                    candListsm[*count].numharm = stage;
                    (*count)++;

                    //ff2++;
                    //done = true;
                    //sem = 0;
                  }
                  //else
                  {
                    //printf("Well this is strange?\n");
                  }
                  //cnt++;
                  //__syncthreads();
                }
              }
            }
            else
            {
              //atomicAdd((unsigned long long int*) &g_canSem, 1);
              //atomicAdd((unsigned long long int*) &sem, 1);
            }

            //__syncthreads();
            //atomicAdd(&ff, 1);
          }
          //else
          {
            //powers[sy][sx] = 0;
          }
        }
        else
        {
          //powers[sy][sx] = 0;
        }
      }
    }
  }
  //if ( ff )
  //printf("Found %i canidates!\n", ff);

  //__syncthreads();

  // Collectively compute the block-wide exclusive prefix sum
  //BlockScan(temp_storage).ExclusiveSum(count, start);

  //__syncthreads();
  //__threadfence();

  //if (threadIdx.x == (SS_X-1) && threadIdx.y == (SS_Y-1) && start > 0 )
  //  printf("Found %i canidates!\n", start);

   //if (tid == (SS_X) * (SS_Y) && *count > 0)
   //printf("Found %i  %i  %i  canidates!\n", *count, (int) sem, spin);

  //for( int i = 0; i < count; i++ )
  {
    //candList[start+i] = lCands[i];
  }
}
*/

/*
template<int stage>
__device__ static inline void searcht(const float powers_t[SS_Y_TILES][SS_X_TILES], cand* d_candList, cand* candListsm, int *count, double rlo, float zlo, int tlx, int tly)
{
  int sx;     // Shared memory x location
  int sy;     // Shared memory y location

  // Specialize BlockScan for a 1D block of 128 threads on type int
  //typedef hipcub::BlockScan<int, SS_X, hipcub::BLOCK_SCAN_RAKING, SS_Y> BlockScan;

  // Allocate shared memory for BlockScan
  //__shared__ typename BlockScan::TempStorage temp_storage;

  const uint tid = threadIdx.y * SS_X + threadIdx.x;

  const float powcut = POWERCUT[stage];
  float mypwer;

  //__shared__ uint ff;

  __shared__ unsigned long long int sem;
  //  __shared__ uint ff2;
  //__shared__ int mutex;
  //__shared__ int cnts[SS_X*SS_Y];

  //if (tid == 0)
  {
    //*count = 0;
    //ff2 = 0;
    //sem = 0;
  }

  //int start = 0;
  //int spin = 0;
  //cand lCands[SS_Y_TILES*SS_X_TILES];

  //__syncthreads();

  // Read data from fundamental into shared memory
#ifndef DEBUG
#pragma unroll
#endif
  for (int y = 0; y < SS_Y_TILES; y++)
  {
    //if ( piy >= 0 && piy < searchList.height.val[0] )
    {
#ifndef DEBUG
#pragma unroll
#endif
      for (int x = 0; x < SS_X_TILES; x++)
      {

        //if (sx > SS_X_OVERLAP && sy > SS_Y_OVERLAP && sx < SS_X_NUM - SS_X_OVERLAP && sy < SS_Y_NUM - SS_Y_OVERLAP)

        //if ( pix >= 0 && pix < searchList.widths.val[0] )
        {
          //mypwer = powers[sy][sx];
          mypwer = powers_t[y][x];
          //int lmax = true;

          if (mypwer > powcut)
          {


            //if (lmax == true)
            {
              //float power =
              //float pow = powers[y][x];
              //float sig = candidate_sigma(pow, numharm, numindep);
              //float rr = (ffdot->rlo + jj * (double) ACCEL_DR) / (double) numharm;
              //float zz = (ffdot->zlo + ii * (double) ACCEL_DZ) / (double) numharm;
              //printf("Found a canidate! Power: %.3f  sig: %.4f \n",pow, sig);
              //          ff++;

              //while ( atomicCAS( &mutex, 0, 1 ) );

              //ff2++;
              //mutex = 0;
              //atomicExch( &mutex, 0 );

              //cnts[tid]++;
              //volatile
              //volatile
              //__syncthreads();
              //while (atomicCAS((unsigned long long int*) &sem, 0, tid + 1) != 0)
              //{
              //}

              //printf("Assuire sem %i \n", tid );

              //acquire_semaphore(&g_canSem);
              // candList[ff2].power =  mypwer;
              //candList[ff2].r = x;
              //candList[ff2].z = y;

              //__syncthreads();

              if (1)
              {
                if (0)
                {
                  volatile bool done = false;

                  //atomicAdd(&g_canCount, 1);
                  // Global synch and add
                  int cnt = 0;
                  while (!done)
                  {
                    volatile int prev = atomicCAS((unsigned long long int*) &g_canSem, SEMFREE, 1);
                    if (prev == SEMFREE )  // || cnt > 5000)
                    {
                      if (g_canCount < MAX_CANDS_PER_BLOCK)
                      {
                        int numharm = 1 << stage;
                        d_candList[g_canCount].power = mypwer;
                        //(ffdot->rlo + jj * (double) ACCEL_DR) / (double) numharm;
                        //zz = plains->h_candidates[i].z;
                        //(ffdot->zlo + ii * (double) ACCEL_DZ) / (double) numharm;

                        d_candList[g_canCount].r = (rlo + sx * (float) ACCEL_DR ) / (float) (numharm);
                        d_candList[g_canCount].z = (zlo + sy * (float) ACCEL_DZ ) / (float) (numharm);
                        d_candList[g_canCount].numharm = numharm;
                        g_canCount++;
                      }
                      else
                      {
                        printf("/rCUDA Candidate list to short, not recording candidate!  lost: %i candidates.\n", g_canCount-MAX_CANDS_PER_BLOCK);
                        //g_canCount++;
                      }
                      //printf("Found a canidate! Power: %.3f  %i  \n", mypwer, g_canCount);
                      //ff2++;
                      done = true;
                      g_canSem = SEMFREE;
                    }
                    //cnt++;
                    //__syncthreads();
                  }
                }
                else
                {
                  //int cnt = 0;
                  // Shared synch and add
                  //while (!done && cnt < 5000)
                  {
                    //volatile int prev = atomicCAS((unsigned long long int*) &sem, 0, 1);
                    //prev = 0;
                    //if (prev == 0)
                    //if ( (*count) < SS_X_TILES*SS_Y_TILES*5 )
                    if ( true )
                    {
                      //atomicAdd(s_count, 1);

                      sy = y * SS_Y + threadIdx.y;
                      sx = x * SS_X + threadIdx.x;

                      int piy = tly + sy;
                      int pix = tlx + sx;

                      int numharm = 1 << stage;
                      candListsm[*count].power = mypwer;
                      candListsm[*count].r = (ACCEL_RDR * (rlo + pix * (float) ACCEL_DR) / (float)numharm + 0.5) * ACCEL_DR;
                      //(rlo + pix * (float) ACCEL_DR) / (float) (numharm);
                      //return (int) (ACCEL_RDR * (rlo + pix * (float) ACCEL_DR) * harm_fract + 0.5) * ACCEL_DR;
                      candListsm[*count].z = (zlo + piy * (float) ACCEL_DZ) / (float) (numharm);
                      candListsm[*count].numharm = numharm;

                      (*count)++;

                      //printf("Found a candidate! Power: %.3f  \n", mypwer);

                      //ff2++;
                      //done = true;
                      //sem = 0;
                    }
                    //else
                    {
                      //printf("Well this is strange?\n");
                    }
                    //cnt++;
                    //__syncthreads();
                  }
                }
              }
              else
              {
                //atomicAdd((unsigned long long int*) &g_canSem, 1);
                //atomicAdd((unsigned long long int*) &sem, 1);
              }

              //__syncthreads();
              //atomicAdd(&ff, 1);
            }
            //else
            {
              //powers[sy][sx] = 0;
            }
          }
          else
          {
            //powers[sy][sx] = 0;
          }

        }
      }
    }
  }
  //if ( ff )
  //printf("Found %i canidates!\n", ff);

  //__syncthreads();

  // Collectively compute the block-wide exclusive prefix sum
  //BlockScan(temp_storage).ExclusiveSum(count, start);

  //__syncthreads();
  //__threadfence();

  //if (threadIdx.x == (SS_X-1) && threadIdx.y == (SS_Y-1) && start > 0 )
  //  printf("Found %i canidates!\n", start);

  //for( int i = 0; i < count; i++ )
  {
    //candList[start+i] = lCands[i];
  }
}
*/

/*
__device__ void sumPlainsSm(float powers[SS_Y_NUM][SS_X_NUM], cuFfdot10 others, int stage, float fRlow, float fZlow, int tlx, int tly)
{
  int tId = threadIdx.y * SS_X + threadIdx.x;       // Index in thread block

  const int threadsInB = SS_X * SS_Y;//
  int batch = ceilf(SS_X_NUM * SS_Y_NUM / (float) (threadsInB));

  int pix;// X location in powers data
  int piy;// Y location in powers data

  int sx;// X location in Shared memory
  int sy;// Y location in Shared memory

  int cmplxX;// X location in the complex data
  int cmplxY;// Y location in the complex data

  float lPwer = 0;// The power calculated
  fcomplexcu* cplxRow;// Row in complex data
  float frac;

  int start, end;

  if (stage > 0)
  {
    start = (1 << (stage - 1));
    end = (1 << stage) - 1;
  }

  //__syncthreads(); // Not necessary

  __shared__ int xInds[16][SS_X_NUM];
  int noBaches = ceilf(SS_X_NUM * start / (float) threadsInB);

  // Calculate xInicies and store in shared memory
  for (int b = 0; b < noBaches; b++)
  {
    int pos = b * threadsInB + tId;
    int harmListIdx = floorf(pos / (float) SS_X_NUM);
    int harm = harmListIdx + start;
    if (harm <= end)
    {
      frac = others.arr[harm].harmFraction;
      sx = pos - harmListIdx * SS_X_NUM;

      if (sx < SS_X_NUM)
      {
        pix = tlx + sx;
        float rr = fRlow + pix * ACCEL_DR;
        float subr = calc_required_r_gpu(frac, rr);
        int sRlow = (int) floorf(calc_required_r_gpu(frac, fRlow));
        int isx = index_from_r(subr, sRlow);
        xInds[harmListIdx][sx] = isx;
      }
    }
  }

  __syncthreads();

  for (int harm = start; harm <= end; harm++)
  {
    int harmListIdx = harm - start;
    int* yInds = &YINDS[others.arr[harm].inds];

    // Read data from fundamental into shared memory
#ifndef DEBUG
#pragma unroll
#endif
    for (int y = 0; y < SS_Y_TILES; y++)
    {
      sy = y * SS_Y + threadIdx.y;
      piy = tly + sy;

      if (piy >= 0 && piy < others.arr[0].ffdotHeight)
      {
        cmplxY = yInds[piy];                        // Y inex lookup from const memory
        cmplxY *= others.arr[harm].ffdotStride;// Stride down
        cplxRow = &others.arr[harm].ffdot[cmplxY + others.arr[harm].ffdBuffre];

#ifndef DEBUG
#pragma unroll
#endif
        for (int x = 0; x < SS_X_TILES; x++)
        {
          sx = x * SS_X + threadIdx.x;
          pix = tlx + sx;                           // Note I should chop off this value as it is "out of bounds" but for the moment I am including it.
          cmplxX = pix;
          cmplxX = xInds[harmListIdx][sx];
          //cmplxX += others.arr[harm].ffdBuffre;
          //cmplxX++;

          //int CStar = others.arr[harm].ffdBuffre;
          //int CEnd  = others.arr[harm].ffdotWidth - others.arr[harm].ffdBuffre;
          if (cmplxX >= 0 && cmplxX < others.arr[harm].ffdotWidth - 2*others.arr[harm].ffdBuffre)
          {
            //if ( cmplxX >= CStar && cmplxX < CEnd )
            {
              fcomplexcu cmp = cplxRow[cmplxX];
              lPwer = cmp.r * cmp.r + cmp.i * cmp.i;
              powers[sy][sx] += lPwer;
              //atomicAdd(&powers[sy][sx], lPwer);
            }
            //else
            {
              //printf("cmplxX out of bounds\n");
            }
          }
          else
          {
            //powers[sy][sx] = 0; // TMP
          }
        }
      }
      else
      {
        for (int x = 0; x < SS_X_TILES; x++)
        {
          //powers[sy][sx] = 0; // TMP
        }
      }
    }
  }

  __syncthreads();
}
*/

template<int stage>
__device__ static inline  void sumPlainsSmTex(float powers_t[SS_Y_TILES][SS_X_TILES], /*uint xInds[8][SS_X_NUM],*/const cuSearchList searchList, /*const primaryInf* pInf, const int stage,*/ int tlx, int tly, cand* candListsm, int *count)
{
  //int tId = threadIdx.y * SS_X + threadIdx.x;       // Index in thread block

  //const int threadsInB = SS_X * SS_Y;                       //
  //int batch = ceilf(SS_X_NUM * SS_Y_NUM / (float) (threadsInB));

  int pix;// X location in powers data
  int piy;// Y location in powers data

  int sx;// X location in Shared memory
  int sy;// Y location in Shared memory

  //int cmplxX;                                       // X location in the complex data
  //uint cmplxY;                                       // Y location in the complex data

  const float powcut = POWERCUT[stage];

  float lPwer = 0;// The power calculated
  //fcomplexcu* cplxRow;                              // Row in complex data
  //float frac2;

  //int start, end;

  //if (stage > 0)
  //{
  //    start = (1 << (stage - 1));
  //end = (1 << stage) - 1;
  //}

  //const int end = start + searchList.cnt.val[stage];

  //__syncthreads(); // Not nessesary

  //int start, end;

  //if ( stage > 0 )
  //{
  const int start = (1 << (stage - 1));
  const int end = (1 << stage) - 1;

  //const int tId = threadIdx.y * SS_X + threadIdx.x;       // Index in thread block
  //const int blkId = blockIdx.y*gridDim.x+blockIdx.x;

  //}

  /*
   __syncthreads(); //make sure no one is still using xInds

   //int noBaches = ceilf(SS_X_NUM * searchList.cnt.val[stage] / (float) (SS_X * SS_Y) );
   const int noBaches = ceilf(SS_X_NUM * (start) / (float) (SS_X * SS_Y));

   // Calculate xInicies and store in shared memory
   for (int b = 0; b < noBaches; b++)
   {
   int pos = b * (SS_X * SS_Y) + tId;
   int harmListIdx = floorf(pos / (float) SS_X_NUM);
   int harm = harmListIdx + start;
   if (harm <= end)
   {
   frac2 = searchList.frac.val[harm];
   sx = pos - harmListIdx * SS_X_NUM;

   if (sx >= 0 && sx < SS_X_NUM)
   {
   pix = tlx + sx;
   float rr = pInf->fRlow + pix * ACCEL_DR;
   float subr = calc_required_r_gpu(frac2, rr);
   int sRlow = (int) floorf(calc_required_r_gpu(frac2, pInf->fRlow));
   int isx = index_from_r(subr, sRlow);
   xInds[harmListIdx][sx] = isx;
   }
   }
   }
   __syncthreads(); // make sure xInds is complete
   */

  int xInds2[SS_X_TILES];
  //int yInds2[SS_Y_TILES];

#ifndef DEBUG
#pragma unroll
#endif
  for (int harm = start; harm <= end; harm++)
  {
    const int* yInds       = &YINDS[searchList.yInds.val[harm]];
    const fcomplexcu* data    = searchList.datas.val[harm];
    const float frac2         = searchList.frac.val[harm];

#ifndef DEBUG
#pragma unroll
#endif
    for (int x = 0; x < SS_X_TILES; x++)
    {
      sx = x * SS_X + threadIdx.x;
      if (sx >= 0 && sx < SS_X_NUM)
      {
        pix         = tlx + sx;
        //double rr    = searchList.searchRLow + pix * ACCEL_DR;
        //rr = searchList.rLow.val[0] + pix * ACCEL_DR;
        //double subr  = calc_required_r_gpu(frac2, rr);
        //subr = rr ;//* frac2   ; // ( (ACCEL_RDR) * rr * frac2 + 0.5) * (ACCEL_DR) ;
        //int isx     = index_from_r(subr, sRlow);
        //xInds2[x]   = isx ;
        //xInds2[x]   += searchList.ffdBuffre.val[harm];
        //xInds2[x]   = ((((( 2.0 * (searchList.searchRLow + pix * ACCEL_DR) * frac2 + 0.5) * ACCEL_DR) - sRlow) * 2.0 + 1e-3 ));
        //xInds2[x] = subr;
        //xInds2[x]   = (int)(pix*frac2+searchList.idxSum.val[harm]) + searchList.ffdBuffre.val[harm];
      }
    }

    /* // Block calculating tiles
     const int harmListIdx = harm - start;
     #pragma unroll SS_Y_TILES
     for (int y = 0; y < SS_Y_TILES; y++)
     {
     sy = y * SS_Y + threadIdx.y;
     piy = tly + sy;
     if (piy >= 0 && piy < pInf->height)
     {
     int zz = pInf->fZlow + piy * ACCEL_DZ;
     int subz = calc_required_z(frac2, zz);
     int zind = index_from_z(subz, searchList.zMax.val[harm]);
     yInds2[y] = zind;
     }
     }
     */

    // Read data from fundamental into shared memory
#ifndef DEBUG
#pragma unroll
#endif
    for (int y = 0; y < SS_Y_TILES; y++)
    {
      sy = y * SS_Y + threadIdx.y;
      piy = tly + sy;

      if (piy >= 0 && piy < searchList.heights.val[0])
      {
        //int cmplxY = yInds[piy];                        // Y index lookup from constant memory
        //cmplxY = yInds2[y];

        //if ( tId == 0 && blkId == 0 )
        //  printf("piy: %-2i  cmplxY: %-2i  %.2f\n", piy, cmplxY, frac2);

        /*
         int zz = pInf->fZlow + piy * ACCEL_DZ;
         int subz = calc_required_z(frac2, zz);
         int zind = index_from_z(subz, searchList.zMax.val[harm]);
         cmplxY = zind;
         */

        //uint add = piy + searchList.yInds.val[harm];
        //int std = searchList.strides.val[harm];
        //if (searchList.yInds.val[harm] + piy > MAX_YINDS )
        //{
        //  int xxvv = searchList.yInds.val[harm] + piy;
        //  printf("yind to large!\n", xxvv);
        //}
        //int sz2 = cmplxY + 0 /*searchList.ffdBuffre.val[harm]*/ ;
        //if (sz2 > searchList.strides.val[harm]*searchList.heights.val[harm] )
        //  printf("sz2 %i to large!\n", sz2);

        const fcomplexcu* cplxRow = &(data[yInds[piy]*searchList.strides.val[harm]]);
        const int cmplxY = yInds[piy]; //*searchList.strides.val[harm];// Stride down

        //--&searchList.datas.val[harm][cmplxY + searchList.ffdBuffre.val[harm]];      //+ searchList.ffdBuffre.val[harm] ];

        //printf("add: %i std: %i \n", add, std);

#ifndef DEBUG
#pragma unroll
#endif
        for (int x = 0; x < SS_X_TILES; x++)
        {
          sx = x * SS_X + threadIdx.x;
          pix = tlx + sx;                           // Note I should chop off this value as it is "out of bounds" but for the moment I am including it.
          //cmplxX = pix;
          //const int cmplxX = xInds[harmListIdx][sx];
          const int cmplxX = xInds2[x];

          if (pix >= 0 && pix < searchList.widths.val[0] /* && cmplxX < searchList.strides.val[harm] */ )
          {
            //kker = tex2D < float2 > (kerTex, tid, y);
            //fcomplexcu cmp = searchList.texs[harm] [cmplxX][cmplxY];

            //const float2 cmp = tex2D < float2 > (searchList.texs.val[harm], cmplxX, cmplxY);
            //lPwer = cmp.x * cmp.x + cmp.y * cmp.y;

            //fcomplexcu cmp = searchList.datas.val[harm][ cmplxY*searchList.strides.val[harm] + cmplxX ];
            //const fcomplexcu cmp {0,0}; // = cplxRow[cmplxX];

            fcomplexcu cmp = cplxRow[cmplxX];
            lPwer = cmp.r * cmp.r + cmp.i * cmp.i;

            /*
            if ( sy == 0 && sx ==0 && blkId == 0 )
              printf("Stage %i - [%.2f]\n", stage, frac2);
            if ( sy == 0 && sx < 10 && blkId == 0 )
              printf("%.4f ", lPwer);
            if ( sy == 0 && sx == 9 && blkId == 0 )
              printf("\n");
             */

            powers_t[y][x] += lPwer;
            //powers_t[y][x] = xInds2[x];

            if ( harm == end )
            {
              if ( powers_t[y][x] >= powcut  )
              {
                int numharm = 1 << stage;
                candListsm[*count].power = powers_t[y][x];
                candListsm[*count].r = ( searchList.rLow.val[0] + pix * (double)ACCEL_DR ) / (double)numharm ;
                candListsm[*count].z = ( piy*(double) ACCEL_DZ - searchList.zMax.val[0]  ) / (double)numharm ;
                candListsm[*count].numharm = numharm;

                (*count)++;

                //printf("Found a candidate! Power: %.3f  \n", powers_t[y][x] );
              }

              //powcut
              //printf("pow: %f \n", powers_t[y][x] );
            }

            //if ( powers_t[y][x] >= powcut )
            //  printf("We found value pow %f\n", powers_t[y][x]);


            //atomicAdd(&powers[sy][sx], lPwer);
          }
          //else
          {
            //powers[sy][sx] = 0; // TMP
          }
        }
      }
      //else
      {
        //for (int x = 0; x < SS_X_TILES; x++)
        {
          //powers[sy][sx] = 0; // TMP
        }
      }
    }
  }

  // */

  //__syncthreads();
}

/*
__global__ void add_and_searchCU(cuFfdot10 others, cand* d_cands, int noStages, float fRlow, float fZlow, int copyBack, int searchP)
{
  __shared__ float powers[SS_Y_NUM][SS_X_NUM];

  // Temporary memory for candidates
  cand candList[SS_X_TILES*SS_Y_TILES*5];
  int candCount = 0;

  int tlx = blockIdx.x * (SS_X_NUM - 2 * SS_X_OVERLAP) - SS_X_OVERLAP;// Top leaf of the block covered by this block
  int tly = blockIdx.y * (SS_Y_NUM - 2 * SS_Y_OVERLAP) - SS_Y_OVERLAP;// Top leaf of the block covered by this block

  int threadsInB = SS_X * SS_Y;//
  int batch = ceilf(SS_X_NUM * SS_Y_NUM / (float) (threadsInB));

  int pix;// X location in powers data
  int piy;// Y location in powers data

  int sx;// X location in Shared memory
  int sy;// Y location in Shared memory

  int cmplxX;// X location in the complex data
  //int cmplxY;                                       // Y location in the complex data

  float lPwer = 0;// The power calculated
  fcomplexcu* cplxRow;// Row in complex data

  // Read data from fundamental into shared memory
  if ( true )
  {
#ifndef DEBUG
#pragma unroll
#endif
    for (int y = 0; y < SS_Y_TILES; y++)
    {
      sy = y * SS_Y + threadIdx.y;
      piy = tly + sy;

      if (piy >= 0 && piy < others.arr[0].ffdotHeight)
      {
        piy *= (others.arr[0].ffdotStride);         // Stride down
        cplxRow = &others.arr[0].ffdot[piy];

#ifndef DEBUG
#pragma unroll
#endif
        for (int x = 0; x < SS_X_TILES; x++)
        {
          sx = x * SS_X + threadIdx.x;
          pix = tlx + sx;                           // Note I should chop off this value as it is "out of bounds" but for the moment I am including it.
          cmplxX = pix + others.arr[0].ffdBuffre;   // Shift the address by ffdBuffre
          lPwer = 0;

          if (cmplxX >= 0 && cmplxX < others.arr[0].ffdotWidth)
          {
            fcomplexcu cmp = cplxRow[cmplxX];
            lPwer = cmp.r * cmp.r + cmp.i * cmp.i;
          }
          powers[sy][sx] = lPwer;


        }
      }
      else
      {
        // Set to zero
        for (int x = 0; x < SS_X_TILES; x++)
        {
          sx = x * SS_X + threadIdx.x;
          powers[sy][sx] = 0;
        }
      }
    }
  }

  if (searchP)
  {
    search(powers, 0, d_cands, candList, &candCount, fRlow, fZlow);
  }

  if ( true )
  {
    for (int stage = 1; stage < noStages; stage++)
    {
      sumPlainsSm(powers, others, stage, fRlow, fZlow, tlx, tly);

      if (searchP)
      {
        search(powers, stage, d_cands, candList, &candCount, fRlow, fZlow);
      }
    }
  }


  //others.arr[0].ffdotPowers[tId] += 1;
  //__threadfence();
  //if (  threadIdx.y == 0 && threadIdx.x ==0 && blockIdx.x == 0 && blockIdx.y == 0 && g_canCount != 0 )
  //  printf("Found %i \n",g_canCount);
}
*/

/** Sum and Search - loop down - find column max
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base
 */
template<int noStages, int FLAGS>
__global__ void add_and_searchCU3(cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base)
{
  const int bidx  = threadIdx.y * SS3_X   +   threadIdx.x;
  const int tid   = blockIdx.x  * SS3_Y*SS3_X + bidx;

  if ( tid < searchList.widths.val[0] )
  {
    const int noHarms = (1 << (noStages - 1));
    int inds[noHarms];
    accelcandBasic candLists[noStages];

    int start   = 0;
    int end     = 0;

    FOLD // Prep
    {
      // Initialise the x indices of this thread
      inds[0] = tid + searchList.ffdBuffre.val[0];

      // Calculate the x indices
#pragma unroll
      for ( int i = 1; i < noHarms; i++ )
      {
        //inds[i]     = (int)(tid*searchList.frac.val[i]+searchList.idxSum.val[i]) + searchList.ffdBuffre.val[i];
      }

      // Set the local and return candidate powers to zero
#pragma unroll
      for ( int i = 0; i < noStages; i++ )
      {
        candLists[i].sigma = 0;

        if ( FLAGS & CU_CAND_SINGLE_G )
          d_cands[tid*noStages+i].sigma = 0;
      }
    }

    FOLD // Prep
    {
      if  ( noStages >= 1 )
      {
        const int nPowers = (noStages)*2;   // The number of powers to batch calculate together, *2 is a "random choice it would be this or noHarms
        float powers[nPowers];              // registers to hold values to increase mem cache hits

        int y;
        for( y = 0; y < searchList.heights.val[0]-nPowers ; y+= nPowers )
        {

#pragma unroll
          for( int i = 0; i < nPowers ; i++ )
          {
            powers[i] = 0;
          }

          // Loop over stages, sum and search
#pragma unroll
          for ( int stage = 0 ; stage < noStages; stage++)
          {
            if      ( stage == 0 )
            {
              start = 0;
              end = 1;
            }
            else if ( stage == 1 )
            {
              start = 1;
              end = 2;
            }
            else if ( stage == 2 )
            {
              start = 2;
              end = 4;
            }
            else if ( stage == 3 )
            {
              start = 4;
              end = 8;
            }
            else if ( stage == 4 )
            {
              start = 8;
              end = 16;
            }

            // Sum set of powers
#pragma unroll
            for ( int harm = start; harm < end; harm++ )
            {

#pragma unroll
              for( int i = 0; i < nPowers; i++ )
              {
                if  ( (FLAGS & FLAG_PLN_TEX ) )
                {
                  const float2 cmpf = tex2D < float2 > (searchList.texs.val[harm], inds[harm], YINDS[searchList.yInds.val[harm]+y+i]);
                  powers[i] += cmpf.x * cmpf.x + cmpf.y * cmpf.y;
                }
                else
                {
                  const fcomplexcu cmpc  = searchList.datas.val[harm][YINDS[searchList.yInds.val[harm]+y+i]*searchList.strides.val[harm]+inds[harm]];
                  powers[i] += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                }
              }

            }

            // Search set of powers
#pragma unroll
            for( int i = 0; i < nPowers ; i++ )
            {
              if  (  powers[i] >  POWERCUT[stage] )
              {
                //if ( lPwer > candLists[stage].sigma )
                if ( powers[i] > candLists[stage].sigma )
                {
                  candLists[stage].sigma  = powers[i];
                  candLists[stage].z      = y;
                }
              }
            }
          }
        }

        // one last loop with the if statement included
        {
          start   = 0;
          end     = 0;

#pragma unroll
          for( int i = 0; i < nPowers ; i++ )
          {
            powers[i] = 0;
          }

#pragma unroll
          for ( int stage = 0 ; stage < noStages; stage++)
          {
            if      ( stage == 0 )
            {
              start = 0;
              end = 1;
            }
            else if ( stage == 1 )
            {
              start = 1;
              end = 2;
            }
            else if ( stage == 2 )
            {
              start = 2;
              end = 4;
            }
            else if ( stage == 3 )
            {
              start = 4;
              end = 8;
            }
            else if ( stage == 4 )
            {
              start = 8;
              end = 16;
            }

#pragma unroll
            for ( int harm = start; harm < end; harm++ )
            {

#pragma unroll
              for( int i = 0; i < nPowers; i++ )
              {
                if ( i + y < searchList.heights.val[0])
                {
                  if  ( ( FLAGS & FLAG_PLN_TEX ) )
                  {
                    const float2 cmpf = tex2D < float2 > (searchList.texs.val[harm], inds[harm], YINDS[searchList.yInds.val[harm]+y+i]);
                    powers[i] += cmpf.x * cmpf.x + cmpf.y * cmpf.y;
                  }
                  else
                  {
                    const fcomplexcu cmpc  = searchList.datas.val[harm][YINDS[searchList.yInds.val[harm]+y+i]*searchList.strides.val[harm]+inds[harm]];
                    powers[i] += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                  }
                }
              }

            }

#pragma unroll
            for( int i = 0; i < nPowers ; i++ )
            {
              if  (  powers[i] >  POWERCUT[stage] )
              {
                //if ( lPwer > candLists[stage].sigma )
                if ( powers[i] > candLists[stage].sigma )
                {
                  candLists[stage].sigma  = powers[i];
                  candLists[stage].z      = y;
                }
              }
            }
          }
        }
      }
      else
      {
        float lPwer = 0;
        int y;
        for( y = 0; y < searchList.heights.val[0] ; y++ )
        {
          lPwer = 0;

#pragma unroll
          for ( int stage = 0 ; stage < noStages; stage++)
          {
            if      ( stage == 0 )
            {
              start = 0;
              end = 1;
            }
            else if ( stage == 1 )
            {
              start = 1;
              end = 2;
            }
            else if ( stage == 2 )
            {
              start = 2;
              end = 4;
            }
            else if ( stage == 3 )
            {
              start = 4;
              end = 8;
            }
            else if ( stage == 4 )
            {
              start = 8;
              end = 16;
            }

#pragma unroll
            for ( int harm = start; harm < end; harm++ )
            {

              //if ( noStages == -1 )
              {
                //cmpc  = searchList.datas.val[0][y*searchList.strides.val[0]+inds[0]];
                //lPwer += cmpc.r * cmpc.r + cmpc.i * cmpc.i;

                //cmpf = tex2D < float2 > (searchList.texs.val[harm], inds[harm], y);
                //lPwer += cmpf.x * cmpf.x + cmpf.y * cmpf.y;

                //const fcomplexcu cmpc  = searchList.datas.val[0][y*searchList.strides.val[0]+searchList.ffdBuffre.val[0] + tid];
                //const fcomplexcu cmpc =  baseaDDS[0][y*searchList.strides.val[0] + searchList.ffdBuffre.val[0] + tid ];
                //lPwer += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
              }
              /*
            else if ( searchList.frac.val[harm] > 0.7  )
            {
              cmpc  = searchList.datas.val[harm][YINDS[searchList.yInds.val[harm]+y]*searchList.strides.val[harm]+inds[harm]];
              lPwer += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
            }
               */
              //else //if ( harm == 1 )
              //else
              {


                //if ( FLAGS & FLAG_PLN_TEX )
                {
                  //const float2 cmpf = tex2D < float2 > (searchList.texs.val[harm], inds[harm], YINDS[searchList.yInds.val[harm]+y]);
                  //lPwer += cmpf.x * cmpf.x + cmpf.y * cmpf.y;
                }
                //else
                {
                  const fcomplexcu cmpc  = searchList.datas.val[harm][YINDS[searchList.yInds.val[harm]+y]*searchList.strides.val[harm]+inds[harm]];
                  //const fcomplexcu cmpc  = searchList.datas.val[0][YINDS[searchList.yInds.val[0]+y]*searchList.strides.val[0]+inds[0]];
                  //const fcomplexcu cmpc =  baseaDDS[harm][YINDS[searchList.yInds.val[harm]+y]*searchList.strides.val[harm]+inds[harm]];
                  //lPwer += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                  lPwer += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                }


                //int cmplxY = YINDS[searchList.yInds.val[harm]+y] ;

                /*
              if ( harm == 0)
                cmplxY =  y ;
                else
                  cmplxY =  YINDS[searchList.yInds.val[harm]+y] ;
                 */

                //const float2 cmpf = tex2D < float2 > (searchList.texs.val[harm], inds[harm], YINDS[searchList.yInds.val[harm]+y]);
                //const float2 cmpf = tex2D < float2 > (searchList.texs.val[0], inds[0], YINDS[searchList.yInds.val[0]+y]);
                //lPwer += cmpf.x * cmpf.x + cmpf.y * cmpf.y;
              }

              //lPwer += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
            }

            if  ( lPwer >  POWERCUT[stage] )
            {
              if ( lPwer > candLists[stage].sigma )
              {
                candLists[stage].sigma  = lPwer;
                candLists[stage].z      = y;
              }
            }
          }
        }
      }
    }

    // Write results back to DRAM and calculate sigma if needed
    if      ( FLAGS & CU_CAND_DEVICE   )
    {
#pragma unroll
      for ( int stage = 0 ; stage < noStages; stage++)
      {
        const short numharm = 1 << stage;

        if  ( candLists[stage].sigma >  POWERCUT[stage] )
        {
          int idx =  (int)(( searchList.rLow.val[0] + tid * (double) ACCEL_DR ) / (double)numharm ) - base ;
          if ( idx >= 0 )
          {
            long long numtrials         = NUMINDEP[stage];
            candLists[stage].numharm    = numharm;
            //candLists[stage].z          = ( candLists[stage].z*(double) ACCEL_DZ - searchList.zMax.val[0]  )  / (double)numharm ;
            candLists[stage].sigma      = (float)candidate_sigma_cu(candLists[stage].sigma, numharm, numtrials);

            FOLD // Atomic write to global list
            {
              volatile bool done = false;
              while (!done)
              {
                volatile int prev = atomicCAS(&d_sem[idx], UINT_MAX, tid );
                if ( prev == UINT_MAX )
                {
                  if ( candLists[stage].sigma > d_cands[idx].sigma )
                  {
                    d_cands[idx]              = candLists[stage];
                  }
                  d_sem[idx]                  = UINT_MAX;
                  done = true;
                }
              }
            }
          }
        }
      }
    }
    else if ( FLAGS & CU_CAND_SINGLE_G )
    {
#pragma unroll
      for ( int stage = 0 ; stage < noStages; stage++)
      {
        //if ( candLists[stage].sigma > 0 )
        if  ( candLists[stage].sigma >  POWERCUT[stage] )
        {
          const short numharm = ( 1 << stage );
          candLists[stage].numharm      = numharm;

          if ( FLAGS & FLAG_SAS_SIG )
          {
            // Calculate sigma value
            long long numtrials         = NUMINDEP[stage];
            candLists[stage].sigma      = (float)candidate_sigma_cu(candLists[stage].sigma, numharm, numtrials);
          }

          // Write to DRAM
          d_cands[tid*noStages + stage] = candLists[stage];
        }
      }
    }
  }
}

__global__ void print_YINDS2(int no)
{
  const int bidx  = threadIdx.y * SS3_X       +   threadIdx.x;
  const int tid   = blockIdx.x  * (SS3_Y*SS3_X) + bidx;

  if ( tid == 0 )
  {
    printf("%p\n", YINDS );

    for(int i = 0 ; i < no; i ++)
    {
      printf("%03i: %-5i  %i \n", i, YINDS[i], sizeof(int)*8 );
    }
    printf("\n");
  }
}

/** Sum and Search - loop down - column max - multi-step .
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base          Used in CU_CAND_DEVICE
 * @param noSteps
 */
#if TEMPLATE_SEARCH == 1
template<uint FLAGS, /*typename sType,*/ int noStages, typename stpType, int noSteps>
__global__ void add_and_searchCU31(cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base/*, sType pd*/, stpType rLows )
#else
template<uint FLAGS, /*typename sType,*/ int noStages, typename stpType>

__global__ void add_and_searchCU31(cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base/*, sType pd*/, stpType rLows, int noSteps )
#endif
{
  const int bidx  = threadIdx.y * SS3_X         +  threadIdx.x;
  const int tid   = blockIdx.x  * (SS3_Y*SS3_X) +  bidx;

  const int width = searchList.widths.val[0];

  if ( tid < width )
  {
    const int noHarms     = ( 1 << (noStages-1) ) ;
    const int nPowers     = 8 ; // (noStages)*2;      // The number of powers to batch calculate together, *2 is a "random choice it would be this or noHarms
    const int zeroHeight  = searchList.heights.val[0] ;

#if TEMPLATE_SEARCH == 1
    accelcandBasic candLists[noStages][noSteps];

    int         inds[noSteps][noHarms];
    fcomplexcu* pData[noSteps][noHarms];
    //float*      pPowr[noSteps][noHarms];
    float       powers[noSteps][nPowers];         // registers to hold values to increase mem cache hits
#else
    accelcandBasic candLists[noStages][MAX_STEPS];

    int         inds[MAX_STEPS][noHarms];
    fcomplexcu* pData[MAX_STEPS][noHarms];
    //float*      pPowr[MAX_STEPS][noHarms];
    float       powers[MAX_STEPS][nPowers];         // registers to hold values to increase mem cache hits
#endif

    int start   = 0;
    int end     = 0;
    int iy, ix;
    int y;

    FOLD // Prep - Initialise the x indices & set candidates to 0 .
    {
      // Calculate the x indices or create a pointer offset by the correct amount
#pragma unroll
      for ( int harm = 0; harm < noHarms; harm++ )      // loop over harmonic
      {
#if TEMPLATE_SEARCH == 1
#pragma unroll
#endif
        for ( int step = 0; step < noSteps; step++)     // Loop over steps
        {
          int drlo = (int) ( ACCEL_RDR * rLows.arry[step] * searchList.frac.val[harm] + 0.5 ) * ACCEL_DR ;
          float srlo = (int) ( ACCEL_RDR * ( rLows.arry[step] + tid * ACCEL_DR ) * searchList.frac.val[harm] + 0.5 ) * ACCEL_DR ;

          ix = (srlo - drlo) * ACCEL_RDR + searchList.ffdBuffre.val[harm] ;

          //float rLow    = rLows.arry[step] * searchList.frac.val[harm];
          //float diff    = rLow - (int)rLow;
          //float idxS    = 0.5f + diff*ACCEL_RDR ;
          //ix = (int)( tid * searchList.frac.val[harm] + idxS ) + searchList.ffdBuffre.val[harm];

          if     (FLAGS & FLAG_PLN_TEX)  // Calculate x index
          {
            inds[step][harm]      = ix;
          }
          else                           // Create a pointer list that is offset by the correct amount
          {
            inds[step][harm]      = ix;

            if        ( FLAGS & FLAG_STP_ROW )
            {
              if      ( FLAGS & FLAG_FFT_OUT )
              {
                //pPowr[step][harm]   = &searchList.powers.val[harm][ ix + searchList.strides.val[harm]*step ] ;
              }
              else
              {
                pData[step][harm]   = &searchList.datas.val[harm][  ix + searchList.strides.val[harm]*step ] ;
              }
            }
            else if   ( FLAGS & FLAG_STP_PLN )
            {
              if      ( FLAGS & FLAG_FFT_OUT )
              {
                //pPowr[step][harm]   = &searchList.powers.val[harm][ ix + searchList.strides.val[harm]*step*searchList.heights.val[harm] ] ;
              }
              else
              {
                pData[step][harm]   = &searchList.datas.val[harm][  ix + searchList.strides.val[harm]*step*searchList.heights.val[harm] ] ;
              }
            }
          }
        }

        // Change the stride for this harmonic
        if     ( FLAGS & FLAG_PLN_TEX )
        {
        }
        else
        {
          if        ( FLAGS & FLAG_STP_ROW )
          {
            if ( FLAGS & FLAG_FFT_OUT )
            {
              //searchList.strides.val[harm] *= noSteps;
            }
            else
            {
              searchList.strides.val[harm] *= noSteps;
            }
          }
        }
      }

      // Set the local and return candidate powers to zero
      FOLD
      {
#pragma unroll
        for ( int stage = 0; stage < noStages; stage++ )
        {
#if TEMPLATE_SEARCH == 1
#pragma unroll
#endif
          for ( int step = 0; step < noSteps; step++)   // Loop over steps

          {
            candLists[stage][step ].sigma = 0 ;

            if ( FLAGS & CU_CAND_SINGLE_G )
            {
              d_cands[step*noStages*width + stage*width + tid ].sigma = 0;
            }
          }
        }
      }
    }

    FOLD // Sum & Search
    {
      FOLD  // Loop over blocks of set length .
      {
        //for( y = 0; y < searchList.heights.val[0] - nPowers ; y += nPowers ) // loop over chunks .
        for( y = 0; y < searchList.heights.val[0] ; y += nPowers ) // loop over chunks .
        {
          // Initialise powers for each section column to 0
#if TEMPLATE_SEARCH == 1
#pragma unroll
#endif
          for ( int step = 0; step < noSteps; step++)       	    // Loop over steps .
          {
#pragma unroll
            for( int i = 0; i < nPowers ; i++ )                   // Loop over powers .
            {
              powers[step][i] = 0;
            }
          }

          // Loop over stages, sum and search
//#pragma unroll
          for ( int stage = 0 ; stage < noStages; stage++)        // Loop over stages .
          {

            if      ( stage == 0 )
            {
              start = 0;
              end = 1;
            }
            else if ( stage == 1 )
            {
              start = 1;
              end = 2;
            }
            else if ( stage == 2 )
            {
              start = 2;
              end = 4;
            }
            else if ( stage == 3 )
            {
              start = 4;
              end = 8;
            }
            else if ( stage == 4 )
            {
              start = 8;
              end = 16;
            }

            // Create a section of summed powers one for each step
//#pragma unroll
            for ( int harm = start; harm < end; harm++ )          // Loop over harmonics (plains) in this stage
            {

//#pragma unroll
              for( int yPlus = 0; yPlus < nPowers; yPlus++ )                // Loop over the chunk
              {
                int trm = y + yPlus ;

                iy            = YINDS[ searchList.yInds.val[harm] + trm ];


#if TEMPLATE_SEARCH == 1
//#pragma unroll
#endif
                for ( int step = 0; step < noSteps; step++)         // Loop over steps
                {
                  if     (FLAGS & FLAG_PLN_TEX)
                  {
                    // Calculate y indice
                    if      ( FLAGS & FLAG_STP_ROW )
                    {
                      iy  = ( iy * noSteps + step );
                    }
                    else if ( FLAGS & FLAG_STP_PLN )
                    {
                      iy  = ( iy + searchList.heights.val[harm]*step ) ;
                    }

                    const float2 cmpf         = tex2D < float2 > (searchList.texs.val[harm], inds[step][harm], iy);
                    powers[step][yPlus]      += cmpf.x * cmpf.x + cmpf.y * cmpf.y;
                  }
                  else
                  {
                    if ( FLAGS & FLAG_FFT_OUT )
                    {
                      float power;
                      if        ( FLAGS & FLAG_STP_ROW )
                      {
                        power = searchList.powers.val[harm][ (inds[step][harm]  + searchList.strides.val[harm]*noSteps*iy + searchList.strides.val[harm]*step) ] ;
                        //power = pPowr[step][harm][searchList.strides.val[harm]*iy] ; // Note stride has been set depending on multi-step type
                      }
                      else if   ( FLAGS & FLAG_STP_PLN )
                      {
                        power = searchList.powers.val[harm][ inds[step][harm]  + searchList.strides.val[harm]*iy + searchList.strides.val[harm]*step*searchList.heights.val[harm] ] ;
                      }
                      powers[step][yPlus]        += power;
                    }
                    else
                    {
                      fcomplexcu cmpc;
                      if        ( FLAGS & FLAG_STP_ROW )
                      {
                        //cmpc = searchList.datas.val[harm][ inds[step][harm]  + searchList.strides.val[harm]*noSteps*iy + searchList.strides.val[harm]*step ] ;
                        cmpc = pData[step][harm][searchList.strides.val[harm]*iy] ; // Note stride has been set depending on multi-step type
                      }
                      else if   ( FLAGS & FLAG_STP_PLN )
                      {
                        cmpc = searchList.datas.val[harm][ inds[step][harm]  + searchList.strides.val[harm]*iy + searchList.strides.val[harm]*step*searchList.heights.val[harm] ] ;
                      }

                      powers[step][yPlus]        += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                    }
                  }
                }
              }
            }

            // Search set of powers

#if TEMPLATE_SEARCH == 1
            //#pragma unroll
#endif
            for ( int step = 0; step < noSteps; step++)         	    // Loop over steps
            {
              //#pragma unroll
              for( int i = 0; i < nPowers ; i++ )                     // Loop over section
              {
                if  (  powers[step][i] > POWERCUT[stage] )
                {
                  if ( powers[step][i] > candLists[stage][step].sigma )
                  {
                    if ( y + i < zeroHeight )
                    {
                      // This is our new max!
                      candLists[stage][step].sigma  = powers[step][i];
                      candLists[stage][step].z      = y+i;
                    }
                  }
                }
              }
            }
          }
        }
      }
    }

    // Write results back to DRAM and calculate sigma if needed
    if      ( FLAGS & CU_CAND_DEVICE   )
    {
//#pragma unroll
      for ( int stage = 0 ; stage < noStages; stage++)
      {
        const short numharm = 1 << stage;

#if TEMPLATE_SEARCH == 1
#pragma unroll
#endif
        for ( int step = 0; step < noSteps; step++)         // Loop over steps
        {

          if  ( candLists[stage][step].sigma >  POWERCUT[stage] )
          {
            //float rLow    = rLows.arry[step] * searchList.frac.val[harm];
            //float diff    = rLow - (int)rLow;
            //float idxS    = 0.5  + diff*ACCEL_RDR ;

            int idx =  (int)(( rLows.arry[step] + tid * (double) ACCEL_DR ) / (double)numharm ) - base ;
            if ( idx >= 0 )
            {
              long long numtrials             = NUMINDEP[stage];
              candLists[stage][step].numharm  = numharm;
              //candLists[stage][step].z      = ( candLists[stage].z*(double) ACCEL_DZ - searchList.zMax.val[0]  )  / (double)numharm ;
              candLists[stage][step].sigma    = (float)candidate_sigma_cu(candLists[stage][step].sigma, numharm, numtrials);

              FOLD // Atomic write to global list
              {
                volatile bool done = false;
                while (!done)
                {
                  volatile int prev = atomicCAS(&d_sem[idx], UINT_MAX, tid );
                  if ( prev == UINT_MAX )
                  {
                    if ( candLists[stage][step].sigma > d_cands[idx].sigma )
                    {
                      d_cands[idx]              = candLists[stage][step];
                    }
                    d_sem[idx]                  = UINT_MAX;
                    done = true;
                  }
                }
              }
            }
          }
        }
      }
    }
    else if ( FLAGS & CU_CAND_SINGLE_G )
    {
#if TEMPLATE_SEARCH == 1
#pragma unroll
#endif
      for ( int step = 0; step < noSteps; step++)             // Loop over steps
      {
#pragma unroll
        for ( int stage = 0 ; stage < noStages; stage++)      // Loop over stages
        {

          if  ( candLists[stage][step].sigma >  POWERCUT[stage] )
          {
            //const short numharm                 = ( 1 << stage );
            //candLists[stage][step].numharm      = numharm;

            if ( FLAGS & FLAG_SAS_SIG && FALSE)
            {
              const short numharm               = ( 1 << stage );

              // Calculate sigma value
              long long numtrials               = NUMINDEP[stage];
              candLists[stage][step].sigma      = (float)candidate_sigma_cu(candLists[stage][step].sigma, numharm, numtrials);
            }

            // Write to DRAM
            d_cands[step*noStages*width + stage*width + tid] = candLists[stage][step];
          }
        }
      }
    }
  }
}

/** Sum and Search - loop down - column max - multi-step - step outer .
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base          Used in CU_CAND_DEVICE
 * @param noSteps
 */
#if TEMPLATE_SEARCH == 1
template<uint FLAGS, typename sType, int noStages, typename stpType, int noSteps>
__global__ void add_and_searchCU311(cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base, sType pd, stpType rLows )
#else
template<uint FLAGS, typename sType, int noStages, typename stpType>
__global__ void add_and_searchCU311(cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base, sType pd, stpType rLows, int noSteps )
#endif
{
  /*
  const int bidx  = threadIdx.y * SS3_X         +  threadIdx.x;
  const int tid   = blockIdx.x  * (SS3_Y*SS3_X) +  bidx;

  const int width = searchList.widths.val[0];

  if ( tid < width )
  {
    const int noHarms   = ( 1 << (noStages-1) ) ;
    const int nPowers   = 17 ; // (noStages)*2;      // The number of powers to batch calculate together, *2 is a "random choice it would be this or noHarms

    //register float power;
    float powers[nPowers];            // registers to hold values to increase mem cache hits

    const int zeroHeight = searchList.heights.val[0] ;

    int nStride[noHarms];

#if TEMPLATE_SEARCH == 1
    accelcandBasic candLists[noStages];
    //register float maxP[noStages];
    //int z[noStages];

    // One of the two variables below should get optimised out depending on FLAG_STP_ROW or FLAG_STP_PLN
    int inds[noHarms];
    fcomplexcu* pData[noHarms];
    //float powers[nPowers];         // registers to hold values to increase mem cache hits
#else
    accelcandBasic candLists[noStages];
    //float maxP[noStages];
    //int z[noStages];

    // One of the two variables below should get optimised out depending on FLAG_STP_ROW or FLAG_STP_PLN
    int inds[noHarms];
    fcomplexcu* pData[noHarms];
    //float powers[nPowers];         // registers to hold values to increase mem cache hits
#endif

//#if TEMPLATE_SEARCH == 1
//#pragma unroll
//#endif
    for ( int step = 0; step < noSteps; step++)     // Loop over steps
    {
      int start   = 0;
      int end     = 0;
      int iy;
      int y;

      FOLD // Prep - Initialise the x indices & set candidates to 0 .
      {
        int ix;

        // Calculate the x indices or create a pointer offset by the correct amount
#pragma unroll
        for ( int harm = 0; harm < noHarms; harm++ )      // loop over harmonic
        {
          float rLow    = rLows.arry[step] * searchList.frac.val[harm];
          float diff    = rLow - (int)rLow;
          float idxS    = 0.5f + diff*ACCEL_RDR ;

          ix = (int)( tid * searchList.frac.val[harm] + idxS ) + searchList.ffdBuffre.val[harm];
          if     (FLAGS & FLAG_PLN_TEX)  // Calculate x index
          {
            inds[harm]      = ix;
          }
          else                           // Create a pointer list that is offset by the correct amount
          {
            inds[harm]      = ix;

            if        ( FLAGS & FLAG_STP_ROW )
            {
              pData[harm]   = &searchList.datas.val[harm][ ix + searchList.strides.val[harm]*step ] ;
            }
            else if   ( FLAGS & FLAG_STP_PLN )
            {
              pData[harm]   = &searchList.datas.val[harm][ ix + searchList.strides.val[harm]*step*searchList.heights.val[harm] ] ;
            }
          }

          // Change the stride for this harmonic
          if     ( !( FLAGS & FLAG_PLN_TEX ) && ( FLAGS & FLAG_STP_ROW ) )
          {
            //searchList.strides.val[harm] *= noSteps;
            nStride[harm] = searchList.strides.val[harm] * noSteps;
          }
        }

        // Set the local and return candidate powers to zero
        FOLD
        {
#pragma unroll
          for ( int stage = 0; stage < noStages; stage++ )
          {
            candLists[stage].sigma    = POWERCUT[stage];
            //maxP[stage]               = POWERCUT[stage];

            if ( FLAGS & CU_CAND_SINGLE_G )
            {
              d_cands[step*noStages*width + stage*width + tid ].sigma = 0;
            }
          }
        }
      }

      FOLD // Sum & Search .
      {
        for( y = 0; y < zeroHeight ; y+=nPowers ) // loop over chunks  .
        {
          FOLD // Initialise powers for each section column to 0  .
          {
#pragma unroll
            for( int yPlus = 0; yPlus < nPowers; yPlus++ )                // Loop over the chunk
            {
              powers[yPlus] = 0;
            }
          }

          // Loop over stages, sum and search
#pragma unroll
          for ( int stage = 0 ; stage < noStages; stage++)        // Loop over stages .
          {
            if      ( stage == 0 )
            {
              start = 0;
              end = 1;
            }
            else if ( stage == 1 )
            {
              start = 1;
              end = 2;
            }
            else if ( stage == 2 )
            {
              start = 2;
              end = 4;
            }
            else if ( stage == 3 )
            {
              start = 4;
              end = 8;
            }
            else if ( stage == 4 )
            {
              start = 8;
              end = 16;
            }

            // Create a section of summed powers one for each step

#pragma unroll
            for ( int harm = start; harm < end; harm++ )          // Loop over harmonics (plains) in this stage
            {

#pragma unroll
              for( int yPlus = 0; yPlus < nPowers; yPlus++ )                // Loop over the chunk
              {
                int trm       = y + yPlus ;
                iy            = YINDS[ searchList.yInds.val[harm] + trm ] ;

                if     (FLAGS & FLAG_PLN_TEX)
                {
                  // Calculate y indice
                  if      ( FLAGS & FLAG_STP_ROW )
                  {
                    iy  = ( iy * noSteps + step );
                  }
                  else if ( FLAGS & FLAG_STP_PLN )
                  {
                    iy  = ( iy + searchList.heights.val[harm]*step ) ;
                  }

                  const float2 cmpf      = tex2D < float2 > (searchList.texs.val[harm], inds[harm], iy);
                  //power                 += cmpf.x * cmpf.x + cmpf.y * cmpf.y;
                  powers[yPlus]         += cmpf.x * cmpf.x + cmpf.y * cmpf.y;
                }
                else
                {
                  fcomplexcu cmpc;
                  if        ( FLAGS & FLAG_STP_ROW )
                  {
                    //cmpc = searchList.datas.val[harm][ inds[harm]  + searchList.strides.val[harm]*noSteps*iy + searchList.strides.val[harm]*step ] ;
                    cmpc = pData[harm][nStride[harm]*iy] ; // Note stride has been set depending on multi-step type
                  }
                  else if   ( FLAGS & FLAG_STP_PLN )
                  {
                    cmpc = searchList.datas.val[harm][ inds[harm]  + searchList.strides.val[harm]*iy + searchList.strides.val[harm]*step*searchList.heights.val[harm] ] ;
                  }
                  //power           += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                  powers[yPlus]   += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                }
              }
            }

            //#pragma unroll
            for( int yPlus = 0; yPlus < nPowers; yPlus++ )                // Loop over the chunk
            {
              if ( powers[yPlus] > candLists[stage].sigma )
              {
                if ( yPlus + y < zeroHeight)
                {
                  // This is our new max!
                  candLists[stage].sigma  = powers[yPlus];
                  candLists[stage].z      = y + yPlus;
                }
              }
            }

          }
        }
      }

      // Write results back to DRAM and calculate sigma if needed
      if      ( FLAGS & CU_CAND_DEVICE   )
      {
        //#pragma unroll
        for ( int stage = 0 ; stage < noStages; stage++)
        {
          const short numharm = 1 << stage;

//#if TEMPLATE_SEARCH == 1
//#pragma unroll
//#endif
//          for ( int step = 0; step < noSteps; step++)         // Loop over steps
          {

            if  ( candLists[stage].sigma >  POWERCUT[stage] )
            {
              //float rLow    = rLows.arry[step] * searchList.frac.val[harm];
              //float diff    = rLow - (int)rLow;
              //float idxS    = 0.5  + diff*ACCEL_RDR ;

              int idx =  (int)(( rLows.arry[step] + tid * (double) ACCEL_DR ) / (double)numharm ) - base ;
              if ( idx >= 0 )
              {
                long long numtrials             = NUMINDEP[stage];
                candLists[stage].numharm  = numharm;
                //candLists[stage][step].z      = ( candLists[stage].z*(double) ACCEL_DZ - searchList.zMax.val[0]  )  / (double)numharm ;
                candLists[stage].sigma    = (float)candidate_sigma_cu(candLists[stage].sigma, numharm, numtrials);

                FOLD // Atomic write to global list
                {
                  volatile bool done = false;
                  while (!done)
                  {
                    volatile int prev = atomicCAS(&d_sem[idx], UINT_MAX, tid );
                    if ( prev == UINT_MAX )
                    {
                      if ( candLists[stage].sigma > d_cands[idx].sigma )
                      {
                        d_cands[idx]              = candLists[stage];
                      }
                      d_sem[idx]                  = UINT_MAX;
                      done = true;
                    }
                  }
                }
              }
            }
          }
        }
      }
      else if ( FLAGS & CU_CAND_SINGLE_G )
      {
//#if TEMPLATE_SEARCH == 1
//#pragma unroll
//#endif
//        for ( int step = 0; step < noSteps; step++)             // Loop over steps
        {
#pragma unroll
          for ( int stage = 0 ; stage < noStages; stage++)      // Loop over stages
          {

            if  ( candLists[stage].sigma >  POWERCUT[stage] )
            {
              const short numharm                 = ( 1 << stage );
              candLists[stage].numharm      = numharm;

              if ( FLAGS & FLAG_SAS_SIG && FALSE)
              {
                // Calculate sigma value
                long long numtrials               = NUMINDEP[stage];
                candLists[stage].sigma      = (float)candidate_sigma_cu(candLists[stage].sigma, numharm, numtrials);
              }

              // Write to DRAM
              d_cands[step*noStages*width + stage*width + tid] = candLists[stage];
            }
          }
        }
      }
    }
  }
*/
}

/** Sum and Search - loop down - column max - multi-step - shared memory .
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base          Used in CU_CAND_DEVICE
 * @param noSteps
 */
#if TEMPLATE_SEARCH == 1
template<uint FLAGS, typename sType, int noStages, typename stpType, int noSteps>
__global__ void add_and_searchCU32(cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base, sType pd, stpType rLows )
#else
template<uint FLAGS, typename sType, int noStages, typename stpType>
__global__ void add_and_searchCU32(cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base, sType pd, stpType rLows, int noSteps )
#endif
{
  /*
  const int bid   = threadIdx.y * SS3_X         +  threadIdx.x;
  const int tid   = blockIdx.x  * (SS3_Y*SS3_X) +  bid;
  const int width = searchList.widths.val[0];

  if ( tid < width )
  {
    const int noHarms   = ( 1 << (noStages-1) ) ;
    const int hlfHarms  = noHarms / 2.0 ;
    const int nPowers   = hlfHarms ;

    accelcandBasic candLists[noStages];

    // One of the two variables below should get optimised out depending on FLAG_STP_ROW or FLAG_STP_PLN
    int inds[noHarms];
    //fcomplexcu* pData[noHarms];
    float powers[nPowers];         // registers to hold values to increase mem cache hits

    __shared__ float smPowers[hlfHarms][hlfHarms][SS3_Y*SS3_X];  //

    int start   = 0;
    int end     = 0;
    int iy;
    int ix;
    int y;
    const int zeroHeight = searchList.heights.val[0] ;

    for ( int step = 0; step < noSteps; step++)     // Loop over steps
    {
      FOLD // Prep - Initialise the x indices & set candidates to 0 .
      {
        // Calculate the x indices or create a pointer offset by the correct amount
#pragma unroll
        for ( int harm = 0; harm < noHarms; harm++ )      // loop over harmonic
        {
          float rLow    = rLows.arry[step] * searchList.frac.val[harm];
          float diff    = rLow - (int)rLow;
          float idxS    = 0.5f + diff*ACCEL_RDR ;

          ix = (int)( tid * searchList.frac.val[harm] + idxS ) + searchList.ffdBuffre.val[harm];
          if     (FLAGS & FLAG_PLN_TEX)  // Calculate x index
          {
            inds[harm]      = ix;
          }
          else                           // Create a pointer list that is offset by the correct amount
          {
            inds[harm]      = ix;
          }

        }

        // Set the local and return candidate powers to zero
        FOLD
        {
          //#if TEMPLATE_SEARCH == 1
          //#pragma unroll
          //#endif
          //for ( int step = 0; step < noSteps; step++)   // Loop over steps
          {
#pragma unroll
            for ( int stage = 0; stage < noStages; stage++ )
            {
              candLists[stage].sigma = 0;

              if ( FLAGS & CU_CAND_SINGLE_G )
              {
                d_cands[step*noStages*width + stage*width + tid ].sigma = 0;
              }
            }
          }
        }
      }

      FOLD // Sum & Search
      {
        FOLD  // Loop over blocks of set length .
        {
          for( y = 0; y < searchList.heights.val[0] ; y += nPowers )  // loop over chunks .
          {
            // Loop over stages, sum and search
#pragma unroll
            for ( int stage = 0 ; stage < noStages; stage++)          // Loop over stages .
            {
              if      ( stage == 0 )
              {
                start = 0;
                end = 1;
              }
              else if ( stage == 1 )
              {
                start = 1;
                end = 2;
              }
              else if ( stage == 2 )
              {
                start = 2;
                end = 4;
              }
              else if ( stage == 3 )
              {
                start = 4;
                end = 8;
              }
              else if ( stage == 4 )
              {
                start = 8;
                end = 16;
              }

              FOLD // Read summed powers into shared memory
              {
#pragma unroll
                for ( int harm = start; harm < end; harm++ )            // Loop over harmonics (plains) in this stage
                {
                  int hi = harm - start;

                  int startY, endY;

                  startY        = YINDS[ searchList.yInds.val[harm] + y ];
                  endY          = YINDS[ searchList.yInds.val[harm] + y + nPowers - 1 ];
                  int yDist     = endY -  startY ;

                  //for (int yy = startY ; yy <= endY; yy++ )
                  for (int yd = 0 ; yd < yDist; yd++ )
                  {
                    if     (FLAGS & FLAG_PLN_TEX)
                    {
                      // Calculate y indice
                      if      ( FLAGS & FLAG_STP_ROW )
                      {
                        iy  = ( yy * noSteps + step );
                      }
                      else if ( FLAGS & FLAG_STP_PLN )
                      {
                        iy  = ( yy + searchList.heights.val[harm]*step ) ;
                      }

                      const float2 cmpf       = tex2D < float2 > (searchList.texs.val[harm], inds[harm], iy);
                      powers[yy-startY]     += cmpf.x * cmpf.x + cmpf.y * cmpf.y;
                    }
                    else
                    {
                      fcomplexcu cmpc;
                      if        ( FLAGS & FLAG_STP_ROW )
                      {
                        cmpc = searchList.datas.val[harm][ inds[harm]  + searchList.strides.val[harm]*((yd+startY)*noSteps + step) ] ;
                        //cmpc = pData[harm][searchList.strides.val[harm]*noSteps*yy] ; // Note stride has been set depending on multi-step type
                      }
                      else if   ( FLAGS & FLAG_STP_PLN )
                      {
                        //cmpc = searchList.datas.val[harm][ inds[harm]  + searchList.strides.val[harm]*yy + searchList.strides.val[harm]*step*searchList.heights.val[harm] ] ;
                      }

                      //if      ( stage == 0 )  // Fundamental Harmonic
                      {
                        powers[yd]               = cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                      }
                      //else                    // Other Harmonics
                      {
                        //smPowers[hi][yd][bid]    = cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                      }
                    }
                  }
                }
              }

              if ( stage != 0 ) // Create summed powers for this stage
              {
                for ( int harm = start; harm < end; harm++ )          // Loop over harmonics (plains) in this stage
                {
                  int startY        = YINDS[ searchList.yInds.val[harm] + y ];

                  for( int yPlus = 0; yPlus < nPowers; yPlus++ )      // Loop over the chunk
                  {
                    int trm = y + yPlus ;

                    if ( trm < zeroHeight )
                    {
                      iy            = YINDS[ searchList.yInds.val[harm] + trm ];

                      int sy = iy - startY;

                      if ( sy >= 0 && sy < hlfHarms && harm-start < hlfHarms  && bid < SS3_Y*SS3_X )
                      {
                        //printf("yPlus %i harm: %i   sy: %i   bid: %i  \n",yPlus, harm-start, sy, bid );

                        //powers[yPlus] += smPowers[harm-start][sy][bid];
                      }
                      else
                      {
                        //printf("Error %i\n",tid);
                        //printf("Error: yPlus %i harm: %i   sy: %i   bid: %i  \n",yPlus, harm-start, sy, bid );
                      }
                    }
                    else
                    {
                      //printf("Error\n");
                    }
                  }
                }
              }

              // Search set of powers
              for( int i = 0; i < nPowers ; i++ )                     // Loop over section
              {
                if  (  powers[i] > POWERCUT[stage] )
                {
                  if ( powers[i] > candLists[stage].sigma )
                  {
                    if ( y + i < zeroHeight )
                    {
                      // This is our new max!
                      candLists[stage].sigma  = powers[i];
                      candLists[stage].z      = y+i;
                    }
                  }
                }
              }

            }
          }
        }
      }

      // Write results back to DRAM and calculate sigma if needed
      if      ( FLAGS & CU_CAND_DEVICE   )
      {
        //#pragma unroll
        for ( int stage = 0 ; stage < noStages; stage++)
        {
          const short numharm = 1 << stage;

          //#if TEMPLATE_SEARCH == 1
          //#pragma unroll
          //#endif
          //          for ( int step = 0; step < noSteps; step++)         // Loop over steps
          {

            if  ( candLists[stage].sigma >  POWERCUT[stage] )
            {
              //float rLow    = rLows.arry[step] * searchList.frac.val[harm];
              //float diff    = rLow - (int)rLow;
              //float idxS    = 0.5  + diff*ACCEL_RDR ;

              int idx =  (int)(( rLows.arry[step] + tid * (double) ACCEL_DR ) / (double)numharm ) - base ;
              if ( idx >= 0 )
              {
                long long numtrials             = NUMINDEP[stage];
                candLists[stage].numharm  = numharm;
                //candLists[stage][step].z      = ( candLists[stage].z*(double) ACCEL_DZ - searchList.zMax.val[0]  )  / (double)numharm ;
                candLists[stage].sigma    = (float)candidate_sigma_cu(candLists[stage].sigma, numharm, numtrials);

                FOLD // Atomic write to global list
                {
                  volatile bool done = false;
                  while (!done)
                  {
                    volatile int prev = atomicCAS(&d_sem[idx], UINT_MAX, tid );
                    if ( prev == UINT_MAX )
                    {
                      if ( candLists[stage].sigma > d_cands[idx].sigma )
                      {
                        d_cands[idx]              = candLists[stage];
                      }
                      d_sem[idx]                  = UINT_MAX;
                      done = true;
                    }
                  }
                }
              }
            }
          }
        }
      }
      else if ( FLAGS & CU_CAND_SINGLE_G )
      {
        //#if TEMPLATE_SEARCH == 1
        //#pragma unroll
        //#endif
        //        for ( int step = 0; step < noSteps; step++)             // Loop over steps
        {
#pragma unroll
          for ( int stage = 0 ; stage < noStages; stage++)      // Loop over stages
          {

            if  ( candLists[stage].sigma >  POWERCUT[stage] )
            {
              const short numharm                 = ( 1 << stage );
              candLists[stage].numharm      = numharm;

              if ( FLAGS & FLAG_SAS_SIG && FALSE)
              {
                // Calculate sigma value
                long long numtrials               = NUMINDEP[stage];
                candLists[stage].sigma      = (float)candidate_sigma_cu(candLists[stage].sigma, numharm, numtrials);
              }

              // Write to DRAM
              d_cands[step*noStages*width + stage*width + tid] = candLists[stage];
            }
          }
        }
      }
    }
  }
  */
}

template<uint FLAGS, /*typename sType,*/ uint noStages>
__host__ void add_and_searchCU31_s(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t cnvlStream,cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base, /*sType pd,*/ float* rLows, int noSteps)
{
#if TEMPLATE_SEARCH == 1
  switch (noSteps)
  {
    case 1:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU31<FLAGS),sType,noStages,f01,1>, hipFuncCachePreferL1);
      f01 tmpArr;
      for (int i = 0; i < noSteps; i++)
        tmpArr.arry[i] = rLows[i];
      add_and_searchCU31<FLAGS,/*sType,*/noStages,f01,1><<<dimGrid,  dimBlock, i1, cnvlStream >>>(searchList, d_cands, d_sem, base, /*pd,*/ tmpArr);
      break;
    }
    case 2:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU31<FLAGS),sType,noStages,f02,2>, hipFuncCachePreferL1);
      f02 tmpArr;
      for (int i = 0; i < noSteps; i++)
        tmpArr.arry[i] = rLows[i];
      add_and_searchCU31<FLAGS,/*sType,*/noStages,f02,2><<<dimGrid,  dimBlock, i1, cnvlStream >>>(searchList, d_cands, d_sem, base, /*pd,*/ tmpArr);
      break;
    }
    case 3:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU31<FLAGS),sType,noStages,f03,3>, hipFuncCachePreferL1);
      f03 tmpArr;
      for (int i = 0; i < noSteps; i++)
        tmpArr.arry[i] = rLows[i];
      add_and_searchCU31<FLAGS,/*sType,*/noStages,f03,3><<<dimGrid,  dimBlock, i1, cnvlStream >>>(searchList, d_cands, d_sem, base, /*pd,*/ tmpArr);
      break;
    }
    case 4:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU31<FLAGS),sType,noStages,f04,4>, hipFuncCachePreferL1);
      f04 tmpArr;
      for (int i = 0; i < noSteps; i++)
        tmpArr.arry[i] = rLows[i];
      add_and_searchCU31<FLAGS,/*sType,*/noStages,f04,4><<<dimGrid,  dimBlock, i1, cnvlStream >>>(searchList, d_cands, d_sem, base, /*pd,*/ tmpArr);
      break;
    }
    case 5:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU31<FLAGS),sType,noStages,f05,5>, hipFuncCachePreferL1);
      f05 tmpArr;
      for (int i = 0; i < noSteps; i++)
        tmpArr.arry[i] = rLows[i];
      add_and_searchCU31<FLAGS,/*sType,*/noStages,f05,5><<<dimGrid,  dimBlock, i1, cnvlStream >>>(searchList, d_cands, d_sem, base, /*pd,*/ tmpArr);
      break;
    }
    case 6:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU31<FLAGS),sType,noStages,f06,6>, hipFuncCachePreferL1);
      f06 tmpArr;
      for (int i = 0; i < noSteps; i++)
        tmpArr.arry[i] = rLows[i];
      add_and_searchCU31<FLAGS,/*sType,*/noStages,f06,6><<<dimGrid,  dimBlock, i1, cnvlStream >>>(searchList, d_cands, d_sem, base, /*pd,*/ tmpArr);
      break;
    }
    case 7:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU31<FLAGS),sType,noStages,f07,7>, hipFuncCachePreferL1);
      f07 tmpArr;
      for (int i = 0; i < noSteps; i++)
        tmpArr.arry[i] = rLows[i];
      add_and_searchCU31<FLAGS,/*sType,*/noStages,f07,7><<<dimGrid,  dimBlock, i1, cnvlStream >>>(searchList, d_cands, d_sem, base, /*pd,*/ tmpArr);
      break;
    }
    case 8:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU31<FLAGS),sType,noStages,f08,8>, hipFuncCachePreferL1);
      f08 tmpArr;
      for (int i = 0; i < noSteps; i++)
        tmpArr.arry[i] = rLows[i];
      add_and_searchCU31<FLAGS,/*sType,*/noStages,f08,8><<<dimGrid,  dimBlock, i1, cnvlStream >>>(searchList, d_cands, d_sem, base, /*pd,*/ tmpArr);
      break;
    }
    default:
      fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for %i steps\n", noSteps);
      exit(EXIT_FAILURE);
  }
#else
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU31<FLAGS),sType,noStages,fMax>, hipFuncCachePreferL1);
  fMax tmpArr;
  for (int i = 0; i < noSteps; i++)
    tmpArr.arry[i] = rLows[i];

  add_and_searchCU31<FLAGS,/*sType,*/noStages,fMax> <<<dimGrid, dimBlock, i1, cnvlStream>>>(searchList, d_cands, d_sem, base, /*pd,*/ tmpArr, noSteps);
#endif
}

template<uint FLAGS >
__host__ void add_and_searchCU31_p(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t cnvlStream,cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base, cuSearchItem* pd, float* rLows, int noSteps, const uint noStages )
{
  switch (noStages)
  {
    case 1:
    {
      add_and_searchCU31_s<FLAGS,/*sch1,*/1> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, /*tmpArr,*/ rLows, noSteps );
      break;
    }
    case 2:
    {
      add_and_searchCU31_s<FLAGS,/*sch2,*/2> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, /*tmpArr,*/ rLows, noSteps );
      break;
    }
    case 3:
    {
      add_and_searchCU31_s<FLAGS,/*sch4,*/3> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, /*tmpArr,*/ rLows, noSteps );
      break;
    }
    case 4:
    {
      add_and_searchCU31_s<FLAGS,/*sch8,*/4> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, /*tmpArr,*/ rLows, noSteps );
      break;
    }
    case 5:
    {
      add_and_searchCU31_s<FLAGS,/*sch16,*/5> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, /*tmpArr,*/ rLows, noSteps );
      break;
    }
    default:
      fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for %i stages\n", noStages);
      exit(EXIT_FAILURE);
  }
}

__host__ void add_and_searchCU31_f(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t cnvlStream,cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base, cuSearchItem* pd, float* rLows, int noSteps, const uint noStages, uint FLAGS )
{
  if        ( FLAGS & FLAG_FFT_OUT )
  {
    if      ( FLAGS & FLAG_STP_ROW )
      add_and_searchCU31_p<FLAG_FFT_OUT | CU_CAND_SINGLE_G | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, pd, rLows, noSteps, noStages );
    //else if ( FLAGS & FLAG_STP_PLN )
    //  add_and_searchCU31_p<FLAG_FFT_OUT | CU_CAND_SINGLE_G | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, pd, rLows, noSteps, noStages );
    //else if ( FLAGS & FLAG_STP_STK )
    //  add_and_searchCU31_p<FLAG_FFT_OUT | CU_CAND_SINGLE_G | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, pd, rLows, noSteps, noStages );
    else
    {
      fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination. \n" );
      exit(EXIT_FAILURE);
    }
  }
  else
  {
    if      ( FLAGS & FLAG_STP_ROW )
      add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, pd, rLows, noSteps, noStages );
    //else if ( FLAGS & FLAG_STP_PLN )
    //  add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, pd, rLows, noSteps, noStages );
    //else if ( FLAGS & FLAG_STP_STK )
    //  add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, pd, rLows, noSteps, noStages );
    else
    {
      fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination. \n" );
      exit(EXIT_FAILURE);
    }
  }

// Uncomenting this block will make compile time VERY long! I mean days!
/*
  if( FLAGS & CU_CAND_DEVICE )
  {
    if( FLAGS & FLAG_PLN_TEX )
    {
      if ( FLAGS & FLAG_SAS_SIG )
      {
        if      ( FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_PLN_TEX | FLAG_SAS_SIG | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_PLN_TEX | FLAG_SAS_SIG | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        //else if ( FLAGS & FLAG_STP_STK )
        //  add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_PLN_TEX | FLAG_SAS_SIG | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination.\n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
      else
      {
        if      ( FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_PLN_TEX | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_PLN_TEX | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        //else if ( FLAGS & FLAG_STP_STK )
        //  add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_PLN_TEX | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination. \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
    }
    else
    {
      if ( FLAGS & FLAG_SAS_SIG )
      {
        if      ( FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_SAS_SIG | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_SAS_SIG | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        //else if ( FLAGS & FLAG_STP_STK )
        //  add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_SAS_SIG | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination. \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
      else
      {
        if      ( FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        //else if ( FLAGS & FLAG_STP_STK )
        //  add_and_searchCU31_p<CU_CAND_DEVICE | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination. \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
    }
  }
  else if ( (FLAGS & CU_CAND_SINGLE_G) || (FLAGS & CU_CAND_HOST) )
  {
    if( FLAGS & FLAG_PLN_TEX )
    {
      if ( FLAGS & FLAG_SAS_SIG )
      {
        if      ( FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_PLN_TEX | FLAG_SAS_SIG | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_PLN_TEX | FLAG_SAS_SIG | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        //else if ( FLAGS & FLAG_STP_STK )
        //  add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_PLN_TEX | FLAG_SAS_SIG | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination.  \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
      else
      {
        if      ( FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_PLN_TEX | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_PLN_TEX | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        //else if ( FLAGS & FLAG_STP_STK )
        //  add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_PLN_TEX | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination. \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
    }
    else
    {
      if ( FLAGS & FLAG_SAS_SIG )
      {
        if      ( FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_SAS_SIG | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_SAS_SIG | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        //else if ( FLAGS & FLAG_STP_STK )
        //  add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_SAS_SIG | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination.  \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
      else
      {
        if      ( FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        //else if ( FLAGS & FLAG_STP_STK )
        //  add_and_searchCU31_p<CU_CAND_SINGLE_G | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination. \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
    }
  }
  else if (  FLAGS & CU_CAND_SINGLE_C )
  {
    if( FLAGS & FLAG_PLN_TEX )
    {
      if ( FLAGS & FLAG_SAS_SIG )
      {
        if      (FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_PLN_TEX | FLAG_SAS_SIG | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_PLN_TEX | FLAG_SAS_SIG | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_STK )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_PLN_TEX | FLAG_SAS_SIG | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination.  \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
      else
      {
        if      (FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_PLN_TEX | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_PLN_TEX | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_STK )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_PLN_TEX | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination. \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
    }
    else
    {
      if ( FLAGS & FLAG_SAS_SIG )
      {
        if      (FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_SAS_SIG | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_SAS_SIG | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_STK )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_SAS_SIG | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination.  \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
      else
      {
        if      (FLAGS & FLAG_STP_ROW )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_STP_ROW> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_PLN )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_STP_PLN> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else if ( FLAGS & FLAG_STP_STK )
          add_and_searchCU31_p<CU_CAND_SINGLE_C | FLAG_STP_STK> (dimGrid, dimBlock, i1, cnvlStream, searchList, d_cands, d_sem, base, noSteps, noPlns );
        else
        {
          fprintf(stderr, "ERROR: add_and_searchCU31 has not been templated for flag combination. \n", noPlns);
          exit(EXIT_FAILURE);
        }
      }
    }
  }
*/
}

/** Sum and Search - loop down - column max - use blocks .
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base
 */
template<int noStages, int canMethoud>
__global__ void add_and_searchCU4(cuSearchList searchList, accelcandBasic* d_cands, uint* d_sem, int base)
{
  const int x   = threadIdx.x;
  const int y   = threadIdx.y;
  const int gx  = blockIdx.x*SS4_X + x;

  if ( gx < searchList.widths.val[0] )
  {
    int batches = searchList.heights.val[0] / (float)( SS4_Y );

    const int noHarms = (1 << (noStages - 1));
    int inds[noHarms];
    int start, end;

    float powerThread[noStages];
    int   z[noStages];

    for ( int stage = 0; stage < noStages; stage++ )
    {
      powerThread[stage]  = 0;
      z[stage]            = 0;
    }

    // Initialise the x indices of this thread
    inds[0] = gx + searchList.ffdBuffre.val[0];

    // Calculate the x indices
#pragma unroll
    for ( int i = 1; i < noHarms; i++ )
    {
      //inds[i]     = (int)(gx*searchList.frac.val[i]+searchList.idxSum.val[i]) + searchList.ffdBuffre.val[i];
    }

    for ( int b = 0;  b < batches;  b++)  // Loop over blocks
    {
      float blockPower = 0;
      int by = b*SS4_Y + y;

#pragma unroll
      for ( int stage = 0; stage < noStages; stage++ ) // Loop over harmonic stages
      {
        if      ( stage == 0 )
        {
          start = 0;
          end = 1;
        }
        else if ( stage == 1 )
        {
          start = 1;
          end = 2;
        }
        else if ( stage == 2 )
        {
          start = 2;
          end = 4;
        }
        else if ( stage == 3 )
        {
          start = 4;
          end = 8;
        }
        else if ( stage == 4 )
        {
          start = 8;
          end = 16;
        }

        // Sum set of powers
#pragma unroll
        for ( int harm = start; harm < end; harm++ ) // Loop over sub harmonics
        {
          if  ( (canMethoud & FLAG_PLN_TEX ) )
          {
            const float2 cmpf = tex2D < float2 > (searchList.texs.val[harm], inds[harm], YINDS[searchList.yInds.val[harm]+by]);
            blockPower += cmpf.x * cmpf.x + cmpf.y * cmpf.y;
          }
          else
          {
            const fcomplexcu cmpc  = searchList.datas.val[harm][YINDS[searchList.yInds.val[harm]+by]*searchList.strides.val[harm]+inds[harm]];
            blockPower += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
          }
        }

        if  (  blockPower >  POWERCUT[stage] )
        {
          if ( blockPower > powerThread[stage] )
          {
            powerThread[stage]  = blockPower;
            z[stage]            = b;
          }
        }
      }
    }

#pragma unroll
    for ( int stage = 0; stage < 1; stage++ ) // Loop over harmonic stages
    {
      accelcandBasic can;
      long long numtrials         = NUMINDEP[stage];
      const short numharm = 1 << stage;

      if  ( powerThread[stage] >  POWERCUT[stage] )
      {
        if ( canMethoud & CU_CAND_SINGLE_G )
        {
          can.numharm = numharm;
          can.sigma   = powerThread[0];
          can.z       = z[0];
          if ( canMethoud & FLAG_SAS_SIG )
          {
            // Calculate sigma value
            can.sigma   = (float)candidate_sigma_cu(powerThread[0], numharm, numtrials);
          }

          // Write to DRAM
          d_cands[ searchList.widths.val[0]*stage*y +  stage*searchList.widths.val[0] + gx ] = can;
        }
      }
    }
  }

  /*

  __shared__ float s_powers[noStages][SS4_Y][SS4_X];
  __shared__ uint  s_z[noStages][SS4_Y][SS4_X];
  __shared__ int sum[noStages];

  if (x < noStages && y == 0)
  {
    sum[x] = 0;
  }

  // __syncthreads();

  // Write all results to shard memory
  for ( int s = 0 ; s <  noStages; s++)
  {
    if (powerThread[s] > 0 )
    {
      s_powers[s][y][x]  = powerThread[s];
      s_z[s][y][x]       = z[s] ; // *SS4_Y+y;
      atomicAdd(&sum[s], 1);
    }
  }

  __syncthreads();

  // Write back to DRAM
  if ( y < noStages && sum[y] > 0 )
  {
    z[0] = 0;
    powerThread[0] = 0;
    int stage = y;

    for ( int by = 0 ; by < SS4_Y; by++ )
    {
      if( s_powers[stage][by][x] > powerThread[0] )
      {
        powerThread[0]  = s_powers[stage][by][x];
        z[0]            = s_z[stage][by][x]*SS4_Y + by;
      }
    }

    if  ( powerThread[0] >  POWERCUT[stage] )
    {
      accelcandBasic can;
      long long numtrials         = NUMINDEP[stage];
      const short numharm = 1 << stage;

      // Write results back to DRAM and calculate sigma if needed
      if      ( canMethoud & CU_CAND_DEVICE   )
      {
        int idx =  (int)(( searchList.rLow.val[0] + gx * (double) ACCEL_DR ) / (double)numharm ) - base ;
        if ( idx >= 0 )
        {
          can.numharm = numharm;
          can.sigma   = (float)candidate_sigma_cu(powerThread[0], numharm, numtrials);
          can.z       = ( z[0]*(float) ACCEL_DZ - searchList.zMax.val[0]  )  / (float)numharm ;

          FOLD // Atomic write to global list
          {
            volatile bool done = false;
            while (!done)
            {
              volatile int prev = atomicCAS(&d_sem[idx], UINT_MAX, gx );
              if ( prev == UINT_MAX )
              {
                if ( can.sigma > d_cands[idx].sigma )
                {
                  d_cands[idx]   = can;
                }
                d_sem[idx]      = UINT_MAX;
                done            = true;
              }
            }
          }
        }
      }
      else if ( canMethoud & CU_CAND_SINGLE_G )
      {
        can.numharm = numharm;
        can.sigma   = powerThread[0];
        can.z       = z[0];
        if ( canMethoud & FLAG_SAS_SIG )
        {
          // Calculate sigma value
          can.sigma   = (float)candidate_sigma_cu(powerThread[0], numharm, numtrials);
        }

        // Write to DRAM
        d_cands[gx*noStages + stage] = can;
      }
    }
  }
   */
}

int setConstVals( cuStackList* stkLst, int numharmstages, float *powcut, long long *numindep )
{
  int noHarms         = (1 << (numharmstages - 1) );

  int szx = sizeof(int)*8;

  if (stkLst->hInfos[0].height* (noHarms /*-1*/ ) > MAX_YINDS)
  {
    printf("ERROR! YINDS to small!");
  }
  int *indsY    = (int*) malloc(stkLst->hInfos[0].height * noHarms * sizeof(int));
  int bace      = 0;
  stkLst->hInfos[0].yInds = 0;
  for (int ii = 0; ii< stkLst->noHarms; ii++)
  {
    if ( ii == 0 )
    {
      for (int j = 0; j< stkLst->hInfos[0].height; j++)
      {
        indsY[bace + j] = j;
      }
    }
    else
    {
      for (int j = 0; j< stkLst->hInfos[0].height; j++)
      {
        int zz    = -stkLst->hInfos[0].zmax+ j* ACCEL_DZ;
        int subz  = calc_required_z(stkLst->hInfos[ii].harmFrac, zz);
        int zind  = index_from_z(subz, -stkLst->hInfos[ii].zmax);
        if (zind< 0|| zind>= stkLst->hInfos[ii].height)
        {
          int Err = 0;
          printf("ERROR! YINDS Wrong!");
        }
        indsY[bace + j] = zind;
      }
    }
    stkLst->hInfos[ii].yInds = bace;
    bace += stkLst->hInfos[0].height;
  }

  void *dcoeffs;

  hipGetSymbolAddress((void **)&dcoeffs, YINDS);
  CUDA_SAFE_CALL(hipMemcpy(dcoeffs, indsY, bace*sizeof(int), hipMemcpyHostToDevice),                      "Copying Y indices to device");

  hipGetSymbolAddress((void **)&dcoeffs, POWERCUT);
  CUDA_SAFE_CALL(hipMemcpy(dcoeffs, powcut, numharmstages * sizeof(float), hipMemcpyHostToDevice),        "Copying power cutoff to device");

  hipGetSymbolAddress((void **)&dcoeffs, NUMINDEP);
  CUDA_SAFE_CALL(hipMemcpy(dcoeffs, numindep, numharmstages * sizeof(long long), hipMemcpyHostToDevice),  "Copying stages to device");

  //CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(YINDS),    indsY,         bace * sizeof(int), hipMemcpyHostToDevice), "Failed to copy Y indices to device memory.");
  //hipMemcpyToSymbol(HIP_SYMBOL(POWERCUT), powcut,    numharmstages * sizeof(float));
  //hipMemcpyToSymbol(HIP_SYMBOL(NUMINDEP), numindep,  numharmstages * sizeof(long long));

  //for(int i = 0 ; i < 400; i ++)
  {
    //printf("%03i:  %-5i  %i \n", i, indsY[i], sizeof(int)*8 );
  }

  //CUDA_SAFE_CALL(hipDeviceSynchronize(),"");

  //print_YINDS<<<1,1>>>(400);

  //CUDA_SAFE_CALL(hipDeviceSynchronize(),"");

  CUDA_SAFE_CALL(hipGetLastError(), "Error Preparing the constant memory.");
}

void sumAndSearch(cuStackList* plains, accelobs* obs, GSList** cands)
{
  //hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  dim3 dimBlock, dimGrid;

  nvtxRangePush("Add & Search");

  if ( plains->haveSData || plains->haveCData ) // previous plain has data data so sum and search
  {
    int noStages = log(plains->noHarms)/log(2) + 1;
    int harmtosum;
    cuSearchList searchList;      // The list of details of all the individual plains
    cuSearchItem* pd;
    float *rLows;
    pd = (cuSearchItem*)malloc(plains->noHarms * sizeof(cuSearchItem));
    rLows = (float*)malloc(plains->noSteps * sizeof(float));

    FOLD // Do synchronisations
    {
      for (int ss = 0; ss< plains->noStacks; ss++)
      {
        cuFfdotStack* cStack = &plains->stacks[ss];

        hipStreamWaitEvent(plains->strmSearch, cStack->plnComp, 0);
      }
    }

    if ( plains->haveCData ) // Sum & search
    {
      FOLD // Create search list
      {
        //printf("\n");

        searchList.searchRLow = plains->searchRLow;
        int i = 0;
        for (int stage = 0; stage < noStages; stage++)
        {
          harmtosum = 1 << stage;

          for (int harm = 1; harm <= harmtosum; harm += 2)
          {
            //printf("Stage  %i harm %i \n", stage, harm);

            float fract = 1-harm/ float(harmtosum);
            int idx = round(fract* plains->noHarms);
            if ( fract == 1 )
              idx = 0;

            searchList.texs.val[i]      = plains->plains[idx].datTex;
            searchList.datas.val[i]     = plains->plains[idx].d_plainData;
            searchList.powers.val[i]    = plains->plains[idx].d_powers;
            searchList.frac.val[i]      = plains->hInfos[idx].harmFrac;
            searchList.yInds.val[i]     = plains->hInfos[idx].yInds;
            searchList.heights.val[i]   = plains->hInfos[idx].height;
            searchList.widths.val[i]    = plains->plains[idx].ffdotPowWidth[0];
            searchList.strides.val[i]   = plains->hInfos[idx].inpStride;
            searchList.ffdBuffre.val[i] = plains->hInfos[idx].halfWidth*ACCEL_NUMBETWEEN;
            searchList.zMax.val[i]      = plains->hInfos[idx].zmax;
            //searchList.fullRLow.val[i]  = plains->plains[idx].searchRlowPrev[0];// .fullRLow[0];
            searchList.rLow.val[i]      = plains->plains[idx].rLow[0];

            /*
            pd[i].tex                   = plains->plains[idx].datTex;
            pd[i].data                  = plains->plains[idx].d_plainData;
            pd[i].frac                  = plains->hInfos[idx].harmFrac;
            pd[i].yInd                  = plains->hInfos[idx].yInds;
            pd[i].height                = plains->hInfos[idx].height;
            pd[i].width                 = plains->plains[idx].ffdotPowWidth[0];
            pd[i].stride                = plains->hInfos[idx].inpStride;
            pd[i].ffdBuffre             = plains->hInfos[idx].halfWidth*ACCEL_NUMBETWEEN;
            pd[i].zMax                  = plains->hInfos[idx].zmax;


            // Values for creating x indices
            //double diff =  plains->plains[idx].rLow[0] * plains->hInfos[idx].harmFrac - floor(plains );
            double diff =  plains->plains[idx].searchRlowPrev[0] * plains->hInfos[idx].harmFrac - floor( plains->plains[idx].searchRlowPrev[0] * plains->hInfos[idx].harmFrac );
            searchList.idxSum.val[i]    = 0.5 + diff*ACCEL_RDR ;

            for ( int step = 0; step < plains->noSteps; step++)         // Loop over steps
            {
              float dd      = plains->plains[idx].rLow[step] - plains->plains[0].rLow[step] * plains->hInfos[idx].harmFrac;

              double p1     = plains->plains[0].rLow[step] * plains->hInfos[idx].harmFrac ;
              double p2     = floor(plains->plains[idx].rLow[step]) ;
              double diff   = plains->plains[0].rLow[step] * plains->hInfos[idx].harmFrac - floor(plains->plains[idx].rLow[step]);
              float idxS    = 0.5 + diff*ACCEL_RDR ;
              //printf("Step %02i    diff: %12.2f    idx: %12.2f \n", step, diff, idxS);
            }*/

            i++;
          }
        }
      }

      FOLD // Call the main sum & search kernel
      {
        /*
        if      (  plains->flag & CU_CAND_DEVICE )
        {
          dimBlock.x = SS3_X;
          dimBlock.y = SS3_Y;

          float bw = SS3_X * SS3_Y;
          float ww = plains->plains[0].ffdotPowWidth / ( bw );

          dimGrid.x = ceil(ww);
          dimGrid.y = 1;

          if( plains->flag & FLAG_PLN_TEX )
          {
            if      ( noStages == 1 )
              add_and_searchCU3<1,CU_CAND_DEVICE | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            else if ( noStages == 2 )
              add_and_searchCU3<2,CU_CAND_DEVICE | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            else if ( noStages == 3 )
              add_and_searchCU3<3,CU_CAND_DEVICE | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            else if ( noStages == 4 )
              add_and_searchCU3<4,CU_CAND_DEVICE | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            else if ( noStages == 5 )
              add_and_searchCU3<5,CU_CAND_DEVICE | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
          }
          else
          {
            if      ( noStages == 1 )
              add_and_searchCU3<1,CU_CAND_DEVICE><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            else if ( noStages == 2 )
              add_and_searchCU3<2,CU_CAND_DEVICE><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            else if ( noStages == 3 )
              add_and_searchCU3<3,CU_CAND_DEVICE><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            else if ( noStages == 4 )
              add_and_searchCU3<4,CU_CAND_DEVICE><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            else if ( noStages == 5 )
              add_and_searchCU3<5,CU_CAND_DEVICE><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );

          }
          plains->haveCData = 0;
        }
        else if ( (plains->flag & CU_CAND_SINGLE_G) || (plains->flag & CU_CAND_HOST) )
        {
          dimBlock.x = SS3_X;
          dimBlock.y = SS3_Y;

          float bw = SS3_X * SS3_Y;
          float ww = plains->plains[0].ffdotPowWidth / ( bw );

          dimGrid.x = ceil(ww);
          dimGrid.y = 1;

          if( plains->flag & FLAG_PLN_TEX )
          {
            if ( plains->flag & FLAG_SAS_SIG )
            {
              if      ( noStages == 1 )
                add_and_searchCU3<1,CU_CAND_SINGLE_G | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 2 )
                add_and_searchCU3<2,CU_CAND_SINGLE_G | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 3 )
                add_and_searchCU3<3,CU_CAND_SINGLE_G | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 4 )
                add_and_searchCU3<4,CU_CAND_SINGLE_G | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 5 )
                add_and_searchCU3<5,CU_CAND_SINGLE_G | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            }
            else
            {
              if      ( noStages == 1 )
                add_and_searchCU3<1,CU_CAND_SINGLE_G | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 2 )
                add_and_searchCU3<2,CU_CAND_SINGLE_G | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 3 )
                add_and_searchCU3<3,CU_CAND_SINGLE_G | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 4 )
                add_and_searchCU3<4,CU_CAND_SINGLE_G | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 5 )
                add_and_searchCU3<5,CU_CAND_SINGLE_G | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            }
          }
          else
          {
            if ( plains->flag & FLAG_SAS_SIG )
            {
              if      ( noStages == 1 )
                add_and_searchCU3<1,CU_CAND_SINGLE_G | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 2 )
                add_and_searchCU3<2,CU_CAND_SINGLE_G | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 3 )
                add_and_searchCU3<3,CU_CAND_SINGLE_G | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 4 )
                add_and_searchCU3<4,CU_CAND_SINGLE_G | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 5 )
                add_and_searchCU3<5,CU_CAND_SINGLE_G | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            }
            else
            {
              if      ( noStages == 1 )
                add_and_searchCU3<1,CU_CAND_SINGLE_G><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 2 )
                add_and_searchCU3<2,CU_CAND_SINGLE_G><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 3 )
                add_and_searchCU3<3,CU_CAND_SINGLE_G><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 4 )
                add_and_searchCU3<4,CU_CAND_SINGLE_G><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 5 )
                add_and_searchCU3<5,CU_CAND_SINGLE_G><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            }
          }

                  }
        else if (  plains->flag & CU_CAND_SINGLE_C )
        {
          dimBlock.x = SS3_X;
          dimBlock.y = SS3_Y;

          float bw = SS3_X * SS3_Y;
          float ww = plains->plains[0].ffdotPowWidth / ( bw );

          dimGrid.x = ceil(ww);
          dimGrid.y = 1;

          if ( plains->flag & FLAG_PLN_TEX )
          {
            if ( plains->flag & FLAG_SAS_SIG )
            {
              if      ( noStages == 1 )
                add_and_searchCU3<1,CU_CAND_SINGLE_C | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 2 )
                add_and_searchCU3<2,CU_CAND_SINGLE_C | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 3 )
                add_and_searchCU3<3,CU_CAND_SINGLE_C | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 4 )
                add_and_searchCU3<4,CU_CAND_SINGLE_C | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 5 )
                add_and_searchCU3<5,CU_CAND_SINGLE_C | FLAG_SAS_SIG | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            }
            else
            {
              if      ( noStages == 1 )
                add_and_searchCU3<1,CU_CAND_SINGLE_C | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 2 )
                add_and_searchCU3<2,CU_CAND_SINGLE_C | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 3 )
                add_and_searchCU3<3,CU_CAND_SINGLE_C | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 4 )
                add_and_searchCU3<4,CU_CAND_SINGLE_C | FLAG_PLN_TEX><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 5 )
                add_and_searchCU3<5,CU_CAND_SINGLE_C><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            }
          }
          else
          {
            if ( plains->flag & FLAG_SAS_SIG )
            {
              if      ( noStages == 1 )
                add_and_searchCU3<1,CU_CAND_SINGLE_C | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 2 )
                add_and_searchCU3<2,CU_CAND_SINGLE_C | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 3 )
                add_and_searchCU3<3,CU_CAND_SINGLE_C | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 4 )
                add_and_searchCU3<4,CU_CAND_SINGLE_C | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 5 )
                add_and_searchCU3<5,CU_CAND_SINGLE_C | FLAG_SAS_SIG><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            }
            else
            {
              if      ( noStages == 1 )
                add_and_searchCU3<1,CU_CAND_SINGLE_C><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 2 )
                add_and_searchCU3<2,CU_CAND_SINGLE_C><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 3 )
                add_and_searchCU3<3,CU_CAND_SINGLE_C><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 4 )
                add_and_searchCU3<4,CU_CAND_SINGLE_C><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
              else if ( noStages == 5 )
                add_and_searchCU3<5,CU_CAND_SINGLE_C><<<dimGrid, dimBlock, 0, plains->strmSearch>>>(searchList, plains->d_bCands, plains->d_candSem, plains->rLow );
            }
          }
        }
         */

        if ( (plains->flag & CU_CAND_SINGLE_G) || (plains->flag & CU_CAND_HOST) ) // Call the templated kernel
        {
          dimBlock.x  = SS3_X;
          dimBlock.y  = SS3_Y;

          float bw    = SS3_X * SS3_Y;
          float ww    = plains->plains[0].ffdotPowWidth[0] / ( bw );

          dimGrid.x   = ceil(ww);
          dimGrid.y   = 1;

          add_and_searchCU31_f(dimGrid, dimBlock, 0, plains->strmSearch, searchList, plains->d_bCands, plains->d_candSem, 0, pd, &plains->plains->rLow[0], plains->noSteps, plains->noHarmStages, plains->flag );
        }

        // Run message
        CUDA_SAFE_CALL(hipGetLastError(), "Error at add_and_searchCU31 kernel launch");

        CUDA_SAFE_CALL(hipEventRecord(plains->searchComp,  plains->strmSearch),"Recording event: searchComp");
      }
    }

    if ( plains->haveSData ) // Process previous results
    {
      if ( plains->flag & CU_CAND_SINGLE_G )
      {
        // A blocking synchronisation to ensure results are ready to be proceeded by the host
        CUDA_SAFE_CALL(hipEventSynchronize(plains->candCpyComp), "ERROR: copying result from device to host.");

        nvtxRangePush("CPU Process results");

        plains->noResults=0;

        long long numindep;

        double poww, sig, sigx, sigc, diff;
        double gpu_p, gpu_q;
        double rr, zz;
        int added = 0;
        int numharm;
        poww = 0;

        double diffRL = plains->plains[0].searchRlow - plains->plains[0].rLow;

        if      ( ( plains->flag & CU_CAND_SINGLE_C ) == CU_CAND_SINGLE_C ) 	  // Process previous results
        {
#pragma omp critical
          for ( int x = 0; x < plains->accelLen; x++ )
          {
            for ( int i = 0; i < noStages; i++ )
            {
              for ( int step = 0; step < plains->mxSteps; step++)         // Loop over steps
              {
                int idx   = x*noStages + i ;
                poww      = plains->h_bCands[idx].sigma;

                if ( poww > 0 )
                {
                  plains->noResults++;

                  numharm   = plains->h_bCands[idx].numharm;
                  numindep  = obs->numindep[twon_to_index(numharm)];

                  if ( plains->flag & FLAG_SAS_SIG )
                    sig     = poww;
                  else
                    sig     = candidate_sigma(poww, numharm, numindep);

                  rr = ( plains->plains[0].searchRlowPrev[step] + x *  ACCEL_DR )        / (double)numharm ;
                  zz = ( plains->h_bCands[idx].z * ACCEL_DZ - plains->hInfos[0].zmax )   / (double)numharm ;

                  FOLD
                  {
                    added = 0;
                    //cands = insert_new_accelcand2(cands, poww, sig, numharm, rr, zz, &added);
                    *cands = insert_new_accelcand(*cands, poww, sig, numharm, rr, zz, &added);
                  }
                }

                if (added && !obs->dat_input)
                {
                  //fprintf(obs->workfile, "%12.2f [ %12.5f %12.5f ]  %3d  %14.4f  %14.4f  %10.4f  GPU\n", poww, sig, sigx, numharm, rr, rr / obs->T, zz);
                }
              }
            }
          }
        }
        else if ( ( plains->flag & CU_CAND_SINGLE_G ) == CU_CAND_SINGLE_G )     // Process previous results
        {
#pragma omp critical
          //for ( int x = 0; x < ACCEL_USELEN; x++ )
          for ( int step = 0; step < plains->mxSteps; step++)         // Loop over steps
          {
            for ( int stage = 0; stage < plains->noHarmStages; stage++ )
            {
              numharm   = 1 << stage;

              for ( int x = 0; x < plains->accelLen; x++ )
              {
                int idx   = step*plains->noHarmStages*plains->accelLen +   stage*plains->accelLen + x;
                poww      = plains->h_bCands[idx].sigma;

                if ( numharm  != plains->h_bCands[idx].numharm )
                {
                  int TMP = 0;
                  /// TODO this can be calculated from the stage we don't need to store it!
                }

                if ( poww > 0 )
                {
                  // We have a candidate
                  rr = ( plains->plains[0].searchRlowPrev[step] + x *  ACCEL_DR )        / (double)numharm ;
                  zz = ( plains->h_bCands[idx].z * ACCEL_DZ - plains->hInfos[0].zmax )   / (double)numharm ;

                  added = 0;
                  long grIdx = floor( rr - plains->rLow );     /// The index of the candidate in the global list
                  if ( grIdx >= 0 )
                  {
                    plains->noResults++;
                    numindep  = obs->numindep[twon_to_index(numharm)];

                    // Calculate sigma of detection
                    if ( plains->flag & FLAG_SAS_SIG )
                      sig     = poww;
                    else
                      sig     = candidate_sigma(poww, numharm, numindep);

                    if ( plains->h_candidates[grIdx].sig < sig )
                    {
                      // this sigma is greater that the current sigma for this r value
                      if ( plains->h_candidates[grIdx].sig < sig )
                      {
                        plains->h_candidates[grIdx].sig      = sig;
                        plains->h_candidates[grIdx].power    = poww;
                        plains->h_candidates[grIdx].numharm  = numharm;
                        plains->h_candidates[grIdx].r        = rr;
                        plains->h_candidates[grIdx].z        = zz;
                        added = 1;
                      }
                    }
                  }

                  if (added && !obs->dat_input)
                  {
                    //fprintf(obs->workfile, "%12.2f  %12.5f  %3d  %14.4f  %14.4f  %10.4f  GPU\n", poww, sig, numharm, rr, rr / obs->T, zz);
                  }
                }
              }
            }
          }
        }

        nvtxRangePop();

        // Do some Synchronisation
        CUDA_SAFE_CALL(hipEventRecord(plains->processComp, plains->strmSearch),"Recording event: searchComp");

        plains->haveSData = 0;
      }
    }

    // Copy results from device to host
    if ( plains->flag & CU_CAND_SINGLE_G || plains->flag & CU_CAND_HOST )
    {
      if ( plains->haveCData )
      {
        hipStreamWaitEvent(plains->strmSearch, plains->searchComp,  0);
        hipStreamWaitEvent(plains->strmSearch, plains->processComp, 0);

        //CUDA_SAFE_CALL(hipMemcpyAsync(plains->h_bCands, plains->d_bCands, ACCEL_USELEN*noStages*sizeof(accelcandBasic), hipMemcpyDeviceToHost, plains->strmSearch), "Failed to copy results back");
        CUDA_SAFE_CALL(hipMemcpyAsync(plains->h_bCands, plains->d_bCands, plains->accelLen*plains->noHarmStages*plains->noSteps*sizeof(accelcandBasic), hipMemcpyDeviceToHost, plains->strmSearch), "Failed to copy results back");

        CUDA_SAFE_CALL(hipEventRecord(plains->candCpyComp, plains->strmSearch),"Recording event: readComp");
        CUDA_SAFE_CALL(hipGetLastError(), "COPY");

        plains->haveCData = 0;
        plains->haveSData = 1;
      }
    }
  }
  nvtxRangePop();
}

