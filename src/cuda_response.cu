#include "hip/hip_runtime.h"
/** @file cuda_response.cu
 *  @brief Utility functions and kernels to calculate response filter coefficients and perfrom correlations
 *
 *  @author Chris Laidler
 *  @bug No known bugs.
 *
 *  This includes fresnel integrals
 *
 *  Change Log
 *
 *  2017-10-20
 *    Start the change log - I know this is a bit late =/
 *    Had a big refactor of all the functions here
 *    Full work up of error and optimisation of speed - See my thesis for full details
 *    In brief:
 *       Max error in "generic" single coefficients of ~1e-5 from offset > 0.04 - Note the error can be > 2e-5 for offset > 1000
 *       Max error in "generic" double coefficients of ~1e-12 from 0.0002 < offset < 0.04 - Error then drops to 2e-15 at offset = 10 then increases to 3e-14 at offset = 1000
 *
 *
 */

#include <iostream>
#include <stdio.h>

#include "cuda_math.h"
#include "cuda_response.h"


/** Point beyond which to return 0.5, the asymptotic value  .
 *
 *  At some point the Fresnel amplitude is smaller than the error
 *
 * @param nothing   This is just a dummy factor for "templating" to floating point
 * @return          The float specific boundary
 */
__host__ __device__ inline float fresLim2(float nothing)
{
  // From testing this value is ~5e6
  return FRESLIM2_F;
}

/** Point beyond which to return 0.5, the asymptotic value  .
 *
 *  At some point the Fresnel amplitude is smaller than the error
 *
 * @param nothing   This is just a dummy factor for "templating" to floating point
 * @return          The double specific boundary
 */
__host__ __device__ inline double fresLim2(double nothing)
{
  // From testing this value is ~5e9
  return FRESLIM2_D;
}

/** Get the limit below which, Fourier Interpolation, returns the actual bin  .
 *
 *  Inlined and templated function, so that it quickly returns a float specific constant value
 *
 * @param nothing   This is just a dummy factor for "templating" to floating point
 * @return          The float specific boundary
 */
__host__ __device__ inline float getRlim(float nothing)
{
  return R_LIM_F;
}

/** Get the limit below which, Fourier Interpolation, returns the actual bin  .
 *
 *  Inlined and templated function, so that it quickly returns a double specific constant value
 *
 * @param nothing   This is just a dummy factor for "templating" to floating point
 * @return          The double specific boundary
 */
__host__ __device__ inline double getRlim(double nothing)
{
  return R_LIM_D;
}

/** Get the limit below which, to do a Fourier Interpolation rather than calculate a the acceleration coefficient  .
 *
 *  Inlined and templated function, so that it quickly returns a float specific constant value
 *
 *  The bound I found by inspection is: 1e-5 + r*r/3e4 + |r|*1e3
 *  This is a quadratic value below which I have found the error in the float coefficient to be greater than float interpolation value
 *
 * @param r         The offset of the bin from the point
 * @return          The quadratic float specific boundary
 */
__host__ __device__ /*inline*/ float getZlim(float r)
{
  return 0.028f + 0.0325f * pow_t(fabs_t(r), 1.25f );
}

/** Get the limit below which, to do a Fourier Interpolation rather than calculate a the acceleration coefficient  .
 *
 *  Inlined and templated function, so that it quickly returns a float specific constant value
 *
 *  The bound I found by inspection is: 1e-5 + r*r/3e4 + |r|*1e3
 *  This is a quadratic value below which I have found the error in the float coefficient to be greater than float interpolation value
 *
 * @param r         The offset of the bin from the point
 * @return          The quadratic float specific boundary
 */
__host__ __device__ /*inline*/ double getZlim(double r)
{
  return 0.00015f + 0.0002256f * pow_t(fabs_t((float)r), 1.25f );
}

/** Get the limit below which, to do a Fourier Interpolation rather than calculate a the acceleration coefficient  .
 *
 *  Inlined and templated function, so that it quickly returns a float specific constant value
 *
 *  The bound I found by inspection is: 1e-5 + r*r/3e4 + |r|*1e3
 *  This is a quadratic value below which I have found the error in the float coefficient to be greater than float interpolation value
 *
 * @param r         The offset of the bin from the point
 * @return          The quadratic float specific boundary
 */
__host__ long double getZlim(long double r)
{
  return 0.0002L + 0.00024L * pow_t(fabs_t(r), 1.5L) ;
}

__host__ __device__ inline void fres_sinecos_phase(float x, float x2, float* sin, float* cos)
{
  float trigT;

  if      ( x2 <= FRES_SINGLE_PHASE  )				// Single Precision - no phase calcs - 1024 - 2048 - 4096
  {
    // This is the simples and most common case.
    // this accounts for the vast majority of the the coefficients between the high accurate bounds
    // In these cases     the trig calculation (assuming float has 6 significant digits)
    trigT 	= x2;
  }
  else								// Single Precision - phase adjust - 2^13 8192 - 2^14 16384 - 2^15 32768
  {
    // My modulus of the square
    trigT	= sqMod4(x);
  }

//  TESTING:							// Double Precision - phase adjust
//  {
//    // This general happens when z is close to zero, a good approximation to the boundary condition (x2 == 1e4) is:
//    // |z| < 2e-4 x (offset)^2
//    // This double precision float modulus can be very computationally expensive and it is thus worth avoiding if possible
//    trigT 	= fmod_t((double)x*(double)x, 4.0);
//  }

  // The actual trigonometric calculation - Using intrinsic function faster but less accurate.
  trigT 	= trigT*(float)PIBYTWO;
  sincos_t(trigT, sin, cos);

  // TESTING: Below is slower than the above trig, as there is no intrinsic single precision sincospif and improves performance only slightly
  //trigT 	= trigT/2.0f;
  //sincospi_t(trigT, sin, cos);
}

__host__ __device__ inline void fres_sinecos_phase(double x, double x2, double* sin, double* cos)
{
  double trigT;

  if      ( x2 <= FRES_SINGLE_PHASE  )				// Double Precision - no phase calcs
  {
    // Strait double implementation
    trigT 	= x2;
  }
  else								// Phase calculation on the x^2 term
  {
    // Do a DP phase correction
    trigT 	= sqMod4(x);
  }

  // Use sincospi it is faster and more accurate, only faster because there is no intrinsic __sincos for double precision
  sincospi_t(trigT/2.0, sin, cos);

  // TESTING: Most basic double
  //sincos_t(x2*(double)PIBYTWO, sin, cos);
}

/** Fresnel integral  .
 *
 * DESCRIPTION:
 *
 * Evaluates the Fresnel integrals
 *
 *           x
 *           -
 *          | |
 * C(x) =   |   cos(pi/2 t**2) dt,
 *        | |
 *         -
 *          0
 *
 *           x
 *           -
 *          | |
 * S(x) =   |   sin(pi/2 t**2) dt.
 *        | |
 *         -
 *          0
 *
 *
 * The integrals are evaluated by a power series for x < 1.
 * For x >= 1 auxiliary functions f(x) and g(x) are employed
 * such that
 *
 * C(x) = 0.5 + f(x) sin( pi/2 x**2 ) - g(x) cos( pi/2 x**2 )
 * S(x) = 0.5 - f(x) cos( pi/2 x**2 ) - g(x) sin( pi/2 x**2 )
 *
 *
 *
 * ACCURACY:
 *
 *  Relative error.
 *
 * Arithmetic  function   domain     # trials      peak         rms
 *   IEEE       S(x)      0, 10       10000       2.0e-15     3.2e-16
 *   IEEE       C(x)      0, 10       10000       1.8e-15     3.3e-16
 *   DEC        S(x)      0, 10        6000       2.2e-16     3.9e-17
 *   DEC        C(x)      0, 10        5000       2.3e-16     3.9e-17
 *
 *   This function is adapted from:
 *   Cephes Math Library Release 2.8:  June, 2000
 *   Copyright 1984, 1987, 1989, 2000 by Stephen L. Moshier
 *
 * @param x   Value to evaluate the Fresnel integral at
 * @param ss  The result S(x)
 * @param cc  The result C(x)
 */
template<typename T, typename idxT>
__host__ __device__ void fresnl(idxT x, T* cc, T* ss)
{
  T f, g, c, s, t, u;
  T absX;					// Absolute value of x
  absX       = fabs_t(x);			// Use templated absolute CUDA function

  /**
   *  In our case x = sqrt(2/|z|) * (-offset - z / 2.0 )
   *
   *  TODO: This function could be templated for accuracy 1-9 determining the number of elements of the polynomials that are used -
   */
  if      ( absX < (T)FREESLIM1  )		// Small so use a polynomial approximation  .
  {
    /*
       This method only gets used about 5 % of the time
       From actual run-tests I found that this cases can run in as little as ~55 clock cycles in single precision and ~1655 clock cycles in double precision on Maxwell Generation

       Op-Count
        * 31
        / 2
        + 21
    */
    T x2	= absX * absX;
    t		= x2 * x2;

    T t01 = t;
    T t02 = t01*t;
    T t03 = t02*t;
    T t04 = t03*t;
    T t05 = t04*t;
    T t06 = t05*t;
    T sn  = (T)3.18016297876567817986e11 + (T)-4.42979518059697779103e10*t01 + (T)2.54890880573376359104e9*t02  + (T)-6.29741486205862506537e7*t03  + (T)7.08840045257738576863e5 *t04 - (T)2.99181919401019853726e3  *t05;
    T sd  = (T)6.07366389490084639049e11 + (T) 2.24411795645340920940e10*t01 + (T)4.19320245898111231129e8*t02  + (T) 5.17343888770096400730e6*t03  + (T)4.55847810806532581675e4 *t04 + (T)2.81376268889994315696e2  *t05 + t06 ;
    T cn  = (T)9.99999999999999998822e-1 + (T)-2.05525900955013891793e-1*t01 + (T)1.88843319396703850064e-2*t02 + (T)-6.45191435683965050962e-4*t03 + (T)9.50428062829859605134e-6*t04 - (T)4.98843114573573548651e-8 *t05;
    T cd  = (T)1.00000000000000000118e0  + (T) 4.12142090722199792936e-2*t01 + (T)8.68029542941784300606e-4*t02 + (T) 1.22262789024179030997e-5*t03 + (T)1.25001862479598821474e-7*t04 + (T)9.15439215774657478799e-10*t05 + (T)3.99982968972495980367e-12*t06 ;

    *ss   = absX * x2 * sn / sd;
    *cc   = absX * cn / cd;
  }
  else if ( absX > fresLim2((T)0.0)  )		// Asymptotic behaviour  .
  {
    // From testing I found the point where the error is greater than the Fresnel amplitude, at that point return the asymptotic value
    *cc   = (T)0.5;
    *ss   = (T)0.5;
  }
  else						// Auxiliary functions for large argument  .
  {
    /*
       This method gets used more than 95 % of the time and is computationally more intensive, with ~115 basic flops as well as trig and fabs

       From actual run-tests I found that this cases can run in as little as ~212-482 clock cycles in single precision and ~3960 clock cycles in double precision on Maxwell Generation
       If the trig calculation is done strait up using the __sincosf intrinsic and no phase correction is done this can run in as little as ~157 clock cycles - this is not suggested as this negatively effects accuracy.
       If single precision phase correction is done, this can run in ~212, cock cycles, if the phase correction is done in double precision ~482 clock cycles are used, almost doubling the run time! (900+ clock cycles if trig is done in double precision)

       Op-Count
        * 59 + 1
        / 6
        + 43
        - 3
        fmod
        sincos
    */

    T x2	= absX * absX;			// x * x ( Standard precision value of x squared )

    t		= (T)PI * x2;
    u		= (T)1.0 / (t * t);
    t		= (T)1.0 / t;

    T u01 = u;
    T u02 = u01*u;
    T u03 = u02*u;
    T u04 = u03*u;
    T u05 = u04*u;
    T u06 = u05*u;
    T u07 = u06*u;
    T u08 = u07*u;
    T u09 = u08*u;
    T u10 = u09*u;
    T u11 = u10*u;
    T fn  = (T)3.76329711269987889006e-20 + (T)1.34283276233062758925e-16*u01 + (T)1.72010743268161828879e-13*u02 + (T)1.02304514164907233465e-10*u03 + (T)3.05568983790257605827e-8 *u04 + (T)4.63613749287867322088e-6*u05 + (T)3.45017939782574027900e-4*u06 + (T)1.15220955073585758835e-2*u07 + (T)1.43407919780758885261e-1*u08 + (T)4.21543555043677546506e-1*u09;
    T fd  = (T)1.25443237090011264384e-20 + (T)4.52001434074129701496e-17*u01 + (T)5.88754533621578410010e-14*u02 + (T)3.60140029589371370404e-11*u03 + (T)1.12699224763999035261e-8 *u04 + (T)1.84627567348930545870e-6*u05 + (T)1.55934409164153020873e-4*u06 + (T)6.44051526508858611005e-3*u07 + (T)1.16888925859191382142e-1*u08 + (T)7.51586398353378947175e-1*u09 + u10;
    T gn  = (T)1.86958710162783235106e-22 + (T)8.36354435630677421531e-19*u01 + (T)1.37555460633261799868e-15*u02 + (T)1.08268041139020870318e-12*u03 + (T)4.45344415861750144738e-10*u04 + (T)9.82852443688422223854e-8*u05 + (T)1.15138826111884280931e-5*u06 + (T)6.84079380915393090172e-4*u07 + (T)1.87648584092575249293e-2*u08 + (T)1.97102833525523411709e-1*u09 + (T)5.04442073643383265887e-1*u10 ;
    T gd  = (T)1.86958710162783236342e-22 + (T)8.39158816283118707363e-19*u01 + (T)1.38796531259578871258e-15*u02 + (T)1.10273215066240270757e-12*u03 + (T)4.60680728146520428211e-10*u04 + (T)1.04314589657571990585e-7*u05 + (T)1.27545075667729118702e-5*u06 + (T)8.14679107184306179049e-4*u07 + (T)2.53603741420338795122e-2*u08 + (T)3.37748989120019970451e-1*u09 + (T)1.47495759925128324529e0 *u10 + u11 ;

    f     = (T)1.0 - u * fn / fd;
    g     =          t * gn / gd;

    // This trig calculation is templated for T precision phase calculations for large x
    // If T precision phase calculations are used this can almost T the run time of this function!
    // Even in the single precision case this accounts for ~0.25 of computation time
    fres_sinecos_phase((T)x, x2, &s, &c);

    // TESTING: Double for comparison of accuracy
    //double ts, tc;
    //sincospi_t((double)x*(double)x/2.0, &ts, &tc);
    //s = ts; c = tc;

    t     = (T)PI * absX;

    *cc   = (T)0.5 + (f * s - g * c) / t;
    *ss   = (T)0.5 - (f * c + g * s) / t;
  }

  if ( x < (idxT)0.0 )				// Swap as function is antisymmetric  .
  {
    *cc   = -*cc;
    *ss   = -*ss;
  }
}

////////////////////  Coefficient - Fourier interpolation

/** Calculate a single Fourier interpolation coefficient at a distance from a point  .
 *
 * If you want a set of coefficients at a point or an Fourier interpolation value of FFT data see rz_coefficients and rz_convolution_cu
 *
 * This function calculates the applicable Fourier interpolation coefficient at a specific distance from a point.
 *
 * This function takes as parameters, previously calculated values.
 * These are the trigonometric values that are phase related to the reference point in unit steps in r from the point
 * These values are pre-divided by PI
 * and need only be scaled by distance
 *
 * This can be efficiently used to calculate a set of coefficients for a reference point
 *
 *
 * @param dist		The "distance" in bins of the reference r-value and a "true" location, measured in bins
 * @param sinsinPI	Phase value, sin*sin/PI, signs of trig values irrelevant
 * @param sincosPI	Phase value, sin*cos/PI, signs of trig values irrelevant
 * @param real		A pointer to the real part of the Fourier value
 * @param imag		A pointer to the real part of the Fourier value
 */
template<typename T>
__host__ __device__ inline void calc_coefficient_r(T offset, T sinsinbyPI, T sincosbyPI, T* real, T* imag)
{
  if ( fabs_t(offset) < getRlim(sinsinbyPI) )			// Check for close to actual bin values - fabs_t 1 op
  {
    // Correct for division by zero ie: sinc(0) = 1
    *real = (T)1.0;
    *imag = (T)0.0;
  }
  else
  {
    *real =  sincosbyPI / offset ;
    *imag = -sinsinbyPI / offset ;
  }
}

/** Calculate Fourier interpolation value at a given distance, in r  .
 *
 * If you want a set of coefficients at a point or an correlation of FFT data see rz_coefficients and rz_convolution_cu
 *
 * This function calculates the applicable Fourier interpolation components at a specific distance from a point.
 * Where the distance is the distance a FFT bin is in from the reference point measured in bin's.
 * where bins with values below the reference point are positive, and points above negative.
 *
 * This function calculates all the "generic" values that are independent of distance for a specific z
 * This is inefficient when requiring all coefficients for a point
 * In those cases use rz_convolution_cu or rz_coefficients
 *
 * @param offset	The distance of the (real) r value from the f-fdot position, negative below the location. Measured in (FFT) bins.
 * @param real		Pointer to the real coefficient
 * @param imag		Pointer to the imaginary coefficient
 */
template<typename T>
__host__ __device__ inline void calc_coefficient_r(T offset, T* resReal, T* resImag)
{
  // Do Fourier interpolation
  T sin, cos;

  /** NOTE: Single precision accuracy
   *   I tested using  sincospif(x)  vs  __sincosf(x*PI)
   *   I found a significant change in accuracy
   *   __sincosf(x*PI):
   *   This faster intrinsic has an error that tails off at around 1e-7,
   *   this is a decreasing error relative to the amplitude of the of values
   *   sincospif():
   *   Has an error that drops off at the same rate as the amplitude,
   *   thus has a roughly constant error of approximately 6 decimal places.
   */
  sincospi_t(offset, &sin, &cos);		// Slightly slower but constant error relative to amplitude
  //sincos_t(offset*(T)PI, &sin, &cos);		// Slightly faster but relative accuracy drops with offset

  calc_coefficient_r(offset, sin*sin/(T)PI, sin*cos/(T)PI, resReal, resImag);
}

////////////////////  Coefficient - Acceleration

__host__ __device__ inline void resp_sinecos_phase(float Qk, float dr, float z, float* sin, float* cos)
{
  //float x_float = fabs_t(z);

  float x_float = dr * dr / z - dr + z/4.0f ;
  if      ( fabs_t(x_float) < RESP_SINGLE_PHASE )
  {
    // Here the final trig term is still be below some power of two term, leaving desired precision for trig
    // Pre-calculated the value because its needed for the check ;)
  }
  else
  {
    // Calculate phase in double precision
    // This should not happen, double precision math that will probably give results worse than a R search
    // This only happens if z is very small, ie very close to zero acceleration, or if the offset is lager than the high accuracy kernel length, which is a bit long at zero actually.
    double x_double	= (double)Qk * (double)Qk / (double)z ;
    x_float		= fmod_t(x_double, 2.0);
  }

  // This could have been done but then have to change the bound multiples crazy factors
  x_float *= (float)PI;

  // The actual trigonometric calculation
  // TODO: Test accuracy using sincospi_t vs sincos_t(PI*)
  sincos_t(x_float, sin, cos);

  // NOTE: Below is slower than the above tig, as there is no intrinsic single precision sincospif func, true sincospif accuracy was not noticed in my application
  //sincospi_t(x_float, sin, cos);

  // NOTE: Tested double trig - Does not improve accuracy
  //double dSin, dCos;
  //sincospi_t((double)Qk * (double)Qk / (double)z, &dSin, &dCos );
  //*sin = dSin;
  //*cos = dCos;
}

__host__ __device__ inline void resp_sinecos_phase(double Qk, double dr, double z, double* sin, double* cos)
{
  double  xx	= (dr * dr / z - dr + z/4.0);

  // Double precision phase clipping (unnecessary as it is trig is done in double)
  //xx		= fmod_t(xx, 2.0);
  
  //xx		*= (double)PI;
  //sincos_t(xx, sin, cos);

  // Use sincospi it is faster and more accurate, only faster because there is no intrinsic __sincos for double precision
  sincospi_t(xx, sin, cos);
}

__host__ __device__ inline void resp_sinecos(float Qk, float dr, float z, float* sin, float* cos)
{
  //sincos_t(Qk * Qk / z * (float)PI, sin, cos);			// High error in places
  //sincos_t((dr*dr-z*dr+z*z/4.0) / z * (float)PI, sin, cos);
  sincos_t((dr * dr / z - dr + z/4.0f )* (float)PI, sin, cos);		// Reduce max error but with slightly higher general error
}

__host__ __device__ inline void resp_sinecos(double Qk, double dr, double z, double* sin, double* cos)
{
  //sincospi_t(Qk*Qk/z, sin, cos);				// High error in places
  sincospi_t((dr * dr / z - dr + z/4.0), sin, cos);		// Reduce max error but with slightly higher general error
}

/** Calculate the correlation coefficient  (z != 0)  .
 *
 * If you want a set of coefficients at a point or an Fourier interpolation value of FFT data see rz_coefficients and rz_convolution_cu
 *
 * This function calculates the applicable Fourier interpolation coefficient at a specific distance from a point, at a given z.
 *
 * It uses evaluating Eq (39) in:
 * Ransom, Scott M., Stephen S. Eikenberry, and John Middleditch. "Fourier techniques for very long astrophysical time-series analysis." The Astronomical Journal 124.3 (2002): 1788.
 *
 * Where: qₖ  is the variable Qk and is: ((the distance from the centre frequency) - ṙ/2)
 *        z   is the variable ṙ
 *
 * This function takes as parameters, previously calculated values.
 * These values are the values specific to the value of z only
 * They need only be calculated one per coefficient and are used as parameters
 * Making this more efficient
 *
 * This function is templated for precision can can be calculated at single or double precision
 *
 * If called as single precision
 * This function may introduce some double procession math to calculate some phase information
 * Similar double precision phase calculations are done in the evaluation of the Fresnel integral
 * This increases the range and accuracy of the single precision value at the minimum required amount double calculations
 *
 *
 * @param Qk			(rc - k ) - z/2  ( The distance of the coefficient from the reference point, scaled for z)
 * @param z			|z| ( The absolute value of z ) [ The assumption is that z != 0 ]
 * @param sq2overAbsZ		The square root of ( 2 / PI / |z| )
 * @param overSq2AbsZ		1/sqrt(|z|)
 * @param sighnZ		z/|z| The sign of z (1 or -1)
 * @param real			A pointer to the real part of the coefficient
 * @param imag			A pointer to the real part of the coefficient
 */
template<typename T, bool phaseCheck>
__host__ __device__ inline void calc_coefficient_z(T Qk, T dr, T z, T sq2overAbsZ, T overSq2AbsZ, int sighnZ, T* real, T* imag)
{
  T sin, cos;
  T Yk,Zk;
  T SZk, CZk, SYk, CYk;
  T Sk, Ck;

  if ( phaseCheck )						// This check should be evaluated at compile time
  {
    // Trig calculations templated for large Qk so phase value is calculated as a double if needed
    // Double will generally happens at very low z an approximation when |z| < 1.02e-4 x (offset)^2
    resp_sinecos_phase(Qk, dr, z, &sin, &cos);
  }
  else
  {
    resp_sinecos(Qk, dr, z, &sin, &cos);
  }

  FOLD // Fresnel calculations  .
  {
    Yk = sq2overAbsZ * Qk;
    fresnl<T, T>(Yk, &CYk, &SYk);

    Zk = sq2overAbsZ * ( Qk + z) ;
    fresnl<T, T>(Zk, &CZk, &SZk);

    Sk =  ( SZk - SYk );
    Ck =  ( CYk - CZk ) * sighnZ ;
  }

  // This is the "correct" version
  *real =  overSq2AbsZ * ( Sk * sin - Ck * cos ) ;
  *imag = -overSq2AbsZ * ( Sk * cos + Ck * sin ) ;

//  // TESTING: Double
//  *real =  rsqrt(2.0*fabs((double)z)) * ( (double)Sk * (double)sin - (double)Ck * (double)cos ) ;
//  *imag = -rsqrt(2.0*fabs((double)z)) * ( (double)Sk * (double)cos + (double)Ck * (double)sin ) ;
}

template<typename T, bool phaseCheck>
__host__ __device__ inline void calc_coefficient_z(T offset, T z, T* resReal, T* resImag)
{
  // Calculate all the "constants"
  // 6 basic fops and fabs and sqrt
  int signZ		= (z < (T)0.0) ? -1 : 1;
  T absZ		= fabs_t(z);
  T sqrtAbsZ		= sqrt_t(absZ);
  T sq2overAbsZ		= (T)SQRT2 / sqrtAbsZ;
  T overSq2AbsZ		= (T)1.0 / (T)SQRT2 / sqrtAbsZ ;
  T Qk			= offset - z / (T)2.0;			// Adjust for acceleration

  calc_coefficient_z<T, phaseCheck>(Qk, offset, z, sq2overAbsZ, overSq2AbsZ, signZ, resReal, resImag);
}

////////////////////  Coefficient - Approximation

/** Calculate Fourier interpolation value at a given distance, in r  .
 *
 * If you want a set of coefficients at a point or an correlation of FFT data see rz_coefficients and rz_convolution_cu
 *
 * This function calculates the applicable Fourier interpolation components at a specific distance from a point.
 * Where the distance is the distance a FFT bin is in from the reference point measured in bin's.
 * where bins with values below the reference point are positive, and points above negative.
 *
 * This function calculates all the "generic" values that are independent of distance for a specific z
 * This is inefficient when requiring all coefficients for a point
 * In those cases use rz_convolution_cu or rz_coefficients
 *
 * @param offset	The distance of the (real) r value from the f-fdot position, negative below the location. Measured in (FFT) bins.
 * @param real		Pointer to the real coefficient
 * @param imag		Pointer to the imaginary coefficient
 */
template<typename T>
__host__ __device__ inline void calc_coefficient_a(T offset, T z, T piR, T sinPiR, T cosPiR, T* resReal, T* resImag)
{
  T r1_abs = fabs_t(offset);
  T r1 = offset;
  T r2 = r1 * r1 ;
  T r3 = r2 * r1 ;
  T term;

  // T0 (constant) ie: Fourier interpolation
  T a0_r	= +cosPiR*sinPiR/piR;
  T a0_i	= -sinPiR*sinPiR/piR;
  if ( r1_abs < getE0lim(r1) )								// Check for close to actual bin values
  {
    a0_r = (T)1.0;
    a0_i = (T)0.0;
  }
  
  // T1 (linear) coefficient
  term		= (cosPiR-sinPiR/piR)/r2/(T)PI/(T)2.0 ;
  T a1_r	= -sinPiR * term;
  T a1_i	= -cosPiR * term;
  if ( r1_abs < getE1rLim(r1) )
  {
    a1_r	= (T)1.64493406512755329404 * r1 ;					// Liner interpolate crossing at 0
  }
  if ( r1_abs < getE1iLim(r1) )
  {
    a1_i	= (T)0.523598775598298873067 - (T)3.10062372500642122663 * r2 ;		// Quadratic interpolate intercept at Pi/6
  }

  // T2 (quadratic) coefficient
  term		= (T)0.25/(T)PI/r3*((T)3.0/(T)PI/r1*(-sinPiR/piR+cosPiR)+sinPiR);
  T a2_r	= +cosPiR*term;
  T a2_i	= -sinPiR*term;
  if ( r1_abs < getE2lim(r1) )
  {
    a2_r	= -(T)0.164493406684822643659 + (T)0.927685388563495792822 * r2 ;	// Quadratic interpolate intercept at Pi^2/60
    a2_i	=  (T)0.516751848464762894412 * r1 ;					// Liner interpolate crossing at 0
  }

  // T3 (cubic) coefficient
  // I had a look at this, I got close, but not quite there - future work?
  //T a3_r	= 0 ; //
  //T a3_i	= 0 ; // +cosPiR/(T)8.0/PI/r/r/r/r*((T)6.0/piR*((T)2.499983/piR*(-sinPiR/piR+cosPiR)+sinPiR)-cosPiR - sinPiR*sinPiR/cosPiR/(T)125.0*r/fabs_t(r) ) ;

  *resReal	= a0_r + a1_r*z + a2_r*z*z  ; //+ a3_r*z*z*z ;
  *resImag	= a0_i + a1_i*z + a2_i*z*z  ; //+ a3_i*z*z*z ;
}

/** Calculate Fourier interpolation value at a given distance, in r  .
 *
 * If you want a set of coefficients at a point or an correlation of FFT data see rz_coefficients and rz_convolution_cu
 *
 * This function calculates the applicable Fourier interpolation components at a specific distance from a point.
 * Where the distance is the distance a FFT bin is in from the reference point measured in bin's.
 * where bins with values below the reference point are positive, and points above negative.
 *
 * This function calculates all the "generic" values that are independent of distance for a specific z
 * This is inefficient when requiring all coefficients for a point
 * In those cases use rz_convolution_cu or rz_coefficients
 *
 * @param offset	The distance of the (real) r value from the f-fdot position, negative below the location. Measured in (FFT) bins.
 * @param real		Pointer to the real coefficient
 * @param imag		Pointer to the imaginary coefficient
 */
template<typename T>
__host__ __device__ inline void calc_coefficient_a(T offset, T z, T* resReal, T* resImag)
{
  T piR, sinPiR, cosPiR;

  // Basic terms
  piR = (T)PI*offset;
  sincospi_t(offset, &sinPiR, &cosPiR);		// Slightly slower but constant error relative to amplitude
  //sincos_t(piR, &sinPiR, &cosPiR);		// Slightly faster but relative accuracy drops with offset

  return calc_coefficient_a<T>(offset, z, piR, sinPiR, cosPiR, resReal, resImag);
}

////////////////////  Coefficient - Generic

/** Calculate a coefficient at a given distance, in r  .
 *
 * If you want a set of coefficients at a point or an correlation of FFT data see rz_coefficients and rz_convolution_cu
 *
 * This function calculates the applicable coefficient at a specific distance from a point.
 * These are used in the correlation to correct FFT values at a given z value and distance in r.
 * Where the distance is the distance a FFT bin is in from the reference point measured in bin's.
 * where bins with values below the reference point are positive, and points above negative.
 *
 * This function calculates all the "generic" values that are independent of distance for a specific z
 * This is inefficient when requiring all coefficients for a point
 * In those cases use rz_convolution_cu or rz_coefficients
 *
 * @param offset	The distance of the (real) r value from the f-fdot position, negative below the location. Measured in (FFT) bins.
 * @param z		The value of fdot, measured in (FFT) bins
 * @param real		Pointer to the real coefficient
 * @param imag		Pointer to the imaginary coefficient
 */
template<typename T>
__host__ __device__ void calc_coefficient(T offset, T z, T* resReal, T* resImag)
{
  T abs_z = fabs_t(z);

  if ( abs_z > getZlim(offset) )				// Calculate raw coefficient .
  {
    calc_coefficient_z<T, false>(offset, z, resReal, resImag);
  }
  else								// Calculate approximation coefficient  .
  {
    calc_coefficient_a<T>(offset, z, resReal, resImag);
  }
}

__host__ __device__ double2 calc_coefficient(double offset, double z)
{
  double2 resp;
  calc_coefficient<double>(offset, z, &resp.x, &resp.y);

  return resp;
}

__host__ __device__ float2  calc_coefficient(float  offset, float  z)
{
  float2 resp;
  calc_coefficient<float>(offset, z, &resp.x, &resp.y);

  return resp;
}

/** Calculate a coefficient at specific bin for a given reference r  .
 *
 * This function calculates the applicable coefficient at a specific distance from a point.
 * These are used in the correlation to correct FFT values at a given z value and distance in r.
 * If z is close to zero, the Fourier interpolation coefficient is given else the correlation coefficient is returned
 *
 *
 * The distance is, the distance a FFT bin is in from the reference point measured in bin's.
 * where bins with values below the reference point are negative, and points above positive.
 *
 * This function calculates all the "generic" values that are independent of distance for a specific z
 * This is inefficient when requiring all coefficients for a point
 * In those cases use rz_convolution_cu or rz_coefficients
 *
 * @param bin		The Fourier bin to be multiplied with the coefficient
 * @param z		The value of fdot, measured in (FFT) bins
 * @param real		Pointer to the real coefficient
 * @param imag		Pointer to the imaginary coefficient
 */
template<typename T>
__host__ __device__ void calc_coefficient_bin(long bin, double r, T z,  T* real, T* imag)
{
  calc_coefficient<T>( r-bin, z, real, imag );
}

////////////////////  Generate an array of coefficients

/** Calculate a set of coefficients for a give f-fdot value
 *
 * @param r			The desired fractional frequency in bins
 * @param z
 * @param kern_half_width
 * @param out
 */
template<typename T, typename outT>
__host__ __device__ void rz_coefficients(double r, T z, int kern_half_width, outT* out)
{
  outT*   resp;							// The input data, this is a complex number stored as, float2 or double2
  long    dintfreq;						// Integer part of r      - double precision
  long    start = 0;
  T	  offset;						// The distance from the centre frequency (r)
  int     numkern;						// The actual number of kernel values to use

  FOLD // Calculate the reference bin (closes integer bin to r)  .
  {
    dintfreq	= r;
    start	= dintfreq + 1 - kern_half_width ;
    offset 	= ( r - start );				// This is rc-k for the first bin
  }

  FOLD // Clamp values to usable bounds  .
  {
    numkern 	= 2 * kern_half_width;
  }

  FOLD // Calculate coefficients  .
  {
    // Calculate all the constants
    int signZ		= (z < (T)0.0) ? -1 : 1;
    T absZ		= fabs_t(z);
    T sqrtAbsZ		= sqrt_t(absZ);
    T sq2overAbsZ	= (T)SQRT2 / sqrtAbsZ;
    T overSq2AbsZ	= (T)1.0 / (T)SQRT2 / sqrtAbsZ ;
    T Qk		= offset - z / (T)2.0;			// Adjust for acceleration

    for ( int i = 0 ; i < numkern; i++, Qk--, offset-- )	// Loop over the kernel elements  .
    {
      //  Get the address of the coefficient  .
      resp	= &out[start+i];

      FOLD // Calculate coefficient  .
      {
	if ( absZ > getZlim(offset) )				// Calculate raw coefficients .
	{
	  calc_coefficient_z<T, false>(Qk, offset, z, sq2overAbsZ, overSq2AbsZ, signZ, &resp->x, &resp->y);
	}
	else							// Calculate approximation coefficients  .
	{
	  calc_coefficient_a<T>(offset, z, &resp->x, &resp->y);
	}
      }
    }
  }
}

////////////////////  Convolution

/** calculate a single point in the f-fdot plain from FFT values
 *
 * This calculation is done by direct application of a convolution
 * It convolves a number of local bins from the FFT with the relevant
 * Response value.
 *
 * It is templated for the precision of the calculation and the input data.
 * Single precision calculations use minimal double point calculation to increase accuracy
 *
 * This is done fairly efficiently, reusing some constants
 *
 * @param inputData
 * @param loR
 * @param noBins
 * @param r
 * @param z
 * @param kern_half_width
 * @param real
 * @param imag
 */
template<typename T, typename dataT>
__host__ __device__ void rz_convolution_cu(const dataT* inputData, long loR, long noBins, double r, T z, int kern_half_width, T* real, T* imag)
{
  *real = (T)0.0;
  *imag = (T)0.0;

  dataT   inp;							// The input data, this is a complex number stored as, float2 or double2
  long    dintfreq;						// Integer part of r      - double precision
  long    start;						// The first bin to use
  T       offset;						// The distance from the centre frequency (r) - NOTE: This could be double, float can get ~5 decimal places for lengths of < 999
  int     numkern;						// The actual number of kernel values to use
  T 	  resReal 	= 0;					// Response value - real
  T 	  resImag 	= 0;					// Response value - imaginary

  FOLD // Calculate the reference bin (closes integer bin to r)  .
  {
    dintfreq	= r;						// This type cast will always be the floor - unless R is negative =/
    start	= dintfreq + 1 - kern_half_width ;
    offset 	= (r - start);					// This is rc-k for the first bin
    numkern 	= 2 * kern_half_width;
  }

  FOLD // Adjust to input Data
  {
    // Adjust to start of input Data
    if ( start >= loR )
    {
      start	-= loR;						// Adjust for accessing the input FFT
    }
    else
    {
      // Start is below beginning of available data so start at available data
      numkern	-= loR - start;
      offset	= ( r - loR);					// This is rc-k for the first bin
      start	= 0;
    }

    // Adjust to end of input Data
    if ( start + numkern >= noBins )
    {
      numkern = noBins - start;
    }
  }

  FOLD // Main loop - Read input, calculate coefficients, multiply and sum results  .
  {
    // Calculate all the constants
    int signZ       = (z < (T)0.0) ? -1 : 1;
    T absZ          = fabs_t(z);
    T sqrtAbsZ      = sqrt_t(absZ);
    T sq2overAbsZ   = (T)SQRT2 / sqrtAbsZ;
    T overSq2AbsZ   = (T)1.0 / (T)SQRT2 / sqrtAbsZ ;
    T Qk            = offset - z / (T)2.0;			// Adjust for acceleration

    for ( int i = 0 ; i < numkern; i++ , Qk--, offset-- )	// Loop over the kernel elements
    {
      FOLD 							//  Read the input value  .
      {
	inp	= inputData[start+i];
      }

      FOLD 							// Calculate coefficients  .
      {
	if ( absZ > getZlim(offset) )				// Calculate raw coefficients .
	{
	  calc_coefficient_z<T, false>(Qk, offset, z, sq2overAbsZ, overSq2AbsZ, signZ, &resReal, &resImag);
	}
	else							// Calculate approximation coefficients  .
	{
	  calc_coefficient_a<T>(offset, z, &resReal, &resImag);
	}
      }

      FOLD 							//  Do the multiplication and sum  accumulate  .
      {
	*real += (resReal * inp.x - resImag * inp.y);
	*imag += (resReal * inp.y + resImag * inp.x);
      }
    }
  }


}

/**  Uses the correlation method to do a Fourier interpolation at a number integer spaced (r) points in the f-fdot plane.
 *
 * It does the correlations manually. (i.e. no FFTs)
 * The kernels can be reused for the same value of z and fraction of r
 * Thus each thread calculates each kernel value once and uses it to calculate the value of
 * a number of integer spaced points in the r direction
 *
 * @param inputData           A pointer to the beginning of the input data
 * @param outData             A pointer to the location of the output complex numbers, this is a thread dependent array of length noBlk
 * @param loR                 The R value of the first bin in the input data
 * @param r                   The R value of the first point to do the interpolation at
 * @param z                   The Z value of the to do the interpolation at
 * @param blkWidth            The width of the blocks in bins
 * @param kern_half_width     The half width of the points to use in the interpolation
 */
template<typename T, typename dataIn, typename dataOut>
__host__ __device__ void rz_convolution_cu(const dataIn* inputData, long loR, long inStride, double r, T z, int kern_half_width, dataOut* outData, int blkWidth, int noBlk)
{
  for ( int blk = 0; blk < noBlk; blk++ )
  {
    outData[blk].x = 0;
    outData[blk].y = 0;
  }

  dataIn  inp;							// The input data, this is a complex number stored as, float2 or double2
  long    dintfreq;						// Integer part of r      - double precision
  long    start;						// The first bin to use
  T       offset;						// The distance from the centre frequency (r) - NOTE: This could be double, float can get ~5 decimal places for lengths of < 999
  int     numkern;						// The actual number of kernel values to use
  T       resReal 	= (T)0.0;				// Response value - real
  T       resImag 	= (T)0.0;				// Response value - imaginary

  FOLD 								// Calculate the reference bin (closes integer bin to r)  .
  {
    dintfreq	= r;						// TODO: Check this when r is < 0 ?????
    start	= dintfreq + 1 - kern_half_width ;
  }

  FOLD 								// Clamp values to usable bounds  .
  {
    numkern	= 2 * kern_half_width;
    offset	= ( r - start);					// This is rc-k for the first bin
  }

  FOLD 								// Adjust for FFT  .
  {
    // Adjust to FFT
    start -= loR;						// Adjust for accessing the input FFT
  }

  FOLD // Main loop - Read input, calculate coefficients, multiply and sum results  .
  {
    // Calculate all the constants
    int signZ		= (z < (T)0.0) ? -1 : 1;
    T absZ		= fabs_t(z);
    T sqrtAbsZ		= sqrt_t(absZ);
    T sq2overAbsZ	= (T)SQRT2 / sqrtAbsZ;
    T overSq2AbsZ	= (T)1.0 / (T)SQRT2 / sqrtAbsZ ;
    T Qk		= offset - z / (T)2.0;			// Adjust for acceleration

    for ( int i = 0 ; i < numkern; i++, Qk--, offset--)		// Loop over the kernel elements
    {

      FOLD 							// Calculate coefficient  .
      {
	if ( fabs_t(z) > getZlim(offset) )			// Calculate raw coefficients .
	{
	  calc_coefficient_z<T, false>(Qk, offset, z, sq2overAbsZ, overSq2AbsZ, signZ, &resReal, &resImag);
	}
	else							// Calculate approximation coefficients  .
	{
	  calc_coefficient_a<T>(offset, z, &resReal, &resImag);
	}
      }

      // Use the coefficient on each input value with the same fractional part

      for ( int blk = 0; blk < noBlk; blk++ )
      {
	FOLD // Clamp values to usable bounds  .
	{
	  int idx = start+i+blk*blkWidth;

	  if ( idx >= 0 && idx < inStride )
	  {
	    FOLD //  Read the input value  .
	    {
	      inp	= inputData[idx];
	    }

	    FOLD //  Do the multiplication  .
	    {
	      outData[blk].x += (resReal * inp.x - resImag * inp.y);
	      outData[blk].y += (resReal * inp.y + resImag * inp.x);
	    }
	  }
	}
      }
    }
  }
}


//#ifdef WITH_OPT_BLK_SHF

/**  Uses the correlation method to do a Fourier interpolation at a number integer spaced (r) points in the f-fdot plane.
 *
 * It does the correlations manually. (i.e. no FFTs)
 * The kernels can be reused for the same value of z and fraction of r
 * Thus each thread calculates each kernel value once and uses it to calculate the value of
 * a number of integer spaced points in the r direction
 *
 * @param inputData           A pointer to the beginning of the input data
 * @param outData             A pointer to the location of the output complex numbers, this is a thread dependent array of length noBlk
 * @param loR                 The R value of the first bin in the input data
 * @param r                   The R value of the first point to do the interpolation at
 * @param z                   The Z value of the to do the interpolation at
 * @param blkWidth            The width of the blocks in bins
 * @param kern_half_width     The half width of the points to use in the interpolation
 */
template<int noColumns>
__host__ __device__ void rz_convolution_sfl(float2* inputData, const long loR, const long inStride, const double r, const float z, const int kern_half_width, float2* outData, const int colWidth, const int ic, const int cIdx)
{
  long    dintfreq;						// Integer part of r      - double precision
  long    start;						// The first bin to use
  float   offset;						// The distance from the centre frequency (r) - NOTE: This could be double, float can get ~5 decimal places for lengths of < 999
  int     numkern;						// The actual number of kernel values to use
  float   resReal;						// Response value - real
  float   resImag;						// Response value - imaginary

  FOLD 								// Calculate the reference bin (closes integer bin to r)  .
  {
    dintfreq	= r;						// TODO: Check this when r is < 0 ?????
    start	= dintfreq + 1 - kern_half_width ;
  }

  FOLD 								// Clamp values to usable bounds  .
  {
    numkern	= 2 * kern_half_width;
    offset	= ( r - cIdx - start);				// This is rc-k for the first bin
  }

  FOLD 								// Adjust for FFT  .
  {
    // Adjust to FFT
    start -= loR;						// Adjust for accessing the input FFT
  }

  FOLD // Zero the output
  {
    outData->x = 0.0f;
    outData->y = 0.0f;
  }

  inputData = &inputData[start+(cIdx)*colWidth];

  FOLD // Main loop - Read input, calculate coefficients, multiply and sum results  .
  {
    // Calculate all the constants
    int signZ		= (z < (float)0.0) ? -1 : 1;
    float absZ		= fabs_t(z);
    float sqrtAbsZ	= sqrt_t(absZ);
    float sq2overAbsZ	= (float)SQRT2 / sqrtAbsZ;
    float overSq2AbsZ	= (float)1.0 / (float)SQRT2 / sqrtAbsZ ;
    float Qk		= offset - z / (float)2.0;		// Adjust for acceleration

    for ( int i = 0 ; i < numkern; i+=noColumns, Qk-=noColumns, offset-=noColumns)		// Loop over the kernel elements
    {
      FOLD 							// Calculate coefficient  .
      {
	//calc_coefficient<float>(offset, z, &resReal, &resImag);
	if ( fabs_t(z) > getZlim(offset) )			// Calculate raw coefficients .
	{
	  calc_coefficient_z<float, false>(Qk, offset, z, sq2overAbsZ, overSq2AbsZ, signZ, &resReal, &resImag);
	}
	else							// Calculate approximation coefficients  .
	{
	  calc_coefficient_a<float>(offset, z, &resReal, &resImag);
	}
      }

      FOLD 							//  Do the multiplication and sum  accumulate  .
      {
	for( int idx = 0; idx < noColumns; idx++)
	{
	  // TODO: May have to do an end condition check here?

	  // Read input - These reads are generally coalesced
	  // I have found they are highly cached, so much so that no manual caching or sharing with shuffle is needed!
	  //float2 inp = inputData[start + i + idx + (cIdx)*colWidth];
	  float2 inp = inputData[i + idx];

#ifdef  __CUDA_ARCH__
	  float resCRea_c = __shfl(resReal, idx, noColumns );
	  float resImag_c = __shfl(resImag, idx, noColumns );
	  outData->x += (resCRea_c * inp.x - resImag_c * inp.y);
	  outData->y += (resCRea_c * inp.y + resImag_c * inp.x);
#else
	  //	  float resCRea_c;
	  //	  float resImag_c;
	  //
	  //	  FOLD 							// Calculate coefficient  .
	  //	  {
	  //	    int adjust = cIdx-idx; // TODO: this needs to be checked, sigh change?
	  //	    if ( fabs_t(z) > getZlim(offset) )			// Calculate raw coefficients .
	  //	    {
	  //	      calc_coefficient_z<float, false>(Qk+adjust, offset+adjust, z, sq2overAbsZ, overSq2AbsZ, signZ, &resCRea_c, &resImag_c);
	  //	    }
	  //	    else							// Calculate approximation coefficients  .
	  //	    {
	  //	      calc_coefficient_a<float>(offset+adjust, z, &resCRea_c, &resImag_c);
	  //	    }
	  //	  }
//	  outData->x += (resCRea_c * inp.x - resImag_c * inp.y);
//	  outData->y += (resCRea_c * inp.y + resImag_c * inp.x);
#endif

	}
      }
    }
  }
}

////////////////////  DBG

// DBG - Testing function
template<typename T>
__global__ void k_fresnlin(kerStruct inf)
{
  T v[DEVIS][3];

  T start	= inf.fList[0];
  T end		= inf.fList[1];
  T off2	= (end-start)/(float)inf.reps/(float)REPS;

  for ( int i = 0 ; i < inf.reps; ++i)
  {
    float off	= start + i/(float)inf.reps*(end-start);

    for ( int ii = 0; ii < REPS; ++ii )
    {
      off += off2;
#pragma unroll
      for ( int x = 0; x < DEVIS; x++ )
      {
	fresnl<T, T>(off + (T)0.00000001*x, &v[x][0], &v[x][1]);
	v[x][2] += v[x][0] + v[x][1];
      }
    }
  }

  if ( threadIdx.y * blockDim.x + threadIdx.x > 1024 )
  {
#pragma unroll
    for ( int x = 0; x < DEVIS; x++ )
    {
      printf("%f %f %f - ", v[x][0], v[x][1], v[x][2] );
    }
    printf("\n");
  }
}

template<typename T, typename T2>
__global__ void k_fresnEval(T* input, T2* output )
{
  const int bid = blockIdx.y  * gridDim.x  + blockIdx.x;		/// Block ID (flat index)
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;		/// Thread ID in block (flat index)
  const int gid = bid * blockDim.x* blockDim.y + tid;			/// Grid ID (flat index)

  T v = input[gid];
  T2 res;

  fresnl<T, T>(v, &res.x, &res.y);

  output[gid] = res;
}

//__global__ void k_fresnEval_d(double* input, double2* output )
//{
//  const int bid = blockIdx.y  * gridDim.x  + blockIdx.x;        	/// Block ID (flat index)
//  const int tid = threadIdx.y * blockDim.x + threadIdx.x;       	/// Thread ID in block (flat index)
//  const int gid = bid * blockDim.x* blockDim.y + tid;			/// Grid ID (flat index)
//
//  double v = input[gid];
//  double2 res;
//
//  fresnl<double, double>(v, &res.x, &res.y);
//
//  output[gid] = res;
//}

// DBG - Testing function
template<typename T>
__global__ void k_finterpin(kerStruct inf)
{
  T v[DEVIS][3];

  T start	= inf.fList[0];
  T end		= inf.fList[1];
  T off2	= (end-start)/(float)inf.reps/(float)REPS;

  for ( int i = 0 ; i < inf.reps; ++i)
  {
    float off	= start + i/(float)inf.reps*(end-start);

    for ( int ii = 0; ii < REPS; ++ii )
    {
      off += off2;
#pragma unroll
      for ( int x = 0; x < DEVIS; x++ )
      {
	calc_coefficient_r<T>( off + (T)0.00000001*x, &v[x][0], &v[x][1] );
	v[x][2] += v[x][0] + v[x][1];
      }
    }
  }

  if ( threadIdx.y * blockDim.x + threadIdx.x > 1024 )
  {
#pragma unroll
    for ( int x = 0; x < DEVIS; x++ )
    {
      printf("%f %f %f - ", v[x][0], v[x][1], v[x][2] );
    }
    printf("\n");
  }
}

template<typename T, typename T2>
__global__ void k_finterpEval(T* input, T2* output )
{
  const int bid = blockIdx.y  * gridDim.x  + blockIdx.x;        	/// Block ID (flat index)
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;       	/// Thread ID in block (flat index)
  const int gid = bid * blockDim.x* blockDim.y + tid;			/// Grid ID (flat index)

  T v = input[gid];
  T2 res;

  calc_coefficient_r<T>(v, &res.x, &res.y);

  output[gid] = res;
}

// DBG - Testing function
template<typename T>
__global__ void k_responsein(kerStruct inf)
{
  T v[DEVIS][3];

  // Standard case - single precision no phase
  T off	= inf.fList[0];
  T z	= inf.fList[1];

  for ( int i = 0 ; i < inf.reps; ++i)
  {
    for ( int ii = 0; ii < REPS; ++ii )
    {
#pragma unroll
      for ( int x = 0; x < DEVIS; x++ )
      {
	calc_coefficient<T>(off+ (T)0.00000001*x, z, &v[x][0], &v[x][1]) ;
	v[x][2] += v[x][0] + v[x][1];
      }
    }
  }

  if ( threadIdx.y * blockDim.x + threadIdx.x > 1024 )
  {
#pragma unroll
    for ( int x = 0; x < DEVIS; x++ )
    {
      printf("%f %f %f - ", v[x][0], v[x][1], v[x][2] );
    }
    printf("\n");
  }
}

// DBG Templates
template __global__ void k_fresnlin<float>(kerStruct inf);
template __global__ void k_fresnlin<double>(kerStruct inf);
template __global__ void k_responsein<float>(kerStruct inf);
template __global__ void k_responsein<double>(kerStruct inf);

template __global__ void k_fresnEval<float , float2 >(float* input,  float2* output  );
template __global__ void k_fresnEval<double, double2>(double* input, double2* output );


template __global__ void k_finterpEval<float , float2 >(float*  input, float2*  output );
template __global__ void k_finterpEval<double, double2>(double* input, double2* output );
template __global__ void k_finterpin<float >(kerStruct inf);
template __global__ void k_finterpin<double>(kerStruct inf);

//////////////////// Templates

template void fresnl<float,  float>  (float  xxa, float*  cc, float*  ss);
template void fresnl<float,  double> (double xxa, float*  cc, float*  ss);
template void fresnl<double, double> (double xxa, double* cc, double* ss);

////////////////////

template void calc_coefficient<float> (float  offset, float  z,  float*  real, float*  imag);
template void calc_coefficient<double>(double offset, double z,  double* real, double* imag);

template void calc_coefficient_bin<float> (long bin, double r, float  z,  float*  real, float*  imag);
template void calc_coefficient_bin<double>(long bin, double r, double z,  double* real, double* imag);


template void calc_coefficient_r<float >(float  dist, float  sinsinPI, float  sincosPI, float*  real, float*  imag);
template void calc_coefficient_r<double>(double dist, double sinsinPI, double sincosPI, double* real, double* imag);

template void calc_coefficient_r<float >(float  offset, float*  real, float*  imag);
template void calc_coefficient_r<double>(double offset, double* real, double* imag);


//template void calc_coefficient_z<float,  true >(float  Qk, float  dr, float  z, float  sq2overAbsZ, float  overSq2AbsZ, int sighnZ, float*  real, float*  imag);
//template void calc_coefficient_z<float,  false>(float  Qk, float  dr, float  z, float  sq2overAbsZ, float  overSq2AbsZ, int sighnZ, float*  real, float*  imag);
//template void calc_coefficient_z<double, true >(double Qk, double dr, double z, double sq2overAbsZ, double overSq2AbsZ, int sighnZ, double* real, double* imag);
//template void calc_coefficient_z<double, false>(double Qk, double dr, double z, double sq2overAbsZ, double overSq2AbsZ, int sighnZ, double* real, double* imag);

template void calc_coefficient_z<float,  true >(float  offset, float  z, float*  real, float*  imag);
template void calc_coefficient_z<float,  false>(float  offset, float  z, float*  real, float*  imag);
template void calc_coefficient_z<double, true >(double offset, double z, double* real, double* imag);
template void calc_coefficient_z<double, false>(double offset, double z, double* real, double* imag);

template inline void calc_coefficient_z<float,  false>(float Qk, float dr, float z, float sq2overAbsZ, float overSq2AbsZ, int sighnZ, float* real, float* imag);


template void calc_coefficient_a<float >(float  offset, float  z, float*  real, float*  imag);
template void calc_coefficient_a<double>(double offset, double z, double* real, double* imag);


template void rz_coefficients<double, double2>(double r, double z, int kern_half_width, double2* out);
template void rz_coefficients<float,  float2> (double r, float  z, int kern_half_width, float2*  out);

////////////////////


//#ifdef WITH_OPT_BLK_SHF
template void rz_convolution_sfl<1 >(float2* inputData, const long loR, const long inStride, const double r, const float z, const int kern_half_width, float2* outData, const int colWidth, const int ic, const int cIdx);
template void rz_convolution_sfl<2 >(float2* inputData, const long loR, const long inStride, const double r, const float z, const int kern_half_width, float2* outData, const int colWidth, const int ic, const int cIdx);
template void rz_convolution_sfl<4 >(float2* inputData, const long loR, const long inStride, const double r, const float z, const int kern_half_width, float2* outData, const int colWidth, const int ic, const int cIdx);
template void rz_convolution_sfl<8 >(float2* inputData, const long loR, const long inStride, const double r, const float z, const int kern_half_width, float2* outData, const int colWidth, const int ic, const int cIdx);
template void rz_convolution_sfl<16>(float2* inputData, const long loR, const long inStride, const double r, const float z, const int kern_half_width, float2* outData, const int colWidth, const int ic, const int cIdx);
template void rz_convolution_sfl<32>(float2* inputData, const long loR, const long inStride, const double r, const float z, const int kern_half_width, float2* outData, const int colWidth, const int ic, const int cIdx);
//#endif

template void rz_convolution_cu<float,  float2> (const float2*  inputData, long loR, long noBins, double r, float  z, int kern_half_width, float*  real, float*  imag);
//template void rz_convolution_cu<float,  double2>(const double2* inputData, long loR, long noBins, double r, float  z, int kern_half_width, float*  real, float*  imag);
template void rz_convolution_cu<double, float2> (const float2*  inputData, long loR, long noBins, double r, double z, int kern_half_width, double* real, double* imag);
//template void rz_convolution_cu<double, double2>(const double2* inputData, long loR, long noBins, double r, double z, int kern_half_width, double* real, double* imag);



template void rz_convolution_cu<float,  float2, float2> (const float2* inputData, long loR, long inStride, double r, float  z, int kern_half_width, float2* outData, int blkWidth, int noBlk);
template void rz_convolution_cu<double, float2, float2> (const float2* inputData, long loR, long inStride, double r, double z, int kern_half_width, float2* outData, int blkWidth, int noBlk);


////////////////////
