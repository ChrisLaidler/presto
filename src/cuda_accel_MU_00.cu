#include "hip/hip_runtime.h"
#include "cuda_accel_MU.h"

/** Kernel for testing best possible performance - Just write to ffdot plane - 1 thread per complex value  .
 *
 * @param kernels
 * @param inpData
 * @param ffdot
 * @param width
 * @param height
 * @param stride
 * @param noSteps
 * @param kerHeight
 */
__global__ void mult00_k(const __restrict__ fcomplexcu* kernels, const __restrict__ fcomplexcu*  inpData, __restrict__ fcomplexcu* ffdot, const int width, const int height, const int stride, const int noSteps, const int noPlns, int kerHeight )
{
  const int ix = blockIdx.x * CNV_DIMX + threadIdx.x;
  const int iy = blockIdx.y * CNV_DIMY + threadIdx.y;

  fcomplexcu ker;                                 /// kernel data
  uint nHeight = height * noSteps;

  ker.i = 0;
  ker.r = 0;

  if (ix < width && iy < nHeight)
  {
    // Calculate flat index
    const int idx = iy * stride + ix;

    ffdot[idx] = ker;
  }
}


/** Kernel for testing best possible performance - Read input, read kernel, write to ffdot plane - 1 thread per column  .
 *
 */
__global__ void mult01_k(const __restrict__ fcomplexcu* kernels, const __restrict__ fcomplexcu* inpData, __restrict__ fcomplexcu* ffdot, const int width, const int height, const int stride, const int noSteps, const int noPlns, int kerHeight )
{
  const int bidx = threadIdx.y * CNV_DIMX + threadIdx.x;          /// Block ID - flat index
  const int tid  = blockIdx.x  * CNV_DIMX * CNV_DIMY + bidx;      /// Global thread ID - flat index ie column index of stack

  fcomplexcu ker;                                                 /// kernel data

  if ( tid < width )  // Valid thread  .
  {
    int idx;                                                      /// flat index of output plane

    FOLD  // Stride, kernel, input data & output data  .
    {
      kernels += tid;
      ffdot   += tid;
      inpData += tid;
    }

    FOLD // Read input data  .
    {
      for (int step = 0; step < noSteps; step++)
      {
        for (int pln = 0; pln < noPlns; pln++)                    // Loop through the planes  .
        {
          fcomplexcu ipd        = inpData[ (int)(pln*noSteps*stride + step*stride) ];

          if ( ipd.r < 0 && ipd.r > 0 ) 	                        // Required so as to not optimise out  .
          {
            printf("ipd \n");
          }
        }
      }
    }

    FOLD // Read kernel  .
    {
      int   lDepth  = ceilf(kerHeight/(float)gridDim.y);
      int   y0      = lDepth*blockIdx.y;
      int   y1      = MIN(y0+lDepth, kerHeight);

      for (int kerY = y0; kerY < y1; kerY++ )
      {
        idx   = kerY * stride;
        ker   = kernels[idx];

        if ( ker.r < 0 && ker.r > 0 )                             // Required so as to not optimise out  .
        {
          printf("ker \n");
        }
      }
    }

    FOLD // Write data to planes  .
    {
      int   nHeight = height * noSteps;
      int   lDepth  = ceilf(nHeight/(float)gridDim.y);
      int   y0      = lDepth*blockIdx.y;
      int   y1      = MIN(y0+lDepth, nHeight);

      ker.i         = 0;
      ker.r         = 0;

      for (int y = y0; y < y1; y++ )
      {
        idx         = y * stride;

        FOLD // Write  .
        {
          ffdot[idx] = ker;
        }
      }
    }
  }
}

/** Kernel for testing best possible performance - Just write to ffdot plane - Each thread loops down over column  .
 *
 * @param kernels
 * @param inpData
 * @param ffdot
 * @param width
 * @param height
 * @param stride
 * @param noSteps
 * @param kerHeight
 */
__host__  void mult00(hipStream_t multStream, cuFFdotBatch* batch, uint stack)
{
  dim3 dimGrid, dimBlock;

  cuFfdotStack* cStack = &batch->stacks[stack];

  dimBlock.x = CNV_DIMX;
  dimBlock.y = CNV_DIMY;

  if (0)
  {
    dimGrid.x = ceil(cStack->width                    / (float) ( CNV_DIMX ));
    dimGrid.y = ceil(cStack->height*batch->noSteps    / (float) ( CNV_DIMX ));

    mult00_k<<<dimGrid, dimBlock, 0, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->height, cStack->strideCmplx, batch->noSteps, cStack->noInStack, cStack->kerHeigth);
  }
  else
  {
    dimGrid.x = ceil(cStack->width / (float) ( CNV_DIMX * CNV_DIMY ));
    dimGrid.y = cStack->mulSlices;

    mult01_k<<<dimGrid, dimBlock, 0, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->height, cStack->strideCmplx, batch->noSteps, cStack->noInStack, cStack->kerHeigth);
  }

}

//-----------------------------------------//


/** Multiplication kernel - Multiply a stack with a kernel - multi-step - Loop ( Pln - Y - step )  .
 * Each thread loops down a column of the plane
 * Reads the input and multiplies it with the kernel and writes result to plane
 */
template<int FLAGS, int noSteps>
__global__ void mult02_k(const fcomplexcu* __restrict__ kernels, const fcomplexcu* __restrict__ inpData, fcomplexcu* __restrict__ ffdot, const int width, const int stride, int noPlns, const int firstPlane )
{
  const int bidx = threadIdx.y * CNV_DIMX + threadIdx.x;          /// Block ID - flat index
  const int tid  = blockIdx.x  * CNV_DIMX * CNV_DIMY + bidx;      /// Global thread ID - flat index ie column index of stack

  if ( tid < width )  // Valid thread  .
  {
    int idx;                                      /// flat index of output plane
    int pHeight = 0;                              /// Height of previous data in the stack
    fcomplexcu ker;                               /// kernel data

    FOLD  // Stride, kernel, input data & output data  .
    {
      kernels += tid;
      ffdot   += tid;
      inpData += tid;
    }

    __restrict__ fcomplexcu inpDat[noSteps];                  // Set of input data for this thread/column

    for (int pln = 0; pln < noPlns; pln++)                    // Loop through the planes  .
    {
      const int plnStrd       = pln*stride*noSteps;
      const int plnHeight     = HEIGHT_HARM[firstPlane + pln];
      const int kerYOffset    = (HEIGHT_HARM[firstPlane] - plnHeight)/2;
      const int ns2           = plnHeight * stride;

      FOLD // Read input data for this plane
      {
        for (int step = 0; step < noSteps; step++)
        {
          fcomplexcu inp    = inpData[ (int)(plnStrd + step*stride) ];
          inp.r             /= (float) width;
          inp.i             /= (float) width;
          inpDat[step]      = inp;
        }
      }

      for (int planeY = 0; planeY < plnHeight; planeY++)      // Loop over the individual plane  .
      {
        FOLD // Read the kernel value  .
        {
          ker   = kernels[(kerYOffset+planeY)*stride];
        }

        int off1;

        FOLD // Calculate partial offset  .
        {
          if      ( FLAGS & FLAG_ITLV_ROW )
          {
            off1  = pHeight + planeY*noSteps*stride;
          }
          else
          {
            off1  = pHeight + planeY*stride;
          }
        }

        for ( int step = 0; step < noSteps; ++step )          // Loop over steps .
        {
          FOLD // Calculate indices  .
          {
            if      ( FLAGS & FLAG_ITLV_ROW )
            {
              idx  = off1 + step * stride;
            }
            else
            {
              idx  = off1 + step * ns2;
            }
          }

          fcomplexcu kv;
          FOLD // Multiply  .
          {
            kv.r = (inpDat[step].r * ker.r + inpDat[step].i * ker.i);
            kv.i = (inpDat[step].i * ker.r - inpDat[step].r * ker.i);
          }

          //ker.r       = tid;
          //ker.i       = planeY;
          //ffdot[idx]  = ker;
          //ffdot[idx]  = inpDat[step];
          //kv.r = width;
          ffdot[idx]  = kv;
        }
      }

      pHeight += plnHeight * noSteps * stride;
    }
  }
}

template<int FLAGS>
__host__  void mult02_s(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t multStream, cuFFdotBatch* batch, uint stack)
{
  cuFfdotStack* cStack  = &batch->stacks[stack];
  int offset            = cStack->startIdx;

  switch (batch->noSteps)
  {
    case 1:
    {
      mult02_k<FLAGS,1><<<dimGrid, dimBlock, i1, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 2:
    {
      mult02_k<FLAGS,2><<<dimGrid, dimBlock, i1, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 3:
    {
      mult02_k<FLAGS,3><<<dimGrid, dimBlock, i1, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 4:
    {
      mult02_k<FLAGS,4><<<dimGrid, dimBlock, i1, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 5:
    {
      mult02_k<FLAGS,5><<<dimGrid, dimBlock, i1, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 6:
    {
      mult02_k<FLAGS,6><<<dimGrid, dimBlock, i1, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 7:
    {
      mult02_k<FLAGS,7><<<dimGrid, dimBlock, i1, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    case 8:
    {
      mult02_k<FLAGS,8><<<dimGrid, dimBlock, i1, multStream>>>(cStack->d_kerData , cStack->d_iData, cStack->d_planeMult, cStack->width, cStack->strideCmplx, cStack->noInStack, offset);
      break;
    }
    default:
    {
      fprintf(stderr, "ERROR: mult11 has not been templated for %i steps\n", batch->noSteps);
      exit(EXIT_FAILURE);
    }
  }
}

__host__  void mult02_f(hipStream_t multStream, cuFFdotBatch* batch, uint stack)
{
  dim3 dimGrid, dimBlock;

  cuFfdotStack* cStack = &batch->stacks[stack];

  dimBlock.x = CNV_DIMX;
  dimBlock.y = CNV_DIMY;

  dimGrid.x = ceil(cStack->width / (float) ( CNV_DIMX * CNV_DIMY ));
  dimGrid.y = 1;

  if      ( batch->flag & FLAG_ITLV_ROW )
    mult02_s<FLAG_ITLV_ROW>(dimGrid, dimBlock, 0, multStream, batch, stack);
  else
    mult02_s<0>(dimGrid, dimBlock, 0, multStream, batch, stack);
}
