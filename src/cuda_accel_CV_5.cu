#include "hip/hip_runtime.h"
#include "cuda_accel_CV.h"

/** Convolution kernel - Convolve an entire batch with convolution kernel
 * Each thread loops down a column of the plains and convolves input with kernel and writes result to plain
 */
template<uint FLAGS, typename hType, int noHars>
__global__ void convolveffdot5_ok(const fcomplexcu *kernels, const fcomplexcu *datas, fcomplexcu *ffdot, iHarmList widths, iHarmList strides, iHarmList heights, uint no, uint hh, uint noSteps, uint step )
{
  const int ix = blockIdx.x * CNV_DIMX * CNV_DIMY + CNV_DIMX * threadIdx.y + threadIdx.x;

  int iy      = 0;
  int height  = 0;
  fcomplexcu ker, dat;

  // Stride
  kernels += ix;
  ffdot   += ix;
  datas   += ix;
  int tid  = 0;
  int idx  = 0;
  int stride;

  if ( FLAGS & FLAG_STP_ROW )
  {
    ffdot  += ix + step * strides.val[0]; ;
  }
  else if ( FLAGS & FLAG_STP_PLN )
  {
    ffdot += ix ; //+ step * strides.val[0] * heights.val[0];
  }

  for (int n = 0; n < no; n++)
  {
    if (ix < widths.val[n])
    {
      stride   = strides.val[n];
      dat      = datas[step*stride];
      datas    += stride*noSteps;

      dat.r   /= (float) widths.val[n] ;
      dat.i   /= (float) widths.val[n] ;

      height  += heights.val[n];

      if ( FLAGS & FLAG_STP_PLN )
      {
        ffdot += step * strides.val[n] * heights.val[n];
      }

      for (; iy < height; iy++)
      {
        ker           =  kernels[tid];
        tid          += stride;


        // Convolve
        ffdot[idx].r = (dat.r * ker.r + dat.i * ker.i);
        ffdot[idx].i = (dat.i * ker.r - dat.r * ker.i);


        //ffdot[tid].r  =  (dat.r * ker.r + dat.i * ker.i);
        //ffdot[tid].i  =  (dat.i * ker.r - dat.r * ker.i);


        // Calculate indices
        if ( FLAGS & FLAG_STP_ROW )
        {
          ffdot  += noSteps * stride;
        }
        else if ( FLAGS & FLAG_STP_PLN )
        {
          ffdot += stride;
        }
      }
      if ( FLAGS & FLAG_STP_PLN )
      {
        ffdot += ( noSteps - 1 - step ) * strides.val[n] * heights.val[n];
      }
    }
  }
}


template<uint FLAGS, int noSteps>
__global__ void convolveffdot5_k(const fcomplexcu *kernels, const fcomplexcu *datas, fcomplexcu *ffdot, int noHars)
{
  const int ix = blockIdx.x * CNV_DIMX * CNV_DIMY + CNV_DIMX * threadIdx.y + threadIdx.x;

  //int iy        = 0;
  //int heightSum = 0;
  //int height    = 0;

  fcomplexcu dat[noSteps];

  // Stride
  //kernels += ix;
  ffdot   += ix;
  datas   += ix;

  for (int n = 0; n < noHars; n++)            // Loop over plains
  {
    const int stride   = STRIDE_FAM_ORDER[n];
    const int height   = HEIGHT_FAM_ORDER[n];
    fcomplexcu* ker    = KERNEL_FAM_ORDER[n] + ix;

    if ( ix < stride )
    {
      // read input into registers
      for (int step = 0; step < noSteps; step++) // Loop over plains
      {
        dat[step]      = datas[step*stride];

        // Normalise
        dat[step].r   /= (float) stride ;
        dat[step].i   /= (float) stride ;
      }

      // Stride input data
      datas        += stride*noSteps;

      for (int iy = 0; iy < height; iy++)
      {
        const int PlnStride = iy*stride;

        // Convolve and write data
        for (int step = 0; step < noSteps; step++) // Loop over steps
        {
          // Convolve
          fcomplexcu val;
          val.r = (dat[step].r * ker->r + dat[step].i * ker->i);
          val.i = (dat[step].i * ker->r - dat[step].r * ker->i);

          if      ( FLAGS & FLAG_STP_ROW )
          {
            *ffdot = val;
            ffdot += stride;  // Stride output pointer to next plain
          }
          else if ( FLAGS & FLAG_STP_PLN )
          {
            ffdot[PlnStride + stride+height ] = val;
          }
        }

        // Stride kernel to next "row"
        ker += stride;
      }

      if ( FLAGS & FLAG_STP_PLN ) // Stride output pointer to next plain
      {
        ffdot += noSteps*height*stride;
      }
    }
  }
}


template<int FLAGS>
__host__  void convolveffdot5_s(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t cnvlStream, cuFFdotBatch* batch)
{
  switch (batch->noSteps)
  {
    case 1:
    {
      convolveffdot5_k<FLAGS,1><<<dimGrid, dimBlock, i1, cnvlStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 2:
    {
      convolveffdot5_k<FLAGS,2><<<dimGrid, dimBlock, i1, cnvlStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 3:
    {
      convolveffdot5_k<FLAGS,3><<<dimGrid, dimBlock, i1, cnvlStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 4:
    {
      convolveffdot5_k<FLAGS,4><<<dimGrid, dimBlock, i1, cnvlStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 5:
    {
      convolveffdot5_k<FLAGS,5><<<dimGrid, dimBlock, i1, cnvlStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 6:
    {
      convolveffdot5_k<FLAGS,6><<<dimGrid, dimBlock, i1, cnvlStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 7:
    {
      convolveffdot5_k<FLAGS,7><<<dimGrid, dimBlock, i1, cnvlStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 8:
    {
      convolveffdot5_k<FLAGS,8><<<dimGrid, dimBlock, i1, cnvlStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    default:
    {
      fprintf(stderr, "ERROR: convolveffdot5 has not been templated for %i steps\n", batch->noSteps);
      exit(EXIT_FAILURE);
    }
  }

  hipDeviceSynchronize();

  int tmp = 0;
}

__host__  void convolveffdot5_f(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t cnvlStream, cuFFdotBatch* batch)
{
  if      ( batch->flag & FLAG_STP_ROW )
    convolveffdot5_s<FLAG_STP_ROW>(dimGrid, dimBlock, i1, cnvlStream, batch);
  else if ( batch->flag & FLAG_STP_PLN )
    convolveffdot5_s<FLAG_STP_PLN>(dimGrid, dimBlock, i1, cnvlStream, batch);
  else
  {
    fprintf(stderr, "ERROR: convolveffdot5 has not been templated for layout.\n");
    exit(EXIT_FAILURE);
  }
}
