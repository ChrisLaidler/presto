#include "cuda_accel_MU.h"

__device__ hipfftCallbackLoadC d_loadCallbackPtr     = CB_MultiplyInput;
__device__ hipfftCallbackStoreC d_storeCallbackPtr   = CB_PowerOut;

__device__ hipfftComplex CB_MultiplyInput( void *dataIn, size_t offset, void *callerInfo, void *sharedPtr)
{
  stackInfo *inf  = (stackInfo*)callerInfo;

  int fIdx        = inf->famIdx;
  int noSteps     = inf->noSteps;
  int noPlains    = inf->noPlains;
  int stackStrd   = STRIDE_FAM_ORDER[fIdx];
  int width       = WIDTH_FAM_ORDER[fIdx];

  int strd        = stackStrd * noSteps ;                 /// Stride taking into acount steps)
  int gRow        = offset / strd;                        /// Row (ignoring steps)
  int col         = offset % stackStrd;                   /// 2D column
  int top         = 0;                                    /// The top of the plain
  int pHeight     = 0;
  int pln         = 0;

  for ( int i = 0; i < noPlains; i++ )
  {
    top += HEIGHT_FAM_ORDER[fIdx+i];

    if ( gRow >= top )
    {
      pln         = i+1;
      pHeight     = top;
    }
  }

  int row         = offset / stackStrd - pHeight*noSteps;
  int pIdx        = fIdx + pln;
  int plnHeight   = HEIGHT_FAM_ORDER[pIdx];
  int step;

  if ( inf->flag & FLAG_ITLV_PLN )
  {
    step = row / plnHeight;
    row  = row % plnHeight;
  }
  else
  {
    step  = row % noSteps;
    row   = row / noSteps;
  }

  hipfftComplex ker = ((hipfftComplex*)(KERNEL_FAM_ORDER[pIdx]))[row*stackStrd + col];      //
  hipfftComplex inp = ((hipfftComplex*)inf->d_iData)[(pln*noSteps+step)*stackStrd + col];   //

  // Do the multiplication
  hipfftComplex out;
  out.x = ( inp.x * ker.x + inp.y * ker.y ) / (float)width;
  out.y = ( inp.y * ker.x - inp.x * ker.y ) / (float)width;

  return out;
}

__device__ void CB_PowerOut( void *dataIn, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr)
{
  // Calculate power
  float power = element.x*element.x + element.y*element.y ;

  // Write result (offsets are the same)
  ((float*)callerInfo)[offset] = power;
}

void copyCUFFT_LD_CB(cuFFdotBatch* batch)
{
  CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_ldCallbackPtr, HIP_SYMBOL(d_loadCallbackPtr),  sizeof(hipfftCallbackLoadC)),   "");
  CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_stCallbackPtr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(hipfftCallbackStoreC)),  "");
}

/** Multiply and inverse FFT the complex f-∂f plain using FFT callback
 * @param batch
 */
void multiplyBatchCUFFT(cuFFdotBatch* batch )
{
#ifdef SYNCHRONOUS
  cuFfdotStack* pStack = NULL;  // Previous stack
#endif

  // Multiply this entire stack in one block
  for (int ss = 0; ss< batch->noStacks; ss++)
  {
    cuFfdotStack* cStack = &batch->stacks[ss];

    FOLD // Synchronisation  .
    {
      CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->fftPStream, cStack->prepComp,0),   "Waiting for GPU to be ready to copy data to device.");  // Need input data
      CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->fftPStream, batch->searchComp, 0), "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete

#ifdef SYNCHRONOUS
      // Wait for all the input FFT's to complete
      for (int ss = 0; ss< batch->noStacks; ss++)
      {
        cuFfdotStack* cStack2 = &batch->stacks[ss];
        hipStreamWaitEvent(cStack->fftPStream, cStack2->prepComp, 0);
      }

      // Wait for the previous multiplication to complete
      if ( pStack != NULL )
        hipStreamWaitEvent(cStack->fftPStream, pStack->plnComp, 0);
#endif
    }

    FOLD // Do the FFT  .
    {
#pragma omp critical
      FOLD
      {
        FOLD // Timing  .
        {
#ifdef TIMING
          hipEventRecord(cStack->invFFTinit, cStack->fftPStream);
#endif
        }

        FOLD // Set store FFT callback  .
        {
          if ( batch->flag & FLAG_MUL_CB_OUT )
          {
            CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&batch->h_stCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&cStack->d_plainPowers ),"");
          }
        }

        FOLD // Set load FFT callback  .
        {
          CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&batch->h_ldCallbackPtr, HIPFFT_CB_LD_COMPLEX, (void**)&cStack->d_sInf ),"");
        }

        CUFFT_SAFE_CALL(hipfftSetStream(cStack->plnPlan, cStack->fftPStream),  "Error associating a CUFFT plan with multStream.");
        CUFFT_SAFE_CALL(hipfftExecC2C(cStack->plnPlan, (hipfftComplex *) cStack->d_plainData, (hipfftComplex *) cStack->d_plainData, HIPFFT_BACKWARD),"Error executing CUFFT plan.");
      }
    }

    FOLD // Synchronisation  .
    {
      hipEventRecord(cStack->plnComp, cStack->fftPStream);

#ifdef SYNCHRONOUS
      pStack = cStack;
#endif
    }
  }
}

/** Multiply and inverse FFT the complex f-∂f plain
 * This assumes the input data is ready and on the device
 * This creates a complex f-∂f plain
 */
void multiplyBatch(cuFFdotBatch* batch)
{
  //hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  if ( batch->haveInput )
  {
    nvtxRangePush("Multiply & FFT");
#ifdef STPMSG
    printf("\tMultiply & FFT\n");
#endif

    dim3 dimBlock, dimGrid;

    if ( batch->flag & FLAG_MUL_CB_IN )  		// Do the multiplication using a CUFFT callback  .
    {
#ifdef STPMSG
      printf("\t\tMultiply with CUFFT\n");
#endif
      multiplyBatchCUFFT( batch );
    }
    else                                    // Do the multiplication and FFT separately  .
    {
      FOLD // Multiply  .
      {
#ifdef STPMSG
        printf("\t\tMultiply\n");
#endif

        // In my testing I found multiplying each plain separately works fastest so it is the "default"
        if      ( batch->flag & FLAG_MUL_BATCH ) 	// Do the multiplications one family at a time  .
        {
          FOLD // Synchronisation  .
          {
            for (int ss = 0; ss < batch->noStacks; ss++) // Synchronise input data preparation for all stacks
            {
              cuFfdotStack* cStack = &batch->stacks[ss];
              CUDA_SAFE_CALL(hipStreamWaitEvent(batch->multStream, cStack->prepComp,0),     "Waiting for GPU to be ready to copy data to device.");    // Need input data
            }

            CUDA_SAFE_CALL(hipStreamWaitEvent(batch->multStream, batch->searchComp, 0),      "Waiting for GPU to be ready to copy data to device.");   // This will overwrite the f-fdot plain so search must be compete
          }

          FOLD // Call kernel  .
          {
#ifdef TIMING // Timing event  .
            CUDA_SAFE_CALL(hipEventRecord(batch->multInit, batch->multStream),"Recording event: multInit");
#endif

            mult50_f(batch->multStream, batch);

            // Run message
            CUDA_SAFE_CALL(hipGetLastError(), "Error at kernel launch");
          }

          FOLD // Synchronisation  .
          {
            CUDA_SAFE_CALL(hipEventRecord(batch->multComp, batch->multStream),"Recording event: multComp");
          }
        }
        else if ( batch->flag & FLAG_MUL_STK ) 	  // Do the multiplications one stack  at a time  .
        {
#ifdef SYNCHRONOUS
          cuFfdotStack* pStack = NULL;  // Previous stack
#endif

          // Multiply this entire stack in one block
          for (int ss = 0; ss < batch->noStacks; ss++)
          {
            cuFfdotStack* cStack = &batch->stacks[ss];

            FOLD // Synchronisation  .
            {
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->multStream, cStack->prepComp,0),    "Waiting for GPU to be ready to copy data to device.");  // Need input data
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->multStream, batch->searchComp, 0),  "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete

#ifdef SYNCHRONOUS
              // Wait for all the input FFT's to complete
              for (int ss = 0; ss< batch->noStacks; ss++)
              {
                cuFfdotStack* cStack2 = &batch->stacks[ss];
                hipStreamWaitEvent(cStack->multStream, cStack2->prepComp, 0);
              }

              // Wait for the previous multiplication to complete
              if ( pStack != NULL )
                hipStreamWaitEvent(cStack->multStream, pStack->multComp, 0);
#endif
            }

            FOLD // Timing event  .
            {
#ifdef TIMING
              CUDA_SAFE_CALL(hipEventRecord(cStack->multInit, cStack->multStream),"Recording event: multInit");
#endif
            }

            FOLD // Call kernel(s)  .
            {
              if      ( batch->flag & FLAG_MUL_00 )
              {
                mult00_f(cStack->multStream, batch, ss);
              }
              else if ( batch->flag & FLAG_MUL_10 )
              {
                mult10_f(cStack->multStream, batch, ss);
              }
              else if ( batch->flag & FLAG_MUL_20 )
              {
                mult20_f(cStack->multStream, batch, ss);
              }
              else if ( batch->flag & FLAG_MUL_30 )
              {
                mult30_f(cStack->multStream, batch, ss);
              }
              /*
              else if ( batch->flag & FLAG_MUL_41 )
              {
                mult41_f(cStack->multStream, batch, ss);
              }
              else if ( batch->flag & FLAG_MUL_42 )
              {
                mult42_f(cStack->multStream, batch, ss);
              }
              else if ( batch->flag & FLAG_MUL_43 )
              {
                mult43_f(cStack->multStream, batch, ss);
              }
              */
              else
              {
                fprintf(stderr,"ERROR: No valid multiply specified. Line %i in %s.\n", __LINE__, __FILE__);
                exit(EXIT_FAILURE);
              }

              // Run message
              CUDA_SAFE_CALL(hipGetLastError(), "Error at kernel launch (mult7)");
            }

            FOLD // Synchronisation  .
            {
              hipEventRecord(cStack->multComp, cStack->multStream);

#ifdef SYNCHRONOUS
              pStack = cStack;
#endif
            }
          }
        }
        else if ( batch->flag & FLAG_MUL_PLN ) 	  // Do the multiplications one plain  at a time  .
        {
          mult40(batch);
        }
        else
        {
          fprintf(stderr, "ERROR: multiplyBatch not templated for this type of multiplication.\n");
        }
      }

      FOLD // Inverse FFT the f-∂f plain  .
      {

#ifdef STPMSG
        printf("\t\tInverse FFT\n");
#endif

#ifdef SYNCHRONOUS
        cuFfdotStack* pStack = NULL;  // Previous stack
#endif

        // Copy fft data to device
        //for (int ss = plains->noStacks-1; ss >= 0; ss-- )
        for (int ss = 0; ss< batch->noStacks; ss++)
        {
          cuFfdotStack* cStack = &batch->stacks[ss];

          FOLD // Synchronisation  .
          {
            hipStreamWaitEvent(cStack->fftPStream, cStack->multComp, 0);
            hipStreamWaitEvent(cStack->fftPStream, batch->multComp,  0);

#ifdef SYNCHRONOUS
            // Wait for all the multiplications to complete
            for (int ss = 0; ss< batch->noStacks; ss++)
            {
              cuFfdotStack* cStack2 = &batch->stacks[ss];
              hipStreamWaitEvent(cStack->fftPStream, cStack2->multComp, 0);
            }

            // Wait for the previous fft to complete
            if ( pStack != NULL )
              hipStreamWaitEvent(cStack->fftPStream, pStack->plnComp, 0);
#endif
          }

          FOLD // Call the inverse CUFFT  .
          {
#pragma omp critical
            {
              FOLD // Timing  .
              {
#ifdef TIMING
                hipEventRecord(cStack->invFFTinit, cStack->fftPStream);
#endif
              }

              if ( batch->flag & FLAG_MUL_CB_OUT ) // Set the CUFFT callback to calculate and store powers  .
              {
                CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&batch->h_stCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&cStack->d_plainPowers ),"");
              }

              CUFFT_SAFE_CALL(hipfftSetStream(cStack->plnPlan, cStack->fftPStream),  "Error associating a CUFFT plan with multStream.");
              CUFFT_SAFE_CALL(hipfftExecC2C(cStack->plnPlan, (hipfftComplex *) cStack->d_plainData, (hipfftComplex *) cStack->d_plainData, HIPFFT_BACKWARD),"Error executing CUFFT plan.");

              FOLD // Synchronisation  .
              {
                hipEventRecord(cStack->plnComp, cStack->fftPStream);

#ifdef SYNCHRONOUS
                pStack = cStack;
#endif
              }
            }
          }
        }
      }
    }

    batch->haveInput    = 0;
    batch->haveConvData = 1;

    nvtxRangePop();
  }

  // Set the r-values and width for the next iteration when we will be doing the actual Add and Search
  cycleRlists(batch);
}

