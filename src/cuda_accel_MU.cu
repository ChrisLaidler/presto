#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "cuda_accel_MU.h"

//====================================== Constant variables  ===============================================\\

__device__ hipfftCallbackLoadC  d_loadCallbackPtr    = CB_MultiplyInput;
__device__ hipfftCallbackStoreC d_storePow_f         = CB_PowerOut_f;
__device__ hipfftCallbackStoreC d_storePow_h         = CB_PowerOut_h;
__device__ hipfftCallbackStoreC d_storeInmemRow      = CB_PowerOutInmem_ROW;
__device__ hipfftCallbackStoreC d_storeInmemPln      = CB_PowerOutInmem_PLN;


//======================================= Global variables  ================================================\\


//========================================== Functions  ====================================================\\

__device__ hipfftComplex CB_MultiplyInput( void *dataIn, size_t offset, void *callerInfo, void *sharedPtr)
{
  stackInfo *inf  = (stackInfo*)callerInfo;

  int fIdx        = inf->famIdx;
  int noSteps     = inf->noSteps;
  int noPlains    = inf->noPlains;
  int stackStrd   = STRIDE_HARM[fIdx];
  int width       = WIDTH_HARM[fIdx];

  int strd        = stackStrd * noSteps ;                 /// Stride taking into acount steps)
  int gRow        = offset / strd;                        /// Row (ignoring steps)
  int col         = offset % stackStrd;                   /// 2D column
  int top         = 0;                                    /// The top of the plain
  int pHeight     = 0;
  int pln         = 0;

  for ( int i = 0; i < noPlains; i++ )
  {
    top += HEIGHT_HARM[fIdx+i];

    if ( gRow >= top )
    {
      pln         = i+1;
      pHeight     = top;
    }
  }

  int row         = offset / stackStrd - pHeight*noSteps;
  int pIdx        = fIdx + pln;
  int plnHeight   = HEIGHT_HARM[pIdx];
  int step;

  if ( inf->flag & FLAG_ITLV_PLN )
  {
    step = row / plnHeight;
    row  = row % plnHeight;
  }
  else
  {
    step  = row % noSteps;
    row   = row / noSteps;
  }

  hipfftComplex ker = ((hipfftComplex*)(KERNEL_HARM[pIdx]))[row*stackStrd + col];      //
  hipfftComplex inp = ((hipfftComplex*)inf->d_iData)[(pln*noSteps+step)*stackStrd + col];   //

  // Do the multiplication
  hipfftComplex out;
  out.x = ( inp.x * ker.x + inp.y * ker.y ) / (float)width;
  out.y = ( inp.y * ker.x - inp.x * ker.y ) / (float)width;

  return out;
}

__device__ void CB_PowerOut_f( void *dataIn, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr)
{
  // Calculate power
  float power = element.x*element.x + element.y*element.y ;

  // Write result (offsets are the same)
  ((float*)callerInfo)[offset] = power;
}

__device__ void CB_PowerOut_h( void *dataIn, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr)
{
  // Calculate power
  float power = element.x*element.x + element.y*element.y ;

  // Write result (offsets are the same)
  ((half*)callerInfo)[offset] = __float2half(power);
}

__device__ void CB_PowerOutInmem_ROW( void *dataIn, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr)
{
  const int hw  = 56 * 2  ; // HWIDTH_STAGE[0];
  const int al  = 3870 ; //ALEN ;
  const int ns  = 4 ; // NO_STEPS
  int row   = offset  / ( INMEM_FFT_WIDTH * ns ) ;
  int col   = offset  % INMEM_FFT_WIDTH;
  int step  = ( offset % ( INMEM_FFT_WIDTH * ns ) ) / INMEM_FFT_WIDTH;


  col      -= hw;

  //if ( col >= 0 && col < al )
  {

    // Calculate power
    float power = element.x*element.x + element.y*element.y ;
    //half  power = __float2half(element.x*element.x + element.y*element.y) ;

    // Write result (offsets are the same)
    //int plnOff = /*row * PLN_STRIDE*/ + step*al + col;
    //PLN_START[plnOff] = power;
    //PLN_START[offset] = power;
    //((float*)callerInfo)[plnOff] = power;
    //((float*)callerInfo)[offset] = power;
    ((half*)callerInfo)[offset] = __float2half(power);
    //((half*)callerInfo)[offset] = power;

    //  if ( offset == 162735 )
    //  {
    //    printf("\n");
    //
    //    printf("PLN_START:  %p \n", PLN_START);
    //    printf("PLN_STRIDE: %i \n", PLN_STRIDE);
    //    printf("NO_STEPS:   %i \n", NO_STEPS);
    //    printf("step0:      %i \n", step0);
    //
    //    printf("row:        %i \n", row);
    //    printf("col:        %i \n", col);
    //    printf("step:       %i \n", step);
    //  }
  }
}

__device__ void CB_PowerOutInmem_PLN( void *dataIn, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr)
{
  //  int step0 = (int)callerInfo; // I know this isn't right but its faster than accessing the pointer =)
  //  int row   = offset  / INMEM_FFT_WIDTH;
  //  int step  = row /  HEIGHT_STAGE[0];
  //  row       = row %  HEIGHT_STAGE[0];  // Assumes plain interleaved!
  //  int col   = offset % INMEM_FFT_WIDTH;
  //int plnOff = row * PLN_STRIDE + step0 + step + col;

  // Calculate power
  float power = element.x*element.x + element.y*element.y ;

  // Write result
  //PLN_START[plnOff] = power;
  //((float*)callerInfo)[offset] = power;
  ((half*)callerInfo)[offset] = __float2half(power);
}

void copyCUFFT_LD_CB(cuFFdotBatch* batch)
{
  CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_ldCallbackPtr, HIP_SYMBOL(d_loadCallbackPtr),  sizeof(hipfftCallbackLoadC)),   "");

  //  if ( batch->flag & FLAG_SS_INMEM  )
  //  {
  //    if      ( batch->flag & FLAG_ITLV_ROW )
  //      CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_stCallbackPtr, HIP_SYMBOL(d_storeInmemRow), sizeof(hipfftCallbackStoreC)),  "");
  //    else if ( batch->flag & FLAG_ITLV_PLN )
  //      CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_stCallbackPtr, HIP_SYMBOL(d_storeInmemPln), sizeof(hipfftCallbackStoreC)),  "");
  //    else
  //    {
  //      fprintf(stderr,"ERROR: invalid memory lay out. Line %i in %s\n", __LINE__, __FILE__);
  //    }
  //  }
  //  else
  //  {
  //    CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_stCallbackPtr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(hipfftCallbackStoreC)),  "");
  //  }

  if (  (batch->flag & FLAG_SS_INMEM) && ( batch->flag & FLAG_HALF) )
  {
    CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_stCallbackPtr, HIP_SYMBOL(d_storePow_h), sizeof(hipfftCallbackStoreC)),  "");
  }
  else
  {
    CUDA_SAFE_CALL(hipMemcpyFromSymbol( &batch->h_stCallbackPtr, HIP_SYMBOL(d_storePow_f), sizeof(hipfftCallbackStoreC)),  "");
  }

}

/** Multiply and inverse FFT the complex f-∂f plain using FFT callback
 * @param batch
 */
void multiplyBatchCUFFT(cuFFdotBatch* batch )
{
#ifdef SYNCHRONOUS
  cuFfdotStack* pStack = NULL;  // Previous stack
#endif

  // Multiply this entire stack in one block
  for (int ss = 0; ss< batch->noStacks; ss++)
  {
    cuFfdotStack* cStack = &batch->stacks[ss];

    FOLD // Synchronisation  .
    {
      CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->fftPStream, cStack->prepComp,0),   "Waiting for GPU to be ready to copy data to device.");  // Need input data
      CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->fftPStream, batch->searchComp, 0), "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete

      if ( batch->retType & CU_STR_PLN )
      {
        CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->fftPStream, batch->candCpyComp, 0), "Waiting for GPU to be ready to copy data to device.");  // Multiplication will change the plain
      }

#ifdef SYNCHRONOUS
      // Wait for all the input FFT's to complete
      for (int ss = 0; ss < batch->noStacks; ss++)
      {
        cuFfdotStack* cStack2 = &batch->stacks[ss];
        hipStreamWaitEvent(cStack->fftPStream, cStack2->prepComp, 0);
      }

      // Wait for the previous multiplication to complete
      if ( pStack != NULL )
        hipStreamWaitEvent(cStack->fftPStream, pStack->ifftComp, 0);
#endif
    }

    FOLD // Do the FFT  .
    {
#pragma omp critical
      FOLD
      {
        FOLD // Timing  .
        {
#ifdef TIMING
          hipEventRecord(cStack->ifftInit, cStack->fftPStream);
#endif
        }

        FOLD // Set store FFT callback  .
        {
          if ( batch->flag & FLAG_CUFFT_CB_OUT )
          {
            if ( batch->flag & FLAG_SS_INMEM  )
            {
              rVals* rVal;
              rVal = &((*batch->rSearch)[0][0]);

              printf("\nRval: %i  adressL %p  \n", rVal->step, &rVal->step );

              CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&batch->h_stCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)rVal->step ),"");
            }
            else
            {
              CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&batch->h_stCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&cStack->d_plainPowers ),"");
            }
          }
        }

        FOLD // Set load FFT callback  .
        {
          CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&batch->h_ldCallbackPtr, HIPFFT_CB_LD_COMPLEX, (void**)&cStack->d_sInf ),"");
        }

        CUFFT_SAFE_CALL(hipfftSetStream(cStack->plnPlan, cStack->fftPStream),  "Error associating a CUFFT plan with multStream.");
        CUFFT_SAFE_CALL(hipfftExecC2C(cStack->plnPlan, (hipfftComplex *) cStack->d_plainData, (hipfftComplex *) cStack->d_plainData, HIPFFT_BACKWARD),"Error executing CUFFT plan.");
      }
    }

    FOLD // Synchronisation  .
    {
      hipEventRecord(cStack->ifftComp, cStack->fftPStream);

#ifdef SYNCHRONOUS
      pStack = cStack;
#endif
    }
  }
}

__global__ void cpyPowers_ker( float*  dst, size_t  dpitch, const float *  src, size_t  spitch, size_t  width, size_t  height)
{
  int ix = blockIdx.x * 16 + threadIdx.x ;
  //int iy = blockIdx.y * 16 + threadIdx.y ;

  for ( int iy = 0 ; iy < height; iy++)
  {
    if ( ix < width && iy < height)
    {
      dst[iy*dpitch + ix] = src[iy*spitch +ix];
    }
  }
}

void cpyPowers( float* __restrict__ dst, size_t  dpitch, const float* __restrict__ src, size_t  spitch, size_t  width, size_t  height, hipStream_t  stream)
{
  dim3 dimBlock, dimGrid;

  dimBlock.x  = 512;
  dimBlock.y  = 1 ; //16;

  float ww    = width  / (float)dimBlock.x ;
  float hh    = height / (float)dimBlock.y ;

  dimGrid.x   = ceil(ww);
  dimGrid.y   = 1 ; // ceil(hh);

  cpyPowers_ker<<<dimGrid,  dimBlock, 0, stream >>>(dst, dpitch, src, spitch, width, height);
}

template<typename T>
void copyIFFTtoPln( cuFFdotBatch* batch, cuFfdotStack* cStack)
{
  rVals* rVal;

  T* dst;
  T* src;
  size_t  dpitch;
  size_t  spitch;
  size_t  width;
  size_t  height;

  int powSz;

  powSz = sizeof(T);

  dpitch  = batch->sInf->mInf->inmemStride * powSz;
  width   = batch->accelLen * powSz;
  height  = cStack->height;
  spitch  = cStack->strideFloat * powSz;

  for ( int step = 0; step < batch->noSteps; step++ )
  {
    rVal = &((*batch->rInput)[step][0]);

    //if ( rVal->step >= 0 && rVal->step < batch->sInf->SrchSz->noSteps )
    if ( rVal->numrs )
    {
      dst     = ((T*)batch->d_plainFull)    + rVal->step * batch->accelLen;

      if      ( batch->flag & FLAG_ITLV_ROW )
      {
        src     = ((T*)cStack->d_plainPowers)  + cStack->strideFloat*step + batch->hInfos->halfWidth * ACCEL_NUMBETWEEN;
        spitch  = cStack->strideFloat*batch->noSteps*powSz;
      }
      else if ( batch->flag & FLAG_ITLV_PLN )
      {
        src     = ((T*)cStack->d_plainPowers)  + cStack->strideFloat*height*step + batch->hInfos->halfWidth * ACCEL_NUMBETWEEN ;
      }
      else
      {
        fprintf(stderr,"ERROR: Invalid interleaving, on line %i in %s.", __LINE__, __FILE__);
      }

      //CUDA_SAFE_CALL(hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToDevice, cStack->fftPStream ),"Error calling hipMemcpy2DAsync after IFFT.");

      CUDA_SAFE_CALL(hipStreamWaitEvent(batch->strmSearch, cStack->ifftComp, 0), "");
      CUDA_SAFE_CALL(hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToDevice, batch->strmSearch ),"Error calling hipMemcpy2DAsync after IFFT.");

      FOLD // Synchronisation  .
      {
        hipEventRecord(cStack->ifftMemComp, batch->strmSearch);
      }
    }
    else
    {
      TMP
    }
  }
}

/** Multiply and inverse FFT the complex f-∂f plain
 * This assumes the input data is ready and on the device
 * This creates a complex f-∂f plain
 */
void multiplyBatch(cuFFdotBatch* batch)
{
  //hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  if ( batch->haveInput )
  {
    nvtxRangePush("Multiply & FFT");
#ifdef STPMSG
    printf("\tMultiply & FFT\n");
#endif

    dim3 dimBlock, dimGrid;

    if ( batch->flag & FLAG_CUFFT_CB_IN )   // Do the multiplication using a CUFFT callback  .
    {
#ifdef STPMSG
      printf("\t\tMultiply with CUFFT\n");
#endif
      multiplyBatchCUFFT( batch );
    }
    else                                    // Do the multiplication and FFT separately  .
    {
      FOLD // Multiply  .
      {
#ifdef STPMSG
        printf("\t\tMultiply\n");
#endif

        // In my testing I found multiplying each plain separately works fastest so it is the "default"
        if      ( batch->flag & FLAG_MUL_BATCH ) 	// Do the multiplications one family at a time  .
        {
          FOLD // Synchronisation  .
          {
            for (int ss = 0; ss < batch->noStacks; ss++) // Synchronise input data preparation for all stacks
            {
              cuFfdotStack* cStack = &batch->stacks[ss];
              CUDA_SAFE_CALL(hipStreamWaitEvent(batch->multStream, cStack->prepComp,0),      "Waiting for GPU to be ready to copy data to device.");    // Need input data

              if ( (batch->flag & FLAG_CUFFT_CB_OUT) )
              {
                // CFF output callback has its own data so can start once FFT is complete
                CUDA_SAFE_CALL(hipStreamWaitEvent(batch->multStream, cStack->ifftComp, 0),  "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete
              }
            }

            if ( !(batch->flag & FLAG_CUFFT_CB_OUT) )
            {
              // Have to wait for search to finish reading data
              CUDA_SAFE_CALL(hipStreamWaitEvent(batch->multStream, batch->searchComp, 0),  "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete
            }

            if ( (batch->retType & CU_STR_PLN) && !(batch->flag & FLAG_CUFFT_CB_OUT) )
            {
              CUDA_SAFE_CALL(hipStreamWaitEvent(batch->multStream, batch->candCpyComp, 0),   "Waiting for GPU to be ready to copy data to device.");   // Multiplication will change the plain
            }
          }

          FOLD // Call kernel  .
          {
#ifdef TIMING // Timing event  .
            CUDA_SAFE_CALL(hipEventRecord(batch->multInit, batch->multStream),"Recording event: multInit");
#endif

            mult30_f(batch->multStream, batch);

            // Run message
            CUDA_SAFE_CALL(hipGetLastError(), "Error at kernel launch");
          }

          FOLD // Synchronisation  .
          {
            CUDA_SAFE_CALL(hipEventRecord(batch->multComp, batch->multStream),"Recording event: multComp");
          }
        }
        else if ( batch->flag & FLAG_MUL_STK   )  // Do the multiplications one stack  at a time  .
        {
#ifdef SYNCHRONOUS
          cuFfdotStack* pStack = NULL;  // Previous stack
#endif

          // Multiply this entire stack in one block
          for (int ss = 0; ss < batch->noStacks; ss++)
          {
            cuFfdotStack* cStack = &batch->stacks[ss];

            FOLD // Synchronisation  .
            {
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->multStream, cStack->prepComp,  0),  "Waiting for GPU to be ready to copy data to device.");  // Need input data

              if ( (batch->flag & FLAG_CUFFT_CB_OUT) )
              {
                // CFF output callback has its own data so can start once FFT is complete
                CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->multStream, cStack->ifftComp, 0),  "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete
              }
              else
              {
                // Have to wait for search to finish reading data
                CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->multStream, batch->searchComp, 0),  "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete
              }

              if ( (batch->retType & CU_STR_PLN) && !(batch->flag & FLAG_CUFFT_CB_OUT) )
              {
                CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->multStream, batch->candCpyComp, 0), "Waiting for GPU to be ready to copy data to device.");  // Multiplication will change the plain
              }

#ifdef SYNCHRONOUS
              // Wait for all the input FFT's to complete
              for (int ss = 0; ss < batch->noStacks; ss++)
              {
                cuFfdotStack* cStack2 = &batch->stacks[ss];
                hipStreamWaitEvent(cStack->multStream, cStack2->prepComp, 0);
              }

              // Wait for the previous multiplication to complete
              if ( pStack != NULL )
                hipStreamWaitEvent(cStack->multStream, pStack->multComp, 0);
#endif
            }

            FOLD // Timing event  .
            {
#ifdef TIMING
              CUDA_SAFE_CALL(hipEventRecord(cStack->multInit, cStack->multStream),"Recording event: multInit");
#endif
            }

            FOLD // Call kernel(s)  .
            {
              if      ( cStack->flag & FLAG_MUL_00 )
              {
                mult00(cStack->multStream, batch, ss);
              }
              else if ( cStack->flag & FLAG_MUL_21 )
              {
                mult21_f(cStack->multStream, batch, ss);
              }
              else if ( cStack->flag & FLAG_MUL_22 )
              {
                mult22_f(cStack->multStream, batch, ss);
              }
              else if ( cStack->flag & FLAG_MUL_23 )
              {
                mult23_f(cStack->multStream, batch, ss);
              }
              else
              {
                fprintf(stderr,"ERROR: No valid stack multiplication specified. Line %i in %s.\n", __LINE__, __FILE__);
                exit(EXIT_FAILURE);
              }

              // Run message
              CUDA_SAFE_CALL(hipGetLastError(), "Error at kernel launch (mult7)");
            }

            FOLD // Synchronisation  .
            {
              hipEventRecord(cStack->multComp, cStack->multStream);

#ifdef SYNCHRONOUS
              pStack = cStack;
#endif
            }
          }
        }
        else if ( batch->flag & FLAG_MUL_PLN ) 	  // Do the multiplications one plain  at a time  .
        {
          mult10(batch);
        }
        else
        {
          fprintf(stderr, "ERROR: multiplyBatch not templated for this type of multiplication.\n");
        }
      }

      FOLD // Inverse FFT the f-∂f plain  .
      {

#ifdef STPMSG
        printf("\t\tInverse FFT\n");
#endif

#ifdef SYNCHRONOUS
        cuFfdotStack* pStack = NULL;  // Previous stack
#endif

        // Copy fft data to device
        //for (int ss = plains->noStacks-1; ss >= 0; ss-- )
        for (int ss = 0; ss < batch->noStacks; ss++)
        {
          cuFfdotStack* cStack = &batch->stacks[ss];

#ifdef STPMSG
          printf("\t\t\tStack %i\n",ss);
#endif

          FOLD // Synchronisation  .
          {
#ifdef STPMSG
            printf("\t\t\t\tSynchronisation\n");
#endif
            hipStreamWaitEvent(cStack->fftPStream, cStack->multComp, 0);
            hipStreamWaitEvent(cStack->fftPStream, batch->multComp,  0);

            if ( (batch->retType & CU_STR_PLN) && (batch->flag & FLAG_CUFFT_CB_OUT) )
            {
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->fftPStream, batch->candCpyComp, 0), "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete
            }

            if ( batch->flag & FLAG_SS_INMEM  )
            {
              CUDA_SAFE_CALL(hipStreamWaitEvent(cStack->fftPStream, cStack->ifftMemComp, 0), "Waiting for GPU to be ready to copy data to device.");  // This will overwrite the plain so search must be compete
            }

#ifdef SYNCHRONOUS
            // Wait for all the multiplications to complete
            for (int ss = 0; ss< batch->noStacks; ss++)
            {
              cuFfdotStack* cStack2 = &batch->stacks[ss];
              hipStreamWaitEvent(cStack->fftPStream, cStack2->multComp, 0);
            }

            // Wait for the previous fft to complete
            if ( pStack != NULL )
              hipStreamWaitEvent(cStack->fftPStream, pStack->ifftComp, 0);
#endif
          }

          FOLD // Call the inverse CUFFT  .
          {
            //#pragma omp critical
            {
#ifdef STPMSG
              printf("\t\t\t\tCall the inverse CUFFT\n");
#endif
              FOLD // Timing  .
              {
#ifdef TIMING
                hipEventRecord(cStack->ifftInit, cStack->fftPStream);
#endif
              }

              rVals* rVal;
              rVal = &((*batch->rInput)[0][0]);

              FOLD // Set store FFT callback  .
              {
                if ( batch->flag & FLAG_CUFFT_CB_OUT )
                {
                  //                  if ( batch->flag & FLAG_SS_INMEM  )
                  //                  {
                  //                    CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&batch->h_stCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&cStack->d_plainPowers ),"");
                  //                  }
                  //                  else
                  {
                    CUFFT_SAFE_CALL(hipfftXtSetCallback(cStack->plnPlan, (void **)&batch->h_stCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&cStack->d_plainPowers ),"");
                  }
                }
              }

              FOLD // Call the FFT  .
              {
                CUFFT_SAFE_CALL(hipfftSetStream(cStack->plnPlan, cStack->fftPStream),  "Error associating a CUFFT plan with multStream.");
                CUFFT_SAFE_CALL(hipfftExecC2C(cStack->plnPlan, (hipfftComplex *) cStack->d_plainData, (hipfftComplex *) cStack->d_plainData, HIPFFT_BACKWARD),"Error executing CUFFT plan.");
              }

              FOLD // Synchronisation  .
              {
                hipEventRecord(cStack->ifftComp, cStack->fftPStream);
              }

              FOLD // Copy data to device plain  .
              {
                if ( batch->flag & FLAG_SS_INMEM  )
                {
                  FOLD // Copy memory on the device  .
                  {
                    if ( batch->flag & FLAG_HALF )
                    {
                      copyIFFTtoPln<half>( batch, cStack );
                    }
                    else
                    {
                      copyIFFTtoPln<float>( batch, cStack );
                    }

                    CUDA_SAFE_CALL(hipGetLastError(), "Error at IFFT - hipMemcpy2DAsync");
                  }
                }

              }

#ifdef SYNCHRONOUS
              pStack = cStack;
#endif
            }
          }

#ifdef STPMSG
          printf("\t\t\tDone\n",ss);
#endif
        }
      }
    }

    batch->haveInput    = 0;
    batch->haveConvData = 1;

    nvtxRangePop();
  }

  // Set the r-values and width for the next iteration when we will be doing the actual Add and Search
  cycleRlists(batch);
}

