#include "hip/hip_runtime.h"
#include "cuda_accel_SS.h"

#include <semaphore.h>

#include <hipfft/hipfft.h>
#include <algorithm>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <nvToolsExt.h>
#include <nvToolsExtCudaRt.h>

#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"
#include "cuda_accel_SS.h"

//======================================= Constant memory =================================================\\

__device__ __constant__ int       YINDS[MAX_YINDS];                   ///< The harmonic related Y index for each plane
__device__ __constant__ float     POWERCUT_STAGE[MAX_HARM_NO];        ///<
__device__ __constant__ float     NUMINDEP_STAGE[MAX_HARM_NO];        ///<

__device__ __constant__ int       HEIGHT_STAGE[MAX_HARM_NO];          ///< Plane heights in stage order
__device__ __constant__ int       STRIDE_STAGE[MAX_HARM_NO];          ///< Plane strides in stage order
__device__ __constant__ int       PSTART_STAGE[MAX_HARM_NO];          ///< Start offset of good points in a plane, stage order

__device__ __constant__ void*     PLN_START;                          ///< A pointer to the start of the in-mem plane
__device__ __constant__ uint      PLN_STRIDE;                         ///< The strided in units of the in-mem plane
__device__ __constant__ int       NO_STEPS;                           ///< The number of steps used in the search  -  NB: this is specific to the batch not the search, but its only used in the inmem search!
__device__ __constant__ int       ALEN;                               ///< CUDA copy of the accelLen used in the search

//====================================== Constant variables  ===============================================\\

__device__ const float FRAC_STAGE[16]     =  { 1.0000f, 0.5000f, 0.7500f, 0.2500f, 0.8750f, 0.6250f, 0.3750f, 0.1250f, 0.9375f, 0.8125f, 0.6875f, 0.5625f, 0.4375f, 0.3125f, 0.1875f, 0.0625f } ;

//__device__ const float FRAC_STAGE[16]     =  { 1.0000f, 0.5000f, 0.2500f, 0.7500f, 0.1250f, 0.3750f, 0.6250f, 0.8750f, 0.0625f, 0.1875f, 0.3125f, 0.4375f, 0.5625f, 0.6875f, 0.8125f, 0.9375f } ;

__device__ const float FRAC_HARM[16]      =  { 1.0f, 0.9375f, 0.875f, 0.8125f, 0.75f, 0.6875f, 0.625f, 0.5625f, 0.5f, 0.4375f, 0.375f, 0.3125f, 0.25f, 0.1875f, 0.125f, 0.0625f } ;
__device__ const short STAGE[5][2]        =  { {0,0}, {1,1}, {2,3}, {4,7}, {8,15} } ;
__device__ const short CHUNKSZE[5]        =  { 4, 8, 8, 8, 8 } ;

//======================================= Global variables  ================================================\\

//int    inMemSrchSz = 32768; // Deprecated

//========================================== Functions  ====================================================\\

/** Return x such that 2**x = n
 *
 * @param n
 * @return
 */
__host__ __device__ inline int twon_to_index(int n)
{
  int x = 0;

  while (n > 1)
  {
    n >>= 1;
    x++;
  }
  return x;
}

template<int64_t FLAGS>
__device__ inline int getY(int planeY, const int noSteps,  const int step, const int planeHeight = 0 )
{
  // Calculate y indice from interleave method
  if      ( FLAGS & FLAG_ITLV_ROW )
  {
    return planeY * noSteps + step;
  }
  else
  {
    return planeY + planeHeight*step;
  }
}

template<int64_t FLAGS>
__device__ inline float getPower(const int ix, const int iy, hipTextureObject_t tex, fcomplexcu* base, const int stride)
{
  if  ( (FLAGS & FLAG_SAS_TEX ) )
  {
    const float2 cmpf = tex2D < float2 > (tex, ix, iy);
    return (cmpf.x * cmpf.x + cmpf.y * cmpf.y);
  }
  else
  {
    const fcomplexcu cmpc  = base[iy*stride+ix];
    return (cmpc.r * cmpc.r + cmpc.i * cmpc.i);
  }
}

/** Main loop down call
 *
 * This will asses and call the correct templated kernel
 *
 * @param dimGrid
 * @param dimBlock
 * @param stream
 * @param batch
 */
__host__ void add_and_searchCU3(hipStream_t stream, cuFFdotBatch* batch )
{
  const int64_t FLAGS = batch->flags ;

  if            ( (FLAGS & FLAG_CUFFT_CB_POW) && (FLAGS & FLAG_SAS_TEX) && (FLAGS & FLAG_TEX_INTERP) )
  {
    fprintf(stderr,"ERROR: Invalid sum and search kernel. Line %i in %s\n", __LINE__, __FILE__ );
    exit(EXIT_FAILURE);
    //add_and_searchCU3_PT_f(stream, batch );
  }
  else
  {
    if      ( FLAGS & FLAG_SS_00 )
    {
      add_and_searchCU00(stream, batch );
    }
    else if ( FLAGS & FLAG_SS_10 )
    {
      add_and_searchCU31(stream, batch );
    }
    //		Depricated
    //
    //    else if ( FLAGS & FLAG_SS_20 )
    //    {
    //      add_and_searchCU32(stream, batch );
    //    }
    //    else if ( FLAGS & FLAG_SS_30 )
    //    {
    //      add_and_searchCU33(stream, batch );
    //    }
    else
    {
      fprintf(stderr,"ERROR: Invalid sum and search kernel.\n");
      exit(EXIT_FAILURE);
    }
  }
}

int setConstVals( cuFFdotBatch* batch, int numharmstages, float *powcut, long long *numindep )
{
  void *dcoeffs;

  FOLD // Calculate Y coefficients and copy to constant memory  .
  {
    int noHarms         = batch->cuSrch->noSrchHarms;

    if ( ((batch->hInfos->height + INDS_BUFF) * noHarms) > MAX_YINDS)
    {
      printf("ERROR! YINDS to small!");
    }

    freeNull(batch->cuSrch->yInds);
    batch->cuSrch->yInds    = (int*) malloc( (batch->hInfos->height + INDS_BUFF) * noHarms * sizeof(int));
    int *indsY            = batch->cuSrch->yInds;
    int bace              = 0;

    batch->hInfos->yInds  = 0;

    int zmax = batch->hInfos->zmax ;

    for (int ii = 0; ii < noHarms; ii++)
    {
      if ( ii == 0 )
      {
        for (int j = 0; j < batch->hInfos->height; j++)
        {
          indsY[bace + j] = j;
        }
      }
      else
      {
        float harmFrac  = HARM_FRAC_STAGE[ii];
        int sZmax;

        if ( batch->flags & FLAG_SS_INMEM )
        {
          sZmax = zmax;
        }
        else
        {
          int sIdx  = batch->cuSrch->sIdx[ii];
          sZmax = batch->hInfos[sIdx].zmax;
        }

        for (int j = 0; j < batch->hInfos->height; j++)
        {
          int zz    = -zmax + j* ACCEL_DZ;
          int subz  = calc_required_z( harmFrac, zz );
          int zind  = index_from_z( subz, -sZmax );

          indsY[bace + j] = zind;
        }
      }

      if ( ii < batch->noSrchHarms)
      {
        batch->hInfos[ii].yInds = bace;
      }

      bace += batch->hInfos->height;

      // Buffer with last value
      for (int j = 0; j < INDS_BUFF; j++)
      {
        indsY[bace + j] = indsY[bace + j-1];
      }

      bace += INDS_BUFF;
    }

    hipGetSymbolAddress((void **)&dcoeffs, YINDS);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, indsY, bace*sizeof(int), hipMemcpyHostToDevice, batch->stacks->initStream),                      "Copying Y indices to device");
  }

  FOLD // copy power cutoff values  .
  {
    if ( powcut )
    {
      hipGetSymbolAddress((void **)&dcoeffs, POWERCUT_STAGE);
      CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, powcut, numharmstages * sizeof(float), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying power cutoff to device");
    }
    else
    {
      float pw[5];
      for ( int i = 0; i < 5; i++)
      {
        pw[i] = 0;
      }
      hipGetSymbolAddress((void **)&dcoeffs, POWERCUT_STAGE);
      CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &pw, 5 * sizeof(float), hipMemcpyHostToDevice, batch->stacks->initStream),         "Copying power cutoff to device");
    }
  }

  FOLD // number of independent values  .
  {
    if (numindep)
    {
      hipGetSymbolAddress((void **)&dcoeffs, NUMINDEP_STAGE);
      CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, numindep, numharmstages * sizeof(long long), hipMemcpyHostToDevice, batch->stacks->initStream),  "Copying stages to device");
    }
    else
    {
      long long numi[5];
      for ( int i = 0; i < 5; i++)
      {
        numi[i] = 0;
      }
      hipGetSymbolAddress((void **)&dcoeffs, NUMINDEP_STAGE);
      CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &numi, 5 * sizeof(long long), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying stages to device");

    }
  }

  FOLD // Some other values  .
  {
    hipGetSymbolAddress((void **)&dcoeffs, NO_STEPS);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs,  &(batch->noSteps),  sizeof(int), hipMemcpyHostToDevice, batch->stacks->initStream),  "Copying number of steps");

    hipGetSymbolAddress((void **)&dcoeffs, ALEN);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs,  &(batch->accelLen), sizeof(int), hipMemcpyHostToDevice, batch->stacks->initStream),  "Copying accelLen");
  }

  FOLD // In-mem plane details  .
  {
    if ( batch->flags & FLAG_SS_INMEM  )
    {
      hipGetSymbolAddress((void **)&dcoeffs, PLN_START);
      CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &(batch->cuSrch->d_planeFull),  sizeof(void*),  hipMemcpyHostToDevice, batch->stacks->initStream),  "Copying accelLen");

      hipGetSymbolAddress((void **)&dcoeffs, PLN_STRIDE);
      CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &(batch->cuSrch->inmemStride),  sizeof(uint),   hipMemcpyHostToDevice, batch->stacks->initStream),  "Copying accelLen");
    }
  }

  FOLD // Set other stage specific values  .
  {
    int height[MAX_HARM_NO];
    int stride[MAX_HARM_NO];
    int pStart[MAX_HARM_NO];

    FOLD // Set values  .
    {
      for (int i = 0; i < batch->noGenHarms; i++)
      {
        int sIdx  = batch->cuSrch->sIdx[i];
        height[i] = batch->hInfos[sIdx].height;
        stride[i] = batch->hInfos[sIdx].width;
        pStart[i] = batch->hInfos[sIdx].kerStart;
      }

      FOLD // The rest  .
      {
        int zeroZMax    = batch->hInfos->zmax;

        presto_interp_acc accuracy = LOWACC;
        if ( batch->flags & FLAG_KER_HIGH )
          accuracy = HIGHACC;

        for (int i = batch->noGenHarms; i < MAX_HARM_NO; i++)
        {
          float harmFrac  = HARM_FRAC_FAM[i];
          int zmax        = calc_required_z(harmFrac, zeroZMax);
          height[i]       = (zmax / ACCEL_DZ) * 2 + 1;
          stride[i]       = calc_fftlen3(harmFrac, zmax, batch->accelLen, accuracy);
          pStart[i]       = -1;
        }
      }
    }

    hipGetSymbolAddress((void **)&dcoeffs, HEIGHT_STAGE);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &height, MAX_HARM_NO * sizeof(int), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying stages to device");

    hipGetSymbolAddress((void **)&dcoeffs, STRIDE_STAGE);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &stride, MAX_HARM_NO * sizeof(int), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying stages to device");

    hipGetSymbolAddress((void **)&dcoeffs, PSTART_STAGE);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &pStart, MAX_HARM_NO * sizeof(int), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying stages to device");
  }

  CUDA_SAFE_CALL(hipGetLastError(), "Preparing the constant memory.");

  return (1);
}

void SSKer(cuFFdotBatch* batch)
{
  infoMSG(2,3,"Sum & Search\n");

  nvtxRangePush("S&S Ker");

  FOLD // Do synchronisations  .
  {
    infoMSG(3,4,"pre synchronisations\n");

    for (int ss = 0; ss < batch->noStacks; ss++)
    {
      cuFfdotStack* cStack = &batch->stacks[ss];

      if ( batch->flags & FLAG_SS_INMEM )
      {
        hipStreamWaitEvent(batch->srchStream, cStack->ifftMemComp,   0);
      }
      else
      {
        hipStreamWaitEvent(batch->srchStream, cStack->ifftComp,      0);
      }
    }
  }

  FOLD // Timing event  .
  {
    if ( batch->flags & FLAG_TIME ) // Timing event
    {
      CUDA_SAFE_CALL(hipEventRecord(batch->searchInit,  batch->srchStream),"Recording event: searchInit");
    }
  }



  FOLD // Call the SS kernel  .
  {
    infoMSG(3,4,"kernel\n");

    if ( batch->retType & CU_POWERZ_S )
    {
      if      ( batch->flags & FLAG_SS_STG )
      {
        add_and_searchCU3(batch->srchStream, batch );
      }
      else if ( batch->flags & FLAG_SS_INMEM )
      {
        add_and_search_IMMEM(batch);
      }
      else
      {
        fprintf(stderr,"ERROR: function %s is not setup to handle this type of search.\n",__FUNCTION__);
        exit(EXIT_FAILURE);
      }
    }
    else
    {
      fprintf(stderr,"ERROR: function %s is not setup to handle this type of return data for GPU accel search\n",__FUNCTION__);
      exit(EXIT_FAILURE);
    }
    CUDA_SAFE_CALL(hipGetLastError(), "At SSKer kernel launch");
  }

  FOLD // Synchronisation  .
  {
    infoMSG(3,4,"post synchronisations\n");

    CUDA_SAFE_CALL(hipEventRecord(batch->searchComp,  batch->srchStream),"Recording event: searchComp");
  }

  nvtxRangePop();
}

/** Process an individual candidate  .
 *
 */
int procesCanidate(resultData* res, double rr, double zz, double poww, double sig, int stage, int numharm)
{
  cuSearch*	cuSrch	= res->cuSrch;

  // Adjust r and z for the number of harmonics
  rr    /=  (double)numharm ;
  zz    =   ( zz * ACCEL_DZ - res->zMax ) / (double)numharm ;

  if ( rr < cuSrch->SrchSz->searchRHigh )
  {
    if ( !(res->flags & FLAG_SIG_GPU) ) // Do the sigma calculation  .
    {
      sig     = candidate_sigma_cu(poww, numharm, cuSrch->numindep[stage]);
    }

    if      ( res->cndType & CU_STR_LST     )
    {
      if ( res->flags & FLAG_THREAD )
      {
        // Thread safe
	pthread_mutex_lock(&cuSrch->threasdInfo->candAdd_mutex);
	GSList *candsGPU	= (GSList*)cuSrch->h_candidates;
	int     added		= 0;
	cuSrch->h_candidates	= insert_new_accelcand(candsGPU, poww, sig, numharm, rr, zz, &added );
        (*res->noResults)++;
        pthread_mutex_unlock(&cuSrch->threasdInfo->candAdd_mutex);
      }
      else
      {
	GSList *candsGPU	= (GSList*)cuSrch->h_candidates;
	int     added		= 0;
	cuSrch->h_candidates	= insert_new_accelcand(candsGPU, poww, sig, numharm, rr, zz, &added );
        (*res->noResults)++;
      }
    }
    else if ( res->cndType & CU_STR_ARR     )
    {
      double  rDiff = rr - cuSrch->SrchSz->searchRLow ;
      long    grIdx;   /// The index of the candidate in the global list

      if ( res->flags & FLAG_STORE_EXP )
      {
        grIdx = floor(rDiff*ACCEL_RDR);
      }
      else
      {
        grIdx = floor(rDiff);
      }

      if ( grIdx >= 0 && grIdx < cuSrch->SrchSz->noOutpR )      // Valid index  .
      {
        if ( res->flags & FLAG_STORE_ALL )                      // Store all stages  .
        {
          grIdx += stage * (cuSrch->SrchSz->noOutpR);           // Stride by size
        }

        if ( res->cndType & CU_CANDFULL )
        {
          initCand* candidate = &((initCand*)cuSrch->h_candidates)[grIdx];

          // this sigma is greater than the current sigma for this r value
          if ( candidate->sig < sig )
          {
            if ( res->flags & FLAG_THREAD )
            {
              pthread_mutex_lock(&cuSrch->threasdInfo->candAdd_mutex);
              if ( candidate->sig < sig ) // Check again
              {
                if ( candidate->sig == 0 )
                  (*res->noResults)++;

                candidate->sig      = sig;
                candidate->power    = poww;
                candidate->numharm  = numharm;
                candidate->r        = rr;
                candidate->z        = zz;
              }
              pthread_mutex_unlock(&cuSrch->threasdInfo->candAdd_mutex);
            }
            else
            {
              if ( candidate->sig == 0 )
                (*res->noResults)++;

              candidate->sig      = sig;
              candidate->power    = poww;
              candidate->numharm  = numharm;
              candidate->r        = rr;
              candidate->z        = zz;
            }
          }
        }
        else
        {
          fprintf(stderr,"ERROR: function %s requires storing full candidates.\n",__FUNCTION__);
          exit(EXIT_FAILURE);
        }
      }
    }
    else if ( res->cndType & CU_STR_QUAD    )
    {
      candTree* qt = (candTree*)cuSrch->h_candidates;

      initCand* candidate     	= new initCand;

      candidate->sig      	= sig;
      candidate->power    	= poww;
      candidate->numharm  	= numharm;
      candidate->r        	= rr;
      candidate->z        	= zz;

      (*res->noResults)++;

      qt->insert(candidate);
    }
    else
    {
      fprintf(stderr,"ERROR: Function %s unknown candidate storage type.\n", __FUNCTION__);
      exit(EXIT_FAILURE);
    }
  }

  return (0);
}

/** Process the results of the search this is usually run in a separate CPU thread  .
 *
 * This function is meant to be the entry of a separate thread
 *
 */
void* processSearchResults(void* ptr)
{
  resultData*	res	= (resultData*)ptr;
  cuSearch*	cuSrch	= res->cuSrch;

  struct timeval start, end;      		// Timing variables

  if ( res->flags & FLAG_TIME ) 		// Timing  .
  {
    gettimeofday(&start, NULL);
  }

  double poww, sig;
  double rr, zz;
  int numharm;
  int idx;

  for ( int stage = 0; stage < cuSrch->noHarmStages; stage++ )
  {
    numharm       = (1<<stage);
    float cutoff  = cuSrch->powerCut[stage];

    for ( int y = res->y0; y < res->y1; y++ )
    {
      for ( int x = res->x0; x < res->x1; x++ )
      {
        poww      = 0;
        sig       = 0;
        zz        = 0;

        idx = stage*res->xStride*res->yStride + y*res->xStride + x ;

        if      ( res->retType & CU_CANDMIN     )
        {
          candMin candM         = ((candMin*)res->retData)[idx];

          if ( candM.power > poww )
          {
            sig                 = candM.power;
            poww                = candM.power;
            zz                  = candM.z;
          }
        }
        else if ( res->retType & CU_POWERZ_S    )
        {
          candPZs candM         = ((candPZs*)res->retData)[idx];

          if ( candM.value > poww )
          {
            sig                 = candM.value;
            poww                = candM.value;
            zz                  = candM.z;
          }
        }
        else if ( res->retType & CU_CANDBASC    )
        {
          accelcandBasic candB  = ((accelcandBasic*)res->retData)[idx];

          if ( candB.sigma > poww )
          {
            poww                = candB.sigma;
            sig                 = candB.sigma;
            zz                  = candB.z;
          }
        }
        else if ( res->retType & CU_FLOAT       )
        {
          float val  = ((float*)res->retData)[idx];

          if ( val > cutoff )
          {
            poww                = val;
            sig                 = val;
            zz                  = y;
          }
        }
        else if ( res->retType & CU_HALF        )
        {
          float val  = half2float( ((ushort*)res->retData)[idx] );

          if ( val > cutoff )
          {
            poww                  = val;
            sig                   = val;
            zz                    = y;
          }
        }
        else
        {
          fprintf(stderr,"ERROR: function %s requires accelcandBasic\n",__FUNCTION__);
          if ( res->flags & FLAG_THREAD )
          {
            sem_trywait(&(cuSrch->threasdInfo->running_threads));
          }
          exit(EXIT_FAILURE);
        }

        if ( poww > 0 )
        {
          if ( isnan(poww) )
          {
            rr      = res->rLow + x * ACCEL_DR / numharm ;
            fprintf(stderr, "CUDA search returned an NAN power at bin %.3f.\n", rr);
          }
          else
          {
            if ( isinf(poww) )
            {
              if ( res->flags & FLAG_HALF )
              {
                poww          = 6.55e4;      // Max 16 bit float value
                double rPos   = res->rLow + x * ACCEL_DR / numharm ;
                fprintf(stderr,"WARNING: Search return inf power at bin %.2f, dropping to %.2e. If this persists consider using single precision floats.\n", rPos, poww);
              }
              else
              {
                poww          = 3.402823e38; // Max 32 bit float value
                double rPos   = res->rLow + x * ACCEL_DR / numharm ;
                fprintf(stderr,"WARNING: Search return inf power at bin %.2f. This is probably an error as you are using single precision floats.\n", rPos);
              }
            }

            if ( zz < 0 || zz >= res->zMax+1)
            {
              double rPos   = res->rLow + x * ACCEL_DR / numharm ;
              fprintf(stderr,"ERROR: invalid z value found at bin %.2f.\n", rPos);
            }
            else
            {
              // This value is above the threshold
              rr      = res->rLow + x * ACCEL_DR ;
              procesCanidate(res, rr, zz, poww, sig, stage, numharm ) ;
            }
          }
        }
      }
    }
  }

  if ( res->flags & FLAG_TIME ) // Timing  .
  {
    gettimeofday(&end, NULL);
    float time =  ((end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec))*1e-3  ;

    if ( res->flags & FLAG_THREAD )
    {
      pthread_mutex_lock(&cuSrch->threasdInfo->candAdd_mutex);
      res->resultTime[0] += time;
      pthread_mutex_unlock(&cuSrch->threasdInfo->candAdd_mutex);
    }
    else
    {
      res->resultTime[0] += time;
    }
  }

  // Decrease the count number of running threads
  if ( res->flags & FLAG_THREAD )
  {
    sem_trywait(&(cuSrch->threasdInfo->running_threads));
  }

  FOLD // Free memory
  {
    if ( res->flags & FLAG_THREAD )
      free (res->retData);
    free (res);
  }

  return (NULL);
}

/** Process the search results for the batch  .
 * This usually spawns a separate CPU thread to do the sigma calculations
 */
void processSearchResults(cuFFdotBatch* batch)
{
  if ( (*batch->rAraays)[batch->rActive][0][0].numrs )
  {
    struct timeval start, end;          // Timing variables
    resultData* thrdDat;

    infoMSG(1,2,"Process previous results\n");

    nvtxRangePush("CPU Process results");

    if ( batch->flags & FLAG_TIME )    // Timing  .
    {
      gettimeofday(&start, NULL);
    }

    FOLD // Allocate temporary memory to copy results back to  .
    {
      nvtxRangePush("malloc");

      thrdDat = new resultData;     // A data structure to hold info for the thread processing the results
      memset(thrdDat, 0, sizeof(resultData) );

      if ( batch->flags & FLAG_THREAD )
      {
        thrdDat->retData = (void*)malloc(batch->retDataSize);
      }

      nvtxRangePop();
    }

    FOLD // Initialise data structure  .
    {
      rVals* rVal = &(*batch->rAraays)[batch->rActive][0][0];

      infoMSG(3,3,"Initialise data structure\n");

      thrdDat->cuSrch		= batch->cuSrch;
      thrdDat->cndType  	= batch->cndType;
      thrdDat->rLow       	= rVal->drlo;
      thrdDat->retType  	= batch->retType;
      thrdDat->flags    	= batch->flags;
      thrdDat->zMax      	= batch->hInfos->zmax;
      thrdDat->resultTime 	= batch->resultTime;
      thrdDat->noResults  	= &batch->noResults;

      thrdDat->x0      		= 0;
      thrdDat->x1		= 0;
      thrdDat->y0		= 0;
      thrdDat->y1		= batch->ssSlices;

      thrdDat->xStride		= batch->strideOut;
      thrdDat->yStride		= batch->ssSlices;

      if ( !(batch->flags & FLAG_SS_INMEM) )
      {
        // Multi-step

        thrdDat->xStride	*= batch->noSteps;

        for ( int step = 0; step < batch->noSteps; step++) // Loop over steps  .
        {
          rVals* rVal		= &(*batch->rAraays)[batch->rActive][step][0];
          thrdDat->x1		+= rVal->numrs;                 // These should all be Acelllen but there may be the case of the last step!
        }
      }
      else
      {
        // NB: In-mem has only one step
        thrdDat->x1		= rVal->numrs;
      }

      if ( thrdDat->x1 > thrdDat->xStride )
      {
        fprintf(stderr,"ERROR: Number of elements of greater than stride. In function %s  \n",__FUNCTION__);
        exit(EXIT_FAILURE);
      }
    }

    FOLD // Timing  .
    {
      if ( batch->flags & FLAG_TIME )
      {
        gettimeofday(&end, NULL);
        float time = ((end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec))*1e-3  ;
        int idx = MIN(2, batch->noStacks-1);

        pthread_mutex_lock(&batch->cuSrch->threasdInfo->candAdd_mutex);
        batch->resultTime[idx] += time;
        pthread_mutex_unlock(&batch->cuSrch->threasdInfo->candAdd_mutex);
      }
    }

    FOLD // Copy data from device  .
    {
      FOLD // A blocking synchronisation to ensure results are ready to be proceeded by the host  .
      {
        infoMSG(3,4,"pre synchronisation [blocking] candCpyComp\n");

        nvtxRangePush("EventSynch");
        CUDA_SAFE_CALL(hipEventSynchronize(batch->candCpyComp), "At a blocking synchronisation. This is probably a error in one of the previous asynchronous CUDA calls.");
        nvtxRangePop();
      }

      FOLD // Timing  .
      {
        if ( batch->flags & FLAG_TIME )
        {
          gettimeofday(&start, NULL);
        }
      }

      FOLD // Copy data  .
      {
        infoMSG(3,3,"copy to temporary memory\n");

        nvtxRangePush("memcpy");

        void *gpuOutput;

        if ( !(batch->flags & FLAG_SYNCH) && (batch->flags & FLAG_SS_INMEM) )
        {
          gpuOutput = batch->h_outData2;
        }
        else
        {
          gpuOutput = batch->h_outData1;
        }

        if ( batch->flags & FLAG_THREAD )
        {
          memcpy(thrdDat->retData, gpuOutput, batch->retDataSize);

          FOLD // Synchronisation  .
          {
            infoMSG(3,4,"synchronise\n");

            // This will allow kernels to run while the CPU continues
            CUDA_SAFE_CALL(hipEventRecord(batch->processComp, batch->srchStream),"Recording event: processComp");
          }
        }
        else
        {
          thrdDat->retData = gpuOutput;
        }

        nvtxRangePop();
      }

      FOLD // Timing 1  .
      {
        if ( batch->flags & FLAG_TIME )
        {
          gettimeofday(&end, NULL);
          float time =  ((end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec))*1e-3  ;
          int idx = MIN(1, batch->noStacks-1);

          pthread_mutex_lock(&batch->cuSrch->threasdInfo->candAdd_mutex);
          batch->resultTime[idx] += time;
          pthread_mutex_unlock(&batch->cuSrch->threasdInfo->candAdd_mutex);
        }
      }
    }

    FOLD // ADD candidates to global list potently in a separate thread  .
    {
      if ( batch->flags & FLAG_SYNCH )
      {
        nvtxRangePush("Thread");
      }

      if ( batch->flags & FLAG_THREAD ) 	// Create thread  .
      {
        infoMSG(3,4,"create thread\n");

        sem_post(&batch->cuSrch->threasdInfo->running_threads); // Increase the count number of running threads, processSearchResults will decrease it when its finished

        pthread_t thread;
        int  iret1 = pthread_create( &thread, NULL, processSearchResults, (void*) thrdDat);

        if (iret1)
        {
          fprintf(stderr,"Error - pthread_create() return code: %d\n", iret1);
          exit(EXIT_FAILURE);
        }

        if ( batch->flags & FLAG_SYNCH )
        {
          void *status;
          if ( pthread_join(thread, &status) )
          {
            fprintf(stderr,"ERROR: Failed to join results thread.\n");
            exit(EXIT_FAILURE);
          }
        }
      }
      else                              	// Just call the function  .
      {
        infoMSG(3,4,"non thread\n");

        processSearchResults( (void*) thrdDat );

        if ( !(batch->flags & FLAG_THREAD) )
        {
          // Not using threading so using original memory location

          FOLD // Synchronisation  .
          {
            infoMSG(3,4,"synchronise\n");

            // This will allow kernels to run while the CPU continues
            CUDA_SAFE_CALL(hipEventRecord(batch->processComp, batch->srchStream),"Recording event: processComp");
          }
        }
      }

      if ( batch->flags & FLAG_SYNCH )
      {
        nvtxRangePop();
      }
    }

    nvtxRangePop();
  }
}

void getResults(cuFFdotBatch* batch)
{
  // Timing
  if ( batch->flags & FLAG_TIME )
  {
    if ( (*batch->rAraays)[batch->rActive+1][0][0].numrs )
    {
      // Sum & Search kernel
      timeEvents( batch->candCpyInit, batch->candCpyComp, &batch->copyD2HTime[0],   "Copy device to host");
    }
  }

  if ( (*batch->rAraays)[batch->rActive][0][0].numrs )
  {
    infoMSG(1,2,"Copy results from device to host\n");

    FOLD // Synchronisations  .
    {
      infoMSG(3,3,"pre synchronise\n");

      CUDA_SAFE_CALL(hipStreamWaitEvent(batch->resStream, batch->searchComp,  0),"Waiting on event searchComp");
      CUDA_SAFE_CALL(hipStreamWaitEvent(batch->resStream, batch->processComp, 0),"Waiting on event processComp");
    }

    FOLD // Timing event  .
    {
      if ( batch->flags & FLAG_TIME )
      {
        CUDA_SAFE_CALL(hipEventRecord(batch->candCpyInit,  batch->srchStream),"Recording event: candCpyInit");
      }
    }

    FOLD // Copy relevant data back  .
    {
      infoMSG(3,3,"Async memcpy\n");

      if      ( batch->retType & CU_STR_PLN )
      {
        CUDA_SAFE_CALL(hipMemcpyAsync(batch->h_outData1, batch->d_planePowr, batch->pwrDataSize, hipMemcpyDeviceToHost, batch->resStream), "Failed to copy results back");
      }
      else
      {
        CUDA_SAFE_CALL(hipMemcpyAsync(batch->h_outData1, batch->d_outData1,  batch->retDataSize, hipMemcpyDeviceToHost, batch->resStream), "Failed to copy results back");
      }

      CUDA_SAFE_CALL(hipGetLastError(), "Copying results back from device.");
    }

    FOLD // Synchronisations  .
    {
      infoMSG(3,3,"post synchronise\n");

      CUDA_SAFE_CALL(hipEventRecord(batch->candCpyComp, batch->resStream),"Recording event: readComp");
    }

    CUDA_SAFE_CALL(hipGetLastError(), "Leaving getResults.");
  }
}

void sumAndSearch(cuFFdotBatch* batch)        // Function to call to SS and process data in normal steps  .
{
  // Timing
  if ( batch->flags & FLAG_TIME )
  {
    if ( (*batch->rAraays)[batch->rActive+1][0][0].numrs )
    {
      // Sum & Search kernel
      timeEvents( batch->searchInit, batch->searchComp, &batch->searchTime[0],   "Sum & Search");
    }
  }

  // Sum and search the IFFT'd data  .
  if ( (*batch->rAraays)[batch->rActive][0][0].numrs )
  {
    infoMSG(1,2,"Sum & Search\n");

    if      ( batch->retType 	& CU_STR_PLN 	  )
    {
      // Nothing!
    }
    else if ( batch->flags    & FLAG_SS_INMEM )
    {
      // NOTHING
    }
    else if ( batch->flags    & FLAG_SS_CPU   )
    {
      // NOTHING
    }
    else
    {
      SSKer(batch);
    }
  }
}

void sumAndSearchOrr(cuFFdotBatch* batch)     // Function to call to SS and process data in normal steps  .
{
  FOLD // Sum and search the IFFT'd data  .
  {
    infoMSG(2,1,"Sum & Search\n");

    if      ( batch->retType & CU_STR_PLN )
    {
      // Nothing!
    }
    else if ( batch->flags & FLAG_SS_INMEM )
    {
      // NOTHING
    }
    else if ( batch->flags & FLAG_SS_CPU )
    {
      // NOTHING
    }
    else
    {
      SSKer(batch);
    }
  }

  if ( batch->flags & FLAG_SYNCH )
  {
    FOLD // Copy results from device to host  .
    {
      if  ( batch->flags & FLAG_SS_INMEM )
      {
        // Nothing
      }
      else
      {
        getResults(batch);
      }
    }

    FOLD // Process previous results  .
    {
      if  ( batch->flags & FLAG_SS_INMEM )
      {
        // Nothing
      }
      else
      {
        processSearchResults(batch);
      }
    }
  }
  else
  {
    FOLD // Process previous results  .
    {
      if  ( batch->flags & FLAG_SS_INMEM )
      {
        // Nothing
      }
      else
      {
        processSearchResults(batch);
      }
    }

    FOLD // Copy results from device to host  .
    {
      if  ( batch->flags & FLAG_SS_INMEM )
      {
        // Nothing
      }
      else
      {
        getResults(batch);
      }
    }
  }
}

void sumAndMax(cuFFdotBatch* batch)
{
  // TODO write this
}

void inMem(cuFFdotBatch* batch)
{
  long long noX = batch->accelLen * batch->cuSrch->SrchSz->noSteps ;
  int       noY = batch->hInfos->height;
  float*    pln = (float*)batch->cuSrch->h_candidates;

  //for ( int stage = 0; stage < batch->noHarmStages; stage++ )
  for ( int stage = 0; stage < 5 ; stage++ )
  {
    omp_set_num_threads(8);

#pragma omp parallel
    {
      int tid = omp_get_thread_num();

      printf("inMem tid %02i \n", tid);

#pragma omp for
      for ( int iy = 0; iy < noY; iy++ )
      {
        int y1 = iy       * noX   ;
        int y2 = (iy*0.5) * noX   ;

        for ( int ix = noX -1; ix >= 0; ix-- )
        {
          int idx1 = y1 +  ix ;
          int idx2 = y2 +  (ix*0.5) ;

          pln[idx1] += pln[idx2];
        }
      }
    }
  }
}

void inmemSS(cuFFdotBatch* batch, double drlo, int len)
{
  infoMSG(1,2,"Inmem Search\n");

  setActiveBatch(batch, 0);
  setSearchRVals(batch, drlo, len);

  if ( batch->flags & FLAG_SYNCH )
  {
    add_and_search_IMMEM(batch);

    getResults(batch);

    processSearchResults(batch);
  }
  else
  {
    setActiveBatch(batch, 0);
    add_and_search_IMMEM(batch);

    setActiveBatch(batch, 1);
    processSearchResults(batch);

    setActiveBatch(batch, 0);
    getResults(batch);
  }
  
  // Cycle r values
  cycleRlists(batch);
  setActiveBatch(batch, 1);

  // Cycle candidate output
  cycleOutput(batch);
}

void inmemSumAndSearch(cuSearch* cuSrch)
{
  infoMSG(1,2,"Inmem Sum And Search\n");

  cuFFdotBatch* master  = &cuSrch->pInf->kernels[0];   // The first kernel created holds global variables
  uint startBin         = cuSrch->SrchSz->searchRLow * ACCEL_RDR;
  uint endBin           = startBin + cuSrch->SrchSz->noSteps * master->accelLen;
  float totaBinsl       = endBin - startBin ;
  int iteration         = 0;
  uint currentBin       = startBin;

  nvtxRangePush("Inmem Search");

  FOLD // Set all r-values to zero  .
  {
    for ( int bIdx = 0; bIdx < cuSrch->pInf->noBatches; bIdx++ )
    {
      cuFFdotBatch* batch = &cuSrch->pInf->batches[bIdx];

      for ( int rIdx = 0; rIdx < batch->noRArryas; rIdx++ )
      {
        for ( int step = 0; step < batch->noSteps; step++ )
        {
          for ( int harm = 0; harm < batch->noGenHarms; harm++ )
          {
            rVals* rVal = &(*batch->rAraays)[rIdx][step][harm];
            memset(rVal, 0, sizeof(rVals) );

            rVal->step = -1;
          }
        }
      }
    }
  }

#ifndef DEBUG   // Parallel if we are not in debug mode  .

  if ( cuSrch->sSpec->flags & FLAG_SYNCH )
  {
    omp_set_num_threads(1);
  }
  else
  {
    omp_set_num_threads(cuSrch->pInf->noBatches);
  }

#pragma omp parallel
#endif
  FOLD  //                              ---===== Main Loop =====---  .
  {
    int tid = omp_get_thread_num();
    cuFFdotBatch* batch = &cuSrch->pInf->batches[tid];

    setDevice(batch->device) ;

    uint firstBin = 0;
    uint len      = 0;

    while ( currentBin < endBin )
    {
#pragma omp critical
      FOLD // Calculate the step  .
      {
        FOLD  // Synchronous behaviour  .
        {
#ifndef  DEBUG
          if ( cuSrch->sSpec->flags & FLAG_SYNCH )
#endif
          {
            // If running in synchronous mode use multiple batches, just synchronously
            tid     = iteration % cuSrch->pInf->noBatches ;
            batch   = &cuSrch->pInf->batches[tid];
            setDevice(batch->device) ;
          }
        }

        iteration++;

        int step    = (currentBin-startBin)/batch->strideOut;
        firstBin    = currentBin;
        len         = MIN(batch->strideOut, endBin - firstBin) ;
        currentBin += len;
        rVals* rVal = &(*batch->rAraays)[0][0][0];
        rVal->step  = step;

        if ( msgLevel >= 1 )
        {
          int tot  = (endBin)/batch->strideOut;

          infoMSG(1,1,"\nStep %4i of %4i thread %02i processing %02i steps on GPU %i\n", step+1, tot, tid, 1, batch->device );
        }
      }

      inmemSS(batch, firstBin * ACCEL_DR, len);

#pragma omp critical
      FOLD // Output  .
      {
        if ( msgLevel == 0  )
        {
          int noTrd;
          sem_getvalue(&master->cuSrch->threasdInfo->running_threads, &noTrd );
          printf("\rSearching  in-mem GPU plane. %5.1f%% ( %3i Active CPU threads processing found candidates)  ", (totaBinsl-endBin+currentBin)/totaBinsl*100.0, noTrd );
          fflush(stdout);
        }
        else
        {

        }
      }

    }

    for ( int step= 0 ; step < batch->noRArryas; step++ )
    {
      inmemSS(batch, 0, 0);
    }
  }

  printf("\rSearching  in-mem GPU plane. %5.1f%%                                                                                    \n\n", 100.0 );

  FOLD // Wait for all processing threads to terminate
  {
    waitForThreads(&master->cuSrch->threasdInfo->running_threads, "Waiting for CPU thread(s) to finish processing returned from the GPU.", 200 );
  }

  nvtxRangePop();
}
