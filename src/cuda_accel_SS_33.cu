#include "hip/hip_runtime.h"
#include "cuda_accel_SS.h"

#define SS33_X           16                    // X Thread Block
#define SS33_Y           8                     // Y Thread Block
#define SS33BS           (SS33_X*SS33_Y)


template<int noStages, int noSteps>
__device__ __forceinline__ int idxSS(int tid, int stage, int step)
{
  return stage * noSteps * SS33BS + SS33BS * step + tid ;
}

/** Sum and Search - loop down - column max - multi-step - step outer .
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base          Used in CU_OUTP_DEVICE
 * @param noSteps
 */
template<uint FLAGS, const int noStages, const int noHarms, const int cunkSize, const int noSteps>
__global__ void add_and_searchCU33_k(const uint width, __restrict__ candMin* d_cands, tHarmList texs, fsHarmList powersArr, cHarmList cmplxArr )
{
  const int tid   = threadIdx.y * SS33_X          +  threadIdx.x;   /// Block index
  const int gid   = blockIdx.x  * (SS33_Y*SS33_X) +  tid;           /// Global thread id (ie column) 0 is the first 'good' column

  if ( tid < width )
  {
    const int zeroHeight  = HEIGHT_STAGE[0];
    const int oStride     = STRIDE_STAGE[0];                            /// The stride of the output data

    //const int zh1         = zeroHeight-1;
    //const int zh2         = zeroHeight-2;

    int                 inds      [noHarms];
    //candMin             candLists [noStages][noSteps];
    float               candPow   [noStages][noSteps];
    //__shared__ float    candPow   [noStages*noSteps*SS33BS];
    int                 candZ     [noStages][noSteps];
    //__shared__ float    candZ     [noStages*noSteps*SS33BS];
    float               powers    [noSteps][cunkSize];                /// registers to hold values to increase mem cache hits
    int                 stride    [noHarms];



    FOLD // Prep - Initialise the x indices & set candidates to 0 .
    {
      FOLD // Calculate the x indices or create a pointer offset by the correct amount  .
      {
#pragma unroll
        for ( int harm = 0; harm < noHarms; harm++ )                /// loop over harmonic  .
        {
          // NOTE: the indexing below assume each plain starts on a multiple of noHarms
          int   ix    = roundf( gid*FRAC_STAGE[harm] ) + HWIDTH_STAGE[harm] ;
          inds[harm]  = ix;

          if ( FLAGS & FLAG_ITLV_ROW )
          {
            stride[harm] = noSteps*STRIDE_STAGE[harm] ;
          }
          else
          {
            stride[harm] = STRIDE_STAGE[harm] ;
          }
        }
      }

      FOLD  // Set the local and return candidate powers to zero  .
      {
#pragma unroll
        for ( int stage = 0; stage < noStages; stage++ )
        {
#pragma unroll
          for ( int step = 0; step < noSteps; step++)               // Loop over steps
          {
            //candLists[stage][step].sigma = 0 ;
            candPow [stage][step]        = 0 ;
            //candPow[idxSS<noStages,noSteps>(tid, stage, step)]          = 0 ;
            d_cands[step*noStages*oStride + stage*oStride + gid ].sigma = 0 ;
          }
        }
      }
    }

    FOLD // Sum & Search - Ignore contaminated ends tid to starts at correct spot  .
    {
      for( int y = 0; y < zeroHeight ; y += cunkSize )              // loop over chunks .
      {
        FOLD // Initialise powers for each section column to 0  .
        {
#pragma unroll
          for ( int step = 0; step < noSteps; step++)                 // Loop over steps .
          {
#pragma unroll
            for( int yPlus = 0; yPlus < cunkSize ; yPlus++ )          // Loop over powers .
            {
              powers[step][yPlus] = 0;
            }
          }
        }

        FOLD // Loop over stages, sum and search  .
        {
#pragma unroll
          for ( int stage = 0 ; stage < noStages; stage++)          // Loop over stages  .
          {
            int start = STAGE[stage][0] ;
            int end   = STAGE[stage][1] ;

            FOLD // Create a section of summed powers one for each step  .
            {
#pragma unroll
              for ( int harm = start; harm <= end; harm++ )           // Loop over harmonics (batch) in this stage  .
              {
                int ix1       = inds[harm] ;
                int ix2       = ix1;
                //int h1      = HEIGHT_STAGE[harm]-1;

#pragma unroll
                for( int yPlus = 0; yPlus < cunkSize; yPlus++ )       // Loop over the chunk  .
                {
                  int trm     = y + yPlus ;                           ///< True Y index in plain

                  int iy1     = YINDS[ zeroHeight*harm + trm ];
                  //  OR
                  //int iy1     = roundf( (HEIGHT_STAGE[harm]-1.0)*trm/(float)(zeroHeight-1.0) ) ;
                  // OR
                  //int iy1     = ( h1 * trm + zh2 ) / zh1;

                  int iy2     = iy1*stride[harm];

#pragma unroll
                  for ( int step = 0; step < noSteps; step++)         // Loop over steps  .
                  {
                    if        ( FLAGS & FLAG_ITLV_PLN )
                    {
                      iy2 = iy1 + step * HEIGHT_STAGE[harm];                // stride step by plain
                    }
                    else
                    {
                      ix2 = ix1 + step * STRIDE_STAGE[harm] ;
                    }

                    if        ( FLAGS & FLAG_SAS_TEX )
                    {
                      if      ( FLAGS & FLAG_MUL_CB_OUT )
                      {
                        const float cmpf      = tex2D < float > (texs.val[harm], ix2+0.5f, iy2+0.5f ); // + 0.5 YES + 0.5 I REALLY wish someone had documented that one, 2 days of debugging to find that!!!!!!
                        powers[step][yPlus]   += cmpf;
                      }
                      else
                      {
                        const float r         = tex2D < float > (texs.val[harm], ix2*2+0.5f, iy2+0.5f ); // + 0.5 YES + 0.5 I REALLY wish someone had documented that one, 2 days of debugging to find that!!!!!!
                        const float i         = tex2D < float > (texs.val[harm], ix2*2+1.5f, iy2+0.5f ); // + 0.5 YES + 0.5 I REALLY wish someone had documented that one, 2 days of debugging to find that!!!!!!
                        powers[step][yPlus]   += r*r+i*i;
                      }
                    }
                    else
                    {
                      if      ( FLAGS & FLAG_MUL_CB_OUT )
                      {
                        float cmpf            = powersArr[harm][ iy2 + ix2 ];
                        powers[step][yPlus]  += cmpf;
                      }
                      else
                      {
                        fcomplexcu cmpc       = cmplxArr[harm][ iy2 + ix2 ];
                        powers[step][yPlus]  += cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                      }
                    }
                  }
                }
              }
            }

            FOLD // Search set of powers  .
            {
#pragma unroll
              for ( int step = 0; step < noSteps; step++)           // Loop over steps  .
              {
#pragma unroll
                for( int yPlus = 0; yPlus < cunkSize ; yPlus++ )     // Loop over section  .
                {
                  if  (  powers[step][yPlus] > POWERCUT_STAGE[stage] )
                  {

                    //if ( powers[step][yPlus] > candLists[stage][step].sigma )
                    if ( powers[step][yPlus] > candPow [stage][step] )
                      //if ( powers[step][yPlus] > candPow[idxSS<noStages,noSteps>(tid, stage, step)] )
                    {
                      if ( y + yPlus < zeroHeight )
                      {
                        // This is our new max!
                        //candLists[stage][step].sigma  = powers[step][yPlus];
                        //candLists[stage][step].z      = y+yPlus;

                        candPow [stage][step]  = powers[step][yPlus];
                        //candPow[idxSS<noStages,noSteps>(tid, stage, step)] = powers[step][yPlus] ;
                        candZ   [stage][step]  = y+yPlus;
                        //candZ[idxSS<noStages,noSteps>(tid, stage, step)] = y+yPlus ;
                      }
                    }
                  }
                }
              }
            }

          }
        }

      }
    }

    FOLD // Write results back to DRAM and calculate sigma if needed  .
    {
#pragma unroll
      for ( int step = 0; step < noSteps; step++)             // Loop over steps
      {
#pragma unroll
        for ( int stage = 0 ; stage < noStages; stage++)      // Loop over stages
        {
          //if  ( candLists[stage][step].sigma >  POWERCUT_STAGE[stage] )
          if  ( candPow [stage][step] >  POWERCUT_STAGE[stage] )
            //if  ( candPow[idxSS<noStages,noSteps>(tid, stage, step)] >  POWERCUT_STAGE[stage] )
          {
            candMin tt;

            tt.sigma = candPow [stage][step];
            //tt.sigma = candPow[idxSS<noStages,noSteps>(tid, stage, step)] ;
            tt.z     = candZ   [stage][step];
            //tt.z     = candZ[idxSS<noStages,noSteps>(tid, stage, step)] ;

            // Write to DRAM
            d_cands[step*noStages*oStride + stage*oStride + gid] = tt;
          }
        }
      }
    }

  }
}

template<uint FLAGS, int noStages, const int noHarms, const int cunkSize>
__host__ void add_and_searchCU33_q(dim3 dimGrid, dim3 dimBlock, hipStream_t stream, cuFFdotBatch* batch )
{
  const int noSteps = batch->noSteps ;

  for ( int step = 0; step < noSteps; step++)
  {
    long long firstBin  = (*batch->rConvld)[step][0].expBin ;

    for (int i = 0; i < noHarms; i++)
    {
      int idx =  batch->stageIdx[i];

      long long binb      = (*batch->rConvld)[step][idx].expBin ;

      if ( firstBin * h_FRAC_STAGE[i] != binb )
      {
        fprintf(stderr,"ERROR, in function %s, R values are not properly aligned! Each step should start on a multiple of (2 x No Harms).\n", __FUNCTION__ );
        fprintf(stderr,"%f != %f.\n", firstBin * h_FRAC_STAGE[i], (float)binb );
        exit(EXIT_FAILURE);
      }
    }
  }

  tHarmList   texs;
  fsHarmList powers;
  cHarmList   cmplx;

  for (int i = 0; i < noHarms; i++)
  {
    int idx         = batch->stageIdx[i];
    texs.val[i]     = batch->plains[idx].datTex;
    powers.val[i]   = batch->plains[idx].d_plainPowers;
    cmplx.val[i]    = batch->plains[idx].d_plainData;
  }

  switch (noSteps)
  {
    case 1:
    {
      add_and_searchCU33_k<FLAGS,noStages,noHarms,cunkSize,1><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candMin*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 2:
    {
      add_and_searchCU33_k<FLAGS,noStages,noHarms,cunkSize,2><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candMin*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 3:
    {
      add_and_searchCU33_k<FLAGS,noStages,noHarms,cunkSize,3><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candMin*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 4:
    {
      //hipFuncSetCacheConfig(reinterpret_cast<const void*>(add_and_searchCU33_k<FLAGS),noStages,noHarms,cunkSize,4>, hipFuncCachePreferL1);
      add_and_searchCU33_k<FLAGS,noStages,noHarms,cunkSize,4><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candMin*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 5:
    {
      add_and_searchCU33_k<FLAGS,noStages,noHarms,cunkSize,5><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candMin*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 6:
    {
      add_and_searchCU33_k<FLAGS,noStages,noHarms,cunkSize,6><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candMin*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 7:
    {
      add_and_searchCU33_k<FLAGS,noStages,noHarms,cunkSize,7><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candMin*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    case 8:
    {
      add_and_searchCU33_k<FLAGS,noStages,noHarms,cunkSize,8><<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (candMin*)batch->d_retData, texs, powers, cmplx );
      break;
    }
    default:
      fprintf(stderr, "ERROR: add_and_searchCU311 has not been templated for %i steps\n", noSteps);
      exit(EXIT_FAILURE);
  }
}

template<uint FLAGS, int noStages, const int noHarms>
__host__ void add_and_searchCU33_c(dim3 dimGrid, dim3 dimBlock, hipStream_t stream, cuFFdotBatch* batch )
{
  switch (globalInt01)
  {
    //    case 1:
    //    {
    //      add_and_searchCU33_q<FLAGS,noStages,noHarms,1>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    case 2:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,2>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 3:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,3>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 4:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,4>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 5:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,5>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 6:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,6>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 7:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,7>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 8:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,8>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 9:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,9>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 10:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,10>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 12:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,12>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 14:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,14>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 16:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,16>(dimGrid, dimBlock, stream, batch);
      break;
    }
    case 18:
    {
      add_and_searchCU33_q<FLAGS,noStages,noHarms,18>(dimGrid, dimBlock, stream, batch);
      break;
    }
    //    case 20:
    //    {
    //      add_and_searchCU33_q<FLAGS,noStages,noHarms,20>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    //    case 24:
    //    {
    //      add_and_searchCU33_q<FLAGS,noStages,noHarms,24>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    default:
      fprintf(stderr, "ERROR: %s has not been templated for %i stages\n", __FUNCTION__, noStages);
      exit(EXIT_FAILURE);
  }

}

template<uint FLAGS >
__host__ void add_and_searchCU33_p(dim3 dimGrid, dim3 dimBlock, hipStream_t stream, cuFFdotBatch* batch )
{
  const int noStages = batch->noHarmStages;

  switch (noStages)
  {
    //    case 1:
    //    {
    //      add_and_searchCU33_c<FLAGS,1,1>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    //    case 2:
    //    {
    //      add_and_searchCU33_c<FLAGS,2,2>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    //    case 3:
    //    {
    //      add_and_searchCU33_c<FLAGS,3,4>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    //    case 4:
    //    {
    //      add_and_searchCU33_c<FLAGS,4,8>(dimGrid, dimBlock, stream, batch);
    //      break;
    //    }
    case 5:
    {
      add_and_searchCU33_c<FLAGS,5,16>(dimGrid, dimBlock, stream, batch);
      break;
    }
    default:
      fprintf(stderr, "ERROR: %s has not been templated for %i stages\n", __FUNCTION__, noStages);
      exit(EXIT_FAILURE);
  }
}

__host__ void add_and_searchCU33(hipStream_t stream, cuFFdotBatch* batch )
{
  const uint FLAGS = batch->flag;
  dim3 dimBlock, dimGrid;

  dimBlock.x  = SS33_X;
  dimBlock.y  = SS33_Y;

  float bw    = SS33_X * SS33_Y;
  float ww    = batch->accelLen / ( bw );

  dimGrid.x   = ceil(ww);
  dimGrid.y   = 1;

  if        ( FLAGS & FLAG_MUL_CB_OUT )
  {
    if      ( FLAGS & FLAG_ITLV_ROW )
      add_and_searchCU33_p<FLAG_MUL_CB_OUT | FLAG_ITLV_ROW> (dimGrid, dimBlock, stream, batch);
    else if ( FLAGS & FLAG_ITLV_PLN )
      add_and_searchCU33_p<FLAG_MUL_CB_OUT | FLAG_ITLV_PLN>  (dimGrid, dimBlock, stream, batch);
    else
    {
      fprintf(stderr, "ERROR: %s has not been templated for flag combination. \n", __FUNCTION__ );
      exit(EXIT_FAILURE);
    }
  }
  else
  {
    if      ( FLAGS & FLAG_ITLV_ROW )
      add_and_searchCU33_p<FLAG_ITLV_ROW> (dimGrid, dimBlock, stream, batch);
    else if ( FLAGS & FLAG_ITLV_PLN )
      add_and_searchCU33_p<FLAG_ITLV_PLN> (dimGrid, dimBlock, stream, batch);
    else
    {
      fprintf(stderr, "ERROR: %s has not been templated for flag combination. \n", __FUNCTION__ );
      exit(EXIT_FAILURE);
    }
  }
}

