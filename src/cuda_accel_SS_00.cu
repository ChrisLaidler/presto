#include "hip/hip_runtime.h"
#include "cuda_accel_SS.h"

#define SS00_X           16                    // X Thread Block
#define SS00_Y           8                     // Y Thread Block
#define SS00BS           (SS00_X*SS00_Y)

/** Sum and Search - loop down - column max - multi-step - step outer .
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base          Used in CU_OUTP_DEVICE
 * @param noSteps
 */
template<uint FLAGS, int noBatch >
__global__ void add_and_searchCU00_k(const uint width, accelcandBasic* d_cands, fsHarmList powersArr, cHarmList cmplxArr, const int noHarms, const int noStages, const int noSteps )
{
  const int bidx  = threadIdx.y * SS00_X  +  threadIdx.x;   /// Block index
  const int tid   = blockIdx.x  * SS00BS  +  bidx;          /// Global thread id (ie column) 0 is the first 'good' column

  if ( tid < width )
  {
    int oStride    = STRIDE_HARM[0];

    FOLD  // Set the local and return candidate powers to zero  .
    {
      for ( int stage = 0; stage < noStages; stage++ )
      {
        for ( int step = 0; step < noSteps; step++)               // Loop over steps
        {
          d_cands[step*noStages*oStride + stage*oStride + tid ].sigma = 0;
        }
      }
    }

    float batch[noBatch];
    for ( int harm = 0; harm < noHarms ; harm++)                // Loop over plains  .
    {
      int maxW      = ceilf(width * FRAC_HARM[harm]);
      int stride    = STRIDE_HARM[harm];

      if ( tid < maxW )
      {
        uint nHeight = HEIGHT_HARM[harm] * noSteps;

        FOLD // Read data from plains  .
        {
          for ( int yBase = 0; yBase < nHeight; yBase += noBatch )
          {
            for ( int yPlus = 0; yPlus < noBatch; yPlus++ )
            {
              int idx  = (yBase+yPlus) * stride;

              FOLD // Read  .
              {
                if      ( FLAGS & FLAG_MUL_CB_OUT )
                {
                  float cmpf            = powersArr[harm][ tid + idx ];
                  batch[yPlus]          = cmpf;
                }
                else
                {
                  fcomplexcu cmpc       = cmplxArr[harm][ tid + idx ];
                  batch[yPlus]          = cmpc.r * cmpc.r + cmpc.i * cmpc.i ;
                }
              }
            }
            for ( int yPlus = 0; yPlus < noBatch; yPlus++ )
            {
              if (yPlus + yBase < nHeight )
              {
                if ( batch[yPlus] < 0 ) // Make sure we don't optimise out the reads
                {
                  printf("SS\n");
                }
              }
            }
          }
        }
      }
    }
  }
}

/** Sum and Search - loop down - column max - multi-step - step outer .
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base          Used in CU_OUTP_DEVICE
 * @param noSteps
 */
template<uint FLAGS, int noBatch >
__global__ void add_and_searchCU01_k(const uint width, accelcandBasic* d_cands, fsHarmList powersArr, cHarmList cmplxArr, const int noHarms, const int noStages, const int noSteps )
{
  const int bidx  = threadIdx.y * SS00_X  +  threadIdx.x;   /// Block index
  const int tid   = blockIdx.x  * SS00BS  +  bidx;          /// Global thread id (ie column) 0 is the first 'good' column

  if ( tid < width )
  {
    int oStride    = STRIDE_STAGE[0];
    FOLD  // Set the local and return candidate powers to zero
    {
      for ( int stage = 0; stage < noStages; stage++ )
      {
        for ( int step = 0; step < noSteps; step++)               // Loop over steps
        {
          d_cands[step*noStages*oStride + stage*oStride + tid ].sigma = 0;
        }
      }
    }

    float batch[noBatch];
    for ( int harm = 0; harm < noHarms ; harm++)  // Loop over plains
    {
      int maxW      = ceilf(width * FRAC_STAGE[harm]);
      int stride    = STRIDE_STAGE[harm];

      if ( tid < maxW )
      {
        uint nHeight = HEIGHT_STAGE[harm] * noSteps;

        FOLD // Read data from plains  .
        {
          for ( int yBase = 0; yBase < nHeight; yBase+=noBatch )
          {
            for ( int yPlus = 0; yPlus < noBatch; yPlus++ )
            {
              int idx  = (yBase+yPlus) * stride;

              FOLD // Read  .
              {
                if      ( FLAGS & FLAG_MUL_CB_OUT )
                {
                  float cmpf            = powersArr[harm][ tid + idx ];
                  batch[yPlus]          = cmpf;
                }
                else
                {
                  fcomplexcu cmpc       = cmplxArr[harm][ tid + idx ];
                  batch[yPlus]          = cmpc.r * cmpc.r + cmpc.i * cmpc.i ;
                }
              }
            }

            for ( int yPlus = 0; yPlus < noBatch; yPlus++ )
            {
              if (yPlus + yBase < nHeight )
              {
                if ( batch[yPlus] < 0 )
                {
                  printf("SS\n");
                }
              }
            }
          }
        }
      }
    }
  }
}

/** Sum and Search - loop down - column max - multi-step - step outer .
 *
 * @param searchList
 * @param d_cands
 * @param d_sem
 * @param base          Used in CU_OUTP_DEVICE
 * @param noSteps
 */
template<uint FLAGS, int noBatch >
__global__ void add_and_searchCU02_k(const uint width, accelcandBasic* d_cands, fsHarmList powersArr, cHarmList cmplxArr, const int noHarms, const int noStages, const int noSteps )
{
  const int bidx  = threadIdx.y * SS00_X  +  threadIdx.x;   /// Block index
  const int tid   = blockIdx.x  * SS00BS  +  bidx;          /// Global thread id (ie column) 0 is the first 'good' column

  if ( tid < width )
  {
    int oStride    = STRIDE_STAGE[0];

    FOLD  // Set the local and return candidate powers to zero  .
    {
      for ( int stage = 0; stage < noStages; stage++ )
      {
        for ( int step = 0; step < noSteps; step++)               // Loop over steps
        {
          d_cands[step*noStages*oStride + stage*oStride + tid ].sigma = 0;
        }
      }
    }

    float batch[noBatch];
    for ( int harm = 0; harm < noHarms ; harm++)  // Loop over plains  .
    {
      int maxW      = ceilf(width * FRAC_STAGE[0]);
      int stride    = STRIDE_STAGE[0];

      if ( tid < maxW )
      {
        uint nHeight = HEIGHT_STAGE[0] * noSteps;

        FOLD // Read data from plains  .
        {
          for ( int yBase = 0; yBase < nHeight; yBase += noBatch )
          {
            for ( int yPlus = 0; yPlus < noBatch; yPlus++ )
            {
              int idx  = (yBase+yPlus) * stride ;

              FOLD // Read  .
              {
                if      ( FLAGS & FLAG_MUL_CB_OUT )
                {
                  float cmpf            = powersArr[0][ tid + idx ];
                  batch[yPlus]          = cmpf;
                }
                else
                {
                  fcomplexcu cmpc       = cmplxArr[0][ tid + idx ];
                  batch[yPlus]          = cmpc.r * cmpc.r + cmpc.i * cmpc.i ;
                }
              }
            }

            for ( int yPlus = 0; yPlus < noBatch; yPlus++ )
            {
              if (yPlus + yBase < nHeight )
              {
                if ( batch[yPlus] < 0 )
                {
                  printf("SS\n");
                }
              }
            }
          }
        }
      }
    }
  }
}

template<uint FLAGS>
__host__ void add_and_searchCU02_c(dim3 dimGrid, dim3 dimBlock, hipStream_t stream, cuFFdotBatch* batch )
{
  const int   noStages  = log2((double)batch->noHarms) + 1 ;
  fsHarmList  powers;
  cHarmList   cmplx;

  for (int i = 0; i < batch->noHarms; i++)
  {
    int idx         = batch->stageIdx[i]; // Stage order
    powers.val[i]   = batch->plains[idx].d_plainPowers;
    cmplx.val[i]    = batch->plains[idx].d_plainData;
  }

  switch (globalInt01)
  {
    case 1:
    {
      add_and_searchCU01_k<FLAGS, 1> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 2:
    {
      add_and_searchCU01_k<FLAGS,2> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
//    case 3:
//    {
//      add_and_searchCU01_k<FLAGS,3> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
//      break;
//    }
    case 4:
    {
      add_and_searchCU01_k<FLAGS,4> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
//    case 5:
//    {
//      add_and_searchCU01_k<FLAGS,5> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
//      break;
//    }
    case 6:
    {
      add_and_searchCU01_k<FLAGS,6> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
//    case 7:
//    {
//      add_and_searchCU01_k<FLAGS,7> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
//      break;
//    }
    case 8:
    {
      add_and_searchCU01_k<FLAGS,8> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
//    case 9:
//    {
//      add_and_searchCU01_k<FLAGS,9> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
//      break;
//    }
    case 10:
    {
      add_and_searchCU01_k<FLAGS,10> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 12:
    {
      add_and_searchCU01_k<FLAGS,12> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 14:
    {
      add_and_searchCU01_k<FLAGS,14> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 16:
    {
      add_and_searchCU01_k<FLAGS,16> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 18:
    {
      add_and_searchCU01_k<FLAGS,18> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 20:
    {
      add_and_searchCU01_k<FLAGS,20> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 24:
    {
      add_and_searchCU01_k<FLAGS,24> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    default:
      fprintf(stderr, "ERROR: %s has not been templated for %i chunk size.\n", __FUNCTION__, globalInt01);
      exit(EXIT_FAILURE);
  }

}

template<uint FLAGS>
__host__ void add_and_searchCU00_c(dim3 dimGrid, dim3 dimBlock, hipStream_t stream, cuFFdotBatch* batch )
{
  const int   noStages  = log2((double)batch->noHarms) + 1 ;
  fsHarmList  powers;
  cHarmList   cmplx;

  for (int i = 0; i < batch->noHarms; i++)
  {
    int idx = i;
    powers.val[i]   = batch->plains[idx].d_plainPowers;
    cmplx.val[i]    = batch->plains[idx].d_plainData;
  }

  switch (globalInt01)
  {
    case 1:
    {
      add_and_searchCU00_k<FLAGS, 1> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 2:
    {
      add_and_searchCU00_k<FLAGS,2> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
//    case 3:
//    {
//      add_and_searchCU00_k<FLAGS,3> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
//      break;
//    }
    case 4:
    {
      add_and_searchCU00_k<FLAGS,4> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 5:
    {
      add_and_searchCU00_k<FLAGS,5> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
//    case 6:
//    {
//      add_and_searchCU00_k<FLAGS,6> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
//      break;
//    }
    case 7:
    {
      add_and_searchCU00_k<FLAGS,7> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 8:
    {
      add_and_searchCU00_k<FLAGS,8> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
//    case 9:
//    {
//      add_and_searchCU00_k<FLAGS,9> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
//      break;
//    }
    case 10:
    {
      add_and_searchCU00_k<FLAGS,10> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 12:
    {
      add_and_searchCU00_k<FLAGS,12> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 14:
    {
      add_and_searchCU00_k<FLAGS,14> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 16:
    {
      add_and_searchCU00_k<FLAGS,16> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 18:
    {
      add_and_searchCU00_k<FLAGS,18> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 20:
    {
      add_and_searchCU00_k<FLAGS,20> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    case 24:
    {
      add_and_searchCU00_k<FLAGS,24> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      break;
    }
    default:
      fprintf(stderr, "ERROR: %s has not been templated for %i chunk size.\n", __FUNCTION__, globalInt01);
      exit(EXIT_FAILURE);
  }

}

__host__ void add_and_searchCU00(hipStream_t stream, cuFFdotBatch* batch )
{
  dim3 dimBlock, dimGrid;

  const uint FLAGS    = batch->flag;

  dimBlock.x  = SS00_X;
  dimBlock.y  = SS00_Y;

  float bw    = SS00BS ;
  float ww    = batch->accelLen / ( bw );

  dimGrid.x   = ceil(ww);
  dimGrid.y   = 1;

  if ( 1 )  // Stage order  .
  {
    if        ( FLAGS & FLAG_MUL_CB_OUT )
    {
      //add_and_searchCU02_k<FLAG_MUL_CB_OUT, 12> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      add_and_searchCU02_c<FLAG_MUL_CB_OUT>(dimGrid,dimBlock,stream, batch );
    }
    else
    {
      //add_and_searchCU02_k< 0, 12 > <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      add_and_searchCU02_c<0>(dimGrid,dimBlock,stream, batch );
    }
  }
  else
  {
    if        ( FLAGS & FLAG_MUL_CB_OUT )
    {
      //add_and_searchCU00_k<FLAG_MUL_CB_OUT, 12> <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      add_and_searchCU00_c<FLAG_MUL_CB_OUT>(dimGrid,dimBlock,stream, batch );
    }
    else
    {
      //add_and_searchCU00_k< 0, 12 > <<<dimGrid,  dimBlock, 0, stream >>>(batch->accelLen, (accelcandBasic*)batch->d_retData, powers, cmplx, batch->noHarms, noStages, batch->noSteps  );
      add_and_searchCU00_c<0>(dimGrid,dimBlock,stream, batch );
    }
  }

}

