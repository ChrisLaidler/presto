#include "hip/hip_runtime.h"
/** @file cuda_accel_MU_31.cu
 *  @brief The implementation of the family multiplication kernel v1
 *
 *  @author Chris Laidler
 *  @bug No known bugs.
 *
 *  Change Log
 *
 *  [0.0.01] []
 *    Beginning of change log
 *    Working version un-numbed
 *
 *  [0.0.01] [2017-02-24]
 *     Added preprocessor directives for steps and chunks
 *
 */
 
#include "cuda_accel_MU.h"

#ifdef WITH_MUL_31

/** Multiplication kernel - Multiply an entire batch with convolution kernel  .
 * Each thread loops down a column of the planes and multiplies input with kernel and writes result to plane
 */
template<int64_t FLAGS, int noSteps>
__global__ void mult31_k(const __restrict__ float2* kernels, const __restrict__ float2* datas, __restrict__ float2* ffdot, int noPlanes)
{
  const int ix = blockIdx.x * CNV_DIMX * CNV_DIMY + CNV_DIMX * threadIdx.y + threadIdx.x;

  float2 input[noSteps];

  // Stride
  ffdot   += ix;
  datas   += ix;

  int pHeight = 0;

  for (int n = 0; n < noPlanes; n++)					// Loop over planes  .
  {
    const int stride      = STRIDE_HARM[n];

    if ( ix < stride )
    {
      const int plnHeight = HEIGHT_HARM[n];
      const short lDepth  = ceilf(plnHeight/(float)gridDim.y);
      const short y0      = lDepth*blockIdx.y;
      const short y1      = MIN(y0+lDepth, plnHeight);
      float2* ker     = (float2*)KERNEL_HARM[n] + y0 * stride + ix;

#ifdef WITH_ITLV_PLN
      const int plnStride = plnHeight*stride;
#endif

      // read input for each step into registers
      for (int step = 0; step < noSteps; step++)			// Loop over planes  .
      {
        input[step]       = datas[step*stride];

        // Normalise
        input[step].x    /= (float) stride ;
        input[step].y    /= (float) stride ;
      }

      // Stride input data
      datas              += stride*noSteps;

      for (int planeY = y0; planeY < y1; planeY++)			// Loop over individual plane  .
      {
        int off1;
        FOLD // Calculate partial offset  .
        {
          if      ( FLAGS & FLAG_ITLV_ROW )
          {
            off1  = pHeight + planeY*noSteps*stride;
          }
#ifdef WITH_ITLV_PLN
          else
          {
            off1  = pHeight + planeY*stride;
          }
#endif
        }

        // Multiply and write data
        for (int step = 0; step < noSteps; step++)			// Loop over steps  .
        {
          //
          float2 out;
          float2 ipd = input[step];

          // Calculate index
          int idx = 0;
          FOLD // Calculate indices  .
          {
            if      ( FLAGS & FLAG_ITLV_ROW )
            {
              idx  = off1 + step * stride;
            }
#ifdef WITH_ITLV_PLN
            else
            {
              idx  = off1 + step * plnStride;
            }
#endif
          }

	  // Multiply
#if CORRECT_MULT
          // This is the "correct" version
          out.x = (ipd.x * ker->x - ipd.y * ker->y);
          out.y = (ipd.x * ker->y + ipd.y * ker->x);
#else
          // This is the version accelsearch uses, ( added for comparison )
          out.x = (ipd.x * ker->x + ipd.y * ker->y);
          out.y = (ipd.y * ker->x - ipd.x * ker->y);
#endif

          // Write the actual value
          ffdot[idx] = out;
        }

        // Stride kernel to next row
        ker += stride;
      }

      // Track plane offset
      pHeight += noSteps*plnHeight*stride;
    }
  }
}

template<int64_t FLAGS>
__host__  void mult31_s(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t multStream, cuFFdotBatch* batch)
{
  switch (batch->noSteps)
  {
#if MIN_STEPS <= 1  and MAX_STEPS >= 1
    case 1:
    {
      mult31_k<FLAGS,1><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 2  and MAX_STEPS >= 2
    case 2:
    {
      mult31_k<FLAGS,2><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 3  and MAX_STEPS >= 3
    case 3:
    {
      mult31_k<FLAGS,3><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 4  and MAX_STEPS >= 4
    case 4:
    {
      mult31_k<FLAGS,4><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 5  and MAX_STEPS >= 5
    case 5:
    {
      mult31_k<FLAGS,5><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 6  and MAX_STEPS >= 6
    case 6:
    {
      mult31_k<FLAGS,6><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 7  and MAX_STEPS >= 7
    case 7:
    {
      mult31_k<FLAGS,7><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 8  and MAX_STEPS >= 8
    case 8:
    {
      mult31_k<FLAGS,8><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 9  and MAX_STEPS >= 9
    case 9:
    {
      mult31_k<FLAGS,9><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 10 and MAX_STEPS >= 10
    case 10:
    {
      mult31_k<FLAGS,10><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 11 and MAX_STEPS >= 11
    case 11:
    {
      mult31_k<FLAGS,11><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

#if MIN_STEPS <= 12 and MAX_STEPS >= 12
    case 12:
    {
      mult31_k<FLAGS,12><<<dimGrid, dimBlock, i1, multStream>>>((float2*)batch->d_kerData, (float2*)batch->d_iData, (float2*)batch->d_planeMult, batch->noGenHarms);
      break;
    }
#endif

    default:
    {
      if      ( batch->noSteps < MIN_STEPS )
	fprintf(stderr, "ERROR: In %s, # steps (%i) less than the compiled minimum %i.\n", __FUNCTION__, batch->noSteps, MIN_STEPS );
      else if ( batch->noSteps > MAX_STEPS )
	fprintf(stderr, "ERROR: In %s, # steps (%i) greater than the compiled maximum %i.\n", __FUNCTION__, batch->noSteps, MIN_STEPS );
      else
	fprintf(stderr, "ERROR: %s has not been templated for %i steps.\n", __FUNCTION__, batch->noSteps);

      exit(EXIT_FAILURE);
    }
  }
}

#endif	// WITH_MUL_31

__host__  void mult31(hipStream_t multStream, cuFFdotBatch* batch)
{
#ifdef WITH_MUL_31

  dim3 dimGrid, dimBlock;

  dimBlock.x = CNV_DIMX;
  dimBlock.y = CNV_DIMY;

  dimGrid.x = ceil(batch->hInfos[0].width / (float) ( CNV_DIMX * CNV_DIMY ));
  dimGrid.y = batch->mulSlices;

  if      ( batch->flags & FLAG_ITLV_ROW )
    mult31_s<FLAG_ITLV_ROW>(dimGrid, dimBlock, 0, multStream, batch);
#ifdef WITH_ITLV_PLN
  else
    mult31_s<0>(dimGrid, dimBlock, 0, multStream, batch);
#else
  else
  {
    fprintf(stderr, "ERROR: functionality disabled in %s.\n", __FUNCTION__);
    exit(EXIT_FAILURE);
  }
#endif

#else
  EXIT_DIRECTIVE("WITH_MUL_31");
#endif
}
