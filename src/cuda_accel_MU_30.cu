#include "hip/hip_runtime.h"
#include "cuda_accel_MU.h"

/** Multiplication kernel - Multiply an entire batch with convolution kernel  .
 * Each thread loops down a column of the plains and multiplies input with kernel and writes result to plain
 */
template<uint FLAGS, int noSteps>
__global__ void mult30_k(const __restrict__ fcomplexcu* kernels, const __restrict__ fcomplexcu* datas, __restrict__ fcomplexcu* ffdot, int noPlains)
{
  const int ix = blockIdx.x * CNV_DIMX * CNV_DIMY + CNV_DIMX * threadIdx.y + threadIdx.x;

  fcomplexcu input[noSteps];

  // Stride
  ffdot   += ix;
  datas   += ix;

  for (int n = 0; n < noPlains; n++)                  // Loop over plains  .
  {
    const int stride   = STRIDE_HARM[n];
    const int height   = HEIGHT_HARM[n];
    fcomplexcu* ker    = KERNEL_HARM[n] + ix;

    if ( ix < stride )
    {
      // read input for each step into registers
      for (int step = 0; step < noSteps; step++)      // Loop over plains  .
      {
        input[step]      = datas[step*stride];

        // Normalise
        input[step].r   /= (float) stride ;
        input[step].i   /= (float) stride ;
      }

      // Stride input data
      datas        += stride*noSteps;

      short   lDepth  = ceilf(height/(float)gridDim.y);
      short   y0      = lDepth*blockIdx.y;
      short   y1      = MIN(y0+lDepth, height);

      //for (int iy = 0; iy < height; iy++)           // Loop over individual plain  .
      for (int iy = y0; iy < y1; iy++)              // Loop over individual plain  .
      {
        const int plnOffset = iy*stride;
        const int PlnStride = height*stride;

        // Multiply and write data
        for (int step = 0; step < noSteps; step++)  // Loop over steps  .
        {
          // Multiply
          fcomplexcu val;
          val.r = (input[step].r * ker->r + input[step].i * ker->i);
          val.i = (input[step].i * ker->r - input[step].r * ker->i);

          if      ( FLAGS & FLAG_ITLV_ROW )
          {
            *ffdot = val;
            ffdot += stride;  // Stride output pointer to next plain
          }
          else if ( FLAGS & FLAG_ITLV_PLN )
          {
            ffdot[plnOffset + step*PlnStride ] = val;
          }
        }

        // Stride kernel to next row
        ker += stride;
      }

      if ( FLAGS & FLAG_ITLV_PLN ) 	                // Stride output pointer to next plain  .
      {
        ffdot += noSteps*height*stride;
      }
    }
  }
}

template<int FLAGS>
__host__  void mult30_s(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t multStream, cuFFdotBatch* batch)
{
  switch (batch->noSteps)
  {
    case 1:
    {
      mult30_k<FLAGS,1><<<dimGrid, dimBlock, i1, multStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 2:
    {
      mult30_k<FLAGS,2><<<dimGrid, dimBlock, i1, multStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 3:
    {
      mult30_k<FLAGS,3><<<dimGrid, dimBlock, i1, multStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 4:
    {
      mult30_k<FLAGS,4><<<dimGrid, dimBlock, i1, multStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 5:
    {
      mult30_k<FLAGS,5><<<dimGrid, dimBlock, i1, multStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 6:
    {
      mult30_k<FLAGS,6><<<dimGrid, dimBlock, i1, multStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 7:
    {
      mult30_k<FLAGS,7><<<dimGrid, dimBlock, i1, multStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    case 8:
    {
      mult30_k<FLAGS,8><<<dimGrid, dimBlock, i1, multStream>>>(batch->d_kerData , batch->d_iData, batch->d_plainData, batch->noHarms);
      break;
    }
    default:
    {
      fprintf(stderr, "ERROR: mult5 has not been templated for %lu steps\n", batch->noSteps);
      exit(EXIT_FAILURE);
    }
  }
}

__host__  void mult30_f(hipStream_t multStream, cuFFdotBatch* batch)
{
  dim3 dimGrid, dimBlock;

  dimBlock.x = CNV_DIMX;
  dimBlock.y = CNV_DIMY;

  dimGrid.x = ceil(batch->hInfos[0].width / (float) ( CNV_DIMX * CNV_DIMY ));
  dimGrid.y = batch->noMulSlices;

  if      ( batch->flag & FLAG_ITLV_ROW )
    mult30_s<FLAG_ITLV_ROW>(dimGrid, dimBlock, 0, multStream, batch);
  else if ( batch->flag & FLAG_ITLV_PLN )
    mult30_s<FLAG_ITLV_PLN>(dimGrid, dimBlock, 0, multStream, batch);
  else
  {
    fprintf(stderr, "ERROR: mult5 has not been templated for layout.\n");
    exit(EXIT_FAILURE);
  }
}
