#include "hip/hip_runtime.h"
#include "cuda_accel_SS.h"

#include <hipfft/hipfft.h>
#include <algorithm>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <nvToolsExt.h>
#include <nvToolsExtCudaRt.h>

#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"
#include "cuda_accel_SS.h"


void add_and_search_CPU(cuFFdotBatch* batch )
{

#ifdef STPMSG
  printf("\t\tSum & search CPU\n");
#endif

#ifdef TIMING // Timing  .
  struct timeval start, end;
#endif

  const int noStages    = batch->noHarmStages;
  const int noHarms     = batch->noHarms;
  const int noSteps     = batch->noSteps;
  const int FLAGS       = batch->flag;
  const int zeroHeight  = batch->hInfos->height;

  float*      pwerPlnF[noHarms];
  fcomplexcu* pwerPlnC[noHarms];

  candPZs     candLists [noStages][noSteps];
  float       pow[noHarms][noSteps];
  short       iyP[noHarms];
  int         inds[noHarms];
  int         sliceSz   = 16;
  int         noSlices  = ceil( zeroHeight / (float)sliceSz );
  int         noCands   = 0;
  cand*       cnd       = (cand*)malloc(sizeof(cand)*noSlices*batch->accelLen*noStages*noSteps);

  FOLD // Sum search data  .
  {
    nvtxRangePush("CPU Sum & search");

#ifdef TIMING // Timing  .
    gettimeofday(&start, NULL);
#endif

    FOLD // Prep - Initialise the x indices  .
    {
      int bace = 0;
      for ( int harm = 0; harm < noHarms; harm++ )                  // loop over harmonic  .
      {
        int stgIDX = batch->stageIdx[harm];

        pwerPlnF[stgIDX] = &((float*)batch->h_retData1)[bace];
        pwerPlnC[stgIDX] = &((fcomplexcu*)batch->h_retData1)[bace];

        bace += batch->hInfos[harm].height * batch->stacks[batch->hInfos[harm].stackNo].strideFloat * noSteps;
      }
    }

    for ( int ix = 0; ix < batch->accelLen; ix++ )
    {
      FOLD // Prep - Initialise the x indices  .
      {
        for ( int harm = 0; harm < noHarms; harm++ )                // loop over harmonic  .
        {
          int stgIDX      = batch->stageIdx[harm];

          //// NOTE: the indexing below assume each plain starts on a multiple of noHarms
          int   hIdx      = round( ix*batch->hInfos[stgIDX].harmFrac ) + batch->hInfos[stgIDX].halfWidth * ACCEL_NUMBETWEEN ;
          inds[harm]      = hIdx;
        }
      }

      FOLD // Set the local and return candidate powers to zero  .
      {
        for ( int stage = 0; stage < noStages; stage++ )
        {
          for ( int step = 0; step < noSteps; step++)               // Loop over steps  .
          {
            candLists[stage][step].value = batch->sInf->powerCut[stage] ;
          }
        }
      }

      FOLD // Set hold values to zero
      {
        for ( int harm = 0; harm < noHarms; harm++ )
        {
          iyP[harm] = -1;
        }
      }

      FOLD // Sum & Search - Ignore contaminated ends tid to starts at correct spot  .
      {
        for( int y = 0, sy = 0; y < zeroHeight; y++, sy++ )         // Loop over the chunk  .
        {
          float powers[noSteps];
          for ( int step = 0; step < noSteps; step++)               // Loop over steps  .
          {
            powers[step] = 0;
          }

          for ( int stage = 0 ; stage < noStages; stage++)          // Loop over stages  .
          {
            short start         = STAGE_CPU[stage][0] ;
            short end           = STAGE_CPU[stage][1] ;

            for ( int harm = start; harm <= end; harm++ )         	// Loop over harmonics (batch) in this stage  .
            {
              int stgIDX        = batch->stageIdx[harm];
              cuHarmInfo* hInf  = &batch->hInfos[stgIDX];

              int     ix1       = inds[harm] ;
              int     ix2       = ix1;
              short   iy1       = batch->sInf->yInds[ (zeroHeight+INDS_BUFF)*harm + y ];

              if ( iyP[harm] != iy1 ) // Only read power if it is not the same as the previous  .
              {
                for ( int step = 0; step < noSteps; step++ )        // Loop over steps  .
                {
                  int iy2;

                  FOLD // Calculate index  .
                  {
                    if        ( FLAGS & FLAG_ITLV_ROW )
                    {
                      ix2 = ix1 + step    * hInf->inpStride;
                      iy2 = iy1 * noSteps * hInf->inpStride;
                    }
                    else
                    {
                      iy2 = ( iy1 + step * hInf->height ) * hInf->inpStride ;
                    }
                  }

                  FOLD // Read powers  .
                  {
                    if      ( FLAGS & FLAG_CUFFT_CB_OUT )
                    {
                      pow[harm][step]         = pwerPlnF[harm][ iy2 + ix2 ];
                    }
                    else
                    {
                      fcomplexcu cmpc         = pwerPlnC[harm][ iy2 + ix2 ];
                      pow[harm][step]         = cmpc.r * cmpc.r + cmpc.i * cmpc.i;
                    }
                  }

                }
                iyP[harm] = iy1;
              }

              for ( int step = 0; step < noSteps; step++)           // Loop over steps  .
              {
                powers[step]  += pow[harm][step];
              }
            }

            for ( int step = 0; step < noSteps; step++)             // Loop over steps  .
            {
              if ( powers[step] > candLists[stage][step].value )
              {
                // This is our new max!
                candLists[stage][step].value  = powers[step];
                candLists[stage][step].z      = y;
              }
            }
          }

          if ( sy > sliceSz || y == zeroHeight - 1 )
          {
            FOLD // Add candidates to list  .
            {
              for ( int stage = 0 ; stage < noStages; stage++)      // Loop over stages  .
              {
                for ( int step = 0; step < noSteps; step++)         // Loop over steps  .
                {
                  if ( candLists[stage][step].value > batch->sInf->powerCut[stage] )
                  {
                    rVals* rVal = &batch->rValues[step][0];

                    int numharm   = (1<<stage);
                    double rr     = rVal->drlo + ix *  ACCEL_DR ;

                    //procesCanidate(batch, rr, y, candLists[stage][step].value, 0, stage, numharm );
                    cnd[noCands].numharm  = numharm;
                    cnd[noCands].power    = candLists[stage][step].value;
                    cnd[noCands].r        = rr;
                    cnd[noCands].sig      = 0;
                    cnd[noCands].z        = y;
                    noCands++;
                  }
                }
              }
            }

            FOLD // Set the local and return candidate powers to zero  .
            {
              for ( int stage = 0; stage < noStages; stage++ )
              {
                for ( int step = 0; step < noSteps; step++)         // Loop over steps  .
                {
                  candLists[stage][step].value = 0 ;
                }
              }
            }

            sy = 0;
          }
        }
      }
    }

#ifdef TIMING // Timing  .
    gettimeofday(&end, NULL);
    float v1 =  ((end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec))*1e-3  ;
    batch->searchTime[0] += v1;
#endif

    nvtxRangePop();
  }

  FOLD // Process candidates  .
  {
    nvtxRangePush("CPU Process results");

#ifdef TIMING // Timing  .
    gettimeofday(&start, NULL);
#endif

    for ( int c = 0; c < noCands; c++ )
    {
      int stage = log2((float)cnd[c].numharm);
      //procesCanidate(batch, cnd[c].r, cnd[c].z, cnd[c].power, 0, stage, cnd[c].numharm );
    }

#ifdef TIMING // Timing  .
    gettimeofday(&end, NULL);
    float v2 =  ((end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec))*1e-3  ;
    batch->resultTime[0] += v2;
#endif

    nvtxRangePop();
  }

  free(cnd);
}
