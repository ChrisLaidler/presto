#include "hip/hip_runtime.h"
/** @file cuda_accel_utils.cu
 *  @brief Utility functions for CUDA accelsearch
 *
 *  This contains the various utility functions for the CUDA accelsearch
 *  These include:
 *    Determining plane - widths and step size and accellen
 *    Generating kernel structures
 *    Generating plane structures
 *
 *  @author Chris Laidler
 *  @bug No known bugs.
 *
 *  Change Log
 *
 *  [0.0.01] []
 *    Beginning of change log
 *    Working version un-numbed
 *
 *  [0.0.02] [2017-01-07 10:25]
 *    Fixed bug in determining optimal plane width - half plane using correct z-max
 *
 *  [0.0.02] [2017-01-31 18:50]
 *    Fixed more bugs in accel len calculation
 *    Caged the way profiling and timing happens, introduced the PROF macro
 *    Changed GPUDefaylts text values
 *    New better ordering for asynchronous & profiling standard search (faster overlap GPU and CPU)
 *    Added many more debug messages in initialisation routines
 *    Fixed bug in iFFT stream creation
 *
 *  [0.0.03] []
 *    Added a new fag to allow separate treatment of input and plane FFT's (separate vs single)
 *    Caged createFFTPlans to allow creating the FFT plans for input and plane separately
 *    Reorder stream creation in initKernel
 *    Synchronous runs now default to one batch and separate FFT's
 *    Added ZBOUND_NORM flag to specify bound to swap over to CPU input normalisation
 *    Added ZBOUND_INP_FFT flag to specify bound to swap over to CPU FFT's for input
 *    Added 3 generic debug flags ( FLAG_DPG_TEST_1, FLAG_DPG_TEST_2, FLAG_DPG_TEST_3 )
 *
 *  [0.0.04] [2017-02-01]
 *    Fixed a bug in the ordering of the process results component in - standard, synchronous mode
 *    Re-ordered things so sum & search slices uses output stride, this means in-mem now uses the correct auto slices for sum and search
 *
 *  [0.0.05] [2017-02-01]
 *    Converted candidate processing to use a circular buffer of results in pinned memory
 *    Added a function to zero r-array, it preserves pointer to pinned host memory
 *
 *  [0.0.03] [2017-02-05]
 *    Reorder in-mem async to slightly faster (3 way)
 *
 *  [0.0.03] [2017-02-10]
 *    Multi batch async fixed finishing off search
 *
 *  [0.0.03] [2017-02-16]
 *    Separated candidate and optimisation CPU threading
 *
 *  [0.0.03] [2017-02-24]
 *     Added preprocessor directives for steps and chunks
 *
 *  [0.0.03] [2017-03-04]
 *     Work on automatic step, batch and chunk selection
 *
 *  [0.0.03] [2017-03-09]
 *     Added slicing exit for testing
 *
 *  [0.0.03] [2017-03-25]
 *  Improved multiplication chunk handling
 *  Added temporary output of chunks and step size
 *  Clamp SAS chunks to SAS slice width
 *
 *  [2017-03-30]
 *  	Fix in-mem plane size estimation to be more accurate
 *  	Added function to calculate in-mem plane size
 *  	Re worked the search size data structure and the manner number of steps is calculated
 *  	Converted some debug messages sizes from GiB to GB and MiB to MB
 *  	Added separate candidate array resolution - Deprecating FLAG_STORE_EXP
 *
 *  [2017-04-17]
 *  	Fixed clipping of multiplication chunks back to max slice height
 *
 *  [2017-04-24]
 *  	Reworked calculating the y-index and added the setPlaneBounds function
 *
 *  [2017-05-12]
 *  	Massive refactor. Moved a bunch of stuff to a separate candidate generation stage file
 *	Added the printErrors capability
 */

#include <hipfft/hipfft.h>
#include <algorithm>

#include <thrust/sort.h>
#include <thrust/device_vector.h>

extern "C"
{
#define __float128 long double
#include "accel.h"
}

#ifdef USEFFTW
#include <fftw3.h>
#endif

#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"
#include "cuda_accel_GEN.h"
#include "cuda_accel_IN.h"
#include "cuda_cand_OPT.h"

#ifdef CBL
#include <unistd.h>
#include "log.h"
#endif

int    globalInt01    = 0;
int    globalInt02    = 0;
int    globalInt03    = 0;
int    globalInt04    = 0;
int    globalInt05    = 0;

float  globalFloat01  = 0;
float  globalFloat02  = 0;
float  globalFloat03  = 0;
float  globalFloat04  = 0;
float  globalFloat05  = 0;

int     useUnopt      = 0;
int     msgLevel      = 0;

double ratioARR[] = {
    3.0 / 2.0,
    5.0 / 2.0,
    2.0 / 3.0,
    4.0 / 3.0,
    5.0 / 3.0,
    3.0 / 4.0,
    5.0 / 4.0,
    2.0 / 5.0,
    3.0 / 5.0,
    4.0 / 5.0,
    5.0 / 6.0,
    2.0 / 7.0,
    3.0 / 7.0,
    4.0 / 7.0,
    3.0 / 8.0,
    5.0 / 8.0,
    2.0 / 9.0,
    3.0 / 10.0,
    2.0 / 11.0,
    3.0 / 11.0,
    2.0 / 13.0,
    3.0 / 13.0,
    2.0 / 15.0
};






///////////////////////// Function prototypes ////////////////////////////////////


void __printErrors( ACC_ERR_CODE value, const char* file, int lineNo, const char* errorMsg)
{
  if (value)
  {
    char msg[1024] = {0};

    if ( errorMsg )
      sprintf(msg, "\n%s", errorMsg );
    else
      sprintf(msg, "ERROR: Unspecified. ( Someone was being lazy... )" );
    sprintf(msg, "\n%s\n  On line: %4i in %s\n  Error codes:\n", msg, lineNo, file );

    if (value & ACC_ERR_NAN )
    {
      value &= (~ACC_ERR_NAN);
      sprintf(msg, "%s     NAN \n", msg);
    }

    if (value & ACC_ERR_NEG )
    {
      value &= (~ACC_ERR_NEG);
      sprintf(msg, "%s     Negative value \n", msg);
    }

    if (value & ACC_ERR_STRIDE )
    {
      value &= (~ACC_ERR_STRIDE);
      sprintf(msg, "%s     Invalid stride \n", msg);
    }

    if (value & ACC_ERR_ALIGHN )
    {
      value &= (~ACC_ERR_ALIGHN);
      sprintf(msg, "%s     Alignment\n", msg);
    }

    if (value & ACC_ERR_OVERFLOW )
    {
      value &= (~ACC_ERR_OVERFLOW);
      sprintf(msg, "%s     Overflow\n", msg );
    }

    if (value & ACC_ERR_OUTOFBOUNDS )
    {
      value &= (~ACC_ERR_OUTOFBOUNDS);
      sprintf(msg, "%s     Out of bounds\n", msg );
    }

    if (value & ACC_ERR_NULL )
    {
      value &= (~ACC_ERR_NULL);
      sprintf(msg, "%s     NULL pointer\n", msg );
    }




    if (value )
    {
      sprintf(msg, "%s     Unknown? %i \n", msg, value);
    }

    fprintf(stderr, "%s\n", msg);
  }
}

void setDebugMsgLevel(int lvl)
{
  msgLevel = lvl;
}

/** Calculate an optimal accellen given a width  .
 *
 * @param width		The width of the plane (usually a power of two) if width < 100 the closes power of 2 to width*1000 will be used ie 8 -> 8024
 * @param zmax		The highest z value being searched for
 * @param accuracy	The accuracy of the kernel
 * @param noResPerBin	The resolution 2 -> interbinning
 * @return
 * If width is not a power of two it will be rounded up to the nearest power of two
 */
uint optAccellen(float width, float zmax, presto_interp_acc accuracy, int noResPerBin)
{
  double halfwidth	= cu_z_resp_halfwidth<double>(zmax, accuracy); /// The halfwidth of the maximum zmax, to calculate step size
  double pow2		= pow(2 , round(log2(width)) );
  uint oAccelLen	= floor(pow2 - 2 - 2 * halfwidth * noResPerBin );	// NOTE: I think the extra ( - 2 ) in here is not needed?

  infoMSG(6,6,"For a width %.0f and z-max %.1f with spacing %i, Halfwidth is %.0f and step size is %u. \n", pow2, zmax, noResPerBin, halfwidth, oAccelLen );
  return oAccelLen;
}

/** Calculate the step size from a width if the width is < 100 it is scaled to be the closest power of two  .
 *
 * @param width		The width of the plane (usually a power of two) if width < 100 the closes power of 2 to width*1000 will be used ie 8 -> 8024
 * @param zmax		The highest z value being searched for
 * @param accuracy	The accuracy of the kernel
 * @param noResPerBin	The resolution 2 -> interbinning
 * @return		The step size
 */
uint calcAccellen(float width, float zmax, presto_interp_acc accuracy, int noResPerBin)
{
  int accelLen;

  if ( width > 100 )
  {
    accelLen = width;
  }
  else
  {
    accelLen = optAccellen(width*1000.0, zmax, accuracy, noResPerBin) ;
  }
  return accelLen;
}

/** Calculate the step size from a width if the width is < 100 it is skate to be the closest power of two  .
 *
 * @param width		The width of the plane (usually a power of two) if width < 100 the closes power of 2 to width*1000 will be used ie 8 -> 8024
 * @param zmax		The highest z value being searched for
 * @param noHarms	The number of harmonics being summed ( power of 2 )
 * @param accuracy	The accuracy of the kernel
 * @param noResPerBin	The resolution 2 -> interbinning
 * @param zRes		The resolution of the z values
 * @param hamrDevis	Make sure the width is divisible by the number of harmonics (needed for CUDA sum and search)
 * @return		The step size
 */
uint calcAccellen(float width, float zmax, int noHarms, presto_interp_acc accuracy, int noResPerBin, float zRes, bool hamrDevis)
{
  infoMSG(5,5,"Calculating step size\n");

  uint	accelLen, oAccelLen1, oAccelLen2;

  oAccelLen1  = calcAccellen(width, zmax, accuracy, noResPerBin);
  infoMSG(6,6,"Initial optimal step size %i for a fundamental plane of width %.0f with z-max %.1f \n", oAccelLen1, width, zmax);

  if ( width > 100 )				// The user specified the exact width they want to use for accellen  .
  {
    accelLen  = oAccelLen1;
    infoMSG(6,6,"User specified step size %.0f - using: %i \n", width, oAccelLen1);
  }
  else						// Determine accellen by, examining the accellen at the second stack  .
  {
    if ( noHarms > 1 )				// Working with a family of planes
    {
      float halfZ	= cu_calc_required_z<double>(0.5, zmax, zRes);
      oAccelLen2	= calcAccellen(width*0.5, halfZ, accuracy, noResPerBin);
      accelLen		= MIN(oAccelLen2*2, oAccelLen1);

      infoMSG(6,6,"Second optimal step size %i from half plane step size of %i.\n", accelLen, oAccelLen2);
    }
    else
    {
      // Just a single plane
      accelLen		= oAccelLen1;
    }

    FOLD // Check  .
    {
      double ss        = cu_calc_fftlen<double>(1, zmax, accelLen, accuracy, noResPerBin, zRes) ;
      double l2        = log2( ss ) - 10 ;
      double fWidth    = pow(2, l2);

      if ( fWidth != width )
      {
	fprintf(stderr,"ERROR: Width calculation did not give the desired value.\n");
	exit(EXIT_FAILURE);
      }
    }
  }

  FOLD						// Ensure divisibility  .
  {
    float devisNo = 2;				// Divisible by 2, not sure why, its not noResPerBin its 2?

    if ( hamrDevis )				// Adjust to be divisible by number of harmonics  .
    {
      devisNo = noResPerBin*noHarms;
    }
    accelLen = floor( accelLen/devisNo ) * (devisNo);

    infoMSG(6,6,"Divisible %i.\n", accelLen);
  }

  return accelLen;
}

fftInfo* readFFT(char* fileName)
{
  char name[1024] = {0};
  char* suffix;
  int loc;

  printf("Opening %s \n", fileName);

  suffix = strrchr(fileName, '.');
  if ( suffix == NULL )
    return NULL;
  if (strcmp(++suffix, "fft") == 0)
  {
    fftInfo* fft = new fftInfo;
    memset(fft, 0, sizeof(fftInfo));

    FILE* fftFile	= chkfopen(fileName, "rb");
    long long filelen	= chkfilelen(fftFile, sizeof(fcomplex));
    fft->data		= (fcomplex*)malloc(sizeof(fcomplex) * filelen );
    long long fileRead	= chkfread(fft->data, sizeof(fcomplex), filelen, fftFile);
    fclose(fftFile);

    infodata * idata = new(infodata);
    //--suffix[0]=0;	// Remove suffix
    strncpy(name, fileName, suffix - fileName-1 );
    readinf(idata, name);

    fft->firstBin = 0;
    fft->lastBin = fileRead-1;
    fft->noBins = fileRead;

    fft->N = idata->N;
    fft->dt = idata->dt;
    fft->T = fft->N * fft->dt;

    return fft;
  }

  return NULL;
}

/** Set the search size parameters
 *
 * This calculates the search size parameters from the FFT, number of harmonics being summed, halfwidth and resolution
 *
 */
void setSrchSize(searchSpecs* SrchSz, int halfWidth, int noHarms, int alighnment)
{
  SrchSz->searchRHigh	= ceil  ( SrchSz->specRHigh );
  SrchSz->searchRLow	= floor ( SrchSz->specRLow  / (double)noHarms / (double)alighnment ) * alighnment ;
  SrchSz->noSearchR	= SrchSz->searchRHigh - SrchSz->searchRLow ;		/// Determine the number of candidate 'r' values

  SrchSz->rLow		= SrchSz->searchRLow  - halfWidth ;
  SrchSz->rHigh		= SrchSz->searchRHigh + halfWidth ;
  SrchSz->noInpR	= SrchSz->rHigh - SrchSz->rLow  ;  			/// The number of input data points
}

/**
 *
 * @param device
 * @param print
 * @return
 */
int selectDevice(int device, int print)
{
  hipDeviceProp_t deviceProp;
  int currentDevvice, deviceCount;  //, device = 0;

  CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount), "Failed to get device count using hipGetDeviceCount");
  //printf("There are %i CUDA capable devices available.");
  if (device>= deviceCount)
  {
    if (deviceCount== 0)
    {
      fprintf(stderr, "ERROR: Could not detect any CUDA capable devices!\n");
      exit(EXIT_FAILURE);
    }
    fprintf(stderr, "ERROR: Attempting to select device %i when I detect only %i devices, using device 0 instead!\n", device, deviceCount);
    device = 0;
  }

  CUDA_SAFE_CALL(hipSetDevice(device), "Failed to set device using hipSetDevice");
  CUDA_SAFE_CALL(hipDeviceReset(), "Failed to set device using : hipDeviceReset");
  CUDA_SAFE_CALL(hipGetLastError(), "At start of everything?.\n");
  CUDA_SAFE_CALL(hipGetDevice(&currentDevvice), "Failed to get device using hipGetDevice");
  if (currentDevvice!= device)
  {
    fprintf(stderr, "ERROR: CUDA Device not set.\n");
    exit(EXIT_FAILURE);
  }

  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, currentDevvice), "Failed to get device properties device using hipGetDeviceProperties");

  if (print)
    printf("\nRunning on device %d: \"%s\"  which has CUDA Capability  %d.%d\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);

  return ((deviceProp.major<< 4)+ deviceProp.minor);
}

void printCands(const char* fileName, GSList *cands, double T)
{
  if ( cands == NULL  )
    return;

  GSList *inp_list = cands ;

  FILE * myfile;                    /// The file being written to
  myfile = fopen ( fileName, "w" );

  if ( myfile == NULL )
    fprintf ( stderr, "ERROR: Unable to open log file %s\n", fileName );
  else
  {
    fprintf(myfile, "%4s\t%14s\t%10s\t%14s\t%13s\t%9s\t%7s\t%2s \n", "#", "r", "f", "z", "fd", "sig", "power", "harm" );
    int i = 0;

    while ( inp_list )
    {
      fprintf(myfile, "%4i\t%14.5f\t%10.6f\t%14.2f\t%13.10f\t%9.4f\t%7.2f\t%2i\n", i+1, ((accelcand *) (inp_list->data))->r, ((accelcand *) (inp_list->data))->r / T, ((accelcand *) (inp_list->data))->z,((accelcand *) (inp_list->data))->z/T/T, ((accelcand *) (inp_list->data))->sigma, ((accelcand *) (inp_list->data))->power, ((accelcand *) (inp_list->data))->numharm );
      inp_list = inp_list->next;
      i++;
    }
    fclose ( myfile );
  }
}

void initGPUs(gpuSpecs* gSpec)
{
  int currentDevvice, deviceCount;
  char txt[1024];

  int major           = 0;
  int minor           = 0;

  CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount), "Failed to get device count using hipGetDeviceCount");

  for (int dIdx = 0; dIdx < gSpec->noDevices; dIdx++)
  {
    int device    = gSpec->devId[dIdx];
    gpuInf* gInf  = &gSpec->devInfo[dIdx];

    CUDA_SAFE_CALL( hipSetDevice ( device ), "Failed to set device using hipSetDevice");

    // Check if the the current device is 'device'
    CUDA_SAFE_CALL( hipGetDevice(&currentDevvice), "Failed to get device using hipGetDevice" );

    if ( currentDevvice != device)
    {
      fprintf(stderr, "ERROR: Device not set.\n");
    }
    else // call something to initialise the device
    {
      sprintf(txt,"Init device %02i", device );

      PROF // Profiling  .
      {
	NV_RANGE_PUSH(txt);
      }

      hipDeviceProp_t deviceProp;
      CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, device), "Failed to get device properties device using hipGetDeviceProperties");

      major                           = deviceProp.major;
      minor                           = deviceProp.minor;
      gInf->capability                = major + minor/10.0f;
      gInf->alignment                 = getMemAlignment();                  // This action will initialise the CUDA context
      gInf->devid                     = device;
      gInf->name                      = (char*)malloc(256*sizeof(char));

      sprintf(gInf->name, "%s", deviceProp.name );

      // TODO: Profile this
      CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1),"Failed to set cache config"); // hipFuncCachePreferNone OR hipFuncCachePreferShared OR hipFuncCachePreferL1 OR hipFuncCachePreferEqual

      PROF // Profiling  .
      {
	NV_RANGE_POP(); // txt
      }
    }
  }
}

gpuSpecs* getGpuSpec(int devID, int batch, int steps, int opts)
{
  gpuSpecs* gSpec = new gpuSpecs;
  memset(gSpec, 0, sizeof(gpuSpecs));

  if (devID < 0 )
  {
    gSpec->noDevices		= getGPUCount();

    for ( int i = 0; i < gSpec->noDevices; i++)
      gSpec->devId[i]		= i;
  }
  else
  {
    gSpec->noDevices		= 1;
    gSpec->devId[0]		= devID;

    gSpec->noDevBatches[0]	= batch;
    gSpec->noDevSteps[0]	= steps;
    gSpec->noDevOpt[0]		= opts;
  }

//  // Set default
//  for ( int i = 0; i < gSpec->noDevices; i++)
//  {
//    gSpec->noDevBatches[i]	= 0;
//    gSpec->noDevSteps[i]	= 0;
//    gSpec->noDevOpt[i]		= 0;
//  }



  return gSpec;
}

/**  Read the GPU details from clig command line  .
 *
 * @param cmd     clig struct
 * @param bInf    A pointer to the accel info struct to fill
 */
gpuSpecs* readGPUcmd(Cmdline *cmd)
{
  gpuSpecs* gpul = new gpuSpecs;
  memset(gpul, 0, sizeof(gpuSpecs));

  CUDA_SAFE_CALL(hipGetLastError(), "Entering readGPUcmd.");

  if ( cmd->gpuP ) // Determine the index and number of devices
  {
    if ( cmd->gpuC == 0 )  // NB: Note using gpuC == 0 requires a change in accelsearch_cmd.c every time clig is run!!!! [ usually line 32 should be "  /* gpuC = */ 0," ]
    {
      // Make a list of all devices
      gpul->noDevices   = getGPUCount();
      for ( int dev = 0 ; dev < gpul->noDevices; dev++ )
	gpul->devId[dev] = dev;
    }
    else
    {
      // User specified devices(s)
      gpul->noDevices		= cmd->gpuC;
      for ( int dev = 0 ; dev < gpul->noDevices; dev++ )
	gpul->devId[dev]	= cmd->gpu[dev];
    }
  }

  for ( int dev = 0 ; dev < gpul->noDevices; dev++ ) // Loop over devices  .
  {
    if ( dev >= cmd->nbatchC )
      gpul->noDevBatches[dev]	= cmd->nbatch[cmd->nbatchC-1];
    else
      gpul->noDevBatches[dev]	= cmd->nbatch[dev];

    if ( dev >= cmd->nstepsC )
      gpul->noDevSteps[dev]	= cmd->nsteps[cmd->nbatchC-1];
    else
      gpul->noDevSteps[dev]	= cmd->nsteps[dev];

    if ( dev >= cmd->numoptC )
      gpul->noDevOpt[dev]	= cmd->numopt[cmd->nbatchC-1];
    else
      gpul->noDevOpt[dev]	= cmd->numopt[dev];

  }

  return gpul;
}

bool strCom(const char* str1, const char* str2)
{
  if ( strncmp(str1,str2, strlen(str2) ) == 0 )
    return 1;
  else
    return 0;
}

bool singleFlag ( int64_t*  flags, const char* str1, const char* str2, int64_t flagVal, const char* onVal, const char* offVal, int lineno, const char* fName )
{
  if      ( strCom("1", str2 ) || strCom(onVal, str2 ) )
  {
    (*flags) |=  flagVal;
    return true;
  }
  else if ( strCom("0", str2 ) || strCom(offVal, str2 ) )
  {
    (*flags) &= ~flagVal;
  }
  else if ( strCom(str2, "#" ) || strCom("", str2 )  )
  {
    // Blank do nothing
  }
  else
  {
    fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
  }
  return false;
}

/** Read accel search details from the text file
 *
 * @param sSpec
 */
void readAccelDefalts(confSpecs *conf)
{
  int64_t*  genFlags = &(conf->gen->flags);
  int64_t*  optFlags = &(conf->opt->flags);

  FILE *file;
  char fName[1024];
  sprintf(fName, "%s/lib/GPU_defaults.txt", getenv("PRESTO"));

  file = fopen(fName, "r");
  if ( file )  // Read candidates from previous search  .
  {
    printf("Reading GPU search settings from %s\n",fName);

    char* line;
    char  line2[1024];
    int   lineno = 0;

    char str1[1024];
    char str2[1024];

    char *rest;

    while (fgets(line2, sizeof(line2), file))
    {
      lineno++;

      line = line2;

      // Strip proceeding white space
      while ( *line <= 32 &&  *line != 10 )
	line++;

      // Set to only be the word
      int flagLen = 0;
      char* flagEnd = line;
      while ( *flagEnd != ' ' && *flagEnd != 0 && *flagEnd != 10 )
      {
	flagLen++;
	flagEnd++;
      }

      int ll = strlen(line);

      str2[0] = 0;
      int pRead = sscanf(line, "%s %s", str1, str2 );
      if ( str2[0] == '#' )
	str2[0] = 0;

      if ( strCom(str1, "#" ) || ( ll == 1 ) )                  // Comment line
      {
	continue;
      }

      else if ( strCom(str1, "DUMMY" ) )                        // Dummy parameter
      {
	continue;
      }

      else if ( strCom("FLAG_SEPSRCH", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_SEPSRCH, "", "0", lineno, fName );
      }

      else if ( strCom("R_RESOLUTION", str1 ) )
      {
	int no1;
	int read1 = sscanf(line, "%s %i %s", str1, &no1, str2 );
	if ( read1 >= 2 )
	{
	  if ( no1 >= 1 && no1 <= 16 )
	  {
	    conf->gen->noResPerBin = no1;
	  }
	  else
	  {
	    fprintf(stderr,"WARNING: Invalid optimisation resolution, it should range between 1 and 16 \n");
	  }
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("Z_RESOLUTIOM", str1 ) )
      {
	float no1;
	int read1 = sscanf(line, "%s %f %s", str1, &no1, str2 );
	if ( read1 >= 2 )
	{
	  if ( no1 > 0 && no1 <= 16 )
	  {
	    conf->gen->zRes = no1;
	  }
	  else
	  {
	    fprintf(stderr,"WARNING: Invalid optimisation resolution, it should range between 1 and 16 \n");
	  }
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("FLAG_Z_SPLIT", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_Z_SPLIT, "", "0", lineno, fName );
      }

      else if ( strCom(line, "RESULTS_RING" ) )			// The size of the per batch results ring buffer
      {
	int no1;
	int read1 = sscanf(line, "%s %i %s", str1, &no1, str2 );
	if ( read1 >= 2 )
	{
	  if ( no1 >= 3 && no1 <= 16 )
	  {
	    conf->gen->ringLength = no1;
	  }
	  else
	  {
	    fprintf(stderr,"WARNING: Invalid ring size (%s), it should range between 3 and 16 \n", str1);
	  }
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("INTERLEAVE", str1 ) ||  strCom("IL", str1 ) )   // Interleaving
      {
	singleFlag ( genFlags, str1, str2, FLAG_ITLV_ROW, "ROW", "PLN", lineno, fName );
      }

      else if ( strCom("RESPONSE", str1 ) )                     // Response shape
      {
	singleFlag ( genFlags, str1, str2, FLAG_KER_HIGH, "HIGH", "STD", lineno, fName );
      }

      else if ( strCom("FLAG_KER_HIGH", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_KER_HIGH, "", "0", lineno, fName );
      }

      else if ( strCom("FLAG_KER_MAX", str1 ) )                 // Kernel
      {
	singleFlag ( genFlags, str1, str2, FLAG_KER_MAX, "", "0", lineno, fName );
      }

      else if ( strCom("CENTER_RESPONSE", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_CENTER, "", "off", lineno, fName );
      }

      else if ( strCom("RESPONSE_PRECISION", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_KER_DOUBGEN, "DOUBLE", "SINGLE", lineno, fName );
      }

      else if ( strCom("KER_FFT_PRECISION", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_KER_DOUBFFT, "DOUBLE", "SINGLE", lineno, fName );
      }

      else if ( strCom("INP_NORM",	str1 ) )
      {
	(*genFlags) &= ~CU_NORM_GPU;	// Clear values

	if      ( strCom("CPU",  str2 ) || strCom(str2, "A" ) )
	{
	  // CPU is no value clear is sufficient
	}
	else if ( strCom("GPU_SM", str2 ) || strCom("GPU", str2 ) )
	{
	  (*genFlags) |= CU_NORM_GPU_SM;
	}
	else if ( strCom("GPU_SM_MIN", str2 ) || strCom("GPU_SM2", str2 ))
	{
	  (*genFlags) |= CU_NORM_GPU_SM_MIN;
	}
	else if ( strCom("GPU_OS", str2 ) )
	{
	  (*genFlags) |= CU_NORM_GPU_OS;
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
	}
      }

      else if ( strCom("ZBOUND_NORM", str1 ) )
      {
	float no1;
	int read1 = sscanf(line, "%s %f %s", str1, &no1, str2 );
	if ( no1 < 0 )
	{
	  if ( no1 < -1 )
	  {
	    fprintf(stderr,"WARNING: Invalid bound (%.1f) on CPU normalisation, value must be >= 0.  Ignoring value.\n", no1 );
	  }
	}
	else
	{
	  conf->gen->inputNormzBound = no1;
	}
      }

      else if ( strCom("INP_FFT", str1 ) )
      {
	if      ( strCom(str2, "A") )
	{
	  // Default to GPU FFT's - CPU FFT's may be worth doing if z-max is lager than 50 or 100 depends on the CPU and GPU
	  (*genFlags) &= ~CU_INPT_FFT_CPU;
	}
	else if ( singleFlag ( genFlags, str1, str2, CU_INPT_FFT_CPU, "CPU", "GPU", lineno, fName ) )
	{
	  if ( (*genFlags) & CU_NORM_GPU )  // TMP REM - Added to mark an error for thesis timing
	  {
	    printf("Temporary exit - input FFT / NORM \n");
	    exit(EXIT_FAILURE);
	  }

	  // IF we are doing CPU FFT's we need to do CPU normalisation
	  (*genFlags) &= ~CU_NORM_GPU;
	}
      }

      else if ( strCom("ZBOUND_FFT", str1 ) )
      {
	float no1;
	int read1 = sscanf(line, "%s %f %s", str1, &no1, str2 );
	if ( no1 < 0 )
	{
	  if ( no1 < -1 )
	  {
	    fprintf(stderr,"WARNING: Invalid bound (%.1f) on input FFT, value must be >= 0.  Ignoring value.\n", no1 );
	  }
	}
	else
	{
	  conf->gen->inputFFFTzBound = no1;
	}
      }

      else if ( strCom("MUL_KER", str1 ) )
      {
	if      ( strCom("00", str2 ) )
	{
#if defined(WITH_MUL_00) || defined(WITH_MUL_01) || defined(WITH_MUL_02)
	  (*genFlags) &= ~FLAG_MUL_ALL;
	  (*genFlags) |=  FLAG_MUL_00;
#else
	  line[flagLen] = 0;
	  fprintf(stderr, "WARNING: Not compiled with multiplication 2.3 kernel.  (FLAG: %s line %i in %s)\n", line, lineno, fName);

	  FOLD  // TMP REM - Added to mark an error for thesis timing
	  {
	    printf("Temporary exit - mult Kernel \n");
	    exit(EXIT_FAILURE);
	  }
#endif	// WITH_MUL_00 WITH_MUL_01 WITH_MUL_02
	}
	else if ( strCom("11", str2 ) )
	{
#ifdef WITH_MUL_11
	  (*genFlags) &= ~FLAG_MUL_ALL;
	  (*genFlags) |=  FLAG_MUL_11;
#else	// WITH_MUL_11
	  line[flagLen] = 0;
	  fprintf(stderr, "WARNING: Not compiled with multiplication 1.1 kernel.  (FLAG: %s line %i in %s)\n", line, lineno, fName);

	  FOLD  // TMP REM - Added to mark an error for thesis timing
	  {
	    printf("Temporary exit - mult Kernel \n");
	    exit(EXIT_FAILURE);
	  }
#endif	// WITH_MUL_11
	}
	else if ( strCom("21", str2 ) )
	{
#ifdef WITH_MUL_21
	  (*genFlags) &= ~FLAG_MUL_ALL;
	  (*genFlags) |=  FLAG_MUL_21;
#else	// WITH_MUL_21
	  line[flagLen] = 0;
	  fprintf(stderr, "WARNING: Not compiled with multiplication 2.1 kernel.  (FLAG: %s line %i in %s)\n", line, lineno, fName);

	  FOLD  // TMP REM - Added to mark an error for thesis timing
	  {
	    printf("Temporary exit - mult Kernel \n");
	    exit(EXIT_FAILURE);
	  }
#endif	// WITH_MUL_21
	}
	else if ( strCom("22", str2 ) )
	{
#ifdef WITH_MUL_22
	  (*genFlags) &= ~FLAG_MUL_ALL;
	  (*genFlags) |=  FLAG_MUL_22;
#else	// WITH_MUL_22
	  line[flagLen] = 0;
	  fprintf(stderr, "WARNING: Not compiled with multiplication 2.2 kernel.  (FLAG: %s line %i in %s)\n", line, lineno, fName);

	  FOLD  // TMP REM - Added to mark an error for thesis timing
	  {
	    printf("Temporary exit - mult Kernel \n");
	    exit(EXIT_FAILURE);
	  }
#endif	// WITH_MUL_22
	}
	else if ( strCom("23", str2 ) )
	{
#ifdef WITH_MUL_23
	  (*genFlags) &= ~FLAG_MUL_ALL;
	  (*genFlags) |=  FLAG_MUL_23;
#else	// WITH_MUL_23
	  line[flagLen] = 0;
	  fprintf(stderr, "WARNING: Not compiled with multiplication 2.3 kernel.  (FLAG: %s line %i in %s)\n", line, lineno, fName);

	  FOLD  // TMP REM - Added to mark an error for thesis timing
	  {
	    printf("Temporary exit - mult Kernel \n");
	    exit(EXIT_FAILURE);
	  }
#endif	// WITH_MUL_23
	}
	else if ( strCom("31", str2 ) )
	{
#ifdef WITH_MUL_31
	  (*genFlags) &= ~FLAG_MUL_ALL;
	  (*genFlags) |=  FLAG_MUL_31;
#else	// WITH_MUL_31
	  line[flagLen] = 0;
	  fprintf(stderr, "WARNING: Not compiled with multiplication 3.1 kernel.  (FLAG: %s line %i in %s)\n", line, lineno, fName);

	  FOLD  // TMP REM - Added to mark an error for thesis timing
	  {
	    printf("Temporary exit - mult Kernel \n");
	    exit(EXIT_FAILURE);
	  }
#endif	// WITH_MUL_31
	}
	else if ( strCom("CB", str2 ) )
	{
#if CUDA_VERSION >= 6050

#ifdef	WITH_MUL_PRE_CALLBACK
	  (*genFlags) &= ~FLAG_MUL_ALL;
	  (*genFlags) |=  FLAG_MUL_CB;
#else
	  line[flagLen] = 0;
	  fprintf(stderr, "WARNING: Not compiled with multiplication through CUFFT callbacks enabled.  (FLAG: %s line %i in %s)\n", line, lineno, fName);

	  FOLD  // TMP REM - Added to mark an error for thesis timing
	  {
	    printf("Temporary exit - mult Kernel \n");
	    exit(EXIT_FAILURE);
	  }
#endif
#else
	  line[flagLen] = 0;
	  fprintf(stderr, "WARNING: Use of CUDA callbacks requires CUDA 6.5 or greater.  (FLAG: %s line %i in %s)\n", line, lineno, fName);
#endif
	}
	else if ( strCom(str2, "A"  ) )
	{
	  (*genFlags) &= ~FLAG_MUL_ALL;
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);

	  FOLD  // TMP REM - Added to mark an error for thesis timing
	  {
	    printf("Temporary exit - mult Kernel \n");
	    exit(EXIT_FAILURE);
	  }
	}
      }

      else if ( strCom("MUL_TEXTURE", str1 ) )
      {
	fprintf(stderr, "WARNING: The flag %s has been deprecated.\n", str1);
      }

      else if ( strCom("MUL_SLICES", str1 ) )
      {
	if ( strCom(str2, "A"   ) )
	{
	  conf->gen->mulSlices = 0;
	}
	else
	{
	  int no;
	  int read1 = sscanf(str2, "%i", &no  );
	  if ( read1 == 1 )
	  {
	    conf->gen->mulSlices = no;
	  }
	  else
	  {
	    line[flagLen] = 0;
	    fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	  }
	}
      }

      else if ( strCom("MUL_CHUNK", str1 ) )
      {
	if ( strCom(str2, "A"   ) )
	{
	  conf->gen->mulChunk = 0;
	}
	else
	{
	  int no;
	  int read1 = sscanf(str2, "%i", &no  );
	  if ( read1 == 1 )
	  {
	    conf->gen->mulChunk = no;
	  }
	  else
	  {
	    line[flagLen] = 0;
	    fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	  }
	}
      }

      else if ( strCom("CONVOLVE", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_CONV, "SEP", "CONT", lineno, fName );
      }

      else if ( strCom("STACK", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_STK_UP, "UP", "DN", lineno, fName );
      }

      else if ( strCom("CUFFT_PLAN_INP", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, CU_FFT_SEP_INP, "SEPARATE", "SINGLE", lineno, fName );
      }

      else if ( strCom("CUFFT_PLAN_PLN", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, CU_FFT_SEP_PLN, "SEPARATE", "SINGLE", lineno, fName );
      }

      else if ( strCom("STD_POWERS", str1 ) )
      {
	if      ( strCom("CB", str2 ) )
	{
#if CUDA_VERSION >= 6050
	  (*genFlags) |=     FLAG_CUFFT_CB_POW;
#else
	  line[flagLen] = 0;
	  fprintf(stderr,"WARNING: Use of CUDA callbacks requires CUDA 6.5 or greater.  (FLAG: %s line %i in %s)\n", line, lineno, fName);
#endif
	}
	else if ( strCom("SS", str2 ) )
	{
	  (*genFlags) &= ~FLAG_CUFFT_CB_POW;
	}
	else if ( strCom(str2, "#" ) || strCom("", str2 ) || strCom(str2, "A" ) )
	{
	  // Blank do nothing
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
	}
      }

      else if ( strCom("IN_MEM_POWERS", str1 ) )
      {
	if      ( strCom("CB", str2 ) )
	{
#if CUDA_VERSION >= 6050
	  (*genFlags) |=     FLAG_CUFFT_CB_INMEM;
#else
	  line[flagLen] = 0;
	  fprintf(stderr,"WARNING: Use of CUDA callbacks requires CUDA 6.5 or greater.  (FLAG: %s %s line %i in %s)\n", str1, str2, lineno, fName);
#endif
	}
	else if ( strCom("MEM_CPY", str2 ) || strCom("", str2 ))
	{
#if CUDA_VERSION >= 6050
	  (*genFlags) &=    ~FLAG_CUFFT_CB_INMEM;
	  (*genFlags) |=     FLAG_CUFFT_CB_POW;
#else
	  line[flagLen] = 0;
	  fprintf(stderr,"WARNING: Use of CUDA callbacks requires CUDA 6.5 or greater.  (FLAG: %s %s line %i in %s)\n", str1, str2, lineno, fName);
#endif
	}
	else if ( strCom("KERNEL", str2 ) )
	{
	  (*genFlags) &=    ~FLAG_CUFFT_CB_INMEM;
	  (*genFlags) &=    ~FLAG_CUFFT_CB_POW;
	}
	else if ( strCom(str2, "#" ) || strCom("", str2 ) || strCom(str2, "A" )  )
	{
	  // Blank do nothing
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
	}
      }

      else if ( strCom("FLAG_NO_CB", str1 ) )
      {
	(*genFlags) &= ~FLAG_CUFFT_ALL;
      }

      else if ( strCom("POWER_PRECISION", str1 ) )
      {
	if      ( strCom("HALF",   str2 ) )
	{
#if CUDA_VERSION >= 7050
	  (*genFlags) |=  FLAG_POW_HALF;
#else
	  (*genFlags) &= ~FLAG_POW_HALF;

	  line[flagLen] = 0;
	  fprintf(stderr,"WARNING: Half precision can only be used with CUDA 7.5 or later! Reverting to single precision. (FLAG: %s line %i in %s)\n", line, lineno, fName);
#endif
	}
	else if ( strCom("SINGLE", str2 ) )
	{
	  (*genFlags) &= ~FLAG_POW_HALF;
	}
	else if ( strCom("DOUBLE", str2 ) )
	{
	  fprintf(stderr,"ERROR: Cannot sore in-mem plane as double! Defaulting to float.\n");
	  (*genFlags) &= ~FLAG_POW_HALF;
	}
	else if ( strCom(str2, "#" ) || strCom("", str2 ) || strCom(str2, "A" )  )
	{
	  // Blank do nothing
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
	}
      }

      else if ( strCom("SS_KER", str1 ) )
      {
	if      ( strCom("00",  str2 ) )
	{
	  (*genFlags) &= ~FLAG_SS_ALL;
	  (*genFlags) |= FLAG_SS_00;
	  (*genFlags) |= FLAG_STAGES;
	}
	else if ( strCom("CPU", str2 ) )
	{
	  fprintf(stderr, "ERROR: CPU Sum and search is no longer supported.\n\n");
	  continue;

	  (*genFlags) &= ~FLAG_SS_ALL;
	  (*genFlags) |= FLAG_SS_CPU;

	  conf->gen->retType &= ~CU_SRT_ALL   ;
	  conf->gen->retType |= CU_STR_PLN    ;

	  if ( (*genFlags) & FLAG_CUFFT_CB_POW )
	  {
	    conf->gen->retType &= ~CU_TYPE_ALLL   ;
	    conf->gen->retType |= CU_FLOAT        ;
	  }
	  else
	  {
	    conf->gen->retType &= ~CU_TYPE_ALLL   ;
	    conf->gen->retType |= CU_CMPLXF       ;
	  }
	}
	else if ( strCom("10",  str2 ) || strCom("31",  str2 ) )
	{
	  (*genFlags) &= ~FLAG_SS_ALL;
	  (*genFlags) |= FLAG_SS_31;
	  (*genFlags) |= FLAG_STAGES;
	}
	else if ( strCom("INMEM", str2 ) || strCom("IM", str2 ) )
	{
	  (*genFlags) |= FLAG_SS_INMEM;
	}
	else if ( strCom(str2, "A"  ) )
	{
	  (*genFlags) &= ~FLAG_SS_ALL;
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
	}
      }

      else if ( strCom("SS_COUNT", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_SS_COUNT, "", "0", lineno, fName );
#ifndef WITH_SAS_COUNT
	fprintf(stderr,"WARNING: Not compiled with Sum & search counting enabled. Config on line %i in %s has no effect.\n", lineno, fName );
#endif
      }

      else if ( strCom("SS_SLICES", str1 ) )
      {
	if ( strCom(str2, "A"   ) )
	{
	  conf->gen->ssSlices = 0;
	}
	else
	{
	  int no;
	  int read1 = sscanf(str2, "%i", &no  );
	  if ( read1 == 1 )
	  {
	    conf->gen->ssSlices = no;
	  }
	  else
	  {
	    line[flagLen] = 0;
	    fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	  }
	}
      }

      else if ( strCom("SS_CHUNK", str1 ) )
      {
	if ( strCom(str2, "A"   ) )
	{
	  conf->gen->ssChunk = 0;
	}
	else
	{
	  int no;
	  int read1 = sscanf(str2, "%i", &no  );
	  if ( read1 == 1 )
	  {
	    if ( no <= 0 )		// Auto
	    {
	      conf->gen->ssChunk = 0;
	    }
	    else if ( (no >= MIN_SAS_CHUNK) and (no <= MAX_SAS_CHUNK) )
	    {
	      conf->gen->ssChunk = no;
	    }
	    else
	    {
	      fprintf(stderr, "WARNING: Sum & search chunk size not in compiled bounds (%i - %i). Line %i of %s.\n", MIN_SAS_CHUNK, MAX_SAS_CHUNK, lineno, fName);
	      conf->gen->ssChunk = 0;

	      FOLD  // TMP REM - Added to mark an error for thesis timing
	      {
		printf("Temporary exit - ssChunk \n");
		exit(EXIT_FAILURE);
	      }
	    }
	  }
	  else
	  {
	    line[flagLen] = 0;
	    fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	  }
	}
      }

      else if ( strCom("SS_COLUMN", str1 ) )
      {
	if ( strCom(str2, "A"   ) )
	{
	  conf->gen->ssColumn = 0;
	}
	else
	{
	  int no;
	  int read1 = sscanf(str2, "%i", &no  );
	  if ( read1 == 1 )
	  {
	    if ( no <= 0 )		// Auto
	    {
	      conf->gen->ssColumn = 0;
	    }
	    else if ( (no >= MIN_SAS_COLUMN) and (no <= MAX_SAS_COLUMN) )
	    {
	      conf->gen->ssColumn = no;
	    }
	    else
	    {
	      fprintf(stderr, "WARNING: Sum & search column size not in compiled bounds (%i - %i). Line %i of %s.\n", MIN_SAS_COLUMN, MAX_SAS_COLUMN, lineno, fName);
	      conf->gen->ssColumn = 0;

	      FOLD  // TMP REM - Added to mark an error for thesis timing
	      {
		printf("Temporary exit - ssColumn \n");
		exit(EXIT_FAILURE);
	      }
	    }
	  }
	  else
	  {
	    line[flagLen] = 0;
	    fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	  }
	}
      }

      else if ( strCom("SS_INMEM_SZ", str1 ) )
      {
	if ( strCom(str2, "A"   ) )
	{
	  conf->gen->ssStepSize = 0;
	}
	else
	{
	  int no;
	  int read1 = sscanf(str2, "%i", &no  );
	  if ( read1 == 1 )
	  {
	    conf->gen->ssStepSize = no;
	  }
	  else
	  {
	    line[flagLen] = 0;
	    fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	  }
	}
      }

      else if ( strCom("CAND_PROCESS", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_CAND_THREAD, "THREAD", "SEQ", lineno, fName );
      }

      else if ( strCom("CAND_MEM", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_CAND_MEM_PRE, "PRE", "RING", lineno, fName );
      }

      else if ( strCom("CAND_STORAGE", str1 ) )
      {
	if      ( strCom("ARR", str2 ) || strCom("", str2 ) )
	{
	  // Return type
	  conf->gen->retType &= ~CU_TYPE_ALLL ;
	  conf->gen->retType &= ~CU_SRT_ALL   ;

	  conf->gen->retType |= CU_POWERZ_S   ;
	  conf->gen->retType |= CU_STR_ARR    ;

	  // Candidate type
	  conf->gen->cndType &= ~CU_TYPE_ALLL ;
	  conf->gen->cndType &= ~CU_SRT_ALL   ;

	  conf->gen->cndType |= CU_CANDFULL   ;
	  conf->gen->cndType |= CU_STR_ARR    ;
	}
	else if ( strCom("LST", str2 ) )
	{
	  // Return type
	  conf->gen->retType &= ~CU_TYPE_ALLL ;
	  conf->gen->retType &= ~CU_SRT_ALL   ;

	  conf->gen->retType |= CU_POWERZ_S   ;
	  conf->gen->retType |= CU_STR_ARR    ;

	  // Candidate type
	  conf->gen->cndType &= ~CU_TYPE_ALLL ;
	  conf->gen->cndType &= ~CU_SRT_ALL   ;

	  conf->gen->cndType |= CU_CANDFULL   ;
	  conf->gen->cndType |= CU_STR_LST    ;
	}
	else if ( strCom("QUAD", str2 ) )
	{
	  fprintf(stderr, "ERROR: Quadtree storage not yet implemented. Doing nothing!\n");
	  continue;

	  // Candidate type
	  conf->gen->cndType &= ~CU_TYPE_ALLL ;
	  conf->gen->cndType &= ~CU_SRT_ALL   ;

	  conf->gen->cndType |= CU_POWERZ_S   ;
	  conf->gen->cndType |= CU_STR_QUAD   ;
	}
	else if ( strCom(str2, "#" ) || strCom("", str2 )  )
	{
	  // Blank do nothing
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
	}
      }

      else if ( strCom("ARR_RES", str1 ) )
      {
	float no1;
	int read1 = sscanf(line, "%s %f %s", str1, &no1, str2 );
	if ( read1 >= 2 )
	{
	  if ( no1 >= 0.1 && no1 <= 1.0 )
	  {
	    conf->gen->candRRes = no1;
	  }
	  else
	  {
	    fprintf(stderr,"WARNING: Invalid candidate array resolution, it should range between 0.1 and 1 \n");
	  }
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("RETURN", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_STAGES, "STAGES", "FINAL", lineno, fName );
      }

      else if ( strCom("FLAG_RET_ARR", str1 ) )
      {
	conf->gen->retType &= ~CU_SRT_ALL   ;
	conf->gen->retType |= CU_STR_ARR    ;
      }
      else if ( strCom("FLAG_RET_PLN", str1 ) )
      {
	conf->gen->retType &= ~CU_SRT_ALL   ;
	conf->gen->retType |= CU_STR_PLN    ;
      }

      else if ( strCom("FLAG_STORE_ALL", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_STORE_ALL, "", "0", lineno, fName );
      }

      //////////////  OPTEMISATION  \\\\\\\\\\\\\\\\\\\\\\\\

      else if ( strCom("OPT_METHOUD", str1 ) )
      {
	if      ( strCom("PLANE", str2 ) )
	{
	  (*optFlags) &= ~FLAG_OPT_ALL;
	}
	else if ( strCom("SWARM", str2 ) )
	{
	  (*optFlags) &= ~FLAG_OPT_ALL;
	  (*optFlags) |= FLAG_OPT_SWARM;
	}
	else if ( strCom("NM", str2 ) )
	{
	  (*optFlags) &= ~FLAG_OPT_ALL;
	  (*optFlags) |= FLAG_OPT_NM;
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
	}
      }

      else if ( strCom("OPT_Z_RATIO", str1 ) )
      {
	float no1;
	int read1 = sscanf(line, "%s %f %s", str1, &no1, str2 );
	if ( read1 >= 2 )
	{
	  if ( no1 >= 0 && no1 <= 100 )
	  {
	    conf->opt->zScale = no1;
	  }
	  else
	  {
	    fprintf(stderr,"WARNING: Invalid optimisation scale, it should range between 0 and 100 \n");
	  }
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("OPT_R_RES", str1 ) )
      {
	int no1;
	int read1 = sscanf(line, "%s %i %s", str1, &no1, str2 );
	if ( read1 >= 2 )
	{
	  if ( no1 >= 1 && no1 <= 128 )
	  {
	    conf->opt->optResolution = no1;
	  }
	  else
	  {
	    fprintf(stderr,"WARNING: Invalid optimisation resolution, it should range between 1 and 128 \n");
	  }
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("OPT_NORM", str1 ) )
      {
	singleFlag ( optFlags, str1, str2, FLAG_OPT_NRM_LOCAVE, "LOCAVE", "MEDIAN", lineno, fName );

	if      ( strCom("NONE", str2 ) )
	{
	  (*optFlags) &= ~FLAG_OPT_NRM_ALL;
	}
	else if ( strCom("LOCAVE", str2 ) )
	{
	  (*optFlags) &= ~FLAG_OPT_NRM_ALL;
	  (*optFlags) |= FLAG_OPT_NRM_LOCAVE;
	}
	else if ( strCom("MEDIAN1D", str2 ) )
	{
	  (*optFlags) &= ~FLAG_OPT_NRM_ALL;
	  (*optFlags) |= FLAG_OPT_NRM_MEDIAN1D;
	}
	else if ( strCom("MEDIAN2D", str2 ) )
	{
	  (*optFlags) &= ~FLAG_OPT_NRM_ALL;
	  (*optFlags) |= FLAG_OPT_NRM_MEDIAN2D;
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
	}
      }

      else if ( strCom("FLAG_OPT_BEST", str1 ) )
      {
	singleFlag ( optFlags, str1, str2, FLAG_OPT_BEST, "", "0", lineno, fName );
      }

      else if ( strCom("OPT_MIN_LOC_HARMS", str1 ) )
      {
	int no;
	int read1 = sscanf(str2, "%i", &no  );
	if ( read1 == 1 )
	{
	  if ( no >= 1 && no <= OPT_MAX_LOC_HARMS )
	  {
	    conf->opt->optMinLocHarms = no;
	  }
	  else
	  {
	    fprintf(stderr,"WARNING: Invalid value, %s should range between 1 and %i \n", str1, OPT_MAX_LOC_HARMS);
	  }
	}
	else
	{
	  line[flagLen] = 0;
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("OPT_MIN_REP_HARMS", str1 ) )
      {
	int no;
	int read1 = sscanf(str2, "%i", &no  );
	if ( read1 == 1 )
	{
	  conf->opt->optMinRepHarms = no;
	}
	else
	{
	  line[flagLen] = 0;
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("optPlnScale", str1 ) )
      {
	float no;
	int read1 = sscanf(str2, "%f", &no  );
	if ( read1 == 1 )
	{
	  conf->opt->optPlnScale = no;
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("FLAG_OPT_DYN_HW", str1 ) )
      {
	singleFlag ( optFlags, str1, str2, FLAG_OPT_DYN_HW, "", "0", lineno, fName );
      }

      else if ( strCom("OPT_NELDER_MEAD_REFINE", str1 ) )
      {
	singleFlag ( optFlags, str1, str2, FLAG_OPT_NM_REFINE, "", "0", lineno, fName );
      }

      else if ( strCom("OPT_PROCESS", str1 ) )
      {
	singleFlag ( optFlags, str1, str2, FLAG_OPT_THREAD, "THREAD", "SEQ", lineno, fName );
      }

      else if ( strCom("optPlnSiz", str1 ) )
      {
	int no1;
	int no2;
	int read1 = sscanf(line, "%s %i %i", str1, &no1, &no2 );
	if ( read1 == 3 )
	{
	  if    ( no1 == 1 )
	  {
	    conf->opt->optPlnSiz[0] = no2;
	  }
	  else if ( no1 == 2 )
	  {
	    conf->opt->optPlnSiz[1] = no2;
	  }
	  else if ( no1 == 4 )
	  {
	    conf->opt->optPlnSiz[2] = no2;
	  }
	  else if ( no1 == 8 )
	  {
	    conf->opt->optPlnSiz[3] = no2;
	  }
	  else if ( no1 == 16 )
	  {
	    conf->opt->optPlnSiz[4] = no2;
	  }
	  else
	  {
	    fprintf(stderr, "WARNING: expecting optplnSiz 01, optplnSiz 02, optplnSiz 04, optplnSiz 08 or optplnSiz 16 \n");
	  }
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("optPlnDim", str1 ) )
      {
	int no1;
	int no2;
	int read1 = sscanf(line, "%s %i %i", str1, &no1, &no2 );
	if ( read1 == 3 )
	{
	  if ( no1 >= 1 && no1 <= NO_OPT_LEVS )
	  {
	    conf->opt->optPlnDim[no1-1] = no2;
	  }
	  else
	  {
	    fprintf(stderr,"WARNING: Invalid optimisation plane number %i numbers should range between 1 and %i \n", no1, NO_OPT_LEVS);
	  }
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom(line, "FLAG_OPT_DYN_HW" ) )
      {
	singleFlag ( optFlags, str1, str2, FLAG_OPT_DYN_HW, "", "0", lineno, fName );
      }

      else if ( strCom("FLAG_DBG_SYNCH", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_SYNCH, "", "0", lineno, fName );
	singleFlag ( optFlags, str1, str2, FLAG_SYNCH, "", "0", lineno, fName );
      }

      else if ( strCom("FLAG_DBG_PROFILING", str1 ) )
      {
#ifdef PROFILING
	singleFlag ( genFlags, str1, str2, FLAG_PROF, "", "0", lineno, fName );
	singleFlag ( optFlags, str1, str2, FLAG_PROF, "", "0", lineno, fName );
#else
	fprintf(stderr, "ERROR: Found %s on line %i of %s, the program has not been compile with profiling enabled. Check the #define in cuda_accel.h.\n", str1, lineno, fName);
	exit(EXIT_FAILURE); // TMP REM - Added to mark an error for thesis timing
#endif
      }

      else if ( strCom("FLAG_DPG_PLT_OPT", str1 ) )
      {
	singleFlag ( optFlags, str1, str2, FLAG_DPG_PLT_OPT, "", "0", lineno, fName );
      }

      else if ( strCom("FLAG_DPG_PLT_POWERS", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_DPG_PLT_POWERS, "", "0", lineno, fName );
	singleFlag ( optFlags, str1, str2, FLAG_DPG_PLT_POWERS, "", "0", lineno, fName );
      }

      else if ( strCom("FLAG_DPG_UNOPT", str1 ) )
      {
	useUnopt    = 1;
      }

      else if ( strCom("FLAG_DBG_SKIP_OPT", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_DPG_SKP_OPT, "", "0", lineno, fName );
	singleFlag ( optFlags, str1, str2, FLAG_DPG_SKP_OPT, "", "0", lineno, fName );
      }

      else if ( strCom("FLAG_DPG_PRNT_CAND", str1 ) )
      {
	singleFlag ( genFlags, str1, str2, FLAG_DPG_PRNT_CAND, "", "0", lineno, fName );
	singleFlag ( optFlags, str1, str2, FLAG_DPG_PRNT_CAND, "", "0", lineno, fName );
      }

      else if ( strCom("DBG_LEV", str1 ) )
      {
	int no;
	int read1 = sscanf(str2, "%i", &no  );
	if ( read1 == 1 )
	{
	  msgLevel = no;
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", str1, lineno, fName);
	}
      }

      else if ( strCom("FLAG_DBG_TEST", str1 ) )
      {
	if      ( strCom(str2, "0") )
	{
	  (*genFlags) &= ~FLAG_DBG_TEST_ALL;
	}
	else if ( strCom(str2, "1") )
	{
	  (*genFlags) |= FLAG_DBG_TEST_1;
	}
	else if ( strCom(str2, "2") )
	{
	  (*genFlags) |= FLAG_DBG_TEST_2;
	}
	else if ( strCom(str2, "3") )
	{
	  (*genFlags) |= FLAG_DBG_TEST_3;
	}
	else
	{
	  fprintf(stderr, "ERROR: Found unknown value \"%s\" for flag \"%s\" on line %i of %s.\n", str2, str1, lineno, fName);
	}
      }

      else if ( strCom(line, "cuMedianBuffSz" ) )             // The size of the sub sections to use in the cuda median selection algorithm
      {
	rest = &line[ strlen("cuMedianBuffSz")+1];
	cuMedianBuffSz = atoi(rest);
      }

      else if ( strCom(line, "globalFloat01" ) )
      {
	rest = &line[ strlen("globalFloat01")+1];
	globalFloat01 = atof(rest);
      }
      else if ( strCom(line, "globalFloat02" ) )
      {
	rest = &line[ strlen("globalFloat02")+1];
	globalFloat02 = atof(rest);
      }
      else if ( strCom(line, "globalFloat03" ) )
      {
	rest = &line[ strlen("globalFloat03")+1];
	globalFloat03 = atof(rest);
      }
      else if ( strCom(line, "globalFloat04" ) )
      {
	rest = &line[ strlen("globalFloat04")+1];
	globalFloat04 = atof(rest);
      }
      else if ( strCom(line, "globalFloat05" ) )
      {
	rest = &line[ strlen("globalFloat05")+1];
	globalFloat05 = atof(rest);
      }

      else if ( strCom(line, "globalInt01" ) )
      {
	rest = &line[ strlen("globalInt01")+1];
	globalInt01 = atoi(rest);
      }
      else if ( strCom(line, "globalInt02" ) )
      {
	rest = &line[ strlen("globalInt02")+1];
	globalInt02 = atoi(rest);
      }
      else if ( strCom(line, "globalInt03" ) )
      {
	rest = &line[ strlen("globalInt03")+1];
	globalInt03 = atoi(rest);
      }
      else if ( strCom(line, "globalInt04" ) )
      {
	rest = &line[ strlen("globalInt04")+1];
	globalInt04 = atoi(rest);
      }
      else if ( strCom(line, "globalInt05" ) )
      {
	rest = &line[ strlen("globalInt05")+1];
	globalInt05 = atoi(rest);
      }

      else
      {
	line[flagLen] = 0;
	fprintf(stderr, "ERROR: Found unknown flag \"%s\" on line %i of %s.\n", line, lineno, fName);
	exit(EXIT_FAILURE); // TMP REM - Added to mark an error for thesis timing
      }
    }

    fclose (file);
  }
  else
  {
    printf("Unable to read GPU accel settings from %s\n", fName);
    exit(EXIT_FAILURE); // TMP REM - Added to mark an error for thesis timing
  }
}

searchSpecs* sSpecsFromObs(Cmdline *cmd, accelobs* obs, confSpecs* conf)
{
  searchSpecs* sSpec = new(searchSpecs);
  memset(sSpec, 0, sizeof(searchSpecs));

  sSpec->specRLow	= obs->rlo;
  sSpec->specRHigh	= obs->rhi;

  sSpec->searchRLow	= obs->rlo;
  sSpec->searchRHigh	= obs->rhi;

  sSpec->noHarmStages	= log2((float)cmd->numharm)+1;
  sSpec->noHarms	= cmd->numharm;
  sSpec->zMax		= cmd->zmax;
  sSpec->sigma		= cmd->sigma;

  conf->gen->planeWidth	= cmd->width;
  conf->gen->normType	= obs->norm_type;

  conf->gen->zMax	= cu_calc_required_z<double>(1, fabs(sSpec->zMax), conf->gen->zRes);

//  REM
//  if ( conf->gen->flags & (FLAG_SS_31 /*| FLAG_SS_20 | FLAG_SS_30 */ ) )
//  {
//    // Round the first bin to a multiple of the number of harmonics this is needed in the s&s kernel
//    sSpec->searchRLow	= floor(sSpec->searchRLow/(float)cmd->numharm)*cmd->numharm;
//  }

  return sSpec;
}

fftInfo* fftFromObs(accelobs* obs)
{
  fftInfo* fftInf = new(fftInfo);
  memset(fftInf, 0, sizeof(fftInfo));

  fftInf->data		= obs->fft;				// Pointer to first memory location of the FT values
  fftInf->T		= obs->T;				// Observation duration
  fftInf->dt		= obs->dt;				// Sampling frequency

  fftInf->firstBin	= 0;					// By default the start of the FT in memory
  fftInf->lastBin	= obs->numbins-1;			// The last bin in memory
  fftInf->noBins	= fftInf->lastBin - fftInf->firstBin + 1;// The number of bins read into memory

  return fftInf;
}

confSpecs* defaultConfig()
{
  confSpecs* conf = new(confSpecs);
  memset(conf, 0, sizeof(confSpecs));

  conf->gen = new(confSpecsGen);
  memset(conf->gen, 0, sizeof(confSpecsGen));

  conf->opt = new(confSpecsOpt);
  memset(conf->opt, 0, sizeof(confSpecsOpt));

  CUDA_SAFE_CALL(hipGetLastError(), "Entering readSrchSpecs.");

  FOLD // Defaults for accel search  .
  {
    conf->gen->flags	|= FLAG_KER_DOUBGEN ;	// Generate the kernels using double precision math (still stored as floats though)
    conf->gen->flags	|= FLAG_ITLV_ROW    ;
    conf->gen->flags	|= FLAG_CENTER      ;	// Centre and align the usable part of the planes
    conf->gen->flags	|= CU_FFT_SEP_INP   ;	// Input is small and separate FFT plans wont take up too much memory

#ifdef WITH_SAS_COUNT
    conf->gen->flags	|= FLAG_SS_COUNT    ;	// Enable counting results in sum & search kernels
#endif

    // NOTE: I found using the strait ring buffer memory is fastest - If the data is very noisy consider using FLAG_CAND_MEM_PRE
#ifndef DEBUG
    conf->gen->flags	|= FLAG_CAND_THREAD ;	// Multithreading really slows down debug so only turn it on by default for release mode, NOTE: This can be over ridden in the defaults file
    conf->opt->flags	|= FLAG_OPT_THREAD  ;	// Do CPU component of optimisation in a separate thread - A very good idea
#endif

#if CUDA_VERSION >= 6050
    conf->gen->flags	|= FLAG_CUFFT_CB_POW;	// CUFFT callback to calculate powers, very efficient so on by default
#endif

#if CUDA_VERSION >= 7050 && defined(WITH_HALF_RESCISION_POWERS)
    conf->gen->flags	|= FLAG_POW_HALF;
#endif

//    if ( obs->inmem )				// Use the command line to select in-mem search, NOTE: this is over ridden by what ever is in the DEFAULTS file (best to comment out this line then!)
//    {
//      sSpec.flags	|= FLAG_SS_INMEM;
//    }

    conf->gen->flags	|= FLAG_STAGES;

    conf->gen->cndType	|= CU_CANDFULL;  	// Candidate data type - CU_CANDFULL this should be the default as it has all the needed data
    conf->gen->cndType	|= CU_STR_ARR;  	// Candidate storage structure - CU_STR_ARR    is generally the fastest

    conf->gen->retType	|= CU_POWERZ_S;  	// Return type
    conf->gen->retType	|= CU_STR_ARR;  	// Candidate storage structure

    conf->gen->noResPerBin	= 2;		// Inter binning
    conf->gen->candRRes		= 0.5;		// 1 Candidate per 2 bins
    conf->gen->zRes		= 2;
    conf->gen->zMax		= 200;
    conf->gen->ringLength	= 7;		// Just a good number
    conf->gen->planeWidth	= 8;		// A good default for newer GPU's

    conf->gen->normType		= 0;
    conf->gen->inputNormzBound	= -1;		// Default to not uses, only used if specified in the defaults file
    conf->gen->inputFFFTzBound	= -1;		// Default to not uses, only used if specified in the defaults file

    conf->gen->ssStepSize	= 32768;	// TODO: Check this, to small may be inefficient too large can make the IM plane to large

    // Default: Auto chose best!
    conf->gen->mulSlices	= 0 ;
    conf->gen->mulChunk		= 0 ;
    conf->gen->ssSlices		= 0 ;
    conf->gen->ssChunk		= 0 ;

    conf->opt->zScale		= 4;
    conf->opt->optResolution	= 16;
    conf->opt->optPlnScale	= 10;
    conf->opt->optMinLocHarms	= 1;
    conf->opt->optMinRepHarms	= 1;


    conf->opt->flags		|= FLAG_OPT_NRM_MEDIAN1D;
    conf->opt->flags		|= FLAG_OPT_BLK_HRM;
    conf->opt->flags		|= FLAG_OPT_PTS_HRM;
    conf->opt->flags		|= FLAG_OPT_NM_REFINE;

    conf->opt->optPlnDim[0]	= 128;
    conf->opt->optPlnDim[1]	= 32;
    conf->opt->optPlnDim[2]	= 16;
    conf->opt->optPlnDim[3]	= 16;
    conf->opt->optPlnDim[4]	= 0;
    conf->opt->optPlnDim[5]	= 0;
    conf->opt->optPlnDim[6]	= 0;

    conf->opt->optPlnSiz[0]	= 16;
    conf->opt->optPlnSiz[1]	= 14;
    conf->opt->optPlnSiz[2]	= 12;
    conf->opt->optPlnSiz[3]	= 10;
    conf->opt->optPlnSiz[4]	= 8;
  }

  return conf;
}

confSpecs* getConfig()
{
  confSpecs* conf = defaultConfig();

  // Now read the
  readAccelDefalts(conf);

  return conf;
}

searchSpecs* getSpec(fftInfo* fft)
{
  searchSpecs* sSpec = new searchSpecs;
  memset(sSpec, 0, sizeof(searchSpecs));

  sSpec->noHarmStages	= 5;
  sSpec->noHarms 	= 16;
  sSpec->sigma		= 2;
  sSpec->zMax		= 200;

  if ( fft )
  {
    if ( fft->N && fft->dt )
    {
      sSpec->specRLow		= 1*fft->N*fft->dt;
      sSpec->specRHigh		= 10000.0*fft->N*fft->dt;
    }
    else
    {
      sSpec->specRLow		= fft->firstBin;
      sSpec->specRHigh		= fft->lastBin;
    }

    sSpec->searchRLow	= sSpec->specRLow;
    sSpec->searchRHigh	= sSpec->specRHigh;
  }

  return sSpec;
}

void intSrchThrd(cuSearch* srch)
{
  resThrds* tInf = srch->threasdInfo;

  if ( !tInf )
  {
    tInf     = new(resThrds);
    memset(tInf, 0, sizeof(resThrds));
  }

  if (pthread_mutex_init(&tInf->candAdd_mutex, NULL))
  {
    printf("Unable to initialise a mutex.\n");
    exit(EXIT_FAILURE);
  }

  if (sem_init(&tInf->running_threads, 0, 0))
  {
    printf("Could not initialise a semaphore\n");
    exit(EXIT_FAILURE);
  }

  srch->threasdInfo = tInf;
}

searchSpecs* duplicate(searchSpecs* sSpec)
{
  searchSpecs* dup = new searchSpecs;
  memcpy(dup,sSpec,sizeof(searchSpecs));
  return dup;
}

confSpecsGen* duplicate(confSpecsGen* conf)
{
  confSpecsGen* dup = new confSpecsGen;
  memcpy(dup, conf, sizeof(confSpecsGen));
  return dup;
}

confSpecsOpt* duplicate(confSpecsOpt* conf)
{
  confSpecsOpt* dup = new confSpecsOpt;
  memcpy(dup, conf, sizeof(confSpecsOpt));
  return dup;
}

confSpecs* duplicate(confSpecs* conf)
{
  confSpecs* dup = new confSpecs;
  dup->gen = duplicate(conf->gen);
  dup->opt = duplicate(conf->opt);
  return dup;
}

gpuSpecs* duplicate(gpuSpecs* gSpec)
{
  gpuSpecs* dup = new gpuSpecs;
  memcpy(dup, gSpec, sizeof(gpuSpecs));
  return dup;
}

bool compare(searchSpecs* sSpec1, searchSpecs* sSpec2)
{
  if ( sSpec1 == sSpec2 )
    return true;

  if ( sSpec1->noHarmStages != sSpec2->noHarmStages )
    return false;

  if ( sSpec1->noHarms != sSpec2->noHarms )
    return false;

  if ( sSpec1->searchRHigh != sSpec2->searchRHigh )
    return false;

  if ( sSpec1->searchRLow != sSpec2->searchRLow )
    return false;

  if ( sSpec1->sigma != sSpec2->sigma )
    return false;

  if ( sSpec1->zMax != sSpec2->zMax )
    return false;

  return true;
}

bool compare(confSpecsGen* conf1, confSpecsGen* conf2)
{
  if ( conf1 == conf2 )
    return true;

  if (conf1->candRRes != conf2->candRRes )
    return false;

  if (conf1->inputFFFTzBound != conf2->inputFFFTzBound )
    return false;

  if (conf1->inputNormzBound != conf2->inputNormzBound )
    return false;

  if (conf1->noResPerBin != conf2->noResPerBin )
    return false;

  if (conf1->planeWidth != conf2->planeWidth )
    return false;

  if (conf1->ssStepSize != conf2->ssStepSize )
    return false;

  if (conf1->zMax != conf2->zMax )
    return false;

  if (conf1->zRes != conf2->zRes )
    return false;

  if ( conf1->flags != conf2->flags )
  {
    if ( (conf1->flags & (FLAG_Z_SPLIT) ) != ( conf2->flags & (FLAG_Z_SPLIT) ) )
      return false;

    if ( (conf1->flags & (CU_NORM_GPU) ) != ( conf2->flags & (CU_NORM_GPU) ) )
      return false;

    int64_t mul1 = conf1->flags & (FLAG_MUL_ALL);
    int64_t mul2 = conf2->flags & (FLAG_MUL_ALL);

    if ( (conf1->flags & (FLAG_MUL_CB) ) != ( conf2->flags & (FLAG_MUL_CB) ) )
      return false;

    if ( (conf1->flags & (CU_FFT_SEP_ALL) ) != ( conf2->flags & (CU_FFT_SEP_ALL) ) )
      return false;

    if ( (conf1->flags & (FLAG_CUFFT_ALL) ) != ( conf2->flags & (FLAG_CUFFT_ALL) ) )
      return false;

    if ( (conf1->flags & (FLAG_POW_HALF) ) != ( conf2->flags & (FLAG_POW_HALF) ) )
      return false;

    if ( (conf1->flags & (FLAG_SS_ALL) ) != ( conf2->flags & (FLAG_SS_ALL) ) )
      return false;

    if ( (conf1->flags & (FLAG_SS_ALL) ) != ( conf2->flags & (FLAG_SS_ALL) ) )
      return false;
  }

  return true;
}

bool compare(confSpecsOpt* conf1, confSpecsOpt* conf2)
{
  if ( conf1 == conf2 )
    return true;

  if (conf1->optMinLocHarms != conf2->optMinLocHarms )
    return false;

  if (conf1->optMinRepHarms != conf2->optMinRepHarms )
    return false;

  if (conf1->optPlnScale != conf2->optPlnScale )
    return false;

  if (conf1->optResolution != conf2->optResolution )
    return false;

  if (conf1->zScale != conf2->zScale )
    return false;

  return true;
}

bool compare(fftInfo* fft1, fftInfo* fft2)
{
  if ( fft1 == fft2 )
    return true;

  if (fft1->N != fft2->N )
    return false;

  if (fft1->T != fft2->T )
    return false;

  if (fft1->dt != fft2->dt )
    return false;

  if (fft1->firstBin != fft2->firstBin )
    return false;

  if (fft1->lastBin != fft2->lastBin )
    return false;

  return true;
}

bool compare(gpuSpecs* gSpec1, gpuSpecs* gSpec2)
{
  if ( gSpec1 == gSpec2 )
    return true;

  if (gSpec1->noDevices != gSpec1->noDevices )
    return false;

  for ( int devNo = 0; devNo < gSpec1->noDevices; devNo++ )
  {
    if ( gSpec1->devId[devNo] != gSpec1->devId[devNo] )
      return false;

    if ( gSpec1->noDevBatches[devNo] != gSpec1->noDevBatches[devNo] )
      return false;

    if ( gSpec1->noDevOpt[devNo] != gSpec1->noDevOpt[devNo] )
      return false;

    if ( gSpec1->noDevSteps[devNo] != gSpec1->noDevSteps[devNo] )
      return false;
  }

  return true;
}

bool compare(cuSearch* search, searchSpecs* sSpec, confSpecs* conf, gpuSpecs* gSpec, fftInfo* fftInf)
{
  CUDA_SAFE_CALL(hipGetLastError(), "Entering initCuSearch.");

  if (!search)
    return false;

  if ( search->noGenHarms != sSpec->noHarms )
    return false;

  if ( search->noHarmStages != sSpec->noHarmStages )
    return false;

  if ( !compare(search->sSpec, sSpec) )
    return false;

  if ( !compare(search->conf->gen, conf->gen) )
    return false;

  if ( !compare(search->conf->opt, conf->opt) )
    return false;

  return true;
}

cuSearch* initSearchInf(searchSpecs* sSpec, confSpecs* conf, gpuSpecs* gSpec, fftInfo* fftInf)
{
  cuSearch* srch = new(cuSearch);
  memset(srch, 0, sizeof(cuSearch));

  srch->noHarmStages		= sSpec->noHarmStages;
  srch->noGenHarms		= ( 1<<(srch->noHarmStages-1) );
  srch->noSrchHarms		= ( 1<<(srch->noHarmStages-1) );

  srch->sIdx			= (int*)malloc(srch->noGenHarms * sizeof(int));
  srch->powerCut		= (float*)malloc(srch->noHarmStages * sizeof(float));
  srch->numindep		= (long long*)malloc(srch->noHarmStages * sizeof(long long));

  srch->sSpec			= sSpec;
  srch->conf			= conf;
  srch->gSpec			= gSpec;
  srch->fft			= fftInf;

  FOLD // Calculate power cutoff and number of independent values  .
  {
    infoMSG(3,2,"Calculate power cutoff and number of independent values\n");

    // Calculate appropriate z-max
    int numz = round(conf->gen->zMax / conf->gen->zRes) * 2 + 1;
    float adjust = 0;

    FOLD // Calculate power cutoff and number of independent values  .
    {
      for (int ii = 0; ii < srch->noHarmStages; ii++)
      {
      if ( numz == 1 )
      {
	srch->numindep[ii]	= (sSpec->searchRHigh - sSpec->searchRLow) / (double)(1<<ii) ;
      }
      else
      {
	srch->numindep[ii]	= (sSpec->searchRHigh - sSpec->searchRLow) * (numz + 1) * ( conf->gen->zRes / 6.95 ) / (double)(1<<ii);
      }

      // Power cutoff
      srch->powerCut[ii]	= power_for_sigma(sSpec->sigma, (1<<ii), srch->numindep[ii]);


      FOLD // Adjust for some lack in precision, if using half precision
      {
	if ( conf->gen->flags & FLAG_POW_HALF )
	{
	  float noP = log10( srch->powerCut[ii] );
	  float dp = pow(10, floor(noP)-4 );  		// "Last" significant value

	  adjust = -dp*(1<<ii);				// Subtract one significant "value" for each harmonic
	  srch->powerCut[ii] += adjust;
	}
      }

      infoMSG(6,6,"Stage %i numindep %12lli  threshold power %9.7f  adjusted %9.7f  \n", ii, srch->numindep[ii], srch->powerCut[ii], adjust);
      }
    }
  }

  FOLD // Set up the CPU threading  .
  {
    infoMSG(3,2,"Set up the CPU threading\n");

    intSrchThrd(srch);
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP();	// init Search inf
  }

  return srch;
}

cuSearch* initSearchInfCMD(Cmdline *cmd, accelobs* obs, gpuSpecs* gSpec)
{
  confSpecs*	conf	= getConfig();
  fftInfo*	fft	= fftFromObs(obs);
  searchSpecs*	sSpec	= sSpecsFromObs(cmd, obs, conf);

  return initSearchInf(sSpec, conf, gSpec, fft);
}

int remOptFlag(cuOptCand* pln, int64_t flag)
{
  if ( pln )
    pln->flags &= ~flag;
  else
  {
    fprintf(stderr, "ERROR: Null pointer");
    return ACC_ERR_NULL;
  }
  return ACC_ERR_NONE;
}

int setOptFlag(cuOptCand* pln, int64_t flag)
{
  if ( pln )
    pln->flags |=  flag;
  else
  {
    fprintf(stderr, "ERROR: Null pointer");
    return ACC_ERR_NULL;
  }
  return ACC_ERR_NONE;
}

int remOptFlag(cuOptInfo* oInf, int64_t flag)
{
  if ( !oInf )
  {
    fprintf(stderr, "ERROR: Null pointer");
    return ACC_ERR_NULL;
  }
  if ( !oInf->opts )
  {
    fprintf(stderr, "ERROR: Null pointer");
    return ACC_ERR_NULL;
  }

  int ret = ACC_ERR_NONE;
  for ( int i =0; i < oInf->noOpts; i++ )
  {
    ret |= remOptFlag(&oInf->opts[i], flag);
  }
  return ret;
}

int setOptFlag(cuOptInfo* oInf, int64_t flag)
{
  if ( !oInf )
  {
    fprintf(stderr, "ERROR: Null pointer");
    return ACC_ERR_NULL;
  }
  if ( !oInf->opts )
  {
    fprintf(stderr, "ERROR: Null pointer");
    return ACC_ERR_NULL;
  }

  int ret = ACC_ERR_NONE;
  for ( int i =0; i < oInf->noOpts; i++ )
  {
    ret |= setOptFlag(&oInf->opts[i], flag);
  }
  return ret;
}

int setOptFlag(cuSearch* cuSrch, int64_t flag)
{
  int ret = ACC_ERR_NONE;
  if ( cuSrch )
    ret |= setOptFlag(cuSrch->oInf, flag);
  else
  {
    fprintf(stderr, "ERROR: Null pointer");
    ret |= ACC_ERR_NULL;
  }
  return ret;
}

int remOptFlag(cuSearch* cuSrch, int64_t flag)
{
  int ret = ACC_ERR_NONE;
  if ( cuSrch )
    ret |= remOptFlag(cuSrch->oInf, flag);
  else
  {
    fprintf(stderr, "ERROR: Null pointer");
    ret |= ACC_ERR_NULL;
  }
  return ret;
}

void freeCuSearch(cuSearch* srch)
{
  if (srch)
  {
    if ( srch->pInf )
      freeCuAccel(srch->pInf);

    freeNull(srch->sIdx);
    freeNull(srch->powerCut);
    freeNull(srch->numindep);

    freeNull(srch)
  }
}

void printBitString( int64_t val )
{
  printf("Value %015ld : ", val );

  for ( int i = 0; i < 64; i++)
  {
    if( val & ( 1ULL << (63-i) ) )
      printf("1");
    else
      printf("0");
  }
  printf("\n");
}

void printCommandLine(int argc, char *argv[])
{
  printf("Command:\t");

  for ( int i =0; i < argc; i ++ )
  {
    printf("%s ",argv[i]);
  }
  printf("\n");
}

GSList* getCanidates(cuFFdotBatch* batch, GSList *cands )
{
  //  gridQuadTree<double, float>* qt = (gridQuadTree<double, float>*)(batch->h_candidates) ;
  //  quadNode<double, float>* head = qt->getHead();
  //
  //  qt->update();
  //
  //  printf("GPU search found %li unique values in tree.\n", head->noEls );

  return cands;
}

int hilClimb(candTree* tree, double tooclose = 5)
{
  container* cont = tree->getSmallest();
  //double tooclose = 5;

  while ( cont )
  {
    container* largest = tree->getLargest(cont, tooclose);
    if ( *largest > *cont )
    {
      tree->markForRemoval(cont);
    }
    cont = cont->larger;
  }

  uint rem = tree->removeMarked();
  printf("hilClimb  Removed %6i - %6i remain \n", rem, tree->noVals() );

  return rem;
}

int eliminate_harmonics(candTree* tree, double tooclose = 1.5)
{
  infoMSG(1,2,"Eliminate harmonics");

  int maxharm = 16;
  int numremoved = 0;

  initCand* tempCand = new(initCand);
  container* next;
  container* close;
  container* serch;

  container* lst = tree->getLargest();

  while ( lst )
  {
    initCand* candidate = (initCand*)lst->data;

    tempCand->power    = candidate->power;
    tempCand->numharm  = candidate->numharm;
    tempCand->r        = candidate->r;
    tempCand->z        = candidate->z;
    tempCand->sig      = candidate->sig;

    // Remove harmonics down
    for (double ii = 1; ii <= maxharm; ii++)
    {
      FOLD // Remove down candidates  .
      {
	tempCand->r  = candidate->r / ii;
	tempCand->z  = candidate->z / ii;
	serch       = contFromCand(tempCand);
	close       =  tree->getAll(serch, tooclose);

	while (close)
	{
	  next = close->smaller;

	  if ( *close != *lst )
	  {
	    tree->remove(close);
	    numremoved++;
	  }

	  close = next;
	}
      }

      FOLD // Remove down up  .
      {
	tempCand->r  = candidate->r * ii;
	tempCand->z  = candidate->z * ii;
	serch       = contFromCand(tempCand);
	close       =  tree->getAll(serch, tooclose/**sqrt(ii)*/);

	while (close)
	{
	  next = close->smaller;

	  if ( *close != *lst )
	  {
	    tree->remove(close);
	    numremoved++;
	  }

	  close = next;
	}
      }
    }

    for (int ii = 1; ii < 23; ii++)
    {
      tempCand->r  = candidate->r * ratioARR[ii];
      tempCand->z  = candidate->z * ratioARR[ii];
      serch       = contFromCand(tempCand);
      close       =  tree->getAll(serch, tooclose);

      while (close)
      {
	next = close->smaller;

	if ( *close != *lst )
	{
	  tree->remove(close);
	  numremoved++;
	}

	close = next;
      }
    }

    lst = lst->smaller;
  }

  printf("Harmonics Removed %6i - %6i remain \n", numremoved, tree->noVals() );

  return (numremoved);
}

/**  Wait for CPU threads to complete  .
 *
 */
int waitForThreads(sem_t* running_threads, const char* msg, int sleepMS )
{
  infoMSG(2,2,"Wait for CPU threads to complete\n");

  int noTrd;
  sem_getvalue(running_threads, &noTrd );

  if (noTrd)
  {
    char waitMsg[1024];
    int ite = 0;

    PROF // Profiling  .
    {
	NV_RANGE_PUSH("Wait on CPU threads");
    }

    while ( noTrd > 0 )
    {
      PROF // Profiling  .
      {
	NV_RANGE_PUSH("Sleep");
      }

      ite++;

      if ( noTrd >= 1 && !(ite % 10) )
      {
	sprintf(waitMsg,"%s  %3i thread still active.", msg, noTrd);

	FOLD  // Spinner  .
	{
	  if      (ite == 1 )
	    printf("\r%s⌜   ", waitMsg);
	  if      (ite == 2 )
	    printf("\r%s⌝   ", waitMsg);
	  if      (ite == 3 )
	    printf("\r%s⌟   ", waitMsg);
	  if      (ite == 4 )
	  {
	    printf("\r%s⌞   ", waitMsg);
	    ite = 0;
	  }
	  fflush(stdout);
	}
      }

      usleep(sleepMS);
      sem_getvalue(running_threads, &noTrd );

      PROF // Profiling  .
      {
	NV_RANGE_POP(); // Sleep
      }
    }

    if (ite >= 10 )
      printf("\n\n");

    PROF // Profiling  .
    {
      NV_RANGE_POP(); // Wait on CPU threads
    }

    return (ite);
  }

  return (0);
}

void* contextInitTrd(void* ptr)
{
  //long long* contextInit = (long long*)malloc(sizeof(long long));
  //*contextInit = 0;

  struct timeval start, end;
  gpuSpecs* gSpec = (gpuSpecs*)ptr;

  TIME // Start the timer  .
  {
    NV_RANGE_PUSH("Context");

    gettimeofday(&start, NULL);
  }

  initGPUs(gSpec);

  TIME // End the timer  .
  {
    NV_RANGE_POP(); // Context

    gettimeofday(&end, NULL);
    gSpec->nctxTime += (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec);
  }

  pthread_exit(&gSpec->nctxTime);

  return (NULL);
}

long long initCudaContext(gpuSpecs* gSpec)
{
  if (gSpec)
  {
    infoMSG(4, 4, "Creating context pthread for CUDA context initialisation.\n");

    int iret1 = 1;

#ifndef DEBUG
    iret1 = pthread_create( &gSpec->cntxThread, NULL, contextInitTrd, (void*) gSpec);
#endif

    if ( iret1 )
    {
      struct timeval start, end;

      fprintf(stderr,"ERROR: Failed to initialise context tread. pthread_create() return code: %d.\n", iret1);
      gSpec->cntxThread = 0;

      TIME // Start the timer  .
      {
	gettimeofday(&start, NULL);

	NV_RANGE_PUSH("Context");
      }

      printf("Initializing CUDA context's\n");
      initGPUs(gSpec);

      TIME // End the timer  .
      {
	NV_RANGE_POP(); // Context

	gettimeofday(&end, NULL);
	gSpec->nctxTime += (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec);
      }
    }
  }

  return 0;
}

long long compltCudaContext(gpuSpecs* gSpec)
{
  if ( gSpec)
  {
    if ( gSpec->cntxThread )
    {
      infoMSG(4, 4, "Wait on CUDA context thread\n");

      PROF // Profiling  .
      {
	NV_RANGE_PUSH("Wait on context thread");
      }

      printf("Waiting for CUDA context initialisation complete ...");
      fflush(stdout);

      void *status;
      struct timespec ts;
      if (clock_gettime(CLOCK_REALTIME, &ts) == -1)
      {
	fprintf(stderr,"ERROR: Failed to get time.\n");
      }
      ts.tv_sec += 10;

      int rr = pthread_timedjoin_np(gSpec->cntxThread, &status, &ts);
      if ( rr )
      {
	fprintf(stderr,"ERROR: Failed to join context thread.\n");
	if ( pthread_kill(gSpec->cntxThread, SIGALRM) )
	{
	  fprintf(stderr,"ERROR: Failed to kill context thread.\n");
	}

	for ( int i = 0; i < gSpec->noDevices; i++)
	{
	  CUDA_SAFE_CALL(hipSetDevice(gSpec->devId[i]), "ERROR in hipSetDevice");
	  CUDA_SAFE_CALL(hipDeviceReset(), "Error in device reset.");
	}

	exit(EXIT_FAILURE);
      }

      printf("\r                                                          ");
      fflush(stdout);

      gSpec->cntxThread = 0;

      infoMSG(4, 4, "Done\n");

      PROF // Profiling  .
      {
	NV_RANGE_POP(); // Wait on context thread
      }
    }

    return gSpec->nctxTime;
  }
  else
  {
    fprintf(stderr,"ERROR: Called %s with NULL pointer.\n", __FUNCTION__ );
  }

  return 0;
}
