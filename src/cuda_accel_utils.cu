#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <algorithm>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <nvToolsExt.h>
#include <nvToolsExtCudaRt.h>

extern "C"
{
#define __float128 long double
#include "accel.h"
}

#ifdef USEFFTW
#include <fftw3.h>
#endif

#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"
#include "cuda_accel_IN.h"

#ifdef CBL
#include <unistd.h>
#include "log.h"
#endif

__device__ __constant__ int           HEIGHT_HARM[MAX_HARM_NO];    ///< Plane  height  in stage order
__device__ __constant__ int           STRIDE_HARM[MAX_HARM_NO];    ///< Plane  stride  in stage order
__device__ __constant__ int           WIDTH_HARM[MAX_HARM_NO];     ///< Plane  strides   in family
__device__ __constant__ fcomplexcu*   KERNEL_HARM[MAX_HARM_NO];    ///< Kernel pointer in stage order
__device__ __constant__ stackInfo     STACKS[64];
__device__ __constant__ int           STK_STRD[4];                 ///< Stride of the stacks
__device__ __constant__ char          STK_INP[4][4069];            ///< input details


int    globalInt01    = 0;
int    globalInt02    = 0;
int    globalInt03    = 0;
int    globalInt04    = 0;
int    globalInt05    = 0;

float  globalFloat01  = 0;
float  globalFloat02  = 0;
float  globalFloat03  = 0;
float  globalFloat04  = 0;
float  globalFloat05  = 0;

int     useUnopt      = 0;
int     msgLevel      = 0;

double ratioARR[] = {
    3.0 / 2.0,
    5.0 / 2.0,
    2.0 / 3.0,
    4.0 / 3.0,
    5.0 / 3.0,
    3.0 / 4.0,
    5.0 / 4.0,
    2.0 / 5.0,
    3.0 / 5.0,
    4.0 / 5.0,
    5.0 / 6.0,
    2.0 / 7.0,
    3.0 / 7.0,
    4.0 / 7.0,
    3.0 / 8.0,
    5.0 / 8.0,
    2.0 / 9.0,
    3.0 / 10.0,
    2.0 / 11.0,
    3.0 / 11.0,
    2.0 / 13.0,
    3.0 / 13.0,
    2.0 / 15.0
};

__global__ void printfData(float* data, int nX, int nY, int stride, int sX = 0, int sY = 0)
{
  //printf("\n");
  for (int x = 0; x < nX; x++)
  {
    printf("---------");
  }
  printf("\n");
  for (int y = 0; y < nY; y++)
  {
    for (int x = 0; x < nX; x++)
    {
      printf("%8.4f ",data[ (y+sY)*stride + sX+ x ]);
    }
    printf("\n");
  }
  for (int x = 0; x < nX; x++)
  {
    printf("---------");
  }
  printf("\n");
}

/** Return the first value of 2^n >= x
 */
__host__ __device__ long long next2_to_n_cu(long long x)
{
  long long i = 1;

  while (i < x)
    i <<= 1;

  return i;
}

void setActiveBatch(cuFFdotBatch* batch, int rIdx)
{
  batch->rActive = rIdx;
}

float half2float(const ushort h)
{
  unsigned int sign     = ((h >> 15) & 1);
  unsigned int exponent = ((h >> 10) & 0x1f);
  unsigned int mantissa = ((h & 0x3ff) << 13);

  if (exponent == 0x1f)     // NaN or Inf
  {
    mantissa = (mantissa ? (sign = 0, 0x7fffff) : 0);
    exponent = 0xff;
  }
  else if (!exponent)       // Denorm or Zero
  {
    if (mantissa)
    {
      unsigned int msb;
      exponent = 0x71;
      do
      {
        msb = (mantissa & 0x400000);
        mantissa <<= 1;  /* normalize */
        --exponent;
      }
      while (!msb);

      mantissa &= 0x7fffff;  /* 1.mantissa is implicit */
    }
  }
  else
  {
    exponent += 0x70;
  }

  uint res = ((sign << 31) | (exponent << 23) | mantissa);
  return  *((float*)(&res));
}

/* The fft length needed to properly process a subharmonic */
int calc_fftlen3(double harm_fract, int max_zfull, uint accelLen, presto_interp_acc accuracy)
{
  int bins_needed, end_effects;

  bins_needed = accelLen * harm_fract + 2;
  end_effects = 2 * ACCEL_NUMBETWEEN * z_resp_halfwidth(calc_required_z(harm_fract, max_zfull), accuracy);
  return next2_to_n_cu(bins_needed + end_effects);
}

/** Calculate an optimal accellen given a width  .
 *
 * @param width the width of the plane usually a power of two
 * @param zmax
 * @return
 * If width is not a power of two it will be rounded up to the nearest power of two
 */
uint optAccellen(float width, int zmax, presto_interp_acc accuracy)
{
  float halfwidth       = z_resp_halfwidth(zmax, accuracy); /// The halfwidth of the maximum zmax, to calculate accel len
  float pow2            = pow(2 , round(log2(width)) );
  uint oAccelLen        = floor(pow2 - 2 - 2 * ACCEL_NUMBETWEEN * halfwidth);

  return oAccelLen;
}

/** Calculate the step size from a width if the width is < 100 it is skate to be the closest power of two  .
 *
 * @param width
 * @param zmax
 * @return
 */
uint calcAccellen(float width, float zmax, presto_interp_acc accuracy)
{
  int accelLen;

  if ( width > 100 )
  {
    accelLen = width;
  }
  else
  {
    accelLen = optAccellen(width*1000.0, zmax, accuracy) ;
  }
  return accelLen;
}

/** Allocate R value array
 *
 */
void createRvals(cuFFdotBatch* batch, rVals** rLev1, rVals**** rAraays )
{
  rVals**   rLev2;

  int oSet                = 0;


  (*rLev1)                = (rVals*)malloc(sizeof(rVals)*batch->noSteps*batch->noGenHarms*batch->noRArryas);
  memset((*rLev1), 0, sizeof(rVals)*batch->noSteps*batch->noGenHarms*batch->noRArryas);
  for (int i1 = 0 ; i1 < batch->noSteps*batch->noGenHarms*batch->noRArryas; i1++)
  {
    (*rLev1)[i1].step     = -1; // Invalid step (0 is a valid value!)
  }

  *rAraays                = (rVals***)malloc(batch->noRArryas*sizeof(rVals**));

  for (int rIdx = 0; rIdx < batch->noRArryas; rIdx++)
  {
    rLev2                 = (rVals**)malloc(sizeof(rVals*)*batch->noSteps);
    (*rAraays)[rIdx]      = rLev2;

    for (int step = 0; step < batch->noSteps; step++)
    {
      rLev2[step]         = &((*rLev1)[oSet]);
      oSet               += batch->noGenHarms;
    }
  }
}

void freeRvals(cuFFdotBatch* batch, rVals** rLev1, rVals**** rAraays )
{
  if (*rAraays)
  {
    for (int rIdx = 0; rIdx < batch->noRArryas; rIdx++)
    {
      rVals**   rLev2;

      rLev2 = (*rAraays)[rIdx];

      freeNull(rLev2);
    }

    freeNull(*rAraays);
  }

  freeNull(*rLev1);
}

void createFFTPlans(cuFFdotBatch* kernel)
{
  char msg[1024];

  nvtxRangePush("FFT plans");

  // Note creating the plans is the most expensive task in the GPU init, I tried doing it in parallel but it was slower
  for (int i = 0; i < kernel->noStacks; i++)
  {
    cuFfdotStack* cStack  = &kernel->stacks[i];

    FOLD //  .
    {
      sprintf(msg,"Stack %i",i);
      nvtxRangePush(msg);

      int n[]             = {cStack->width};

      int inembed[]       = {cStack->strideCmplx * sizeof(fcomplexcu)};         /// Storage dimensions of the input data in memory
      int istride         = 1;                                                  /// The distance between two successive input elements in the least significant (i.e., innermost) dimension
      int idist           = cStack->strideCmplx;                                /// The distance between the first element of two consecutive signals in a batch of the input data

      int onembed[]       = {cStack->strideCmplx * sizeof(fcomplexcu)};
      int ostride         = 1;
      int odist           = cStack->strideCmplx;

      FOLD // Create the input FFT plan  .
      {
        if ( kernel->flags & CU_INPT_FFT_CPU )
        {
          nvtxRangePush("FFTW");
          cStack->inpPlanFFTW = fftwf_plan_many_dft(1, n, cStack->noInStack*kernel->noSteps, (fftwf_complex*)cStack->h_iData, n, istride, idist, (fftwf_complex*)cStack->h_iData, n, ostride, odist, -1, FFTW_ESTIMATE);
          nvtxRangePop();
        }
        else
        {
          nvtxRangePush("CUFFT Inp");
          CUFFT_SAFE_CALL(hipfftPlanMany(&cStack->inpPlan,  1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, cStack->noInStack*kernel->noSteps), "Creating plan for input data of stack.");
          nvtxRangePop();
        }
      }

      FOLD // Create the stack iFFT plan  .
      {
        nvtxRangePush("CUFFT Pln");
        CUFFT_SAFE_CALL(hipfftPlanMany(&cStack->plnPlan,  1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, cStack->height*kernel->noSteps), "Creating plan for complex data of stack.");
        nvtxRangePop();
      }

      nvtxRangePop();
    }

    CUDA_SAFE_CALL(hipGetLastError(), "Creating FFT plans for the stacks.");
  }

  nvtxRangePop();
}

/** Initialise a kernel data structure and values on a given device  .
 *
 * First Initialise kernel data structure (this is just a batch)
 *
 * Next create kernel values
 * If master is NULL this is the first device so calculate the actual kernel values
 * If master != NULL copy the kernel values from the master on another device
 *
 * @param kernel
 * @param master
 * @param numharmstages
 * @param zmax
 * @param fftinf
 * @param device
 * @param noBatches
 * @param noSteps
 * @param width
 * @param powcut
 * @param numindep
 * @param flags
 * @param outType
 * @param outData
 * @return
 */
int initKernel(cuFFdotBatch* kernel, cuFFdotBatch* master, cuSearch*   sInf, int devID )
{
  std::cout.flush();

  size_t free, total;                           ///< GPU memory
  int noInStack[MAX_HARM_NO];

  //int noSrchHarms     = noGenHarms;

  noInStack[0]        = 0;
  size_t batchSize    = 0;                      ///< Total size (in bytes) of all the data need by a family (ie one step) excluding FFT temporary
  size_t fffTotSize   = 0;                      ///< Total size (in bytes) of FFT temporary memory
  size_t planeSize    = 0;                      ///< Total size (in bytes) of memory required independently of batch(es)
  float plnElsSZ      = 0;                      ///< The size of an element of the in-mem ff plane (generally the size of float complex)
  float powElsSZ      = 0;                      ///< The size of an element of the powers plane

  gpuInf* gInf        = &sInf->gSpec->devInfo[devID];
  int device          = gInf->devid;
  int noBatches       = sInf->gSpec->noDevBatches[devID];
  int noSteps         = sInf->gSpec->noDevSteps[devID];
  int alignment       = gInf->alignment;

  presto_interp_acc  accuracy = LOWACC;

  CUDA_SAFE_CALL(hipGetLastError(), "Entering initKernel.");

  infoMSG(3,3,"%s device %i\n",__FUNCTION__, device);

  char msg[1024];
  sprintf(msg, "Dev %02i", device );
  nvtxRangePush(msg);

  FOLD // See if we can use the cuda device and whether it may be possible to do GPU in-mem search .
  {
    infoMSG(3,4,"access device %i\n", device);

    nvtxRangePush("Get Device");

    if ( device >= getGPUCount() )
    {
      fprintf(stderr, "ERROR: There is no CUDA device %i.\n", device);
      return (0);
    }
    int currentDevvice;
    CUDA_SAFE_CALL(hipSetDevice(device), "Failed to set device using hipSetDevice");
    CUDA_SAFE_CALL(hipGetDevice(&currentDevvice), "Failed to get device using hipGetDevice");
    if (currentDevvice != device)
    {
      fprintf(stderr, "ERROR: CUDA Device not set.\n");
      return (0);
    }
    else
    {
      CUDA_SAFE_CALL(hipMemGetInfo ( &free, &total ), "Getting Device memory information");
    }

    nvtxRangePop();
  }

  FOLD // Now see if this device could do a GPU in-mem search  .
  {
    if ( master == NULL ) // For the moment lets try this on only the first card!
    {
      infoMSG(3,4,"in-mem?\n");

      int noarms        = (1 << (sInf->noHarmStages - 1) );

      double plnX       = ( sInf->sSpec->fftInf.rhi - sInf->sSpec->fftInf.rlo/(double)noarms ) / (double)( ACCEL_DR ) ; // The number of bins
      int    plnY       = calc_required_z(1.0, (float)sInf->sSpec->zMax );

      if ( sInf->sSpec->flags & FLAG_HALF )
      {
#if CUDA_VERSION >= 7050
        plnElsSZ = sizeof(half);
#else
        plnElsSZ = sizeof(float);
        fprintf(stderr, "WARNING: Half precision can only be used with CUDA 7.5 or later! Reverting to single precision!\n");
        sInf->sSpec->flags &= ~FLAG_HALF;
#endif
      }
      else
      {
        plnElsSZ = sizeof(float);
      }

      if ( sInf->sSpec->flags & FLAG_KER_HIGH )
        accuracy = HIGHACC;

      // Calculate "approximate" plane width
      uint accelLen     = calcAccellen(sInf->sSpec->pWidth, sInf->sSpec->zMax, accuracy );
      float fftLen      = calc_fftlen3(1, sInf->sSpec->zMax, accelLen, accuracy );

      double totalSize  = plnX * plnY * plnElsSZ ;
      double appRoxWrk  = plnY * fftLen * ( 4 * 3 + 1) ; // 4 planes * ( input + CUFFT )

      if ( totalSize + appRoxWrk < free )
      {
        if ( !(sInf->sSpec->flags & FLAG_SS_ALL) || (sInf->sSpec->flags & FLAG_SS_INMEM) )
        {
          printf("Device %i can do a in-mem GPU search.\n", device);
          printf("  There is %.2fGB free memory.\n  The entire f-∂f plane requires %.2f GB and the workspace ~%.2f MB.\n\n", free*1e-9, totalSize*1e-9, appRoxWrk*1e-6 );
        }

        if ( (sInf->sSpec->flags & FLAG_SS_ALL) && !(sInf->sSpec->flags & FLAG_SS_INMEM) )
        {
          fprintf(stderr,"WARNING: Opting to NOT do a in-mem search when you could!\n");
        }
        else
        {
          sInf->noGenHarms        = 1;

          if ( sInf->gSpec->noDevices > 1 )
          {
            fprintf(stderr,"  Warning: Reverting to single device search.\n");
            sInf->gSpec->noDevices = 1;
          }

          sInf->sSpec->flags |= FLAG_SS_INMEM ;

#if CUDA_VERSION >= 6050
          if ( !(sInf->sSpec->flags & FLAG_CUFFT_CB_POW) )
            fprintf(stderr,"  Warning: Doing an in-mem search with no CUFFT callbacks, this is not ideal.\n"); // It should be on by default the user must have disabled it
#else
          fprintf(stderr,"  Warning: Doing an in-mem search with no CUFFT callbacks, this is not ideal. Try upgrading to CUDA 6.5 or later.\n");
          sInf->sSpec->flags &= ~FLAG_CUFFT_ALL;
#endif

#if CUDA_VERSION >= 7050
          if ( !(sInf->sSpec->flags & FLAG_HALF) )
            fprintf(stderr,"  Warning: You could be using half precision.\n"); // They should be on by default the user must have disabled them
#else
          fprintf(stderr,"  Warning: You could be using half precision. Try upgrading to CUDA 7.5 or later.\n");
#endif

          FOLD // Set types  .
          {
            sInf->sSpec->retType &= ~CU_TYPE_ALLL;
            sInf->sSpec->retType |= CU_POWERZ_S;

            sInf->sSpec->retType &= ~CU_SRT_ALL;
            sInf->sSpec->retType |= CU_STR_ARR;
          }
        }
      }
      else
      {
        if ( !(sInf->sSpec->flags & FLAG_SS_ALL) || (sInf->sSpec->flags & FLAG_SS_INMEM) )
        {
          printf("Device %i can not do a in-mem GPU search.\n", device);
          printf("  There is %.2fGB free memory.\n  The entire f-∂f plane requires %.2f GB and the workspace ~%.2f MB.\n\n", free*1e-9, totalSize*1e-9, appRoxWrk*1e-6 );
        }

        if ( sInf->sSpec->flags & FLAG_SS_INMEM  )
        {
          fprintf(stderr,"ERROR: Requested an in-memory GPU search, this is not possible\n\tThere is %.2f GB of free memory.\n\tIn-mem GPU search would require ~%.2f GB\n\n", free*1e-9, (totalSize + appRoxWrk)*1e-9 );
        }
        sInf->sSpec->flags &= ~FLAG_SS_INMEM ;
      }

      if ( !(sInf->sSpec->flags & FLAG_SS_ALL) )
      {
        // Default to S&S 1.
        sInf->sSpec->flags |= FLAG_SS_10;
        sInf->sSpec->flags |= FLAG_RET_STAGES;
      }
    }
  }

  FOLD // Do a sanity check on Flags and CUDA version  .
  {
    // TODO: do a check whether there is enough precision in an int to store the index of the largest point

    if ( master == NULL ) // For the moment lets try this on only the first card!
    {
      infoMSG(3,4,"FLAGS\n");

      // CUFFT callbacks
#if CUDA_VERSION < 6050
      sInf->sSpec->flags &= ~FLAG_CUFFT_ALL;
#endif

      if ( (sInf->sSpec->flags & FLAG_HALF) && !(sInf->sSpec->flags & FLAG_SS_INMEM) && !(sInf->sSpec->flags & FLAG_CUFFT_CB_POW) )
      {
#if CUDA_VERSION >= 7050
        fprintf(stderr, "WARNING: Can't use half precision with out of memory search and no CUFFT callbacks. Reverting to single precision!\n");
#endif
        sInf->sSpec->flags &= ~FLAG_HALF;
      }

      if ( !(sInf->sSpec->flags & FLAG_SS_INMEM) && (sInf->sSpec->flags & FLAG_CUFFT_CB_INMEM) )
      {
        fprintf(stderr, "WARNING: Can't use inmem callback with out of memory search. Disabling in-mem callback.\n");
        sInf->sSpec->flags &= ~FLAG_CUFFT_CB_INMEM;
      }

      if ( (sInf->sSpec->flags & FLAG_CUFFT_CB_POW) && (sInf->sSpec->flags & FLAG_CUFFT_CB_INMEM) )
      {
        fprintf(stderr, "WARNING: in-mem CUFFT callback will supersede power callback, I have found power callbacks to be the best.\n");
        sInf->sSpec->flags &= ~FLAG_CUFFT_CB_POW;
      }

      if ( (sInf->sSpec->flags & FLAG_SS_10) || (sInf->sSpec->flags & FLAG_SS_INMEM) )
      {
        sInf->sSpec->flags |= FLAG_RET_STAGES;
      }

      char typeS[1024];
      sprintf(typeS, "Doing");

      if ( sInf->sSpec->flags & FLAG_SS_INMEM )
        sprintf(typeS, "%s a in-memory", typeS);
      else
        sprintf(typeS, "%s an out of memory", typeS);

      sprintf(typeS, "%s search using", typeS);
      if ( sInf->sSpec->flags & FLAG_HALF )
        sprintf(typeS, "%s half", typeS);
      else
        sprintf(typeS, "%s single", typeS);

      sprintf(typeS, "%s precision", typeS);
      if ( sInf->sSpec->flags & FLAG_CUFFT_CB_POW )
        sprintf(typeS, "%s and CUFFT callbacks to calculate powers.", typeS);
      else if ( sInf->sSpec->flags & FLAG_CUFFT_CB_INMEM )
        sprintf(typeS, "%s and CUFFT callbacks to calculate powers and store in the full plane.", typeS);
      else
        sprintf(typeS, "%s and no CUFFT callbacks.", typeS);

      printf("\n%s\n\n", typeS);
    }

    FOLD // Determine the size of the elements of the planes  .
    {
      // Half precision?
      if ( sInf->sSpec->flags & FLAG_HALF )
      {
#if CUDA_VERSION >= 7050
        plnElsSZ = sizeof(half);
#else
        plnElsSZ = sizeof(float);
        fprintf(stderr, "WARNING: Half precision can only be used with CUDA 7.5 or later! Reverting to single precision!\n");
        sInf->sSpec->flags &= ~FLAG_HALF;
#endif
      }
      else
      {
        plnElsSZ = sizeof(float);
      }

      // Set power plane size
      if ( sInf->sSpec->flags & FLAG_CUFFT_CB_POW )
      {
        powElsSZ = plnElsSZ;
      }
      else
      {
        powElsSZ = sizeof(fcomplexcu);
      }
    }
  }

  FOLD // Allocate and zero some structures  .
  {
    infoMSG(3,4,"Allocate and zero structures\n");

    FOLD // Initialise main pointer to this kernel  .
    {
      memset(kernel, 0, sizeof(cuFFdotBatch));

      if ( master != NULL )  // Copy all pointers and sizes from master. All non global pointers must be overwritten.
      {
        memcpy(kernel,  master,  sizeof(cuFFdotBatch));
        kernel->srchMaster  = 0;
      }
      else
      {
        kernel->flags         = sInf->sSpec->flags;
        kernel->srchMaster    = 1;
        kernel->noHarmStages  = sInf->noHarmStages;
        kernel->noGenHarms    = sInf->noGenHarms;
        kernel->noSrchHarms   = sInf->noSrchHarms;
      }
    }

    FOLD // Set the device specific parameters  .
    {
      kernel->sInf          = sInf;
      kernel->device        = device;
      kernel->isKernel      = 1;                // This is the device master
      kernel->capability    = gInf->capability;
    }

    FOLD // Allocate memory  .
    {
      kernel->hInfos        = (cuHarmInfo*) malloc(kernel->noSrchHarms * sizeof(cuHarmInfo));
      kernel->kernels       = (cuKernel*)   malloc(kernel->noGenHarms * sizeof(cuKernel));

      // Zero memory for kernels and harmonics
      memset(kernel->hInfos,  0, kernel->noSrchHarms * sizeof(cuHarmInfo));
      memset(kernel->kernels, 0, kernel->noGenHarms  * sizeof(cuKernel));
    }
  }

  FOLD // Determine how many stacks and how many planes in each stack  .
  {
    if ( master == NULL ) 	// Calculate details for the batch  .
    {
      infoMSG(3,4,"Determine number of stacks and planes\n");

      FOLD // Determine accellen and step size  .
      {
        infoMSG(3,5,"Determining step size and width\n");

        printf("Determining GPU step size and plane width:\n");

        if ( kernel->noSrchHarms > 1 )
        {
          // Working with a family of planes

          int   oAccelLen1, oAccelLen2;

          // This adjustment makes sure no more than half the harmonics are in the largest stack (reduce waisted work - gives a 0.01 - 0.12 speed increase )
          oAccelLen1  = calcAccellen(sInf->sSpec->pWidth,     sInf->sSpec->zMax, accuracy);
          oAccelLen2  = calcAccellen(sInf->sSpec->pWidth/2.0, sInf->sSpec->zMax/2.0, accuracy);

          if ( sInf->sSpec->pWidth > 100 )
          {
            // The user specified the exact width they want to use for accellen
            kernel->accelLen  = oAccelLen1;
          }
          else
          {
            // Use double the accellen of the half plane
            kernel->accelLen  = MIN(oAccelLen2*2, oAccelLen1);
          }

          if ( sInf->sSpec->pWidth < 100 ) // Check  .
          {
            float fWidth    = floor(calc_fftlen3(1, sInf->sSpec->zMax, kernel->accelLen, accuracy)/1000.0);

            float ss        = calc_fftlen3(1, sInf->sSpec->zMax, kernel->accelLen, accuracy) ;
            float l2        = log2( ss );

            if      ( l2 == 10 )
              fWidth = 1 ;
            else if ( l2 == 11 )
              fWidth = 2 ;
            else if ( l2 == 12 )
              fWidth = 4 ;
            else if ( l2 == 13 )
              fWidth = 8 ;
            else if ( l2 == 14 )
              fWidth = 16 ;
            else if ( l2 == 15 )
              fWidth = 32 ;
            else if ( l2 == 16 )
              fWidth = 64 ;

            if ( fWidth != sInf->sSpec->pWidth )
            {
              fprintf(stderr,"ERROR: Width calculation did not give the desired value.\n");
              exit(EXIT_FAILURE);
            }
          }
        }
        else
        {
          // Just a single plane
          kernel->accelLen = calcAccellen(sInf->sSpec->pWidth, sInf->sSpec->zMax, accuracy);
        }

        FOLD // Now make sure that accelLen is divisible by (noSrchHarms*ACCEL_RDR) this "rule" is used for indexing in the sum and search kernel
        {
          kernel->accelLen = floor( kernel->accelLen/(float)(kernel->noSrchHarms*ACCEL_RDR) ) * (kernel->noSrchHarms*ACCEL_RDR);

          if ( sInf->sSpec->pWidth > 100 ) // Check  .
          {
            if ( sInf->sSpec->pWidth != kernel->accelLen )
            {
              fprintf(stderr,"ERROR: Using manual step size, value must be divisible by numharm * %i (%i) try %i.\n", ACCEL_RDR, kernel->noSrchHarms*ACCEL_RDR, kernel->accelLen );
              exit(EXIT_FAILURE);
            }
          }
        }

        FOLD // Print kernel accuracy  .
        {
          printf(" • Using ");

          if ( sInf->sSpec->flags & FLAG_KER_HIGH )
          {
            printf("high ");
          }
          else
          {
            printf("standard ");
          }
          printf("accuracy response functions.\n");

          if ( sInf->sSpec->flags & FLAG_KER_MAX )
            printf(" • Using maximum response function length for entire kernel.\n");
        }

        if ( kernel->accelLen > 100 ) // Print output
        {
          float fftLen      = calc_fftlen3(1, sInf->sSpec->zMax, kernel->accelLen, accuracy);
          int   oAccelLen   = optAccellen(fftLen, sInf->sSpec->zMax, accuracy);
          float ratio       = kernel->accelLen/float(oAccelLen);

          printf(" • Using max plane width of %.0f and thus", fftLen);

          if    	( ratio < 0.90 )
          {
            printf(" an non-optimal step-size of %i.\n", kernel->accelLen );
            if ( sInf->sSpec->pWidth > 100 )
            {
              int K              = round(fftLen/1000.0);
              fprintf(stderr,"    WARNING: Using manual width\\step-size is not advised rather set width to one of 2 4 8 46 32.\n    For a zmax of %i using %iK FFTs the optimal step-size is %i.\n", sInf->sSpec->zMax, K, oAccelLen);
            }
          }
          else if ( ratio < 0.95 )
          {
            printf(" an close to optimal step-size of %i.\n", kernel->accelLen );
          }
          else
          {
            printf(" an optimal step-size of %i.\n", kernel->accelLen );
          }
        }
        else
        {
          fprintf(stderr,"ERROR: With a width of %i, the step-size would be %i and this is too small, try with a wider width or lower z-max.\n", sInf->sSpec->pWidth, kernel->accelLen);
          exit(EXIT_FAILURE);
        }
      }

      FOLD // Set some harmonic related values  .
      {
        int prevWidth       = 0;
        int noStacks        = 0;
        int stackHW         = 0;
        int hIdx, sIdx;
        float hFrac;

        FOLD // Set up basic details of all the harmonics  .
        {
        for (int i = kernel->noSrchHarms; i > 0; i--)
        {
          cuHarmInfo* hInfs;
          hFrac               = (i) / (float)kernel->noSrchHarms;
          hIdx                = kernel->noSrchHarms-i;
          hInfs               = &kernel->hInfos[hIdx];                              // Harmonic index

          hInfs->harmFrac     = hFrac;
          hInfs->zmax         = calc_required_z(hInfs->harmFrac, sInf->sSpec->zMax);
          hInfs->height       = (hInfs->zmax / ACCEL_DZ) * 2 + 1;
          hInfs->width        = calc_fftlen3(hInfs->harmFrac, kernel->hInfos[0].zmax, kernel->accelLen, accuracy);
          hInfs->halfWidth    = z_resp_halfwidth(hInfs->zmax, accuracy);

          if ( prevWidth != hInfs->width )
          {
            // We have a new stack
            noStacks++;

            if ( hIdx < kernel->noGenHarms )
            {
              kernel->noStacks = noStacks;
            }

            noInStack[noStacks - 1]       = 0;
            prevWidth                     = hInfs->width;
            stackHW                       = z_resp_halfwidth(hInfs->zmax, accuracy);

            // Maximise, centre and align halfwidth
            int   sWidth                  = (int) ( ceil(kernel->accelLen * hInfs->harmFrac * ACCEL_DR ) * ACCEL_RDR + DBLCORRECT ) + 1 ;     // Width of usable data for this plane
            float centHW                  = (hInfs->width  - sWidth)/2.0/(float)ACCEL_NUMBETWEEN;                                             //
            float noAlg                   = alignment / float(sizeof(fcomplex)) / (float)ACCEL_NUMBETWEEN ;                                   // halfWidth will be multiplied by ACCEL_NUMBETWEEN so can divide by it here!
            float centAlgnHW              = floor(centHW/noAlg) * noAlg ;                                                                     // Centre and aligned half width

            if ( stackHW > centAlgnHW )
            {
              stackHW                     = floor(centHW);
            }
            else
            {
              stackHW                     = centAlgnHW;
            }
          }

          hInfs->stackNo      = noStacks-1;

          if ( kernel->flags & FLAG_CENTER )
          {
            hInfs->kerStart   = stackHW*ACCEL_NUMBETWEEN;
          }
          else
          {
            hInfs->kerStart   = hInfs->halfWidth*ACCEL_NUMBETWEEN;
          }

          if ( hIdx < kernel->noGenHarms )
          {
            noInStack[noStacks - 1]++;
          }
        }
        }

        FOLD // Set up the indexing details of all the harmonics  .
        {
          // Calculate the stage order of the harmonics
          sIdx = 0;

          for ( int stage = 0; stage < kernel->noHarmStages; stage++ )
          {
            int harmtosum = 1 << stage;
            for (int harm = 1; harm <= harmtosum; harm += 2, sIdx++)
            {
              hFrac     = harm/float(harmtosum);
              hIdx      = hFrac == 1 ? 0 : round(hFrac*kernel->noSrchHarms);



              kernel->hInfos[hIdx].stageIndex   = sIdx;
              sInf->sIdx[sIdx]                  = hIdx;
            }
          }
        }
      }
    }
    else                    // Copy details from the master batch  .
    {
      // Copy memory from kernels and harmonics
      memcpy(kernel->hInfos,  master->hInfos,  kernel->noSrchHarms * sizeof(cuHarmInfo));
      memcpy(kernel->kernels, master->kernels, kernel->noGenHarms  * sizeof(cuKernel));
    }
  }

  FOLD // Allocate all the memory for the stack data structures  .
  {
    infoMSG(3,4,"Allocate memory for stacks\n");

    long long neede = kernel->noStacks * sizeof(cuFfdotStack) + kernel->noSrchHarms * sizeof(cuHarmInfo) + kernel->noGenHarms * sizeof(cuKernel);

    if ( neede > getFreeRamCU() )
    {
      fprintf(stderr, "ERROR: Not enough host memory for search.\n");
    }
    else
    {
      // Set up stacks
      kernel->stacks = (cuFfdotStack*) malloc(kernel->noStacks* sizeof(cuFfdotStack));

      if ( master == NULL )
      {
        memset(kernel->stacks, 0, kernel->noStacks * sizeof(cuFfdotStack));
      }
      else
      {
        memcpy(kernel->stacks, master->stacks, kernel->noStacks * sizeof(cuFfdotStack));

        FOLD // Zero some of the relevant values in the stack
        {
          for (int i = 0; i < kernel->noStacks; i++)           // Loop through Stacks  .
          {
            cuFfdotStack* cStack  = &kernel->stacks[i];

            cStack->plnPlan       = 0;
            cStack->inpPlan       = 0;
          }
        }
      }
    }
  }

  FOLD // Set up the basic details of all the stacks and calculate the stride  .
  {
    FOLD // Set up the basic details of all the stacks  .
    {
      if ( master == NULL )
      {
        infoMSG(3,4,"Stack details\n");

        int prev                = 0;
        for (int i = 0; i < kernel->noStacks; i++)           // Loop through Stacks  .
        {
          cuFfdotStack* cStack  = &kernel->stacks[i];
          cStack->height        = 0;
          cStack->noInStack     = noInStack[i];
          cStack->startIdx      = prev;
          cStack->harmInf       = &kernel->hInfos[cStack->startIdx];
          cStack->kernels       = &kernel->kernels[cStack->startIdx];
          cStack->width         = cStack->harmInf->width;
          cStack->kerHeigth     = cStack->harmInf->height;
          cStack->flags         = kernel->flags;               // Used to create the kernel, will be over written later

          for (int j = 0; j < cStack->noInStack; j++)
          {
            cStack->startZ[j]   = cStack->height;
            cStack->height     += cStack->harmInf[j].height;
          }

          prev                 += cStack->noInStack;
        }
      }
    }

    FOLD // Calculate the stride and data thus data size of the stacks  .
    {
      // This is device specific so done on each card

      infoMSG(3,4,"Stride details\n");

      kernel->inpDataSize     = 0;
      kernel->kerDataSize     = 0;
      kernel->plnDataSize     = 0;
      kernel->pwrDataSize     = 0;

      for (int i = 0; i < kernel->noStacks; i++)          // Loop through Stacks  .
      {
        cuFfdotStack* cStack  = &kernel->stacks[i];

        FOLD // Compute size of
        {
          // Compute stride  .
          cStack->strideCmplx =   getStrie(cStack->width, sizeof(hipfftComplex), alignment);
          cStack->stridePower =   getStrie(cStack->width, powElsSZ,             alignment);

          kernel->inpDataSize +=  cStack->strideCmplx * cStack->noInStack * sizeof(hipfftComplex);
          kernel->kerDataSize +=  cStack->strideCmplx * cStack->kerHeigth * sizeof(hipfftComplex);
          kernel->plnDataSize +=  cStack->strideCmplx * cStack->height    * sizeof(hipfftComplex);

          if ( !(kernel->flags & FLAG_CUFFT_CB_INMEM) )
            kernel->pwrDataSize +=  cStack->stridePower * cStack->height    * powElsSZ;
        }
      }
    }
  }

  FOLD // Batch specific streams  .
  {
    nvtxRangePush("streams");

    infoMSG(3,4,"Batch streams\n");

    char strBuff[1024];

    if ( kernel->flags & FLAG_SYNCH )
    {
      cuFfdotStack* fStack = &kernel->stacks[0];

      CUDA_SAFE_CALL(hipStreamCreate(&fStack->initStream),"Creating CUDA stream for initialisation");

      sprintf(strBuff,"%i.0.0.0 Initialisation", device );
      nvtxNameCudaStreamA(fStack->initStream, strBuff);
      //printf("hipStreamCreate: %s\n", strBuff);

      for (int i = 0; i < kernel->noStacks; i++)
      {
        cuFfdotStack* cStack = &kernel->stacks[i];

        cStack->initStream = fStack->initStream;
      }
    }
    else
    {
      for (int i = 0; i < kernel->noStacks; i++)
      {
        cuFfdotStack* cStack = &kernel->stacks[i];

        CUDA_SAFE_CALL(hipStreamCreate(&cStack->initStream),"Creating CUDA stream for initialisation");

        sprintf(strBuff,"%i.0.0.%i Initialisation", device, i);
        nvtxNameCudaStreamA(cStack->initStream, strBuff);
        //printf("hipStreamCreate: %s\n", strBuff);
      }
    }

    if ( !(kernel->flags & CU_FFT_SEP) )
    {
      if ( !(kernel->flags & CU_INPT_FFT_CPU) )
      {
        for (int i = 0; i < kernel->noStacks; i++)
        {
          cuFfdotStack* cStack = &kernel->stacks[i];

          CUDA_SAFE_CALL(hipStreamCreate(&cStack->fftIStream),"Creating CUDA stream for fft's");
          sprintf(strBuff,"%i.0.2.%i FFT Input Dev", device, i);
          nvtxNameCudaStreamA(cStack->fftIStream, strBuff);
          //printf("hipStreamCreate: %s\n", strBuff);
        }
      }

      for (int i = 0; i < kernel->noStacks; i++)
      {
        cuFfdotStack* cStack = &kernel->stacks[i];

        CUDA_SAFE_CALL(hipStreamCreate(&cStack->fftPStream),"Creating CUDA stream for fft's");
        sprintf(strBuff,"%i.0.4.%i FFT Plane Dev", device, i);
        nvtxNameCudaStreamA(cStack->fftPStream, strBuff);
        //printf("hipStreamCreate: %s\n", strBuff);
      }
    }

    nvtxRangePop();
  }

  FOLD // Allocate device memory for all the kernels data  .
  {
    nvtxRangePush("kernel malloc");

    infoMSG(3,4,"Allocate device memory for all the kernels data\n");

    if ( kernel->kerDataSize > free )
    {
      fprintf(stderr, "ERROR: Not enough device memory for GPU multiplication kernels. There is only %.2f MB free and you need %.2f MB \n", free / 1048576.0, kernel->kerDataSize / 1048576.0 );
      freeKernel(kernel);
      return (0);
    }
    else
    {
      CUDA_SAFE_CALL(hipMalloc((void**)&kernel->d_kerData, kernel->kerDataSize), "Failed to allocate device memory for kernel stack.");
      CUDA_SAFE_CALL(hipGetLastError(), "Allocation of device memory for kernel?.\n");
    }

    nvtxRangePop();
  }

  FOLD // Set the sizes values of the harmonics and kernels and pointers to kernel data  .
  {
    infoMSG(3,4,"Set the sizes values of the harmonics and kernels and pointers to kernel data\n");

    size_t kerSiz = 0;

    for (int i = 0; i < kernel->noStacks; i++)
    {
      cuFfdotStack* cStack            = &kernel->stacks[i];
      cStack->d_kerData               = &kernel->d_kerData[kerSiz];

      // Set the stride
      for (int j = 0; j< cStack->noInStack; j++)
      {
        // Point the plane kernel data to the correct position in the "main" kernel
        int iDiff                     = cStack->kerHeigth - cStack->harmInf[j].height ;
        float fDiff                   = iDiff / 2.0;
        cStack->kernels[j].d_kerData  = &cStack->d_kerData[cStack->strideCmplx*(int)fDiff];
        cStack->kernels[j].harmInf    = &cStack->harmInf[j];
      }
      kerSiz                          += cStack->strideCmplx * cStack->kerHeigth;
    }

  }

  FOLD // Initialise the multiplication kernels  .
  {
    if ( master == NULL )     // Create the kernels  .
    {
      infoMSG(3,4,"Initialise the multiplication kernels\n");

      // Run message
      CUDA_SAFE_CALL(hipGetLastError(), "Before creating GPU kernels");

      FOLD // Check contamination of the largest stack  .
      {
        float contamination = (kernel->hInfos->halfWidth*2*ACCEL_NUMBETWEEN)/(float)kernel->hInfos->width*100 ;
        if ( contamination > 25 )
        {
          fprintf(stderr, "WARNING: Contamination is high, consider increasing width with the -width flag.\n");
        }
      }

      printf("\nGenerating GPU multiplication kernels using device %i\n", device);

      FOLD // Calculate the response values  .
      {
        infoMSG(3,5,"Calculate the response values\n");

        nvtxRangePush("Calc response");

        int hh      = 1;
        for (int i = 0; i < kernel->noStacks; i++)
        {
          cuFfdotStack* cStack = &kernel->stacks[i];

          float contamination = (cStack->harmInf->halfWidth*2*ACCEL_NUMBETWEEN)/(float)cStack->harmInf->width*100 ;
          float padding       = (1-(kernel->accelLen*cStack->harmInf->harmFrac + cStack->harmInf->halfWidth*2*ACCEL_NUMBETWEEN ) / cStack->harmInf->width)*100.0 ;

          printf("  ■ Stack %i has %02i f-∂f plane(s). width: %5li  stride: %5li  Height: %6li  Memory size: %7.1f MB \n", i+1, cStack->noInStack, cStack->width, cStack->strideCmplx, cStack->height, cStack->height*cStack->strideCmplx*sizeof(fcomplex)/1024.0/1024.0);

          // Call the CUDA kernels
          // Only need one kernel per stack
          createStackKernel(cStack);

          printf("    ► Created kernel %i  Size: %7.1f MB  Height %4i   Contamination: %5.2f %%  Padding: %5.2f %%\n", i+1, cStack->harmInf->height*cStack->strideCmplx*sizeof(fcomplex)/1024.0/1024.0, cStack->harmInf->zmax, contamination, padding);

          for (int j = 0; j < cStack->noInStack; j++)
          {
            printf("      • Harmonic %02i  Fraction: %5.3f   Z-Max: %4i   Half Width: %4i  Start offset: %4i \n", hh, cStack->harmInf[j].harmFrac, cStack->harmInf[j].zmax, cStack->harmInf[j].halfWidth, cStack->harmInf[j].kerStart / ACCEL_NUMBETWEEN  );
            hh++;
          }
        }

        nvtxRangePop();
      }

      FOLD // FFT the kernels  .
      {
        infoMSG(3,5,"FFT the  response values\n");

        nvtxRangePush("FFT kernels");

        fflush(stdout);
        printf("  FFT'ing the kernels ");
        fflush(stdout);

        for (int i = 0; i < kernel->noStacks; i++)
        {
          infoMSG(4,6,"Stack %i\n",i);

          cuFfdotStack* cStack = &kernel->stacks[i];

          FOLD // Create the plan  .
          {
            infoMSG(4,6,"Create plan\n");

            sprintf(msg,"Plan %i",i);
            nvtxRangePush(msg);

            int n[]             = {cStack->width};
            int inembed[]       = {cStack->strideCmplx* sizeof(fcomplexcu)};
            int istride         = 1;
            int idist           = cStack->strideCmplx;
            int onembed[]       = {cStack->strideCmplx* sizeof(fcomplexcu)};
            int ostride         = 1;
            int odist           = cStack->strideCmplx;
            int height          = cStack->kerHeigth;

            // Normal plans
            CUFFT_SAFE_CALL(hipfftPlanMany(&cStack->plnPlan,  1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, height), "Creating plan for FFT'ing the kernel.");
            CUDA_SAFE_CALL(hipGetLastError(), "Creating FFT plans for the stacks.");

            nvtxRangePop();
          }

          FOLD // Call the plan  .
          {
            infoMSG(4,6,"Call the plan\n");

            sprintf(msg,"Call %i",i);
            nvtxRangePush(msg);

            CUFFT_SAFE_CALL(hipfftSetStream(cStack->plnPlan, cStack->initStream),  "Error associating a CUFFT plan with multStream.");
            CUFFT_SAFE_CALL(hipfftExecC2C(cStack->plnPlan, (hipfftComplex *) cStack->d_kerData, (hipfftComplex *) cStack->d_kerData, HIPFFT_FORWARD), "FFT'ing the kernel data. [hipfftExecC2C]");
            CUDA_SAFE_CALL(hipGetLastError(), "FFT'ing the multiplication kernels.");

            nvtxRangePop();
          }

          FOLD // Destroy the plan  .
          {
            infoMSG(4,6,"Destroy the plan\n");

            sprintf(msg,"Dest %i",i);
            nvtxRangePush(msg);

            CUFFT_SAFE_CALL(hipfftDestroy(cStack->plnPlan), "Destroying plan for complex data of stack. [hipfftDestroy]");
            CUDA_SAFE_CALL(hipGetLastError(), "Destroying the plan.");

            nvtxRangePop();
          }

          printf("•");
          fflush(stdout);
        }

        CUDA_SAFE_CALL(hipGetLastError(), "FFT'ing the multiplication kernels.");

        printf("\n");

        nvtxRangePop();
      }

      printf("Done generating GPU multiplication kernels\n");
    }
    else
    {
      infoMSG(3,4,"Copy multiplication kernels\n");

      // TODO: Check this works in this location

      printf("• Copying multiplication kernels from device %i.\n", master->device);
      //CUDA_SAFE_CALL(hipMemcpyPeer(kernel->d_kerData, kernel->device, master->d_kerData, master->device, master->kerDataSize ), "Copying multiplication kernels between devices.");
      CUDA_SAFE_CALL(hipMemcpyPeerAsync(kernel->d_kerData, kernel->device, master->d_kerData, master->device, master->kerDataSize, master->stacks->initStream ), "Copying multiplication kernels between devices.");
    }
  }

  FOLD // Decide how to handle input and output and allocate required memory  .
  {
    infoMSG(3,4,"Input and output.\n");

    nvtxRangePush("data");

    printf("\nInitializing GPU %i (%s)\n", device, sInf->gSpec->devInfo[devID].name );

    printf("• Examining GPU memory of device %2i:\n", kernel->device);

    ulong freeRam;          /// The amount if free host memory
    int retSZ     = 0;      /// The size in byte of the returned data
    int candSZ    = 0;      /// The size in byte of the candidates
    int retY      = 0;      /// The number of candidates return per family (one step)
    ulong hostC   = 0;      /// The size in bytes of device memory used for candidates

    FOLD // Calculate the search size in bins  .
    {
      if ( master == NULL )
      {
        int minR              = floor ( sInf->sSpec->fftInf.rlo /(double) kernel->noSrchHarms - kernel->hInfos->halfWidth );
        int maxR              = ceil  ( sInf->sSpec->fftInf.rhi  + kernel->hInfos->halfWidth );

        searchScale* SrchSz   = new searchScale;
        sInf->SrchSz          = SrchSz;
        memset(SrchSz, 0, sizeof(searchScale));

        SrchSz->searchRLow    = sInf->sSpec->fftInf.rlo / (double)kernel->noSrchHarms;
        SrchSz->searchRHigh   = sInf->sSpec->fftInf.rhi;
        SrchSz->rLow          = minR;
        SrchSz->rHigh         = maxR;
        SrchSz->noInpR        = maxR - minR  ;  /// The number of input data points
        SrchSz->noSteps       = ( sInf->sSpec->fftInf.rhi - sInf->sSpec->fftInf.rlo ) / (float)( kernel->accelLen * ACCEL_DR ) ; // The number of planes to make

        if ( kernel->flags & FLAG_SS_INMEM   )
        {
          SrchSz->noSteps     = ( SrchSz->searchRHigh - SrchSz->searchRLow ) / (float)( kernel->accelLen * ACCEL_DR ) ; // The number of planes to make
        }

        if ( kernel->flags  & FLAG_STORE_EXP )
        {
          SrchSz->noOutpR     = ceil( (SrchSz->searchRHigh - SrchSz->searchRLow)/ACCEL_DR );
        }
        else
        {
          SrchSz->noOutpR     = ceil(SrchSz->searchRHigh - SrchSz->searchRLow);
        }

        if ( (kernel->flags & FLAG_STORE_ALL) && !( kernel->flags  & FLAG_RET_STAGES) )
        {
          printf("   Storing all results implies returning all results so adding FLAG_RET_STAGES to flags!\n");
          kernel->flags  |= FLAG_RET_STAGES;
        }
      }
    }

    FOLD // Chunks and Slices  .
    {
      FOLD // Multiplication defaults are set per batch  .
      {
        kernel->mulSlices         = sInf->sSpec->mulSlices;
        kernel->mulChunk          = sInf->sSpec->mulChunk;

        FOLD // Set stack multiplication slices
        {
          for (int i = 0; i < kernel->noStacks; i++)
          {
            cuFfdotStack* cStack  = &kernel->stacks[i];
            cStack->mulSlices     = sInf->sSpec->mulSlices;
            cStack->mulChunk      = sInf->sSpec->mulChunk;
          }
        }
      }

      FOLD // Sum  & search  .
      {
        kernel->ssChunk           = sInf->sSpec->ssChunk;
        kernel->ssSlices          = sInf->sSpec->ssSlices;

        if ( kernel->ssSlices <= 0 )
        {
          if      ( kernel->stacks->width <= 1024 )
          {
            kernel->ssSlices      = 8 ; // Default value
          }
          else if ( kernel->stacks->width <= 2048 )
          {
            kernel->ssSlices      = 4 ; // Default value
          }
          else if ( kernel->stacks->width <= 4096 )
          {
            kernel->ssSlices      = 2 ;
          }
          else
          {
            kernel->ssSlices      = 1 ;
          }

        }
        kernel->ssSlices          = MIN(kernel->ssSlices, ceil(kernel->hInfos->height/20.0) );
      }
    }

    FOLD // Calculate candidate type  .
    {
      if ( master == NULL )   // There is only one list of candidates per search so only do this once!
      {
        kernel->cndType         = sInf->sSpec->cndType;

        if      ( !(kernel->cndType & CU_TYPE_ALLL) )
        {
          fprintf(stderr,"Warning: No candidate data type specified in %s. Setting to default.\n",__FUNCTION__);
          kernel->cndType = CU_CANDFULL;
        }

        if      (kernel->cndType & CU_CMPLXF   )
        {
          candSZ = sizeof(fcomplexcu);
        }
        else if (kernel->cndType & CU_INT      )
        {
          candSZ = sizeof(int);
        }
        else if (kernel->cndType & CU_FLOAT    )
        {
          candSZ = sizeof(float);
        }
        else if (kernel->cndType & CU_POWERZ_S )
        {
          candSZ = sizeof(candPZs);
        }
        else if (kernel->cndType & CU_POWERZ_I )
        {
          candSZ = sizeof(candPZi);
        }
        else if (kernel->cndType & CU_CANDMIN  )
        {
          candSZ = sizeof(candMin);
        }
        else if (kernel->cndType & CU_CANDSMAL )
        {
          candSZ = sizeof(candSml);
        }
        else if (kernel->cndType & CU_CANDBASC )
        {
          candSZ = sizeof(accelcandBasic);
        }
        else if (kernel->cndType & CU_CANDFULL )  // This should be the default
        {
          candSZ = sizeof(cand);
        }
        else
        {
          fprintf(stderr,"ERROR: No output type specified in %s setting to default.\n", __FUNCTION__);
          kernel->cndType |= CU_CANDFULL;
          candSZ = sizeof(cand);
        }

        if      ( !(kernel->cndType & CU_SRT_ALL   ) ) // Set defaults  .
        {
          fprintf(stderr,"Warning: No candidate storage type specified in %s. Setting to default.\n",__FUNCTION__);
          kernel->cndType = CU_STR_ARR   ;
        }
      }
    }

    FOLD // Calculate return type, size and data structure  .
    {
      kernel->retType       = sInf->sSpec->retType;

      if      (kernel->retType & CU_STR_PLN   )
      {
        if (  (kernel->flags & FLAG_CUFFT_CB_POW) && ( !( (kernel->retType & CU_HALF) || (kernel->retType & CU_FLOAT)))   )
        {
          fprintf(stderr,"WARNING: Returning plane and CUFFT output requires float return type.\n");
          kernel->retType &= ~CU_TYPE_ALLL;
          kernel->retType |= CU_FLOAT;
        }

        if ( !(kernel->flags & FLAG_CUFFT_CB_POW) && !(kernel->retType & CU_CMPLXF) )
        {
          fprintf(stderr,"WARNING: Returning plane requires complex float return type.\n");
          kernel->retType &= ~CU_TYPE_ALLL;
          kernel->retType |= CU_CMPLXF;
        }

        if ( kernel->flags & FLAG_SIG_GPU )
        {
          fprintf(stderr,"WARNING: Cannot do GPU sigma calculations when returning plane data.\n");
          kernel->flags &= ~FLAG_SIG_GPU;
        }
      }

      if      (kernel->retType & CU_CMPLXF    )
      {
        retSZ = sizeof(fcomplexcu);
      }
      else if (kernel->retType & CU_INT       )
      {
        retSZ = sizeof(int);
      }
      else if (kernel->retType & CU_HALF      )
      {
#if CUDA_VERSION >= 7050
        retSZ = sizeof(half);
#else
        fprintf(stderr,"ERROR: Half precision can only be used with CUDA 7.5 or later!\n");
        exit(EXIT_FAILURE);
#endif
      }
      else if (kernel->retType & CU_FLOAT     )
      {
        retSZ = sizeof(float);
      }
      else if (kernel->retType & CU_DOUBLE    )
      {
        retSZ = sizeof(double);
      }
      else if (kernel->retType & CU_POWERZ_S  )
      {
        retSZ = sizeof(candPZs);
      }
      else if (kernel->retType & CU_POWERZ_I  )
      {
        retSZ = sizeof(candPZi);
      }
      else if (kernel->retType & CU_CANDMIN   )
      {
        retSZ = sizeof(candMin);
      }
      else if (kernel->retType & CU_CANDSMAL  )
      {
        retSZ = sizeof(candSml);
      }
      else if (kernel->retType & CU_CANDBASC  )
      {
        retSZ = sizeof(accelcandBasic);
      }
      else if (kernel->retType & CU_CANDFULL  )
      {
        retSZ = sizeof(cand);
      }
      else
      {
        fprintf(stderr,"ERROR: No output type specified in %s\n",__FUNCTION__);
        kernel->retType &= ~CU_TYPE_ALLL ;
        kernel->retType |=  CU_POWERZ_S ;
        retSZ = sizeof(candPZs);
      }

      FOLD // Sum and search slices  .
      {
        if      ( kernel->retType & CU_STR_PLN )
        {
          // Each stage returns a plane the size of the fundamental
          retY = kernel->hInfos->height;
        }
        else
        {
          retY = kernel->ssSlices;
        }
      }

      FOLD // Return data structure  .
      {
        if      ( kernel->flags & FLAG_SS_INMEM )
        {
          kernel->strideOut = sInf->sSpec->ssStepSize;
        }
        else if ( (kernel->retType & CU_STR_ARR) )
        {
          //kernel->strideOut = kernel->hInfos->width;  // NOTE: This could be accellen rather than width, but to allow greater flexibility keep it at width. CU_STR_PLN    requires width
          kernel->strideOut = getStrie(kernel->accelLen, retSZ, alignment);
        }
        else if (  kernel->retType & CU_STR_PLN  )
        {
          if      ( kernel->retType & CU_FLOAT  )
          {
            kernel->strideOut = kernel->stacks->stridePower ;
          }
          else if ( kernel->retType & CU_HALF   )
          {
            kernel->strideOut = kernel->stacks->stridePower ;
          }
          else if ( kernel->retType & CU_CMPLXF )
          {
            kernel->strideOut = kernel->stacks->strideCmplx ;
          }
          else
          {
            fprintf(stderr,"ERROR: CUDA return type not compatible with returning plane.\n");
            exit(EXIT_FAILURE);
          }
        }
        else
        {
          fprintf(stderr,"ERROR: CUDA return structure not specified.\n");
          exit(EXIT_FAILURE);
        }
      }

      // Calculate return data size for one step
      kernel->retDataSize   = retY*kernel->strideOut*retSZ;

      if ( kernel->flags & FLAG_RET_STAGES )
        kernel->retDataSize *= kernel->noHarmStages;
    }

    FOLD // Calculate batch size and number of steps and batches on this device  .
    {
      nvtxRangePush("Calc steps");

      CUDA_SAFE_CALL(hipMemGetInfo ( &free, &total ), "Getting Device memory information"); // TODO: This call may not be necessary we could calculate this from previous values
      freeRam = getFreeRamCU();

      printf("   There is a total of %.2f GiB of device memory of which there is %.2f GiB free and %.2f GiB free host memory.\n",total / 1073741824.0, (free )  / 1073741824.0, freeRam / 1073741824.0 );

      FOLD // Calculate size of various memory's'  .
      {
        batchSize             = kernel->inpDataSize + kernel->plnDataSize + kernel->pwrDataSize + kernel->retDataSize;  // This is currently the size of one step
        fffTotSize            = kernel->inpDataSize + kernel->plnDataSize;                                              // FFT data treated separately because there will be only one set per device

        if ( kernel->flags & FLAG_SS_INMEM  ) // Size of memory for plane full ff plane  .
        {
          uint noStepsP       =  ceil(sInf->SrchSz->noSteps / (float)noSteps) * noSteps;
          uint nX             = noStepsP * kernel->accelLen;
          uint nY             = kernel->hInfos->height;
          planeSize          += nX * nY * plnElsSZ ;
        }
      }

      FOLD // Calculate how many batches and steps to do  .
      {
        float possSteps;
        char hipfftType[1024];

        if ( kernel->flags & CU_FFT_SEP )
        {
          possSteps = ( free - planeSize ) / (double) ( (fffTotSize + batchSize) * noBatches ) ;
        }
        else
        {
          possSteps = ( free - planeSize ) / (double) (  fffTotSize + batchSize  * noBatches ) ;  // (fffTotSize * possSteps) for the CUFFT memory for FFT'ing the plane(s) and (totSize * noThreads * possSteps) for each thread(s) plan(s)
        }

        printf("     Requested %i batches on this device.\n", noBatches);
        if ( possSteps > 1 )
        {
          if ( noSteps > floor(possSteps) )
          {
            printf("      Requested %i steps per batch, but with %i batches we can only do %.2f steps per batch. \n", noSteps, noBatches, possSteps );
            noSteps = floor(possSteps);
          }

          if ( floor(possSteps) > noSteps + 1 && (noSteps < MAX_STEPS) )
          {
            printf("       Note: requested %i steps per batch, you could do up to %.2f steps per batch. \n", noSteps, possSteps );
          }

          kernel->noSteps = noSteps;

          if ( kernel->noSteps > MAX_STEPS )
          {
            kernel->noSteps = MAX_STEPS;
            printf("      Trying to use more steps that the maximum number (%i) this code is compiled with.\n", kernel->noSteps );
          }
        }
        else
        {
          printf("      There is not enough memory to crate %i batches with one plane each.\n", noBatches);

          float noSteps1    = ( free ) / (double) ( fffTotSize + batchSize ) ;
          noSteps           = MIN(MAX_STEPS, floor(noSteps1));
          kernel->noSteps   = noSteps;
          noBatches         = 1;

          printf("        Throttling to %i steps in 1 batch.\n", kernel->noSteps);
        }

        if ( noBatches <= 0 || kernel->noSteps <= 0 )
        {
          fprintf(stderr, "ERROR: Insufficient memory to make make any planes on this device. One step would require %.2fGiB of device memory.\n", ( fffTotSize + batchSize )/1073741824.0 );

          // TODO: check flags here!

          freeKernel(kernel);
          return (0);
        }

        if ( kernel->flags & CU_FFT_SEP )
        {
          // one CUFFT plan per batch
          fffTotSize *= noBatches;
          sprintf(hipfftType, "( separate plans for each batch )");
        }
        else
        {
          sprintf(hipfftType, "( single plan for all batches )");
        }

        float  totUsed = ( kernel->kerDataSize + planeSize + ( fffTotSize + batchSize * noBatches ) * kernel->noSteps ) ;

        printf("     Processing %i steps with each of the %i batch(s)\n", kernel->noSteps, noBatches );

        printf("    -----------------------------------------------\n" );
        printf("    Kernels        use: %5.2f GiB of device memory.\n", (kernel->kerDataSize) / 1073741824.0 );
        printf("    CUFFT         uses: %5.2f GiB of device memory, %s\n", (fffTotSize*kernel->noSteps) / 1073741824.0, hipfftType );
        if ( planeSize )
        {
          printf("    In-mem plane  uses: %5.2f GiB of device memory.", (planeSize) / 1073741824.0 );

          if ( kernel->flags & FLAG_HALF )
          {
            printf(" (using half precision)\n");
          }
          else
          {
            printf("\n");
          }
        }
        printf("    Each batch    uses: %5.2f GiB of device memory.\n", (batchSize*kernel->noSteps) / 1073741824.0 );
        printf("                 Using: %5.2f GiB of %.2f [%.2f%%] of GPU memory for search.\n",  totUsed / 1073741824.0, total / 1073741824.0, totUsed / (float)total * 100.0f );
      }

      nvtxRangePop();
    }

    FOLD // Scale data sizes by number of steps  .
    {
      kernel->inpDataSize *= kernel->noSteps;
      kernel->plnDataSize *= kernel->noSteps;
      kernel->pwrDataSize *= kernel->noSteps;
      if ( !(kernel->flags & FLAG_SS_INMEM)  )
        kernel->retDataSize *= kernel->noSteps;       // In-mem search stage does not use steps
    }

    float fullCSize     = sInf->SrchSz->noOutpR * candSZ;               /// The full size of all candidate data

    if ( kernel->flags  & FLAG_STORE_ALL )
      fullCSize *= kernel->noHarmStages; // Store  candidates for all stages

    FOLD // DO a sanity check on flags  .
    {
      FOLD // How to handle input  .
      {
        if ( (kernel->flags & CU_INPT_FFT_CPU) && !(kernel->flags & CU_NORM_CPU) )
        {
          fprintf(stderr, "WARNING: Using CPU FFT of the input data necessitate doing the normalisation on CPU.\n");
          kernel->flags |= CU_NORM_CPU;
        }
      }

      FOLD // Set the stack flags  .
      {
        for (int i = 0; i < kernel->noStacks; i++)
        {
          cuFfdotStack* cStack  = &kernel->stacks[i];
          cStack->flags         = kernel->flags;
        }
      }
    }

    FOLD // Batch independent device memory  .
    {
      if ( kernel->flags & FLAG_SS_INMEM  )
      {
        nvtxRangePush("in-mem alloc");

        uint    noStepsP =  ceil(sInf->SrchSz->noSteps / (float)kernel->noSteps) * kernel->noSteps ;
        uint    nX       = noStepsP * kernel->accelLen;
        uint    nY       = kernel->hInfos->height;
        size_t  stride;

        CUDA_SAFE_CALL(hipMallocPitch(&sInf->d_planeFull,    &stride, plnElsSZ*nX, nY),   "Failed to allocate device memory for getMemAlignment.");
        CUDA_SAFE_CALL(hipMemsetAsync(sInf->d_planeFull, 0, stride*nY, kernel->stacks->initStream),"Failed to initiate plane memory to zero");

        sInf->inmemStride = stride / plnElsSZ;

        nvtxRangePop();
      }

    }

    FOLD // Allocate global (device independent) host memory  .
    {
      // One set of global set of "candidates" for all devices
      if ( master == NULL )
      {
        nvtxRangePush("host alloc");

        if      ( kernel->cndType & CU_STR_ARR  )
        {
          if ( sInf->sSpec->outData == NULL   )
          {
            // Have to allocate the array!

            freeRam  = getFreeRamCU();
            if ( fullCSize < freeRam*0.90 )
            {
              // Same host candidates for all devices
              // This can use a lot of memory for long searches!
              sInf->h_candidates = malloc( fullCSize );
              memset(sInf->h_candidates, 0, fullCSize );
              hostC += fullCSize;
            }
            else
            {
              fprintf(stderr, "ERROR: Not enough host memory for candidate list array. Need %.2fGiB there is %.2fGiB.\n", fullCSize / 1073741824.0, freeRam / 1073741824.0 );
              fprintf(stderr, "       Try set -fhi to a lower value. ie: numharm*1000. ( or buy more RAM, or close Chrome ;)\n");
              fprintf(stderr, "       Will continue trying to use a dynamic list.\n");

              kernel->cndType &= ~CU_SRT_ALL ;
              kernel->cndType |= CU_STR_LST ;
            }
          }
          else
          {
            // This memory has already been allocated
            sInf->h_candidates = sInf->sSpec->outData;
            memset(sInf->h_candidates, 0, fullCSize ); // NOTE: this may error if the preallocated memory int karge enough!
          }
        }
        else if ( kernel->cndType & CU_STR_QUAD )
        {
          if ( sInf->sSpec->outData == NULL )
          {
            candTree* qt = new candTree;
            sInf->h_candidates = qt;
          }
          else
          {
            sInf->h_candidates = sInf->sSpec->outData;
          }
        }
        else if ( kernel->cndType & CU_STR_LST  )
        {
          // Nothing really to do here =/
          GSList* lst = g_slist_alloc();
          lst->data = NULL;
          lst->next = NULL;

          sInf->h_candidates    = lst;
        }
        else if ( kernel->cndType & CU_STR_PLN  )
        {
          fprintf(stderr,"WARNING: The case of candidate planes has not been implemented!\n");

          // This memory has already been allocated
          sInf->h_candidates = sInf->sSpec->outData;
        }

        nvtxRangePop();
      }
    }

    if ( hostC )
    {
      printf("    Input and candidates use and additional:\n");
      if ( hostC )
        printf("                        %5.2f GiB of host   memory\n", hostC / 1073741824.0 );
    }
    printf("    -----------------------------------------------\n" );

    CUDA_SAFE_CALL(hipGetLastError(), "Failed to create memory for candidate list or input data.");

    printf("  Done\n");

    nvtxRangePop();
  }

  FOLD // Create FFT plans, ( 1 - set per device )  .
  {
    nvtxRangePush("FFT plans");

    if ( ( kernel->flags & CU_INPT_FFT_CPU ) && master == NULL )
    {
      nvtxRangePush("read_wisdom");

      read_wisdom();

      nvtxRangePop();
    }

    if ( !(kernel->flags & CU_FFT_SEP) )
    {
      infoMSG(3,4,"Create FFT plans\n");

      createFFTPlans(kernel);
    }

    nvtxRangePop();
  }

  FOLD // Create texture memory from kernels  .
  {
    if ( kernel->flags & FLAG_TEX_MUL )
    {
      infoMSG(3,4,"Create texture memory\n");

      nvtxRangePush("text mem");

      hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);

      CUDA_SAFE_CALL(hipGetLastError(), "Creating texture from kernel data.");

      for (int i = 0; i < kernel->noStacks; i++)           // Loop through Stacks
      {
        cuFfdotStack* cStack = &kernel->stacks[i];

        struct hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        texDesc.addressMode[0]            = hipAddressModeClamp;
        texDesc.addressMode[1]            = hipAddressModeClamp;
        texDesc.filterMode                = hipFilterModePoint;
        texDesc.readMode                  = hipReadModeElementType;
        texDesc.normalizedCoords          = 0;

        hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType                   = hipResourceTypePitch2D;
        resDesc.res.pitch2D.desc          = channelDesc;
        resDesc.res.pitch2D.devPtr        = cStack->d_kerData;
        resDesc.res.pitch2D.width         = cStack->width;
        resDesc.res.pitch2D.pitchInBytes  = cStack->strideCmplx * sizeof(fcomplex);
        resDesc.res.pitch2D.height        = cStack->kerHeigth;

        CUDA_SAFE_CALL(hipCreateTextureObject(&cStack->kerDatTex, &resDesc, &texDesc, NULL), "Creating texture from kernel data.");

        CUDA_SAFE_CALL(hipGetLastError(), "Creating texture from the stack of kernel data.");

        // Create the actual texture object
        for (int j = 0; j< cStack->noInStack; j++)        // Loop through planes in stack
        {
          cuKernel* cKer = &cStack->kernels[j];

          resDesc.res.pitch2D.devPtr        = cKer->d_kerData;
          resDesc.res.pitch2D.height        = cKer->harmInf->height;
          resDesc.res.pitch2D.width         = cKer->harmInf->width;
          resDesc.res.pitch2D.pitchInBytes  = cStack->strideCmplx * sizeof(fcomplex);

          CUDA_SAFE_CALL(hipCreateTextureObject(&cKer->kerDatTex, &resDesc, &texDesc, NULL), "Creating texture from kernel data.");
          CUDA_SAFE_CALL(hipGetLastError(), "Creating texture from kernel data.");
        }
      }

      nvtxRangePop();
    }
  }

  FOLD // Set constant memory values  .
  {
    infoMSG(3,4,"Set constant memory values\n");

    nvtxRangePush("const mem");

    setConstVals( kernel,  sInf->noHarmStages, sInf->powerCut, sInf->numindep );

    setConstVals_Fam_Order( kernel );                            // Constant values for multiply

    setStackVals( kernel );

    FOLD // // CUFFT callbacks
    {
      if ( !(kernel->flags & CU_FFT_SEP) )
      {
#if CUDA_VERSION >= 6050        // CUFFT callbacks only implemented in CUDA 6.5
        copyCUFFT_LD_CB(kernel);
#endif
      }
    }

    nvtxRangePop();
  }

  printf("Done initializing GPU %i.\n",device);

  std::cout.flush();
  nvtxRangePop();

  return noBatches;
}

void freeKernelGPUmem(cuFFdotBatch* kernrl)
{
  cudaFreeNull(kernrl->d_kerData);

  CUDA_SAFE_CALL(hipGetLastError(), "Freeing device memory for kernel.\n");
}

/** Free kernel data structure  .
 *
 * @param kernel
 * @param master
 */
void freeKernel(cuFFdotBatch* kernrl)
{
  freeKernelGPUmem(kernrl);

  freeNull(kernrl->stacks);
  freeNull(kernrl->hInfos);
  freeNull(kernrl->kernels);
}

/** Initialise the pointers of the planes data structures of a batch  .
 *
 * This assumes the stack pointers have already been setup
 *
 * @param batch
 */
void setPlanePointers(cuFFdotBatch* batch)
{
  infoMSG(4,5,"setPlanePointers\n");

  for (int i = 0; i < batch->noStacks; i++)
  {
    infoMSG(4,6,"stack %i\n", i);

    // Set stack pointers
    cuFfdotStack* cStack  = &batch->stacks[i];

    for (int j = 0; j < cStack->noInStack; j++)
    {
      infoMSG(4,7,"plane %i\n", i);

      cuFFdot* cPlane           = &cStack->planes[j];

      cPlane->d_planeMult       = &cStack->d_planeMult[ cStack->startZ[j] * batch->noSteps * cStack->strideCmplx ];

      if (cStack->d_planePowr)
      {
        if ( batch->flags & FLAG_HALF )
        {
#if CUDA_VERSION >= 7050
          cPlane->d_planePowr   = &((half*)         cStack->d_planePowr)[ cStack->startZ[j] * batch->noSteps * cStack->stridePower ];
#else
          fprintf(stderr,"ERROR: Half precision can only be used with CUDA 7.5 or later!\n");
          exit(EXIT_FAILURE);
#endif
        }
        else
        {
          if ( batch->flags & FLAG_CUFFT_CB_POW )
            cPlane->d_planePowr = &((float*)      cStack->d_planePowr)[ cStack->startZ[j] * batch->noSteps * cStack->stridePower ];
          else
            cPlane->d_planePowr = &((fcomplexcu*) cStack->d_planePowr)[ cStack->startZ[j] * batch->noSteps * cStack->stridePower ];
        }
      }

      cPlane->d_iData           = &cStack->d_iData[cStack->strideCmplx*j*batch->noSteps];
      cPlane->harmInf           = &cStack->harmInf[j];
      cPlane->kernel            = &cStack->kernels[j];
    }
  }
}

/** Initialise the pointers of the stacks data structures of a batch  .
 *
 * This assumes the various memory blocks of the batch have been created
 *
 * @param batch
 */
void setStkPointers(cuFFdotBatch* batch)
{
  infoMSG(4,5,"setStkPointers\n");

  size_t cmplStart  = 0;
  size_t pwrStart   = 0;
  size_t idSiz      = 0;            /// The size in bytes of input data for one stack
  int harm          = 0;            /// The harmonic index of the first plane the the stack

  for (int i = 0; i < batch->noStacks; i++) // Set the various pointers of the stacks  .
  {
    infoMSG(4,6,"stack %i\n", i);

    cuFfdotStack* cStack  = &batch->stacks[i];

    cStack->d_iData       = &batch->d_iData[idSiz];
    cStack->h_iData       = &batch->h_iData[idSiz];
    cStack->planes        = &batch->planes[harm];
    cStack->kernels       = &batch->kernels[harm];
    cStack->d_planeMult   = &batch->d_planeMult[cmplStart];
    if (batch->d_planePowr)
    {
      if ( batch->flags & FLAG_HALF )
      {
#if CUDA_VERSION >= 7050
        cStack->d_planePowr     = &((half*)       batch->d_planePowr)[ pwrStart ];
#else
        fprintf(stderr,"ERROR: Half precision can only be used with CUDA 7.5 or later!\n");
        exit(EXIT_FAILURE);
#endif
      }
      else
      {
        if ( batch->flags & FLAG_CUFFT_CB_POW )
          cStack->d_planePowr   = &((float*)      batch->d_planePowr)[ pwrStart ];
        else
          cStack->d_planePowr   = &((fcomplexcu*) batch->d_planePowr)[ pwrStart ];
      }
    }

    // Increment the various values used for offset
    harm                 += cStack->noInStack;
    idSiz                += batch->noSteps  * cStack->strideCmplx * cStack->noInStack;
    cmplStart            += cStack->height  * cStack->strideCmplx * batch->noSteps ;
    pwrStart             += cStack->height  * cStack->stridePower * batch->noSteps ;
  }
}

/** Initialise the pointers of the stacks and planes data structures of a batch  .
 *
 * This assumes the various memory blocks of the batch have been created
 *
 * @param batch
 */
void setBatchPointers(cuFFdotBatch* batch)
{
  // First initialise the various pointers of the stacks
  setStkPointers(batch);

  // Now initialise the various pointers of the planes
  setPlanePointers(batch);
}

/** Initialise a batch using details from the device kernel  .
 *
 * @param batch
 * @param kernel
 * @param no
 * @param of
 * @return
 */
int initBatch(cuFFdotBatch* batch, cuFFdotBatch* kernel, int no, int of)
{
  char msg[1024];
  sprintf(msg,"%i of %i", no, of);
  nvtxRangePush(msg);

  char strBuff[1024];
  size_t free, total;

  FOLD // See if we can use the cuda device  .
  {
    infoMSG(3,4,"Device %i\n", kernel->device);

    setDevice(kernel->device) ;

    CUDA_SAFE_CALL(hipMemGetInfo ( &free, &total ), "Getting Device memory information");
  }

  FOLD // Copy details from kernel and allocate stacks .
  {
    infoMSG(3,4,"Copy kernel\n");

    // Copy the basic batch parameters from the kernel
    memcpy(batch, kernel, sizeof(cuFFdotBatch));

    batch->srchMaster   = 0;
    batch->isKernel     = 0;

    infoMSG(3,4,"Create and copy stacks\n");

    // Allocate memory for the stacks
    batch->stacks = (cuFfdotStack*) malloc(batch->noStacks * sizeof(cuFfdotStack));

    // Copy the actual stacks
    memcpy(batch->stacks, kernel->stacks, batch->noStacks  * sizeof(cuFfdotStack));
  }

  FOLD // Set the batch specific flags  .
  {
    infoMSG(3,4,"Set flags\n");

    FOLD // Multiplication flags  .
    {
      for ( int i = 0; i < batch->noStacks; i++ )   // Multiplication is generally stack specific so loop through stacks  .
      {
        cuFfdotStack* cStack  = &batch->stacks[i];

        FOLD // multiplication kernel  .
        {
          if ( !(cStack->flags & FLAG_MUL_ALL ) )   // Default to multiplication  .
          {
            int64_t mFlag = 0;

            // In my testing I found multiplying each plane separately works fastest so it is the "default"
            int noInp =  cStack->noInStack * kernel->noSteps ;

            if ( batch->capability > 3.0 )
            {
              // Lots of registers per thread so 2.1 is good
              mFlag |= FLAG_MUL_21;
            }
            else
            {
              // We require fewer registers per thread, so use Multiplication kernel 2.1
              if ( noInp <= 20 )
              {
                // TODO: Check small, looks like some times 22 may be faster.
                mFlag |= FLAG_MUL_21;
              }
              else
              {
                if ( kernel->noSteps <= 4 )
                {
                  // very few steps so 2.2 not always the best option
                  if ( kernel->hInfos->zmax > 100 )
                  {
                    // This only really holds for 16 harmonics summed with 3 or 4 steps
                    // In my testing it is generally true for zmax greater than 100
                    mFlag |= FLAG_MUL_23;
                  }
                  else
                  {
                    // Here 22 is usually better
                    mFlag |= FLAG_MUL_22;
                  }
                }
                else
                {
                  // Enough steps to justify Multiplication kernel 2.1
                  mFlag |= FLAG_MUL_22;
                }
              }
            }

            // Set the stack and batch flag
            cStack->flags |= mFlag;
            batch->flags  |= mFlag;
          }
        }

        FOLD // Slices  .
        {
          if ( cStack->mulSlices <= 0 )
          {
            // Multiplication slices not specified so use logical values

            if      ( cStack->width <= 256  )
            {
              cStack->mulSlices = 10;
            }
            else if ( cStack->width <= 512  )
            {
              cStack->mulSlices = 8;
            }
            else if ( cStack->width <= 1024 )
            {
              cStack->mulSlices = 6;
            }
            else if ( cStack->width <= 2048 )
            {
              cStack->mulSlices = 4;
            }
            else if ( cStack->width <= 4096 )
            {
              cStack->mulSlices = 2;
            }
            else
            {
              // TODO: check with a card with many SM's
              cStack->mulSlices = 1;
            }
          }

          // Clamp to size of kernel (ie height of the largest plane)
          cStack->mulSlices = MIN(cStack->mulSlices,cStack->kerHeigth/2.0);
        }

        FOLD // Chunk size  .
        {
          if ( cStack->mulChunk <= 0 )
          {
            cStack->mulChunk = 4;
          }

          // Clamp to size of kernel (ie height of the largest plane)
          cStack->mulChunk = MIN( cStack->mulChunk, ceil(cStack->kerHeigth/2.0) );
        }
      }
    }

    FOLD // Sum and search flags  .
    {
      if ( !(batch->flags & FLAG_SS_ALL ) )   // Default to multiplication  .
      {
        batch->flags |= FLAG_SS_10;
      }

      if ( batch->ssChunk <= 0 )
      {
        //kernel->ssChunk         = 8 ;
        float val = 30.0 / (float) batch->noSteps ;

        batch->ssChunk = MAX(MIN(floor(val), 9),1);
      }
    }
  }

  FOLD // Create FFT plans  .
  {
    if ( kernel->flags & CU_FFT_SEP )
    {
      infoMSG(3,4,"Create FFT plans\n");

      createFFTPlans(batch);

      FOLD // Set CUFFT callbacks
      {
#if CUDA_VERSION >= 6050        // CUFFT callbacks only implemented in CUDA 6.5
        copyCUFFT_LD_CB(batch);
#endif
      }
    }
  }

  FOLD // Allocate all device and host memory for the batch  .
  {
    infoMSG(3,4,"Allocate memory for the batch\n");

    FOLD // Allocate page-locked host memory for input data  .
    {
      nvtxRangePush("Host");

      CUDA_SAFE_CALL(hipHostMalloc(&batch->h_iData, batch->inpDataSize ), "Failed to create page-locked host memory plane input data." );

      if ( batch->flags & CU_NORM_CPU ) // Allocate memory for normalisation
        batch->h_normPowers = (float*) malloc(batch->hInfos->width * sizeof(float));

      nvtxRangePop();
    }

    FOLD // Allocate R value lists  .
    {
      batch->noRArryas        = 5; // This is just a convenient value

      createRvals(batch, &batch->rArr1, &batch->rArraysPlane);
      batch->rAraays = &batch->rArraysPlane;

      if ( batch->flags & FLAG_SEPRVAL )
        createRvals(batch, &batch->rArr2, &batch->rArraysSrch);

//      rVals*    rLev1;
//      rVals**   rLev2;
//
//      int oSet                = 0;
//      batch->noRArryas        = 5; // This is just a convenient value
//
//      rLev1                   = (rVals*)malloc(sizeof(rVals)*batch->noSteps*batch->noHarms*batch->noRArryas);
//      memset(rLev1, 0, sizeof(rVals)*batch->noSteps*batch->noHarms*batch->noRArryas);
//      for (int i1 = 0 ; i1 < batch->noSteps*batch->noHarms*batch->noRArryas; i1++)
//      {
//        rLev1[i1].step = -1; // Invalid step (0 is a valid value!)
//      }
//
//      (*batch->rAraays)          = (rVals***)malloc(batch->noRArryas*sizeof(rVals**));
//
//      for (int rIdx = 0; rIdx < batch->noRArryas; rIdx++)
//      {
//        rLev2                 = (rVals**)malloc(sizeof(rVals*)*batch->noSteps);
//        (*batch->rAraays)[rIdx]  = rLev2;
//
//        for (int step = 0; step < batch->noSteps; step++)
//        {
//          rLev2[step]         = &rLev1[oSet];
//          oSet               += batch->noHarms;
//        }
//      }
    }

    FOLD // Allocate device Memory for Planes, Stacks & Input data (steps)  .
    {
      nvtxRangePush("device");

      size_t req = batch->inpDataSize + batch->plnDataSize + batch->pwrDataSize;

      if ( req > free ) // Not enough memory =(
      {
        printf("Not enough GPU memory to create any more batches.\n");
        return 0;
      }
      else
      {
        if ( batch->inpDataSize )
        {
          CUDA_SAFE_CALL(hipMalloc((void** )&batch->d_iData,       batch->inpDataSize ), "Failed to allocate device memory for batch input.");
          free -= batch->inpDataSize;
        }

        if ( batch->plnDataSize )
        {
          CUDA_SAFE_CALL(hipMalloc((void** )&batch->d_planeMult,   batch->plnDataSize ), "Failed to allocate device memory for batch complex plane.");
          free -= batch->plnDataSize;
        }

        if ( batch->pwrDataSize )
        {
          CUDA_SAFE_CALL(hipMalloc((void** )&batch->d_planePowr,   batch->pwrDataSize ), "Failed to allocate device memory for batch powers plane.");
          free -= batch->pwrDataSize;
        }
      }

      nvtxRangePop();
    }

    FOLD // Allocate device & page-locked host memory for return data  .
    {
      nvtxRangePush("Host");

      FOLD // Allocate device memory  .
      {
        if ( kernel->retDataSize && !(kernel->retType & CU_STR_PLN) )
        {
          if ( batch->retDataSize > free )
          {
            // Not enough memory =(
            printf("Not enough GPU memory for return data.\n");
            return 0;
          }
          else
          {
            CUDA_SAFE_CALL(hipMalloc((void** ) &batch->d_outData1, batch->retDataSize ), "Failed to allocate device memory for return values.");
            free -= batch->retDataSize;

            if ( batch->flags & FLAG_SS_INMEM )
            {
              if ( batch->flags & FLAG_SEPSRCH )
              {
                // Create a separate output space
                CUDA_SAFE_CALL(hipMalloc((void** ) &batch->d_outData2, batch->retDataSize ), "Failed to allocate device memory for return values.");
                free -= batch->retDataSize;
              }
              else
              {
                batch->d_outData2 = batch->d_planeMult;
              }
            }
          }
        }
      }

      FOLD // Allocate page-locked host memory to copy the candidates back to  .
      {
        if ( kernel->retDataSize )
        {
          CUDA_SAFE_CALL(hipHostMalloc(&batch->h_outData1, kernel->retDataSize), "Failed to create page-locked host memory plane for return data.");
          memset(batch->h_outData1, 0, kernel->retDataSize );

          if ( kernel->flags & FLAG_SS_INMEM )
          {
            CUDA_SAFE_CALL(hipHostMalloc(&batch->h_outData2, kernel->retDataSize), "Failed to create page-locked host memory plane for return data.");
            memset(batch->h_outData2, 0, kernel->retDataSize );
          }
        }
      }

      nvtxRangePop();
    }

    FOLD // Create the planes structures
    {
      if ( batch->noGenHarms* sizeof(cuFFdot) > getFreeRamCU() )
      {
        fprintf(stderr, "ERROR: Not enough host memory for search.\n");
        return 0;
      }
      else
      {
        batch->planes = (cuFFdot*) malloc(batch->noGenHarms* sizeof(cuFFdot));
        memset(batch->planes, 0, batch->noGenHarms* sizeof(cuFFdot));
      }
    }

    FOLD // Create timing arrays  .
    {
      if ( batch->flags & FLAG_TIME )
      {
        int sz = batch->noStacks*sizeof(float) ;

        batch->copyH2DTime    = (float*)malloc(sz);
        batch->normTime       = (float*)malloc(sz);
        batch->InpFFTTime     = (float*)malloc(sz);
        batch->multTime       = (float*)malloc(sz);
        batch->InvFFTTime     = (float*)malloc(sz);
        batch->copyToPlnTime  = (float*)malloc(sz);
        batch->searchTime     = (float*)malloc(sz);
        batch->resultTime     = (float*)malloc(sz);
        batch->copyD2HTime    = (float*)malloc(sz);

        memset(batch->copyH2DTime,    0, sz);
        memset(batch->normTime,       0, sz);
        memset(batch->InpFFTTime,     0, sz);
        memset(batch->multTime,       0, sz);
        memset(batch->InvFFTTime,     0, sz);
        memset(batch->copyToPlnTime,  0, sz);
        memset(batch->searchTime,     0, sz);
        memset(batch->resultTime,     0, sz);
        memset(batch->copyD2HTime,    0, sz);
      }
    }
  }

  FOLD // Setup the pointers for the stacks and planes of this batch  .
  {
    infoMSG(3,4,"Setup the pointers\n");

    setBatchPointers(batch);
  }

  FOLD // Set up the batch streams and events  .
  {
    infoMSG(3,4,"Set up the batch streams and events\n");

    FOLD // Create Streams  .
    {
      FOLD // Input streams  .
      {
        // Batch input ( Always needed, for copying input to device )
        CUDA_SAFE_CALL(hipStreamCreate(&batch->inpStream),"Creating input stream for batch.");
        sprintf(strBuff,"%i.%i.1.0 Batch Input", batch->device, no);
        nvtxNameCudaStreamA(batch->inpStream, strBuff);
        //printf("hipStreamCreate: %s\n", strBuff);

        // Stack input
        if ( !(batch->flags & CU_NORM_CPU)  )
        {
          for (int i = 0; i < batch->noStacks; i++)
          {
            cuFfdotStack* cStack  = &batch->stacks[i];

            CUDA_SAFE_CALL(hipStreamCreate(&cStack->inptStream), "Creating input data multStream for stack");
            sprintf(strBuff,"%i.%i.1.%i Stack Input", batch->device, no, i);
            nvtxNameCudaStreamA(cStack->inptStream, strBuff);
            //printf("hipStreamCreate: %s\n", strBuff);
          }
        }
      }

      FOLD // Input FFT streams  .
      {
        for (int i = 0; i < kernel->noStacks; i++)
        {
          cuFfdotStack* cStack = &batch->stacks[i];

          if ( kernel->flags & CU_FFT_SEP )       // Create stream  .
          {
            if ( !(kernel->flags & CU_INPT_FFT_CPU) )
            {
              CUDA_SAFE_CALL(hipStreamCreate(&cStack->fftIStream),"Creating CUDA stream for input fft's");

              sprintf(strBuff,"%i.%i.2.%i Inp FFT", batch->device, no, i);
              nvtxNameCudaStreamA(cStack->fftIStream, strBuff);
              //printf("hipStreamCreate: %s\n", strBuff);
            }
          }
          else                                    // Copy stream of the kernel  .
          {
            cuFfdotStack* kStack  = &kernel->stacks[i];
            cStack->fftIStream    = kStack->fftIStream;
          }
        }
      }

      FOLD // Multiply streams  .
      {
        if      ( batch->flags & FLAG_MUL_BATCH )
        {
          CUDA_SAFE_CALL(hipStreamCreate(&batch->multStream),"Creating multiplication stream for batch.");
          sprintf(strBuff,"%i.%i.3.0 Batch Multiply", batch->device, no);
          nvtxNameCudaStreamA(batch->multStream, strBuff);
          //printf("hipStreamCreate: %s\n", strBuff);
        }

        if ( (batch->flags & FLAG_MUL_STK) || (batch->flags & FLAG_MUL_PLN)  )
        {
          for (int i = 0; i< batch->noStacks; i++)
          {
            cuFfdotStack* cStack  = &batch->stacks[i];

            CUDA_SAFE_CALL(hipStreamCreate(&cStack->multStream), "Creating multStream for stack");
            sprintf(strBuff,"%i.%i.3.%i Stack Multiply", batch->device, no, i);
            nvtxNameCudaStreamA(cStack->multStream, strBuff);
            //printf("hipStreamCreate: %s\n", strBuff);
          }
        }
      }

      FOLD // Inverse FFT streams  .
      {
        for (int i = 0; i < kernel->noStacks; i++)
        {
          cuFfdotStack* cStack = &batch->stacks[i];

          if ( batch->flags & CU_FFT_SEP )           // Create stream
          {
            CUDA_SAFE_CALL(hipStreamCreate(&cStack->fftPStream),"Creating CUDA stream for fft's");

            sprintf(strBuff,"%i.%i.4.%i Stack iFFT", batch->device, no, i);
            nvtxNameCudaStreamA(cStack->fftPStream, strBuff);
            //printf("hipStreamCreate: %s\n", strBuff);
          }
          else                                        // Copy stream of the kernel
          {
            cuFfdotStack* kStack  = &kernel->stacks[i];
            cStack->fftPStream    = kStack->fftPStream;
          }
        }
      }

      FOLD // Search stream  .
      {
        CUDA_SAFE_CALL(hipStreamCreate(&batch->srchStream), "Creating strmSearch for batch.");
        sprintf(strBuff,"%i.%i.5.0 Batch Search", batch->device, no);
        nvtxNameCudaStreamA(batch->srchStream, strBuff);
        //printf("hipStreamCreate: %s\n", strBuff);
      }

      FOLD // Result stream  .
      {
        // Batch output ( Always needed, for copying results from device )
        CUDA_SAFE_CALL(hipStreamCreate(&batch->resStream), "Creating strmSearch for batch.");
        sprintf(strBuff,"%i.%i.6.0 Batch result", batch->device, no);
        nvtxNameCudaStreamA(batch->resStream, strBuff);
        //printf("hipStreamCreate: %s\n", strBuff);
      }

      CUDA_SAFE_CALL(hipGetLastError(), "Creating streams for the batch.");
    }

    FOLD // Create Events  .
    {
      FOLD // Create batch events  .
      {
        if ( batch->flags & FLAG_TIME )
        {
          CUDA_SAFE_CALL(hipEventCreate(&batch->iDataCpyComp), "Creating input event iDataCpyComp.");
          CUDA_SAFE_CALL(hipEventCreate(&batch->candCpyComp),  "Creating input event candCpyComp.");
          CUDA_SAFE_CALL(hipEventCreate(&batch->normComp),     "Creating input event normComp.");
          CUDA_SAFE_CALL(hipEventCreate(&batch->multComp),     "Creating input event multComp.");
          CUDA_SAFE_CALL(hipEventCreate(&batch->searchComp),   "Creating input event searchComp.");
          CUDA_SAFE_CALL(hipEventCreate(&batch->processComp),  "Creating input event processComp.");

          CUDA_SAFE_CALL(hipEventCreate(&batch->iDataCpyInit), "Creating input event iDataCpyInit.");
          CUDA_SAFE_CALL(hipEventCreate(&batch->candCpyInit),  "Creating input event candCpyInit.");
          CUDA_SAFE_CALL(hipEventCreate(&batch->multInit),     "Creating input event multInit.");
          CUDA_SAFE_CALL(hipEventCreate(&batch->searchInit),   "Creating input event searchInit.");
        }
        else
        {
          CUDA_SAFE_CALL(hipEventCreateWithFlags(&batch->iDataCpyComp,   hipEventDisableTiming ), "Creating input event iDataCpyComp.");
          CUDA_SAFE_CALL(hipEventCreateWithFlags(&batch->candCpyComp,    hipEventDisableTiming ), "Creating input event candCpyComp.");
          CUDA_SAFE_CALL(hipEventCreateWithFlags(&batch->normComp,       hipEventDisableTiming ), "Creating input event normComp.");
          CUDA_SAFE_CALL(hipEventCreateWithFlags(&batch->multComp,       hipEventDisableTiming ), "Creating input event searchComp.");
          CUDA_SAFE_CALL(hipEventCreateWithFlags(&batch->searchComp,     hipEventDisableTiming ), "Creating input event searchComp.");
          CUDA_SAFE_CALL(hipEventCreateWithFlags(&batch->processComp,    hipEventDisableTiming ), "Creating input event processComp.");
        }
      }

      FOLD // Create stack events  .
      {
        for (int i = 0; i< batch->noStacks; i++)
        {
          cuFfdotStack* cStack  = &batch->stacks[i];

          if ( batch->flags & FLAG_TIME )
          {
            // in  events (with timing)
            CUDA_SAFE_CALL(hipEventCreate(&cStack->normInit),    "Creating input normalisation event");
            CUDA_SAFE_CALL(hipEventCreate(&cStack->inpFFTinit),  "Creating input FFT initialisation event");
            CUDA_SAFE_CALL(hipEventCreate(&cStack->multInit),    "Creating multiplication initialisation event");
            CUDA_SAFE_CALL(hipEventCreate(&cStack->ifftInit), 	  "Creating inverse FFT initialisation event");
            CUDA_SAFE_CALL(hipEventCreate(&cStack->ifftMemInit), "Creating inverse FFT copy initialisation event");

            // out events (with timing)
            CUDA_SAFE_CALL(hipEventCreate(&cStack->normComp),    "Creating input normalisation event");
            CUDA_SAFE_CALL(hipEventCreate(&cStack->prepComp), 		"Creating input data preparation complete event");
            CUDA_SAFE_CALL(hipEventCreate(&cStack->multComp), 		"Creating multiplication complete event");
            CUDA_SAFE_CALL(hipEventCreate(&cStack->ifftComp),    "Creating IFFT complete event");
            CUDA_SAFE_CALL(hipEventCreate(&cStack->ifftMemComp), "Creating IFFT memory copy complete event");
          }
          else
          {
            // out events (without timing)
            CUDA_SAFE_CALL(hipEventCreateWithFlags(&cStack->normComp,    hipEventDisableTiming), "Creating input data preparation complete event");
            CUDA_SAFE_CALL(hipEventCreateWithFlags(&cStack->prepComp,    hipEventDisableTiming), "Creating input data preparation complete event");
            CUDA_SAFE_CALL(hipEventCreateWithFlags(&cStack->multComp,    hipEventDisableTiming), "Creating multiplication complete event");
            CUDA_SAFE_CALL(hipEventCreateWithFlags(&cStack->ifftComp,    hipEventDisableTiming), "Creating IFFT complete event");
            CUDA_SAFE_CALL(hipEventCreateWithFlags(&cStack->ifftMemComp, hipEventDisableTiming), "Creating IFFT memory copy complete event");
          }
        }
      }
      CUDA_SAFE_CALL(hipGetLastError(), "Creating events for the batch.");
    }

    //CUDA_SAFE_CALL(hipGetLastError(), "Creating streams and events for the batch.");
  }

  FOLD // Create textures for the f-∂f planes  .
  {
    if ( (batch->flags & FLAG_TEX_INTERP) && !( (batch->flags & FLAG_CUFFT_CB_POW) && (batch->flags & FLAG_SAS_TEX) ) )
    {
      fprintf(stderr, "ERROR: Cannot use texture memory interpolation without CUFFT callback to write powers. NOT using texture memory interpolation\n");
      batch->flags &= ~FLAG_TEX_INTERP;
    }

    if ( batch->flags & FLAG_SAS_TEX )
    {
      infoMSG(3,4,"Create textures\n");

      hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

      struct hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.addressMode[0]    = hipAddressModeClamp;
      texDesc.addressMode[1]    = hipAddressModeClamp;
      texDesc.readMode          = hipReadModeElementType;
      texDesc.normalizedCoords  = 0;

      if ( batch->flags & FLAG_TEX_INTERP )
      {
        texDesc.filterMode        = hipFilterModeLinear;   /// Liner interpolation
      }
      else
      {
        texDesc.filterMode        = hipFilterModePoint;
      }

      for (int i = 0; i< batch->noStacks; i++)
      {
        cuFfdotStack* cStack = &batch->stacks[i];

        hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType           = hipResourceTypePitch2D;
        resDesc.res.pitch2D.desc  = channelDesc;

        for (int j = 0; j< cStack->noInStack; j++)
        {
          cuFFdot* cPlane = &cStack->planes[j];

          if ( batch->flags & FLAG_CUFFT_CB_POW ) // float input
          {
            if      ( batch->flags & FLAG_ITLV_ROW )
            {
              resDesc.res.pitch2D.height          = cPlane->harmInf->height;
              resDesc.res.pitch2D.width           = cPlane->harmInf->width * batch->noSteps;
              resDesc.res.pitch2D.pitchInBytes    = cStack->harmInf->width * batch->noSteps * sizeof(float);
              resDesc.res.pitch2D.devPtr          = cPlane->d_planePowr;
            }
            else
            {
              resDesc.res.pitch2D.height          = cPlane->harmInf->height * batch->noSteps ;
              resDesc.res.pitch2D.width           = cPlane->harmInf->width;
              resDesc.res.pitch2D.pitchInBytes    = cStack->harmInf->width * sizeof(float);
              resDesc.res.pitch2D.devPtr          = cPlane->d_planePowr;
            }
          }
          else // Implies complex numbers
          {
            if      ( batch->flags & FLAG_ITLV_ROW )
            {
              resDesc.res.pitch2D.height          = cPlane->harmInf->height;
              resDesc.res.pitch2D.width           = cPlane->harmInf->width * batch->noSteps * 2;
              resDesc.res.pitch2D.pitchInBytes    = cStack->harmInf->width * batch->noSteps * 2 * sizeof(float);
              resDesc.res.pitch2D.devPtr          = cPlane->d_planePowr;
            }
            else
            {
              resDesc.res.pitch2D.height          = cPlane->harmInf->height * batch->noSteps ;
              resDesc.res.pitch2D.width           = cPlane->harmInf->width * 2;
              resDesc.res.pitch2D.pitchInBytes    = cStack->harmInf->width * 2 * sizeof(float);
              resDesc.res.pitch2D.devPtr          = cPlane->d_planePowr;
            }
          }

          CUDA_SAFE_CALL(hipCreateTextureObject(&cPlane->datTex, &resDesc, &texDesc, NULL), "Creating texture from the plane data.");
        }
      }
      CUDA_SAFE_CALL(hipGetLastError(), "Creating textures from the plane data.");
    }
  }

  nvtxRangePop();

  return batch->noSteps;
}

/** Free batch data structure  .
 *
 * @param batch
 */
void freeBatchGPUmem(cuFFdotBatch* batch)
{
  CUDA_SAFE_CALL(hipGetLastError(), "Entering freeBatchGPUmem.");

  setDevice(batch->device) ;

  FOLD // Free host memory
  {
    infoMSG(2,2,"Free host memory\n");

    freeNull(batch->h_normPowers);
  }

  FOLD // Free pinned memory
  {
    infoMSG(2,2,"Free pinned memory\n");

    cudaFreeHostNull(batch->h_iData);
    cudaFreeHostNull(batch->h_outData1);
  }

  FOLD // Free device memory
  {
    infoMSG(2,2,"Free device memory\n");

    // Free the output memory
    if ( batch->d_outData1 == batch->d_planeMult )
    {
      batch->d_outData1 = NULL;
    }
    else if ( batch->d_outData2 == batch->d_planeMult )
    {
      batch->d_outData2 = NULL;
    }

    if ( batch->d_outData1 == batch->d_outData2 )
    {
      cudaFreeNull(batch->d_outData1);
      batch->d_outData2 = NULL;
    }
    else
    {
      cudaFreeNull(batch->d_outData1);
      cudaFreeNull(batch->d_outData2);
    }

    // Free the input and planes
    cudaFreeNull(batch->d_iData);
    cudaFreeNull(batch->d_planeMult );
    cudaFreeNull(batch->d_planePowr );

    // Free the rval arrays used during generation and search stages
    freeRvals(batch, &batch->rArr1, &batch->rArraysPlane);
    if ( batch->flags & FLAG_SEPRVAL )
      freeRvals(batch, &batch->rArr2, &batch->rArraysSrch);
  }

  FOLD // Free textures for the f-∂f planes  .
  {
    if ( batch->flags & FLAG_SAS_TEX )
    {
      infoMSG(2,2,"Free textures\n");

      for (int i = 0; i < batch->noStacks; i++)
      {
        cuFfdotStack* cStack = &batch->stacks[i];

        for (int j = 0; j< cStack->noInStack; j++)
        {
          cuFFdot* cPlane = &cStack->planes[j];

          if ( cPlane->datTex )
          {
            CUDA_SAFE_CALL(hipDestroyTextureObject(cPlane->datTex), "Creating texture from the plane data.");
            cPlane->datTex = (fCplxTex)0;
          }
        }
      }
      CUDA_SAFE_CALL(hipGetLastError(), "Creating textures from the plane data.");
    }
  }

  CUDA_SAFE_CALL(hipGetLastError(), "Exiting freeBatchGPUmem.");
}

/** Free batch data structure  .
 *
 * @param batch
 */
void freeBatch(cuFFdotBatch* batch)
{
  freeBatchGPUmem(batch);

  FOLD // Free host memory
  {
    freeNull(batch->stacks);
    freeNull(batch->planes);

    if ( batch->flags & FLAG_TIME )
    {
      freeNull(batch->copyH2DTime   );
      freeNull(batch->normTime      );
      freeNull(batch->InpFFTTime    );
      freeNull(batch->multTime      );
      freeNull(batch->InvFFTTime    );
      freeNull(batch->copyToPlnTime );
      freeNull(batch->searchTime    );
      freeNull(batch->resultTime    );
      freeNull(batch->copyD2HTime   );
    }
  }

}

/** Initiate a optimisation plane
 * If oPln has not been pre initialised and is NULL it will create a new data structure.
 * If oPln has been pre initialised the device ID and Idx are used!
 *
 */
cuOptCand* initOptCand(cuSearch* sSrch, cuOptCand* oPln = NULL, int devLstId = 0 )
{
  searchSpecs* sSpec = sSrch->sSpec;

  if ( !oPln )
  {
    oPln = (cuOptCand*)malloc(sizeof(cuOptCand));
    memset(oPln,0,sizeof(cuOptCand));

    if ( devLstId < MAX_GPUS )
    {
      oPln->device = sSrch->gSpec->devId[devLstId];
    }
    else
    {
      fprintf(stderr, "ERROR: Device list index is greater that the list length, in function: %s.\n", __FUNCTION__);
      exit(EXIT_FAILURE);
    }
  }
  else
  {
    if ( oPln->device != sSrch->gSpec->devId[devLstId] )
    {
      bool found = false;

      for ( int lIdx = 0; lIdx < MAX_GPUS; lIdx++ )
      {
        if ( sSrch->gSpec->devId[lIdx] == oPln->device )
        {
          devLstId = lIdx;
          found = true;
          break;
        }
      }

      if (!found)
      {
        if (devLstId < MAX_GPUS )
        {
          oPln->device = sSrch->gSpec->devId[devLstId];
        }
        else
        {
          fprintf(stderr, "ERROR: Device list index is greater that the list length, in function: %s.\n", __FUNCTION__);
          exit(EXIT_FAILURE);
        }

      }
    }
  }

  FOLD // Create stuff  .
  {
    setDevice(oPln->device) ;

    int   noHarms       = (1<<(sSpec->noHarmStages-1));
    float zMax          = MAX(sSpec->zMax+50, sSpec->zMax*2);
    zMax                = MAX(zMax, 60 * noHarms );
    //zMax                = MAX(zMax, sSpec->zMax * 34 + 50 );  // This may be a bit high!

    oPln->maxHalfWidth  = z_resp_halfwidth( zMax, HIGHACC );
    oPln->maxNoR        = 512;
    oPln->maxNoZ        = 512;
    oPln->outSz         = oPln->maxNoR * oPln->maxNoZ ;       // This needs to be multiplied by the size of the output element
    oPln->alignment     = sSrch->gSpec->devInfo[devLstId].alignment; //getMemAlignment();

    // Create streams
    CUDA_SAFE_CALL(hipStreamCreate(&oPln->stream),"Creating stream for candidate optimisation.");
    char nmStr[1024];
    sprintf(nmStr,"Optimisation Stream %02i", oPln->pIdx);
    nvtxNameCudaStreamA(oPln->stream, nmStr);
    //printf("hipStreamCreate: %s\n", nmStr);

    // Events
    CUDA_SAFE_CALL(hipEventCreate(&oPln->inpInit),     "Creating input event inpInit." );
    CUDA_SAFE_CALL(hipEventCreate(&oPln->inpCmp),      "Creating input event inpCmp."  );
    CUDA_SAFE_CALL(hipEventCreate(&oPln->compInit),    "Creating input event compInit.");
    CUDA_SAFE_CALL(hipEventCreate(&oPln->compCmp),     "Creating input event compCmp." );
    CUDA_SAFE_CALL(hipEventCreate(&oPln->outInit),     "Creating input event outInit." );
    CUDA_SAFE_CALL(hipEventCreate(&oPln->outCmp),      "Creating input event outCmp."  );

    size_t freeMem, totalMem;

    oPln->outSz        *= sizeof(float);
    oPln->inpSz         = (oPln->maxNoR + 2*oPln->maxHalfWidth)*noHarms*sizeof(hipfftComplex)*2;

    CUDA_SAFE_CALL(hipMemGetInfo ( &freeMem, &totalMem ), "Getting Device memory information");

    if ( (oPln->inpSz + oPln->outSz) > freeMem )
    {
      printf("Not enough GPU memory to create any more stacks.\n");
      free(oPln);
      return NULL;
    }
    else
    {
      // Allocate device memory
      CUDA_SAFE_CALL(hipMalloc(&oPln->d_out,  oPln->outSz),   "Failed to allocate device memory for kernel stack.");
      CUDA_SAFE_CALL(hipMalloc(&oPln->d_inp,  oPln->inpSz),   "Failed to allocate device memory for kernel stack.");

      // Allocate host memory
      CUDA_SAFE_CALL(hipHostMalloc(&oPln->h_out,  oPln->outSz), "Failed to allocate device memory for kernel stack.");
      CUDA_SAFE_CALL(hipHostMalloc(&oPln->h_inp,  oPln->inpSz), "Failed to allocate device memory for kernel stack.");
    }
  }
  return oPln;
}

int setStackInfo(cuFFdotBatch* batch, stackInfo* h_inf, int offset)
{
  infoMSG(4,4,"setStackInfo\n" );

  stackInfo* dcoeffs;
  hipGetSymbolAddress((void **)&dcoeffs, STACKS );

  for (int i = 0; i < batch->noStacks; i++)
  {
    infoMSG(4,5,"stack %i\n",i);

    cuFfdotStack* cStack  = &batch->stacks[i];
    stackInfo*    cInf    = &h_inf[i];

    cInf->noSteps         = batch->noSteps;
    cInf->noPlanes        = cStack->noInStack;
    cInf->famIdx          = cStack->startIdx;
    cInf->flags           = batch->flags;

    cInf->d_iData         = cStack->d_iData;
    cInf->d_planeData     = cStack->d_planeMult;
    cInf->d_planePowers   = cStack->d_planePowr;

    // Set the pointer to constant memory
    cStack->stkIdx        = offset+i;
    cStack->d_sInf        = dcoeffs + offset+i ;
  }

  return batch->noStacks;
}

int setConstVals_Fam_Order( cuFFdotBatch* batch )
{
  FOLD // Set other constant values
  {
    void *dcoeffs;

    int           height[MAX_HARM_NO];
    int           stride[MAX_HARM_NO];
    int            width[MAX_HARM_NO];
    fcomplexcu*   kerPnt[MAX_HARM_NO];

    FOLD // Set values  .
    {
      for (int i = 0; i < batch->noGenHarms; i++)
      {
        cuFfdotStack* cStack  = &batch->stacks[ batch->hInfos[i].stackNo];

        height[i] = batch->hInfos[i].height;
        stride[i] = cStack->strideCmplx;
        width[i]  = batch->hInfos[i].width;
        kerPnt[i] = batch->kernels[i].d_kerData;

        if ( (i>=batch->noGenHarms) &&  (batch->hInfos[i].width != cStack->strideCmplx) )
        {
          fprintf(stderr,"ERROR: Width is not the same as stride, using width this may case errors in the multiplication.\n");
        }
      }

      // Rest
      for (int i = batch->noGenHarms; i < MAX_HARM_NO; i++)
      {
        height[i] = 0;
        stride[i] = 0;
        width[i]  = 0;
        kerPnt[i] = 0;
      }
    }

    hipGetSymbolAddress((void **)&dcoeffs, HEIGHT_HARM);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &height, MAX_HARM_NO * sizeof(int), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying stages to device");

    hipGetSymbolAddress((void **)&dcoeffs, STRIDE_HARM);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &stride, MAX_HARM_NO * sizeof(int), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying stages to device");

    hipGetSymbolAddress((void **)&dcoeffs, WIDTH_HARM);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &width,  MAX_HARM_NO * sizeof(int), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying stages to device");

    hipGetSymbolAddress((void **)&dcoeffs, KERNEL_HARM);
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, &kerPnt, MAX_HARM_NO * sizeof(fcomplexcu*), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying stages to device");
  }

  CUDA_SAFE_CALL(hipGetLastError(), "Preparing the constant memory values for the multiplications.");

  return 1;
}

int setStackVals( cuFFdotBatch* batch )
{
  stackInfo* dcoeffs;
  //if ( batch->isKernel )
  {
    int         l_STK_STRD[4];
    char        l_STK_INP[4][4069];

    for (int i = 0; i < batch->noStacks; i++)
    {
      cuFfdotStack* cStack  = &batch->stacks[i];

      l_STK_STRD[i] = cStack->strideCmplx;

      int         off     = 0;
      char        inpIdx  = 0;

      // Create the actual texture object
      for (int j = 0; j < cStack->noInStack; j++)        // Loop through planes in stack
      {
        cuHarmInfo*  hInf = &cStack->harmInf[j];

        // Create the actual texture object
        for (int k = 0; k < batch->noSteps; k++)        // Loop through planes in stack
        {
          for ( int h = 0; h < hInf->height; h++ )
          {
            l_STK_INP[i][off++] = inpIdx;
          }
          inpIdx++;
        }
      }
    }

    hipGetSymbolAddress((void **)&dcoeffs, STK_STRD );
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, l_STK_STRD, sizeof(l_STK_STRD), hipMemcpyHostToDevice, batch->stacks->initStream),              "Copying stack info to device");

    hipGetSymbolAddress((void **)&dcoeffs, STK_INP );
    CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, l_STK_INP, sizeof(l_STK_INP), hipMemcpyHostToDevice, batch->stacks->initStream),      "Copying stack info to device");
  }

  return 1;
}

/** Copy host stack info to the device constant memory
 *
 * NOTE: The device should already be set!
 *
 * @param h_inf
 * @param noStacks
 * @return
 */
int setConstStkInfo(stackInfo* h_inf, int noStacks,  hipStream_t stream)
{
  infoMSG(3,4,"set ConstStkInfo(%i)\n", noStacks );

  void *dcoeffs;

  // TODO: Do a test to see if  we are on the correct device

  hipGetSymbolAddress((void **)&dcoeffs, STACKS);
  CUDA_SAFE_CALL(hipMemcpyAsync(dcoeffs, h_inf, noStacks * sizeof(stackInfo), hipMemcpyHostToDevice, stream),      "Copying stack info to device");

  return 1;
}

void drawPlaneCmplx(fcomplexcu* ffdotPlane, char* name, int stride, int height)
{
  float *h_fArr = (float*) malloc(stride * height * sizeof(fcomplexcu));
  //float DestS   = ffdotPlane->ffPowWidth*sizeof(float);
  //float SourceS = ffdotPlane->ffPowStride;
  CUDA_SAFE_CALL(hipMemcpy2D(h_fArr, stride * sizeof(fcomplexcu), ffdotPlane, stride * sizeof(fcomplexcu), stride * sizeof(fcomplexcu), height, hipMemcpyDeviceToHost), "Failed to copy data from device to host");

  //draw2DArray(name, h_fArr, stride*2, height);
  free(h_fArr);
}

void timeSynch(cuFFdotBatch* batch)
{
  if ( (*batch->rAraays)[batch->rActive][0][0].numrs )
  {
    if ( batch->flags & FLAG_TIME ) // Timing  .
    {
      infoMSG(1,2,"Timing\n");

      float time;         // Time in ms of the thing
      hipError_t ret;    // Return status of hipEventElapsedTime

      FOLD // Norm Timing  .
      {
        if ( !(batch->flags & CU_NORM_CPU) )
        {
          for (int ss = 0; ss < batch->noStacks; ss++)
          {
            cuFfdotStack* cStack = &batch->stacks[ss];

            ret = hipEventElapsedTime(&time, cStack->normInit, cStack->normComp);

            if ( ret != hipErrorNotReady )
            {
#pragma omp atomic
              batch->normTime[ss] += time;
            }
          }

          CUDA_SAFE_CALL(hipGetLastError(), "Norm Timing");
        }
      }

      FOLD // Input FFT timing  .
      {
        if ( !(batch->flags & CU_INPT_FFT_CPU) )
        {
          for (int ss = 0; ss < batch->noStacks; ss++)
          {
            cuFfdotStack* cStack = &batch->stacks[ss];

            ret = hipEventElapsedTime(&time, cStack->inpFFTinit, cStack->prepComp);

            if ( ret != hipErrorNotReady )
            {
#pragma omp atomic
              batch->InpFFTTime[ss] += time;
            }
          }

          CUDA_SAFE_CALL(hipGetLastError(), "Input FFT timing");
        }
      }

      FOLD // Copy input data  .
      {
        ret = hipEventElapsedTime(&time, batch->iDataCpyInit, batch->iDataCpyComp);

        if ( ret != hipErrorNotReady )
        {
#pragma omp atomic
          batch->copyH2DTime[0] += time;
        }

        CUDA_SAFE_CALL(hipGetLastError(), "Copy input timing");
      }

      FOLD // Multiplication timing  .
      {
        if ( !(batch->flags & FLAG_MUL_CB) )
        {
          // Did the convolution by separate kernel

          if ( batch->flags & FLAG_MUL_BATCH )   	// Convolution was done on the entire batch  .
          {
            ret = hipEventElapsedTime(&time, batch->multInit, batch->multComp);

            if ( ret != hipErrorNotReady )
            {
#pragma omp atomic
              batch->multTime[0] += time;
            }
          }
          else                                    // Convolution was on a per stack basis  .
          {
            for (int ss = 0; ss < batch->noStacks; ss++)              // Loop through Stacks
            {
              cuFfdotStack* cStack = &batch->stacks[ss];

              ret = hipEventElapsedTime(&time, cStack->multInit, cStack->multComp);

              if ( ret != hipErrorNotReady )
              {
#pragma omp atomic
                batch->multTime[ss] += time;
              }
            }
          }

          CUDA_SAFE_CALL(hipGetLastError(), "Multiplication timing");
        }
      }

      FOLD // Inverse FFT timing  .
      {
        for (int ss = 0; ss < batch->noStacks; ss++)
        {
          cuFfdotStack* cStack = &batch->stacks[ss];

          ret = hipEventElapsedTime(&time, cStack->ifftInit, cStack->ifftComp);
          if ( ret != hipErrorNotReady )
          {
#pragma omp atomic
            batch->InvFFTTime[ss] += time;
          }
        }

        CUDA_SAFE_CALL(hipGetLastError(), "Inverse FFT timing");
      }

      FOLD // Copy to in-mem plane timing  .
      {
        if ( batch->flags & FLAG_SS_INMEM )
        {
          for (int ss = 0; ss < batch->noStacks; ss++)
          {
            cuFfdotStack* cStack = &batch->stacks[ss];

            ret = hipEventElapsedTime(&time, cStack->ifftMemInit, cStack->ifftMemComp);
            if ( ret != hipErrorNotReady )
            {
#pragma omp atomic
              batch->copyToPlnTime[ss] += time;
            }
          }

          CUDA_SAFE_CALL(hipGetLastError(), "Copy to in-mem plane timing");
        }
      }

      FOLD // Search Timing  .
      {
        if ( !(batch->flags & FLAG_SS_CPU) && !(batch->flags & FLAG_SS_INMEM ) )
        {
          ret = hipEventElapsedTime(&time, batch->searchInit, batch->searchComp);

          if ( ret != hipErrorNotReady )
          {
#pragma omp atomic
            batch->searchTime[0] += time;
          }

          CUDA_SAFE_CALL(hipGetLastError(), "Search Timing");
        }
      }

      FOLD // Copy D2H  .
      {
        if ( !(batch->flags & FLAG_SS_INMEM ) )
        {
          ret = hipEventElapsedTime(&time, batch->candCpyInit, batch->candCpyComp);

          if ( ret != hipErrorNotReady )
          {
#pragma omp atomic
            batch->copyD2HTime[0] += time;
          }

          CUDA_SAFE_CALL(hipGetLastError(), "Copy D2H Timing");
        }
      }
    }
  }
}

/** Cycle the arrays of r-values  .
 *
 * @param batch
 */
void cycleRlists(cuFFdotBatch* batch)
{
  infoMSG(2,2,"Cycle R lists\n");

  rVals** hold = (*batch->rAraays)[batch->noRArryas-1];
  for ( int i = batch->noRArryas-1; i > 0; i-- )
  {
    (*batch->rAraays)[i] =  (*batch->rAraays)[i - 1];
  }
  (*batch->rAraays)[0] = hold;

//  if ( msgLevel >= 3 )
//  {
//    for ( int i = 0 ; i < batch->noRArryas; i++ )
//    {
//      rVals* rVal = &(*batch->rAraays)[i][0][0];
//
//      printf("%i  step: %03i  r-low: %8.1f  numrs: %06ld\n", i, rVal->step, rVal->drlo, rVal->numrs );
//    }
//  }
}

/** Cycle the arrays of r-values  .
 *
 * @param batch
 */
void CycleBackRlists(cuFFdotBatch* batch)
{
  infoMSG(2,1,"CycleBackRlists\n");

  rVals** hold = (*batch->rAraays)[0];
  for ( int i = 0; i < batch->noRArryas-1; i++ )
  {
    (*batch->rAraays)[i] =  (*batch->rAraays)[i + 1];
  }

  (*batch->rAraays)[batch->noRArryas-1] = hold;
}

void cycleOutput(cuFFdotBatch* batch)
{
  infoMSG(2,2,"Cycle output\n");

  void* d_hold = batch->d_outData1;
  void* h_hold = batch->h_outData1;

  batch->d_outData1 = batch->d_outData2;
  batch->h_outData1 = batch->h_outData2;

  batch->d_outData2 = d_hold;
  batch->h_outData2 = h_hold;
}

void search_ffdot_batch_CU(cuFFdotBatch* batch, double* searchRLow, double* searchRHi, int norm_type )
{
  infoMSG(1,1,"search_ffdot_batch_CU\n");

  CUDA_SAFE_CALL(hipGetLastError(), "Entering search_ffdot_batch_CU.");

  // Calculate R values
  setActiveBatch(batch, 0);
  setGenRVals(batch, searchRLow, searchRHi );

  if ( batch->flags & FLAG_SYNCH )
  {
    initInput(batch, norm_type);

    multiplyBatch(batch);

    IFFTBatch(batch);

    if  ( batch->flags & FLAG_SS_INMEM )
    {
      copyToInMemPln(batch);
    }
    else
    {
      sumAndSearch(batch);

      getResults(batch);

      processSearchResults(batch);
    }
  }
  else
  {
    setActiveBatch(batch, 0);
    initInput(batch, norm_type);

    if  ( batch->flags & FLAG_SS_INMEM )
    {
      setActiveBatch(batch, 0);
      multiplyBatch(batch);

      setActiveBatch(batch, 1);
      copyToInMemPln(batch);

      setActiveBatch(batch, 0);
      IFFTBatch(batch);
    }
    else
    {
      setActiveBatch(batch, 1);
      sumAndSearch(batch);

      setActiveBatch(batch, 2);
      processSearchResults(batch);

      setActiveBatch(batch, 1);
      getResults(batch);

      setActiveBatch(batch, 0);
      convolveBatch(batch);
    }
  }

  // Change R-values
  cycleRlists(batch);
  setActiveBatch(batch, 1);
}

void finish_Search(cuFFdotBatch* batch)
{
  infoMSG(1,1,"Finish search\n");

  FOLD // A blocking synchronisation to ensure results are ready to be proceeded by the host
  {
    infoMSG(3,4,"pre synchronisation [blocking] ifftMemComp - stack\n");

    for (int ss = 0; ss < batch->noStacks; ss++)
    {
      infoMSG(4,5,"Stack %i\n", ss);

      nvtxRangePush("EventSynch");
      cuFfdotStack* cStack = &batch->stacks[ss];
      CUDA_SAFE_CALL(hipEventSynchronize(cStack->ifftMemComp), "At a blocking synchronisation. This is probably a error in one of the previous asynchronous CUDA calls.");
      nvtxRangePop();
    }

    infoMSG(3,4,"pre synchronisation [blocking] processComp\n");

    nvtxRangePush("EventSynch");
    CUDA_SAFE_CALL(hipEventSynchronize(batch->processComp), "At a blocking synchronisation. This is probably a error in one of the previous asynchronous CUDA calls.");
    nvtxRangePop();
  }
}

void max_ffdot_planeCU(cuFFdotBatch* batch, double* searchRLow, double* searchRHi, int norm_type, fcomplexcu* fft, long long* numindep, float* powers)
{
  CUDA_SAFE_CALL(hipGetLastError(), "Entering ffdot_planeCU2.");

  FOLD // Initialise input data  .
  {
    setActiveBatch(batch, 0);
    initInput(batch, norm_type);
  }

  if ( batch->flags & FLAG_SYNCH )
  {

    FOLD // Multiply & inverse FFT  .
    {
      convolveBatch(batch);
    }

    FOLD // Sum & Max
    {
      //sumAndMax(batch, numindep, powers);
    }

  }
  else
  {

    FOLD // Sum & Max
    {
      //sumAndMax(batch, numindep, powers);
    }

    FOLD // Multiply & inverse FFT  .
    {
      convolveBatch(batch);
    }

  }

}

int selectDevice(int device, int print)
{
  hipDeviceProp_t deviceProp;
  int currentDevvice, deviceCount;  //, device = 0;

  CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount), "Failed to get device count using hipGetDeviceCount");
  //printf("There are %i CUDA capable devices available.");
  if (device>= deviceCount)
  {
    if (deviceCount== 0)
    {
      fprintf(stderr, "ERROR: Could not detect any CUDA capable devices!\n");
      exit(EXIT_FAILURE);
    }
    fprintf(stderr, "ERROR: Attempting to select device %i when I detect only %i devices, using device 0 instead!\n", device, deviceCount);
    device = 0;
  }

  CUDA_SAFE_CALL(hipSetDevice(device), "Failed to set device using hipSetDevice");
  CUDA_SAFE_CALL(hipDeviceReset(), "Failed to set device using : hipDeviceReset");
  CUDA_SAFE_CALL(hipGetLastError(), "At start of everything?.\n");
  CUDA_SAFE_CALL(hipGetDevice(&currentDevvice), "Failed to get device using hipGetDevice");
  if (currentDevvice!= device)
  {
    fprintf(stderr, "ERROR: CUDA Device not set.\n");
    exit(EXIT_FAILURE);
  }

  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, currentDevvice), "Failed to get device properties device using hipGetDeviceProperties");

  if (print)
    printf("\nRunning on device %d: \"%s\"  which has CUDA Capability  %d.%d\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);

  return ((deviceProp.major<< 4)+ deviceProp.minor);
}

void printCands(const char* fileName, GSList *cands, double T)
{
  if ( cands == NULL  )
    return;

  GSList *inp_list = cands ;

  FILE * myfile;                    /// The file being written to
  myfile = fopen ( fileName, "w" );

  if ( myfile == NULL )
    fprintf ( stderr, "ERROR: Unable to open log file %s\n", fileName );
  else
  {
    fprintf(myfile, "#\tr\tf\tz\tfd\tsig\tpower\tharm \n");
    int i = 0;

    while ( inp_list->next )
    {
      fprintf(myfile, "%i\t%14.5f\t%10.6f\t%14.2f\t%13.10f\t%-7.4f\t%7.2f\t%i \n", i+1, ((accelcand *) (inp_list->data))->r, ((accelcand *) (inp_list->data))->r / T, ((accelcand *) (inp_list->data))->z,((accelcand *) (inp_list->data))->z/T/T, ((accelcand *) (inp_list->data))->sigma, ((accelcand *) (inp_list->data))->power, ((accelcand *) (inp_list->data))->numharm );
      inp_list = inp_list->next;
      i++;
    }
    fclose ( myfile );
  }
}

void printContext()
{
  int currentDevvice;
  hipCtx_t pctx;
  hipCtxGetCurrent ( &pctx );
  CUDA_SAFE_CALL(hipGetDevice(&currentDevvice), "Failed to get device using hipGetDevice");

  int trd;
#ifdef WITHOMP
  trd = omp_get_thread_num();
#else
  trd = 0;
#endif

  printf("Thread %02i  currentDevvice: %i Context %p \n", trd, currentDevvice, pctx);
}

int setDevice(int device)
{
  int dev;

  CUDA_SAFE_CALL(hipGetDevice(&dev), "Failed to get device using hipGetDevice");

  if ( dev != device )
  {
    CUDA_SAFE_CALL(hipSetDevice(device), "Failed to set device using hipSetDevice");
    CUDA_SAFE_CALL(hipGetDevice(&dev), "Failed to get device using hipGetDevice");
    if ( dev != device )
    {
      fprintf(stderr, "ERROR: CUDA Device not set.\n");
      exit(EXIT_FAILURE);
    }
  }

  return dev;
}

gpuSpecs gSpec(int devID = -1 )
{
  gpuSpecs gSpec;
  memset(&gSpec, 0 , sizeof(gpuSpecs));

  if (devID < 0 )
  {
    gSpec.noDevices      = getGPUCount();

    for ( int i = 0; i < gSpec.noDevices; i++)
      gSpec.devId[i]        = i;
  }
  else
  {
    gSpec.noDevices      = 1;
    gSpec.devId[0]       = devID;
  }

  // Set default
  for ( int i = 0; i < gSpec.noDevices; i++)
  {
    gSpec.noDevBatches[i] = 2;
    gSpec.noDevSteps[i]   = 4;
  }

  return gSpec;
}

/**  Read the GPU details from clig command line  .
 *
 * @param cmd     clig struct
 * @param bInf    A pointer to the accel info struct to fill
 */
gpuSpecs readGPUcmd(Cmdline *cmd)
{
  gpuSpecs gpul;

  CUDA_SAFE_CALL(hipGetLastError(), "Entering readGPUcmd.");

  if ( cmd->gpuP ) // Determine the index and number of devices
  {
    if ( cmd->gpuC == 0 )  // NB: Note using gpuC == 0 requires a change in accelsearch_cmd.c every time clig is run!!!! [ usually line 32 should be "  /* gpuC = */ 0," ]
    {
      // Make a list of all devices
      gpul.noDevices   = getGPUCount();
      for ( int dev = 0 ; dev < gpul.noDevices; dev++ )
        gpul.devId[dev] = dev;
    }
    else
    {
      // User specified devices(s)
      gpul.noDevices   = cmd->gpuC;
      for ( int dev = 0 ; dev < gpul.noDevices; dev++ )
        gpul.devId[dev] = cmd->gpu[dev];
    }
  }

  for ( int dev = 0 ; dev < gpul.noDevices; dev++ ) // Loop over devices  .
  {
    if ( dev >= cmd->nbatchC )
      gpul.noDevBatches[dev] = cmd->nbatch[cmd->nbatchC-1];
    else
      gpul.noDevBatches[dev] = cmd->nbatch[dev];

    if ( dev >= cmd->nstepsC )
      gpul.noDevSteps[dev] = cmd->nsteps[cmd->nbatchC-1];
    else
      gpul.noDevSteps[dev] = cmd->nsteps[dev];

    if ( dev >= cmd->numoptC )
      gpul.noDevOpt[dev] = cmd->numopt[cmd->nbatchC-1];
    else
      gpul.noDevOpt[dev] = cmd->numopt[dev];

  }

  return gpul;
}

bool strCom(const char* str1, const char* str2)
{
  if ( strncmp(str1,str2, strlen(str2) ) == 0 )
    return 1;
  else
    return 0;
}

/** Read accel search details from the text file
 *
 * @param sSpec
 */
void readAccelDefalts(searchSpecs *sSpec)
{
  int64_t*  flags = &(sSpec->flags);
  FILE *file;
  char fName[1024];
  sprintf(fName, "%s/lib/GPU_defaults.txt", getenv("PRESTO"));

  if ( file = fopen(fName, "r") )  // Read candidates from previous search  .
  {
    printf("Reading GPU search settings from %s\n",fName);

    char* line;
    char  line2[1024];
    int   lineno = 0;

    char *rest;

    while (fgets(line2, sizeof(line2), file))
    {
      lineno++;

      line = line2;

      // Strip proceeding white space
      while ( *line <= 32 &&  *line != 10 )
        line++;

      // Set to only be the word
      int flagLen = 0;
      char* flagEnd = line;
      while ( *flagEnd != ' ' && *flagEnd != 0 && *flagEnd != 10 )
      {
        flagLen++;
        flagEnd++;
      }

      int ll = strlen(line);

      if      ( strCom(line, "FLAG_ITLV_ROW" ) || strCom(line, "INTERLEAVE_ROW" ) ||  strCom(line, "IL_ROW" ) )
      {
        (*flags) |= FLAG_ITLV_ROW;
      }
      else if ( strCom(line, "FLAG_ITLV_PLN" ) || strCom(line, "INTERLEAVE_PLN" ) ||  strCom(line, "IL_PLN" ) )
      {
        (*flags) &= ~FLAG_ITLV_ROW;
      }

      else if ( strCom(line, "FLAG_KER_STD"  ) )
      {
        (*flags) &= ~FLAG_KER_HIGH;
      }
      else if ( strCom(line, "FLAG_KER_HIGH" ) )
      {
        (*flags) |= FLAG_KER_HIGH;
      }
      else if ( strCom(line, "FLAG_KER_MAX"  ) )
      {
        (*flags) |= FLAG_KER_MAX;
      }
      else if ( strCom(line, "FLAG_CENTER"   ) )
      {
        (*flags) |= FLAG_CENTER;
      }

      else if ( strCom(line, "CU_NORM_CPU" ) || strCom(line, "NORM_CPU" ) )
      {
        (*flags) |= CU_NORM_CPU;
      }
      else if ( strCom(line, "CU_NORM_GPU" ) || strCom(line, "NORM_GPU" ) )
      {
        (*flags) &= ~CU_NORM_CPU;
      }

      else if ( strCom(line, "CU_INPT_FFT_CPU" ) || strCom(line, "CPU_FFT" ) || strCom(line, "FFT_CPU" ) )
      {
        (*flags) |= CU_NORM_CPU;
        (*flags) |= CU_INPT_FFT_CPU;
      }
      else if ( strCom(line, "CU_INPT_GPU_FFT" ) || strCom(line, "GPU_FFT" ) || strCom(line, "FFT_GPU" ) )
      {
        (*flags) &= ~CU_INPT_FFT_CPU;
      }

      else if ( strCom(line, "FLAG_MUL_00" ) || strCom(line, "MUL_00" ) )
      {
        (*flags) &= ~FLAG_MUL_ALL;
        (*flags) |=  FLAG_MUL_00;
      }
      else if ( strCom(line, "FLAG_MUL_11" ) || strCom(line, "MUL_11" ) )
      {
        (*flags) &= ~FLAG_MUL_ALL;
        (*flags) |=  FLAG_MUL_11;
      }
      else if ( strCom(line, "FLAG_MUL_21" ) || strCom(line, "MUL_21" ) )
      {
        (*flags) &= ~FLAG_MUL_ALL;
        (*flags) |=  FLAG_MUL_21;
      }
      else if ( strCom(line, "FLAG_MUL_22" ) || strCom(line, "MUL_22" ) )
      {
        (*flags) &= ~FLAG_MUL_ALL;
        (*flags) |=  FLAG_MUL_22;
      }
      else if ( strCom(line, "FLAG_MUL_23" ) || strCom(line, "MUL_23" ) )
      {
        (*flags) &= ~FLAG_MUL_ALL;
        (*flags) |=  FLAG_MUL_23;
      }
      else if ( strCom(line, "FLAG_MUL_30" ) || strCom(line, "MUL_30" ) )
      {
        (*flags) &= ~FLAG_MUL_ALL;
        (*flags) |=  FLAG_MUL_30;
      }
      else if ( strCom(line, "FLAG_MUL_CB" ) || strCom(line, "MUL_CB" ) )
      {
#if CUDA_VERSION >= 6050
        (*flags) &= ~FLAG_MUL_ALL;
        (*flags) |=  FLAG_MUL_CB;
#else
        line[flagLen] = 0;
        fprintf(stderr, "WARNING: Use of CUDA callbacks requires CUDA 6.5 or greater.  (FLAG: %s line %i in %s)\n", line, lineno, fName);
#endif
      }
      else if ( strCom(line, "FLAG_MUL_A"  ) || strCom(line, "MUL_A"  ) )
      {
        (*flags) &= ~FLAG_MUL_ALL;
      } 

      else if ( strCom(line, "FLAG_FFT_SEPERATE"  ) || strCom(line, "FLAG_FFT_SEP"  ) )
      {
        (*flags) |= CU_FFT_SEP;
      }

      else if ( strCom(line, "FLAG_TEX_MUL" ) )
      {
        fprintf(stderr, "WARNING: The flag FLAG_TEX_MUL has been deprecated.\n");
        //(*flags) |= FLAG_TEX_MUL;
      }

      else if ( strCom(line, "MUL_Chunk"  ) || strCom(line, "MUL_CHUNK"  ) )
      {
        char str1[1024];
        char str2[1024];
        int no;
        int read1 = sscanf(line, "%s %i ", str1, &no  );
        int read2 = sscanf(line, "%s %s ", str1, str2 );

        if ( read1 == 2 )
        {
          sSpec->mulChunk = no;
        }
        else if ( strCom(str2, "AA"  ) || strCom(str2, "A"   ) )
        {
          sSpec->mulChunk = 0;
        }
        else
        {
          line[flagLen] = 0;
          fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", line, lineno, fName);
        }
      }
      else if ( strCom(line, "MUL_Slices" ) || strCom(line, "MUL_SLICES" ) )
      {
        char str1[1024];
        char str2[1024];
        int no;
        int read1 = sscanf(line, "%s %i ", str1, &no  );
        int read2 = sscanf(line, "%s %s ", str1, str2 );

        if ( read1 == 2 )
        {
          sSpec->mulSlices = no;
        }
        else if ( strCom(str2, "AA"  ) || strCom(str2, "A"   ) )
        {
          sSpec->mulSlices = 0;
        }
        else
        {
          line[flagLen] = 0;
          fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", line, lineno, fName);
        }
      }

      else if ( strCom(line, "FLAG_CUFFT_CB_POW" ) 		|| strCom(line, "CB_POW"   ) )
      {
#if CUDA_VERSION >= 6050
        (*flags) |= FLAG_CUFFT_CB_POW;
#else
        line[flagLen] = 0;
        fprintf(stderr,"WARNING: Use of CUDA callbacks requires CUDA 6.5 or greater.  (FLAG: %s line %i in %s)\n", line, lineno, fName);
#endif
      }
      else if ( strCom(line, "FLAG_CUFFT_CB_INMEM" )  || strCom(line, "CB_INMEM" ) )
      {
#if CUDA_VERSION >= 6050
        (*flags) |= FLAG_CUFFT_CB_INMEM;
#else
        line[flagLen] = 0;
        fprintf(stderr,"WARNING: Use of CUDA callbacks requires CUDA 6.5 or greater.  (FLAG: %s line %i in %s)\n", line, lineno, fName);
#endif
      }
      else if ( strCom(line, "FLAG_NO_CB" )           || strCom(line, "NO_CB" 	 ) )
      {
        (*flags) &= ~FLAG_CUFFT_ALL;
      }

      else if ( strCom(line, "FLAG_SAS_TEX" ) )
      {
        (*flags) |= FLAG_SAS_TEX;
      }

      else if ( strCom(line, "FLAG_TEX_INTERP" ) )
      {
        (*flags) |= FLAG_SAS_TEX;
        (*flags) |= FLAG_TEX_INTERP;
      }

      else if ( strCom(line, "FLAG_SIG_GPU" ) || strCom(line, "SIG_GPU" ) )
      {
        (*flags) |= FLAG_SIG_GPU;
      }
      else if ( strCom(line, "FLAG_SIG_CPU" ) || strCom(line, "SIG_CPU" ) )
      {
        (*flags) &= ~FLAG_SIG_GPU;
      }

      else if ( strCom(line, "SS_INMEM_SZ" ) )
      {
        rest                = &line[ strlen("inMemSrchSz")+1];
        sSpec->ssStepSize   = atoi(rest);
      }

      else if ( strCom(line, "FLAG_SS_CPU" 	) || strCom(line, "SS_CPU" 	) )
      {
        (*flags) &= ~FLAG_SS_ALL;
        (*flags) |= FLAG_SS_CPU;

        // CPU Significance
        (*flags) &= ~FLAG_SIG_GPU;

        sSpec->retType &= ~CU_SRT_ALL   ;
        sSpec->retType |= CU_STR_PLN    ;

        if ( (*flags) & FLAG_CUFFT_CB_POW )
        {
          sSpec->retType &= ~CU_TYPE_ALLL   ;
          sSpec->retType |= CU_FLOAT        ;
        }
        else
        {
          sSpec->retType &= ~CU_TYPE_ALLL   ;
          sSpec->retType |= CU_CMPLXF       ;
        }
      }
      else if ( strCom(line, "FLAG_SS_00"  	) || strCom(line, "SS_00"  	) )
      {
        (*flags) &= ~FLAG_SS_ALL;
        (*flags) |= FLAG_SS_00;
        (*flags) |= FLAG_RET_STAGES;
      }
      else if ( strCom(line, "FLAG_SS_10"  	) || strCom(line, "SS_10"  	) )
      {
        (*flags) &= ~FLAG_SS_ALL;
        (*flags) |= FLAG_SS_10;
        (*flags) |= FLAG_RET_STAGES;
      }
      else if ( strCom(line, "FLAG_SS_INMEM") || strCom(line, "SS_INMEM") )
      {
        (*flags) |= FLAG_SS_INMEM;
      }
      else if ( strCom(line, "FLAG_SS_A"    ) || strCom(line, "SS_A"   	) )
      {
        (*flags) &= ~FLAG_SS_ALL;
      }
      else if ( strCom(line, "FLAG_SS "    	) || strCom(line, "SS "     ) )
      {
        char str1[1024];
        char str2[1024];
        int no;
        sscanf(line, "%s %i ", str1, &no  );
        sscanf(line, "%s %s ", str1, str2 );

        if      ( no == 0 )
        {
          (*flags) &= ~FLAG_SS_ALL;
          (*flags) |= FLAG_SS_00;
          (*flags) |= FLAG_RET_STAGES;
        }
        else if ( no == 1 )
        {
          (*flags) &= ~FLAG_SS_ALL;
          (*flags) |= FLAG_SS_10;
          (*flags) |= FLAG_RET_STAGES;
        }
        else if ( strCom(str2, "AA"  ) || strCom(str2, "A"   ) )
        {
          (*flags) &= ~FLAG_SS_ALL;
        }
        else if ( strCom(line, "CPU" ) || strCom(line, "cpu" ) )
        {
          (*flags) &= ~FLAG_SS_ALL;
          (*flags) |= FLAG_SS_CPU;

          // CPU Significance
          (*flags) &= ~FLAG_SIG_GPU;

          sSpec->retType &= ~CU_SRT_ALL   ;
          sSpec->retType |= CU_STR_PLN    ;

          if ( (*flags) & FLAG_CUFFT_CB_POW )
          {
            sSpec->retType &= ~CU_TYPE_ALLL   ;
            sSpec->retType |= CU_FLOAT        ;
          }
          else
          {
            sSpec->retType &= ~CU_TYPE_ALLL   ;
            sSpec->retType |= CU_CMPLXF       ;
          }
        }
        else if ( strCom(line, "INMEM" ) || strCom(line, "inmem" ) )
        {
          (*flags) |= FLAG_SS_INMEM;
        }
        else
        {
          line[flagLen] = 0;
          fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", line, lineno, fName);
        }
      }

      else if ( strCom(line, "SS_Chunk"  ) || strCom(line, "SS_CHUNK"  ) )
      {
        char str1[1024];
        char str2[1024];
        int no;
        int read1 = sscanf(line, "%s %i ", str1, &no  );
        int read2 = sscanf(line, "%s %s ", str1, str2 );

        if ( read1 == 2 )
        {
          sSpec->ssChunk = no;
        }
        else if ( strCom(str2, "AA"  ) || strCom(str2, "A"   ) )
        {
          sSpec->ssChunk = 0;
        }
        else
        {
          line[flagLen] = 0;
          fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", line, lineno, fName);
        }
      }
      else if ( strCom(line, "SS_Slices" ) || strCom(line, "SS_SLICES" ) )
      {
        char str1[1024];
        char str2[1024];
        int no;
        int read1 = sscanf(line, "%s %i ", str1, &no  );
        int read2 = sscanf(line, "%s %s ", str1, str2 );

        if ( read1 == 2 )
        {
          sSpec->ssSlices = no;
        }
        else if ( strCom(str2, "AA"  ) || strCom(str2, "A"   ) )
        {
          sSpec->ssSlices = 0;
        }
        else
        {
          line[flagLen] = 0;
          fprintf(stderr, "ERROR: Found unknown value for %s on line %i of %s.\n", line, lineno, fName);
        }
      }

      else if ( strCom(line, "CU_CAND_ARR"  ) || strCom(line, "CAND_ARR"  ) )
      {
        // Return type
        sSpec->retType &= ~CU_TYPE_ALLL ;
        sSpec->retType &= ~CU_SRT_ALL   ;

        sSpec->retType |= CU_POWERZ_S   ;
        sSpec->retType |= CU_STR_ARR    ;

        // Candidate type
        sSpec->cndType &= ~CU_TYPE_ALLL ;
        sSpec->cndType &= ~CU_SRT_ALL   ;

        sSpec->cndType |= CU_CANDFULL   ;
        sSpec->cndType |= CU_STR_ARR    ;
      }
      else if ( strCom(line, "CU_CAND_LST"  ) || strCom(line, "CAND_LST"  ) )
      {
        // Return type
        sSpec->retType &= ~CU_TYPE_ALLL ;
        sSpec->retType &= ~CU_SRT_ALL   ;

        sSpec->retType |= CU_POWERZ_S   ;
        sSpec->retType |= CU_STR_ARR    ;

        // Candidate type
        sSpec->cndType &= ~CU_TYPE_ALLL ;
        sSpec->cndType &= ~CU_SRT_ALL   ;

        sSpec->cndType |= CU_CANDFULL   ;
        sSpec->cndType |= CU_STR_LST    ;
      }
      else if ( strCom(line, "CU_CAND_QUAD" ) || strCom(line, "CAND_QUAD" ) )
      {
        // Candidate type
        sSpec->cndType &= ~CU_TYPE_ALLL ;
        sSpec->cndType &= ~CU_SRT_ALL   ;

        sSpec->cndType |= CU_POWERZ_S   ;
        sSpec->cndType |= CU_STR_QUAD   ;
      }

      else if ( strCom(line, "FLAG_HALF" 	  ) )
      {
#if CUDA_VERSION >= 7050
        (*flags) |=  FLAG_HALF;
#else
        (*flags) &= ~FLAG_HALF;

        line[flagLen] = 0;
        fprintf(stderr,"WARNING: Half precision can only be used with CUDA 7.5 or later! Reverting to single precision. (FLAG: %s line %i in %s)\n", line, lineno, fName);
#endif
      }
      else if ( strCom(line, "FLAG_SINGLE" 	) )
      {
        (*flags) &= ~FLAG_HALF;
      }
      else if ( strCom(line, "FLAG_DOUBLE"  ) )
      {
        fprintf(stderr,"ERROR: Cannot sore in-mem plane as double! Defaulting to float.\n");
        (*flags) &= ~FLAG_HALF;
      }

      else if ( strCom(line, "FLAG_RET_STAGES" ) )
      {
        (*flags) |= FLAG_RET_STAGES;
      }
      else if ( strCom(line, "FLAG_RETURN_FINAL" ) )
      {
        (*flags) &= ~FLAG_RET_STAGES;
      }

      else if ( strCom(line, "FLAG_RET_ARR" ) )
      {
        sSpec->retType &= ~CU_SRT_ALL   ;
        sSpec->retType |= CU_STR_ARR    ;
      }
      else if ( strCom(line, "FLAG_RET_PLN" ) )
      {
        sSpec->retType &= ~CU_SRT_ALL   ;
        sSpec->retType |= CU_STR_PLN    ;
      }

      else if ( strCom(line, "FLAG_STORE_ALL" ) )
      {
        (*flags) |= FLAG_STORE_ALL;
      }

      else if ( strCom(line, "FLAG_THREAD" ) )
      {
        (*flags) |= FLAG_THREAD;
      }
      else if ( strCom(line, "FLAG_SEQ" ) )
      {
        (*flags) &= ~FLAG_THREAD;
      }

      else if ( strCom(line, "FLAG_STK_UP" ) )
      {
        (*flags) |= FLAG_STK_UP;
      }
      else if ( strCom(line, "FLAG_STK_DOWN" ) )
      {
        (*flags) &= ~FLAG_STK_UP;
      }

      else if ( strCom(line, "FLAG_CONV" ) )
      {
        (*flags) |= FLAG_CONV;
      }
      else if ( strCom(line, "FLAG_SEP" ) )
      {
        (*flags) &= ~FLAG_CONV;
      }

      else if ( strCom(line, "FLAG_STORE_EXP" ) )
      {
        (*flags) |= FLAG_STORE_EXP;
      }

      else if ( strCom(line, "FLAG_RAND_1" ) || strCom(line, "RAND_1" ) )
      {
        (*flags) |= FLAG_RAND_1;
      }


      else if ( strCom(line, "FLAG_DBG_SYNCH" ) )
      {
        (*flags) |= FLAG_SYNCH;
      }
      else if ( strCom(line, "FLAG_DBG_TIMING" ) )
      {
        //(*flags) |= FLAG_SYNCH; // Timing relies on synchronous search
        (*flags) |= FLAG_TIME;
      }

      else if ( strCom(line, "FLAG" ) || strCom(line, "CU_" ) )
      {
        line[flagLen] = 0;
        fprintf(stderr, "ERROR: Found unknown flag %s on line %i of %s.\n", line, lineno, fName);
      }

      else if ( strCom(line, "cuMedianBuffSz" ) )             // The size of the sub sections to use in the cuda median selection algorithm
      {
        rest = &line[ strlen("cuMedianBuffSz")+1];
        cuMedianBuffSz = atoi(rest);
      }

      else if ( strCom(line, "globalFloat01" ) )
      {
        rest = &line[ strlen("globalFloat01")+1];
        globalFloat01 = atof(rest);
      }
      else if ( strCom(line, "globalFloat02" ) )
      {
        rest = &line[ strlen("globalFloat02")+1];
        globalFloat02 = atof(rest);
      }
      else if ( strCom(line, "globalFloat03" ) )
      {
        rest = &line[ strlen("globalFloat03")+1];
        globalFloat03 = atof(rest);
      }
      else if ( strCom(line, "globalFloat04" ) )
      {
        rest = &line[ strlen("globalFloat04")+1];
        globalFloat04 = atof(rest);
      }
      else if ( strCom(line, "globalFloat05" ) )
      {
        rest = &line[ strlen("globalFloat05")+1];
        globalFloat05 = atof(rest);
      }

      else if ( strCom(line, "globalInt01" ) )
      {
        rest = &line[ strlen("globalInt01")+1];
        globalInt01 = atoi(rest);
      }
      else if ( strCom(line, "globalInt02" ) )
      {
        rest = &line[ strlen("globalInt02")+1];
        globalInt02 = atoi(rest);
      }
      else if ( strCom(line, "globalInt03" ) )
      {
        rest = &line[ strlen("globalInt03")+1];
        globalInt03 = atoi(rest);
      }
      else if ( strCom(line, "globalInt04" ) )
      {
        rest = &line[ strlen("globalInt04")+1];
        globalInt04 = atoi(rest);
      }
      else if ( strCom(line, "globalInt05" ) )
      {
        rest = &line[ strlen("globalInt05")+1];
        globalInt05 = atoi(rest);
      }

      // Optimisation vars
      else if ( strCom(line, "optpln01" ) )
      {
        rest      = &line[ strlen("optpln01")+1];
        optpln01  = atoi(rest);
      }
      else if ( strCom(line, "optpln02" ) )
      {
        rest      = &line[ strlen("optpln02")+1];
        optpln02  = atoi(rest);
      }
      else if ( strCom(line, "optpln03" ) )
      {
        rest      = &line[ strlen("optpln03")+1];
        optpln03  = atoi(rest);
      }
      else if ( strCom(line, "optpln04" ) )
      {
        rest      = &line[ strlen("optpln04")+1];
        optpln04  = atoi(rest);
      }
      else if ( strCom(line, "optpln05" ) )
      {
        rest      = &line[ strlen("optpln05")+1];
        optpln05  = atoi(rest);
      }
      else if ( strCom(line, "optpln06" ) )
      {
        rest      = &line[ strlen("optpln06")+1];
        optpln06  = atoi(rest);
      }

      else if ( strCom(line, "downScale" ) )
      {
        rest      = &line[ strlen("downScale")+1];
        downScale = atof(rest);
      }

      else if ( strCom(line, "optSz01" ) )
      {
        rest      = &line[ strlen("optSz01")+1];
        optSz01   = atof(rest);
      }
      else if ( strCom(line, "optSz02" ) )
      {
        rest      = &line[ strlen("optSz02")+1];
        optSz02   = atof(rest);
      }
      else if ( strCom(line, "optSz04" ) )
      {
        rest      = &line[ strlen("optSz04")+1];
        optSz04   = atof(rest);
      }
      else if ( strCom(line, "optSz08" ) )
      {
        rest      = &line[ strlen("optSz08")+1];
        optSz08   = atof(rest);
      }
      else if ( strCom(line, "optSz16" ) )
      {
        rest      = &line[ strlen("optSz16")+1];
        optSz16   = atof(rest);
      }

      else if ( strCom(line, "pltOpt"  ) || strCom(line, "PLT_OPT" ) )
      {
        pltOpt    = 1;
      }

      else if ( strCom(line, "UNOPT" ) )
      {
        useUnopt    = 1;
      }

      else if ( strCom(line, "DBG_LEV" ) )
      {
        rest      = &line[ strlen("DBG_LEV")+1];
        msgLevel  = atoi(rest);
      }


      else if ( strCom(line, "skpOpt"  ) || strCom(line, "SKP_OPT" ) || strCom(line, "FLAG_DBG_SKIP_OPT" ) )
      {
        skpOpt  = 1;
      }

      else if ( strCom(line, "#" ) || ll == 1 )
      {
        // Comment line !
      }

      else
      {
        line[flagLen] = 0;
        fprintf(stderr, "ERROR: Found unknown flag \"%s\" on line %i of %s.\n", line, lineno, fName);
      }
    }

    fclose (file);
  }
  else
  {
    printf("Unable to read GPU accel settings from %s\n", fName);
  }
}

searchSpecs readSrchSpecs(Cmdline *cmd, accelobs* obs)
{
  searchSpecs sSpec;
  memset(&sSpec, 0, sizeof(sSpec));

  CUDA_SAFE_CALL(hipGetLastError(), "Entering readSrchSpecs.");

  // Defaults for accel search
  sSpec.flags         |= FLAG_RET_STAGES  ;
  sSpec.flags         |= FLAG_ITLV_ROW    ;

#ifndef DEBUG
  sSpec.flags         |= FLAG_THREAD      ; 	// Multithreading really slows down debug so only turn it on by default for release mode, NOTE: This can be over ridden in the defaults file
#endif

#if CUDA_VERSION >= 6050
  sSpec.flags         |= FLAG_CUFFT_CB_POW; 	// CUFFT callback to calculate powers, very efficient so on by default
#endif

#if CUDA_VERSION >= 7050
  sSpec.flags         |= FLAG_HALF;
#endif

  if ( obs->inmem )
  {
    sSpec.flags       |= FLAG_SS_INMEM;
  }

  sSpec.cndType       |= CU_CANDFULL    ;   	// Candidate data type - CU_CANDFULL this should be the default as it has all the needed data
  sSpec.cndType       |= CU_STR_ARR     ;   	// Candidate storage structure - CU_STR_ARR    is generally the fastest

  sSpec.retType       |= CU_POWERZ_S    ;   	// Return type
  sSpec.retType       |= CU_STR_ARR     ;   	// Candidate storage structure

  sSpec.fftInf.fft    = obs->fft;
  sSpec.fftInf.nor    = obs->numbins;
  sSpec.fftInf.rlo    = obs->rlo;
  sSpec.fftInf.rhi    = obs->rhi;

  sSpec.noHarmStages  = obs->numharmstages;
  sSpec.zMax          = obs->zhi;
  sSpec.sigma         = cmd->sigma;
  sSpec.pWidth        = cmd->width;

  readAccelDefalts(&sSpec);

  if ( sSpec.flags & (FLAG_SS_10 /*| FLAG_SS_20 | FLAG_SS_30 */ ) )
  {
    // Round the first bin to a multiple of the number of harmonics this is needed in the s&s kernel
    sSpec.fftInf.rlo  = floor(obs->rlo/(float)cmd->numharm)*cmd->numharm;
  }

  return sSpec;
}

/** Create multiplication kernel and allocate memory for planes on all devices  .
 *
 * Create the kernels on the first device and then copy it to all others
 *
 * @param sSrch     A pointer to the search structure
 *
 * @return
 */
void initPlanes(cuSearch* sSrch )
{
  infoMSG(2,1,"Create all planes.\n");

  sSrch->pInf = new cuPlnInfo;
  memset(sSrch->pInf, 0, sizeof(cuPlnInfo));

  CUDA_SAFE_CALL(hipGetLastError(), "Entering initCuAccel.");

  FOLD // Create the primary stack on each device, this contains the kernel  .
  {
    infoMSG(2,2,"Create the primary stack/kernel on each device\n");

    nvtxRangePush("Initialise Kernels");

    sSrch->pInf->kernels = (cuFFdotBatch*)malloc(sSrch->gSpec->noDevices*sizeof(cuFFdotBatch));

    int added;
    cuFFdotBatch* master = NULL;

    for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) // Loop over devices  .
    {
      added = initKernel(&sSrch->pInf->kernels[sSrch->pInf->noDevices], master, sSrch, dev );

      if ( added && !master ) // This was the first batch so it is the master
      {
        master = &sSrch->pInf->kernels[0];
      }

      if ( added )
      {
        sSrch->pInf->noBatches += added;
        sSrch->pInf->noDevices++;
      }
      else
      {
        sSrch->gSpec->noDevBatches[dev] = 0;
        fprintf(stderr, "ERROR: failed to set up a kernel on device %i, trying to continue... \n", sSrch->gSpec->devId[dev]);
      }
    }

    nvtxRangePop();

    if ( sSrch->pInf->noDevices <= 0 ) // Check if we got any devices  .
    {
      fprintf(stderr, "ERROR: Failed to set up a kernel on any device. Try -lsgpu to see what devices there are.\n");
      exit (EXIT_FAILURE);
    }

  }

  FOLD // Create planes for calculations  .
  {
    infoMSG(2,2,"Create planes\n");

    nvtxRangePush("Initialise Batches");

    sSrch->pInf->noSteps       = 0;
    sSrch->pInf->batches       = (cuFFdotBatch*)malloc(sSrch->pInf->noBatches*sizeof(cuFFdotBatch));
    sSrch->pInf->devNoStacks   = (int*)malloc(sSrch->gSpec->noDevices*sizeof(int));
    sSrch->pInf->h_stackInfo   = (stackInfo**)malloc(sSrch->gSpec->noDevices*sizeof(stackInfo*));

    memset(sSrch->pInf->devNoStacks,0,sSrch->gSpec->noDevices*sizeof(int));

    int bNo = 0;
    int ker = 0;

    for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) // Loop over devices  .
    {
      int noSteps = 0;
      if ( sSrch->gSpec->noDevBatches[dev] > 0 )
      {
        int firstBatch = bNo;

        for ( int batch = 0 ; batch < sSrch->gSpec->noDevBatches[dev]; batch++ )
        {
          infoMSG(3,3,"Initialise batch %02i\n", bNo );

          noSteps = initBatch(&sSrch->pInf->batches[bNo], &sSrch->pInf->kernels[ker], batch, sSrch->gSpec->noDevBatches[dev]-1);

          if ( noSteps == 0 )
          {
            if ( batch == 0 )
            {
              fprintf(stderr, "ERROR: Failed to create at least one batch on device %i.\n", sSrch->pInf->kernels[dev].device);
            }
            break;
          }
          else
          {
            sSrch->pInf->noSteps           += noSteps;
            sSrch->pInf->devNoStacks[dev]  += sSrch->pInf->batches[bNo].noStacks;
            bNo++;
          }
        }

        int noStacks = sSrch->pInf->devNoStacks[dev] ;
        if ( noStacks )
        {
          infoMSG(3,3,"Initialise constant memory for stacks\n" );

          sSrch->pInf->h_stackInfo[dev] = (stackInfo*)malloc(noStacks*sizeof(stackInfo));
          int idx = 0;

          // Set the values of the host data structures
          for (int batch = firstBatch; batch < bNo; batch++)
          {
            idx += setStackInfo(&sSrch->pInf->batches[batch], sSrch->pInf->h_stackInfo[dev], idx);
          }

          if ( idx != noStacks )
          {
            fprintf (stderr,"ERROR: in %s line %i, The number of stacks on device do not match.\n.",__FILE__, __LINE__);
          }
          else
          {
            setConstStkInfo(sSrch->pInf->h_stackInfo[dev], idx, sSrch->pInf->batches->stacks->initStream);
          }
        }

        ker++;
      }
    }

    if ( bNo != sSrch->pInf->noBatches )
    {
      fprintf(stderr, "WARNING: Number of batches created does not match the number anticipated.\n");
      sSrch->pInf->noBatches = bNo;
    }

    nvtxRangePop();
  }
}

/** Create multiplication kernel and allocate memory for planes on all devices  .
 *
 * Create the kernels on the first device and then copy it to all others
 *
 * @param sSrch     A pointer to the search structure
 *
 * @return
 */
void initOptimisers(cuSearch* sSrch )
{
  sSrch->oInf = new cuOptInfo;
  memset(sSrch->oInf, 0, sizeof(cuOptInfo));

  CUDA_SAFE_CALL(hipGetLastError(), "Entering initOptimisers.");

  FOLD // Create the primary stack on each device, this contains the kernel  .
  {
    nvtxRangePush("Initialise Optimisers");

    sSrch->oInf->noOpts = 0;

    for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) // Loop over devices  .
    {
      if ( sSrch->gSpec->noDevOpt[dev] > 0 )
      {
        sSrch->oInf->noOpts+=sSrch->gSpec->noDevOpt[dev];
      }
    }

    sSrch->oInf->opts = (cuOptCand*)malloc(sSrch->oInf->noOpts*sizeof(cuOptCand));
    memset(sSrch->oInf->opts, 0, sSrch->oInf->noOpts*sizeof(cuOptCand));

    int idx = 0;
    for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) // Loop over devices  .
    {
      for ( int oo = 0 ; oo < sSrch->gSpec->noDevOpt[dev]; oo++ )
      {
        // Setup some basic info
        sSrch->oInf->opts[idx].pIdx     = idx;
        sSrch->oInf->opts[idx].device   = sSrch->gSpec->devId[dev];

        initOptCand(sSrch, &sSrch->oInf->opts[idx], dev );
        idx++;
      }
    }

    nvtxRangePop();
  }
}

void freeAccelGPUMem(cuPlnInfo* aInf)
{
  infoMSG(2,0,"FreeAccelGPUMem\n");

  FOLD // Free planes  .
  {
    for ( int batch = 0 ; batch < aInf->noBatches; batch++ )  // Batches
    {
      infoMSG(2,1,"freeBatchGPUmem %i\n", batch);

      freeBatchGPUmem(&aInf->batches[batch]);
    }
  }

  FOLD // Free kernels  .
  {
    for ( int dev = 0 ; dev < aInf->noDevices; dev++)         // Loop over devices
    {
      infoMSG(2,1,"freeKernelGPUmem device: %i\n", dev);

      freeKernelGPUmem(&aInf->kernels[dev]);
    }
  }
}

void freeCuAccel(cuPlnInfo* mInf)
{
  if ( mInf )
  {
    FOLD // Free planes  .
    {
      for ( int batch = 0 ; batch < mInf->noBatches; batch++ )  // Batches
      {
        freeBatch(&mInf->batches[batch]);
      }
    }

    FOLD // Free kernels  .
    {
      for ( int dev = 0 ; dev < mInf->noDevices; dev++)  // Loop over devices
      {
        freeKernel(&mInf->kernels[dev] );
      }
    }

    freeNull(mInf->batches);
    freeNull(mInf->kernels);

    //    for ( int i = 0; i < MAX_GPUS; i++ )
    //      freeNull(mInf->name[i]);

    freeNull(mInf->devNoStacks);

    FOLD // Stack infos  .
    {
      for ( int dev = 0 ; dev < mInf->noDevices; dev++)  // Loop over devices
      {
        freeNull(mInf->h_stackInfo[dev]);
      }

      freeNull(mInf->h_stackInfo);
    }
  }
}

void intSrchThrd(cuSearch* srch)
{
  //if ( srch->sSpec->flags & FLAG_THREAD )
  {
    resThrds* tInf = srch->threasdInfo;

    if ( !tInf )
    {
      tInf     = new resThrds;
      memset(tInf, 0, sizeof(cuSearch));
    }

    if (pthread_mutex_init(&tInf->candAdd_mutex, NULL))
    {
      printf("Unable to initialise a mutex.\n");
      exit(EXIT_FAILURE);
    }

    if (sem_init(&tInf->running_threads, 0, 0))
    {
      printf("Could not initialise a semaphore\n");
      exit(EXIT_FAILURE);
    }

    srch->threasdInfo = tInf;
  }
}

cuSearch* initSearchInf(searchSpecs* sSpec, gpuSpecs* gSpec, cuSearch* srch)
{
  infoMSG(2,1,"Initialise search data structure\n");

  bool same   = true;

  CUDA_SAFE_CALL(hipGetLastError(), "Entering initCuSearch.");

  if ( srch ) 	                    // Check if the search values have been pre-initialised  .
  {
    if ( srch->noHarmStages != sSpec->noHarmStages )
    {
      same = false;
      // ERROR recreate everything
    }

    if ( srch->pInf )
    {
      if ( srch->pInf->kernels->hInfos->zmax != sSpec->zMax )
      {
        same = false;
        // Have to recreate
      }

      presto_interp_acc accuracy = LOWACC;
      if ( sSpec->flags & FLAG_KER_HIGH )
        accuracy = HIGHACC;

      if ( srch->pInf->kernels->accelLen != optAccellen(sSpec->pWidth,sSpec->zMax, accuracy) )
      {
        same = false;
        // Have to recreate
      }

      if ( !same )
      {
        fprintf(stderr,"ERROR: Call to %s with differing GPU search parameters. Will have to allocate new GPU memory and kernels.\n      NB: Not freeing the old memory!", __FUNCTION__);
      }
      else
      {
        // NB Assuming the GPU specks are all the same
      }
    }
  }

  if ( !srch || same == false)      // Create a new search data structure  .
  {
    infoMSG(2,2,"Create a new search data structure\n");

    srch = new cuSearch;
    memset(srch, 0, sizeof(cuSearch));

    srch->noHarmStages    = sSpec->noHarmStages;
    srch->noGenHarms      = ( 1<<(srch->noHarmStages-1) );
    srch->noSrchHarms     = ( 1<<(srch->noHarmStages-1) );
    srch->sIdx            = (int*)malloc(srch->noGenHarms * sizeof(int));
    srch->powerCut        = (float*)malloc(srch->noHarmStages * sizeof(float));
    srch->numindep        = (long long*)malloc(srch->noHarmStages * sizeof(long long));
  }
  else
  {
    infoMSG(2,2,"Using the existing search data structure\n");
  }

  srch->sSpec             = sSpec;
  srch->gSpec             = gSpec;

  FOLD // Calculate power cutoff and number of independent values  .
  {
    infoMSG(3,2,"Calculate power cutoff and number of independent values\n");

    // Calculate appropriate z-max
    if ( sSpec->zMax % ACCEL_DZ )
      sSpec->zMax = (sSpec->zMax / ACCEL_DZ + 1) * ACCEL_DZ;

    int numz = (sSpec->zMax / ACCEL_DZ) * 2 + 1;

    FOLD // Calculate power cutoff and number of independent values  .
    {
      for (int ii = 0; ii < srch->noHarmStages; ii++)
      {
        if ( sSpec->zMax == 1 )
        {
          srch->numindep[ii]  = (sSpec->fftInf.rhi - sSpec->fftInf.rlo) / srch->noGenHarms;
        }
        else
        {
          srch->numindep[ii]  = (sSpec->fftInf.rhi - sSpec->fftInf.rlo) * (numz + 1) * ( ACCEL_DZ / 6.95) / (double)(1<<ii);
        }

        // Power cutoff
        // TODO: Check if using half precision may affect this
        srch->powerCut[ii]  = power_for_sigma(sSpec->sigma, (1<<ii), srch->numindep[ii]);
      }
    }
  }

  FOLD // Set up the CPU threading  .
  {
    infoMSG(3,2,"Set up the CPU threading\n");

    intSrchThrd(srch);
  }

  return srch;
}

cuSearch* initCuKernels(searchSpecs* sSpec, gpuSpecs* gSpec, cuSearch* srch)
{
  infoMSG(1,0,"Initialise CU search data structures\n");

  if ( !srch )
  {
    srch = initSearchInf(sSpec, gSpec, srch);
  }

  if ( !srch->pInf )
  {
    initPlanes( srch ); // This initialises the plane info
  }
  else
  {
    // TODO: Do a whole bunch of checks here!
    fprintf(stderr, "ERROR: %s has not been set up to handle a pre-initialised memory info data structure.\n", __FUNCTION__);
    exit(EXIT_FAILURE);
  }

  return srch;
}

cuSearch* initCuOpt(searchSpecs* sSpec, gpuSpecs* gSpec, cuSearch* srch)
{
  if ( !srch )
    srch = initSearchInf(sSpec, gSpec, srch);

  if ( !srch->oInf )
  {
    initOptimisers( srch );
  }
  else
  {
    // TODO: Do a whole bunch of checks here!
    fprintf(stderr, "ERROR: %s has not been set up to handle a pre-initialised memory info data structure.\n", __FUNCTION__);
    exit(EXIT_FAILURE);
  }

  return srch;
}

void freeCuSearch(cuSearch* srch)
{
  if (srch)
  {
    if ( srch->pInf )
      freeCuAccel(srch->pInf);

    freeNull(srch->sIdx);
    freeNull(srch->powerCut);
    freeNull(srch->numindep);

    freeNull(srch)
  }
}

void accelMax(cuSearch* srch)
{
  /*
  bool newKer = false;


  if ( aInf == NULL )
  {
    newKer = true;
    aInf = oneDevice(-1, fftinf, numharmstages, zMax, 8, 2, 4, CU_CAND_ARR | FLAG_STORE_EXP, CU_FLOAT, CU_FLOAT, (void*)powers );
  }

  master = &srch->mInf->kernels[0];
   */

  cuFFdotBatch* master   = NULL;    // The first kernel stack created
  master = srch->pInf->kernels;

#ifdef WITHOMP
  omp_set_num_threads(srch->pInf->noBatches);
#endif

  int ss = 0;
  int maxxx = ( srch->sSpec->fftInf.rhi - srch->sSpec->fftInf.rlo ) / (float)( master->accelLen * ACCEL_DR ) ; /// The number of planes we can work with

  if ( maxxx < 0 )
    maxxx = 0;

  int firstStep = 0;

#ifndef DEBUG
#pragma omp parallel
#endif
  FOLD
  {
#ifdef WITHOMP
    int tid = omp_get_thread_num();
#else
    int tid = 0;
#endif

    cuFFdotBatch* trdBatch = &srch->pInf->batches[tid];

    double*  startrs = (double*)malloc(sizeof(double)*trdBatch->noSteps);
    double*  lastrs  = (double*)malloc(sizeof(double)*trdBatch->noSteps);
    size_t rest = trdBatch->noSteps;

    setDevice(trdBatch->device) ;

    while ( ss < maxxx )
    {
#pragma omp critical
      {
        firstStep = ss;
        ss       += trdBatch->noSteps;
        printf("\r   Step %07i of %-i %7.2f%%      \r", firstStep, maxxx,  firstStep/(float)maxxx*100);
        std::cout.flush();
      }

      if ( firstStep >= maxxx )
      {
        break;
      }

      for ( int step = 0; step < trdBatch->noSteps ; step ++)
      {
        if ( step < rest )
        {
          startrs[step] = srch->sSpec->fftInf.rlo   + (firstStep+step) * ( master->accelLen * ACCEL_DR );
          lastrs[step]  = startrs[step] + master->accelLen * ACCEL_DR - ACCEL_DR;
        }
        else
        {
          startrs[step] = 0 ;
          lastrs[step]  = 0 ;
        }
      }
      //max_ffdot_planeCU(trdBatch, startrs, lastrs, 1, (fcomplexcu*)fftinf->fft, numindep, powers );
    }

    for ( int step = 0; step < trdBatch->noSteps ; step ++)
    {
      startrs[step] = 0;
      lastrs[step]  = 0;
    }

    // Finish searching the planes, this is required because of the out of order asynchronous calls
    for ( int pln = 0 ; pln < 2; pln++ )
    {
      //max_ffdot_planeCU(trdBatch, startrs, lastrs, 1,(fcomplexcu*)fftinf->fft, numindep, powers );

      //trdBatch->mxSteps = rest;
    }
    printf("\n");
  }

  /*
  printf("Free planes \n");

  FOLD // Free planes
  {
    for ( int pln = 0 ; pln < nPlanes; pln++ )  // Batches
    {
      freeBatch(planesj[pln]);
    }
  }

  printf("Free kernels \n");

  FOLD // Free kernels
  {
    for ( int dev = 0 ; dev < noKers; dev++)  // Loop over devices
    {
      freeHarmonics(&kernels[dev], master, (void*)powers );
    }
  }
   */

#ifndef DEBUG
  //printCands("GPU_Cands.csv", candsGPU);
#endif
}

void plotPlanes(cuFFdotBatch* batch)
{
  //#ifdef CBL
  //  printf("\n Creating data sets...\n");
  //
  //  nDarray<2, float>gpuCmplx [batch->noSteps][batch->noHarms];
  //  nDarray<2, float>gpuPowers[batch->noSteps][batch->noHarms];
  //  for ( int si = 0; si < batch->noSteps ; si ++)
  //  {
  //    for (int harm = 0; harm < batch->noGenHarms; harm++)
  //    {
  //      cuHarmInfo *hinf  = &batch[0].hInfos[harm];
  //
  //      gpuCmplx[si][harm].addDim(hinf->width*2, 0, hinf->width);
  //      gpuCmplx[si][harm].addDim(hinf->height, -hinf->zmax, hinf->zmax);
  //      gpuCmplx[si][harm].allocate();
  //
  //      gpuPowers[si][harm].addDim(hinf->width, 0, hinf->width);
  //      gpuPowers[si][harm].addDim(hinf->height, -hinf->zmax, hinf->zmax);
  //      gpuPowers[si][harm].allocate();
  //    }
  //  }
  //
  //  for ( int step = 0; step < batch->noSteps ; step ++)
  //  {
  //    for ( int stack = 0 ; stack < batch->noStacks; stack++ )
  //    {
  //      for (int harm = 0; harm < batch->noGenHarms; harm++)
  //      {
  //        cuHarmInfo   *cHInfo  = &batch->hInfos[harm];
  //        cuFfdotStack *cStack  = &batch->stacks[cHInfo->stackNo];
  //        rVals* rVal           = &batch->rArrays[batch->rActive][step][harm];
  //
  //        for( int y = 0; y < cHInfo->height; y++ )
  //        {
  //
  //          fcomplexcu *cmplxData;
  //          float *powers;
  //
  //          if ( batch->flag & FLAG_ITLV_ROW )
  //          {
  //            cmplxData = &batch->d_planeMult[  (y*batch->noSteps + step)*cStack->strideCmplx ];
  //            powers    = &batch->d_planePowr[ ((y*batch->noSteps + step)*cStack->strideFloat + cHInfo->halfWidth * 2 ) ];
  //          }
  //          else
  //          {
  //            cmplxData = &batch->d_planeMult[  (y + step*cHInfo->height)*cStack->strideCmplx ];
  //            powers    = &batch->d_planePowr[ ((y + step*cHInfo->height)*cStack->strideFloat  + cHInfo->halfWidth * 2 ) ];
  //          }
  //
  //          cmplxData += cHInfo->halfWidth*2;
  //          //CUDA_SAFE_CALL(hipMemcpyAsync(gpuCmplx[step][harm].getP(0,y), cmplxData, (cHInfo->width-2*2*cHInfo->halfWidth)*2*sizeof(float), hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
  //          //CUDA_SAFE_CALL(hipMemcpyAsync(gpuCmplx[step][harm].getP(0,y), cmplxData, (cPlane->numrs[step])*2*sizeof(float), hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
  //          CUDA_SAFE_CALL(hipMemcpyAsync(gpuCmplx[step][harm].getP(0,y), cmplxData, (rVal->numrs)*2*sizeof(float), hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
  //          if ( batch->flag & FLAG_CUFFT_CB_OUT )
  //          {
  //            //CUDA_SAFE_CALL(hipMemcpyAsync(gpuPowers[step][harm].getP(0,y), powers, (cPlane->numrs[step])*sizeof(float),   hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
  //            CUDA_SAFE_CALL(hipMemcpyAsync(gpuPowers[step][harm].getP(0,y), powers, (rVal->numrs)*sizeof(float),   hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
  //            /*
  //            for( int jj = 0; jj < plan->numrs[step]; jj++)
  //            {
  //              float *add = gpuPowers[step][harm].getP(jj*2+1,y);
  //              gpuPowers[step][harm].setPoint<ARRAY_SET>(add, 0);
  //            }
  //             */
  //          }
  //        }
  //      }
  //    }
  //  }
  //#else
  //  fprintf(stderr,"ERROR: Not compiled with debug libraries.\n");
  //#endif
}

void printBitString( int64_t val )
{
  printf("Value %015ld : ", val );

  for ( int i = 0; i < 64; i++)
  {
    if( val & ( 1ULL << (63-i) ) )
      printf("1");
    else
      printf("0");
  }
  printf("\n");
}

void printCommandLine(int argc, char *argv[])
{
  printf("Command:\t");

  for ( int i =0; i < argc; i ++ )
  {
    printf("%s ",argv[i]);
  }
  printf("\n");
}

void writeLogEntry(char* fname, accelobs* obs, cuSearch* cuSrch, long long prepTime, long long cpuKerTime, long long cupTime, long long gpuKerTime, long long gpuTime, long long optTime, long long cpuOptTime, long long gpuOptTime)
{
#ifdef CBL
  searchSpecs* sSpec;         ///< Specifications of the search
  cuPlnInfo* mInf;            ///< The allocated Device and host memory and data structures to create planes including the kernels
  cuFFdotBatch* batch;

  sSpec         = cuSrch->sSpec;
  mInf          = cuSrch->pInf;

  if ( !cuSrch || !sSpec || !mInf  )
    return;

  batch         = cuSrch->pInf->batches;
  double noRR   = sSpec->fftInf.rhi - sSpec->fftInf.rlo;

  char hostname[1024];
  gethostname(hostname, 1024);

  Logger* cvsLog = new Logger(fname, 1);
  cvsLog->sedCsvDeliminator('\t');

  // Get the current time
  time_t rawtime;
  tm* ptm;
  time(&rawtime);
  ptm = gmtime(&rawtime);

  FOLD // Basics  .
  {
    cvsLog->csvWrite("Width",     "#", "%4i",     sSpec->pWidth);
    cvsLog->csvWrite("Stride",    "#", "%5i",     batch->stacks->strideCmplx);
    cvsLog->csvWrite("A-Len",     "#", "%5i",     batch->accelLen);

    cvsLog->csvWrite("Z max",     "#", "%03i",    sSpec->zMax);

    cvsLog->csvWrite("Devices",   "#", "%2i",     mInf->noDevices);
    cvsLog->csvWrite("GPU",       "#", "%2i",     batch->device);

    cvsLog->csvWrite("Har",       "#", "%2li",    cuSrch->noGenHarms);
    cvsLog->csvWrite("Plns",      "#", "%2i",     batch->stacks->noInStack);

    cvsLog->csvWrite("Obs N",     "#", "%7.3f",   obs->N  * 1e-6);
    cvsLog->csvWrite("R bins",    "#", "%7.3f",   noRR    * 1e-6);

    cvsLog->csvWrite("Batches",   "#", "%2i",     mInf->noBatches);

    cvsLog->csvWrite("Steps",     "#", "%2i",     batch->noSteps);

    cvsLog->csvWrite("MU Slices", "#", "%2i",     batch->mulSlices);
    cvsLog->csvWrite("MU Chunk",  "#", "%2i",     batch->mulChunk);

    cvsLog->csvWrite("SS Slices", "#", "%2i",     batch->ssSlices);
    cvsLog->csvWrite("SS Chunk",  "#", "%2i",     batch->ssChunk);

    cvsLog->csvWrite("Sigma",     "#", "%4.2f",   sSpec->sigma);
    cvsLog->csvWrite("Time", "-", "%04i/%02i/%02i %02i:%02i:%02i", 1900 + ptm->tm_year, ptm->tm_mon, ptm->tm_mday, ptm->tm_hour, ptm->tm_min, ptm->tm_sec);
    cvsLog->csvWrite("hostname",  "s", "%s",      hostname);
  }

  FOLD // Flags  .
  {
    if ( batch->flags & FLAG_ITLV_ROW )
      cvsLog->csvWrite("IL",      "flg", "ROW");
    else
      cvsLog->csvWrite("IL",      "flg", "PLN");

    if ( batch->flags & CU_NORM_CPU )
      cvsLog->csvWrite("NORM",    "flg", "CPU");
    else
      cvsLog->csvWrite("NORM",    "flg", "GPU");

    if ( batch->flags & CU_INPT_FFT_CPU )
      cvsLog->csvWrite("Inp FFT", "flg", "CPU");
    else
      cvsLog->csvWrite("Inp FFT", "flg", "GPU");

    if      ( batch->flags & FLAG_MUL_00 )
      cvsLog->csvWrite("MUL",    "flg", "00");
    else if ( batch->flags & FLAG_MUL_11 )
      cvsLog->csvWrite("MUL",    "flg", "11");
    else if ( batch->flags & FLAG_MUL_21 )
      cvsLog->csvWrite("MUL",    "flg", "21");
    else if ( batch->flags & FLAG_MUL_22 )
      cvsLog->csvWrite("MUL",    "flg", "22");
    else if ( batch->flags & FLAG_MUL_23 )
      cvsLog->csvWrite("MUL",    "flg", "23");
    else if ( batch->flags & FLAG_MUL_30 )
      cvsLog->csvWrite("MUL",    "flg", "30");
    else if ( batch->flags & FLAG_MUL_CB )
      cvsLog->csvWrite("MUL",    "flg", "CB");
    else
      cvsLog->csvWrite("MUL",    "flg", "?");

    if      ( batch->flags & FLAG_SS_00  )
      cvsLog->csvWrite("SS",    "flg", "00");
    else if ( batch->flags & FLAG_SS_10  )
      cvsLog->csvWrite("SS",    "flg", "10");
    //    else if ( batch->flag & FLAG_SS_20  )
    //      cvsLog->csvWrite("SS",    "flg", "20");
    //    else if ( batch->flag & FLAG_SS_30  )
    //      cvsLog->csvWrite("SS",    "flg", "30");
    else if ( batch->flags & FLAG_SS_INMEM )
      cvsLog->csvWrite("SS",    "flg", "In-Mem");
    else if ( batch->flags & FLAG_SS_CPU )
      cvsLog->csvWrite("SS",    "flg", "CPU");
    else
      cvsLog->csvWrite("SS",    "flg", "?");

    cvsLog->csvWrite("in-mem ss",  "#", "%i", batch->strideOut );


    cvsLog->csvWrite("CB POW",    "flg", "%i", (bool)(batch->flags & FLAG_CUFFT_CB_POW));
    cvsLog->csvWrite("CB INMEM",  "flg", "%i", (bool)(batch->flags & FLAG_CUFFT_CB_INMEM));

    cvsLog->csvWrite("MUL_TEX",   "flg", "%i", (bool)(batch->flags & FLAG_TEX_MUL));
    cvsLog->csvWrite("SAS_TEX",   "flg", "%i", (bool)(batch->flags & FLAG_SAS_TEX));
    cvsLog->csvWrite("INTERP",    "flg", "%i", (bool)(batch->flags & FLAG_TEX_INTERP));
    if ( batch->flags & FLAG_SIG_GPU )
      cvsLog->csvWrite("SIG",    "flg", "GPU");
    else
      cvsLog->csvWrite("SIG",    "flg", "CPU");

    FOLD // Return details  .
    {
      if      ( batch->retType & CU_STR_ARR   )
        cvsLog->csvWrite("RET",  "strct", "ARR");
      else if ( batch->retType & CU_STR_LST  	)
        cvsLog->csvWrite("RET",  "strct", "LST");
      else if ( batch->retType & CU_STR_QUAD  )
        cvsLog->csvWrite("RET",  "strct", "QUAD");
      else
        cvsLog->csvWrite("RET",  "strct", "?");

      if      ( batch->retType & CU_POWERZ_S  )
        cvsLog->csvWrite("RET",  "type", "POWERZ_S");
      else if ( batch->retType & CU_POWERZ_I  )
        cvsLog->csvWrite("RET",  "type", "CU_POWERZ_I");
      else if ( batch->retType & CU_FLOAT  	  )
        cvsLog->csvWrite("RET",  "type", "FLOAT");
      else if ( batch->retType & CU_CANDFULL  )
        cvsLog->csvWrite("RET",  "type", "CU_CANDFULL");
      else
        cvsLog->csvWrite("RET",  "type", "?");
    }

    FOLD // Candidate storage  .
    {
      if      ( batch->cndType & CU_STR_ARR   )
        cvsLog->csvWrite("CAND",  "strct", "ARR");
      else if ( batch->cndType & CU_STR_LST  	)
        cvsLog->csvWrite("CAND",  "strct", "LST");
      else if ( batch->cndType & CU_STR_QUAD  )
        cvsLog->csvWrite("CAND",  "strct", "QUAD");
      else
        cvsLog->csvWrite("CAND",  "strct", "?");

      if      ( batch->cndType & CU_POWERZ_S  )
        cvsLog->csvWrite("CAND",  "type", "POWERZ_S");
      else if ( batch->cndType & CU_POWERZ_I  )
        cvsLog->csvWrite("CAND",  "type", "CU_POWERZ_I");
      else if ( batch->cndType & CU_FLOAT  	  )
        cvsLog->csvWrite("CAND",  "type", "FLOAT");
      else if ( batch->cndType & CU_CANDFULL  )
        cvsLog->csvWrite("CAND",  "type", "CU_CANDFULL");
      else
        cvsLog->csvWrite("CAND",  "type", "?");
    }

    cvsLog->csvWrite("RET_ALL",     "flg", "%i", (bool)(batch->flags & FLAG_RET_STAGES));
    cvsLog->csvWrite("STR_ALL",     "flg", "%i", (bool)(batch->flags & FLAG_STORE_ALL));
    cvsLog->csvWrite("STR_EXP",     "flg", "%i", (bool)(batch->flags & FLAG_STORE_EXP));

    if      ( batch->cndType & FLAG_KER_HIGH  )
      cvsLog->csvWrite("KER_HW",  "type", "HIGH");
    else
      cvsLog->csvWrite("KER_HW",  "type", "STD");

    cvsLog->csvWrite("KER_MAX",     "flg", "%i", (bool)(batch->flags & FLAG_KER_MAX) );
    cvsLog->csvWrite("KER_CENT",    "flg", "%i", (bool)(batch->flags & FLAG_CENTER)  );
  }

  FOLD // Timing  .
  {
    cvsLog->csvWrite("Prep",      "s", "%9.4f",   prepTime    * 1e-6);
    cvsLog->csvWrite("CPU ker",   "s", "%9.4f",   cpuKerTime  * 1e-6);
    cvsLog->csvWrite("CPU Srch",  "s", "%9.4f",   cupTime     * 1e-6);
    cvsLog->csvWrite("GPU ker",   "s", "%9.4f",   gpuKerTime  * 1e-6);
    cvsLog->csvWrite("GPU Srch",  "s", "%9.4f",   gpuTime     * 1e-6);
    cvsLog->csvWrite("Opt",       "s", "%9.4f",   optTime     * 1e-6);
    cvsLog->csvWrite("CPU Opt",   "s", "%9.4f",   cpuOptTime  * 1e-6);
    cvsLog->csvWrite("GPU Opt",   "s", "%9.4f",   gpuOptTime  * 1e-6);
  }

  FOLD // Advanced Timing  .
  {
    float copyH2DT  = 0;
    float InpNorm   = 0;
    float InpFFT    = 0;
    float multT     = 0;
    float InvFFT    = 0;
    float plnCpy    = 0;
    float ss        = 0;
    float resultT   = 0;
    float copyD2HT  = 0;

    if ( batch->flags & FLAG_TIME )
    {
      for (int batch = 0; batch < cuSrch->pInf->noBatches; batch++)
      {
        float l_copyH2DT  = 0;
        float l_InpNorm   = 0;
        float l_InpFFT    = 0;
        float l_multT     = 0;
        float l_InvFFT    = 0;
        float l_plnCpy    = 0;
        float l_ss        = 0;
        float l_resultT   = 0;
        float l_copyD2HT  = 0;

        for (int stack = 0; stack < cuSrch->pInf->batches[batch].noStacks; stack++)
        {
          cuFFdotBatch* batches = &cuSrch->pInf->batches[batch];
          l_copyH2DT  += batches->copyH2DTime[stack];
          l_InpNorm   += batches->normTime[stack];
          l_InpFFT    += batches->InpFFTTime[stack];
          l_multT     += batches->multTime[stack];
          l_InvFFT    += batches->InvFFTTime[stack];
          l_plnCpy    += batches->copyToPlnTime[stack];
          l_ss        += batches->searchTime[stack];
          l_resultT   += batches->resultTime[stack];
          l_copyD2HT  += batches->copyD2HTime[stack];
        }
        copyH2DT  += l_copyH2DT;
        InpNorm   += l_InpNorm;
        InpFFT    += l_InpFFT;
        multT     += l_multT;
        InvFFT    += l_InvFFT;
        plnCpy    += l_plnCpy;
        ss        += l_ss;
        resultT   += l_resultT;
        copyD2HT  += l_copyD2HT;
      }
    }
    cvsLog->csvWrite("copyH2D",     "ms", "%12.6f", copyH2DT);
    cvsLog->csvWrite("InpNorm",     "ms", "%12.6f", InpNorm);
    cvsLog->csvWrite("InpFFT",      "ms", "%12.6f", InpFFT);
    cvsLog->csvWrite("Mult",        "ms", "%12.6f", multT);
    cvsLog->csvWrite("InvFFT",      "ms", "%12.6f", InvFFT);
    cvsLog->csvWrite("plnCpy",      "ms", "%12.6f", plnCpy);
    cvsLog->csvWrite("Sum & Srch",  "ms", "%12.6f", ss);
    cvsLog->csvWrite("result",      "ms", "%12.6f", resultT);
    cvsLog->csvWrite("copyD2H",     "ms", "%12.6f", copyD2HT);
  }

  cvsLog->csvEndLine();
#endif
}

GSList* getCanidates(cuFFdotBatch* batch, GSList *cands )
{
  //  gridQuadTree<double, float>* qt = (gridQuadTree<double, float>*)(batch->h_candidates) ;
  //  quadNode<double, float>* head = qt->getHead();
  //
  //  qt->update();
  //
  //  printf("GPU search found %li unique values in tree.\n", head->noEls );

  return cands;
}

int hilClimb(candTree* tree, double tooclose = 5)
{
  container* cont = tree->getSmallest();
  //double tooclose = 5;

  while ( cont )
  {
    container* largest = tree->getLargest(cont, tooclose);
    if ( *largest > *cont )
    {
      tree->markForRemoval(cont);
    }
    cont = cont->larger;
  }

  uint rem = tree->removeMarked();
  printf("hilClimb  Removed %6i - %6i remain \n", rem, tree->noVals() );

  return rem;
}

int eliminate_harmonics(candTree* tree, double tooclose = 1.5)
{
  infoMSG(1,2,"Eliminate harmonics");

  int maxharm = 16;
  int numremoved = 0;

  cand* tempCand = new cand;
  container* next;
  container* close;
  container* serch;

  container* lst = tree->getLargest();

  while ( lst )
  {
    cand* candidate = (cand*)lst->data;

    tempCand->power    = candidate->power;
    tempCand->numharm  = candidate->numharm;
    tempCand->r        = candidate->r;
    tempCand->z        = candidate->z;
    tempCand->sig      = candidate->sig;

    // Remove harmonics down
    for (double ii = 1; ii <= maxharm; ii++)
    {
      FOLD // Remove down candidates  .
      {
        tempCand->r  = candidate->r / ii;
        tempCand->z  = candidate->z / ii;
        serch       = contFromCand(tempCand);
        close       =  tree->getAll(serch, tooclose);

        while (close)
        {
          next = close->smaller;

          if ( *close != *lst )
          {
            tree->remove(close);
            numremoved++;
          }

          close = next;
        }
      }

      FOLD // Remove down up  .
      {
        tempCand->r  = candidate->r * ii;
        tempCand->z  = candidate->z * ii;
        serch       = contFromCand(tempCand);
        close       =  tree->getAll(serch, tooclose/**sqrt(ii)*/);

        while (close)
        {
          next = close->smaller;

          if ( *close != *lst )
          {
            tree->remove(close);
            numremoved++;
          }

          close = next;
        }
      }
    }

    for (int ii = 1; ii < 23; ii++)
    {
      tempCand->r  = candidate->r * ratioARR[ii];
      tempCand->z  = candidate->z * ratioARR[ii];
      serch       = contFromCand(tempCand);
      close       =  tree->getAll(serch, tooclose);

      while (close)
      {
        next = close->smaller;

        if ( *close != *lst )
        {
          tree->remove(close);
          numremoved++;
        }

        close = next;
      }
    }

    lst = lst->smaller;
  }

  printf("Harmonics Removed %6i - %6i remain \n", numremoved, tree->noVals() );

  return (numremoved);
}

//GSList *testTest(cuFFdotBatch* batch, GSList *candsGPU)
//{
//  candTree optemised;
//
//  candTree trees[batch->noHarmStages];
//
//  candTree* qt =(candTree*)batch->h_candidates;
//
//  hilClimb(qt, 5);
//  eliminate_harmonics(qt);
//
//  cuOptCand* oPlnPln;
//  oPlnPln   = initOptPln(batch->sInf->sSpec);
//
//  container* cont = qt->getLargest();
//
//  int i = 0;
//
//  while ( cont )
//  {
//    i++;
//    printf("\n");
//    //if ( i == 12 )
//    {
//      cand*   candidate = (cand*)cont->data;
//      cont->flag &= ~OPTIMISED_CONTAINER;
//
//      printf("Candidate %03i  harm: %2i   pow: %9.3f   r: %9.4f  z: %7.4f\n",i, candidate->numharm, candidate->power, candidate->r, candidate->z );
//
//      //
//      //    numharm   = candidate->numharm;
//      //    sig       = candidate->sig;
//      //    rr        = candidate->r;
//      //    zz        = candidate->z;
//      //    poww      = candidate->power;
//      //
//      //    candsGPU  = insert_new_accelcand(candsGPU, poww, sig, numharm, rr, zz, &added );
//
//      //accelcand *cand = new accelcand;
//      //memset(cand, 0, sizeof(accelcand));
//      //cand->power   = candidate->power;
//      //cand->r       = candidate->r;
//      //cand->sigma   = candidate->sig;
//      //cand->z       = candidate->z;
//      //cand->numharm = candidate->numharm;
//
//      //accelcand* cand = create_accelcand(candidate->power, candidate->sig, candidate->numharm, candidate->r, candidate->z);
//
//      //candsGPU = insert_accelcand(candsGPU, cand),
//
//      int stg = log2((float)candidate->numharm);
//      candTree* ret = opt_cont(&trees[stg], oPlnPln, cont, &batch->sInf->sSpec->fftInf, i);
//
//      trees[stg].add(ret);
//
//      delete(ret);
//
//      if ( cont->flag & OPTIMISED_CONTAINER )
//      {
//        candidate->sig = candidate_sigma_cl(candidate->power, candidate->numharm,  batch->sInf->numindep[stg] );
//        container* cont = optemised.insert(candidate, 0.1);
//
//        if ( cont )
//        {
//          printf("          %03i  harm: %2i   pow: %9.3f   r: %9.4f  z: %7.4f\n",i, candidate->numharm, candidate->power, candidate->r, candidate->z );
//        }
//        else
//        {
//          printf("          NO\n");
//        }
//      }
//      else
//      {
//        printf("          Already Done\n");
//      }
//    }
//
//    cont = cont->smaller;
//  }
//
//  printf("Optimisation Removed %6i - %6i remain \n", qt->noVals() - optemised.noVals(), optemised.noVals() );
//
//  eliminate_harmonics(&optemised);
//
//  return candsGPU;
//}

//
//uint getOffset(int height, int stride, int harmNo, int stepNo, int rowNo, void* data = NULL)
//{
//  offset    = (rowNo + stepNo*height)*stride + cHInfo->halfWidth * 2
//}
//
//void* getPowerRow(cuFFdotBatch* batch, int harmNo, int stepNo, int rowNo, void* data = NULL)
//{
//  int stackNo = batch->hInfos[harmNo].stackNo;
//
//  cuFfdotStack* cStack    = &batch->stacks[stackNo];
//  cuFFdot*      plan      = &batch->planes[harmNo];
//  cuHarmInfo*   cHInfo    = &batch->hInfos[harmNo];      // The current harmonic we are working on
//
//  if ( data == NULL )
//    data = batch->d_planePowr;
//
//  void* plnData;
//  int   offset = 0;
//
//  if      ( batch->flag & FLAG_ITLV_ROW )
//  {
//    offset    = (rowNo*batch->noSteps + stepNo)*cStack->stridePower + cHInfo->halfWidth * 2 ;
//    //powers    = &((float*)plan->d_planePowr)[offset];
//  }
//  else
//  {
//    offset    = (rowNo + stepNo*cHInfo->height)*cStack->stridePower + cHInfo->halfWidth * 2
//    //powers    = &((float*)plan->d_planePowr)[offset];
//  }
//
//  if      ( batch->flag & FLAG_HALF )
//  {
//    retrun &((half*)      plan->d_planePowr)[offset];
//  }
//  else if ( batch->flag & FLAG_CUFFT_CB_POW )
//  {
//    retrun &((float*)     plan->d_planePowr)[offset];
//  }
//  else if ( batch->flag & FLAG_CUFFT_CB_POW )
//  {
//    retrun &((cmplxData*) plan->d_planePowr)[offset];
//  }
//
//}
//
//// Copy data from device  .
//int getPowers(cuFFdotBatch* batch, float* dst)
//{
//  ulong sz  = 0;
//  harm      = 0;
//
//  void* out;
//
//  if      ( batch->flag & FLAG_HALF )         // half output
//  {
//    out = malloc( batch->pwrDataSize * 2 );
//  }
//  else if ( batch->flag & FLAG_CUFFT_CB_POW ) // float output
//  {
//    out = malloc( batch->pwrDataSize );
//  }
//  else
//  {
//    out = malloc( batch->pwrDataSize / 2.0 ); // fcomplexcu output
//  }
//
//  // Read data from device
//  CUDA_SAFE_CALL(hipMemcpyAsync(out, batch->d_planePowr, batch->pwrDataSize,   hipMemcpyDeviceToHost, batch->resStream), "Failed to copy input data from device.");
//
//  // Write data to page locked memory
//  for ( int stackNo = 0; stackNo < batch->noStacks; stackNo++ )
//  {
//    cuFfdotStack* cStack = &batch->stacks[stackNo];
//
//    for ( int plainNo = 0; plainNo < cStack->noInStack; plainNo++ )
//    {
//      cuHarmInfo* cHInfo    = &batch->hInfos[harm];      // The current harmonic we are working on
//      cuFFdot*    plan      = &cStack->planes[plainNo];          // The current plane
//
//      for ( int stepNo = 0; stepNo < batch->noSteps; stepNo ++) // Loop over steps
//      {
//        rVals* rVal = &((batch->rValues)[stepNo][harm]);
//
//        if ( rVal->numdata )
//        {
//          //// Copy input data from GPU
//          //fcomplexcu *data = &batch->d_iData[sz];
//          //CUDA_SAFE_CALL(hipMemcpyAsync(out, data, cStack->strideCmplx*2*sizeof(float), hipMemcpyDeviceToHost, cStack->fftIStream), "Failed to copy input data from device.");
//
//          CUDA_SAFE_CALL(hipMemcpyAsync(out, plan->d_planePowr, (rVal->numrs)*sizeof(float),   hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
//
//          // Copy pain from GPU
//          for( int y = 0; y < cHInfo->height; y++ )
//          {
//            fcomplexcu *cmplxData;
//            float *powers;
//
//            if      ( batch->flag & FLAG_ITLV_ROW )
//            {
//              cmplxData = &plan->d_planeMult[(y*batch->noSteps + stepNo)*cStack->strideCmplx   + cHInfo->halfWidth * 2 ];
//              powers    = &((float*)plan->d_planePowr)[(y*batch->noSteps + stepNo)*cStack->stridePower + cHInfo->halfWidth * 2 ];
//            }
//            else
//            {
//              cmplxData = &plan->d_planeMult[(y + stepNo*cHInfo->height)*cStack->strideCmplx   + cHInfo->halfWidth * 2 ];
//              powers    = &((float*)plan->d_planePowr)[(y + stepNo*cHInfo->height)*cStack->stridePower + cHInfo->halfWidth * 2 ];
//            }
//
//            if      ( batch->flag & FLAG_CUFFT_CB_OUT )
//            {
//              //CUDA_SAFE_CALL(hipMemcpyAsync(gpuPowers[step][harm].getP(0,y), powers, (plan->numrs[step])*sizeof(float),   hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
//              CUDA_SAFE_CALL(hipMemcpyAsync(gpuPowers[stepNo][harm].getP(0,y), powers, (rVal->numrs)*sizeof(float),   hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
//              /*
//                                   for( int jj = 0; jj < plan->numrs[step]; jj++)
//                                   {
//                                     float *add = gpuPowers[step][harm].getP(jj*2+1,y);
//                                     gpuPowers[step][harm].setPoint<ARRAY_SET>(add, 0);
//                                   }
//               */
//            }
//            else
//            {
//              //cmplxData += cHInfo->halfWidth*ACCEL_RDR;
//              //CUDA_SAFE_CALL(hipMemcpyAsync(gpuCmplx[step][harm].getP(0,y), cmplxData, (plan->numrs[step])*2*sizeof(float), hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
//              CUDA_SAFE_CALL(hipMemcpyAsync(gpuCmplx[stepNo][harm].getP(0,y), cmplxData, (rVal->numrs)*2*sizeof(float), hipMemcpyDeviceToHost, cStack->fftPStream), "Failed to copy input data from device.");
//            }
//          }
//        }
//
//        sz += cStack->strideCmplx;
//      }
//      harm++;
//    }
//
//    // New events for Synchronisation (this event will override the previous event)
//    hipEventRecord(cStack->prepComp, cStack->fftIStream);
//    hipEventRecord(cStack->ifftComp,  cStack->fftPStream);
//  }
//
//  free(out);
//}

/**  Wait for CPU threads to complete  .
 *
 */
int waitForThreads(sem_t* running_threads, const char* msg, int sleepMS )
{
  infoMSG(1,2,"Wait for CPU threads to complete\n");

  int noTrd;
  sem_getvalue(running_threads, &noTrd );

  if (noTrd)
  {
    char waitMsg[1024];
    int ite = 0;

    nvtxRangePush("Wait on CPU threads");

    while ( noTrd > 0 )
    {
      nvtxRangePush("Sleep");

      ite++;

      if ( noTrd >= 1 && !(ite % 10) )
      {
        sprintf(waitMsg,"%s  %3i thread still active.", msg, noTrd);

        FOLD  // Spinner  .
        {
          if      (ite == 1 )
            printf("\r%s⌜   ", waitMsg);
          if      (ite == 2 )
            printf("\r%s⌝   ", waitMsg);
          if      (ite == 3 )
            printf("\r%s⌟   ", waitMsg);
          if      (ite == 4 )
          {
            printf("\r%s⌞   ", waitMsg);
            ite = 0;
          }
          fflush(stdout);
        }
      }

      usleep(sleepMS);
      sem_getvalue(running_threads, &noTrd );

      nvtxRangePop();
    }

    if (ite >= 10 )
      printf("\n\n");

    nvtxRangePop();

    return (ite);
  }

  return (0);
}
