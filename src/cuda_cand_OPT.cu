#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <math.h>             // log
#include <hiprand/hiprand_kernel.h>

#include "cuda_math_ext.h"
#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"


#define FTLIM 1e-6
//#define DLIM  0.4
#define DLIM  0.0


extern "C"
{
#define __float128 long double
#include "accel.h"
}

int     optpln01  = 50;
int     optpln02  = 30;
int     optpln03  = 20;
int     optpln04  = 20;
int     optpln05  = 20;
int     optpln06  = 0;

float   downScale = 6;

float   optSz01   = 16;
float   optSz02   = 14;
float   optSz04   = 12;
float   optSz08   = 10;
float   optSz16   = 8;


__device__ inline double cos_t(double x)
{
  return cos(x);
}
__device__ inline float cos_t(float x)
{
  return cosf(x);
}

__device__ inline double sin_t(double x)
{
  return sin(x);
}
__device__ inline float sin_t(float x)
{
  return sinf(x);
}

__device__ inline double sqrt_t(double x)
{
  return sqrt(x);
}
__device__ inline float sqrt_t(float x)
{
  return sqrtf(x);
}

__device__ inline void sincos_t(double x, double* s, double* c )
{
  sincos(x, s, c);
}
__device__ inline void sincos_t(float x, float* s, float* c )
{
  sincosf(x, s, c);
}


template<typename T>
__device__ void fresnl(T xxa, T* ss, T* cc)
{
  T f, g, c, s, t, u;
  T x, x2;

  x       = fabs(xxa);
  x2      = x * x;

  if      ( x2 < 2.5625   )    	// Small so use a polynomial approximation  .
  {
    t     = x2 * x2;

    T t01 = t;
    T t02 = t01*t;
    T t03 = t02*t;
    T t04 = t03*t;
    T t05 = t04*t;
    T t06 = t05*t;
    T sn  = (T)3.18016297876567817986e11 + (T)-4.42979518059697779103e10*t01 + (T)2.54890880573376359104e9*t02  + (T)-6.29741486205862506537e7*t03  + (T)7.08840045257738576863e5 *t04 - (T)2.99181919401019853726e3  *t05;
    T sd  = (T)6.07366389490084639049e11 + (T) 2.24411795645340920940e10*t01 + (T)4.19320245898111231129e8*t02  + (T) 5.17343888770096400730e6*t03  + (T)4.55847810806532581675e4 *t04 + (T)2.81376268889994315696e2  *t05 + t06 ;
    T cn  = (T)9.99999999999999998822e-1 + (T)-2.05525900955013891793e-1*t01 + (T)1.88843319396703850064e-2*t02 + (T)-6.45191435683965050962e-4*t03 + (T)9.50428062829859605134e-6*t04 - (T)4.98843114573573548651e-8 *t05;
    T cd  = (T)1.00000000000000000118e0  + (T) 4.12142090722199792936e-2*t01 + (T)8.68029542941784300606e-4*t02 + (T) 1.22262789024179030997e-5*t03 + (T)1.25001862479598821474e-7*t04 + (T)9.15439215774657478799e-10*t05 + (T)3.99982968972495980367e-12*t06 ;

    *ss   = x * x2 * sn / sd;
    *cc   = x * cn / cd;
  }
  else if ( x  > 36974.0  )     // Asymptotic behaviour  .
  {
    *cc   = 0.5;
    *ss   = 0.5;
  }
  else                          // Auxiliary functions for large argument  .
  {
    x2    = x * x;
    t     = (T)PI * x2;
    u     = 1.0 / (t * t);
    t     = 1.0 / t;

//    T fn  = (T)3.76329711269987889006e-20+((T)1.34283276233062758925e-16+((T)1.72010743268161828879e-13+((T)1.02304514164907233465e-10+((T)3.05568983790257605827e-8 +((T)4.63613749287867322088e-6+((T)3.45017939782574027900e-4+((T)1.15220955073585758835e-2+((T)1.43407919780758885261e-1+ (T)4.21543555043677546506e-1*u)*u)*u)*u)*u)*u)*u)*u)*u;
//    T fd  = (T)1.25443237090011264384e-20+((T)4.52001434074129701496e-17+((T)5.88754533621578410010e-14+((T)3.60140029589371370404e-11+((T)1.12699224763999035261e-8 +((T)1.84627567348930545870e-6+((T)1.55934409164153020873e-4+((T)6.44051526508858611005e-3+((T)1.16888925859191382142e-1+((T)7.51586398353378947175e-1+u)*u)*u)*u)*u)*u)*u)*u)*u)*u ;
//    T gn  = (T)1.86958710162783235106e-22+((T)8.36354435630677421531e-19+((T)1.37555460633261799868e-15+((T)1.08268041139020870318e-12+((T)4.45344415861750144738e-10+((T)9.82852443688422223854e-8+((T)1.15138826111884280931e-5+((T)6.84079380915393090172e-4+((T)1.87648584092575249293e-2+((T)1.97102833525523411709e-1+ (T)5.04442073643383265887e-1*u)*u)*u)*u)*u)*u)*u)*u)*u)*u ;
//    T gd  = (T)1.86958710162783236342e-22+((T)8.39158816283118707363e-19+((T)1.38796531259578871258e-15+((T)1.10273215066240270757e-12+((T)4.60680728146520428211e-10+((T)1.04314589657571990585e-7+((T)1.27545075667729118702e-5+((T)8.14679107184306179049e-4+((T)2.53603741420338795122e-2+((T)3.37748989120019970451e-1+((T)1.47495759925128324529e0 +u)*u)*u)*u)*u)*u)*u)*u)*u)*u)*u ;

    T u01 = u;
    T u02 = u01*u;
    T u03 = u02*u;
    T u04 = u03*u;
    T u05 = u04*u;
    T u06 = u05*u;
    T u07 = u06*u;
    T u08 = u07*u;
    T u09 = u08*u;
    T u10 = u09*u;
    T u11 = u10*u;
    T fn  = (T)3.76329711269987889006e-20 + (T)1.34283276233062758925e-16*u01 + (T)1.72010743268161828879e-13*u02 + (T)1.02304514164907233465e-10*u03 + (T)3.05568983790257605827e-8 *u04 + (T)4.63613749287867322088e-6*u05 + (T)3.45017939782574027900e-4*u06 + (T)1.15220955073585758835e-2*u07 + (T)1.43407919780758885261e-1*u08 + (T)4.21543555043677546506e-1*u09;
    T fd  = (T)1.25443237090011264384e-20 + (T)4.52001434074129701496e-17*u01 + (T)5.88754533621578410010e-14*u02 + (T)3.60140029589371370404e-11*u03 + (T)1.12699224763999035261e-8 *u04 + (T)1.84627567348930545870e-6*u05 + (T)1.55934409164153020873e-4*u06 + (T)6.44051526508858611005e-3*u07 + (T)1.16888925859191382142e-1*u08 + (T)7.51586398353378947175e-1*u09 + u10;
    T gn  = (T)1.86958710162783235106e-22 + (T)8.36354435630677421531e-19*u01 + (T)1.37555460633261799868e-15*u02 + (T)1.08268041139020870318e-12*u03 + (T)4.45344415861750144738e-10*u04 + (T)9.82852443688422223854e-8*u05 + (T)1.15138826111884280931e-5*u06 + (T)6.84079380915393090172e-4*u07 + (T)1.87648584092575249293e-2*u08 + (T)1.97102833525523411709e-1*u09 + (T)5.04442073643383265887e-1*u10 ;
    T gd  = (T)1.86958710162783236342e-22 + (T)8.39158816283118707363e-19*u01 + (T)1.38796531259578871258e-15*u02 + (T)1.10273215066240270757e-12*u03 + (T)4.60680728146520428211e-10*u04 + (T)1.04314589657571990585e-7*u05 + (T)1.27545075667729118702e-5*u06 + (T)8.14679107184306179049e-4*u07 + (T)2.53603741420338795122e-2*u08 + (T)3.37748989120019970451e-1*u09 + (T)1.47495759925128324529e0 *u10 + u11 ;


    f     = 1.0 - u * fn / fd;
    g     =       t * gn / gd;

    t     = (T)PIBYTWO * x2;
    sincos(t, &s, &c);
    t     = (T)PI * x;

    *cc   = 0.5 + (f * s - g * c) / t;
    *ss   = 0.5 - (f * c + g * s) / t;
  }

  if (xxa < 0.0)                // Swap as function is antisymmetric  .
  {
    *cc   = -*cc;
    *ss   = -*ss;
  }
}


/** Generate the complex response value for Fourier f-dot interpolation  .
 *
 * This is based on gen_z_response in responce.c
 *
 * @param rx            The x index of the value in the kernel
 * @param z             The Fourier Frequency derivative (# of bins the signal smears over during the observation)
 * @param absz          Is the absolute value of z
 * @param roffset       Is the offset in Fourier bins for the full response (i.e. At this point, the response would equal 1.0)
 * @param numbetween    Is the number of points to interpolate between each standard FFT bin. (i.e. 'numbetween' = 2 = interbins, this is the standard)
 * @param numkern       Is the number of complex points that the kernel will contain.
 * @param rr            A pointer to the real part of the complex response for rx
 * @param ri            A pointer to the imaginary part of the complex response for rx
 */
template<typename T>
__device__ inline void gen_z_response(int rx, T z,  T absz, T numbetween, int numkern, float* rr, float* ri)
{
  int signz;
  T zd, r, xx, yy, zz, startr, startroffset;
  T fressy, frescy, fressz, frescz, tmprl, tmpim;
  T s, c, pibyz, cons, delta;

  T zT = z;
  T rT = r;

  startr        = 0 - (0.5 * zT);
  startroffset  = (startr < 0) ? 1.0 + modf(startr, &tmprl) : modf(startr, &tmprl);

  if (rx == numkern / 2.0 && startroffset < 1E-3 && absz < 1E-3)
  {
    T nr, ni;

    zz      = zT * zT;
    xx      = startroffset * startroffset;
    nr      = (T)1.0 - (T)0.16449340668482264365 * zz;
    ni      = (T)-0.5235987755982988731 * zT;
    nr      += startroffset * (T)1.6449340668482264365 * zT;
    ni      += startroffset * ((T)PI - (T)0.5167712780049970029 * zz);
    nr      += xx * ((T)-6.579736267392905746 + (T)0.9277056288952613070 * zz);
    ni      += xx * ((T)3.1006276680299820175 * zT);

    *rr     = nr;
    *ri     = ni;
  }
  else
  {
    /* This is evaluating Eq (39) in:
     * Ransom, Scott M., Stephen S. Eikenberry, and John Middleditch. "Fourier techniques for very long astrophysical time-series analysis." The Astronomical Journal 124.3 (2002): 1788.
     *
     * Where: qᵣ  is the variable r and represents the distance from the centre frequency
     *        |ṙ| is the variable z which is ḟ
     */

    signz   = (zT < 0.0) ? -1 : 1;
    zd      = signz * (T)SQRT2 / sqrt(absz);
    zd      = signz * sqrt(2.0 / absz);
    cons    = zd / 2.0;                             // 1 / sqrt(2*r')

    startr  += numkern / (T) (2 * numbetween);
    delta   = -1.0 / numbetween;
    r       = startr + rx * delta;

    pibyz   = (T)PI / zT;
    yy      = rT * zd;
    zz      = yy + zT * zd;
    xx      = pibyz * rT * rT;

    sincos_t(xx, &s, &c);
    fresnl<T>(yy, &fressy, &frescy);
    fresnl<T>(zz, &fressz, &frescz);

    tmprl   = signz * (frescz - frescy);
    tmpim   = fressy - fressz;

    *rr     =  (tmprl * c - tmpim * s) * cons;
    *ri     = -(tmprl * s + tmpim * c) * cons;
  }
}

/* This routine uses the correlation method to do a Fourier        */
/* complex interpolation at a single point in the f-fdot plane.    */
/* It does the correlations manually. (i.e. no FFTs)               */
/* Arguments:                                                      */
/*   'data' is a complex array of the data to be interpolated.     */
/*   'numdata' is the number of complex points (bins) in data.     */
/*   'r' is the Fourier frequency in data that we want to          */
/*      interpolate.  This can (and should) be fractional.         */
/*   'z' is the fdot to use (z=f-dot*T^2 (T is integration time)). */
/*   'kern_half_width' is the half-width of the kernel in bins.    */
/*   'ans' is the complex answer.                                  */
template<typename T>
__device__ fcomplexcu rz_interp_cu(fcomplexcu* data, int loR, int noBins, double r, double z, int kern_half_width)
{
  int numkern, intfreq;
  double  fracfreq;
  double  dintfreq;
  int signz;
  int ii, lodata;
  T absz, zd, q_r, xx, Yr, Zr, startr;
  T fressy, frescy, fressz, frescz;
  T s, c, pibyz, cons, sinc;
  T tR, tI;     // Response values

  T zT = z;
  T rT = r;

  fcomplexcu inp;
  fcomplexcu ans;

  ans.r = 0.0;
  ans.i = 0.0;

  if ( r > 0 )
  {
    // Split 'r' into integer and fractional parts
    fracfreq          = modf(r, &dintfreq); // This has to be double precision
    intfreq           = (int) dintfreq;
    numkern           = 2 * kern_half_width;
    lodata            = intfreq - kern_half_width;

    // Set up values dependent on Z alone
    absz              = fabs(zT);
    startr            = fracfreq - (0.5 * z);
    signz             = (zT < 0.0) ? -1 : 1;
    zd                = signz * (T)SQRT2 / sqrt(absz);
    cons              = zd / 2.0;
    pibyz             = (T)PI / zT;
    startr            += kern_half_width;

    if ( absz < FTLIM )
    {
//      const int ix        = blockIdx.x * blockDim.x + threadIdx.x;
//      const int iy = blockIdx.y * blockDim.y + threadIdx.y;
//
//      if ( ix == 0 )
//      {
//        printf("absz < FTLIM   iy: %03i\n", iy);
//      }
      //double v1   = r - lodata ;
      //startr      = v1;
      startr = (r - lodata);
    }

    FOLD // Clamp values to usable bounds  .
    {
      if ( lodata < 0 )
      {
        //printf("lodata < 0\n");
        numkern += lodata;
        startr  += lodata;
        lodata  = 0;
      }

      //printf("lodata: %i    loR: %i  \n", lodata, loR);
      lodata -= loR;

      //printf("lodata + numkern: %i  noR: %i \n", lodata + numkern, noBins );
      if ( lodata + numkern >= noBins )
      {
        //        printf("lodata + numkern >= noBins\n");
        //        printf("%i + %i >= %i\n", lodata, numkern, noBins );
        numkern = noBins - lodata;
      }

      //printf("numkern: %i\n", numkern );
    }

    // Loop over positions, calculate response values and do multiplications
    for ( ii = 0, q_r = startr; ii < numkern; q_r--, ii++ )
    {
      FOLD //  Read the input value  .
      {
        inp             = data[lodata+ii];
      }

      FOLD //  Calculate response value  .
      {
        if ( absz < FTLIM ) // Just do a Fourier Interpolation
        {
          xx              = (T)PI*q_r ;
          sincos_t(xx, &s, &c);

          if ( q_r == 0.0 )
            sinc = 1.0;
          else
            sinc = s / xx;

          tR              = c * sinc;
          tI              = s * sinc;

//          const int ix        = blockIdx.x * blockDim.x + threadIdx.x;
//          const int iy = blockIdx.y * blockDim.y + threadIdx.y;
//
//          double  dqr   = r - lodata + ii ;
//          double  dxx   = PI*dqr ;
//          double  sd, cd, sincd, tRd, tId;
//          sincos_t(xx, &sd, &cd);
//
//          if ( dqr == 0.0 )
//            sincd = 1.0;
//          else
//            sincd = sd / dxx;
//
//          tRd             = cd * sincd;
//          tId             = sd * sincd;
//
//          if ( ix == 0 )
//          {
//            printf("-- %15.8f  %15.8f\n   %15.8f  %15.8f\n", tR, tI, tRd, tId);
//          }
//
//          tR = tRd;
//          tI = tId;

          //printf("%04i response: %15.10f %15.10f  r: %15.10f  c: %15.10f s: %15.10f sinc: %15.10f\n", ii, tR, tI, q_r, c, s, sinc );
        }
        else
        {
          Yr              = q_r * zd;
          Zr              = Yr + zT * zd;
          xx              = pibyz * q_r * q_r;

          sincos_t(xx, &s, &c);
          fresnl<T>(Yr, &fressy, &frescy);
          fresnl<T>(Zr, &fressz, &frescz);

          T Ster          = fressz - fressy;
          T Cter          = frescy - frescz;
          tR              = cons * (c*Ster + signz*s*Cter);
          tI              = cons * (s*Ster - signz*c*Cter);

//          const int ix    = blockIdx.x * blockDim.x + threadIdx.x;
//          const int iy    = blockIdx.y * blockDim.y + threadIdx.y;
//          if ( ix == 0 && iy == 0 && ii == 0 )
//          {
//            printf("Yr: %20.10f  Zr: %20.10f  xx: %20.10f  tR: %20.10f  tI: %20.10f  xx: %20.10f  xx: %20.10f  \n", Yr, Zr, xx, tR, tI);
//          }
        }
      }

      FOLD //  Do the multiplication  .
      {
        ans.r           += tR * inp.r - tI*inp.i;
        ans.i           += tR * inp.i + tI*inp.r;
      }

      //printf("%03i %05i Data %12.2f %12.2f  Response: %13.10f %13.10f   %12.2f \n", ii, loR+lodata+ii, inp.r, inp.i, tR, tI, POWERR(ans.r, ans.i) );
    }
  }
  else
  {
    //printf("r < 0: %.6f\n", r );
  }

  return ans;
}

template<typename T>
__global__ void ffdotPln_ker(float* powers, fcomplexcu* fft, int noHarms, int halfwidth, double firstR, double firstZ, double rSZ, double zSZ, int noR, int noZ, int iStride, int oStride, int16 loR, float16 norm, int16 hw)
{
  const int ix = blockIdx.x * blockDim.x + threadIdx.x;
  const int iy = blockIdx.y * blockDim.y + threadIdx.y;

  if ( ix < noR && iy < noZ)
  {
//    if ( ix ==0 && iy == 0 )
//    {
//      printf("\n");
//    }
    double r            = firstR + ix/(double)(noR-1) * rSZ ;
    double z            = firstZ - iy/(double)(noZ-1) * zSZ ;

    double total_power  = 0;
    fcomplexcu ans;

    //double absz         = fabs(z);

    for( int i = 1; i <= noHarms; i++ )
    {
      double absz         = fabs(z*i);
//      if(ix ==0 && iy == 0 )
//      {
//        printf("%02i absz: %.5f\n",i, absz);
//      }
      if( absz < DLIM && absz > FTLIM )
      {
        //ans  = rz_interp_cu<double>(&fft[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, halfwidth);
        ans  = rz_interp_cu<double>(&fft[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, hw.val[i-1] );
      }
      else
      {
        //ans  = rz_interp_cu<T>(&fft[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, halfwidth);
        ans  = rz_interp_cu<T>(&fft[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, hw.val[i-1] );
      }

      //total_power     += POWERR(ans.r, ans.i)/norm.val[i-1];
      total_power     += POWERR(ans.r, ans.i);
    }

    //powers[iy*noR + ix] = total_power;
    powers[iy*oStride + ix] = total_power;
  }
}

__global__ void rz_interp_ker(double r, double z, fcomplexcu* fft, int loR, int noBins, int halfwidth, double normFactor)
{
  float total_power   = 0;

  fcomplexcu ans      = rz_interp_cu<float>(fft, loR, noBins, r, z, halfwidth);
  //fcomplexcu ans      = rz_interp_cu<double>(fft, loR, noBins, r, z, halfwidth);
  total_power         += POWERR(ans.r, ans.i)/normFactor;

  //printf("rz_interp_ker r: %.4f  z: %.4f  Power: %.4f  ( %.4f, %.4f )\n", r, z, POWERR(ans.r, ans.i), ans.r, ans.i);
}

/*

 hiprandState *d_state;
  hipMalloc(&d_state, sizeof(hiprandState));


 */
template<typename T>
__global__ void ffdotSwarm_ker(unsigned long long seed, candOpt* out, fcomplexcu* fft, int loR, int noBins, int noHarms, int noReps, int halfwidth, double firstR, double firstZ, double rSZ, double zSZ, int noR, int noZ, float16 norm)
{
  const int ix        = blockIdx.x * blockDim.x + threadIdx.x;
  const int iy        = blockIdx.y * blockDim.y + threadIdx.y;
  const int idx       = iy * noR + ix;
  const int wrpNo     = floor(idx/32.0);
  const int sz        = 32 ; // noR * noZ - wrpNo * 32 ;
  const int lane      = idx % sz;
  const int oLane     = (lane+1) % sz;

  hiprandState state;

  float a = 0.8;
  float b = 0.4;
  float c = 0.7;

  float   velocityMax       = rSZ / 20.0;

  float   power             = 0;
  float   gBestP            = 0;
  float   lBestP            = 0;

  double  nBestR, nBestZ;
  float   nBestP;

  double2 lGlo;
  double2 lLoc;
  double2 pos;
  double2 vel;
  double2 dGlo;
  double2 dLoc;

  lGlo.x = 0;
  lGlo.y = 0;

  if ( ix < noR && iy < noZ)
  {
    hiprand_init(seed, idx, 0, &state);

    FOLD // Initial values  .
    {
      pos.x       = firstR + ix/(double)(noR-1) * rSZ ;
      pos.y       = firstZ - iy/(double)(noZ-1) * zSZ ;
      power       = 0;

      for( int i = 1; i <= noHarms ; i++ )
      {
        //if ( idx == 1001 )
        {
          fcomplexcu ans   = rz_interp_cu<T>(fft, loR, noBins, pos.x*i, pos.y*i, halfwidth);
          power            += POWERR(ans.r, ans.i)/norm.val[i-1];

          //          if ( idx == 1001 )  // TMP
          //          {
          //            printf(" Pow: %10.2f  %10.2f %10.2f Norm: %10.3f   Accum: %10.3f\n", POWERR(ans.r, ans.i), ans.r, ans.i, norm.val[i-1], power );
          //          }
        }
      }

      // Set local best
      lLoc     = pos;
      lBestP   = power;

      // Global local best
      lGlo      = lLoc;
      gBestP    = lBestP;

      FOLD // Velocity  .
      {
        vel.x = hiprand_uniform(&state);
        vel.y = hiprand_uniform(&state);
        float lenn = sqrt(vel.x*vel.x+vel.y*vel.y) ; // len(vel);
        vel *= velocityMax/lenn;
      }
    }

    //    if ( idx == 1001 ) // TMP
    //    {
    //      double gDst = len(lGlo - pos);
    //      printf("%03i Current r: %10.5f z: %10.5f  power: %20.6f  -  Local r: %10.5f z: %10.5f  power: %20.6f  -  Best r: %10.5f z: %10.5f  power: %20.6f    Dist %9.6f\n", 0, pos.x, pos.y, power, lLoc.x, lLoc.y, lBestP, lGlo.x, lGlo.y, gBestP, gDst);
    //    }

    if(1)
    {
      for (int rep = 0; rep < 10; rep++)
      {
        //d2 = gBestR - pos.x;
        //d3 = gBestZ - pos.y;

        dLoc = lLoc - pos;
        dGlo = lGlo - pos;

        //double lDst = len(dLoc);
        //double gDst = len(dGlo);

        //r1 = hiprand_uniform(&state);
        //r2 = hiprand_uniform(&state);

        vel = a*vel+b*dLoc+c*dGlo;

        float lenn = len(vel);
        if ( lenn > velocityMax )
        {
          vel *= velocityMax/lenn;
        }

        pos += vel;

        power = 0;
        for( int i = 1; i <= noHarms; i++ )
        {
          fcomplexcu ans   = rz_interp_cu<T>(fft, loR, noBins, pos.x*i, pos.y*i, halfwidth);
          power           += POWERR(ans.r, ans.i)/norm.val[i-1];
        }

        //        if ( isnan(power) ) // TMP  .
        //        {
        //          printf("idx: %03i   r: %.1f \n", idx, pos.x );
        //        }

        if ( power > lBestP ) // Update Local bets  .
        {
          lLoc      = pos;
          lBestP    = power;
        }

        if ( power > gBestP ) // Update Global bets  .
        {
          lGlo      = pos;
          gBestP    = power;
        }


        FOLD // Check Global best with neighbour  .
        {
          //nBestR = lGlo.x;
          //nBestZ = lGlo.y;
          //nBestP = gBestP;
          //nBestR = __shfl(lGlo.x,oLane);
          //nBestZ = __shfl(lGlo.y,oLane);

          int2 tmpForExchIn, tmpForExchOut;
          float tt = gBestP;

          for ( int ln = 0; ln < 32; ln++) // Shuffle with all elements in the warp
          {
            // get R
            tmpForExchIn = *(int2 *)(&lGlo.x);
            tmpForExchOut.x = __shfl(tmpForExchIn.x, ln);
            tmpForExchOut.y = __shfl(tmpForExchIn.y, ln);
            nBestR = *(double *)(&tmpForExchOut);

            // get Z
            tmpForExchIn = *(int2 *)(&lGlo.y);
            tmpForExchOut.x = __shfl(tmpForExchIn.x, ln);
            tmpForExchOut.y = __shfl(tmpForExchIn.y, ln);
            nBestZ = *(double *)(&tmpForExchOut);

            // power
            nBestP = __shfl(gBestP, ln);

            if ( nBestP > gBestP )
            {
              //            if ( idx == 0 ) // TMP
              //            {
              //              printf("Got a new best!\n");
              //            }
              lGlo.x   = nBestR;
              lGlo.y   = nBestZ;
              gBestP   = nBestP;
            }
            else if ( idx == 0 ) // TMP
            {
              //printf("Shuffle got Current r: %.5f z: %.5f   power %15.6f vs %15.6f!\n", nBestR, nBestZ, nBestP, gBestP );
            }
          }
        }

        //        if ( idx == 1001 ) // TMP
        //        {
        //          double gDst = len(lGlo - pos);
        //          printf("%03i Current r: %10.5f z: %10.5f  power: %20.6f  -  Local r: %10.5f z: %10.5f  power: %20.6f  -  Best r: %10.5f z: %10.5f  power: %20.6f    Dist %9.6f\n", rep+1, pos.x, pos.y, power, lLoc.x, lLoc.y, lBestP, lGlo.x, lGlo.y, gBestP, gDst);
        //        }
      }
    }

    FOLD // Output  .
    {
      candOpt outP;
      outP.r     = lGlo.x;
      outP.z     = lGlo.y;
      outP.power = gBestP;
      out[idx]   = outP;
    }
  }
}

int ffdotPln(float* powers, fcomplex* fft, int loR, int noBins, int noHarms, double centR, double centZ, double rSZ, double zSZ, int noR, int noZ, int halfwidth, float* fac)
{
  double log2 = log(2.0);

  double maxZ = (centZ + zSZ/2.0);
  double minZ = (centZ - zSZ/2.0);
  double minR = (centR - rSZ/2.0);
  double maxR = (centR + rSZ/2.0);

  int halfwidth2    = z_resp_halfwidth(MAX(fabs(maxZ*noHarms), fabs(minZ*noHarms))+4, HIGHACC);
  //halfwidth         = MAX(halfwidth,halfwidth2);

  double rSpread    = ceil(maxR*noHarms  + halfwidth) - floor(minR*noHarms - halfwidth);

  size_t iStride, pStride;
  float *cuPowers;
  fcomplexcu *cuInp;
  fcomplexcu *cpuInp;
  double factor;

  CUDA_SAFE_CALL(hipMallocPitch(&cuPowers,  &pStride, noR     * sizeof(float),             noZ),   "Failed to allocate device memory for kernel stack.");
  CUDA_SAFE_CALL(hipMallocPitch(&cuInp,     &iStride, rSpread * sizeof(hipfftComplex),  noHarms),   "Failed to allocate device memory for kernel stack.");

  int noInp = iStride/sizeof(hipfftComplex);
  int noPow = pStride/sizeof(float);

  int16   rOff;
  int16   hw;
  float16 norm;

  cpuInp = (fcomplexcu*) malloc(iStride*noHarms);

  for( int h = 0; h < 16; h++)
  {
    rOff.val[h] = 0;
    hw.val[h]   = 0;
  }

  for( int h = 0; h < noHarms; h++)
  {
    rOff.val[h]   = floor( minR*(h+1) - halfwidth );
    hw.val[h]     = z_resp_halfwidth(MAX(fabs(maxZ*(h+1)), fabs(minZ*(h+1)))+2, HIGHACC);
    //printf("%i  %f   %i\n", (int)floor(minR*(h+1)), minR*(h+1), halfwidth );

    int datStart  = floor( minR*(h+1) - halfwidth );
    int datEnd    = ceil ( maxR*(h+1) + halfwidth );
    int novals    = datEnd - datStart;
    int noPowers, off;
    float medianv;

    FOLD // Calculate normalisation factor  .
    {
      if ( fac == NULL) // Calculate the normalisation factor  .
      {
        float*  normPow = (float*) malloc(noInp*sizeof(float));

        if ( datStart < loR )
        {
          novals    -=  (loR - datStart);
          datStart  =   loR;
        }
        if ( datEnd >= noBins )
        {
          novals    -=  (datEnd - noBins - 1);
          datEnd    =   noBins-1;
        }

        noPowers = 0;
        for ( int i = 0; i < noInp; i++)
        {
          off = rOff.val[h] - loR + i;
          if (off >= 0 && off < noBins )
          {
            normPow[noPowers++] = POWERR(fft[off].r, fft[off].i ) ;
          }
        }

        medianv       = median(normPow, noPowers);
        factor        = sqrt(medianv/log2);
        //printf("  %02i  %8.3f \n", h+1, factor );

        free(normPow);
      }
      else              // Use precalcualted normalisation factor  .
      {
        factor = sqrt(fac[h]);
      }
      norm.val[h] = fac[h];
      //factor = 1.0;
    }

    for ( int i = 0; i < noInp; i++) // Normalise input  .
    {
      off = rOff.val[h] - loR + i;
      if (off >= 0 && off < noBins && i < novals)
      {
        cpuInp[h*noInp + i].r = fft[off].r / factor ;
        cpuInp[h*noInp + i].i = fft[off].i / factor ;
      }
      else
      {
        cpuInp[h*noInp + i].r = 0;
        cpuInp[h*noInp + i].i = 0;
      }
    }
  }

  CUDA_SAFE_CALL(hipMemcpy(cuInp, cpuInp, iStride*noHarms, hipMemcpyHostToDevice), "Copying optimisation input to the device");

  FOLD // Call kernel  .
  {
    dim3 dimBlock, dimGrid;

    // Blocks of 1024 threads ( the maximum number of threads per block )
    dimBlock.x = 16;
    dimBlock.y = 16;
    dimBlock.z = 1;

    // One block per harmonic, thus we can sort input powers in Shared memory
    dimGrid.x = ceil(noR/(float)dimBlock.x);
    dimGrid.y = ceil(noZ/(float)dimBlock.y);

    // Call the kernel to normalise and spread the input data
    ffdotPln_ker<float><<<dimGrid, dimBlock, 0, 0>>>(cuPowers, cuInp, noHarms, halfwidth, minR, maxZ, rSZ, zSZ, noR, noZ, noInp, noPow, rOff, norm, hw);

    CUDA_SAFE_CALL(hipGetLastError(), "Calling the ffdotPln_ker kernel.");
  }

  CUDA_SAFE_CALL(hipMemcpy(powers, cuPowers, pStride*noZ, hipMemcpyDeviceToHost), "Copying optimisation results back from the device.");

  //hipDeviceSynchronize();          // TMP
  int TMPP = 0;

  FOLD // Write CVS
  {
    char tName[1024];
    sprintf(tName,"/home/chris/accel/lrg_2_GPU.csv");
    FILE *f2 = fopen(tName, "w");

    fprintf(f2,"%i",noHarms);

    for (int indx = 0; indx < noR ; indx++ )
    {
      double r = minR + indx/(double)(noR-1) * (rSZ) ;
      fprintf(f2,"\t%.6f",r);
    }
    fprintf(f2,"\n");

    for (int indy = 0; indy < noZ; indy++ )
    {
      double z = maxZ - indy/(double)(noZ-1) * (zSZ) ;

      fprintf(f2,"%.6f",z);

      for (int indx = 0; indx < noR ; indx++ )
      {
        float yy2 = powers[indy*noPow+indx];
        fprintf(f2,"\t%.6f",yy2);
      }
      fprintf(f2,"\n");
    }
    fclose(f2);

    FOLD // Make image
    {
      printf("Making lrg_GPU.png    \t... ");
      fflush(stdout);
      char cmd[1024];
      sprintf(cmd,"python ~/bin/bin/plt_ffd.py %s", tName);
      system(cmd);
      printf("Done\n");
    }

    int tmp = 0;
  }

  CUDA_SAFE_CALL(hipFree(cuPowers),    "Failed free device memory for optimisation powers.");
  CUDA_SAFE_CALL(hipFree(cuInp),       "Failed free device memory for optimisation inputs.");

  return noPow;
}

template<typename T>
void ffdotPln( cuOptCand* pln, fftInfo* fft )
{
  double maxZ       = (pln->centZ + pln->zSize/2.0);
  double minZ       = (pln->centZ - pln->zSize/2.0);
  double maxR       = (pln->centR + pln->rSize/2.0);
  double minR       = (pln->centR - pln->rSize/2.0);

  //if ( pln->halfWidth <= 0 )
  pln->halfWidth    = z_resp_halfwidth(MAX(fabs(maxZ*pln->noHarms), fabs(minZ*pln->noHarms)) + 4, HIGHACC);
  double rSpread    = ceil(maxR*pln->noHarms  + pln->halfWidth) - floor(minR*pln->noHarms - pln->halfWidth);
  int    inpStride  = getStrie(rSpread, sizeof(hipfftComplex), pln->alignment);
  pln->outStride    = getStrie(pln->noR,  sizeof(float), pln->alignment);

  if ( pln->inpStride*pln->noHarms*sizeof(hipfftComplex) > pln->inpSz )
  {
    fprintf(stderr, "ERROR: In function %s, cuOptCand not created with large enough input buffer.", __FUNCTION__);
    exit(EXIT_FAILURE);
  }

  int datStart,  datEnd, noDat;
  int16   rOff;
  int16   hw;
  float16 norm;
  int     off;
  int     newInp = 0;

  for( int h = 0; (h < pln->noHarms) && !newInp; h++)
  {
    datStart        = floor( minR*(h+1) - pln->halfWidth );
    datEnd          = ceil(  maxR*(h+1) + pln->halfWidth );
    noDat           = datEnd - datStart;

    if ( datStart < pln->loR[h] )
    {
      newInp = 1;
    }
    else if ( pln->loR[h] + pln->inpStride < datEnd )
    {
      newInp = 1;
    }
  }

  for( int h = 0; h < 16; h++)
  {
    rOff.val[h] = 0;
    hw.val[h]   = 0;
  }

  if ( newInp )
  {
    pln->inpStride = inpStride;
  }

  for( int h = 0; h < pln->noHarms; h++)
  {
    datStart        = floor( minR*(h+1) - pln->halfWidth );
    datEnd          = ceil(  maxR*(h+1) + pln->halfWidth );
    noDat           = datEnd - datStart;
    hw.val[h]       = z_resp_halfwidth(MAX(fabs(maxZ*(h+1)), fabs(minZ*(h+1))) + 4, HIGHACC);
    rOff.val[h]     = pln->loR[h];

    if ( newInp )
    {
      rOff.val[h]     = datStart;
      pln->loR[h]     = datStart;
      double factor   = sqrt(pln->norm[h]);
      norm.val[h]     = factor;

      for ( int i = 0; i < pln->inpStride; i++ ) // Normalise input  .
      {
        off = rOff.val[h] - fft->rlo + i;

        if ( off >= 0 && off < fft->nor /* && i < noDat */ )
        {
          pln->h_inp[h*pln->inpStride + i].r = fft->fft[off].r / factor ;
          pln->h_inp[h*pln->inpStride + i].i = fft->fft[off].i / factor ;
        }
        else
        {
          pln->h_inp[h*pln->inpStride + i].r = 0;
          pln->h_inp[h*pln->inpStride + i].i = 0;
        }
      }
    }
  }

  if ( newInp )
  {
    //CUDA_SAFE_CALL(hipEventRecord(pln->inpInit, pln->stream),"Recording event: inpInit");
    CUDA_SAFE_CALL(hipMemcpyAsync(pln->d_inp, pln->h_inp, pln->inpStride*pln->noHarms*sizeof(fcomplexcu), hipMemcpyHostToDevice, pln->stream), "Copying optimisation input to the device");
    //CUDA_SAFE_CALL(hipEventRecord(pln->inpCmp, pln->stream),"Recording event: inpCmp");
  }

  FOLD // Call kernel  .
  {
    dim3 dimBlock, dimGrid;

    // Event
    //CUDA_SAFE_CALL(hipEventRecord(pln->compInit, pln->stream),"Recording event: inpInit");

    // Blocks of 1024 threads ( the maximum number of threads per block )
    dimBlock.x = 16;
    dimBlock.y = 16;
    dimBlock.z = 1;

    // One block per harmonic, thus we can sort input powers in Shared memory
    dimGrid.x = ceil(pln->noR/(float)dimBlock.x);
    dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

    // Call the kernel to normalise and spread the input data
    ffdotPln_ker<T><<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, pln->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->inpStride, pln->outStride, rOff, norm, hw);

    CUDA_SAFE_CALL(hipGetLastError(), "Calling the ffdot_ker kernel.");

    // Event
    //CUDA_SAFE_CALL(hipEventRecord(pln->compCmp, pln->stream),"Recording event: inpInit");
  }

  //CUDA_SAFE_CALL(hipEventRecord(pln->outInit, pln->stream),"Recording event: outInit");
  CUDA_SAFE_CALL(hipMemcpyAsync(pln->h_out, pln->d_out, pln->outStride*pln->noZ*sizeof(float), hipMemcpyDeviceToHost, pln->stream), "Copying optimisation results back from the device.");
  CUDA_SAFE_CALL(hipEventRecord(pln->outCmp, pln->stream),"Recording event: outCmp");

  int TMPP = 0;
}

template<typename T>
void ffdotSwrm( cuOptCand* pln, fftInfo* fft )
{
  double maxZ       = (pln->centZ + pln->zSize/2.0);
  double minZ       = (pln->centZ - pln->zSize/2.0);
  double maxR       = (pln->centR + pln->rSize/2.0);
  double minR       = (pln->centR - pln->rSize/2.0);

  pln->halfWidth    = z_resp_halfwidth(MAX(fabs(maxZ*pln->noHarms), fabs(minZ*pln->noHarms)) + 4, HIGHACC);
  double rSpread    = ceil(maxR*pln->noHarms  + pln->halfWidth) - floor(minR*pln->noHarms - pln->halfWidth);
  //pln->inpStride    = getStrie(rSpread, sizeof(hipfftComplex), pln->alignment);
  //pln->outStride    = getStrie(pln->noR,  sizeof(float), pln->alignment);
  //  if ( pln->inpStride*pln->noHarms*sizeof(hipfftComplex) > pln->inpSz )
  //  {
  //    fprintf(stderr, "ERROR: In function %s, cuOptCand not created with large enough input buffer.", __FUNCTION__);
  //    exit(EXIT_FAILURE);
  //  }

  int16   rOff;
  float16 norm;
  int     off;
  int datStart,  datEnd, noDat;

  for( int h = 0; h < 16; h++)
  {
    rOff.val[h] = 0;
  }

  for( int h = 0; h < pln->noHarms; h++)
  {
    norm.val[h]     = pln->norm[h];
  }

  FOLD // Call kernel  .
  {
    dim3 dimBlock, dimGrid;

    // Event
    CUDA_SAFE_CALL(hipEventRecord(pln->compInit, pln->stream),"Recording event: inpInit");

    // Blocks of 1024 threads ( the maximum number of threads per block )
    dimBlock.x = 16;
    dimBlock.y = 16;
    dimBlock.z = 1;

    // One block per harmonic, thus we can sort input powers in Shared memory
    dimGrid.x = ceil(pln->noR/(float)dimBlock.x);
    dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

    // Call the kernel to normalise and spread the input data
    ffdotSwarm_ker<T><<<dimGrid, dimBlock, 0, pln->stream >>>(time(NULL), (candOpt*)pln->d_out, pln->d_inp, fft->idx, fft->nor, pln->noHarms, 10, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, norm);

    CUDA_SAFE_CALL(hipGetLastError(), "Calling the ffdot_ker kernel.");

    // Event
    CUDA_SAFE_CALL(hipEventRecord(pln->compCmp, pln->stream),"Recording event: inpInit");
  }

  //hipDeviceSynchronize();          // TMP

  if ( pln->noZ*pln->noR*sizeof(candOpt) > pln->outSz )
  {
    fprintf(stderr,"ERROR, not enough space for output!\n");
  }
  else
  {
    CUDA_SAFE_CALL(hipEventRecord(pln->outInit, pln->stream),"Recording event: outInit");
    CUDA_SAFE_CALL(hipMemcpyAsync(pln->h_out, pln->d_out, pln->noZ*pln->noR*sizeof(candOpt), hipMemcpyDeviceToHost, pln->stream ), "Copying optimisation results back from the device.");
    CUDA_SAFE_CALL(hipEventRecord(pln->outCmp, pln->stream),"Recording event: outCmp");
  }

  int TMPP = 0;
}

void rz_interp_cu(fcomplex* fft, int loR, int noBins, double centR, double centZ, int halfwidth)
{
  FOLD // TMP: CPU equivalent  .
  {
    double total_power = 0.;
    double powargr, powargi;
    fcomplex ans;

    rz_interp((fcomplex*)fft, noBins, centR, centZ, halfwidth, &ans);
  }

  float *cuPowers;
  fcomplexcu *cuInp;
  fcomplexcu *cpuInp;
  int     rOff, lodata;
  double factor;
  double log2 = log(2.0);

  //halfwidth       = z_resp_halfwidth(fabs(centZ), HIGHACC);
  int noInp       = 2*halfwidth;
  lodata          = floor( centR ) - halfwidth ;
  rOff            = lodata - loR ;

  FOLD // Clamp size  .
  {
    if ( lodata < 0 )
    {
      noInp         += lodata;
      rOff          -= lodata;
    }

    if ( rOff + noInp >= noBins )
    {
      fprintf(stderr, "WARNING: attempting to do a f-∂f interpolation beyond the end of the FFT.\n");
      noInp = noBins - rOff;
    }
  }

  FOLD // GPU Memory operations  .
  {
    CUDA_SAFE_CALL(hipMalloc((void** )&cuInp, noInp * sizeof(hipfftComplex) ),   "Failed to allocate device memory for kernel stack.");
    CUDA_SAFE_CALL(hipMemcpy(cuInp, &fft[rOff], noInp * sizeof(hipfftComplex), hipMemcpyHostToDevice), "Copying convolution kernels between devices.");
  }

  FOLD // Calculate normalisation factor  .
  {
    float*  normPow = (float*) malloc(noInp*sizeof(float));

    for ( int i = 0; i < noInp; i++ )
    {
      normPow[i] = POWERR(fft[rOff+i].r, fft[rOff+i].i ) ;
    }

    float medianv   = median(normPow, noInp);
    factor          = sqrt(medianv/log2);

    free(normPow);
  }

  FOLD // Call kernel  .
  {
    dim3 dimBlock, dimGrid;

    // Blocks of 1024 threads ( the maximum number of threads per block )
    dimBlock.x = 1;
    dimBlock.y = 1;
    dimBlock.z = 1;

    // One block per harmonic, thus we can sort input powers in Shared memory
    dimGrid.x = 1;
    dimGrid.y = 1;

    // Call the kernel to normalise and spread the input data
    rz_interp_ker<<<dimGrid, dimBlock, 0, 0>>>(centR, centZ, cuInp, rOff, noInp, halfwidth, factor);

    //hipDeviceSynchronize();          // TMP
    int TMPP = 0;
  }
}

template<typename T>
void opt_candByPln(accelcand* cand, fftInfo* fft, cuOptCand* pln, int noP, double scale, int plt = -1, int nn = 0 )
{
  FOLD // Large points  .
  {
    pln->centR          = cand->r;
    pln->centZ          = cand->z;
    pln->noZ            = noP*2 + 1;
    pln->noR            = noP*2 + 1;
    pln->rSize          = scale;
    pln->zSize          = scale*4.0;

    //          gettimeofday(&start, NULL);       // TMP

    ffdotPln<T>(pln, fft);

    //          gettimeofday(&end, NULL);         // TMP
    //          timev1 = ((end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec)); // TMP
    //          printf("%.5f\t",timev1);          // TMP
  }

  // A blocking synchronisation to ensure results are ready to be proceeded by the host
  CUDA_SAFE_CALL(hipEventSynchronize(pln->outCmp), "ERROR: copying result from device to host.");

  if ( pltOpt > 0 ) // Write CVS & plot output  .
  {
    nvtxRangePush("Write CVS");

    char tName[1024];
    sprintf(tName,"/home/chris/accel/Cand_%05i_Rep_%02i_h%02i.csv", nn, plt, cand->numharm );
    FILE *f2 = fopen(tName, "w");

    fprintf(f2,"%i",pln->noHarms);

    for (int indx = 0; indx < pln->noR ; indx++ )
    {
      double r = pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
      fprintf(f2,"\t%.6f",r);
    }
    fprintf(f2,"\n");

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      double z = pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;

      fprintf(f2,"%.6f",z);

      for (int indx = 0; indx < pln->noR ; indx++ )
      {
        float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
        fprintf(f2,"\t%.15f",yy2);
      }
      fprintf(f2,"\n");
    }
    fclose(f2);

    FOLD // Make image  .
    {
      nvtxRangePush("Image");
      char cmd[1024];
      sprintf(cmd,"python ~/bin/bin/plt_ffd.py %s", tName);
      system(cmd);
      nvtxRangePop();
    }

    nvtxRangePop();

    int tmp = 0;
  }

  FOLD // Get new max  .
  {
    nvtxRangePush("Get Max");

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      for (int indx = 0; indx < pln->noR ; indx++ )
      {
        float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
        if ( yy2 > cand->power )
        {
          cand->power   = yy2;
          cand->r       = pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
          cand->z       = pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;
        }
      }
    }

    nvtxRangePop();
  }
}

template<typename T>
void opt_candBySwrm(accelcand* cand, fftInfo* fft, cuOptCand* pln, int noP, double scale, int plt = -1, int nn = 0 )
{
  FOLD // Large points  .
  {
    pln->centR          = cand->r;
    pln->centZ          = cand->z;
    pln->noZ            = noP*2 + 1;
    pln->noR            = noP*2 + 1;
    pln->rSize          = scale;
    pln->zSize          = scale*4.0;

    ffdotSwrm<T>(pln, fft);
  }

  // A blocking synchronisation to ensure results are ready to be proceeded by the host
  CUDA_SAFE_CALL(hipEventSynchronize(pln->outCmp), "ERROR: copying result from device to host.");

  FOLD // Get new max  .
  {
    float max = ((candOpt*)pln->h_out)[0].power;

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      for (int indx = 0; indx < pln->noR ; indx++ )
      {
        float yy2 = ((candOpt*)pln->h_out)[indy*pln->noR+indx].power;
        if ( yy2 > max )
        {
          max = yy2;
          cand->r     = ((candOpt*)pln->h_out)[indy*pln->noR+indx].r;
          cand->z     = ((candOpt*)pln->h_out)[indy*pln->noR+indx].z;
          cand->power = yy2;

          //printf("New max at %04i r: %15.3f   z: %15.3f \n", indy*pln->noR+indx, cand->r, cand->z );
        }
      }
    }
  }

  //printf("Best point Current r: %10.5f z: %10.5f  power: %20.6f \n", cand->r, cand->z, cand->power);
}

void opt_candPlns(accelcand* cand, accelobs* obs, int nn, cuOptCand* pln)
{
  int ii;
  int *r_offset;
  fcomplex **data;
  double r, z;
  int noP;
  float scale;

  struct timeval start, end, start1, end1;
  double timev1, timev2, timev3;

  int numdata   = obs->numbins;

  cand->pows    = gen_dvect(cand->numharm);
  cand->hirs    = gen_dvect(cand->numharm);
  cand->hizs    = gen_dvect(cand->numharm);
  cand->derivs  = (rderivs *)  malloc(sizeof(rderivs) * cand->numharm);
  r_offset      = (int*) malloc(sizeof(int)*cand->numharm);
  data          = (fcomplex**) malloc(sizeof(fcomplex*)*cand->numharm);

  pln->centR    = cand->r ;
  pln->centZ    = cand->z ;
  pln->noHarms  = cand->numharm ;

  fftInfo fft;
  fft.fft       = obs->fft;
  fft.rlo       = obs->lobin;
  fft.nor       = obs->numbins;
  fft.idx       = obs->lobin;
  fft.rhi       = obs->lobin + obs->numbins;

  //printf("%4i  optimize_accelcand  harm %2i   r %20.4f   z %7.3f  pow: %8.3f \n", nn, pln->noHarms, pln->centR, pln->centZ, 0 );

  for ( int i=1; i <= cand->numharm; i++ )
  {
    pln->norm[i-1]  = get_scaleFactorZ(fft.fft, fft.nor, (fft.idx+pln->centR)*i-fft.rlo, pln->centZ*i, 0.0);
  }

  if ( obs->use_harmonic_polishing )
  {
    if ( obs->mmap_file || obs->dat_input )
    {
      for( ii=0; ii<cand->numharm; ii++ )
      {
        r_offset[ii]   = obs->lobin;
        data[ii]       = obs->fft;
      }

      FOLD // GPU grid  .
      {
        int rep       = 0;
        int lrep      = 0;
        int noP       = 30;
        float snoop   = 0.3;
        float sz;
        float v1, v2;

        if ( cand->numharm == 1  )
          sz = optSz01;
        if ( cand->numharm == 2  )
          sz = optSz02;
        if ( cand->numharm == 4  )
          sz = optSz04;
        if ( cand->numharm == 8  )
          sz = optSz08;
        if ( cand->numharm == 16 )
          sz = optSz16;

        //printf("\n%03i  r: %15.6f   z: %12.6f \n", nn, cand->r, cand->z);

        pln->halfWidth = 0;

        if ( optpln01 > 0 )
        {
          noP           = optpln01 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;
            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            if ( ++lrep > 10 )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale;
        }

        if ( optpln02 > 0 )
        {
          noP           = optpln02 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;
            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            if ( ++lrep > 10 )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale;
        }

        if ( optpln03 > 0 )
        {
          noP           = optpln03 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;
            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            if ( ++lrep > 10 )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale*2;
        }

        if ( optpln04 > 0 )
        {
          noP           = optpln04 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            //printf("cand->r: %.9f  cand->z: %.9f  sz   %.9f\n", cand->r, cand->z, sz );

            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;

//            noP       = 100;
//            cand->r   = 52.567083 ;
//            cand->r   = 52.5 ;
//            cand->z   = 0 ;
//            sz        = 0.5;

            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            if ( ++lrep > 10 )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale*2;
        }

        if ( optpln05 > 0 )
        {
          noP           = optpln05 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            //printf("cand->r: %.9f  cand->z: %.9f  sz   %.9f\n", cand->r, cand->z, sz );
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;

//            noP       = 100;
//            cand->r   = 184.06;
//            cand->z   = 0.32;
//            sz        = 0.01;

            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            if ( ++lrep > 10 )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale*2;
        }

        if ( optpln06 > 0 )
        {
          noP           = optpln06 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;

            opt_candByPln<double>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            if ( ++lrep > 10 )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale*2;
        }

        int tmp = 0;
      }

      FOLD // Optimise derivatives  .
      {
        nvtxRangePush("Opt derivs");

        optemiseDerivs(data, cand->numharm, r_offset, numdata, cand->r, cand->z, cand->derivs, cand->pows, nn);

        for( ii=0; ii < cand->numharm; ii++ )
        {
          cand->hirs[ii]=(cand->r+obs->lobin)*(ii+1);
          cand->hizs[ii]=cand->z*(ii+1);
        }

        FOLD // Update fundamental values to the optimised ones
        {
          cand->power = 0;
          for( ii=0; ii < cand->numharm; ii++ )
          {
            cand->power += cand->derivs[ii].pow/cand->derivs[ii].locpow;
          }
        }

        int noStages = log2((double)cand->numharm);
        cand->sigma = candidate_sigma(cand->power, cand->numharm, obs->numindep[noStages]);

        nvtxRangePop();
      }

    }
  }
}

void opt_candSwrm(accelcand* cand, accelobs* obs, int nn, cuOptCand* pln)
{
  int ii;
  int *r_offset;
  fcomplex **data;

  int numdata   = obs->numbins;

  cand->pows    = gen_dvect(cand->numharm);
  cand->hirs    = gen_dvect(cand->numharm);
  cand->hizs    = gen_dvect(cand->numharm);
  cand->derivs  = (rderivs *)  malloc(sizeof(rderivs) * cand->numharm);
  r_offset      = (int*) malloc(sizeof(int)*cand->numharm);
  data          = (fcomplex**) malloc(sizeof(fcomplex*)*cand->numharm);

  pln->centR    = cand->r ;
  pln->centZ    = cand->z ;
  pln->noHarms  = cand->numharm ;

  fftInfo fft;
  fft.fft       = obs->fft;
  fft.rlo       = obs->lobin;
  fft.nor       = obs->numbins;
  fft.idx       = obs->lobin;
  fft.rhi       = obs->lobin + obs->numbins;

  for ( int i=1; i <= cand->numharm; i++ )
  {
    pln->norm[i-1]  = get_scaleFactorZ(fft.fft, fft.nor, (fft.idx+pln->centR)*i-fft.rlo, pln->centZ*i, 0.0);
  }

  if ( obs->use_harmonic_polishing )
  {
    if ( obs->mmap_file || obs->dat_input )
    {
      for( ii=0; ii<cand->numharm; ii++ )
      {
        r_offset[ii]   = obs->lobin;
        data[ii]       = obs->fft;
      }

      FOLD // GPU swarm  .
      {
        int rep = 0;
        int noP = 20;
        float sz;

        if ( cand->numharm == 1 )
          sz = 16;
        if ( cand->numharm == 2 )
          sz = 14;
        if ( cand->numharm == 4 )
          sz = 12;
        if ( cand->numharm == 8 )
          sz = 10;
        if ( cand->numharm == 16 )
          sz = 8;

        //printf("\n%03i  r: %15.6f   z: %12.6f \n", nn, cand->r, cand->z);

        opt_candBySwrm<float>(cand, &fft, pln, noP, sz,  rep++, nn );

        int tmp = 0;
      }

      FOLD // Optimise derivatives  .
      {
        optemiseDerivs(data, cand->numharm, r_offset, numdata, cand->r, cand->z, cand->derivs, cand->pows, nn);

        for( ii=0; ii < cand->numharm; ii++ )
        {
          cand->hirs[ii]=(cand->r+obs->lobin)*(ii+1);
          cand->hizs[ii]=cand->z*(ii+1);
        }

        FOLD // Update fundamental values to the optimised ones
        {
          cand->power = 0;
          for( ii=0; ii < cand->numharm; ii++ )
          {
            cand->power += cand->derivs[ii].pow/cand->derivs[ii].locpow;
          }
        }

        int noStages = log2((double)cand->numharm);
        cand->sigma = candidate_sigma(cand->power, cand->numharm, obs->numindep[noStages]);
      }

      //printf("Opt point          r: %10.5f z: %10.5f  power: %20.6f   sigma: %6.3f \n", cand->r, cand->z, cand->power, cand->sigma);
    }
  }
}
