#include "hip/hip_runtime.h"
/** @file cuda_cand_OPT.cu
 *  @brief Utility functions and kernels for GPU optimisation
 *
 *
 *  @author Chris Laidler
 *  @bug No known bugs.
 *
 *  Change Log
 *
 *  [0.0.01] []
 *    Beginning of change log
 *    Working version un-numbed
 *
 *  [0.0.02] [2017-02-16]
 *    Separated candidate and optimisation CPU threading
 *
 */

#include <hiprand.h>
#include <math.h>		// log
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <stdint.h>		// uint64_t

#include "cuda_math_ext.h"
#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"
#include "candTree.h"
#include "cuda_response.h"
#include "cuda_cand_OPT.h"
#include "cuda_accel_PLN.h"


#define SCALE_AUT       (1000000000)

extern "C"
{
#define __float128 long double
#include "accel.h"
}



#define		NM_BEST		0
#define		NM_MIDL		1
#define		NM_WRST		2


#define SWAP_PTR(p1, p2) do { initCand* tmp = p1; p1 = p2; p2 = tmp; } while (0)


template<typename T>
T pow(double r, double z, int numharm, cuHarmInput* inp)
{
  int halfW;

  T total_power  = 0;
  T real = 0;
  T imag = 0;

  for( int hIdx = 1; hIdx <= numharm; hIdx++ )
  {
    // Determine half width - high precision
    halfW = cu_z_resp_halfwidth_high<float>(z*hIdx);

    rz_convolution_cu<T, float2>(&((float2*)inp->h_inp)[(hIdx-1)*inp->stride], inp->loR[hIdx-1], inp->stride, r*hIdx, z*hIdx, halfW, &real, &imag);

    total_power     += POWERCU(real, imag);
  }

  return total_power;
}

template<typename T>
T pow(initCand* cand, cuHarmInput* inp)
{
  double total_power = pow<T>(cand->r, cand->z, cand->numharm, inp);

  cand->power =  total_power;

  return total_power;
}

template<typename T>
T pow(accelcand* cand, cuHarmInput* inp)
{
  double total_power = pow<T>(cand->r, cand->z, cand->numharm, inp);

  cand->power =  total_power;

  return total_power;
}


/** Check if the plane, with current settings, requires new input
 *
 * This does not load the actual input
 * This check the input in the input data structure of the plane
 *
 * @param pln     The plane to check, current settings ( centZ, centR, zSize, rSize, etc.) used
 * @param fft     The FFT data that will make up the input
 * @param newInp  Set to 1 if new input is needed
 * @return        ACC_ERR_NONE on success or a collection of error values if full or partial failure
 */
ACC_ERR_CODE chkInput_cand( initCand* cand, cuHarmInput* input, fftInfo* fft, double rSize, double zSize, int* newInp )
{
  return  chkInput(input, cand->r, cand->z, rSize, zSize, cand->numharm, newInp);
}

/** Copy relevant input from FFT to data structure normalising as needed
 *
 *  Note this contains a blocking synchronisation to make sure the pinned host memory is free
 *
 * @param pln     The plane to check
 * @param fft     The FFT data that will make up the input
 * @return        ACC_ERR_NONE on success or a collection of error values if full or partial failure
 */
ACC_ERR_CODE prepInput_cand( initCand* cand, cuHarmInput* input, fftInfo* fft, double rSize, double zSize, int64_t flags )
{
  return loadHostHarmInput(input, fft, cand->r, cand->z, rSize, zSize, cand->numharm, flags, NULL );
}

/** Make sure the input is for the current plane settings is ready in device memory
 *
 * This checks if new memory is needed
 * Normalises it and copies it to the device
 *
 * @param pln	  optimiser
 * @param fft	  FFT data structure
 * @param newInp  Set to 1 if new input is needed
 * @return        ACC_ERR_NONE on success or a collection of error values if full or partial failure
 */
ACC_ERR_CODE prepInput_cand( initCand* cand, cuHarmInput* input, fftInfo* fft, double rSize, double zSize, int* newInp, int64_t flags )
{
  ACC_ERR_CODE	err		= ACC_ERR_NONE;

  // Check input
  int newInp_l;
  err += chkInput_cand( cand, input, fft, rSize, zSize, &newInp_l );

  if ( newInp_l )
  {
    // load normalised data into host memory
    err += prepInput_cand( cand, input, fft, rSize, zSize, flags );
  }

  if ( newInp )
    *newInp = newInp_l;

  return err;
}

candTree* opt_cont(candTree* oTree, cuPlnGen* pln, container* cont, fftInfo* fft, int nn)
{
  //  PROF // Profiling  .
  //  {
  //    NV_RANGE_PUSH("opt_cont");
  //  }
  //
  //  confSpecsGen*  sSpec   = pln->cuSrch->sSpec;
  //  initCand* iCand 	= (initCand*)cont->data;

  //
  //  optInitCandLocPlns(iCand, pln, nn );
  //
  //  accelcand* cand = new accelcand;
  //  memset(cand, 0, sizeof(accelcand));
  //
  //  int lrep      = 0;
  //  int noP       = 30;
  //  float snoop   = 0.3;
  //  float sz;
  //  float v1, v2;
  //
  //  const int mxRep = 10;
  //
  //  initCand* canidate = (initCand*)cont->data;
  //
  //  candTree* thisOpt = new candTree;
  //
  //  if ( canidate->numharm == 1  )
  //    sz = conf->optPlnSiz[0];
  //  if ( canidate->numharm == 2  )
  //    sz = conf->optPlnSiz[1];
  //  if ( canidate->numharm == 4  )
  //    sz = conf->optPlnSiz[2];
  //  if ( canidate->numharm == 8  )
  //    sz = conf->optPlnSiz[3];
  //  if ( canidate->numharm == 16 )
  //    sz = conf->optPlnSiz[4];
  //
  //  //int numindep        = (obs->rhi - obs->rlo ) * (obs->zhi +1 ) * (ACCEL_DZ / 6.95) / pln->noHarms ;
  //
  //  //printf("\n%03i  r: %15.6f   z: %12.6f \n", nn, cand->r, cand->z);
  //
  //  pln->halfWidth = 0;
  //
  //  int plt = 0;
  //
  //  if ( optpln01 > 0 )
  //  {
  //    noP               = optpln01 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale;
  //  }
  //
  //  if ( optpln02 > 0 )
  //  {
  //    noP               = optpln02 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale;
  //  }
  //
  //  if ( optpln03 > 0 )
  //  {
  //    noP               = optpln03 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale*2;
  //  }
  //
  //  if ( optpln04 > 0 )
  //  {
  //    noP               = optpln04 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale*2;
  //  }
  //
  //  if ( optpln05 > 0 )
  //  {
  //    noP               = optpln05 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale*2;
  //  }
  //
  //  if ( optpln06 > 0 )
  //  {
  //    noP               = optpln06 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<double>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale*2;
  //  }
  //
  //  cont->flag |= OPTIMISED_CONTAINER;
  //
  //  NV_RANGE_POP();
  //  return thisOpt;
  return NULL;
}

/** Refine candidate location using repetitive planes
 *
 * @param cand
 * @param opt
 * @param noP
 * @param scale
 * @param plt
 * @param nn
 * @param lv
 * @return
 */
template<typename T>
ACC_ERR_CODE optRefinePosPln(initCand* cand, cuOpt* opt, int noP, double scale, int plt = -1, int nn = 0, int lv = 0 )
{
  ACC_ERR_CODE err = ACC_ERR_NONE;
  int newInput = 0;

  fftInfo*	fft	= opt->cuSrch->fft;
  confSpecsOpt*	conf	= opt->conf;
  cuRzHarmPlane* pln	= opt->plnGen->pln;

  // Number of harmonics to check, I think this could go up to 32!
  int maxHarms	= MAX(cand->numharm, conf->optMinLocHarms);

  FOLD // Generate plain points  .
  {
    pln->noZ		= noP;
    pln->noR		= noP;
    pln->rSize		= scale;
    pln->zSize		= scale*conf->zScale;

    err += centerPlaneOnCand(pln, cand);
    ERROR_MSG(err, "ERROR: Placing ffdot plane.");

    // Over ride the candidate number of harmonics (this must be done after centring the plane)
    pln->noHarms	= maxHarms;

    err += ffdotPln<T>(opt->plnGen, fft, &newInput);

    if ( newInput ) // Create the section of ff plane  .
    {
      // New input was used so don't maintain the old max, as different normalisation may cause minor differences making the powers incomparable
      cand->power	= 0;
    }
    ERROR_MSG(err, "ERROR: Generating f-fdot plane.");

    PROF // Profiling - Time components  .
    {
      if ( (opt->flags & FLAG_PROF) )
      {
  	infoMSG(5,5,"Time components");

  	// Time batch multiply
  	timeEvents( opt->plnGen->inpInit,  opt->plnGen->inpCmp,  &opt->compTime[COMP_OPT_H2D], "Copy H2D");
  	timeEvents( opt->plnGen->compInit, opt->plnGen->compCmp, &opt->compTime[COMP_OPT_PLN1+lv], "Optimisation plane calculations");
  	timeEvents( opt->plnGen->outInit,  opt->plnGen->outCmp,  &opt->compTime[COMP_OPT_D2H], "Copy D2H");
      }
    }
  }

  FOLD // Get new max  .
  {
    PROF // Profiling  .
    {
      NV_RANGE_PUSH("Get Max");
    }

    int noStrHarms = 0;
    if      ( pln->type == CU_STR_HARMONICS )
      noStrHarms = pln->noHarms;
    else if ( pln->type == CU_STR_INCOHERENT_SUM )
      noStrHarms = 1;
    else
    {
      infoMSG(6,6,"Plane type has not been initialised.\n" );
      err += ACC_ERR_UNINIT;
    }

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      for (int indx = 0; indx < pln->noR ; indx++ )
      {
	float yy2 = 0;

	for ( int hIdx = 0; hIdx < noStrHarms; hIdx++)
	{
	  if      ( pln->type == CU_CMPLXF )
	    yy2 +=  POWERF(((float2*)pln->h_data)[indy*pln->zStride + indx*noStrHarms + hIdx]);
	  else if ( pln->type == CU_FLOAT )
	    yy2 +=  ((float*)pln->h_data)[indy*pln->zStride + indx*noStrHarms + hIdx];
	  else
	  {
	    infoMSG(6,6,"Plane type has not been initialised.\n" );
	    err += ACC_ERR_DATA_TYPE;
	    break;
	  }
	}

	if ( yy2 > cand->power )
	{
	  cand->power	= yy2;
	  cand->r	= pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
	  cand->z	= pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;
	  if ( pln->noZ	== 1 )
	    cand->z = pln->centZ;
	  if ( pln->noR	== 1 )
	    cand->r = pln->centR;
	}
      }
    }

    infoMSG(4,4,"Max Power %8.5f at (%.6f %.6f)\n", cand->power, cand->r, cand->z);

    PROF // Profiling  .
    {
      NV_RANGE_POP("Get Max");
    }
  }

  FOLD // Write CVS & plot output  .
  {
#ifdef CBL
    if ( conf->flags & FLAG_DPG_PLT_OPT ) // Write CVS & plot output  .
    {
      // TODO: Check if we can get the dir name and then this can be added into standard accelsearch
      char tName[1024];
      sprintf(tName,"Cand_%05i_Rep_%02i_Lv_%i_h%02i.csv", nn, plt, lv, cand->numharm );

      ffdotPln_plotPln( pln, "/home/chris/accel/", tName );
    }
#endif
  }

  return err;
}

/** Refine candidate location using simplex
 *
 * @param cand
 * @param inp
 * @param rSize
 * @param zSize
 * @param plt
 * @param nn
 * @param lv
 * @return
 */
template<typename T>
ACC_ERR_CODE optInitCandPosSim(initCand* cand, cuHarmInput* inp, double rSize = 1.0, double zSize = 1.0, int plt = 0, int nn = 0, int lv = 0 )
{
  ACC_ERR_CODE err = ACC_ERR_NONE;

  infoMSG(3,3,"Simplex refine position - lvl %i  size %f by %f \n", lv+1, rSize, zSize);

  // These are the Nelder–Mead parameter values
  double reflect	= 1.0;
  double expand		= 2.0;
  double contract	= 0.4;
  double shrink		= 0.3;

  initCand  cnds[3];
  initCand* olst[3];

  initCand  centroid    = *cand;
  initCand  reflection  = *cand;
  initCand  expansion   = *cand;
  initCand  contraction = *cand;

  cnds[0] = *cand;
  cnds[1] = *cand;
  cnds[2] = *cand;

  pow<T>(&cnds[0], inp);
  double inpPow = cnds[0].power;

  cnds[1].r += rSize;
  pow<T>(&cnds[1], inp);

  cnds[2].z += zSize;
  pow<T>(&cnds[2], inp);

  olst[NM_BEST] = &cnds[0];
  olst[NM_MIDL] = &cnds[1];
  olst[NM_WRST] = &cnds[2];

  int ite = 0;
  double rtol;			///< Ratio of low to high

  infoMSG(4,4,"Start - Power: %8.3f at (%.6f %.6f)", cnds[0].power, cnds[0].r, cnds[0].z);

  while (1)
  {
    FOLD // Order
    {
      if (olst[NM_WRST]->power > olst[NM_MIDL]->power )
	SWAP_PTR(olst[NM_WRST], olst[NM_MIDL]);

      if (olst[NM_MIDL]->power > olst[NM_BEST]->power )
      {
	SWAP_PTR(olst[NM_MIDL], olst[NM_BEST]);

	if (olst[NM_WRST]->power > olst[NM_MIDL]->power )
	SWAP_PTR(olst[NM_WRST], olst[NM_MIDL]);
      }
    }

    FOLD // Centroid  .
    {
      centroid.r = ( olst[NM_BEST]->r + olst[NM_MIDL]->r ) / 2.0  ;
      centroid.z = ( olst[NM_BEST]->z + olst[NM_MIDL]->z ) / 2.0  ;
      // Don't calculate the power as it's not used
    }

    ite++;

    rtol = 2.0 * fabs(olst[NM_BEST]->power - olst[NM_WRST]->power) / (fabs(olst[NM_BEST]->power) + fabs(olst[NM_MIDL]->power) + 1.0e-15) ;

    if (rtol < 1.0e-7 )  // Within error so leave  .
    {
      break;
    }

    if ( ite == 100 )
    {
      break;
    }

    FOLD // Reflection  .
    {
      reflection.r = centroid.r + reflect*(centroid.r - olst[NM_WRST]->r ) ;
      reflection.z = centroid.z + reflect*(centroid.z - olst[NM_WRST]->z ) ;
      pow<T>(&reflection, inp);

      if ( olst[NM_BEST]->power <= reflection.power && reflection.power < olst[NM_MIDL]->power )
      {
	*olst[NM_WRST] = reflection;
	continue;
      }
    }

    FOLD // Expansion  .
    {
      if ( reflection.power > olst[NM_BEST]->power )
      {
	expansion.r = centroid.r + expand*(reflection.r - centroid.r ) ;
	expansion.z = centroid.z + expand*(reflection.z - centroid.z ) ;
	pow<T>(&expansion, inp);

	if (expansion.power > reflection.power)
	{
	  *olst[NM_WRST] = expansion;
	}
	else
	{
	  *olst[NM_WRST] = reflection;
	}
	continue;
      }
    }

    FOLD // Contraction  .
    {
      contraction.r = centroid.r + contract*(olst[NM_WRST]->r - centroid.r) ;
      contraction.z = centroid.z + contract*(olst[NM_WRST]->z - centroid.z) ;
      pow<T>(&contraction, inp);

      if ( contraction.power > olst[NM_WRST]->power )
      {
	*olst[NM_WRST] = contraction;
	continue;
      }
    }

    FOLD // Shrink  .
    {
      olst[NM_MIDL]->r = olst[NM_BEST]->r + shrink*(olst[NM_MIDL]->r - olst[NM_BEST]->r);
      olst[NM_MIDL]->z = olst[NM_BEST]->z + shrink*(olst[NM_MIDL]->z - olst[NM_BEST]->z);
      pow<T>(olst[NM_MIDL], inp);

      olst[NM_WRST]->r = olst[NM_BEST]->r + shrink*(olst[NM_WRST]->r - olst[NM_BEST]->r);
      olst[NM_WRST]->z = olst[NM_BEST]->z + shrink*(olst[NM_WRST]->z - olst[NM_BEST]->z);
      pow<T>(olst[NM_WRST], inp);
    }
  }

  double dist = sqrt( (cand->r-olst[NM_BEST]->r)*(cand->r-olst[NM_BEST]->r) + (cand->z-olst[NM_BEST]->z)*(cand->z-olst[NM_BEST]->z) );
  double powInc  = olst[NM_BEST]->power - inpPow;

  cand->r = olst[NM_BEST]->r;
  cand->z = olst[NM_BEST]->z;
  cand->power = olst[NM_BEST]->power;

  infoMSG(4,4,"End   - Power: %8.3f at (%.6f %.6f) %3i iterations moved %9.7f  power inc: %9.7f", cand->power, cand->r, cand->z, ite, dist, powInc);

  return err;
}

/** Initiate a optimisation plane
 * If oPln has not been pre initialised and is NULL it will create a new data structure.
 * If oPln has been pre initialised the device ID and Idx are used!
 *
 */
cuOpt* initOptimiser(cuSearch* sSrch, cuOpt* opt, gpuInf* gInf )
{
  confSpecsOpt*	conf	= sSrch->conf->opt;

  infoMSG(5,5,"Initialising optimiser.\n");

  int	maxHarms	= MAX(sSrch->noSrchHarms, conf->optMinLocHarms);
  
  if (!opt)
  {
    infoMSG(5,5,"Allocating new optimiser\n");
    opt = new cuOpt;
    memset(opt, 0, sizeof(cuOpt));
  }

  FOLD // Create all sub structures  .
  {
    opt->cuSrch		= sSrch;					// Set the pointer t the search specifications
    opt->conf		= conf;						// Should this rather be a duplicate?
    opt->gInf 		= gInf;

    if      ( conf->flags & FLAG_OPT_NM )
    {
      opt->input	= initHarmInput(20, sSrch->sSpec->zMax, maxHarms, gInf);
    }
    else if ( conf->flags & FLAG_OPT_SWARM )
    {
      fprintf(stderr,"ERROR: Particle swarm optimisation has been removed.\n");
      exit(EXIT_FAILURE);
    }
    else // Default use planes
    {
      opt->plnGen	= initPlnGen(maxHarms, sSrch->sSpec->zMax, conf, gInf);
    }
  }

  FOLD // Allocate struct specify memory  .
  {
    int sz = sizeof(long long)*(COMP_OPT_MAX) ;
    opt->compTime       = (long long*)malloc(sz);
    memset(opt->compTime,    0, sz);
  }

  return opt;
}

/** Free individual optimiser
 *
 * @param opt	The optimisers
 * @return
 */
ACC_ERR_CODE freeOptimiser(cuOpt* opt)
{
  ACC_ERR_CODE err	= ACC_ERR_NONE;

  err += freePlnGen(opt->plnGen);

  freeNull(opt->compTime);

  return err;
}

/** Create multiplication kernel and allocate memory for planes on all devices  .
 *
 * Create the kernels on the first device and then copy it to all others
 *
 * @param sSrch     A pointer to the search structure
 *
 * @return
 */
ACC_ERR_CODE initOptimisers(cuSearch* sSrch )
{
  ACC_ERR_CODE err = ACC_ERR_NONE;

  infoMSG(2,2,"Initialise all optimisers.\n");

  sSrch->oInf = new cuOptInfo;
  memset(sSrch->oInf, 0, sizeof(cuOptInfo));

  CUDA_SAFE_CALL(hipGetLastError(), "Entering initOptimisers.");

  double halfWidth = cu_z_resp_halfwidth<double>(sSrch->sSpec->zMax+10, HIGHACC)+10;	// Candidate may be on the z-max border so buffer a bit

  FOLD // Create the primary stack on each device, this contains the kernel  .
  {
    PROF // Profiling  .
    {
      NV_RANGE_PUSH("Init Optimisers");
    }

    FOLD // Determine the number of optimisers to make
    {
      sSrch->oInf->noOpts = 0;
      for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) // Loop over devices  .
      {
	if ( sSrch->gSpec->noDevOpt[dev] <= 0 )
	{
	  // Use the default of 4
	  sSrch->gSpec->noDevOpt[dev] = 4;

	  infoMSG(5,5,"Using the default %i optimisers per GPU.\n", sSrch->gSpec->noDevOpt[dev]);
	}
	sSrch->oInf->noOpts += sSrch->gSpec->noDevOpt[dev];
      }
    }

    infoMSG(5,5,"Initialising %i optimisers on %i devices.\n", sSrch->oInf->noOpts, sSrch->gSpec->noDevices);

    // Initialise the individual optimisers
    sSrch->oInf->opts = (cuOpt*)malloc(sSrch->oInf->noOpts*sizeof(cuOpt));
    memset(sSrch->oInf->opts, 0, sSrch->oInf->noOpts*sizeof(cuOpt));

    int idx = 0;
    for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) // Loop over devices  .
    {
      for ( int oo = 0 ; oo < sSrch->gSpec->noDevOpt[dev]; oo++ )
      {
	initOptimiser(sSrch, &sSrch->oInf->opts[idx], &sSrch->gSpec->devInfo[dev] );
	sSrch->oInf->opts[idx].pIdx = idx;

	idx++;
      }
    }

    PROF // Profiling  .
    {
      NV_RANGE_POP("Init Optimisers");
    }
  }
  
  return err;
}

/** Free all the optimisers of a search  .
 *
 * @param sSrch
 * @return
 */
ACC_ERR_CODE freeOptimisers(cuSearch* sSrch )
{
  ACC_ERR_CODE err	= ACC_ERR_NONE;

  infoMSG(4,4,"Freeing all optimisers.\n");

  if ( sSrch->oInf )
  {
    if ( sSrch->oInf->opts )
    {
      int idx = 0;
      for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) // Loop over devices  .
      {
	for ( int oo = 0 ; oo < sSrch->gSpec->noDevOpt[dev]; oo++ )
	{
	  freeOptimiser(&sSrch->oInf->opts[idx] );
	  idx++;
	}
      }

      freeNull(sSrch->oInf->opts);
    }

    freeNull(sSrch->oInf);
  }

  return err;
}

/** Initialise all the optimisers for the entire search
 *
 * @param srch
 * @return
 */
cuSearch* initCuOpt(cuSearch* srch)
{
  PROF // Profiling  .
  {
    NV_RANGE_PUSH("Init CUDA optimisers");
  }

  if ( !srch->oInf )
  {
    initOptimisers( srch );
  }
  else
  {
    // TODO: Do a whole bunch of checks here!
    fprintf(stderr, "ERROR: %s has not been set up to handle a pre-initialised memory info data structure.\n", __FUNCTION__);
    exit(EXIT_FAILURE);
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP("Init CUDA optimisers");
  }

  return srch;
}

/** Optimise derivatives of a candidate  .
 *
 */
void* optCandDerivs(accelcand* cand, cuSearch* srch )
{
  int ii;
  struct timeval start, end;    // Profiling variables

  fftInfo*	fft	= srch->fft;
  confSpecsOpt*	conf	= srch->conf->opt;
  searchSpecs*	sSpec	= srch->sSpec;

  FOLD // Update fundamental values to the optimised ones  .
  {
    infoMSG(5,5,"DERIVS\n");

    float	maxSig		= 0;
    int		bestH		= 0;
    float	bestP		= 0;
    double  	sig		= 0; // can be a float
    long long	numindep;
    float	candHPower	= 0;
    int		noStages	= 0;
    int 	kern_half_width;
    double	locpow;
    double	real;
    double	imag;
    double	power;
    int		maxHarms  	= MAX(cand->numharm, conf->optMinRepHarms) ;

    PROF // Profiling  .
    {
      if ( !(!(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD)) )
      {
	NV_RANGE_PUSH("DERIVS");
      }

      if ( conf->flags & FLAG_PROF )
      {
	gettimeofday(&start, NULL);
      }
    }

    cand->power   = 0;

    // Set up candidate
    cand->pows    = gen_dvect(maxHarms);
    cand->hirs    = gen_dvect(maxHarms);
    cand->hizs    = gen_dvect(maxHarms);
    cand->derivs  = (rderivs *)   malloc(sizeof(rderivs)  * maxHarms  );

    // Initialise values
    for( ii=0; ii < maxHarms; ii++ )
    {
      cand->hirs[ii]  = cand->r*(ii+1);
      cand->hizs[ii]  = cand->z*(ii+1);
    }

    for( ii = 1; ii <= maxHarms; ii++ )			// Calculate derivatives, powers and sigma for all harmonics  .
    {
      if      ( conf->flags & FLAG_OPT_NRM_LOCAVE   )
      {
	locpow = get_localpower3d(fft->data, fft->noBins, cand->r*ii, cand->z*ii, 0.0);
      }
      else if ( conf->flags & FLAG_OPT_NRM_MEDIAN1D )
      {
	locpow = get_scaleFactorZ(fft->data, fft->noBins, cand->r*ii, cand->z*ii, 0.0);
      }
      else if ( conf->flags & FLAG_OPT_NRM_MEDIAN2D )
      {
	fprintf(stderr,"ERROR: 2D median normalisation has not been written yet.\n");
	exit(EXIT_FAILURE);
      }
      else
      {
	// No normalisation this is plausible but not recommended

	// TODO: This should error if it is out of bounds?
	locpow = 1;
      }

      if ( locpow )
      {
	kern_half_width   = cu_z_resp_halfwidth<double>(fabs(cand->z*ii), HIGHACC);

	rz_convolution_cu<double, float2>((float2*)fft->data, fft->firstBin, fft->noBins, cand->r*ii, cand->z*ii, kern_half_width, &real, &imag);

	// Normalised power
	power = POWERCU(real, imag) / locpow ;

	cand->pows[ii-1] = power;

	get_derivs3d(fft->data, fft->noBins, cand->r*ii, cand->z*ii, 0.0, locpow, &cand->derivs[ii-1] );

	cand->power	+= power;
	int numz 	= round(srch->conf->gen->zMax / srch->conf->gen->zRes) * 2 + 1;
	if ( numz == 1 )
	{
	  numindep	= (sSpec->searchRHigh - sSpec->searchRLow) / (double)(ii) ;
	}
	else
	{
	  numindep	= (sSpec->searchRHigh - sSpec->searchRLow) * (numz + 1) * ( srch->conf->gen->zRes / 6.95 ) / (double)(ii);
	}

	sig		= candidate_sigma_cu(cand->power, (ii), numindep );

	infoMSG(6,6,"Harm %2i  local power %6.3f, normalised power %8.3f,   sigma %5.2f \n", ii, locpow, power, sig );

	if ( sig > maxSig || ii == 1 )
	{
	  maxSig        = sig;
	  bestP         = cand->power;
	  bestH         = (ii);
	}

	if ( ii == cand->numharm )
	{
	  candHPower    = cand->power;

	  if ( !(conf->flags & FLAG_OPT_BEST) )
	  {
	    break;
	  }
	}
      }
    }

    // Final values
    if ( bestP && (conf->flags & FLAG_OPT_BEST) && ( maxSig > 0.001 ) )
    {
      cand->numharm	= bestH;
      cand->sigma	= maxSig;
      cand->power	= bestP;

      infoMSG(4,4,"Cand best val Sigma: %5.2f Power: %6.4f  %i harmonics summed.", maxSig, bestP, bestH);
    }
    else
    {
      cand->power	= candHPower;
      noStages		= log2((double)cand->numharm);
      numindep		= srch->numindep[noStages];
      cand->sigma	= candidate_sigma_cu(candHPower, cand->numharm, numindep);

      infoMSG(4,4,"Cand harm val Sigma: %5.2f Power: %6.4f  %i harmonics summed.", cand->sigma, cand->power, cand->numharm);
    }

    PROF // Profiling  .
    {
      if ( !(!(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD)) )
      {
	NV_RANGE_POP("DERIVS");
      }

      if ( conf->flags & FLAG_PROF )
      {
	gettimeofday(&end, NULL);
	float v1 =  (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec);

	// Thread (pthread) safe add to timing value
	pthread_mutex_lock(&srch->threasdInfo->candAdd_mutex);
	srch->timings[COMP_OPT_DERIVS] += v1;
	pthread_mutex_unlock(&srch->threasdInfo->candAdd_mutex);
      }
    }
  }

  return (NULL);
}

/** CPU process results
 *
 * This function is meant to be the entry of a separate thread
 *
 */
void* cpuProcess(void* ptr)
{
  struct timeval start, end;    // Profiling variables

  ACC_ERR_CODE	err	= ACC_ERR_NONE;
  candSrch*	res	= (candSrch*)ptr;
  cuSearch*	srch	= res->cuSrch;
  accelcand*	cand	= res->cand;
  confSpecsOpt*	conf	= srch->conf->opt;

  // Yes we use two different types of candidates =/
  initCand iCand;
  iCand.numharm		= cand->numharm;
  iCand.power		= cand->power;
  iCand.r		= cand->r;
  iCand.z		= cand->z;

  if ( conf->flags & FLAG_OPT_NM_REFINE )
  {
    FOLD // Prep input
    {
      double sz = 5;	// This size could be a configurable parameter
      err += prepInput_cand( &iCand, res->input, srch->fft, sz, sz*conf->zScale, NULL, conf->flags );
    }

    if ( !ERROR_MSG(err, "ERROR: Preparing input for fine NM refinement.") )
    {
      PROF // Profiling  .
      {
	if ( !(!(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD)) )
	{
	  NV_RANGE_PUSH("NM_REFINE");
	}

	if ( conf->flags & FLAG_PROF )
	{
	  gettimeofday(&start, NULL);
	}
      }

      // Run the NM
      optInitCandPosSim<double>(&iCand,  res->input, 0.0005, 0.0005*conf->optPlnScale );

      cand->r		= iCand.r;
      cand->z		= iCand.z;
      cand->power	= iCand.power;

      // Free thread specific input memory
      freeHarmInput(res->input);
      res->input = NULL;

      PROF // Profiling  .
      {
	if ( !(!(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD)) )
	{
	  NV_RANGE_POP("NM_REFINE");
	}

	if ( conf->flags & FLAG_PROF )
	{
	  gettimeofday(&end, NULL);
	  float v1 =  (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec);

	  // Thread (pthread) safe add to timing value
	  pthread_mutex_lock(&res->cuSrch->threasdInfo->candAdd_mutex);
	  srch->timings[COMP_OPT_REFINE_2] += v1;
	  pthread_mutex_unlock(&res->cuSrch->threasdInfo->candAdd_mutex);
	}
      }
    }
  }

  optCandDerivs(cand, srch);

  // Decrease the count number of running threads
  sem_trywait(&srch->threasdInfo->running_threads);

  free(res);

  return (NULL);
}

/** Optimise derivatives of a candidate Using the CPU  .
 * This usually spawns a separate CPU thread to do the sigma calculations
 */
ACC_ERR_CODE processCandDerivs(accelcand* cand, cuSearch* srch, cuHarmInput* inp = NULL, int candNo = -1)
{
  ACC_ERR_CODE	err		= ACC_ERR_NONE;

  infoMSG(2,2,"Calc Cand Derivatives. r: %.6f  z: %.6f  harm: %i  power: %.2f \n", cand->r, cand->z, cand->numharm, cand->power);

  candSrch*     thrdDat  = new candSrch;
  memset(thrdDat, 0, sizeof(candSrch));

  confSpecsOpt*	conf	= srch->conf->opt;

  thrdDat->cand		= cand;
  thrdDat->cuSrch	= srch;
  thrdDat->candNo	= candNo;

  if ( conf->flags & FLAG_OPT_NM_REFINE )
  {
    // Make a copy of the input data for the thread to use
    thrdDat->input = duplicateHostInput(inp);
  }

  PROF // Profiling  .
  {
    NV_RANGE_PUSH("Post Thread");
  }

  // Increase the count number of running threads
  sem_post(&srch->threasdInfo->running_threads);

  if ( !(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD) )  // Create thread  .
  {
    pthread_t thread;
    int  iret1 = pthread_create( &thread, NULL, cpuProcess, (void*) thrdDat);

    if (iret1)	// Check return status
    {
      fprintf(stderr,"Error - pthread_create() return code: %d\n", iret1);
      exit(EXIT_FAILURE);
    }
  }
  else                              // Just call the function  .
  {
    cpuProcess( (void*) thrdDat );
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP("Post Thread");
  }

  infoMSG(2,2,"Done");

  return err;
}

/** Optimise a candidate location using ffdot planes  .
 *
 * @param cand		The candidate to refine
 * @param pln		The plane data structure to use for the GPU position refinement
 * @param candNo	The index of the candidate being optimised
 */
ACC_ERR_CODE optInitCandLocPlns(initCand* cand, cuOpt* opt, int candNo )
{
  infoMSG(2,2,"Refine location by plain\n");

  PROF // Profiling  .
  {
    NV_RANGE_PUSH("Plns");
  }

  confSpecsOpt*	conf	= opt->conf;
  cuRzHarmPlane* pln	= opt->plnGen->pln;

  FOLD // Get best candidate location using iterative GPU planes  .
  {
    int depth;
    int noP;
    int rep	= 0;
    int lrep	= 0;
    bool doub	= false;
    const int	mxRep		= 10;
    const float moveBound	= 0.67;
    const float outBound	= 0.9;
    double sz;
    float posR, posZ;

    if ( cand->numharm == 1  )
      sz = conf->optPlnSiz[0];
    if ( cand->numharm == 2  )
      sz = conf->optPlnSiz[1];
    if ( cand->numharm == 4  )
      sz = conf->optPlnSiz[2];
    if ( cand->numharm == 8  )
      sz = conf->optPlnSiz[3];
    if ( cand->numharm == 16 )
      sz = conf->optPlnSiz[4];

    cand->power		= 0;				// Set initial power to zero

    for ( int lvl = 0; lvl < NO_OPT_LEVS; lvl++ )
    {
      noP		= conf->optPlnDim[lvl];		// Set in the defaults text file

      if ( opt->plnGen->accu != conf->accu[lvl])
      {
	opt->plnGen->accu	= conf->accu[lvl];
	cand->power		= 0;			// Reset cand power as we are now using a different half-width
      }

      lrep		= 0;
      depth		= 1;

      if ( ( lvl == NO_OPT_LEVS-1 ) || (sz < 0.002) || ( (sz < 0.03) && (abs(pln->centZ) < 0.05) ) )	// Potently force double precision
      {
	// If last plane is not 0, it will be done with double precision
	if (!doub)
	  cand->power = 0;

	doub = true;
      }

      if ( noP )					// Check if there are points in this plane ie. are we optimising position at this level  .
      {
	while ( (depth > 0) && (lrep < mxRep) )		// Recursively make planes at this scale  .
	{
	  infoMSG(3,3,"-----------------------------------------------------\n");

	  if ( doub )
	  {
	    infoMSG(3,3,"Generate double precision plane - lvl %i  depth: %i  iteration %2i  size: %6.4f  dimension: %4i\n", lvl+1, depth, lrep, sz, noP );

	    // Double precision
	    optRefinePosPln<double>(cand, opt, noP, sz,  rep++, candNo, lvl + 1 );
	  }
	  else
	  {
	    infoMSG(3,3,"Generate single precision plane - lvl %i  depth: %i  iteration %2i  size: %6.4f  dimension: %4i\n", lvl+1, depth, lrep, sz, noP );

	    // Standard single precision
	    optRefinePosPln<float>(cand, opt, noP, sz,  rep++, candNo, lvl + 1 );
	  }

	  posR = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
	  posZ = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

	  if ( posR || posZ )
	    infoMSG(4,4,"Plane max absolute offset at %.4f %.4f of plane.\n", posR, posZ );
	  else
	    infoMSG(4,4,"Plane max in same position as current.\n");

	  if ( posR > moveBound || posZ > moveBound )
	  {
	    if ( ( (posR > outBound) || (posZ > outBound) ) && ( depth < lvl+1) )
	    {
	      // Zoom out by half
	      sz *= conf->optPlnScale / 2.0 ;
	      depth++;
	      infoMSG(5,5,"Zoom out");
	    }
	    else
	    {
	      // we'r just going to move the plane
	      infoMSG(5,5,"Move plain");
	    }
	  }
	  else
	  {
	    double rRes = pln->rSize/(double)(pln->noR-1) ;

	    // Break condition
	    if ( rRes < 1e-5 )
	    {
	      infoMSG(5,5,"Break size is small enough\n");
	      break;
	    }

	    // Zoom in
	    sz /= conf->optPlnScale;
	    depth--;
	    infoMSG(5,5,"Zoom in\n");
	    if ( sz < 2.0*rRes )
	      sz = rRes*2.0;
	  }

	  ++lrep;
	}
      }
      else
      {
	if ( doub )
	  infoMSG(3,3,"Skip plane lvl %i (double precision)", lvl+1);
	else
	  infoMSG(3,3,"Skip plane lvl %i (single precision)", lvl+1);
      }
    }
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP("Plns");
  }
}

/** This is the main function called by external elements  .
 *
 * @param cand
 * @param pln
 * @param nn
 */
ACC_ERR_CODE opt_accelcand(accelcand* cand, cuOpt* opt, int candNo)
{
  ACC_ERR_CODE	err	= ACC_ERR_NONE;

  confSpecsOpt*  conf	= opt->conf;
  char Txt[128];

  PROF // Profiling  .
  {
    sprintf(Txt, "Opt Cand %03i", candNo);
    NV_RANGE_PUSH(Txt);
  }

  initCand iCand;				// plane refining uses an initial candidate data structure
  iCand.r 		= cand->r;
  iCand.z 		= cand->z;
  iCand.power		= cand->power;
  iCand.numharm 	= cand->numharm;

  FOLD // Refine position in ff space  .
  {
    struct timeval start, end;    // Profiling variables

    PROF // Profiling  .
    {
      NV_RANGE_PUSH("Refine pos");

      if ( conf->flags & FLAG_PROF )
      {
	gettimeofday(&start, NULL);
      }
    }

    if      ( conf->flags & FLAG_OPT_NM    )
    {
      double sz = 15;	// This size could be a configurable parameter
      prepInput_cand( &iCand, opt->input, opt->cuSrch->fft, sz, sz*conf->zScale, NULL, opt->flags );
      optInitCandPosSim<double>(&iCand, opt->input, 0.5, 0.5*conf->zScale);
    }
    else if ( conf->flags & FLAG_OPT_SWARM )
    {
      fprintf(stderr,"ERROR: Particle swarm optimisation has been removed.\n");
      exit(EXIT_FAILURE);
    }
    else // Default use planes
    {
      err += optInitCandLocPlns(&iCand, opt, candNo);
    }

    PROF // Profiling  .
    {
      NV_RANGE_POP("Refine pos");

      if ( conf->flags & FLAG_PROF )
      {
	gettimeofday(&end, NULL);
	float v1 =  (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec) ;

	// Thread (omp) safe add to timing value
#pragma omp atomic
	opt->cuSrch->timings[COMP_OPT_REFINE_1] += v1;
      }
    }
  }

  // Update the details of the final candidate from the updated initial candidate
  cand->r 		= iCand.r;
  cand->z 		= iCand.z;
  cand->power		= iCand.power;
  cand->numharm 	= iCand.numharm;

  FOLD // Optimise derivatives  .
  {
    err += processCandDerivs(cand, opt->cuSrch, opt->plnGen->input, candNo);
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP(Txt);
  }

  return err;
}

/** Optimise all the candidates in a list
 *
 * @param listptr
 * @param cuSrch
 * @return
 */
int optList(GSList *listptr, cuSearch* cuSrch)
{
  struct timeval start, end;
  slog.setCsvDeliminator('|'); // TMP

  TIME //  Timing  .
  {
      NV_RANGE_PUSH("GPU Kernels");
  }

  int numcands 	= g_slist_length(listptr);

  int ii	= 0;
  int comp	= 0;

#if	!defined(DEBUG) && defined(WITHOMP)   // Parallel if we are not in debug mode  .
  if ( cuSrch->conf->opt->flags & FLAG_SYNCH )
  {
    omp_set_num_threads(1);
  }
  else
  {
    omp_set_num_threads(cuSrch->oInf->noOpts);
  }
#pragma omp parallel
#endif	// !DEBUG && WITHOMP
  FOLD  	// Main GPU loop  .
  {
    accelcand *candGPU;

    int tid	= 0;
    int ti	= 0; // tread specific index

#ifdef	WITHOMP
    tid = omp_get_thread_num();
#endif	// WITHOMP

    cuOpt* opt = &(cuSrch->oInf->opts[tid]);

    setDevice(opt->gInf->devid) ;

    // Make sure all initialisation and other stuff on the device is complete
    CUDA_SAFE_CALL(hipDeviceSynchronize(), "Synchronising device before candidate generation");

    while (listptr)  // Main Loop  .
    {
#pragma omp critical
      FOLD  // Synchronous behaviour  .
      {
#ifndef  DEBUG
	if ( cuSrch->conf->opt->flags & FLAG_SYNCH )
#endif
	{
	  tid 		= ii % cuSrch->oInf->noOpts ;
	  opt 		= &(cuSrch->oInf->opts[tid]);
	  setDevice(opt->gInf->devid);
	}

	FOLD // Calculate candidate  .
	{
	  if ( listptr )
	  {
	    candGPU	= (accelcand *) (listptr->data);
	    listptr	= listptr->next;
	    ii++;
	    ti = ii;
#ifdef CBL
	    FOLD // TMP: This can get removed
	    {
	      candGPU->init_power    = candGPU->power;
	      candGPU->init_sigma    = candGPU->sigma;
	      candGPU->init_numharm  = candGPU->numharm;
	      candGPU->init_r        = candGPU->r;
	      candGPU->init_z        = candGPU->z;
	    }
#endif
	  }
	  else
	  {
	    candGPU = NULL;
	  }
	}
      }

      if ( candGPU ) // Optimise  .
      {
	infoMSG(2,2,"\nOptimising initial candidate %i/%i, Power: %.3f  Sigma %.2f  Harm %i at (%.3f %.3f)\n", ti, numcands, candGPU->power, candGPU->sigma, candGPU->numharm, candGPU->r, candGPU->z );

	accelcand candCPU = *candGPU; // TMP Duplicate canidate for comparison later

	opt_accelcand(candGPU, opt, ti);

#pragma omp atomic
	comp++;


	  Fout // DBG - compare results
	  {
#ifdef CBL
	  slog.csvWrite("idx",	"%5i",    ti);
	  slog.csvWrite("r",	"%15.6f", candCPU.r);
	  slog.csvWrite("z",	"%12.6f", candCPU.z);
	  slog.csvWrite("pow",	"%12.6f", candCPU.power);
	  slog.csvWrite("sig",	"%12.6f", candCPU.sigma);

	  int *r_offset;
	  fcomplex **data;
	  double r, z;

	  r_offset     = (int*) malloc(sizeof(int)*candCPU.numharm);
	  data         = (fcomplex**) malloc(sizeof(fcomplex*)*candCPU.numharm);

	  //optimize_accelcand(candCPU, &obs, ii+1);
	  candCPU.pows   = gen_dvect(candCPU.numharm*2);
	  candCPU.hirs   = gen_dvect(candCPU.numharm*2);
	  candCPU.hizs   = gen_dvect(candCPU.numharm*2);
	  candCPU.derivs = (rderivs *)  malloc(sizeof(rderivs) * candCPU.numharm);
	  //norm	      = gen_dvect(candCPU.numharm);

	  for( int ii=0; ii<candCPU.numharm; ii++ )
	  {
	    r_offset[ii]   = 0;
	    data[ii]       = opt->cuSrch->fft->data;
	    //norm[ii]		= 0;
	  }
	  max_rz_arr_harmonics(data,
	      candCPU.numharm,
	      r_offset,
	      opt->cuSrch->fft->noBins,
	      candCPU.r,
	      candCPU.z,
	      &r,
	      &z,
	      candCPU.derivs,
	      candCPU.pows,
	      opt->plnGen->input->norm);
	  candCPU.r = r;
	  candCPU.z = z;
	  candCPU.power = 0;
	  candCPU.sigma = 0;
	  FOLD
	  {
	    pow<double>(&candCPU, opt->plnGen->input);
	    pow<double>(candGPU, opt->plnGen->input);

	    int noStages	= log2((double)candGPU->numharm);
	    long long numindep	= cuSrch->numindep[noStages];
	    candGPU->sigma	= candidate_sigma_cu(candGPU->power, candGPU->numharm, numindep);
	    candCPU.sigma	= candidate_sigma_cu(candCPU.power,  candCPU.numharm,  numindep);
	  }

	  slog.csvWrite("GPU r",	"%15.6f", candGPU->r);
	  slog.csvWrite("GPU z",	"%12.6f", candGPU->z);
	  slog.csvWrite("GPU Pow",	"%12.6f", candGPU->power );
	  slog.csvWrite("GPU sig",	"%12.6f", candGPU->sigma );

	  slog.csvWrite("CPU r",	"%15.6f", candCPU.r );
	  slog.csvWrite("CPU z",	"%12.6f", candCPU.z );
	  slog.csvWrite("CPU pow",	"%12.6f", candCPU.power );
	  slog.csvWrite("CPU sig",	"%12.6f", candCPU.sigma );

	  double rDist = candCPU.r - candGPU->r ;
	  double zDist = candCPU.z - candGPU->z ;

	  slog.csvWrite("Dist",		"%12.6f", sqrt(rDist*rDist + zDist*zDist) );
	  slog.csvWrite("Pow diff",	"%12.6f", candGPU->power - candCPU.power  );
	  slog.csvWrite("Sig diff",	"%12.6f", candGPU->sigma - candCPU.sigma  );
	  slog.csvWrite("Neg Sig diff",	"%12.6f", -(candGPU->sigma - candCPU.sigma) );

	  slog.csvEndLine();
#endif
	  }

	if ( msgLevel == 0 )
	{
	  printf("\rGPU optimisation %5.1f%% complete   ", comp / (float)numcands * 100.0f );
	  fflush(stdout);
	}
      }
    }
  }

  printf("\rGPU optimisation %5.1f%% complete                      \n", 100.0f );

  TIME //  Timing  .
  {
    NV_RANGE_POP("GPU Kernels");
    gettimeofday(&start, NULL);
  }

  // Wait for CPU derivative threads to finish
  waitForThreads(&cuSrch->threasdInfo->running_threads, "Waiting for CPU threads to complete.", 200 );

  TIME //  Timing  .
  {
    gettimeofday(&end, NULL);
    cuSrch->timings[TIME_OPT_WAIT] += (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec);
  }

  return 0;
}



