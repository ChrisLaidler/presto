#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <math.h>             // log
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#include "cuda_math_ext.h"
#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"

#include <boost/math/special_functions/gamma.hpp>
#include <boost/math/special_functions/erf.hpp>
#include <boost/math/special_functions/binomial.hpp>

#define FTLIM 1e-6
//#define DLIM  0.4
#define DLIM  0.0

#define OPT_INP_BUF   10

extern "C"
{
#define __float128 long double
#include "accel.h"
}

int     optpln01  = 50;
int     optpln02  = 30;
int     optpln03  = 20;
int     optpln04  = 20;
int     optpln05  = 20;
int     optpln06  = 0;

float   downScale = 6;

float   optSz01   = 16;
float   optSz02   = 14;
float   optSz04   = 12;
float   optSz08   = 10;
float   optSz16   = 8;


__device__ inline double cos_t(double x)
{
  return cos(x);
}
__device__ inline float cos_t(float x)
{
  return cosf(x);
}

__device__ inline double sin_t(double x)
{
  return sin(x);
}
__device__ inline float sin_t(float x)
{
  return sinf(x);
}

__device__ inline double sqrt_t(double x)
{
  return sqrt(x);
}
__device__ inline float sqrt_t(float x)
{
  return sqrtf(x);
}

__device__ inline void sincos_t(double x, double* s, double* c )
{
  sincos(x, s, c);
}
__device__ inline void sincos_t(float x, float* s, float* c )
{
  sincosf(x, s, c);
}


template<typename T>
__device__ void fresnl(T xxa, T* ss, T* cc)
{
  T f, g, c, s, t, u;
  T x, x2;

  x       = fabs(xxa);
  x2      = x * x;

  if      ( x2 < 2.5625   )    	// Small so use a polynomial approximation  .
  {
    t     = x2 * x2;

    T t01 = t;
    T t02 = t01*t;
    T t03 = t02*t;
    T t04 = t03*t;
    T t05 = t04*t;
    T t06 = t05*t;
    T sn  = (T)3.18016297876567817986e11 + (T)-4.42979518059697779103e10*t01 + (T)2.54890880573376359104e9*t02  + (T)-6.29741486205862506537e7*t03  + (T)7.08840045257738576863e5 *t04 - (T)2.99181919401019853726e3  *t05;
    T sd  = (T)6.07366389490084639049e11 + (T) 2.24411795645340920940e10*t01 + (T)4.19320245898111231129e8*t02  + (T) 5.17343888770096400730e6*t03  + (T)4.55847810806532581675e4 *t04 + (T)2.81376268889994315696e2  *t05 + t06 ;
    T cn  = (T)9.99999999999999998822e-1 + (T)-2.05525900955013891793e-1*t01 + (T)1.88843319396703850064e-2*t02 + (T)-6.45191435683965050962e-4*t03 + (T)9.50428062829859605134e-6*t04 - (T)4.98843114573573548651e-8 *t05;
    T cd  = (T)1.00000000000000000118e0  + (T) 4.12142090722199792936e-2*t01 + (T)8.68029542941784300606e-4*t02 + (T) 1.22262789024179030997e-5*t03 + (T)1.25001862479598821474e-7*t04 + (T)9.15439215774657478799e-10*t05 + (T)3.99982968972495980367e-12*t06 ;

    *ss   = x * x2 * sn / sd;
    *cc   = x * cn / cd;
  }
  else if ( x  > 36974.0  )     // Asymptotic behaviour  .
  {
    *cc   = 0.5;
    *ss   = 0.5;
  }
  else                          // Auxiliary functions for large argument  .
  {
    x2    = x * x;
    t     = (T)PI * x2;
    u     = 1.0 / (t * t);
    t     = 1.0 / t;

    //    T fn  = (T)3.76329711269987889006e-20+((T)1.34283276233062758925e-16+((T)1.72010743268161828879e-13+((T)1.02304514164907233465e-10+((T)3.05568983790257605827e-8 +((T)4.63613749287867322088e-6+((T)3.45017939782574027900e-4+((T)1.15220955073585758835e-2+((T)1.43407919780758885261e-1+ (T)4.21543555043677546506e-1*u)*u)*u)*u)*u)*u)*u)*u)*u;
    //    T fd  = (T)1.25443237090011264384e-20+((T)4.52001434074129701496e-17+((T)5.88754533621578410010e-14+((T)3.60140029589371370404e-11+((T)1.12699224763999035261e-8 +((T)1.84627567348930545870e-6+((T)1.55934409164153020873e-4+((T)6.44051526508858611005e-3+((T)1.16888925859191382142e-1+((T)7.51586398353378947175e-1+u)*u)*u)*u)*u)*u)*u)*u)*u)*u ;
    //    T gn  = (T)1.86958710162783235106e-22+((T)8.36354435630677421531e-19+((T)1.37555460633261799868e-15+((T)1.08268041139020870318e-12+((T)4.45344415861750144738e-10+((T)9.82852443688422223854e-8+((T)1.15138826111884280931e-5+((T)6.84079380915393090172e-4+((T)1.87648584092575249293e-2+((T)1.97102833525523411709e-1+ (T)5.04442073643383265887e-1*u)*u)*u)*u)*u)*u)*u)*u)*u)*u ;
    //    T gd  = (T)1.86958710162783236342e-22+((T)8.39158816283118707363e-19+((T)1.38796531259578871258e-15+((T)1.10273215066240270757e-12+((T)4.60680728146520428211e-10+((T)1.04314589657571990585e-7+((T)1.27545075667729118702e-5+((T)8.14679107184306179049e-4+((T)2.53603741420338795122e-2+((T)3.37748989120019970451e-1+((T)1.47495759925128324529e0 +u)*u)*u)*u)*u)*u)*u)*u)*u)*u)*u ;

    T u01 = u;
    T u02 = u01*u;
    T u03 = u02*u;
    T u04 = u03*u;
    T u05 = u04*u;
    T u06 = u05*u;
    T u07 = u06*u;
    T u08 = u07*u;
    T u09 = u08*u;
    T u10 = u09*u;
    T u11 = u10*u;
    T fn  = (T)3.76329711269987889006e-20 + (T)1.34283276233062758925e-16*u01 + (T)1.72010743268161828879e-13*u02 + (T)1.02304514164907233465e-10*u03 + (T)3.05568983790257605827e-8 *u04 + (T)4.63613749287867322088e-6*u05 + (T)3.45017939782574027900e-4*u06 + (T)1.15220955073585758835e-2*u07 + (T)1.43407919780758885261e-1*u08 + (T)4.21543555043677546506e-1*u09;
    T fd  = (T)1.25443237090011264384e-20 + (T)4.52001434074129701496e-17*u01 + (T)5.88754533621578410010e-14*u02 + (T)3.60140029589371370404e-11*u03 + (T)1.12699224763999035261e-8 *u04 + (T)1.84627567348930545870e-6*u05 + (T)1.55934409164153020873e-4*u06 + (T)6.44051526508858611005e-3*u07 + (T)1.16888925859191382142e-1*u08 + (T)7.51586398353378947175e-1*u09 + u10;
    T gn  = (T)1.86958710162783235106e-22 + (T)8.36354435630677421531e-19*u01 + (T)1.37555460633261799868e-15*u02 + (T)1.08268041139020870318e-12*u03 + (T)4.45344415861750144738e-10*u04 + (T)9.82852443688422223854e-8*u05 + (T)1.15138826111884280931e-5*u06 + (T)6.84079380915393090172e-4*u07 + (T)1.87648584092575249293e-2*u08 + (T)1.97102833525523411709e-1*u09 + (T)5.04442073643383265887e-1*u10 ;
    T gd  = (T)1.86958710162783236342e-22 + (T)8.39158816283118707363e-19*u01 + (T)1.38796531259578871258e-15*u02 + (T)1.10273215066240270757e-12*u03 + (T)4.60680728146520428211e-10*u04 + (T)1.04314589657571990585e-7*u05 + (T)1.27545075667729118702e-5*u06 + (T)8.14679107184306179049e-4*u07 + (T)2.53603741420338795122e-2*u08 + (T)3.37748989120019970451e-1*u09 + (T)1.47495759925128324529e0 *u10 + u11 ;


    f     = 1.0 - u * fn / fd;
    g     =       t * gn / gd;

    t     = (T)PIBYTWO * x2;
    sincos(t, &s, &c);
    t     = (T)PI * x;

    *cc   = 0.5 + (f * s - g * c) / t;
    *ss   = 0.5 - (f * c + g * s) / t;
  }

  if (xxa < 0.0)                // Swap as function is antisymmetric  .
  {
    *cc   = -*cc;
    *ss   = -*ss;
  }
}

const double EPS    = std::numeric_limits<double>::epsilon();
const double FPMIN  = std::numeric_limits<double>::min()/EPS;


/** Generate the complex response value for Fourier f-dot interpolation  .
 *
 * This is based on gen_z_response in responce.c
 *
 * @param rx            The x index of the value in the kernel
 * @param z             The Fourier Frequency derivative (# of bins the signal smears over during the observation)
 * @param absz          Is the absolute value of z
 * @param roffset       Is the offset in Fourier bins for the full response (i.e. At this point, the response would equal 1.0)
 * @param numbetween    Is the number of points to interpolate between each standard FFT bin. (i.e. 'numbetween' = 2 = interbins, this is the standard)
 * @param numkern       Is the number of complex points that the kernel will contain.
 * @param rr            A pointer to the real part of the complex response for rx
 * @param ri            A pointer to the imaginary part of the complex response for rx
 */
template<typename T>
__device__ inline void gen_z_response(int rx, T z,  T absz, T numbetween, int numkern, float* rr, float* ri)
{
  int signz;
  T zd, r, xx, yy, zz, startr, startroffset;
  T fressy, frescy, fressz, frescz, tmprl, tmpim;
  T s, c, pibyz, cons, delta;

  T zT = z;
  T rT = r;

  startr        = 0 - (0.5 * zT);
  startroffset  = (startr < 0) ? 1.0 + modf(startr, &tmprl) : modf(startr, &tmprl);

  if (rx == numkern / 2.0 && startroffset < 1E-3 && absz < 1E-3)
  {
    T nr, ni;

    zz      = zT * zT;
    xx      = startroffset * startroffset;
    nr      = (T)1.0 - (T)0.16449340668482264365 * zz;
    ni      = (T)-0.5235987755982988731 * zT;
    nr      += startroffset * (T)1.6449340668482264365 * zT;
    ni      += startroffset * ((T)PI - (T)0.5167712780049970029 * zz);
    nr      += xx * ((T)-6.579736267392905746 + (T)0.9277056288952613070 * zz);
    ni      += xx * ((T)3.1006276680299820175 * zT);

    *rr     = nr;
    *ri     = ni;
  }
  else
  {
    /* This is evaluating Eq (39) in:
     * Ransom, Scott M., Stephen S. Eikenberry, and John Middleditch. "Fourier techniques for very long astrophysical time-series analysis." The Astronomical Journal 124.3 (2002): 1788.
     *
     * Where: qᵣ  is the variable r and represents the distance from the centre frequency
     *        |ṙ| is the variable z which is ḟ
     */

    signz   = (zT < 0.0) ? -1 : 1;
    zd      = signz * (T)SQRT2 / sqrt(absz);
    zd      = signz * sqrt(2.0 / absz);
    cons    = zd / 2.0;                             // 1 / sqrt(2*r')

    startr  += numkern / (T) (2 * numbetween);
    delta   = -1.0 / numbetween;
    r       = startr + rx * delta;

    pibyz   = (T)PI / zT;
    yy      = rT * zd;
    zz      = yy + zT * zd;
    xx      = pibyz * rT * rT;

    sincos_t(xx, &s, &c);
    fresnl<T>(yy, &fressy, &frescy);
    fresnl<T>(zz, &fressz, &frescz);

    tmprl   = signz * (frescz - frescy);
    tmpim   = fressy - fressz;

    *rr     =  (tmprl * c - tmpim * s) * cons;
    *ri     = -(tmprl * s + tmpim * c) * cons;
  }
}

/* This routine uses the correlation method to do a Fourier        */
/* complex interpolation at a single point in the f-fdot plane.    */
/* It does the correlations manually. (i.e. no FFTs)               */
/* Arguments:                                                      */
/*   'data' is a complex array of the data to be interpolated.     */
/*   'numdata' is the number of complex points (bins) in data.     */
/*   'r' is the Fourier frequency in data that we want to          */
/*      interpolate.  This can (and should) be fractional.         */
/*   'z' is the fdot to use (z=f-dot*T^2 (T is integration time)). */
/*   'kern_half_width' is the half-width of the kernel in bins.    */
/*   'ans' is the complex answer.                                  */
template<typename T>
__device__ fcomplexcu rz_interp_cu(fcomplexcu* data, int loR, int noBins, double r, double z, int kern_half_width)
{
  int numkern, intfreq;
  double  fracfreq;
  double  dintfreq;
  int signz;
  int ii, lodata;
  T absz, zd, q_r, xx, Yr, Zr, startr;
  T fressy, frescy, fressz, frescz;
  T s, c, pibyz, cons, sinc;
  T tR, tI;     // Response values

  T zT = z;
  //T rT = r;

  fcomplexcu inp;
  fcomplexcu ans;

  ans.r = 0.0;
  ans.i = 0.0;

  if ( r > 0 )
  {
    // Split 'r' into integer and fractional parts
    fracfreq          = modf(r, &dintfreq); // This has to be double precision
    intfreq           = (int) dintfreq;
    numkern           = 2 * kern_half_width;
    lodata            = intfreq - kern_half_width;

    // Set up values dependent on Z alone
    absz              = fabs(zT);
    startr            = fracfreq - (0.5 * z);
    signz             = (zT < 0.0) ? -1 : 1;
    zd                = signz * (T)SQRT2 / sqrt(absz);
    cons              = zd / 2.0;
    pibyz             = (T)PI / zT;
    startr            += kern_half_width;

    if ( absz < FTLIM )
    {
      //      const int ix        = blockIdx.x * blockDim.x + threadIdx.x;
      //      const int iy = blockIdx.y * blockDim.y + threadIdx.y;
      //
      //      if ( ix == 0 )
      //      {
      //        printf("absz < FTLIM   iy: %03i\n", iy);
      //      }
      //double v1   = r - lodata ;
      //startr      = v1;
      startr = (r - lodata);
    }

    FOLD // Clamp values to usable bounds  .
    {
      if ( lodata < 0 )
      {
        //printf("lodata < 0\n");
        numkern += lodata;
        startr  += lodata;
        lodata  = 0;
      }

      //printf("lodata: %i    loR: %i  \n", lodata, loR);
      lodata -= loR;

      //printf("lodata + numkern: %i  noR: %i \n", lodata + numkern, noBins );
      if ( lodata + numkern >= noBins )
      {
        //        printf("lodata + numkern >= noBins\n");
        //        printf("%i + %i >= %i\n", lodata, numkern, noBins );
        numkern = noBins - lodata;
      }

      //printf("numkern: %i\n", numkern );
    }

    // Loop over positions, calculate response values and do multiplications
    for ( ii = 0, q_r = startr; ii < numkern; q_r--, ii++ )
    {
      FOLD //  Read the input value  .
      {
        inp             = data[lodata+ii];
      }

      FOLD //  Calculate response value  .
      {
        if ( absz < FTLIM ) // Just do a Fourier Interpolation
        {
          xx              = (T)PI*q_r ;
          sincos_t(xx, &s, &c);

          if ( q_r == 0.0 )
            sinc = 1.0;
          else
            sinc = s / xx;

          tR              = c * sinc;
          tI              = s * sinc;

          //          const int ix        = blockIdx.x * blockDim.x + threadIdx.x;
          //          const int iy = blockIdx.y * blockDim.y + threadIdx.y;
          //
          //          double  dqr   = r - lodata + ii ;
          //          double  dxx   = PI*dqr ;
          //          double  sd, cd, sincd, tRd, tId;
          //          sincos_t(xx, &sd, &cd);
          //
          //          if ( dqr == 0.0 )
          //            sincd = 1.0;
          //          else
          //            sincd = sd / dxx;
          //
          //          tRd             = cd * sincd;
          //          tId             = sd * sincd;
          //
          //          if ( ix == 0 )
          //          {
          //            printf("-- %15.8f  %15.8f\n   %15.8f  %15.8f\n", tR, tI, tRd, tId);
          //          }
          //
          //          tR = tRd;
          //          tI = tId;

          //printf("%04i response: %15.10f %15.10f  r: %15.10f  c: %15.10f s: %15.10f sinc: %15.10f\n", ii, tR, tI, q_r, c, s, sinc );
        }
        else
        {
          Yr              = q_r * zd;
          Zr              = Yr + zT * zd;
          xx              = pibyz * q_r * q_r;

          sincos_t(xx, &s, &c);
          fresnl<T>(Yr, &fressy, &frescy);
          fresnl<T>(Zr, &fressz, &frescz);

          T Ster          = fressz - fressy;
          T Cter          = frescy - frescz;
          tR              = cons * (c*Ster + signz*s*Cter);
          tI              = cons * (s*Ster - signz*c*Cter);

          //          const int ix    = blockIdx.x * blockDim.x + threadIdx.x;
          //          const int iy    = blockIdx.y * blockDim.y + threadIdx.y;
          //          if ( ix == 0 && iy == 0 && ii == 0 )
          //          {
          //            printf("Yr: %20.10f  Zr: %20.10f  xx: %20.10f  tR: %20.10f  tI: %20.10f  xx: %20.10f  xx: %20.10f  \n", Yr, Zr, xx, tR, tI);
          //          }
        }
      }

      FOLD //  Do the multiplication  .
      {
        ans.r           += tR * inp.r - tI*inp.i;
        ans.i           += tR * inp.i + tI*inp.r;
      }

      //printf("%03i %05i Data %12.2f %12.2f  Response: %13.10f %13.10f   %12.2f \n", ii, loR+lodata+ii, inp.r, inp.i, tR, tI, POWERCU(ans.r, ans.i) );
    }
  }
  else
  {
    //printf("r < 0: %.6f\n", r );
  }

  return ans;
}

template<typename T>
__global__ void ffdotPln_ker(float* powers, fcomplexcu* fft, int noHarms, int halfwidth, double firstR, double firstZ, double rSZ, double zSZ, int noR, int noZ, int iStride, int oStride, int32 loR, float32 norm, int32 hw)
{
  const int ix = blockIdx.x * blockDim.x + threadIdx.x;
  const int iy = blockIdx.y * blockDim.y + threadIdx.y;

  if ( ix < noR && iy < noZ)
  {
    //    if ( ix ==0 && iy == 0 )
    //    {
    //      printf("\n");
    //    }
    double r            = firstR + ix/(double)(noR-1) * rSZ ;
    double z            = firstZ - iy/(double)(noZ-1) * zSZ ;

    double total_power  = 0;
    fcomplexcu ans;

    //double absz         = fabs(z);

    for( int i = 1; i <= noHarms; i++ )
    {
      double absz         = fabs(z*i);
      //      if(ix ==0 && iy == 0 )
      //      {
      //        printf("%02i absz: %.5f\n",i, absz);
      //      }
      if( absz < DLIM && absz > FTLIM )
      {
        //ans  = rz_interp_cu<double>(&fft[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, halfwidth);
        ans  = rz_interp_cu<double>(&fft[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, hw.val[i-1] );
      }
      else
      {
        //ans  = rz_interp_cu<T>(&fft[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, halfwidth);
        ans  = rz_interp_cu<T>(&fft[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, hw.val[i-1] );
      }

      //total_power     += POWERCU(ans.r, ans.i)/norm.val[i-1];
      total_power     += POWERCU(ans.r, ans.i);
    }

    //powers[iy*noR + ix] = total_power;
    powers[iy*oStride + ix] = total_power;
  }
}

__global__ void rz_interp_ker(double r, double z, fcomplexcu* fft, int loR, int noBins, int halfwidth, double normFactor)
{
  float total_power   = 0;

  fcomplexcu ans      = rz_interp_cu<float>(fft, loR, noBins, r, z, halfwidth);
  //fcomplexcu ans      = rz_interp_cu<double>(fft, loR, noBins, r, z, halfwidth);
  total_power         += POWERCU(ans.r, ans.i)/normFactor;

  //printf("rz_interp_ker r: %.4f  z: %.4f  Power: %.4f  ( %.4f, %.4f )\n", r, z, POWERCU(ans.r, ans.i), ans.r, ans.i);
}

/*

 hiprandState *d_state;
  hipMalloc(&d_state, sizeof(hiprandState));


 */
template<typename T>
__global__ void ffdotSwarm_ker(unsigned long long seed, candOpt* out, fcomplexcu* fft, int loR, int noBins, int noHarms, int noReps, int halfwidth, double firstR, double firstZ, double rSZ, double zSZ, int noR, int noZ, float16 norm)
{
  const int ix        = blockIdx.x * blockDim.x + threadIdx.x;
  const int iy        = blockIdx.y * blockDim.y + threadIdx.y;
  const int idx       = iy * noR + ix;
  const int wrpNo     = floor(idx/32.0);
  //const int sz        = 32 ; // noR * noZ - wrpNo * 32 ;
  //const int lane      = idx % sz;
  //const int oLane     = (lane+1) % sz;

  hiprandState state;

  float a = 0.8;
  float b = 0.4;
  float c = 0.7;

  float   velocityMax       = rSZ / 20.0;

  float   power             = 0;
  float   gBestP            = 0;
  float   lBestP            = 0;

  double  nBestR, nBestZ;
  float   nBestP;

  double2 lGlo;
  double2 lLoc;
  double2 pos;
  double2 vel;
  double2 dGlo;
  double2 dLoc;

  lGlo.x = 0;
  lGlo.y = 0;

  if ( ix < noR && iy < noZ)
  {
    hiprand_init(seed, idx, 0, &state);

    FOLD // Initial values  .
    {
      pos.x       = firstR + ix/(double)(noR-1) * rSZ ;
      pos.y       = firstZ - iy/(double)(noZ-1) * zSZ ;
      power       = 0;

      for( int i = 1; i <= noHarms ; i++ )
      {
        //if ( idx == 1001 )
        {
          fcomplexcu ans   = rz_interp_cu<T>(fft, loR, noBins, pos.x*i, pos.y*i, halfwidth);
          power            += POWERCU(ans.r, ans.i)/norm.val[i-1];

          //          if ( idx == 1001 )  // TMP
          //          {
          //            printf(" Pow: %10.2f  %10.2f %10.2f Norm: %10.3f   Accum: %10.3f\n", POWERCU(ans.r, ans.i), ans.r, ans.i, norm.val[i-1], power );
          //          }
        }
      }

      // Set local best
      lLoc     = pos;
      lBestP   = power;

      // Global local best
      lGlo      = lLoc;
      gBestP    = lBestP;

      FOLD // Velocity  .
      {
        vel.x = hiprand_uniform(&state);
        vel.y = hiprand_uniform(&state);
        float lenn = sqrt(vel.x*vel.x+vel.y*vel.y) ; // len(vel);
        vel *= velocityMax/lenn;
      }
    }

    //    if ( idx == 1001 ) // TMP
    //    {
    //      double gDst = len(lGlo - pos);
    //      printf("%03i Current r: %10.5f z: %10.5f  power: %20.6f  -  Local r: %10.5f z: %10.5f  power: %20.6f  -  Best r: %10.5f z: %10.5f  power: %20.6f    Dist %9.6f\n", 0, pos.x, pos.y, power, lLoc.x, lLoc.y, lBestP, lGlo.x, lGlo.y, gBestP, gDst);
    //    }

    if(1)
    {
      for (int rep = 0; rep < 10; rep++)
      {
        //d2 = gBestR - pos.x;
        //d3 = gBestZ - pos.y;

        dLoc = lLoc - pos;
        dGlo = lGlo - pos;

        //double lDst = len(dLoc);
        //double gDst = len(dGlo);

        //r1 = hiprand_uniform(&state);
        //r2 = hiprand_uniform(&state);

        vel = a*vel+b*dLoc+c*dGlo;

        float lenn = len(vel);
        if ( lenn > velocityMax )
        {
          vel *= velocityMax/lenn;
        }

        pos += vel;

        power = 0;
        for( int i = 1; i <= noHarms; i++ )
        {
          fcomplexcu ans   = rz_interp_cu<T>(fft, loR, noBins, pos.x*i, pos.y*i, halfwidth);
          power           += POWERCU(ans.r, ans.i)/norm.val[i-1];
        }

        //        if ( isnan(power) ) // TMP  .
        //        {
        //          printf("idx: %03i   r: %.1f \n", idx, pos.x );
        //        }

        if ( power > lBestP ) // Update Local bets  .
        {
          lLoc      = pos;
          lBestP    = power;
        }

        if ( power > gBestP ) // Update Global bets  .
        {
          lGlo      = pos;
          gBestP    = power;
        }


        FOLD // Check Global best with neighbour  .
        {
          //nBestR = lGlo.x;
          //nBestZ = lGlo.y;
          //nBestP = gBestP;
          //nBestR = __shfl(lGlo.x,oLane);
          //nBestZ = __shfl(lGlo.y,oLane);

          int2 tmpForExchIn, tmpForExchOut;
          //float tt = gBestP;

          for ( int ln = 0; ln < 32; ln++) // Shuffle with all elements in the warp
          {
            // get R
            tmpForExchIn = *(int2 *)(&lGlo.x);
            tmpForExchOut.x = __shfl(tmpForExchIn.x, ln);
            tmpForExchOut.y = __shfl(tmpForExchIn.y, ln);
            nBestR = *(double *)(&tmpForExchOut);

            // get Z
            tmpForExchIn = *(int2 *)(&lGlo.y);
            tmpForExchOut.x = __shfl(tmpForExchIn.x, ln);
            tmpForExchOut.y = __shfl(tmpForExchIn.y, ln);
            nBestZ = *(double *)(&tmpForExchOut);

            // power
            nBestP = __shfl(gBestP, ln);

            if ( nBestP > gBestP )
            {
              //            if ( idx == 0 ) // TMP
              //            {
              //              printf("Got a new best!\n");
              //            }
              lGlo.x   = nBestR;
              lGlo.y   = nBestZ;
              gBestP   = nBestP;
            }
            else if ( idx == 0 ) // TMP
            {
              //printf("Shuffle got Current r: %.5f z: %.5f   power %15.6f vs %15.6f!\n", nBestR, nBestZ, nBestP, gBestP );
            }
          }
        }

        //        if ( idx == 1001 ) // TMP
        //        {
        //          double gDst = len(lGlo - pos);
        //          printf("%03i Current r: %10.5f z: %10.5f  power: %20.6f  -  Local r: %10.5f z: %10.5f  power: %20.6f  -  Best r: %10.5f z: %10.5f  power: %20.6f    Dist %9.6f\n", rep+1, pos.x, pos.y, power, lLoc.x, lLoc.y, lBestP, lGlo.x, lGlo.y, gBestP, gDst);
        //        }
      }
    }

    FOLD // Output  .
    {
      candOpt outP;
      outP.r     = lGlo.x;
      outP.z     = lGlo.y;
      outP.power = gBestP;
      out[idx]   = outP;
    }
  }
}

int ffdotPln(float* powers, fcomplex* fft, int loR, int noBins, int noHarms, double centR, double centZ, double rSZ, double zSZ, int noR, int noZ, int halfwidth, float* fac)
{
  double log2 = log(2.0);

  double maxZ = (centZ + zSZ/2.0);
  double minZ = (centZ - zSZ/2.0);
  double minR = (centR - rSZ/2.0);
  double maxR = (centR + rSZ/2.0);

  int halfwidth2    = z_resp_halfwidth(MAX(fabs(maxZ*noHarms), fabs(minZ*noHarms))+4, HIGHACC);
  //halfwidth         = MAX(halfwidth,halfwidth2);

  double rSpread    = ceil(maxR*noHarms  + halfwidth) - floor(minR*noHarms - halfwidth);

  size_t iStride, pStride;
  float *cuPowers;
  fcomplexcu *cuInp;
  fcomplexcu *cpuInp;
  double factor;

  CUDA_SAFE_CALL(hipMallocPitch(&cuPowers,  &pStride, noR     * sizeof(float),             noZ),   "Failed to allocate device memory for kernel stack.");
  CUDA_SAFE_CALL(hipMallocPitch(&cuInp,     &iStride, rSpread * sizeof(hipfftComplex),  noHarms),   "Failed to allocate device memory for kernel stack.");

  int noInp = iStride/sizeof(hipfftComplex);
  int noPow = pStride/sizeof(float);

  int32   rOff;
  int32   hw;
  float32 norm;

  cpuInp = (fcomplexcu*) malloc(iStride*noHarms);

  for( int h = 0; h < 32; h++)
  {
    rOff.val[h] = 0;
    hw.val[h]   = 0;
  }

  for( int h = 0; h < noHarms; h++)
  {
    rOff.val[h]   = floor( minR*(h+1) - halfwidth );
    hw.val[h]     = z_resp_halfwidth(MAX(fabs(maxZ*(h+1)), fabs(minZ*(h+1)))+2, HIGHACC);
    //printf("%i  %f   %i\n", (int)floor(minR*(h+1)), minR*(h+1), halfwidth );

    int datStart  = floor( minR*(h+1) - halfwidth );
    int datEnd    = ceil ( maxR*(h+1) + halfwidth );
    int novals    = datEnd - datStart;
    int noPowers, off;
    float medianv;

    FOLD // Calculate normalisation factor  .
    {
      if ( fac == NULL) // Calculate the normalisation factor  .
      {
        float*  normPow = (float*) malloc(noInp*sizeof(float));

        if ( datStart < loR )
        {
          novals    -=  (loR - datStart);
          datStart  =   loR;
        }
        if ( datEnd >= noBins )
        {
          novals    -=  (datEnd - noBins - 1);
          datEnd    =   noBins-1;
        }

        noPowers = 0;
        for ( int i = 0; i < noInp; i++)
        {
          off = rOff.val[h] - loR + i;
          if (off >= 0 && off < noBins )
          {
            normPow[noPowers++] = POWERCU(fft[off].r, fft[off].i ) ;
          }
        }

        medianv       = median(normPow, noPowers);
        factor        = sqrt(medianv/log2);
        //printf("  %02i  %8.3f \n", h+1, factor );

        free(normPow);
      }
      else              // Use precalcualted normalisation factor  .
      {
        factor = sqrt(fac[h]);
      }
      norm.val[h] = fac[h];
      //factor = 1.0;
    }

    for ( int i = 0; i < noInp; i++) // Normalise input  .
    {
      off = rOff.val[h] - loR + i;
      if (off >= 0 && off < noBins && i < novals)
      {
        cpuInp[h*noInp + i].r = fft[off].r / factor ;
        cpuInp[h*noInp + i].i = fft[off].i / factor ;
      }
      else
      {
        cpuInp[h*noInp + i].r = 0;
        cpuInp[h*noInp + i].i = 0;
      }
    }
  }

  CUDA_SAFE_CALL(hipMemcpy(cuInp, cpuInp, iStride*noHarms, hipMemcpyHostToDevice), "Copying optimisation input to the device");

  FOLD // Call kernel  .
  {
    dim3 dimBlock, dimGrid;

    // Blocks of 1024 threads ( the maximum number of threads per block )
    dimBlock.x = 16;
    dimBlock.y = 16;
    dimBlock.z = 1;

    // One block per harmonic, thus we can sort input powers in Shared memory
    dimGrid.x = ceil(noR/(float)dimBlock.x);
    dimGrid.y = ceil(noZ/(float)dimBlock.y);

    // Call the kernel to normalise and spread the input data
    ffdotPln_ker<float><<<dimGrid, dimBlock, 0, 0>>>(cuPowers, cuInp, noHarms, halfwidth, minR, maxZ, rSZ, zSZ, noR, noZ, noInp, noPow, rOff, norm, hw);

    CUDA_SAFE_CALL(hipGetLastError(), "Calling the ffdotPln_ker kernel.");
  }

  CUDA_SAFE_CALL(hipMemcpy(powers, cuPowers, pStride*noZ, hipMemcpyDeviceToHost), "Copying optimisation results back from the device.");

  FOLD // Write CVS
  {
    char tName[1024];
    sprintf(tName,"/home/chris/accel/lrg_2_GPU.csv");
    FILE *f2 = fopen(tName, "w");

    fprintf(f2,"%i",noHarms);

    for (int indx = 0; indx < noR ; indx++ )
    {
      double r = minR + indx/(double)(noR-1) * (rSZ) ;
      fprintf(f2,"\t%.6f",r);
    }
    fprintf(f2,"\n");

    for (int indy = 0; indy < noZ; indy++ )
    {
      double z = maxZ - indy/(double)(noZ-1) * (zSZ) ;

      fprintf(f2,"%.6f",z);

      for (int indx = 0; indx < noR ; indx++ )
      {
        float yy2 = powers[indy*noPow+indx];
        fprintf(f2,"\t%.6f",yy2);
      }
      fprintf(f2,"\n");
    }
    fclose(f2);

    FOLD // Make image
    {
      printf("Making lrg_GPU.png    \t... ");
      fflush(stdout);
      char cmd[1024];
      sprintf(cmd,"python ~/bin/bin/plt_ffd.py %s", tName);
      system(cmd);
      printf("Done\n");
    }
  }

  CUDA_SAFE_CALL(hipFree(cuPowers),    "Failed free device memory for optimisation powers.");
  CUDA_SAFE_CALL(hipFree(cuInp),       "Failed free device memory for optimisation inputs.");

  return noPow;
}

template<typename T>
void ffdotPln( cuOptCand* pln, fftInfo* fft )
{
  double maxZ       = (pln->centZ + pln->zSize/2.0);
  double minZ       = (pln->centZ - pln->zSize/2.0);
  double maxR       = (pln->centR + pln->rSize/2.0);
  double minR       = (pln->centR - pln->rSize/2.0);

  CUDA_SAFE_CALL(hipGetLastError(), "Entering ffdotPln.");

  pln->halfWidth    = z_resp_halfwidth(MAX(fabs(maxZ*pln->noHarms), fabs(minZ*pln->noHarms)) + 4, HIGHACC);
  double rSpread    = ceil((maxR+OPT_INP_BUF)*pln->noHarms  + pln->halfWidth) - floor((minR-OPT_INP_BUF)*pln->noHarms - pln->halfWidth);
  int    inpStride  = getStrie(rSpread, sizeof(hipfftComplex), pln->alignment);
  pln->outStride    = getStrie(pln->noR,  sizeof(float), pln->alignment);

  int datStart,  datEnd, noDat;
  int32   rOff;
  int32   hw;
  float32 norm;
  int     off;
  int     newInp = 0;

  // Determine if new input is needed
  for( int h = 0; (h < pln->noHarms) && !newInp; h++)
  {
    datStart        = floor( minR*(h+1) - pln->halfWidth );
    datEnd          = ceil(  maxR*(h+1) + pln->halfWidth );
    noDat           = datEnd - datStart;

    if ( datStart < pln->loR[h] )
    {
      newInp = 1;
    }
    else if ( pln->loR[h] + pln->inpStride < datEnd )
    {
      newInp = 1;
    }
  }

  // Initialise values to 0
  for( int h = 0; h < 32; h++)
  {
    rOff.val[h] = 0;
    hw.val[h]   = 0;
  }

  if ( newInp )
  {
    //printf("New input  old stride  %4i   new Stride %4i \n", pln->inpStride, inpStride );

    pln->inpStride = inpStride;

    if ( pln->inpStride*pln->noHarms*sizeof(hipfftComplex) > pln->inpSz )
    {
      fprintf(stderr, "ERROR: In function %s, cuOptCand not created with large enough input buffer.", __FUNCTION__);
      exit(EXIT_FAILURE);
    }

    FOLD // Calculate normalisation factor  .
    {
      nvtxRangePush("Calc Norm factor");

      for ( int i = 1; i <= pln->noHarms; i++ )
      {
        pln->norm[i-1]  = get_scaleFactorZ(fft->fft, fft->nor, (fft->idx+pln->centR)*i-fft->rlo, pln->centZ*i, 0.0);
      }

      nvtxRangePop();
    }
  }

  for( int h = 0; h < pln->noHarms; h++)
  {
    datStart          = floor( minR*(h+1) - pln->halfWidth );
    datEnd            = ceil(  maxR*(h+1) + pln->halfWidth );
    noDat             = datEnd - datStart;
    hw.val[h]         = z_resp_halfwidth(MAX(fabs(maxZ*(h+1)), fabs(minZ*(h+1))) + 4, HIGHACC);
    rOff.val[h]       = pln->loR[h];

    if ( newInp )
    {
      int startV = MIN( ((datStart + datEnd - pln->inpStride ) / 2.0), datStart );

      rOff.val[h]     = startV;
      pln->loR[h]     = startV;
      double factor   = sqrt(pln->norm[h]);
      norm.val[h]     = factor;

      for ( int i = 0; i < pln->inpStride; i++ ) // Normalise input  .
      {
        off = rOff.val[h] - fft->rlo + i;

        if ( off >= 0 && off < fft->nor /* && i < noDat */ )
        {
          pln->h_inp[h*pln->inpStride + i].r = fft->fft[off].r / factor ;
          pln->h_inp[h*pln->inpStride + i].i = fft->fft[off].i / factor ;
        }
        else
        {
          pln->h_inp[h*pln->inpStride + i].r = 0;
          pln->h_inp[h*pln->inpStride + i].i = 0;
        }
      }
    }
  }

  if ( newInp )
  {
    //CUDA_SAFE_CALL(hipEventRecord(pln->inpInit, pln->stream),"Recording event: inpInit");
    CUDA_SAFE_CALL(hipMemcpyAsync(pln->d_inp, pln->h_inp, pln->inpStride*pln->noHarms*sizeof(fcomplexcu), hipMemcpyHostToDevice, pln->stream), "Copying optimisation input to the device");
    //CUDA_SAFE_CALL(hipEventRecord(pln->inpCmp, pln->stream),"Recording event: inpCmp");
  }

  FOLD // Call kernel  .
  {
    dim3 dimBlock, dimGrid;

    // Event
    //CUDA_SAFE_CALL(hipEventRecord(pln->compInit, pln->stream),"Recording event: inpInit");

    // Blocks of 1024 threads ( the maximum number of threads per block )
    dimBlock.x = 16;
    dimBlock.y = 16;
    dimBlock.z = 1;

    // One block per harmonic, thus we can sort input powers in Shared memory
    dimGrid.x = ceil(pln->noR/(float)dimBlock.x);
    dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

    // Call the kernel to normalise and spread the input data
    ffdotPln_ker<T><<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, pln->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->inpStride, pln->outStride, rOff, norm, hw);

    CUDA_SAFE_CALL(hipGetLastError(), "Calling the ffdot_ker kernel.");

    // Event
    //CUDA_SAFE_CALL(hipEventRecord(pln->compCmp, pln->stream),"Recording event: inpInit");
  }

  //CUDA_SAFE_CALL(hipEventRecord(pln->outInit, pln->stream),"Recording event: outInit");
  CUDA_SAFE_CALL(hipMemcpyAsync(pln->h_out, pln->d_out, pln->outStride*pln->noZ*sizeof(float), hipMemcpyDeviceToHost, pln->stream), "Copying optimisation results back from the device.");
  CUDA_SAFE_CALL(hipEventRecord(pln->outCmp, pln->stream),"Recording event: outCmp");
}

template<typename T>
void ffdotSwrm( cuOptCand* pln, fftInfo* fft )
{
  double maxZ       = (pln->centZ + pln->zSize/2.0);
  double minZ       = (pln->centZ - pln->zSize/2.0);
  double maxR       = (pln->centR + pln->rSize/2.0);
  double minR       = (pln->centR - pln->rSize/2.0);

  pln->halfWidth    = z_resp_halfwidth(MAX(fabs(maxZ*pln->noHarms), fabs(minZ*pln->noHarms)) + 4, HIGHACC);
  double rSpread    = ceil(maxR*pln->noHarms  + pln->halfWidth) - floor(minR*pln->noHarms - pln->halfWidth);
  //pln->inpStride    = getStrie(rSpread, sizeof(hipfftComplex), pln->alignment);
  //pln->outStride    = getStrie(pln->noR,  sizeof(float), pln->alignment);
  //  if ( pln->inpStride*pln->noHarms*sizeof(hipfftComplex) > pln->inpSz )
  //  {
  //    fprintf(stderr, "ERROR: In function %s, cuOptCand not created with large enough input buffer.", __FUNCTION__);
  //    exit(EXIT_FAILURE);
  //  }

  //int16   rOff;
  //int     off;
  //int datStart,  datEnd, noDat;

//  for( int h = 0; h < 16; h++)
//  {
//    rOff.val[h] = 0;
//  }

  float16 norm;

  for( int h = 0; h < pln->noHarms; h++)
  {
    norm.val[h]     = pln->norm[h];
  }

  FOLD // Call kernel  .
  {
    dim3 dimBlock, dimGrid;

    // Event
    CUDA_SAFE_CALL(hipEventRecord(pln->compInit, pln->stream),"Recording event: inpInit");

    // Blocks of 1024 threads ( the maximum number of threads per block )
    dimBlock.x = 16;
    dimBlock.y = 16;
    dimBlock.z = 1;

    // One block per harmonic, thus we can sort input powers in Shared memory
    dimGrid.x = ceil(pln->noR/(float)dimBlock.x);
    dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

    // Call the kernel to normalise and spread the input data
    ffdotSwarm_ker<T><<<dimGrid, dimBlock, 0, pln->stream >>>(time(NULL), (candOpt*)pln->d_out, pln->d_inp, fft->idx, fft->nor, pln->noHarms, 10, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, norm);

    CUDA_SAFE_CALL(hipGetLastError(), "Calling the ffdot_ker kernel.");

    // Event
    CUDA_SAFE_CALL(hipEventRecord(pln->compCmp, pln->stream),"Recording event: inpInit");
  }

  //hipDeviceSynchronize();          // TMP

  if ( pln->noZ*pln->noR*sizeof(candOpt) > pln->outSz )
  {
    fprintf(stderr,"ERROR, not enough space for output!\n");
  }
  else
  {
    CUDA_SAFE_CALL(hipEventRecord(pln->outInit, pln->stream),"Recording event: outInit");
    CUDA_SAFE_CALL(hipMemcpyAsync(pln->h_out, pln->d_out, pln->noZ*pln->noR*sizeof(candOpt), hipMemcpyDeviceToHost, pln->stream ), "Copying optimisation results back from the device.");
    CUDA_SAFE_CALL(hipEventRecord(pln->outCmp, pln->stream),"Recording event: outCmp");
  }
}

void rz_interp_cu(fcomplex* fft, int loR, int noBins, double centR, double centZ, int halfwidth)
{
  FOLD // TMP: CPU equivalent  .
  {
    fcomplex ans;
    rz_interp((fcomplex*)fft, noBins, centR, centZ, halfwidth, &ans);
  }

  fcomplexcu *cuInp;
  int     rOff, lodata;
  double factor;
  double log2 = log(2.0);

  int noInp       = 2*halfwidth;
  lodata          = floor( centR ) - halfwidth ;
  rOff            = lodata - loR ;

  FOLD // Clamp size  .
  {
    if ( lodata < 0 )
    {
      noInp         += lodata;
      rOff          -= lodata;
    }

    if ( rOff + noInp >= noBins )
    {
      fprintf(stderr, "WARNING: attempting to do a f-∂f interpolation beyond the end of the FFT.\n");
      noInp = noBins - rOff;
    }
  }

  FOLD // GPU Memory operations  .
  {
    CUDA_SAFE_CALL(hipMalloc((void** )&cuInp, noInp * sizeof(hipfftComplex) ),   "Failed to allocate device memory for kernel stack.");
    CUDA_SAFE_CALL(hipMemcpy(cuInp, &fft[rOff], noInp * sizeof(hipfftComplex), hipMemcpyHostToDevice), "Copying convolution kernels between devices.");
  }

  FOLD // Calculate normalisation factor  .
  {
    float*  normPow = (float*) malloc(noInp*sizeof(float));

    for ( int i = 0; i < noInp; i++ )
    {
      normPow[i] = POWERCU(fft[rOff+i].r, fft[rOff+i].i ) ;
    }

    float medianv   = median(normPow, noInp);
    factor          = sqrt(medianv/log2);

    free(normPow);
  }

  FOLD // Call kernel  .
  {
    dim3 dimBlock, dimGrid;

    // Blocks of 1024 threads ( the maximum number of threads per block )
    dimBlock.x = 1;
    dimBlock.y = 1;
    dimBlock.z = 1;

    // One block per harmonic, thus we can sort input powers in Shared memory
    dimGrid.x = 1;
    dimGrid.y = 1;

    // Call the kernel to normalise and spread the input data
    rz_interp_ker<<<dimGrid, dimBlock, 0, 0>>>(centR, centZ, cuInp, rOff, noInp, halfwidth, factor);
  }
}

template<typename T>
void generatePln(cand* cand, fftInfo* fft, cuOptCand* pln, int noP, double scale , int plt = -1, int nn = 0 )
{
  nvtxRangePush("generatePln");

  pln->centR          = cand->r;
  pln->centZ          = cand->z;
  pln->noZ            = noP*2 + 1;
  pln->noR            = noP*2 + 1;
  pln->rSize          = scale;
  pln->zSize          = scale*4.0;
  pln->noHarms        = cand->numharm;

  ffdotPln<T>(pln, fft);

  FOLD // A blocking synchronisation to ensure results are ready to be proceeded by the host
  {
    nvtxRangePush("EventSynch");
    CUDA_SAFE_CALL(hipEventSynchronize(pln->outCmp), "Synchronising using hipEventSynchronize.");
    nvtxRangePop();
  }

  if ( pltOpt > 0 ) // Write CVS & plot output  .
  {
    nvtxRangePush("Write CVS");

    char tName[1024];
    sprintf(tName,"/home/chris/accel/Cand_%05i_Rep_%02i_h%02i.csv", nn, plt, cand->numharm );
    FILE *f2 = fopen(tName, "w");

    fprintf(f2,"%i",pln->noHarms);

    for (int indx = 0; indx < pln->noR ; indx++ )
    {
      double r = pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
      fprintf(f2,"\t%.6f",r);
    }
    fprintf(f2,"\n");

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      double z = pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;

      fprintf(f2,"%.6f",z);

      for (int indx = 0; indx < pln->noR ; indx++ )
      {
        float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
        fprintf(f2,"\t%.15f",yy2);
      }
      fprintf(f2,"\n");
    }
    fclose(f2);

    FOLD // Make image  .
    {
      nvtxRangePush("Image");
      char cmd[1024];
      sprintf(cmd,"python ~/bin/bin/plt_ffd.py %s", tName);
      system(cmd);
      nvtxRangePop();
    }

    nvtxRangePop();
  }

  FOLD // Get new max  .
  {
    nvtxRangePush("Get Max");

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      for (int indx = 0; indx < pln->noR ; indx++ )
      {
        float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
        if ( yy2 > cand->power )
        {
          cand->power   = yy2;
          cand->r       = pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
          cand->z       = pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;
        }
      }
    }

    nvtxRangePop();
  }

  nvtxRangePop();
}

void optemiseTree(candTree* tree, cuOptCand* oPlnPln)
{
  container* cont = tree->getLargest();

  while (cont)
  {
    cont = cont->smaller;
  }
}

int addPlnToTree(candTree* tree, cuOptCand* pln)
{
  nvtxRangePush("addPlnToTree");

  FOLD // Get new max  .
  {
    int ggr = 0;

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      for (int indx = 0; indx < pln->noR ; indx++ )
      {
        float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
        //if ( yy2 > cand->power )
        {
          cand* canidate = new cand;

          canidate->numharm = pln->noHarms;
          canidate->power   = yy2;
          canidate->r       = pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
          canidate->z       = pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;
          canidate->sig     = yy2;

          ggr++;

          //          printf("\r ggr: %i", ggr);
          //
          //          if ( ggr == 786 )
          //            int tmp = 0;

          tree->insert(canidate, 0.2 );
        }
      }
    }
    //printf("\n");
  }

  nvtxRangePop();

  return 0;
}

candTree* opt_cont(candTree* oTree, cuOptCand* pln, container* cont, fftInfo* fft, int nn)
{
  nvtxRangePush("opt_cont");

  int lrep      = 0;
  int noP       = 30;
  float snoop   = 0.3;
  float sz;
  float v1, v2;

  const int mxRep = 10;

  cand* canidate = (cand*)cont->data;

  candTree* thisOpt = new candTree;

  if ( canidate->numharm == 1  )
    sz = optSz01;
  if ( canidate->numharm == 2  )
    sz = optSz02;
  if ( canidate->numharm == 4  )
    sz = optSz04;
  if ( canidate->numharm == 8  )
    sz = optSz08;
  if ( canidate->numharm == 16 )
    sz = optSz16;

  //int numindep        = (obs->rhi - obs->rlo ) * (obs->zhi +1 ) * (ACCEL_DZ / 6.95) / pln->noHarms ;

  //printf("\n%03i  r: %15.6f   z: %12.6f \n", nn, cand->r, cand->z);

  pln->halfWidth = 0;

  int plt = 0;

  if ( optpln01 > 0 )
  {
    noP               = optpln01 ;
    lrep              = 0;
    canidate->power   = 0;     // Set initial power to zero
    do
    {
      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );

      container* optC =  oTree->getLargest(canidate, 1);

      if ( optC )
      {
        // This has feature has already been optimised!
        cont->flag |= REMOVE_CONTAINER;
        nvtxRangePop();
        return thisOpt;
      }

      //addPlnToTree(thisOpt, pln);

      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));

      if ( ++lrep > mxRep )
      {
        break;
      }
    }
    while ( v1 > snoop || v2 > snoop );
    sz /= downScale;
  }

  if ( optpln02 > 0 )
  {
    noP               = optpln02 ;
    lrep              = 0;
    canidate->power   = 0;     // Set initial power to zero
    do
    {
      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );

      container* optC =  oTree->getLargest(canidate, 1);

      if ( optC )
      {
        // This has feature has already been optimised!
        cont->flag |= REMOVE_CONTAINER;
        nvtxRangePop();
        return thisOpt;
      }

      //addPlnToTree(thisOpt, pln);

      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));

      if ( ++lrep > mxRep )
      {
        break;
      }
    }
    while ( v1 > snoop || v2 > snoop );
    sz /= downScale;
  }

  if ( optpln03 > 0 )
  {
    noP               = optpln03 ;
    lrep              = 0;
    canidate->power   = 0;     // Set initial power to zero
    do
    {
      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );

      container* optC =  oTree->getLargest(canidate, 1);

      if ( optC )
      {
        // This has feature has already been optimised!
        cont->flag |= REMOVE_CONTAINER;
        nvtxRangePop();
        return thisOpt;
      }

      //addPlnToTree(thisOpt, pln);

      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));

      if ( ++lrep > mxRep )
      {
        break;
      }
    }
    while ( v1 > snoop || v2 > snoop );
    sz /= downScale*2;
  }

  if ( optpln04 > 0 )
  {
    noP               = optpln04 ;
    lrep              = 0;
    canidate->power   = 0;     // Set initial power to zero
    do
    {
      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );

      container* optC =  oTree->getLargest(canidate, 1);

      if ( optC )
      {
        // This has feature has already been optimised!
        cont->flag |= REMOVE_CONTAINER;
        nvtxRangePop();
        return thisOpt;
      }

      //addPlnToTree(thisOpt, pln);

      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));

      if ( ++lrep > mxRep )
      {
        break;
      }
    }
    while ( v1 > snoop || v2 > snoop );
    sz /= downScale*2;
  }

  if ( optpln05 > 0 )
  {
    noP               = optpln05 ;
    lrep              = 0;
    canidate->power   = 0;     // Set initial power to zero
    do
    {
      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );

      container* optC =  oTree->getLargest(canidate, 1);

      if ( optC )
      {
        // This has feature has already been optimised!
        cont->flag |= REMOVE_CONTAINER;
        nvtxRangePop();
        return thisOpt;
      }

      //addPlnToTree(thisOpt, pln);

      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));

      if ( ++lrep > mxRep )
      {
        break;
      }
    }
    while ( v1 > snoop || v2 > snoop );
    sz /= downScale*2;
  }

  if ( optpln06 > 0 )
  {
    noP               = optpln06 ;
    lrep              = 0;
    canidate->power   = 0;     // Set initial power to zero
    do
    {
      generatePln<double>(canidate, fft, pln, noP, sz, plt++, nn );

      container* optC =  oTree->getLargest(canidate, 1);

      if ( optC )
      {
        // This has feature has already been optimised!
        cont->flag |= REMOVE_CONTAINER;
        nvtxRangePop();
        return thisOpt;
      }

      //addPlnToTree(thisOpt, pln);

      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));

      if ( ++lrep > mxRep )
      {
        break;
      }
    }
    while ( v1 > snoop || v2 > snoop );
    sz /= downScale*2;
  }

  cont->flag |= OPTIMISED_CONTAINER;
  nvtxRangePop();
  return thisOpt;
}

template<typename T>
void opt_candByPln(accelcand* cand, fftInfo* fft, cuOptCand* pln, int noP, double scale, int plt = -1, int nn = 0 )
{
  FOLD // Large points  .
  {
    pln->centR          = cand->r;
    pln->centZ          = cand->z;
    pln->noZ            = noP*2 + 1;
    pln->noR            = noP*2 + 1;
    pln->rSize          = scale;
    pln->zSize          = scale*4.0;

    //          gettimeofday(&start, NULL);       // TMP
#ifdef DEBUG
    fprintf(stderr, "SORRY: ffdotPln kills debug! Skipping it!\n");
#else
    ffdotPln<T>(pln, fft);
#endif
    //          gettimeofday(&end, NULL);         // TMP
    //          timev1 = ((end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec)); // TMP
    //          printf("%.5f\t",timev1);          // TMP
  }

  FOLD // A blocking synchronisation to ensure results are ready to be proceeded by the host
  {
    nvtxRangePush("EventSynch");
    CUDA_SAFE_CALL(hipEventSynchronize(pln->outCmp), "Synchronising using hipEventSynchronize.");
    nvtxRangePop();
  }

  if ( pltOpt > 0 ) // Write CVS & plot output  .
  {
    nvtxRangePush("Write CVS");

    char tName[1024];
    sprintf(tName,"/home/chris/accel/Cand_%05i_Rep_%02i_h%02i.csv", nn, plt, cand->numharm );
    FILE *f2 = fopen(tName, "w");

    fprintf(f2,"%i",pln->noHarms);

    for (int indx = 0; indx < pln->noR ; indx++ )
    {
      double r = pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
      fprintf(f2,"\t%.6f",r);
    }
    fprintf(f2,"\n");

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      double z = pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;

      fprintf(f2,"%.6f",z);

      for (int indx = 0; indx < pln->noR ; indx++ )
      {
        float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
        fprintf(f2,"\t%.15f",yy2);
      }
      fprintf(f2,"\n");
    }
    fclose(f2);

    FOLD // Make image  .
    {
      nvtxRangePush("Image");
      char cmd[1024];
      sprintf(cmd,"python ~/bin/bin/plt_ffd.py %s", tName);
      system(cmd);
      nvtxRangePop();
    }

    nvtxRangePop();
  }

  FOLD // Get new max  .
  {
    nvtxRangePush("Get Max");

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      for (int indx = 0; indx < pln->noR ; indx++ )
      {
        float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
        if ( yy2 > cand->power )
        {
          cand->power   = yy2;
          cand->r       = pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
          cand->z       = pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;
        }
      }
    }

    nvtxRangePop();
  }
}

template<typename T>
void opt_candBySwrm(accelcand* cand, fftInfo* fft, cuOptCand* pln, int noP, double scale, int plt = -1, int nn = 0 )
{
  FOLD // Large points  .
  {
    pln->centR          = cand->r;
    pln->centZ          = cand->z;
    pln->noZ            = noP*2 + 1;
    pln->noR            = noP*2 + 1;
    pln->rSize          = scale;
    pln->zSize          = scale*4.0;

    ffdotSwrm<T>(pln, fft);
  }

  FOLD // A blocking synchronisation to ensure results are ready to be proceeded by the host
  {
    nvtxRangePush("EventSynch");
    CUDA_SAFE_CALL(hipEventSynchronize(pln->outCmp), "Synchronising using hipEventSynchronize");
    nvtxRangePop();
  }

  FOLD // Get new max  .
  {
    float max = ((candOpt*)pln->h_out)[0].power;

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      for (int indx = 0; indx < pln->noR ; indx++ )
      {
        float yy2 = ((candOpt*)pln->h_out)[indy*pln->noR+indx].power;
        if ( yy2 > max )
        {
          max = yy2;
          cand->r     = ((candOpt*)pln->h_out)[indy*pln->noR+indx].r;
          cand->z     = ((candOpt*)pln->h_out)[indy*pln->noR+indx].z;
          cand->power = yy2;

          //printf("New max at %04i r: %15.3f   z: %15.3f \n", indy*pln->noR+indx, cand->r, cand->z );
        }
      }
    }
  }

  //printf("Best point Current r: %10.5f z: %10.5f  power: %20.6f \n", cand->r, cand->z, cand->power);
}

template<int n>
void cdfgam_d(double x, double *p, double* q)
{
  if      ( n == 1  )
  {
    *q = exp(-x);
  }
  else if ( n == 2  )
  {
    *q = exp(-x)*( x + 1.0 );
  }
  else if ( n == 4  )
  {
    *q = exp(-x)*( x*(x*(x/6.0 + 0.5) + 1.0 ) + 1.0 );
  }
  else if ( n == 8  )
  {
    *q = exp(-x)*( x*(x*(x*(x*(x*(x*(x/5040.0 + 1.0/720.0 ) + 1.0/120.0 ) + 1.0/24.0 ) + 1.0/6.0 ) + 0.5 ) + 1.0 ) + 1.0 );
  }
  else if ( n == 16 )
  {
    *q = exp(-x)*( x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x*(x/1.307674368e12 +  1.0/8.71782912e10 ) \
        + 1.0/6227020800.0 )+ 1.0/479001600.0 ) \
        + 1.0/39916800.0 )+ 1.0/3628800.0 )     \
        + 1.0/362880.0 ) + 1.0/40320.0 )        \
        + 1.0/5040.0 ) + 1.0/720.0 ) + 1.0/120.0 ) + 1.0/24.0 ) + 1.0/6.0 ) + 0.5 ) + 1.0 )  + 1.0 );
  }
  else
  {
    *q = 1.0 + x ;
    double numerator    = x;
    double denominator  = 1.0;

#pragma unroll
    for ( int i = 2 ; i < n ; i ++ )
    {
      denominator *= i;
      numerator   *= x;
      *q += numerator/denominator;
    }
  }
  *p = 1-*q;
}

double gammln(double xx)
{
  double x,tmp,ser;
  static double cof[6]= { 76.18009173, -86.50532033, 24.01409822, -1.231739516, 0.120858003e-2, -0.536382e-5 };
  int j;

  x       =   xx - 1.0;
  tmp     =   x + 5.5;
  tmp     -=  (x+0.5)*log(tmp);
  ser     =   1.0;

  for ( j=0; j<=5;  j++ )
  {
    x     += 1.0;
    ser   += cof[j]/x;
  }

  return -tmp + log(2.50662827465*ser);
}

static const int ngau = 18;

const double y[18] = {0.0021695375159141994,0.011413521097787704,0.027972308950302116,0.051727015600492421,0.082502225484340941,0.12007019910960293, 0.16415283300752470, 0.21442376986779355, 0.27051082840644336, 0.33199876341447887, 0.39843234186401943, 0.46931971407375483, 0.54413605556657973, 0.62232745288031077, 0.70331500465597174, 0.78649910768313447, 0.87126389619061517, 0.95698180152629142  };
const double w[18] = {0.0055657196642445571,0.012915947284065419,0.020181515297735382,0.027298621498568734,0.034213810770299537,0.040875750923643261,0.047235083490265582,0.053244713977759692,0.058860144245324798,0.064039797355015485,0.068745323835736408,0.072941885005653087,0.076598410645870640,0.079687828912071670,0.082187266704339706,0.084078218979661945,0.085346685739338721,0.085983275670394821 };

//Incomplete gamma by quadrature. Returns P .a; x/ or Q.a; x/, when psig is 1 or 0,
//respectively. User should not call directly.
double gammpapprox(double a, double x, int psig)
{
  double  xu,t,sum,ans;
  double  a1      = a-1.0;
  double  lna1    = log(a1);
  double  sqrta1  = sqrt(a1);
  double  gln     = gammln(a);

  //Set how far to integrate into the tail:
  if (x > a1)
    xu = MAX(a1 + 11.5*sqrta1, x + 6.0*sqrta1);
  else
    xu = MAX(0.,MIN(a1 - 7.5*sqrta1, x - 5.0*sqrta1));

  sum = 0;

  for ( int j=0; j < ngau; j++) // Gauss-Legendre
  {
    t = x + (xu-x)*y[j];
    sum += w[j]*exp(-(t-a1)+a1*(log(t)-lna1));
  }
  ans = sum*(xu-x)*exp(a1*(lna1-1.)-gln);
  return (psig?(ans>0.0? 1.0-ans:-ans):(ans>=0.0? ans:1.0+ans));
}

double gser(const double a, const double x)
{
  //Returns the incomplete gamma function P .a; x/ evaluated by its series representation.
  //Also sets ln .a/ as gln. User should not call directly.
  double sum,del,ap, gln;

  gln=gammln(a);
  ap=a;
  del=sum=1.0/a;
  for (;;)
  {
    ++ap;
    del *= x/ap;
    sum += del;
    if (fabs(del) < fabs(sum)*EPS)
    {
      return sum*exp(-x+a*log(x)-gln);
    }
  }
}

double gcf(const double a, const double x)
{
  //Returns the incomplete gamma function Q.a; x/ evaluated by its continued fraction rep-
  //resentation. Also sets ln .a/ as gln. User should not call directly.
  int i;
  double an,b,c,d,del,h;

  double gln  = gammln(a);
  b           = x+1.0-a;
  //Set up for evaluating continued fraction
  c           = 1.0/FPMIN;
  //by modified Lentz’s method (5.2)
  d           = 1.0/b;
  //with b0 D 0.
  h           = d;

  for (i=1;;i++)
  {
    //Iterate to convergence.
    an = -i*(i-a);
    b += 2.0;
    d=an*d+b;
    if (fabs(d) < FPMIN)
      d=FPMIN;
    c=b+an/c;
    if (fabs(c) < FPMIN)
      c=FPMIN;
    d=1.0/d;
    del=d*c;
    h *= del;
    if (fabs(del-1.0) <= EPS)
      break;
  }

  return exp(-x+a*log(x)-gln)*h;
  //Put factors in front.
}

//Returns the incomplete gamma function P .a; x/.
double gammp(const double a, const double x)
{
  if (x < 0.0 || a <= 0.0)
  {
    throw("bad args in gammp");
  }
  if (x == 0.0)
  {
    return 0.0;
  }
  else if ((int)a >= 100 )                      // Quadrature  .
  {
    return gammpapprox(a,x,1);
  }
  else if (x < a+1.0)                           // Use the series representation  .
  {
    return gser(a,x);
  }
  else                                          // Use the continued fraction representation  .
  {
    return 1.0-gcf(a,x);
  }

}

double gammq(const double a, const double x)
{
  //Returns the incomplete gamma function Q.a; x/ Á 1 P .a; x/.
  if (x < 0.0 || a <= 0.0)
    throw("bad args in gammq");
  if (x == 0.0)
    return 1.0;
  else if ((int)a >= 100)         // Quadrature.
    return gammpapprox(a,x,0);
  else if (x < a+1.0)             // Use the series representation.
    return 1.0-gser(a,x);
  else                            // Use the continued fraction representation.
    return gcf(a,x);
}

double logIGamma_i(int s, double x )
{
  //double x = 1.592432984e8 ;
  //int s = 10;

  double num = pow(x,0) ;
  double den = 1;

  double sum = num/den;
  double trm;

  for( int k = 1; k <= s-1; k++ )
  {
    num   = pow(x,k) ;
    den  *= k;

    trm   = num/den;

    sum  += trm;

    printf("%03i  trm %6e   sum: %6e \n", k, trm, sum );
  }

  double t1 = lgamma((double)s) ;
  double t2 = -x ;
  double t3 = log(sum) ;
  return t1 + t2 + t3 ;
}

double logQChi2_i(int s, double x )
{
  double sum = 0 ;
  double num;
  double den;
  double trm;
  double sum0;

  for( int k = s-1; k >= 0 ; k-- )
  {
    sum0  = sum;
    num   = pow(x,k) ;
    den   = boost::math::factorial<double>(k);
    trm   = num/den;
    sum  += trm;

    if ( sum-sum0 == 0 )
      break;
  }

  double t2 = -x ;
  double t3 = log(sum) ;
  return t2 + t3 ;
}

void calcNQ(double x, long long n, double* p, double* q)
{
  double qq  = 0;
  double pp  = 1;

  double trueV = 1-pow((1-x),n);

  if ( trueV > 0.95 )
  {
    *q = 1.0-pow((long double)(1.0-x),(long double)n);
    *p =     (long double)pow((long double)(1.0-x),(long double)n);
    return;
  }

  //  if ( trueV > 0.9 )
  //  {
  //    int tmp = 0;
  //
  //    {
  //      double term = 1;
  //      long long k = 0;
  //      double sum0 = sum;
  //      double dff ;
  //      double coef = 1;
  //      double fact = 1;
  //
  //      int sz1 = sizeof(double);
  //      int sz2 = sizeof(coef);
  //
  //      do
  //      {
  //        sum0 = sum;
  //        coef *= ( n - (k) );
  //        k++;
  //        fact *= k;
  //        double bcoef1 = coef / fact ;
  //        double bcoef2 = boost::math::binomial_coefficient<double>(n,k);
  //
  //        double t1   = pow(-x,k);
  //        term = bcoef1*t1;
  //        sum -= term;
  //        dff = fabs(sum0-sum);
  //
  //        printf("calcNQ %03i sum: %9.4e  term: %9.6e   dff: %7.3e  bcoef1 %12.8e    bcoef2: %12.8e  \n", k-1, sum, term, dff, bcoef1, bcoef2 );
  //      }
  //      while ( dff > 0 && k < n && k <= 10 );
  //    }
  //  }

  //  if ( x > 1e-8 || trueV > 0.4 )
  //  {
  //    return trueV;
  //  }

  FOLD // Else do a series expansion  .
  {
    double term = 1;
    long long k = 0;
    double  sum0 = qq;
    double  dff ;
    double  coef = 1;
    double  fact = 1;

    qq = 0;

    do
    {
      sum0 = qq;
      coef *= ( n - (k) );
      k++;
      fact *= k;
      double bcoef = coef / fact ;

      double t1   = pow(-x,k);

      if( t1 == 0 )
      {
        if ( k > 1 )
        {
          *p = pp ;
          *q = qq ;
          return;
        }
        else
        {
          *p = 1 - n * x;
          *q =     n * x;
          return;
        }
      }


      term = bcoef*t1;
      qq -= term;
      pp  += term;
      dff = fabs(sum0-qq);

      //      if ( trueV > 0.5 )
      //printf("calcNQ %03i sum: %.4e  term: %.6e   dff: %.3e\n", k-1, pp, term, dff );
    }
    while ( dff > 0 && k < n && k <= 20 );

    *p = pp ;
    *q = qq ;
  }
}

/**
 * This is thread safe!
 * @param poww
 * @param numharm
 * @param numindep
 * @return
 */
double candidate_sigma_cl(double poww, int numharm, long long numindep)
{
  int     k       = numharm * 2.0 ;     // Each harm is 2 powers
  double  gamP    = poww * 2.0 ;        // A just for normalisation of powers

  double logQ, gpu_p, gpu_q, sigc ;

  int     n       = numharm;

  if ( poww > 100 )
  {
    if      ( n == 1  )
    {
      logQ = -poww;
    }
    else if ( n == 2  )
    {
      logQ = -poww+log( poww + 1.0 );
    }
    else if ( n == 4  )
    {
      logQ = -poww + log( poww*(poww*(poww/6.0 + 0.5) + 1.0 ) + 1.0 );
    }
    else if ( n == 8  )
    {
      logQ = -poww + log( poww*(poww*(poww*(poww*(poww*(poww*(poww/5040.0 + 1.0/720.0 ) + 1.0/120.0 ) + 1.0/24.0 ) + 1.0/6.0 ) + 0.5 ) + 1.0 ) + 1.0 );
    }
    else if ( n == 16 )
    {
      logQ = -poww + log( poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww*(poww/1.307674368e12 +  1.0/8.71782912e10 ) \
          + 1.0/6227020800.0 )+ 1.0/479001600.0 ) \
          + 1.0/39916800.0 )+ 1.0/3628800.0 ) \
          + 1.0/362880.0 ) + 1.0/40320.0 ) \
          + 1.0/5040.0 ) + 1.0/720.0 ) + 1.0/120.0 ) + 1.0/24.0 ) + 1.0/6.0 ) + 0.5 ) + 1.0 )  + 1.0 );
    }
    else
    {
      logQ = logQChi2_i(k / 2.0, gamP / 2.0 ) ;
    }

    logQ    += log( (double)numindep );

    double l = sqrt(-2.0*logQ);
    sigc     = l - ( 2.515517 + l * (0.802853 + l * 0.010328) ) / ( 1.0 + l * (1.432788 + l * (0.189269 + l * 0.001308)) ) ;

    return sigc;
  }
  else
  {
    if      ( numharm == 1 )
      cdfgam_d<1>(poww, &gpu_p, &gpu_q );
    else if ( numharm == 2 )
      cdfgam_d<2>(poww, &gpu_p, &gpu_q );
    else if ( numharm == 4 )
      cdfgam_d<4>(poww, &gpu_p, &gpu_q );
    else if ( numharm == 8 )
      cdfgam_d<8>(poww, &gpu_p, &gpu_q );
    else if ( numharm == 16)
      cdfgam_d<16>(poww, &gpu_p, &gpu_q );
    else
    {
      gpu_p = boost::math::gamma_p<double>(k / 2.0, gamP / 2.0 ) ;
      gpu_q = boost::math::gamma_q<double>(k / 2.0, gamP / 2.0 ) ;
    }

    // Correct q for number of trials
    calcNQ(gpu_q, numindep, &gpu_p, &gpu_q);

    sigc = incdf(gpu_p, gpu_q);

    return sigc;
  }
}

void opt_candPlns(accelcand* cand, accelobs* obs, int nn, cuOptCand* pln)
{
  int ii;
  int *r_offset;
  fcomplex **data;

  //double r, z;
  //int noP;
  //float scale;
  //struct timeval start, end, start1, end1;
  //double timev1, timev2, timev3;

  //printf("%4i  optimize_accelcand  harm %2i   r %20.4f   z %7.3f  pow: %8.3f  sig: %8.4f\n", nn, cand->numharm, cand->r, cand->z, cand->power, cand->sigma );

  int maxHarms  = 16;
  maxHarms      = cand->numharm ;

  int numdata   = obs->numbins;

  cand->pows    = gen_dvect(maxHarms);
  cand->hirs    = gen_dvect(maxHarms);
  cand->hizs    = gen_dvect(maxHarms);
  cand->derivs  = (rderivs *)   malloc(sizeof(rderivs)  * maxHarms  );
  r_offset      = (int*)        malloc(sizeof(int)      * maxHarms  );
  data          = (fcomplex**)  malloc(sizeof(fcomplex*)* maxHarms  );

  pln->centR    = cand->r ;
  pln->centZ    = cand->z ;
  //pln->noHarms  = cand->numharm ;
  pln->noHarms  = maxHarms ;

  fftInfo fft;
  fft.fft       = obs->fft;
  fft.rlo       = obs->lobin;
  fft.nor       = obs->numbins;
  fft.idx       = obs->lobin;
  fft.rhi       = obs->lobin + obs->numbins;

  for ( int i=1; i <= maxHarms; i++ )
  {
    pln->norm[i-1]  = get_scaleFactorZ(fft.fft, fft.nor, (fft.idx+pln->centR)*i-fft.rlo, pln->centZ*i, 0.0);
  }

  if ( obs->use_harmonic_polishing )
  {
    //if ( obs->mmap_file || obs->dat_input )
    {
      for( ii=0; ii < maxHarms; ii++ )
      {
        r_offset[ii]  = obs->lobin;
        data[ii]      = obs->fft;
      }

      FOLD // GPU grid  .
      {
        int rep       = 0;
        int lrep      = 0;
        int noP       = 30;
        float snoop   = 0.3;
        float sz;
        float v1, v2;

        const int mxRep = 10;

        if ( cand->numharm == 1  )
          sz = optSz01;
        if ( cand->numharm == 2  )
          sz = optSz02;
        if ( cand->numharm == 4  )
          sz = optSz04;
        if ( cand->numharm == 8  )
          sz = optSz08;
        if ( cand->numharm == 16 )
          sz = optSz16;

        //int numindep        = (obs->rhi - obs->rlo ) * (obs->zhi +1 ) * (ACCEL_DZ / 6.95) / pln->noHarms ;
        //printf("\n%03i  r: %15.6f   z: %12.6f \n", nn, cand->r, cand->z);

        pln->halfWidth = 0;

        if ( optpln01 > 0 )
        {
          noP           = optpln01 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;
            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            //            cand->sigma   = candidate_sigma_cl(cand->power, pln->noHarms, numindep );
            //            printf("      optimize_accelcand  harm %2i   r %20.4f   z %7.3f  pow: %8.3f  sig: %8.4f\n", pln->noHarms, pln->centR, pln->centZ, cand->power, cand->sigma );

            if ( ++lrep > mxRep )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale;
        }

        if ( optpln02 > 0 )
        {
          noP           = optpln02 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;
            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            //            cand->sigma   = candidate_sigma_cl(cand->power, pln->noHarms, numindep );
            //            printf("      optimize_accelcand  harm %2i   r %20.4f   z %7.3f  pow: %8.3f  sig: %8.4f\n", pln->noHarms, pln->centR, pln->centZ, cand->power, cand->sigma );

            if ( ++lrep > mxRep )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale;
        }

        if ( optpln03 > 0 )
        {
          noP           = optpln03 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;
            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            //            cand->sigma   = candidate_sigma_cl(cand->power, pln->noHarms, numindep );
            //            printf("      optimize_accelcand  harm %2i   r %20.4f   z %7.3f  pow: %8.3f  sig: %8.4f\n", pln->noHarms, pln->centR, pln->centZ, cand->power, cand->sigma );

            if ( ++lrep > mxRep )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale*2;
        }

        if ( optpln04 > 0 )
        {
          noP           = optpln04 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;

            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            //            cand->sigma   = candidate_sigma_cl(cand->power, pln->noHarms, numindep );
            //            printf("      optimize_accelcand  harm %2i   r %20.4f   z %7.3f  pow: %8.3f  sig: %8.4f\n", pln->noHarms, pln->centR, pln->centZ, cand->power, cand->sigma );

            if ( ++lrep > mxRep )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale*2;
        }

        if ( optpln05 > 0 )
        {
          noP           = optpln05 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;

            opt_candByPln<float>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            //            cand->sigma   = candidate_sigma_cl(cand->power, pln->noHarms, numindep );
            //            printf("      optimize_accelcand  harm %2i   r %20.4f   z %7.3f  pow: %8.3f  sig: %8.4f\n", pln->noHarms, pln->centR, pln->centZ, cand->power, cand->sigma );

            if ( ++lrep > mxRep )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale*2;
        }

        if ( optpln06 > 0 )
        {
          noP           = optpln06 ;
          lrep          = 0;
          cand->power   = 0;     // Set initial power to zero
          do
          {
            pln->centR    = cand->r ;
            pln->centZ    = cand->z ;

            opt_candByPln<double>(cand, &fft, pln, noP, sz,  rep++, nn );
            v1 = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
            v2 = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

            //            cand->sigma   = candidate_sigma_cl(cand->power, pln->noHarms, numindep );
            //            printf("      optimize_accelcand  harm %2i   r %20.4f   z %7.3f  pow: %8.3f  sig: %8.4f\n", pln->noHarms, pln->centR, pln->centZ, cand->power, cand->sigma );

            if ( ++lrep > mxRep )
            {
              break;
            }
          }
          while ( v1 > snoop || v2 > snoop );
          sz /= downScale*2;
        }
      }

      FOLD // Optimise derivatives  .
      {
        nvtxRangePush("Opt derivs");

        optemiseDerivs(data, maxHarms, r_offset, numdata, cand->r, cand->z, cand->derivs, cand->pows, nn);

        for( ii=0; ii < maxHarms; ii++ )
        {
          cand->hirs[ii]  = (cand->r+obs->lobin)*(ii+1);
          cand->hizs[ii]  = cand->z*(ii+1);
        }

        FOLD // Update fundamental values to the optimised ones  .
        {
          float   maxSig      = 0;
          int     bestH       = 0;
          float   bestP       = 0;
          double  sig         = 0; // can be a float
          int     numindep;

          //double sSig;
          //int     numindepS   = (obs->rhi - obs->rlo ) * (obs->zhi +1 ) * (ACCEL_DZ / 6.95) ;
          //float   sPower      = 0;
          //int     noS         = 0;

          cand->power         = 0;
          for( ii=0; ii < maxHarms; ii++ )
          {
            if ( cand->derivs[ii].locpow > 0 )
            {
              float lPower    =  cand->derivs[ii].pow/cand->derivs[ii].locpow;
              cand->power     += lPower;
              numindep        = (obs->rhi - obs->rlo ) * (obs->zhi +1 ) * (ACCEL_DZ / 6.95) / (ii+1) ;
              sig             = candidate_sigma_cl(cand->power, (ii+1), numindep );

              //              sSig           = candidate_sigma_cl(lPower, 1, numindepS );
              //              if ( lPower > 3 )
              //              {
              //                sPower      += lPower;
              //                noS++;
              //              }
              //              printf("          %02i  pow: %8.3f  sig: %8.4f   Sum: pow: %8.3f  sig: %8.4f\n", ii+1, lPower, sSig, cand->power, sig );

              if ( sig > maxSig )
              {
                maxSig        = sig;
                bestP         = cand->power;
                bestH         = (ii+1);
              }
            }
          }


          //          numindep        = (obs->rhi - obs->rlo ) * (obs->zhi +1 ) * (ACCEL_DZ / 6.95) / (maxHarms) ;
          //          sSig            = candidate_sigma(sPower, maxHarms, numindep );
          //          printf("\n" );
          //          printf("              pow: %8.3f  sig: %8.4f\n", sPower, sSig );
          //          printf("---------------------\n" );

          cand->numharm = bestH;
          cand->sigma   = maxSig;
          cand->power   = bestP;

        }

        //        noStages      = log2((double)cand->numharm);
        //        cand->sigma   = candidate_sigma(cand->power, cand->numharm, obs->numindep[noStages]);

        nvtxRangePop();
      }

    }
  }
}

void opt_candSwrm(accelcand* cand, accelobs* obs, int nn, cuOptCand* pln)
{
  int ii;
  int *r_offset;
  fcomplex **data;

  int numdata   = obs->numbins;

  cand->pows    = gen_dvect(cand->numharm);
  cand->hirs    = gen_dvect(cand->numharm);
  cand->hizs    = gen_dvect(cand->numharm);
  cand->derivs  = (rderivs *)  malloc(sizeof(rderivs) * cand->numharm);
  r_offset      = (int*) malloc(sizeof(int)*cand->numharm);
  data          = (fcomplex**) malloc(sizeof(fcomplex*)*cand->numharm);

  pln->centR    = cand->r ;
  pln->centZ    = cand->z ;
  pln->noHarms  = cand->numharm ;

  fftInfo fft;
  fft.fft       = obs->fft;
  fft.rlo       = obs->lobin;
  fft.nor       = obs->numbins;
  fft.idx       = obs->lobin;
  fft.rhi       = obs->lobin + obs->numbins;

  for ( int i=1; i <= cand->numharm; i++ )
  {
    pln->norm[i-1]  = get_scaleFactorZ(fft.fft, fft.nor, (fft.idx+pln->centR)*i-fft.rlo, pln->centZ*i, 0.0);
  }

  if ( obs->use_harmonic_polishing )
  {
    if ( obs->mmap_file || obs->dat_input )
    {
      for( ii=0; ii<cand->numharm; ii++ )
      {
        r_offset[ii]   = obs->lobin;
        data[ii]       = obs->fft;
      }

      FOLD // GPU swarm  .
      {
        int rep = 0;
        int noP = 20;
        float sz;

        if ( cand->numharm == 1 )
          sz = 16;
        if ( cand->numharm == 2 )
          sz = 14;
        if ( cand->numharm == 4 )
          sz = 12;
        if ( cand->numharm == 8 )
          sz = 10;
        if ( cand->numharm == 16 )
          sz = 8;

        //printf("\n%03i  r: %15.6f   z: %12.6f \n", nn, cand->r, cand->z);

        opt_candBySwrm<float>(cand, &fft, pln, noP, sz,  rep++, nn );
      }

      FOLD // Optimise derivatives  .
      {
        optemiseDerivs(data, cand->numharm, r_offset, numdata, cand->r, cand->z, cand->derivs, cand->pows, nn);

        for( ii=0; ii < cand->numharm; ii++ )
        {
          cand->hirs[ii]=(cand->r+obs->lobin)*(ii+1);
          cand->hizs[ii]=cand->z*(ii+1);
        }

        FOLD // Update fundamental values to the optimised ones
        {
          cand->power = 0;
          for( ii=0; ii < cand->numharm; ii++ )
          {
            cand->power += cand->derivs[ii].pow/cand->derivs[ii].locpow;
          }
        }

        int noStages = log2((double)cand->numharm);
        cand->sigma = candidate_sigma_cl(cand->power, cand->numharm, obs->numindep[noStages]);
      }

      //printf("Opt point          r: %10.5f z: %10.5f  power: %20.6f   sigma: %6.3f \n", cand->r, cand->z, cand->power, cand->sigma);
    }
  }
}
