#include "hip/hip_runtime.h"
/** @file cuda_cand_OPT.cu
 *  @brief Utility functions and kernels for GPU optimisation
 *
 *
 *  @author Chris Laidler
 *  @bug No known bugs.
 *
 *  Change Log
 *
 *  [0.0.01] []
 *    Beginning of change log
 *    Working version un-numbed
 *
 *  [0.0.02] [2017-02-16]
 *    Separated candidate and optimisation CPU threading
 *
 */

#include <hiprand.h>
#include <math.h>		// log
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <stdint.h>		// uint64_t

#include "cuda_math_ext.h"
#include "cuda_accel.h"
#include "cuda_utils.h"
#include "cuda_accel_utils.h"
#include "candTree.h"
#include "cuda_response.h"
#include "cuda_cand_OPT.h"


#define SCALE_AUT       (1000000000)

#define OPT_INP_BUF   25

extern "C"
{
#define __float128 long double
#include "accel.h"
}

#define		NM_BEST		0
#define		NM_MIDL		1
#define		NM_WRST		2

#define		WITH_

#define SWAP_PTR(p1, p2) do { initCand* tmp = p1; p1 = p2; p2 = tmp; } while (0)

#ifdef WITH_OPT_BLK1

template<typename T, int noBlk>
__global__ void ffdotPlnByBlk_ker(float* powers, float2* data, int noHarms, int halfwidth, double firstR, double firstZ, double zSZ, int noR, int noZ, int blkWidth, int iStride, int oStride, optLocInt_t loR, optLocFloat_t norm, optLocInt_t hw)
{
  const int ix = blockIdx.x * blockDim.x + threadIdx.x;
  const int iy = blockIdx.y * blockDim.y + threadIdx.y;

  if ( ix < noR && iy < noZ)
  {
    double r            = firstR + ix*blkWidth/(double)(noR) ;
    double z            = firstZ - iy/(double)(noZ-1) * zSZ ;
    if (noZ == 1)
      z = 0;

    float       total_power[noBlk];
    float2      ans[noBlk];
    int halfW;

    for( int blk = 0; blk < noBlk; blk++ )
    {
      total_power[blk] = 0;
    }

    FOLD
    {
      for( int i = 1; i <= noHarms; i++ )           // Loop over harmonics
      {
	double absz         = fabs(z*i);

	FOLD // Determine half width
	{
	  if ( hw.val[i-1] )
	  {
	    halfW	= hw.val[i-1];
	  }
	  else
	  {
	    halfW       = cu_z_resp_halfwidth_high<float>(z*i); // NB this was (z*i+4) I'm not sure why?
	  }
	}

	// Set complex values to 0 for this harmonic
	for( int blk = 0; blk < noBlk; blk++ )
	{
	  ans[blk].x = 0;
	  ans[blk].y = 0;
	}

	FOLD // Calculate complex value, using direct application of the convolution
	{
	  rz_convolution_cu<T, float2, float2, noBlk>(&data[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, halfW, ans, blkWidth*i);
	}

	// Calculate power for the harmonic
	for( int blk = 0; blk < noBlk; blk++ )
	{
	  total_power[blk] += POWERF(ans[blk]);
	}
      }
    }

    // Write values back to
    for( int blk = 0; blk < noBlk; blk++ )
    {
      powers[iy*oStride + blk*noR + ix] = total_power[blk];
    }
  }
}

#endif

#ifdef WITH_OPT_BLK2

__global__ void opt_genResponse_ker(cuRespPln pln)
{
  const int ix  = blockIdx.x * blockDim.x + threadIdx.x;
  const int iy  = blockIdx.y * blockDim.y + threadIdx.y;

  const int firstBin = ix / pln.noRpnts;
  const int is  = ix % pln.noRpnts;

  if ( iy < pln.noZ && ix < pln.noR )
  {
    const double frac =  is / (double)pln.noRpnts ;

    double     zVal   = pln.zMax - (double)iy*pln.dZ ;
    double     offSet = -pln.halfWidth - frac  +  firstBin ;

    double2 response = calc_response_off(offSet, zVal);

    // Write values to memory
    pln.d_pln[iy*pln.oStride + ix ].x = (float)response.x;
    pln.d_pln[iy*pln.oStride + ix ].y = (float)response.y;
  }
}

template<typename T, int noBlk>
__global__ void ffdotPlnByBlk_ker2(float2* powers, float2* data, cuRespPln pln, int noHarms, int halfwidth, int zIdxTop, int rIdxLft, double zSZ, int noR, int noZ, int blkWidth, int iStride, int oStride, optLocInt_t loR, optLocFloat_t norm, optLocInt_t hw)
{
  const int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int ty = blockIdx.y * blockDim.y + threadIdx.y;

  const int is = tx % pln.noRpnts ;		///<
  const int ix = tx / pln.noRpnts ;
  const int iy = ty;

  if ( iy < noZ )
  {
    int plnCent = (pln.noZ)/2 ;

    int zIdx = plnCent - zIdxTop - iy ;

    float2	kerVal;
    float2	outVal;
    int 	halfW;

    FOLD
    {
      for( int i = 0; i < noHarms; i++ )           // Loop over harmonics
      {
	int 	hrm 	= i+1;

	int	zhIdx;		///< The harmonic specific Z index in the response
	int	shIdx;		///< The harmonic specific response step index
	int 	rhIdx;		///< The harmonic specific

	zhIdx 	= plnCent - zIdx * hrm;
	shIdx 	= is * hrm;					// Multiply we need the
	rhIdx 	= rIdxLft * hrm + shIdx / pln.noRpnts;		// Need the int part
	shIdx 	= shIdx % pln.noRpnts ;				// Adjust the remainder

	double 	zh = -1;

	FOLD // Determine half width
	{
	  if ( hw.val[i] )
	  {
	    halfW	= hw.val[i];
	  }
	  else
	  {
	    zh	= zIdx * hrm * pln.dZ;
	    halfW       = cu_z_resp_halfwidth_high<float>(zh); // NB this was (z*i+4) I'm not sure why?
	  }
	}

	if ( ix < halfW*2 )
	{
	  double off	= (ix - halfW);

	  int	hRidx 	= pln.halfWidth + (ix - halfW);

	  int khIdx = (hRidx) * pln.noRpnts + shIdx ;

	  if ( zhIdx  >= 0 && zhIdx < pln.noZ && hRidx >= 0 && hRidx < pln.halfWidth*2 )
	  {
	    kerVal = pln.d_pln[zhIdx*pln.oStride + khIdx ];
	  }
	  else
	  {
	    kerVal = calc_response_off((float)off, (float)zh);
	  }

	  int start = rhIdx - halfW - loR.val[i]  ;

	  // Calculate power for the harmonic
	  for( int blk = 0; blk < noBlk; blk++ )
	  {
	    float2 inp = data[iStride*i + start + blk*hrm + ix ];

#if CORRECT_MULT
	    // This is the "correct" version
	    outVal.x = (kerVal.x * inp.x - kerVal.y * inp.y);
	    outVal.y = (kerVal.x * inp.y + kerVal.y * inp.x);
#else
	    // This is the version accelsearch uses, ( added for comparison )
	    outVal.x = (kerVal.x * inp.x + kerVal.y * inp.y);
	    outVal.y = (kerVal.y * inp.x - kerVal.x * inp.y);
#endif

	    // if ( ix == 0 )
	    {
	      atomicAdd(&(powers[iy*oStride*noHarms + (is + blk*pln.noRpnts)*noHarms + i].x), (float)(outVal.x));
	      atomicAdd(&(powers[iy*oStride*noHarms + (is + blk*pln.noRpnts)*noHarms + i].y), (float)(outVal.y));
	    }
	  }
	}
      }
    }
  }
}

#endif

#ifdef WITH_OPT_BLK3
template<typename T, int noBlk>
__global__ void ffdotPlnByBlk_ker3(float* powers, float2* fft, int noHarms, int harmWidth, double firstR, double firstZ, double zSZ, int noR, int noZ, int blkWidth, int iStride, int oStride, optLocInt_t loR, optLocFloat_t norm, optLocInt_t hw)
{
  const int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int ty = blockIdx.y * blockDim.y + threadIdx.y;

  const int i  = tx / harmWidth;
  const int ix = tx % harmWidth;
  const int iy = ty;

  if ( ix < noR && iy < noZ)
  {
    double r            = firstR + ix*blkWidth/(double)(noR) ;
    double z            = firstZ - iy/(double)(noZ-1) * zSZ ;
    if (noZ == 1)
      z = 0;

    float2      ans[noBlk];
    int halfW;

    FOLD
    {
      //for( int i = 0; i < noHarms; i++ )           // Loop over harmonics
      {
	int hrm = i+1;

	double absz         = fabs(z*hrm);

	FOLD // Determine half width
	{
	  if ( hw.val[i] )
	  {
	    halfW	= hw.val[i];
	  }
	  else
	  {
	    halfW       = cu_z_resp_halfwidth_high<float>(z*hrm); // NB this was (z*hrm+4) I'm not sure why?
	  }
	}

	// Set complex values to 0 for this harmonic
	for( int blk = 0; blk < noBlk; blk++ )
	{
	  ans[blk].x = 0;
	  ans[blk].y = 0;
	}

	FOLD // Calculate complex value, using direct application of the convolution
	{
	  rz_convolution_cu<T, float2, float2, noBlk>(&fft[iStride*i], loR.val[i], iStride, r*hrm, z*hrm, halfW, ans, blkWidth*hrm);
	}
      }
    }

    // Write values back to
    for( int blk = 0; blk < noBlk; blk++ )
    {
      float power = POWERF(ans[blk]);
      atomicAdd(&(powers[iy*oStride + blk*noR + ix]), power);
    }
  }
}
#endif


#ifdef WITH_OPT_PLN1
template<typename T>
__global__ void ffdotPln_ker(float* powers, float2* data, int noHarms, int halfwidth, double firstR, double firstZ, double rSZ, double zSZ, int noR, int noZ, int iStride, int oStride, optLocInt_t loR, optLocFloat_t norm, optLocInt_t hw)
{
  const int ix = blockIdx.x * blockDim.x + threadIdx.x;
  const int iy = blockIdx.y * blockDim.y + threadIdx.y;

  if ( ix < noR && iy < noZ)
  {
    int halfW;
    double r            = firstR + ix/(double)(noR-1) * rSZ ;
    double z            = firstZ - iy/(double)(noZ-1) * zSZ ;
    if (noZ == 1)
      z = 0;

    T total_power  = 0;
    T real = 0;
    T imag = 0;

    for( int i = 0; i < noHarms; i++ )
    {
      int hrm = i+1;
      FOLD // Determine half width
      {
	if ( hw.val[i] )
	  halfW	= hw.val[i];
	else
	  halfW	= cu_z_resp_halfwidth_high<float>(z*hrm);
      }

      rz_convolution_cu<T, float2>(&data[iStride*i], loR.val[i], iStride, r*hrm, z*hrm, halfW, &real, &imag);

      total_power     += POWERCU(real, imag);
    }

    powers[iy*oStride + ix] = total_power;
  }
}
#endif

#ifdef WITH_OPT_PLN2
template<typename T>
__global__ void ffdotPln_ker2(float2* powers, float2* data, int noHarms, int halfwidth, double firstR, double firstZ, double rSZ, double zSZ, int noR, int noZ, int iStride, int oStride, optLocInt_t loR, optLocFloat_t norm, optLocInt_t hw)
{
  const int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int ty = blockIdx.y * blockDim.y + threadIdx.y;

  int off = tx % halfwidth ;
  int hrm = tx / halfwidth ;

  const int ix = ty % noR;
  const int iy = ty / noR;

  if ( ix < noR && iy < noZ)
  {
    int halfW;
    double r            = firstR + ix/(double)(noR-1) * rSZ ;
    double z            = firstZ - iy/(double)(noZ-1) * zSZ ;
    if (noZ == 1)
      z = 0;

    T real = 0;
    T imag = 0;

//    FOLD
//    {
//      FOLD // Determine half width
//      {
//        if ( hw.val[hrm] )
//          halfW = hw.val[hrm];
//        else
//          halfW = cu_z_resp_halfwidth_high<float>(z*(hrm+1));
//      }
//
//      if (off < halfW*2 )
//      {
//        rz_single_mult_cu<T, float2>(&fft[iStride*hrm], loR.val[hrm], iStride, r*(hrm+1), z*(hrm+1), halfW, &real, &imag, off);
//
//        atomicAdd(&(powers[iy*oStride*noHarms + ix*noHarms+hrm].x), (float)(real));
//        atomicAdd(&(powers[iy*oStride*noHarms + ix*noHarms+hrm].y), (float)(imag));
//      }
//    }

    for( int i = 1; i <= noHarms; i++ )
    {
      hrm = i-1;

      FOLD // Determine half width
      {
	if ( hw.val[hrm] )
	  halfW = hw.val[hrm];
	else
	  halfW = cu_z_resp_halfwidth_high<float>(z*i);
      }

      if (off < halfW*2 )
      {
	rz_single_mult_cu<T, float2>(&data[iStride*hrm], loR.val[hrm], iStride, r*i, z*i, halfW, &real, &imag, off);

	atomicAdd(&(powers[iy*oStride*noHarms + ix*noHarms + hrm].x), (float)(real));
	atomicAdd(&(powers[iy*oStride*noHarms + ix*noHarms + hrm].y), (float)(imag));
      }
    }
  }
}
#endif

#ifdef WITH_OPT_PLN3
template<typename T>
__global__ void ffdotPln_ker3(float* powers, float2* fft, int noHarms, int harmWidth, double firstR, double firstZ, double rSZ, double zSZ, int noR, int noZ, int iStride, int oStride, optLocInt_t loR, optLocFloat_t norm, optLocInt_t hw)
{
  const int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int ty = blockIdx.y * blockDim.y + threadIdx.y;

  const int i  = tx / harmWidth ;
  const int ix = tx % harmWidth ;
  const int iy = ty;

  if ( ix < noR && iy < noZ)
  {
    int halfW;
    double r            = firstR + ix/(double)(noR-1) * rSZ ;
    double z            = firstZ - iy/(double)(noZ-1) * zSZ ;
    if (noZ == 1)
      z = 0;

    T hrm_power  = 0;			///< The power of a single point for the harmonic
    T real = 0;
    T imag = 0;

    const int hrm = i+1;
    FOLD // Determine half width
    {
      if ( hw.val[i] )
	halfW	= hw.val[i];
      else
	halfW	= cu_z_resp_halfwidth_high<float>(z*hrm);
    }

    rz_convolution_cu<T, float2>(&fft[iStride*i], loR.val[i], iStride, r*hrm, z*hrm, halfW, &real, &imag);

    hrm_power     = POWERCU(real, imag);

    atomicAdd(&(powers[iy*oStride + ix]), hrm_power);
  }
}
#endif

#ifdef WITH_OPT_PLN4
#ifdef CBL

// This function is under development, for some strange reason synchthreads is not working
template<typename T, int noHarms>
__global__ void ffdotPlnSM_ker(float* powers, float2* data, int halfwidth, double firstR, double firstZ, double rSZ, double zSZ, int noR, int noZ, int iStride, int oStride, int smLen, optLocInt_t loR, optLocInt_t hw)
{
  const int ix = blockIdx.x * blockDim.x + threadIdx.x;
  const int iy = blockIdx.y * blockDim.y + threadIdx.y;

  const int	blkSz	= blockDim.x * blockDim.y;
  const int	tid	= blockDim.x * threadIdx.y + threadIdx.x;
  //const int	bid 	= blockIdx.y * gridDim.x + blockIdx.x;

  extern __shared__ float2 smmm[];

  //__shared__ unsigned int sSum;

  __syncthreads();

  //  if ( tid == 0 )
  //    sSum = 0;

  __syncthreads();

  float2* sm = smmm;

  int halfW;
  double r            = firstR + ix/(double)(noR-1) * rSZ ;
  double z            = firstZ - iy/(double)(noZ-1) * zSZ ;
  if (noZ == 1)
    z = 0;

  T total_power  = 0;
  T real = (T)0;
  T imag = (T)0;

  //T real_O = (T)0;
  //T imag_O = (T)0;

  int 	width;
  long	first;
  long	last;
  int	noStp;
  int	bOff;

  //double bwidth = (blockDim.x) / (double)(noR-1) * rSZ ;

  int buff = 1;

  double fP = firstR + (blockIdx.x * blockDim.x) /(double)(noR-1) * rSZ ;
  double lP = firstR + MIN(noR, ((blockIdx.x+1) * blockDim.x - 1 ) ) /(double)(noR-1) * rSZ ;

  //  if ( ix < noR && iy < noZ)
  //    //if( total_power != 0 )
  //  {
  //    //      if ( blockIdx.y == 0 )
  //    powers[iy*oStride + ix] = 0;
  //    //      else
  //    //	powers[iy*oStride + ix] = 172 ;
  //  }

  //int nno = 0;

  int bIdx = 0;
  for( int i = 1; i <= noHarms; i++ )
  {
    sm = &smmm[bIdx];

    FOLD // Calc vlas
    {
      halfW	= hw.val[i-1];
      //first	= MAX(loR.val[i-1], floor_t( (firstR + blockIdx.x * bwidth )*i ));
      //double fR = (fP)*i;
      //first	= MAX(loR.val[i-1], floor_t(fR) - halfW - buff );
      //first	= MAX(loR.val[i-1], floor_t(fP*i) - halfW - buff );
      first	= floor(fP*i) - halfW - buff ;
      last	= ceil(lP*i)  + halfW + buff ;
      //first	= floor(fR) - halfW ;
      //width	= halfW*2 + ceil_t(bwidth*i) + buff*2 ;
      //width	= halfW*2 + rSZ*i + 5;
      width 	= last - first;
      bOff	= first - loR.val[i-1];
      noStp	= ceilf( width / (float)blkSz );
      //nno	+= width;
      bIdx	+= width;
    }

    FOLD // // Load input into SM  .
    {
      //      if ( width > smLen )
      //      {
      //	printf(" width > smLen  %i > %i   tid %i  \n", width, smLen, tid );
      //      }

      //      if ( ix == 16 && iy == 16 )
      //      {
      //	printf("h: %2i  smLen: %4i  width: %4i  halfW: %4i  bwidth: %8.4f  first: %7i  loR: %7i  bOff: %3i  len: %3i r: %10.4f fr: %10.4f\n", i, smLen, width, halfW, bwidth*i, first, loR.val[i-1], bOff, bOff + width, r*i, fR );
      //      }

      __syncthreads();

      for ( int stp = 0; stp < noStp ; stp++)
      {
	int odd = stp*blkSz + tid;
	if ( odd < width /* && odd < smLen */ )
	{
	  int o2 = bOff + odd;
	  //if ( o2 < iStride )
	  {
	    //	      int tmp = 0;
	    //	    }
	    //	    else
	    //	    {
	    //	    if ( bid == 0 && i == 16 )
	    //	    {
	    //	      printf("tid: %i odd: %i \n",tid,odd);
	    //	    }

	    sm[odd] = data[(i-1)*iStride + o2 ];

	    //atomicInc(&sSum, 1000000 );
	  }
	}
      }

      //	noStp	= ceil_t(iStride / (float)blkSz);
      //	for ( int stp = 0; stp < noStp ; stp++)
      //	{
      //	  int odd = stp*blkSz + tid;
      //
      //	  if ( odd < iStride )
      //	    sm[odd] = fft[(i-1)*iStride + odd ];
      //	}

      //      if ( ix == 20 )
      //      {
      //	printf(" %03i %2i %8li %4i %4i \n", iy, i, first, width, halfW );
      //      }

      __syncthreads(); // Make sure data is written before doing the convolutions

      //      if ( ix < noR && iy < noZ)
      //      {
      //	__syncthreads(); // Make sure data is written before doing the convolutions
      //	rz_convolution_cu<T, float2>(sm, first, width, r*i, z*i, halfW, &real, &imag);
      //	total_power     += POWERCU(real, imag);
      //      }
      //
      //      __syncthreads(); // Make sure data is written before doing the convolutions

    }
  }

  __syncthreads(); // Make sure data is written before doing the convolutions


  if ( ix < noR && iy < noZ)
  {

    bIdx = 0;
    //#pragma unroll
    for( int i = 1; i <= noHarms; i++ )
    {
      sm = &smmm[bIdx];

      FOLD // Calc vlas
      {
	halfW	= hw.val[i-1];
	//first	= MAX(loR.val[i-1], floor_t( (firstR + blockIdx.x * bwidth )*i ));
	//double fR = (fP)*i;
	//first	= MAX(loR.val[i-1], floor_t(fR) - halfW - buff );
	//first	= MAX(loR.val[i-1], floor_t(fP*i) - halfW - buff );
	first	= floor(fP*i) - halfW - buff ;
	last	= ceil(lP*i)  + halfW + buff ;
	//first	= floor(fR) - halfW ;
	//width	= halfW*2 + ceil_t(bwidth*i) + buff*2 ;
	//width	= halfW*2 + rSZ*i + 5;
	width 	= last - first;
	bOff	= first - loR.val[i-1];
	noStp	= ceilf( width / (float)blkSz );
	//nno	+= width;
	bIdx	+= width;
      }

      //    if ( i != 8 )
      //      continue;

      //sm = &smmm[(i-1)*smLen];

      //      __syncthreads(); // Make sure data is written before doing the convolutions
      //
      //      FOLD // Zero
      //      {
      //	noStp	= ceil_t( smLen / (float)blkSz );
      //	float2 zz;
      //	zz.x = 0;
      //	zz.y = 0;
      //
      //	for ( int stp = 0; stp < noStp ; stp++)
      //	{
      //	  int odd = stp*blkSz + tid;
      //	  if ( odd < smLen /* && odd < smLen */ )
      //	  {
      //	    sm[odd] = zz;
      //	  }
      //	}
      //      }
      //
      //__syncthreads(); // Make sure data is written before doing the convolutions
      //__threadfence_block();
      //__threadfence();

      //    if ( ix >= noR || iy >= noZ)
      //      continue;

      //    real = (T)0.0;
      //    imag = (T)0.0;

      //__syncblocks_atomic();

      //    if ( sSum != nno )
      //    {
      //      printf("Bad2 h: %2i  tid: %3i  %5i %5i\n", i, tid, sSum, nno);
      //    }

      //halfW	= cu_z_resp_halfwidth_high<float>(z*i);

      rz_convolution_cu<T, float2>(sm, first, width, r*i, z*i, halfW, &real, &imag);

      //rz_convolution_cu<T, float2>(&fft[iStride*(i-1)], loR.val[i-1], iStride, r*i, z*i, halfW, &real, &imag);
      //rz_convolution_cu<T, float2>(&fft[iStride*(i-1)+bOff], first, width, r*i, z*i, halfW, &real, &imag);

      //      for ( int ic = 0; ic < width; ic++)
      //      {
      //	real += sm[ic].x;
      //	imag += sm[ic].y;
      //      }

      //      rz_convolution_cu<T, float2>(&fft[iStride*(i-1)+bOff], first, width, r*i, z*i, halfW, &real_O, &imag_O);
      //      if ( real != real_O || imag != imag_O )
      //      {
      //	int tmp = 0;
      //      }

      __syncthreads(); // Make sure data is written before doing the convolutions

      total_power     += POWERCU(real, imag);
    }

    //    if ( ix < noR && iy < noZ)
    //    {}
    //    else
    //    {
    //      real = (T)0.0;
    //      imag = (T)0.0;
    //    }

    //__syncthreads(); // Make sure has all been read before writing

    //__syncthreads(); // Make sure has all been read before writing

    //if ( ix < noR && iy < noZ)
    //if( total_power != 0 )
    //{
    //      if ( blockIdx.y == 0 )
    powers[iy*oStride + ix] = total_power;
    //      else
    //	powers[iy*oStride + ix] = 172 ;
  }
}

#endif

#endif

void opt_genResponse(cuRespPln* pln, hipStream_t stream)
{
#ifdef WITH_OPT_BLK2
  infoMSG(5, 5, "Generating optimisation response function values.\n" );

  dim3 dimBlock, dimGrid;

  dimBlock.x = 16;
  dimBlock.y = 16;
  dimBlock.z = 1;

  hipDeviceSynchronize();
  CUDA_SAFE_CALL(hipGetLastError(), "Calling the ffdot_ker kernel.");

  infoMSG(6, 6, "1 Synch.\n" );

  // One block per harmonic, thus we can sort input powers in Shared memory
  dimGrid.x = ceil(pln->noR / (float)dimBlock.x);
  dimGrid.y = ceil(pln->noZ / (float)dimBlock.y);

  opt_genResponse_ker<<<dimGrid, dimBlock, 0, stream >>>(*pln);

  hipDeviceSynchronize();
  CUDA_SAFE_CALL(hipGetLastError(), "Calling the ffdot_ker kernel.");

#else
  fprintf(stderr, "ERROR: Not compiled with response using block optimising kernel.\n");
  exit(EXIT_FAILURE);
#endif
}

int chKpn( cuOptCand* pln, fftInfo* fft )
{
  PROF // Profiling  .
  {
    NV_RANGE_PUSH("Harm INP");
  }

  confSpecsOpt*  conf	= pln->conf;

  double	maxZ	= (pln->centZ + pln->zSize/2.0);
  double	minZ	= (pln->centZ - pln->zSize/2.0);
  double	maxR	= (pln->centR + pln->rSize/2.0);
  double	minR	= (pln->centR - pln->rSize/2.0);

  CUDA_SAFE_CALL(hipGetLastError(), "Entering ffdotPln.");

  pln->halfWidth	= cu_z_resp_halfwidth_high<double>(MAX(fabs(maxZ*pln->noHarms), fabs(minZ*pln->noHarms)) + 4 );
  double rSpread	= ceil((maxR+OPT_INP_BUF)*pln->noHarms  + pln->halfWidth) - floor((minR-OPT_INP_BUF)*pln->noHarms - pln->halfWidth);
  int	inpStride	= getStride(rSpread, sizeof(hipfftComplex), pln->gInf->alignment);

  int	datStart;		// The start index of the input data
  int	datEnd;			// The end   index of the input data
  int	off;			// Offset
  int	newInp		= 0;	// Flag whether new input is needed

  if ( pln->noHarms != pln->input->noHarms )
  {
    newInp = 1;
  }

  // Determine if new input is needed
  for( int h = 0; (h < pln->noHarms) /* && !newInp */ ; h++ )
  {
    datStart        = floor( minR*(h+1) - pln->halfWidth );
    datEnd          = ceil(  maxR*(h+1) + pln->halfWidth );

    if ( datStart > fft->noBins || datEnd <= fft->firstBin )
    {
      if ( h == 0 )
      {
	fprintf(stderr, "ERROR: Trying to optimise a candidate beyond scope of the FFT?");
	return 0;
      }
      pln->noHarms = h; // use previous harmonic
      break;
    }

    if ( datStart < pln->input->loR[h] )
    {
      newInp = 1;
    }
    else if ( pln->input->loR[h] + pln->input->stride < datEnd )
    {
      newInp = 1;
    }
  }

  // Initialise values to 0
  for( int h = 0; h < OPT_MAX_LOC_HARMS; h++)
  {
    pln->hw[h] = 0;
  }

  if ( newInp ) // Calculate normalisation factor  .
  {
    infoMSG(4,4,"New Input\n");

    pln->input->stride = inpStride;
    pln->input->noHarms = pln->noHarms;

    if ( pln->input->stride*pln->noHarms*sizeof(hipfftComplex) > pln->input->size )
    {
      fprintf(stderr, "ERROR: In function %s, cuOptCand not created with large enough input buffer.\n", __FUNCTION__);
      //fprintf(stderr, "maxZ: %.3f  minZ: %f  minR: %.1f maxR: %.1f  rSpread: %.1f  half width: %i  Harms: %i   \n", maxZ, minZ, minR, maxR, rSpread, pln->halfWidth, pln->noHarms );
      exit (EXIT_FAILURE);
    }

    FOLD // Calculate normalisation factor  .
    {
      PROF // Profiling  .
      {
	NV_RANGE_PUSH("Calc Norm factor");
      }

      for ( int i = 1; i <= pln->noHarms; i++ )
      {
	if      ( conf->flags & FLAG_OPT_NRM_LOCAVE   )
	{
	  pln->input->norm[i-1]  = get_localpower3d(fft->data, fft->noBins, (pln->centR-fft->firstBin)*i, pln->centZ*i, 0.0);
	}
	else if ( conf->flags & FLAG_OPT_NRM_MEDIAN1D )
	{
	  pln->input->norm[i-1]  = get_scaleFactorZ(fft->data, fft->noBins, (pln->centR-fft->firstBin)*i, pln->centZ*i, 0.0);
	}
	else if ( conf->flags & FLAG_OPT_NRM_MEDIAN2D )
	{
	  fprintf(stderr,"ERROR: 2D median normalisation has not been written yet.\n");
	  exit(EXIT_FAILURE);
	}
	else
	{
	  // No normalisation this is plausible but not recommended
	  pln->input->norm[i-1] = 1;
	}
	infoMSG(6,6,"Harm %2i Normalisation %6.4f\n", i, pln->input->norm[i-1]);
      }

      PROF // Profiling  .
      {
	NV_RANGE_POP(); // Calc Norm factor
      }
    }
  }

  if ( newInp ) // A blocking synchronisation to make sure we can write to host memory  .
  {
    infoMSG(4,4,"Blocking synchronisation on %s", "inpCmp" );

    CUDA_SAFE_CALL(hipEventSynchronize(pln->inpCmp), "At a blocking synchronisation. This is probably a error in one of the previous asynchronous CUDA calls.");
  }

  // Calculate values for harmonics     and   normalise input and write data to host memory
  for( int h = 0; h < pln->noHarms; h++)
  {
    datStart            = floor( minR*(h+1) - pln->halfWidth );
    datEnd              = ceil(  maxR*(h+1) + pln->halfWidth );

    pln->hw[h]          = cu_z_resp_halfwidth<double>(MAX(fabs(maxZ*(h+1)), fabs(minZ*(h+1))), HIGHACC);

    if ( pln->hw[h] > pln->halfWidth )
    {
      fprintf(stderr, "ERROR: Harmonic half-width is greater than plain maximum.\n");
      pln->hw[h] = pln->halfWidth;
    }

    if ( newInp ) // Normalise input and Write data to host memory  .
    {
      int startV = MIN( ((datStart + datEnd - pln->input->stride ) / 2.0), datStart ); //Start value if the data is centred

      pln->input->loR[h]     = startV;
      double factor   = sqrt(pln->input->norm[h]);		// Correctly normalise input by the sqrt of the local power

      for ( int i = 0; i < pln->input->stride; i++ )		// Normalise input  .
      {
	off = startV - fft->firstBin + i;

	if ( off >= 0 && off < fft->noBins )
	{
	  pln->input->h_inp[h*pln->input->stride + i].r = fft->data[off].r / factor ;
	  pln->input->h_inp[h*pln->input->stride + i].i = fft->data[off].i / factor ;
	}
	else
	{
	  pln->input->h_inp[h*pln->input->stride + i].r = 0;
	  pln->input->h_inp[h*pln->input->stride + i].i = 0;
	}
      }
    }
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP(); // Harm INP
  }

  return newInp;
}

/**
 *
 * Note the configuration flags are used to set the optimiser flags
 *
 * @param pln	optimiser
 * @param fft	FFT data structure
 * @return
 */
int ffdotPln_prep( cuOptCand* pln, fftInfo* fft )
{
  confSpecsOpt*	conf		= pln->conf;
  cuRespPln* 	rpln 		= pln->responsePln;

  pln->blkCnt = 1;
  pln->blkWidth = 0;

  infoMSG(4,4,"Generating ff section, Centred on (%.6f, %.6f)\n", pln->centR, pln->centZ );

  FOLD // Determine optimisation kernels  .
  {
    if ( pln->rSize > 1.5 &&  (conf->flags & FLAG_OPT_BLK) ) // Use the block kernel  .
    {
      /*	NOTE:	Chris Laidler	22/06/2016
       *
       * The per harmonic blocked kernel is fastest in my testing
       */

      remOptFlag(pln, FLAG_OPT_KER_ALL);
      setOptFlag(pln, (conf->flags & FLAG_OPT_BLK) );

      // New method finer granularity
      if ( pln->flags & FLAG_OPT_BLK_EXP )
      {
	if ( !rpln )
	{
	  fprintf(stderr, "ERROR, optimising with NULL response plane, reverting to standard block method.\n");
	  remOptFlag(pln, FLAG_OPT_BLK);

#ifdef 	WITH_OPT_BLK3
	  setOptFlag(pln, FLAG_OPT_BLK_HRM );
#elif	defined(WITH_OPT_BLK1)
	  setOptFlag(pln, FLAG_OPT_BLK_NRM );
#endif
	}
	else
	{
	  // NOTE: I think this pre-calculated resposen value kernel has been removed?

	  pln->blkWidth	= 1;
	  pln->blkDimX	= rpln->noRpnts;
	  pln->blkCnt	= ceil(pln->rSize);

	  pln->noR	= pln->blkDimX * pln->blkCnt;
	  pln->rSize	= (pln->noR-1)/(double)rpln->noRpnts;
	  pln->lftIdx	= round( pln->centR - pln->rSize/2.0 ) ;
	  pln->centR	= pln->lftIdx + pln->rSize/2.0;

	  pln->noZ	= pln->noR;
	  pln->zSize	= (pln->noZ-1)*rpln->dZ;
	  pln->topZidx	= round( (pln->centZ + pln->zSize/2.0 )/rpln->dZ );
	  double top	= pln->topZidx*rpln->dZ;
	  pln->topZidx	= rpln->noZ/2-pln->topZidx;
	  pln->centZ	= top - pln->zSize/2.0;
	}
      }

      if ( pln->flags & ( FLAG_OPT_BLK_NRM | FLAG_OPT_BLK_HRM ) )
      {
	// TODO: Check noR on fermi cards, the increased registers may justify using larger blocks widths
	do
	{
	  pln->blkWidth++;
	  pln->blkDimX	= pln->blkWidth / ( pln->rSize / pln->noR );
	  //blkDimX	= MIN(ceil(blkDimX),32);                      // The max of 32 is not strictly necessary
	  pln->blkCnt	= ceil(pln->rSize / (float)pln->blkWidth );
	}
	while ( pln->blkCnt > 10 ); // TODO: Make this 10 a hash define

	pln->rSize	= pln->blkCnt*pln->blkWidth - pln->blkWidth/(double)pln->blkDimX;
	pln->noR	= pln->blkDimX * pln->blkCnt;
      }
    }
    else
    {
      /*	NOTE:	Chris Laidler	22/06/2016
       *
       * I found 16 testing on a 750ti, running in synchronous mode.
       * This could probably be tested on more cards but I expect similar results
       * This relates to a optPlnDim of 16, I found anything less than 20 shows
       * significant speed up using the finer granularity kernel.
       */

      remOptFlag(pln, FLAG_OPT_KER_ALL);
      setOptFlag(pln, (conf->flags & FLAG_OPT_PTS) );

      if ( !(pln->flags&FLAG_OPT_PTS) )
      {
#ifdef 	WITH_OPT_PLN3
	setOptFlag(pln, FLAG_OPT_PTS_HRM );
#elif	defined(WITH_OPT_PLN1)
	setOptFlag(pln, FLAG_OPT_PTS_NRM );
#elif	defined(WITH_OPT_PLN2)
	setOptFlag(pln, FLAG_OPT_PTS_EXP );
#endif
      }
    }

    // All kernels use the same output stride
    pln->outStride    = pln->noR;
  }
}

int ffdotPln_input( cuOptCand* pln, fftInfo* fft )
{
  // Check input
  int newInp = chKpn( pln, fft );

  if ( newInp ) // Copy input data to the device  .
  {
    infoMSG(4,4,"1D async memory copy H2D");

    CUDA_SAFE_CALL(hipMemcpyAsync(pln->input->d_inp, pln->input->h_inp, pln->input->stride*pln->noHarms*sizeof(fcomplexcu), hipMemcpyHostToDevice, pln->stream), "Copying optimisation input to the device");
    CUDA_SAFE_CALL(hipEventRecord(pln->inpCmp, pln->stream),"Recording event: inpCmp");
  }

  return newInp;
}

template<typename T>
int ffdotPln_ker( cuOptCand* pln, fftInfo* fft)
{
  confSpecsOpt*	conf		= pln->conf;
  cuRespPln* 	rpln 		= pln->responsePln;

  int		maxHW 		= 0;	// The maximum possible halfwidth of the elements being tested

  optLocInt_t	rOff;			// Row offset
  optLocInt_t	hw;			// The halfwidth for each harmonic
  optLocFloat_t	norm;			// Normalisation factor for each harmonic
  //size_t	pln->resSz		= 0;	//

  infoMSG(5,5,"Height: %5.4f z - Width: %5.4f \n", pln->zSize, pln->rSize );

  // Calculate bounds on potently newly scaled plane
  double maxZ		= (pln->centZ + pln->zSize/2.0);
  double minR		= (pln->centR - pln->rSize/2.0);

  // Initialise values to 0
  for( int h = 0; h < OPT_MAX_LOC_HARMS; h++)
  {
    rOff.val[h]		= pln->input->loR[h];
    hw.val[h]		= pln->hw[h];
    norm.val[h]		= sqrt(pln->input->norm[h]);             // Correctly normalised by the sqrt of the local power

    MAXX(maxHW, hw.val[h]);
  }

  // Halfwidth stuff
  if ( (conf->flags & FLAG_OPT_DYN_HW) || (pln->zSize >= 2) )
  {
    infoMSG(5,5,"Using dynamic half Width");
    for( int h = 0; h < pln->noHarms; h++)
    {
      hw.val[h] = 0;
    }
    maxHW = pln->halfWidth;
  }
  else
  {
    infoMSG(5,5,"Using constant half Width of %i", maxHW);
  }

  FOLD // Check output size  .
  {
    // One float per point
    pln->resSz = pln->outStride*pln->noZ*sizeof(float) ;
    if ( pln->flags & ( FLAG_OPT_BLK_EXP | FLAG_OPT_PTS_EXP ) )
      pln->resSz = pln->outStride*pln->noZ*pln->noHarms*sizeof(fcomplex);	// One point per harmonic

    if ( pln->resSz > pln->outSz )
    {
      fprintf(stderr, "ERROR: Optimisation plane larger than allocated memory.\n");
      exit(EXIT_FAILURE);
    }
  }

  FOLD // Call kernel  .
  {
    dim3 dimBlock, dimGrid;

    if ( conf->flags & FLAG_SYNCH )
      CUDA_SAFE_CALL(hipEventRecord(pln->compInit, pln->stream),"Recording event: compInit");

    if ( pln->flags &  FLAG_OPT_BLK )			// Use block kernel
    {
      infoMSG(4,4,"Block kernel [ No threads %i  Width %i no Blocks %i]", (int)pln->blkDimX, pln->blkWidth, pln->blkCnt);

      if      ( pln->flags &  FLAG_OPT_BLK_NRM )		// Use block kernel
      {
#ifdef WITH_OPT_BLK1

	infoMSG(5,5,"Block kernel 1 - Standard");

	// Thread blocks
	dimBlock.x = pln->blkDimX;
	dimBlock.y = 16;
	dimBlock.z = 1;

	// One block per harmonic, thus we can sort input powers in Shared memory
	dimGrid.x = 1;
	dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

	CUDA_SAFE_CALL(hipEventRecord(pln->tInit1, pln->stream),"Recording event: tInit1");

	// Call the kernel to normalise and spread the input data
	switch (pln->blkCnt)
	{
	  case 2:
	    ffdotPlnByBlk_ker<T,2> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 3:
	    ffdotPlnByBlk_ker<T,3> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 4:
	    ffdotPlnByBlk_ker<T,4> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 5:
	    ffdotPlnByBlk_ker<T,5> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 6:
	    ffdotPlnByBlk_ker<T,6> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 7:
	    ffdotPlnByBlk_ker<T,7> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 8:
	    ffdotPlnByBlk_ker<T,8> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 9:
	    ffdotPlnByBlk_ker<T,9> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 10:
	    ffdotPlnByBlk_ker<T,10><<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  default:
	  {
	    fprintf(stderr, "ERROR: %s has not been templated for %i blocks.\n", __FUNCTION__, pln->blkCnt );
	    exit(EXIT_FAILURE);
	  }
	}

	CUDA_SAFE_CALL(hipEventRecord(pln->tComp1, pln->stream),"Recording event: tComp1");
#else
	fprintf(stderr, "ERROR: Not compiled with WITH_OPT_BLK1.\n");
	exit(EXIT_FAILURE);
#endif
      }
      else if ( pln->flags &  FLAG_OPT_BLK_EXP )
      {
#ifdef WITH_OPT_BLK2

	infoMSG(5,5,"Block kernel 2 - Expanded");

	dimBlock.x = 16;
	dimBlock.y = 16;

	CUDA_SAFE_CALL(hipEventRecord(pln->tInit2, pln->stream),"Recording event: tInit2");

	hipMemsetAsync ( pln->d_out, 0, pln->resSz, pln->stream );
	CUDA_SAFE_CALL(hipGetLastError(), "Zeroing the output memory");

	maxHW *=2 ;

	// One block per harmonic, thus we can sort input powers in Shared memory
	dimGrid.x = ceil(maxHW*rpln->noRpnts/(float)dimBlock.x);
	dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

	// Call the kernel to normalise and spread the input data
	switch (pln->blkCnt)
	{
	  case 2:
	    ffdotPlnByBlk_ker2<T, 2> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 3:
	    ffdotPlnByBlk_ker2<T, 3> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 4:
	    ffdotPlnByBlk_ker2<T, 4> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 5:
	    ffdotPlnByBlk_ker2<T, 5> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 6:
	    ffdotPlnByBlk_ker2<T, 6> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 7:
	    ffdotPlnByBlk_ker2<T, 7> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 8:
	    ffdotPlnByBlk_ker2<T, 8> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 9:
	    ffdotPlnByBlk_ker2<T, 9> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 10:
	    ffdotPlnByBlk_ker2<T,10> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 11:
	    ffdotPlnByBlk_ker2<T,11> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 12:
	    ffdotPlnByBlk_ker2<T,12> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 13:
	    ffdotPlnByBlk_ker2<T,13> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 14:
	    ffdotPlnByBlk_ker2<T,14> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 15:
	    ffdotPlnByBlk_ker2<T,15> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 16:
	    ffdotPlnByBlk_ker2<T,16> <<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, *rpln, pln->noHarms, pln->halfWidth, pln->topZidx, pln->lftIdx, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  default:
	  {
	    fprintf(stderr, "ERROR: %s has not been templated for %i blocks.\n", __FUNCTION__, pln->blkCnt );
	    exit(EXIT_FAILURE);
	  }
	}

	CUDA_SAFE_CALL(hipEventRecord(pln->tComp2, pln->stream),"Recording event: tComp1");
#else
	fprintf(stderr, "ERROR: Not compiled with WITH_OPT_BLK2.\n");
	exit(EXIT_FAILURE);
#endif
      }
      else if ( pln->flags & FLAG_OPT_BLK_HRM )
      {
#ifdef WITH_OPT_BLK3
	infoMSG(5,5,"Block kernel 3 - Harms");

	dimBlock.x = 16;
	dimBlock.y = 16;

	CUDA_SAFE_CALL(hipEventRecord(pln->tInit2, pln->stream),"Recording event: tInit2");

	int noX = ceil(pln->blkDimX / (float)dimBlock.x);
	int harmWidth = noX*dimBlock.x;

	hipMemsetAsync ( pln->d_out, 0, pln->resSz, pln->stream );
	CUDA_SAFE_CALL(hipGetLastError(), "Zeroing the output memory");

	// One block per harmonic, thus we can sort input powers in shared memory
	dimGrid.x = noX * pln->noHarms ;
	dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

	// Call the kernel to normalise and spread the input data
	switch (pln->blkCnt)
	{
	  case 2:
	    ffdotPlnByBlk_ker3<T, 2> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 3:
	    ffdotPlnByBlk_ker3<T, 3> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 4:
	    ffdotPlnByBlk_ker3<T, 4> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 5:
	    ffdotPlnByBlk_ker3<T, 5> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 6:
	    ffdotPlnByBlk_ker3<T, 6> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 7:
	    ffdotPlnByBlk_ker3<T, 7> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 8:
	    ffdotPlnByBlk_ker3<T, 8> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 9:
	    ffdotPlnByBlk_ker3<T, 9> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  case 10:
	    ffdotPlnByBlk_ker3<T,10> <<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->zSize, pln->blkDimX, pln->noZ, pln->blkWidth, pln->input->stride, pln->outStride, rOff, norm, hw);
	    break;
	  default:
	  {
	    fprintf(stderr, "ERROR: %s has not been templated for %i blocks.\n", __FUNCTION__, pln->blkCnt );
	    exit(EXIT_FAILURE);
	  }
	}

	CUDA_SAFE_CALL(hipEventRecord(pln->tComp2, pln->stream),"Recording event: tComp1");
#else
	fprintf(stderr, "ERROR: Not compiled with WITH_OPT_BLK3.\n");
	exit(EXIT_FAILURE);
#endif
      }
      else
      {
	fprintf(stderr, "ERROR: No block optimisation specified.\n");
	exit(EXIT_FAILURE);
      }
    }
    else                  // Use normal kernel
    {
      infoMSG(4,4,"Grid kernel");

      dimBlock.x = 16;
      dimBlock.y = 16;
      dimBlock.z = 1;

      maxHW = ceil(maxHW*2/(float)dimBlock.x)*dimBlock.x;

      if      ( pln->flags &  FLAG_OPT_PTS_SHR ) // Shared mem  .
      {
#ifdef WITH_OPT_PLN4
#ifdef CBL
	float smSz = 0 ;

	for( int h = 0; h < pln->noHarms; h++)
	{
	  smSz += ceil(hw.val[h]*2 + pln->rSize*(h+1) + 4 );
	}

	if ( smSz < 6144*0.9 ) // ~% of SM	10: 4915
	{

	  infoMSG(5,5,"Flat kernel 5 - SM \n");

	  // One block per harmonic, thus we can sort input powers in Shared memory
	  dimGrid.x = ceil(pln->noR/(float)dimBlock.x);
	  dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

	  //int noTB = dimGrid.x * dimGrid.y ;

	  // Call the kernel to normalise and spread the input data
	  switch (pln->noHarms)
	  {
	    case 1:
	      ffdotPlnSM_ker<T,1 ><<<dimGrid, dimBlock, smSz*sizeof(float2), pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->input->stride, pln->outStride, smSz, rOff, hw);
	      break;
	    case 2:
	      ffdotPlnSM_ker<T,2 ><<<dimGrid, dimBlock, smSz*sizeof(float2), pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->input->stride, pln->outStride, smSz, rOff, hw);
	      break;
	    case 4:
	      ffdotPlnSM_ker<T,4 ><<<dimGrid, dimBlock, smSz*sizeof(float2), pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->input->stride, pln->outStride, smSz, rOff, hw);
	      break;
	    case 8:
	      ffdotPlnSM_ker<T,8 ><<<dimGrid, dimBlock, smSz*sizeof(float2), pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->input->stride, pln->outStride, smSz, rOff, hw);
	      break;
	    case 16:
	      ffdotPlnSM_ker<T,16><<<dimGrid, dimBlock, smSz*sizeof(float2), pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->input->stride, pln->outStride, smSz, rOff, hw);
	      break;
	  }
	  //ffdotPlnSM_ker<T><<<dimGrid, dimBlock, smSz*sizeof(float2)*pln->noHarms*1.2, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->input->stride, pln->outStride, smSz, rOff, hw);
	}
#endif
#else
	fprintf(stderr, "ERROR: Not compiled with WITH_OPT_PLN4.\n");
	exit(EXIT_FAILURE);
#endif
      }
      else if ( pln->flags &  FLAG_OPT_PTS_NRM ) // Thread point  .
      {
#ifdef WITH_OPT_PLN1
	infoMSG(5,5,"Flat kernel 1 - Standard\n");

	CUDA_SAFE_CALL(hipEventRecord(pln->tInit1, pln->stream),"Recording event: tInit1");

	// One block per harmonic, thus we can sort input powers in Shared memory
	dimGrid.x = ceil(pln->noR/(float)dimBlock.x);
	dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

	// Call the kernel create a section of the f-fdot plane
	ffdotPln_ker<T><<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, pln->halfWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->input->stride, pln->outStride, rOff, norm, hw);

	CUDA_SAFE_CALL(hipEventRecord(pln->tComp1, pln->stream),"Recording event: tComp1");
#else
	fprintf(stderr, "ERROR: Not compiled with WITH_OPT_PLN1.\n");
	exit(EXIT_FAILURE);
#endif
      }
      else if ( pln->flags &  FLAG_OPT_PTS_EXP ) // Thread response pos  .
      {
#ifdef WITH_OPT_PLN2
	infoMSG(5,5,"Flat kernel 2 - Expanded\n");

	CUDA_SAFE_CALL(hipEventRecord(pln->tInit2, pln->stream),"Recording event: tInit2");

	hipMemsetAsync ( pln->d_out, 0, pln->resSz, pln->stream );
	//hipMemset ( pln->d_out, 0, pln->resSz );
	CUDA_SAFE_CALL(hipGetLastError(), "Zeroing the output memory");

	// One block per harmonic, thus we can sort input powers in Shared memory
	//dimGrid.x = ceil(maxHW*pln->noHarms/(float)dimBlock.x);
	dimGrid.x = ceil(maxHW/(float)dimBlock.x);
	dimGrid.y = ceil(pln->noZ*pln->noR/(float)dimBlock.y);

	// Call the kernel create a section of the f-fdot plane
	ffdotPln_ker2<T><<<dimGrid, dimBlock, 0, pln->stream >>>((float2*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, maxHW, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->input->stride, pln->outStride, rOff, norm, hw);

	CUDA_SAFE_CALL(hipEventRecord(pln->tComp2, pln->stream),"Recording event: tComp2");
#else
	fprintf(stderr, "ERROR: Not compiled with WITH_OPT_PLN2.\n");
	exit(EXIT_FAILURE);
#endif
      }
      else if ( pln->flags &  FLAG_OPT_PTS_HRM ) // Thread point of harmonic  .
      {
#ifdef WITH_OPT_PLN3
	infoMSG(5,5,"Flat kernel 3 - Harmonics\n");

	int noX = ceil(pln->noR / (float)dimBlock.x);
	int harmWidth = noX*dimBlock.x;

	CUDA_SAFE_CALL(hipEventRecord(pln->tInit3, pln->stream),"Recording event: tInit3");

	hipMemsetAsync ( pln->d_out, 0, pln->resSz, pln->stream );
	CUDA_SAFE_CALL(hipGetLastError(), "Zeroing the output memory");

	// One block per harmonic, thus we can sort input powers in Shared memory
	//dimGrid.x = ceil(maxHW*pln->noHarms/(float)dimBlock.x);
	dimGrid.x = noX * pln->noHarms ;
	dimGrid.y = ceil(pln->noZ/(float)dimBlock.y);

	// Call the kernel create a section of the f-fdot plane
	ffdotPln_ker3<T><<<dimGrid, dimBlock, 0, pln->stream >>>((float*)pln->d_out, (float2*)pln->input->d_inp, pln->noHarms, harmWidth, minR, maxZ, pln->rSize, pln->zSize, pln->noR, pln->noZ, pln->input->stride, pln->outStride, rOff, norm, hw);

	CUDA_SAFE_CALL(hipEventRecord(pln->tComp3, pln->stream),"Recording event: tComp3");
#else
	fprintf(stderr, "ERROR: Not compiled with WITH_OPT_PLN3.\n");
	exit(EXIT_FAILURE);
#endif
      }
      else
      {
	fprintf(stderr, "ERROR: No optimisation plane kernel specified.\n");
	exit(EXIT_FAILURE);
      }
    }

    CUDA_SAFE_CALL(hipGetLastError(), "Calling the ffdot_ker kernel.");

    if ( conf->flags & FLAG_SYNCH )
      CUDA_SAFE_CALL(hipEventRecord(pln->compCmp, pln->stream), "Recording event: compCmp");
  }

}

int ffdotPln_get( cuOptCand* pln, fftInfo* fft )
{
  confSpecsOpt*	conf		= pln->conf;
  cuRespPln* 	rpln 		= pln->responsePln;

  FOLD // Copy data back to host  .
  {
    infoMSG(4,4,"1D async memory copy D2H");

    CUDA_SAFE_CALL(hipMemcpyAsync(pln->h_out, pln->d_out, pln->resSz, hipMemcpyDeviceToHost, pln->stream), "Copying optimisation results back from the device.");
    CUDA_SAFE_CALL(hipEventRecord(pln->outCmp, pln->stream),"Recording event: outCmp");
  }
}

int ffdotPln_process( cuOptCand* pln, fftInfo* fft )
{
  confSpecsOpt*	conf		= pln->conf;
  cuRespPln* 	rpln 		= pln->responsePln;

  FOLD // Wait  .
  {
    FOLD // A blocking synchronisation to ensure results are ready to be proceeded by the host  .
    {
      infoMSG(4,4,"Blocking synchronisation on %s", "outCmp" );

      PROF // Profiling  .
      {
	NV_RANGE_PUSH("EventSynch");
      }

      CUDA_SAFE_CALL(hipEventSynchronize(pln->outCmp), "At a blocking synchronisation. This is probably a error in one of the previous asynchronous CUDA calls.");

      PROF // Profiling  .
      {
	NV_RANGE_POP(); // EventSynch
      }
    }
  }

  FOLD // Calc Powers  .
  {
    if ( pln->flags & ( FLAG_OPT_BLK_EXP | FLAG_OPT_PTS_EXP ) )
    {
      PROF // Profiling  .
      {
	NV_RANGE_PUSH("Calc Powers");
      }

      int noHarms = pln->noHarms;

      // Complex harmonic output
      for (int indy = 0; indy < pln->noZ; indy++ )
      {
	for (int indx = 0; indx < pln->noR ; indx++ )
	{
	  float yy2 = 0;
	  for (int i = 0; i < pln->noHarms ; i++ )
	  {
	    float2 p1 = ((float2*)pln->h_out)[ indy*pln->outStride*noHarms + indx*noHarms + i ];
	    yy2 += POWERF(p1);
	  }
	}
      }

      PROF // Profiling  .
      {
	NV_RANGE_POP(); // Calc Powers
      }
    }
  }
}

template<typename T>
int ffdotPln( cuOptCand* pln, fftInfo* fft )
{
  ffdotPln_prep( pln,  fft );

  int newInp = ffdotPln_input( pln, fft );

  ffdotPln_ker<T>( pln, fft );

  ffdotPln_get( pln, fft );

  ffdotPln_process( pln, fft );

  return newInp;
}

void optemiseTree(candTree* tree, cuOptCand* oPlnPln)
{
  container* cont = tree->getLargest();

  while (cont)
  {
    cont = cont->smaller;
  }
}

int addPlnToTree(candTree* tree, cuOptCand* pln)
{
  PROF // Profiling  .
  {
    NV_RANGE_PUSH("addPlnToTree");
  }

  FOLD // Get new max  .
  {
    int ggr = 0;

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      for (int indx = 0; indx < pln->noR ; indx++ )
      {
	float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
	{
	  initCand* canidate = new initCand;

	  canidate->numharm = pln->noHarms;
	  canidate->power   = yy2;
	  canidate->r       = pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
	  canidate->z       = pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;
	  canidate->sig     = yy2;
	  if ( pln->noZ == 1 )
	    canidate->z = 0;

	  ggr++;

	  tree->insert(canidate, 0.2 );
	}
      }
    }
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP(); // addPlnToTree
  }

  return 0;
}

candTree* opt_cont(candTree* oTree, cuOptCand* pln, container* cont, fftInfo* fft, int nn)
{
  //  PROF // Profiling  .
  //  {
  //    NV_RANGE_PUSH("opt_cont");
  //  }
  //
  //  confSpecsGen*  sSpec   = pln->cuSrch->sSpec;
  //  initCand* iCand 	= (initCand*)cont->data;

  //
  //  optInitCandLocPlns(iCand, pln, nn );
  //
  //  accelcand* cand = new accelcand;
  //  memset(cand, 0, sizeof(accelcand));
  //
  //  int lrep      = 0;
  //  int noP       = 30;
  //  float snoop   = 0.3;
  //  float sz;
  //  float v1, v2;
  //
  //  const int mxRep = 10;
  //
  //  initCand* canidate = (initCand*)cont->data;
  //
  //  candTree* thisOpt = new candTree;
  //
  //  if ( canidate->numharm == 1  )
  //    sz = conf->optPlnSiz[0];
  //  if ( canidate->numharm == 2  )
  //    sz = conf->optPlnSiz[1];
  //  if ( canidate->numharm == 4  )
  //    sz = conf->optPlnSiz[2];
  //  if ( canidate->numharm == 8  )
  //    sz = conf->optPlnSiz[3];
  //  if ( canidate->numharm == 16 )
  //    sz = conf->optPlnSiz[4];
  //
  //  //int numindep        = (obs->rhi - obs->rlo ) * (obs->zhi +1 ) * (ACCEL_DZ / 6.95) / pln->noHarms ;
  //
  //  //printf("\n%03i  r: %15.6f   z: %12.6f \n", nn, cand->r, cand->z);
  //
  //  pln->halfWidth = 0;
  //
  //  int plt = 0;
  //
  //  if ( optpln01 > 0 )
  //  {
  //    noP               = optpln01 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale;
  //  }
  //
  //  if ( optpln02 > 0 )
  //  {
  //    noP               = optpln02 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale;
  //  }
  //
  //  if ( optpln03 > 0 )
  //  {
  //    noP               = optpln03 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale*2;
  //  }
  //
  //  if ( optpln04 > 0 )
  //  {
  //    noP               = optpln04 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale*2;
  //  }
  //
  //  if ( optpln05 > 0 )
  //  {
  //    noP               = optpln05 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<float>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale*2;
  //  }
  //
  //  if ( optpln06 > 0 )
  //  {
  //    noP               = optpln06 ;
  //    lrep              = 0;
  //    canidate->power   = 0;     // Set initial power to zero
  //    do
  //    {
  //      generatePln<double>(canidate, fft, pln, noP, sz, plt++, nn );
  //
  //      container* optC =  oTree->getLargest(canidate, 1);
  //
  //      if ( optC )
  //      {
  //        // This has feature has already been optimised!
  //        cont->flag |= REMOVE_CONTAINER;
  //        NV_RANGE_POP();
  //        return thisOpt;
  //      }
  //
  //      //addPlnToTree(thisOpt, pln);
  //
  //      v1 = fabs(( pln->centR - canidate->r )/(pln->rSize/2.0));
  //      v2 = fabs(( pln->centZ - canidate->z )/(pln->zSize/2.0));
  //
  //      if ( ++lrep > mxRep )
  //      {
  //        break;
  //      }
  //    }
  //    while ( v1 > snoop || v2 > snoop );
  //    sz /= downScale*2;
  //  }
  //
  //  cont->flag |= OPTIMISED_CONTAINER;
  //
  //  NV_RANGE_POP();
  //  return thisOpt;
  return NULL;
}

template<typename T>
int optInitCandPosPln(initCand* cand, cuOptCand* pln, int noP, double scale, int plt = -1, int nn = 0, int lv = 0 )
{
  int newInput = 0;

  fftInfo*	fft	= pln->cuSrch->fft;
  confSpecsOpt*	conf	= pln->conf;

  FOLD // Generate plain points  .
  {
    pln->noZ		= noP;
    pln->noR		= noP;
    pln->rSize		= scale;
    pln->zSize		= scale*conf->zScale;
    double rRes		= pln->rSize / (double)(noP-1);
    double zRes		= pln->zSize / (double)(noP-1);

    if ( noP % 2 )
    {
      // Odd
      pln->centR	= cand->r;
      pln->centZ	= cand->z;
    }
    else
    {
      // Even
      pln->centR	= cand->r + rRes/2.0;
      pln->centZ	= cand->z - zRes/2.0;
    }

    if ( ffdotPln<T>(pln, fft) ) // Create the section of ff plane  .
    {
      // New input was used so don't maintain the old max
      cand->power	= 0;
      newInput		= 1;
    }
  }

  FOLD // Get new max  .
  {
    PROF // Profiling  .
    {
      NV_RANGE_PUSH("Get Max");
    }

    for (int indy = 0; indy < pln->noZ; indy++ )
    {
      for (int indx = 0; indx < pln->noR ; indx++ )
      {
	float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
	if ( yy2 > cand->power )
	{
	  cand->power	= yy2;
	  cand->r	= pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
	  cand->z	= pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;
	  if ( pln->noZ	== 1 )
	    cand->z = 0;
	}
      }
    }

    infoMSG(4,4,"Max Power %8.3f at (%.6f %.6f)\n", cand->power, cand->r, cand->z);

    PROF // Profiling  .
    {
      NV_RANGE_POP(); // Get Max
    }
  }

  FOLD // Write CVS & plot output  .
  {
#ifdef CBL
    if ( conf->flags & FLAG_DPG_PLT_OPT ) // Write CVS & plot output  .
    {
      infoMSG(4,4,"Write CVS\n");

      char tName[1024];
      sprintf(tName,"/home/chris/accel/Cand_%05i_Rep_%02i_Lv_%i_h%02i.csv", nn, plt, lv, cand->numharm );
      FILE *f2 = fopen(tName, "w");

      FOLD // Write CSV
      {

	PROF // Profiling  .
	{
	  NV_RANGE_PUSH("Write CVS");
	}

	// Add number of hmonics summed as the first line
	fprintf(f2,"%i",pln->noHarms);

	// Print R values
	for (int indx = 0; indx < pln->noR ; indx++ )
	{
	  double r = pln->centR - pln->rSize/2.0 + indx/(double)(pln->noR-1) * (pln->rSize) ;
	  fprintf(f2,"\t%.6f",r);
	}
	fprintf(f2,"\n");

	for (int indy = 0; indy < pln->noZ; indy++ )
	{
	  // Print Z value
	  double z = pln->centZ + pln->zSize/2.0 - indy/(double)(pln->noZ-1) * (pln->zSize) ;
	  if ( pln->noZ == 1 )
	    z = 0;
	  fprintf(f2,"%.15f",z);
	  
	  // Print power
	  for (int indx = 0; indx < pln->noR ; indx++ )
	  {
	    float yy2 = ((float*)pln->h_out)[indy*pln->outStride+indx];
	    fprintf(f2,"\t%.20f",yy2);
	  }
	  fprintf(f2,"\n");
	}
	fclose(f2);

	PROF // Profiling  .
	{
	  NV_RANGE_POP(); // Write CVS
	}
      }

      FOLD // Make image  .
      {
	infoMSG(4,4,"Image\n");

	PROF // Profiling  .
	{
	  NV_RANGE_PUSH("Image");
	}

	char cmd[1024];
	sprintf(cmd,"python ~/bin/bin/plt_ffd.py %s > /dev/null 2>&1", tName);
	system(cmd);

	PROF // Profiling  .
	{
	  NV_RANGE_POP(); // Image
	}
      }
    }
#endif
  }

  return newInput;
}

template<typename T>
T pow(initCand* cand, cuHarmInput* inp)
{
  int halfW;
  double r            = cand->r;
  double z            = cand->z;

  T total_power  = 0;
  T real = 0;
  T imag = 0;

  for( int i = 1; i <= cand->numharm; i++ )
  {
    // Determine half width - high precision
    halfW = cu_z_resp_halfwidth_high<float>(z*i);

    rz_convolution_cu<T, float2>(&((float2*)inp->h_inp)[(i-1)*inp->stride], inp->loR[i-1], inp->stride, r*i, z*i, halfW, &real, &imag);

    total_power     += POWERCU(real, imag);
  }

  cand->power =  total_power;

  return total_power;
}

int prepInput(initCand* cand, cuOptCand* pln, double sz)
{
  fftInfo*      fft     = pln->cuSrch->fft;

  FOLD // Large points  .
  {
    pln->noHarms	= cand->numharm;
    pln->centR          = cand->r;
    pln->centZ          = cand->z;
    pln->rSize          = sz;
    pln->zSize          = sz*pln->conf->zScale;
  }

  // Check the input
  int newInp = chKpn( pln, fft );

  return newInp;
}

// Simplex method
template<typename T>
int optInitCandPosSim(initCand* cand, cuHarmInput* inp, double rSize = 1.0, double zSize = 1.0, int plt = 0, int nn = 0, int lv = 0 )
{
  infoMSG(3,3,"Simplex refine position - lvl %i  size %f by %f \n", lv+1, rSize, zSize);

  // These are the Nelder–Mead parameter values
  double reflect	= 1.0;
  double expand		= 2.0;
  double contract	= 0.4;
  double shrink		= 0.3;

  initCand  cnds[3];
  initCand* olst[3];

  initCand  centroid    = *cand;
  initCand  reflection  = *cand;
  initCand  expansion   = *cand;
  initCand  contraction = *cand;

  cnds[0] = *cand;
  cnds[1] = *cand;
  cnds[2] = *cand;

  pow<T>(&cnds[0], inp);
  double inpPow = cnds[0].power;

  cnds[1].r += rSize;
  pow<T>(&cnds[1], inp);

  cnds[2].z += zSize;
  pow<T>(&cnds[2], inp);

  olst[NM_BEST] = &cnds[0];
  olst[NM_MIDL] = &cnds[1];
  olst[NM_WRST] = &cnds[2];

  int ite = 0;
  double rtol;			///< Ratio of low to high

  infoMSG(4,4,"Start - Power: %8.3f at (%.6f %.6f)", cnds[0].power, cnds[0].r, cnds[0].z);

  while (1)
  {
    FOLD // Order
    {
      if (olst[NM_WRST]->power > olst[NM_MIDL]->power )
	SWAP_PTR(olst[NM_WRST], olst[NM_MIDL]);

      if (olst[NM_MIDL]->power > olst[NM_BEST]->power )
      {
	SWAP_PTR(olst[NM_MIDL], olst[NM_BEST]);

	if (olst[NM_WRST]->power > olst[NM_MIDL]->power )
	  SWAP_PTR(olst[NM_WRST], olst[NM_MIDL]);
      }
    }

    FOLD // Centroid  .
    {
      centroid.r = ( olst[NM_BEST]->r + olst[NM_MIDL]->r ) / 2.0  ;
      centroid.z = ( olst[NM_BEST]->z + olst[NM_MIDL]->z ) / 2.0  ;
      //pow<T>(&centroid, inp);
    }

    ite++;

    rtol = 2.0 * fabs(olst[NM_BEST]->power - olst[NM_WRST]->power) / (fabs(olst[NM_BEST]->power) + fabs(olst[NM_MIDL]->power) + 1.0e-15) ;

    if (rtol < 1.0e-7 )  // Within error so leave  .
    {
      break;
    }

    if ( ite == 100 )
    {
      break;
    }

    FOLD // Reflection  .
    {
      reflection.r = centroid.r + reflect*(centroid.r - olst[NM_WRST]->r ) ;
      reflection.z = centroid.z + reflect*(centroid.z - olst[NM_WRST]->z ) ;
      pow<T>(&reflection, inp);

      if ( olst[NM_BEST]->power <= reflection.power && reflection.power < olst[NM_MIDL]->power )
      {
	*olst[NM_WRST] = reflection;
	continue;
      }
    }

    FOLD // Expansion  .
    {
      if ( reflection.power > olst[NM_BEST]->power )
      {
	expansion.r = centroid.r + expand*(reflection.r - centroid.r ) ;
	expansion.z = centroid.z + expand*(reflection.z - centroid.z ) ;
	pow<T>(&expansion, inp);

	if (expansion.power > reflection.power)
	{
	  *olst[NM_WRST] = expansion;
	}
	else
	{
	  *olst[NM_WRST] = reflection;
	}
	continue;
      }
    }

    FOLD // Contraction  .
    {
      contraction.r = centroid.r + contract*(olst[NM_WRST]->r - centroid.r) ;
      contraction.z = centroid.z + contract*(olst[NM_WRST]->z - centroid.z) ;
      pow<T>(&contraction, inp);

      if ( contraction.power > olst[NM_WRST]->power )
      {
	*olst[NM_WRST] = contraction;
	continue;
      }
    }

    FOLD // Shrink  .
    {
      olst[NM_MIDL]->r = olst[NM_BEST]->r + shrink*(olst[NM_MIDL]->r - olst[NM_BEST]->r);
      olst[NM_MIDL]->z = olst[NM_BEST]->z + shrink*(olst[NM_MIDL]->z - olst[NM_BEST]->z);
      pow<T>(olst[NM_MIDL], inp);

      olst[NM_WRST]->r = olst[NM_BEST]->r + shrink*(olst[NM_WRST]->r - olst[NM_BEST]->r);
      olst[NM_WRST]->z = olst[NM_BEST]->z + shrink*(olst[NM_WRST]->z - olst[NM_BEST]->z);
      pow<T>(olst[NM_WRST], inp);
    }
  }

  double dist = sqrt( (cand->r-olst[NM_BEST]->r)*(cand->r-olst[NM_BEST]->r) + (cand->z-olst[NM_BEST]->z)*(cand->z-olst[NM_BEST]->z) );
  double powInc  = olst[NM_BEST]->power - inpPow;

  cand->r = olst[NM_BEST]->r;
  cand->z = olst[NM_BEST]->z;
  cand->power = olst[NM_BEST]->power;

  infoMSG(4,4,"End   - Power: %8.3f at (%.6f %.6f) %3i iterations moved %9.7f  power inc: %9.7f", cand->power, cand->r, cand->z, ite, dist, powInc);

  return 1;
}

cuHarmInput* duplicateHost(cuHarmInput* orr)
{
  if ( orr )
  {
    size_t sz = MIN(orr->size, orr->noHarms * orr->stride * sizeof(fcomplexcu) * 1.1);

    PROF // Profiling  .
    {
      NV_RANGE_PUSH("Opt derivs");
    }

    cuHarmInput* res = (cuHarmInput*)malloc(sizeof(cuHarmInput));

    memcpy(res, orr, sizeof(cuHarmInput));
    res->d_inp = NULL;
    res->h_inp = (fcomplexcu*)malloc(sz);

    memcpy(res->h_inp, orr->h_inp, res->noHarms * res->stride * sizeof(fcomplexcu));

    PROF // Profiling  .
    {
      NV_RANGE_POP(); //Opt derivs
    }

    return res;
  }
  else
  {
    return NULL;
  }
}

/** Initiate a optimisation plane
 * If oPln has not been pre initialised and is NULL it will create a new data structure.
 * If oPln has been pre initialised the device ID and Idx are used!
 *
 */
cuOptCand* initOptCand(cuSearch* sSrch, cuOptCand* oPln, int devLstId )
{
  //confSpecsGen* sSpec = sSrch->genConf;
  confSpecsOpt*	conf	= sSrch->conf->opt;

  infoMSG(5,5,"Initialising optimiser.\n");

  FOLD // Get the possibly pre-initialised optimisation plane  .
  {
    if ( !oPln )
    {
      infoMSG(5,5,"Allocating optimisation plane.\n");

      oPln = (cuOptCand*)malloc(sizeof(cuOptCand));
      memset(oPln,0,sizeof(cuOptCand));

      if ( devLstId < MAX_GPUS )
      {
	oPln->gInf = &sSrch->gSpec->devInfo[devLstId];
      }
      else
      {
	fprintf(stderr, "ERROR: Device list index is greater that the list length, in function: %s.\n", __FUNCTION__);
	exit(EXIT_FAILURE);
      }
    }
    else
    {
      infoMSG(5,5,"Checking existing optimisation plane.\n");

      if ( oPln->gInf != &sSrch->gSpec->devInfo[devLstId] )
      {
	bool found = false;

	for ( int lIdx = 0; lIdx < MAX_GPUS; lIdx++ )
	{
	  if ( sSrch->gSpec->devInfo[lIdx].devid == oPln->gInf->devid )
	  {
	    devLstId 	= lIdx;
	    found 	= true;
	    break;
	  }
	}

	if (!found)
	{
	  if (devLstId < MAX_GPUS )
	  {
	    oPln->gInf = &sSrch->gSpec->devInfo[devLstId];
	  }
	  else
	  {
	    fprintf(stderr, "ERROR: Device list index is greater that the list length, in function: %s.\n", __FUNCTION__);
	    exit(EXIT_FAILURE);
	  }
	}
      }
    }
  }

  FOLD // Create all stuff  .
  {
    setDevice(oPln->gInf->devid) ;

    int maxSz = 0;
    int maxWidth = 0;
    float zMax;

    FOLD // Determine the largest zMax  .
    {
      zMax	= MAX(sSrch->sSpec->zMax+50, sSrch->sSpec->zMax*2);
      zMax	= MAX(zMax, 60 * sSrch->noSrchHarms );
      zMax	= MAX(zMax, sSrch->sSpec->zMax * 34 + 50 );  		// TODO: This may be a bit high!
    }

    FOLD // Determine max plane size  .
    {
      for ( int i=0; i < sSrch->noHarmStages; i++ )
      {
	MAXX(maxWidth, conf->optPlnSiz[i] );
      }
      for ( int i=0; i < NO_OPT_LEVS; i++ )
      {
	MAXX(maxSz, conf->optPlnDim[i]);
      }
#ifdef WITH_OPT_BLK2
      MAXX(maxSz, maxWidth * conf->optResolution);
#endif
      oPln->maxNoR	= maxSz*1.15;					// The maximum number of r points we can handle
      oPln->maxNoZ 	= maxSz;					// The maximum number of z points we can handle
    }

    oPln->cuSrch	= sSrch;					// Set the pointer t the search specifications
    oPln->maxHalfWidth	= cu_z_resp_halfwidth<double>( zMax, HIGHACC );	// The halfwidth of the largest plane we think we may handle
    oPln->conf		= conf;						// Should this rather be a duplicate?
    oPln->flags		= oPln->conf->flags;				// Individual flags allows separate configuration

    FOLD // Create streams  .
    {
      infoMSG(5,6,"Create streams.\n");

      CUDA_SAFE_CALL(hipStreamCreate(&oPln->stream),"Creating stream for candidate optimisation.");

      PROF // Profiling, name stream  .
      {
	char nmStr[1024];
	sprintf(nmStr,"Optimisation Stream %02i", oPln->pIdx);
	NV_NAME_STREAM(oPln->stream, nmStr);
      }
    }

    FOLD // Create events  .
    {
      if ( oPln->flags & FLAG_PROF )
      {
	infoMSG(5,5,"Create Events.\n");

	CUDA_SAFE_CALL(hipEventCreate(&oPln->inpInit),     "Creating input event inpInit." );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->inpCmp),      "Creating input event inpCmp."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->compInit),    "Creating input event compInit.");
	CUDA_SAFE_CALL(hipEventCreate(&oPln->compCmp),     "Creating input event compCmp." );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->outInit),     "Creating input event outInit." );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->outCmp),      "Creating input event outCmp."  );

	CUDA_SAFE_CALL(hipEventCreate(&oPln->tInit1),      "Creating input event tInit1."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tComp1),      "Creating input event tComp1."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tInit2),      "Creating input event tInit2."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tComp2),      "Creating input event tComp2."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tInit3),      "Creating input event tInit3."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tComp3),      "Creating input event tComp3."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tInit4),      "Creating input event tInit4."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tComp4),      "Creating input event tComp4."  );
      }
      else
      {
	CUDA_SAFE_CALL(hipEventCreate(&oPln->inpInit,	hipEventDisableTiming),	"Creating input event inpInit." );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->inpCmp,	hipEventDisableTiming),	"Creating input event inpCmp."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->compInit,	hipEventDisableTiming),	"Creating input event compInit.");
	CUDA_SAFE_CALL(hipEventCreate(&oPln->compCmp,	hipEventDisableTiming),	"Creating input event compCmp." );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->outInit,	hipEventDisableTiming),	"Creating input event outInit." );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->outCmp,	hipEventDisableTiming),	"Creating input event outCmp."  );

	CUDA_SAFE_CALL(hipEventCreate(&oPln->tInit1, hipEventDisableTiming),      "Creating input event tInit1."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tComp1, hipEventDisableTiming),      "Creating input event tComp1."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tInit2, hipEventDisableTiming),      "Creating input event tInit2."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tComp2, hipEventDisableTiming),      "Creating input event tComp2."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tInit3, hipEventDisableTiming),      "Creating input event tInit3."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tComp3, hipEventDisableTiming),      "Creating input event tComp3."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tInit4, hipEventDisableTiming),      "Creating input event tInit4."  );
	CUDA_SAFE_CALL(hipEventCreate(&oPln->tComp4, hipEventDisableTiming),      "Creating input event tComp4."  );
      }
    }

    FOLD // Allocate device memory  .
    {
      infoMSG(5,6,"Allocate device memory.\n");

      size_t freeMem, totalMem;
      int maxHarm = 1;

      oPln->input	= (cuHarmInput*)malloc(sizeof(cuHarmInput));
      oPln->outSz	= (oPln->maxNoR * oPln->maxNoZ ) * sizeof(float);

#ifdef	WITH_OPT_PLN2
      maxHarm		= MAX(conf->optMinLocHarms, sSrch->noSrchHarms );
      oPln->outSz	= (oPln->maxNoR * maxHarm * oPln->maxNoZ ) * sizeof(hipfftComplex);
#endif	// WITH_OPT_PLN2

#ifdef	WITH_OPT_PLN3
      oPln->input->size	= (maxWidth*10 + 2*oPln->maxHalfWidth) * sSrch->noSrchHarms * sizeof(hipfftComplex)*2; // The noR is oversized to allow for moves of the plane without getting new input
#endif	// WITH_OPT_PLN3

      CUDA_SAFE_CALL(hipMemGetInfo ( &freeMem, &totalMem ), "Getting Device memory information");
#ifdef MAX_GPU_MEM
      long  Diff = totalMem - MAX_GPU_MEM;
      if( Diff > 0 )
      {
	freeMem-= Diff;
	totalMem-=Diff;
      }
#endif

      if ( (oPln->input->size + oPln->outSz) > freeMem )
      {
	printf("Not enough GPU memory to create any more stacks.\n");
	free(oPln);
	return NULL;
      }
      else
      {
	infoMSG(6,6,"Input %.2f MB output %.2f MB.\n", oPln->input->size*1e-6, oPln->outSz*1e-6 );

	// Allocate device memory
	CUDA_SAFE_CALL(hipMalloc(&oPln->d_out,  oPln->outSz),   "Failed to allocate device memory for kernel stack.");
	CUDA_SAFE_CALL(hipMalloc(&oPln->input->d_inp,  oPln->input->size),   "Failed to allocate device memory for kernel stack.");

	// Allocate host memory
	CUDA_SAFE_CALL(hipHostMalloc(&oPln->h_out,  oPln->outSz), "Failed to allocate device memory for kernel stack.");
	CUDA_SAFE_CALL(hipHostMalloc(&oPln->input->h_inp,  oPln->input->size), "Failed to allocate device memory for kernel stack.");
      }
    }
  }

  return oPln;
}

/** Create multiplication kernel and allocate memory for planes on all devices  .
 *
 * Create the kernels on the first device and then copy it to all others
 *
 * @param sSrch     A pointer to the search structure
 *
 * @return
 */
void initOptimisers(cuSearch* sSrch )
{
  size_t free, total;                           ///< GPU memory

  infoMSG(4,4,"Initialise all optimisers.\n");

  sSrch->oInf = new cuOptInfo;
  memset(sSrch->oInf, 0, sizeof(cuOptInfo));

  CUDA_SAFE_CALL(hipGetLastError(), "Entering initOptimisers.");

  double halfWidth = cu_z_resp_halfwidth<double>(sSrch->sSpec->zMax+10, HIGHACC)+10;	// Candidate may be on the z-max border so buffer a bit

  cuOptCand*	devOpts[MAX_GPUS];

  FOLD // Create the primary stack on each device, this contains the kernel  .
  {
    PROF // Profiling  .
    {
      NV_RANGE_PUSH("Init Optimisers");
    }

    // Determine the number of optimisers to make
    sSrch->oInf->noOpts = 0;
    for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) // Loop over devices  .
    {
      if ( sSrch->gSpec->noDevOpt[dev] <= 0 )
      {
	// Use the default of 4
	sSrch->gSpec->noDevOpt[dev] = 4;

	infoMSG(5,5,"Using the default %i optimisers per GPU.\n", sSrch->gSpec->noDevOpt[dev]);
      }
      sSrch->oInf->noOpts += sSrch->gSpec->noDevOpt[dev];
    }

    infoMSG(5,5,"Initialising %i optimisers on %i devices.\n", sSrch->oInf->noOpts, sSrch->gSpec->noDevices);

    // Initialise the individual optimisers
    sSrch->oInf->opts = (cuOptCand*)malloc(sSrch->oInf->noOpts*sizeof(cuOptCand));
    memset(sSrch->oInf->opts, 0, sSrch->oInf->noOpts*sizeof(cuOptCand));

    int idx = 0;
    for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) // Loop over devices  .
    {
      for ( int oo = 0 ; oo < sSrch->gSpec->noDevOpt[dev]; oo++ )
      {
	// Setup some basic info
	sSrch->oInf->opts[idx].pIdx	= idx;
	sSrch->oInf->opts[idx].gInf	= &sSrch->gSpec->devInfo[dev];

	initOptCand(sSrch, &sSrch->oInf->opts[idx], dev );

	// Initialise device
	if ( oo == 0 )
	{
	  devOpts[dev] = &sSrch->oInf->opts[idx];
	}

	idx++;
      }
    }

    PROF // Profiling  .
    {
      NV_RANGE_POP(); // Init Optimisers
    }
  }

  // Note I found the response plane method to be slower or just equivalent
  Fout // Setup response plane  .
  {
    // Set up planes
    int sz = sSrch->gSpec->noDevices*sizeof(cuRespPln); 	// The size in bytes if the plane
    sSrch->oInf->responsePlanes =  (cuRespPln*)malloc(sz);
    memset(sSrch->oInf->responsePlanes, 0, sz);
    for ( int dev = 0 ; dev < sSrch->gSpec->noDevices; dev++ ) 	// Loop over devices  .
    {
      gpuInf* gInf     	= &sSrch->gSpec->devInfo[dev];
      int device	= gInf->devid;
      cuRespPln* resp	= &sSrch->oInf->responsePlanes[dev];

      FOLD // See if we can use the cuda device and whether it may be possible to do GPU in-mem search .
      {
	infoMSG(5,6,"access device %i\n", device);

	PROF // Profiling  .
	{
	  NV_RANGE_PUSH("Get Device");
	}

	if ( device >= getGPUCount() )
	{
	  fprintf(stderr, "ERROR: There is no CUDA device %i.\n", device);
	  continue;
	}
	int currentDevvice;
	CUDA_SAFE_CALL(hipSetDevice(device), "Failed to set device using hipSetDevice");
	CUDA_SAFE_CALL(hipGetDevice(&currentDevvice), "Failed to get device using hipGetDevice");
	if (currentDevvice != device)
	{
	  fprintf(stderr, "ERROR: CUDA Device not set.\n");
	  continue;
	}
	else
	{
	  CUDA_SAFE_CALL(hipMemGetInfo ( &free, &total ), "Getting Device memory information");
#ifdef MAX_GPU_MEM
	  long  Diff = total - MAX_GPU_MEM;
	  if( Diff > 0 )
	  {
	    free-= Diff;
	    total-=Diff;
	  }
#endif
	}

	PROF // Profiling  .
	{
	  NV_RANGE_POP(); // Get Device
	}
      }

      FOLD // Calculate the size of a response function plane  .
      {
	resp->zMax	= (ceil(sSrch->sSpec->zMax/sSrch->noSrchHarms)+20)*sSrch->noSrchHarms ;
	resp->dZ 	= sSrch->conf->opt->zScale / (double)sSrch->conf->opt->optResolution;
	resp->noRpnts	= sSrch->conf->opt->optResolution;
	resp->noZ	= resp->zMax * 2 / resp->dZ + 1 ;
	resp->halfWidth = halfWidth;
	resp->noR	= sSrch->conf->opt->optResolution*halfWidth*2 ;
	resp->oStride 	= getStride( resp->noR, sizeof(float2), sSrch->gSpec->devInfo[dev].alignment);
	resp->size	= resp->oStride * resp->noZ * sizeof(float2);
      }

      if ( resp->size < free*0.95 )
      {
	printf("Allocating optimisation response function plane %.2f MB\n", resp->size/1e6 );

	infoMSG(5, 5, "Allocating optimisation response function plane %.2f MB\n", resp->size/1e6 );

	CUDA_SAFE_CALL(hipMalloc(&resp->d_pln,  resp->size), "Failed to allocate device memory optimisation response plane.");
	CUDA_SAFE_CALL(hipMemsetAsync(resp->d_pln, 0, resp->size, devOpts[dev]->stream), "Failed to initiate optimisation response plane to zero");

	// This kernel isn't really necessary anymore
	//opt_genResponse(resp, devOpts[dev]->stream);

	for ( int optN = 0; optN < sSrch->oInf->noOpts; optN++ )
	{
	  cuOptCand* oCnd = &sSrch->oInf->opts[optN];

	  if ( oCnd->gInf->devid == devOpts[dev]->gInf->devid )
	  {
	    oCnd->responsePln = resp;
	  }
	}
      }
      else
      {
	fprintf(stderr,"WARNING: Not enough free GPU memory to use a response plane for optimisation. Pln needs %.2f GB there is %.2f GB. \n", resp->size/1e9, free/1e9 );
	memset(resp, 0, sizeof(cuRespPln) );
      }
    }
  }

}

cuSearch* initCuOpt(cuSearch* srch)
{
  //if ( !srch )
  //  srch = initSearchInf(sSpec, gSpec, srch);

  PROF // Profiling  .
  {
    NV_RANGE_PUSH("Init CUDA optimisers");
  }

  if ( !srch->oInf )
  {
    initOptimisers( srch );
  }
  else
  {
    // TODO: Do a whole bunch of checks here!
    fprintf(stderr, "ERROR: %s has not been set up to handle a pre-initialised memory info data structure.\n", __FUNCTION__);
    exit(EXIT_FAILURE);
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP();	// Init CUDA optimisers
  }

  return srch;
}

void freeHarmInput(cuHarmInput* inp)
{
  if ( inp )
  {
    cudaFreeNull(inp->d_inp);
    freeNull(inp->h_inp);
    freeNull(inp);
  }
}

/** Optimise derivatives of a candidate  .
 *
 */
void* optCandDerivs(accelcand* cand, cuSearch* srch )
{
  int ii;
  struct timeval start, end;    // Profiling variables

  fftInfo*	fft	= srch->fft;
  confSpecsOpt*	conf	= srch->conf->opt;
  searchSpecs*	sSpec	= srch->sSpec;

  FOLD // Update fundamental values to the optimised ones  .
  {
    infoMSG(5,5,"DERIVS\n");

    float	maxSig		= 0;
    int		bestH		= 0;
    float	bestP		= 0;
    double  	sig		= 0; // can be a float
    long long	numindep;
    float	candHPower	= 0;
    int		noStages	= 0;
    int 	kern_half_width;
    double	locpow;
    double	real;
    double	imag;
    double	power;
    int		maxHarms  	= MAX(cand->numharm, conf->optMinRepHarms) ;

    PROF // Profiling  .
    {
      if ( !(!(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD)) )
      {
	NV_RANGE_PUSH("DERIVS");
      }

      if ( conf->flags & FLAG_PROF )
      {
	gettimeofday(&start, NULL);
      }
    }

    cand->power   = 0;

    // Set up candidate
    cand->pows    = gen_dvect(maxHarms);
    cand->hirs    = gen_dvect(maxHarms);
    cand->hizs    = gen_dvect(maxHarms);
    cand->derivs  = (rderivs *)   malloc(sizeof(rderivs)  * maxHarms  );

    // Initialise values
    for( ii=0; ii < maxHarms; ii++ )
    {
      cand->hirs[ii]  = cand->r*(ii+1);
      cand->hizs[ii]  = cand->z*(ii+1);
    }

    for( ii = 1; ii <= maxHarms; ii++ )			// Calculate derivatives, powers and sigma for all harmonics  .
    {
      if      ( conf->flags & FLAG_OPT_NRM_LOCAVE   )
      {
	locpow = get_localpower3d(fft->data, fft->noBins, cand->r*ii, cand->z*ii, 0.0);
      }
      else if ( conf->flags & FLAG_OPT_NRM_MEDIAN1D )
      {
	locpow = get_scaleFactorZ(fft->data, fft->noBins, cand->r*ii, cand->z*ii, 0.0);
      }
      else if ( conf->flags & FLAG_OPT_NRM_MEDIAN2D )
      {
	fprintf(stderr,"ERROR: 2D median normalisation has not been written yet.\n");
	exit(EXIT_FAILURE);
      }
      else
      {
	// No normalisation this is plausible but not recommended

	// TODO: This should error if it is out of bounds?
	locpow = 1;
      }

      if ( locpow )
      {
	kern_half_width   = cu_z_resp_halfwidth<double>(fabs(cand->z*ii), HIGHACC);

	rz_convolution_cu<double, float2>((float2*)fft->data, fft->firstBin, fft->noBins, cand->r*ii, cand->z*ii, kern_half_width, &real, &imag);

	// Normalised power
	power = POWERCU(real, imag) / locpow ;

	cand->pows[ii-1] = power;

	get_derivs3d(fft->data, fft->noBins, cand->r*ii, cand->z*ii, 0.0, locpow, &cand->derivs[ii-1] );

	cand->power	+= power;
	int numz 	= round(srch->conf->gen->zMax / srch->conf->gen->zRes) * 2 + 1;
	if ( numz == 1 )
	{
	  numindep	= (sSpec->searchRHigh - sSpec->searchRLow) / (double)(ii) ;
	}
	else
	{
	  numindep	= (sSpec->searchRHigh - sSpec->searchRLow) * (numz + 1) * ( srch->conf->gen->zRes / 6.95 ) / (double)(ii);
	}

	sig		= candidate_sigma_cu(cand->power, (ii), numindep );

	infoMSG(6,6,"Harm %2i  local power %6.3f, normalised power %8.3f,   sigma %5.2f \n", ii, locpow, power, sig );

	if ( sig > maxSig || ii == 1 )
	{
	  maxSig        = sig;
	  bestP         = cand->power;
	  bestH         = (ii);
	}

	if ( ii == cand->numharm )
	{
	  candHPower    = cand->power;

	  if ( !(conf->flags & FLAG_OPT_BEST) )
	  {
	    break;
	  }
	}
      }
    }

    // Final values
    if ( bestP && (conf->flags & FLAG_OPT_BEST) && ( maxSig > 0.001 ) )
    {
      cand->numharm	= bestH;
      cand->sigma	= maxSig;
      cand->power	= bestP;

      infoMSG(4,4,"Cand best val Sigma: %5.2f Power: %6.4f  %i harmonics summed.", maxSig, bestP, bestH);
    }
    else
    {
      cand->power	= candHPower;
      noStages		= log2((double)cand->numharm);
      numindep		= srch->numindep[noStages];
      cand->sigma	= candidate_sigma_cu(candHPower, cand->numharm, numindep);

      infoMSG(4,4,"Cand harm val Sigma: %5.2f Power: %6.4f  %i harmonics summed.", cand->sigma, cand->power, cand->numharm);
    }

    PROF // Profiling  .
    {
      if ( !(!(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD)) )
      {
	NV_RANGE_POP(); // DERIVS
      }

      if ( conf->flags & FLAG_PROF )
      {
	gettimeofday(&end, NULL);
        float v1 =  (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec);

        // Thread (pthread) safe add to timing value
        pthread_mutex_lock(&srch->threasdInfo->candAdd_mutex);
        srch->timings[COMP_OPT_DERIVS] += v1;
        pthread_mutex_unlock(&srch->threasdInfo->candAdd_mutex);
      }
    }
  }

  return (NULL);
}

/** CPU process results
 *
 * This function is meant to be the entry of a separate thread
 *
 */
void* cpuProcess(void* ptr)
{
  candSrch*	res	= (candSrch*)ptr;
  cuSearch*	srch	= res->cuSrch;

  struct timeval start, end;    // Profiling variables

  accelcand*    cand	= res->cand;
  confSpecsOpt*	conf	= srch->conf->opt;

  if ( conf->flags & FLAG_OPT_NM_REFINE )
  {
    PROF // Profiling  .
    {
      if ( !(!(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD)) )
      {
	NV_RANGE_PUSH("NM_REFINE");
      }

      if ( conf->flags & FLAG_PROF )
      {
        gettimeofday(&start, NULL);
      }
    }

    initCand iCand;
    iCand.numharm	= cand->numharm;
    iCand.power		= cand->power;
    iCand.r		= cand->r;
    iCand.z		= cand->z;

    // Run the NM
    optInitCandPosSim<double>(&iCand,  res->input, 0.0005, 0.0005*conf->optPlnScale );

    cand->r		= iCand.r;
    cand->z		= iCand.z;
    cand->power		= iCand.power;

    // Free thread specific input memory
    freeHarmInput(res->input);
    res->input = NULL;

    PROF // Profiling  .
    {
      if ( !(!(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD)) )
      {
	NV_RANGE_POP(); // NM_REFINE
      }

      if ( conf->flags & FLAG_PROF )
      {
	gettimeofday(&end, NULL);
        float v1 =  (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec);

        // Thread (pthread) safe add to timing value
        pthread_mutex_lock(&res->cuSrch->threasdInfo->candAdd_mutex);
        srch->timings[COMP_OPT_REFINE_2] += v1;
        pthread_mutex_unlock(&res->cuSrch->threasdInfo->candAdd_mutex);
      }
    }
  }

  optCandDerivs(cand, srch);

  // Decrease the count number of running threads
  sem_trywait(&srch->threasdInfo->running_threads);

  free(res);

  return (NULL);
}

/** Optimise derivatives of a candidate Using the CPU  .
 * This usually spawns a separate CPU thread to do the sigma calculations
 */
void processCandDerivs(accelcand* cand, cuSearch* srch, cuHarmInput* inp = NULL, int candNo = -1)
{
  infoMSG(2,2,"Calc Cand Derivatives.\n");

  candSrch*     thrdDat  = new candSrch;
  memset(thrdDat, 0, sizeof(candSrch));

  confSpecsOpt*	conf	= srch->conf->opt;

  thrdDat->cand   = cand;
  thrdDat->cuSrch = srch;
  thrdDat->candNo = candNo;

  if ( conf->flags & FLAG_OPT_NM_REFINE )
  {
    // Make a copy of the input data for the thread to use
    thrdDat->input = duplicateHost(inp);
  }

  PROF // Profiling  .
  {
    NV_RANGE_PUSH("Post Thread");
  }

  // Increase the count number of running threads
  sem_post(&srch->threasdInfo->running_threads);

  if ( !(conf->flags & FLAG_SYNCH) && (conf->flags & FLAG_OPT_THREAD) )  // Create thread  .
  {
    pthread_t thread;
    int  iret1 = pthread_create( &thread, NULL, cpuProcess, (void*) thrdDat);

    if (iret1)	// Check return status
    {
      fprintf(stderr,"Error - pthread_create() return code: %d\n", iret1);
      exit(EXIT_FAILURE);
    }
  }
  else                              // Just call the function  .
  {
    cpuProcess( (void*) thrdDat );
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP(); // Post Thread
  }

  infoMSG(2,2,"Done");
}

/** This is the main function called by external elements  .
 *
 * @param cand		The canidate to refine
 * @param pln		The plane data structure to use for the GPU position refinement
 * @param candNo	The index of the candidate being optimised
 */
void optInitCandLocPlns(initCand* cand, cuOptCand* pln, int candNo )
{
  infoMSG(2,2,"Refine location by plain\n");

  PROF // Profiling  .
  {
    NV_RANGE_PUSH("Plns");
  }

  confSpecsOpt*	conf	= pln->conf;

  // Number of harmonics to check, I think this could go up to 32!
  int maxHarms	= MAX(cand->numharm, conf->optMinLocHarms);

  // Setup GPU plane
  pln->centR	= cand->r ;
  pln->centZ	= cand->z ;
  pln->noHarms	= maxHarms ;

  FOLD // Get best candidate location using iterative GPU planes  .
  {
    int depth;
    int noP;
    int rep	= 0;
    int lrep	= 0;
    bool doub	= false;
    const int	mxRep		= 10;
    const float moveBound	= 0.67;
    const float outBound	= 0.9;
    double sz;
    float posR, posZ;

    if ( pln->noHarms == 1  )
      sz = conf->optPlnSiz[0];
    if ( pln->noHarms == 2  )
      sz = conf->optPlnSiz[1];
    if ( pln->noHarms == 4  )
      sz = conf->optPlnSiz[2];
    if ( pln->noHarms == 8  )
      sz = conf->optPlnSiz[3];
    if ( pln->noHarms == 16 )
      sz = conf->optPlnSiz[4];

    pln->halfWidth 	= 0;
    cand->power	= 0;					// Set initial power to zero

    for ( int lvl = 0; lvl < NO_OPT_LEVS; lvl++ )
    {
      noP		= conf->optPlnDim[lvl] ;	// Set in the defaults text file

      lrep		= 0;
      depth		= 1;

      if ( noP )					// Check if there are points in this plane ie. are we optimising position at this level  .
      {
	if ( ( lvl == NO_OPT_LEVS-1 ) || (sz < 0.002) /*|| ( (sz < 0.06) && (abs(pln->centZ) < 0.05) )*/ )	// Potently force double precision
	{
	  // If last plane is not 0, it will be done with double precision
	  if (!doub)
	    cand->power = 0;

	  doub = true;
	}

	while ( (depth > 0) && (lrep < mxRep) )		// Recursively make planes at this scale  .
	{
	  if ( doub )
	  {
	    infoMSG(3,3,"Generate double precision plane - lvl %i  depth: %i  iteration %2i\n", lvl+1, depth, lrep);

	    // Double precision
	    optInitCandPosPln<double>(cand, pln, noP, sz,  rep++, candNo, lvl + 1 );
	  }
	  else
	  {
	    infoMSG(3,3,"Generate single precision plane - lvl %i  depth: %i  iteration %2i\n", lvl+1, depth, lrep);

	    // Standard single precision
	    optInitCandPosPln<float>(cand, pln, noP, sz,  rep++, candNo, lvl + 1 );
	  }

	  posR = fabs(( pln->centR - cand->r )/(pln->rSize/2.0));
	  posZ = fabs(( pln->centZ - cand->z )/(pln->zSize/2.0));

	  double rRes = pln->rSize/(double)(pln->noR-1) ;
	  
	  if ( posR > moveBound || posZ > moveBound )
	  {
	    if ( ( (posR > outBound) || (posZ > outBound) ) && ( depth < lvl+1) )
	    {
	      // Zoom out by half
	      sz *= conf->optPlnScale / 2.0 ;
	      depth++;
	      infoMSG(5,5,"Zoom out");
	    }
	    else
	    {
	      // we'r just going to move the plane
	      infoMSG(5,5,"Move plain");
	    }
	  }
	  else
	  {
	    // Break condition
	    if ( rRes < 1e-5 )
	    {
	      infoMSG(5,5,"Break size is small enough\n");
	      break;
	    }

	    // Zoom in
	    sz /= conf->optPlnScale;
	    depth--;
	    infoMSG(5,5,"Zoom in\n");
	    if ( sz < 2.0*rRes )
	      sz = rRes*2.0;
	  }

	  ++lrep;
	}
      }
      else
      {
	if ( doub )
	  infoMSG(3,3,"Skip plane lvl %i (double precision)", lvl+1);
	else
	  infoMSG(3,3,"Skip plane lvl %i (single precision)", lvl+1);
      }
    }
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP(); // Plns
  }
}

/** This is the main function called by external elements  .
 *
 * @param cand
 * @param pln
 * @param nn
 */
void opt_accelcand(accelcand* cand, cuOptCand* pln, int candNo)
{
  confSpecsOpt*  conf	= pln->conf;

  PROF // Profiling  .
  {
    char Txt[1024];
    sprintf(Txt, "Opt Cand %03i", candNo);

    NV_RANGE_PUSH(Txt);
  }

  initCand iCand;				// plane refining uses an initial candidate data structure
  iCand.r 		= cand->r;
  iCand.z 		= cand->z;
  iCand.power		= cand->power;
  iCand.numharm 	= cand->numharm;

  FOLD // Refine position in ff space  .
  {
    struct timeval start, end;    // Profiling variables

    PROF // Profiling  .
    {
      NV_RANGE_PUSH("Refine pos");

      if ( conf->flags & FLAG_PROF )
      {
	gettimeofday(&start, NULL);
      }
    }

    if      ( conf->flags & FLAG_OPT_NM )
    {
      prepInput(&iCand, pln, 15);
      optInitCandPosSim<double>(&iCand, pln->input, 0.5, 0.5*conf->optPlnScale);
    }
    else if ( conf->flags & FLAG_OPT_SWARM )
    {
      fprintf(stderr,"ERROR: Particle swarm optimisation has been removed.\n");
      exit(EXIT_FAILURE);
    }
    else // Default use planes
    {
      optInitCandLocPlns(&iCand, pln, candNo);
    }

    PROF // Profiling  .
    {
      NV_RANGE_POP();	// Refine pos

      if ( conf->flags & FLAG_PROF )
      {
	gettimeofday(&end, NULL);
	float v1 =  (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec) ;

	// Thread (omp) safe add to timing value
#pragma omp atomic
	pln->cuSrch->timings[COMP_OPT_REFINE_1] += v1;
      }
    }
  }

  // Update the details of the final candidate from the updated initial candidate
  cand->r 		= iCand.r;
  cand->z 		= iCand.z;
  cand->power		= iCand.power;
  cand->numharm 	= iCand.numharm;

  FOLD // Optimise derivatives  .
  {
    prepInput(&iCand, pln, 15);
    processCandDerivs(cand, pln->cuSrch, pln->input,  candNo);
  }

  PROF // Profiling  .
  {
    NV_RANGE_POP(); // Txt
  }
}

int optList(GSList *listptr, cuSearch* cuSrch)
{
  struct timeval start, end;

  TIME //  Timing  .
  {
    NV_RANGE_PUSH("GPU Kernels");
  }

  int numcands 	= g_slist_length(listptr);

  int ii	= 0;
  int comp	= 0;

#if	!defined(DEBUG) && defined(WITHOMP)   // Parallel if we are not in debug mode  .
  if ( cuSrch->conf->opt->flags & FLAG_SYNCH )
  {
    omp_set_num_threads(1);
  }
  else
  {
    omp_set_num_threads(cuSrch->oInf->noOpts);
  }
#pragma omp parallel
#endif	// !DEBUG && WITHOMP
  FOLD  	// Main GPU loop  .
  {
    accelcand *candGPU;

    int tid         = 0;
    int ti          = 0; // tread specific index
#ifdef	WITHOMP
    omp_get_thread_num();
#endif	// WITHOMP

    cuOptCand* oPlnPln = &(cuSrch->oInf->opts[tid]);

    setDevice(oPlnPln->gInf->devid) ;

    // Make sure all initialisation and other stuff on the device is complete
    CUDA_SAFE_CALL(hipDeviceSynchronize(), "Synchronising device before candidate generation");

    while (listptr)  // Main Loop  .
    {
#pragma omp critical
      FOLD  // Synchronous behaviour  .
      {
#ifndef  DEBUG
	if ( cuSrch->conf->opt->flags & FLAG_SYNCH )
#endif
	{
	  tid 		= ii % cuSrch->oInf->noOpts ;
	  oPlnPln 	= &(cuSrch->oInf->opts[tid]);
	  setDevice(oPlnPln->gInf->devid);
	}

	FOLD // Calculate candidate  .
	{
	  if ( listptr )
	  {
	    candGPU	= (accelcand *) (listptr->data);
	    listptr	= listptr->next;
	    ii++;
	    ti = ii;
#ifdef CBL
	    FOLD // TMP: This can get removed
	    {
	      candGPU->init_power    = candGPU->power;
	      candGPU->init_sigma    = candGPU->sigma;
	      candGPU->init_numharm  = candGPU->numharm;
	      candGPU->init_r        = candGPU->r;
	      candGPU->init_z        = candGPU->z;
	    }
#endif
	  }
	  else
	  {
	    candGPU = NULL;
	  }
	}
      }

      if ( candGPU ) // Optimise  .
      {
	infoMSG(2,2,"\nOptimising initial candidate %i/%i, Power: %.3f  Sigma %.2f  Harm %i at (%.3f %.3f)\n", ti, numcands, candGPU->power, candGPU->sigma, candGPU->numharm, candGPU->r, candGPU->z );

	opt_accelcand(candGPU, oPlnPln, ti);

#pragma omp atomic
	comp++;

	if ( msgLevel == 0 )
	{
	  printf("\rGPU optimisation %5.1f%% complete   ", comp / (float)numcands * 100.0f );
	  fflush(stdout);
	}
      }
    }
  }

  printf("\rGPU optimisation %5.1f%% complete                      \n", 100.0f );

  TIME //  Timing  .
  {
    NV_RANGE_POP(); // GPU Kernels
    gettimeofday(&start, NULL);
  }

  // Wait for CPU derivative threads to finish
  waitForThreads(&cuSrch->threasdInfo->running_threads, "Waiting for CPU threads to complete.", 200 );

  TIME //  Timing  .
  {
    gettimeofday(&end, NULL);
    cuSrch->timings[TIME_OPT_WAIT] += (end.tv_sec - start.tv_sec) * 1e6 + (end.tv_usec - start.tv_usec);
  }

  return 0;
}


template int ffdotPln<float >( cuOptCand* pln, fftInfo* fft );
template int ffdotPln<double>( cuOptCand* pln, fftInfo* fft );

template int ffdotPln_ker<float >( cuOptCand* pln, fftInfo* fft );
template int ffdotPln_ker<double>( cuOptCand* pln, fftInfo* fft );
