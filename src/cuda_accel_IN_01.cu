#include "hip/hip_runtime.h"
#include "cuda_accel_IN.h"

__device__ inline float median(float* data, int arrayLength, int eType = 0 )
{
  float medianValue;

  int idx = arrayLength/2.0;

  if ( (arrayLength & 1) )    // odd  .
  {
    medianValue = data[idx];
  }
  else                        //even  .
  {
    if ( eType == -1 )        // lower  .
    {
      medianValue = data[idx-1];
    }
    else if ( eType == 1 )    // upper  .
    {
      medianValue = data[idx];
    }
    else                      // mean  .
    {
      medianValue = ( data[idx-1] + data[idx] ) / 2.0f;
    }
  }

  return medianValue;
}

__device__ inline int midpoint(int imin, int imax)
{
  return (imin + imax) / 2.0 ;
}

__device__ int binSearch(const float* data, float key, int arrayLength)
{
  const int bid = blockIdx.y  * gridDim.x  + blockIdx.x;        /// Block ID (flat index) // TMP
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;       /// Block ID - flat index

  int imax = arrayLength-1;
  int imin = 0;

//  int ite = 0;

  while ( imax >= imin )
  {
    int imid = midpoint(imin, imax) ;

//    if ( bid == 0 && tid == 0 )
//      printf("%02i imin %05i  imin %05i imax %05i   %22.6f  %22.6f  %22.6f    key %22.6f  \n", ite, imin, imid, imax, data[imin], data[imid], data[imax],  key);

    if      ( data[imid] == key )
      return imid;
    else if ( data[imid] <  key )
      imin = imid + 1;
    else
      imax = imid - 1;
  }

//  if ( bid == 0 && tid == 0 )
//    printf("Returning %02i  \n", imin);

  return imin;
}

/** XOR swap two integer values
 *
 * @param a integer a
 * @param b integer b
 */
__device__ inline void swap(int & a, int & b)
{
  a = a ^ b;
  b = a ^ b;
  a = a ^ b;
}

/** Compare and swap two values (if they are in the wrong order).
 *
 * @param valA The first value
 * @param valB The second value
 * @param dir direction to sort data ( 1 -> smallest to largest AND -1 -> largest to smallest )
 */
__device__ inline void Comparator(float &valA, float &valB, uint dir)
{
  if ((valA > valB) == dir)
  {
    register float t;
    //swap(*(int*)&valA, *(int*)&valB );
    t     = valA;
    valA  = valB;
    valB  = t;
  }
}

/** In-place Bitonic sort a float array.
 * @param data A pointer to an shared memory array containing elements to be sorted.
 * @param arrayLength The number of elements in the array
 * @param trdId the index of the calling thread (1 thread for 2 items in data)
 * @param noThread The number of thread that are sorting this data
 * @param dir direction to sort data ( 1 -> smallest to largest AND -1 -> largest to smallest )
 *
 * This is an in-place bitonic sort.
 * This is very fast for small numbers of items, ie; when they can all fit in shared memory, ie < ~12K
 *
 * It has a constant performance of \f$ O\left(n\ \log^2 n \right)\f$ where n is the number of items to be sorted.
 * It requires the sort to be performed by only one block, as it requires synchronisation.
 * But this allows for the use of SM
 *
 * Each thread counts for to items in the array, as each thread performs comparisons between to elements.
 * Generally there is ~48.0 KBytes of shared memory, thus could sort up to 12288 items. However there is a
 * maximum of 1024 thread per block, thus if there are more that 2048 threads each thread must do multiple comparisons at
 * each step. These are refereed to as batches.
 *
 */
__device__ void bitonicSort(float *data, const uint arrayLength, const uint trdId, const uint noThread, const int dir )
{
  const uint noBatch = ceilf(arrayLength / 2.0f / noThread);     // Number of comparisons each thread must do
  uint idx;                               // The index including batch adjustment
  const uint max = arrayLength * 2;       // The maximum distance a thread could compare
  uint bIdx;                              // The thread position in the block
  uint hSz = 1;                           // half block size
  uint pos1, pos2, blk;                   // index of points to be compared
  uint len;                               // The distance between items to swap
  uint bach;                              // The batch we are processing
  uint shift = 32;                        // Amount to bitshift by to calculate remainders
  uint shift2;
  uint hsl1;

  // Incrementally sort blocks of 2 then 4 then 8 ... items
  for (uint size = 2; size < max; size <<= 1, shift--)
  {
    hSz = (size >> 1);
    hsl1 = hSz - 1;

    __syncthreads();

    // Bitonic sort, two Bitonic sorted list into Bitonic list
    for (bach = 0; bach < noBatch; bach++)
    {
      idx = (trdId + bach * noThread);

      //bIdx = hSz - 1 - idx % hSz;
      //bIdx = hsl1 - (idx << shift) >> shift;  // My method
      bIdx = hsl1 - idx & (hSz - 1);// x mod y == x & (y-1), where y is 2^n.

      blk = idx / hSz;

      len = size - 1 - bIdx * 2;
      pos1 = blk * size + bIdx;
      pos2 = pos1 + len;

      if (pos2 < arrayLength)
        Comparator(data[pos1], data[pos2], dir);
    }

    // Bitonic Merge
    for (len = (hSz >>= 1), shift2 = shift + 1; len > 0; len >>= 1, shift2++)
    {
      hSz = (len << 1);

      __syncthreads();
      for (bach = 0; bach < noBatch; bach++)
      {
        idx = (trdId + bach * noThread);

        //bIdx  = idx % len;
        //bIdx = (idx << shift2) >> shift2;
        bIdx = idx & (len - 1);// x mod y == x & (y-1), where y is 2^n.

        blk = idx / len;

        pos1 = blk * hSz + bIdx;
        pos2 = pos1 + len;

        if (pos2 < arrayLength)
          Comparator(data[pos1], data[pos2], dir);
      }
    }
  }

  __syncthreads();  // Ensure all data is sorted before we return
}

/** In-place Bitonic sort a float array.
 * @param data A pointer to an shared memory array containing elements to be sorted.
 * @param arrayLength The number of elements in the array
 * @param trdId the index of the calling thread (1 thread for 2 items in data)
 * @param noThread The number of thread that are sorting this data
 * @param dir direction to sort data ( 1 -> smallest to largest AND -1 -> largest to smallest )
 *
 * This is an in-place bitonic sort.
 * This is very fast for small numbers of items, ie; when they can all fit in shared memory, ie < ~12K
 *
 * It has a constant performance of \f$ O\left(n\ \log^2 n \right)\f$ where n is the number of items to be sorted.
 * It requires the sort to be performed by only one block, as it requires synchronisation.
 * But this allows for the use of SM
 *
 * Each thread counts for to items in the array, as each thread performs comparisons between to elements.
 * Generally there is ~48.0 KBytes of shared memory, thus could sort up to 12288 items. However there is a
 * maximum of 1024 thread per block, thus if there are more that 2048 threads each thread must do multiple comparisons at
 * each step. These are refereed to as batches.
 *
 */
__device__ void bitonicSort1Warp(float *data, const uint arrayLength, const uint trdId, const uint noThread, const int dir )
{
  const uint noBatch = ceilf(arrayLength / 2.0f / noThread);     // Number of comparisons each thread must do
  uint idx;                               // The index including batch adjustment
  const uint max = arrayLength * 2;       // The maximum distance a thread could compare
  uint bIdx;                              // The thread position in the block
  uint hSz = 1;                           // half block size
  uint pos1, pos2, blk;                   // index of points to be compared
  uint len;                               // The distance between items to swap
  uint bach;                              // The batch we are processing
  uint shift = 32;                        // Amount to bitshift by to calculate remainders
  uint shift2;
  uint hsl1;

  // Incrementally sort blocks of 2 then 4 then 8 ... items
  for (uint size = 2; size < max; size <<= 1, shift--)
  {
    hSz = (size >> 1);
    hsl1 = hSz - 1;

    // Bitonic sort, two Bitonic sorted list into Bitonic list
    for (bach = 0; bach < noBatch; bach++)
    {
      idx = (trdId + bach * noThread);

      //bIdx = hSz - 1 - idx % hSz;
      //bIdx = hsl1 - (idx << shift) >> shift;  // My method
      bIdx = hsl1 - idx & (hSz - 1);// x mod y == x & (y-1), where y is 2^n.

      blk = idx / hSz;

      len = size - 1 - bIdx * 2;
      pos1 = blk * size + bIdx;
      pos2 = pos1 + len;

      if (pos2 < arrayLength)
        Comparator(data[pos1], data[pos2], dir);
    }

    // Bitonic Merge
    for (len = (hSz >>= 1), shift2 = shift + 1; len > 0; len >>= 1, shift2++)
    {
      hSz = (len << 1);

      for (bach = 0; bach < noBatch; bach++)
      {
        idx = (trdId + bach * noThread);

        //bIdx  = idx % len;
        //bIdx = (idx << shift2) >> shift2;
        bIdx = idx & (len - 1);// x mod y == x & (y-1), where y is 2^n.

        blk = idx / len;

        pos1 = blk * hSz + bIdx;
        pos2 = pos1 + len;

        if (pos2 < arrayLength)
          Comparator(data[pos1], data[pos2], dir);
      }
    }
  }
}

/** Calculate the median of an array of float values  .
 *
 * This sorts the actual array so the values will be reordered
 * This uses a bitonicSort which is very fast if the array is in SM
 * This means that there
 *
 * @param array array of floats to search, this will be reordered should be in SM
 * @param arrayLength the number of floats in the array
 * @param dir the direction to sort the array 1 = increasing
 * @return the median value
 */
__device__ float cuMedianOne(float *array, uint arrayLength)
{
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;       /// Block ID - flat index
  const int bSz = blockDim.x  * blockDim.y;                     /// Block size

  __shared__ float  medianValue;

  FOLD // Sort  .
  {
    __syncthreads();

    bitonicSort(array, arrayLength, tid, bSz, 1);
  }

  FOLD // Calculate the median  .
  {
    if ( tid == 0 )
    {
      int idx = arrayLength / 2.0f;

      if ((arrayLength & 1))      // odd
      {
        medianValue = array[idx];
      }
      else                        //even
      {
        // mean
        //medianValue = (smBuffer[idx-1] + smBuffer[idx])/2.0f;

        // lower
        medianValue = array[idx - 1];

        // upper
        //medianValue = smBuffer[idx];
      }
    }
  }

  __syncthreads();

  return medianValue;
}

/** Calculate the median of up to 16*bufferSz float values  .
 *
 * This Sorts sections of the array, and then find the median by extracting and combining the
 * centre chunk(s) of these and sorting that. To find the median.
 *
 * Note this reorders the original array
 *
 * @param data the value to find the median of
 * @param buffer to do the sorting in this is bufferSz long and should be in SM
 * @param arrayLength the length of the data array
 * @return The median of data
 */
template< int bufferSz >
__device__ float cuMedianBySection(float *data, float *buffer, uint arrayLength)
{
  const int bid = blockIdx.y  * gridDim.x  + blockIdx.x;        /// Block ID (flat index) // TMP

  const int tid = threadIdx.y * blockDim.x + threadIdx.x;       /// Block ID - flat index
  const int bSz = blockDim.x  * blockDim.y;                     /// Block size

  const int maxSub = 32;

  __shared__ float  lower[maxSub];
  __shared__ float  upper[maxSub];
  __shared__ float  medianA[maxSub];

  __shared__ float  medianValue;

  __shared__ float  maxLower;
  __shared__ float  minUpper;
  __shared__ int    locLower;
  __shared__ int    locUpper;
  __shared__ bool   found;
  __shared__ int    belowN;

  int noSections        = ceilf( arrayLength/(float)bufferSz );
  int noBatches         = ceilf( bufferSz/(float)(bSz) );

  if ( noSections <= maxSub )
  {
    //if ( tid < bufferSz)
    {
      int midLen          = floorf( bufferSz/(float)noSections );   // The length of middle sections
      float hSz           = ceilf( midLen/2.0f );                   // Half the length of the middle sections
      int ite             = 0;                                      // Iteration
      float rMedian;                                                // The number of medians to consider
      int len;                                                      // Only really needed by tid 0
      int before;                                                   // Only really needed by tid 0
      float pPos;

      float lowerVal;
      float upperVal;

      if ( tid == 0 )
      {
        found = false;
      }

      FOLD // Sort each section and write back to device memory  .
      {
        for ( int sec = 0; sec < noSections; sec++ )
        {
          int sStart      = MIN(bufferSz*sec,     arrayLength);
          int sEnd        = MIN(bufferSz*(sec+1), arrayLength);
          int sLen        = sEnd - sStart;

          int mMid        = sLen / 2.0 ;

          FOLD // Load section into shared memory  .
          {
            __syncthreads();

            for ( int batch = 0; batch < noBatches; batch++)
            {
              int dataIdx   = sec*bufferSz + batch*bSz + tid ;
              int bufferIdx = batch*bSz + tid ;

              if ( dataIdx < arrayLength && bufferIdx < sLen )
              {
                buffer[bufferIdx] = data[dataIdx];
              }
            }
          }

          FOLD // Sort  .
          {
            __syncthreads();

            bitonicSort(buffer, sLen, tid, bSz, 1);
          }

          FOLD // Write section from shared memory main memory  .
          {
            __syncthreads();

            for ( int batch = 0; batch < noBatches; batch++)
            {
              int dataIdx   = sec*bufferSz + batch*bSz + tid ;
              int bufferIdx = batch*bSz + tid ;

              if ( dataIdx < arrayLength && bufferIdx < sLen )
              {
                float val       = buffer[bufferIdx];
                data[dataIdx]   = val;

                if ( bufferIdx == mMid )
                {
                  medianA[sec]  = val;
                }
              }
            }
          }
        }
      }

      FOLD // Calculate the median of the median  .
      {
        FOLD // Sort the medians  .
        {
          __syncthreads();

          if ( tid < noSections )
          {
            bitonicSort1Warp(medianA, noSections, tid, bSz, 1);
          }
        }

        FOLD // Calculate the median of medians  .
        {
          __syncthreads();

          if ( tid == 0 )
          {
            medianValue           = median(medianA, noSections, 0 );
            pPos                  = (noSections-1)/2.0;
            lowerVal              = medianA[0];
            upperVal              = medianA[noSections-1];
          }
        }
      }

      FOLD // Get median from sections  .
      {
        __syncthreads();

        while ( !found )
        {
          FOLD // Initialise values  .
          {
            len         = 0;
            before      = 0;
            rMedian     = medianValue;
          }

          FOLD // Load the middle of each section into shared memory and save lower and upper values  .
          {
            __syncthreads();

            for ( int sec = 0; sec < noSections; sec++)
            {
              int sStart      = MIN(bufferSz*sec,     arrayLength);
              int sEnd        = MIN(bufferSz*(sec+1), arrayLength);
              int sLen        = sEnd - sStart;

              FOLD // Find out how many points below the pivot  .
              {
                __syncthreads();

                if ( tid == 0 )
                {
                  belowN      = binSearch(&data[sec*bufferSz], rMedian, sLen);
                }
              }

              __syncthreads();

              int mStart      = MAX(0,    belowN - hSz );
              int mEnd        = MIN(sLen, belowN + hSz );
              int mLen        = mEnd    - mStart;
              int dataStart   = sStart  + mStart;

              for ( int batch = 0; batch < noBatches; batch++ )
              {
                int bIdx      = batch*bSz + tid ;

                if ( bIdx < mLen )
                {
                  int dataIdx   = dataStart + bIdx ;
                  int bufferIdx = len       + bIdx ;

                  float val         = data[dataIdx];
                  buffer[bufferIdx] = val;

                  FOLD  // Set the max and min for this section  .
                  {
                    // Lower
                    if ( bIdx == 0 )
                    {
                      if ( mStart > 0 )
                        lower[sec] = val;
                      else
                        lower[sec] = -1;
                    }

                    // Upper
                    if ( bIdx == mLen - 1 )
                    {
                      if ( mEnd < sLen )
                        upper[sec] = val;
                      else
                        upper[sec] = -1;
                    }
                  }
                }
              }

              len             += mLen;
              before          += mStart;
            }
          }

          FOLD // Sort the collection of mid sections in SM  .
          {
            __syncthreads();

            bitonicSort(buffer, len, tid, bSz, 1);
          }

          FOLD // Find the bounding vales  .
          {
            __syncthreads();

            if ( tid == 0 )
            {
              maxLower = lower[0];
              minUpper = upper[0];
              locLower = 0;
              locUpper = len;

              for ( int sec = 0; sec < noSections; sec++)
              {
                if ( lower[sec] >= 0 )
                {
                  if ( lower[sec] > maxLower || maxLower == -1 )
                    maxLower = lower[sec];
                }

                if ( upper[sec] >= 0 )
                {
                  if ( upper[sec] < minUpper || minUpper == -1 )
                    minUpper = upper[sec];
                }
              }
            }
          }

          FOLD // Find the location of the bounding vales  .
          {
            __syncthreads();

            int noBatchesMid  = ceilf( len/(float)(bSz) );

            for ( int batch = 0; batch < noBatchesMid; batch++)
            {
              int idx = batch*bSz + tid;

              if ( idx < len )
              {
                if ( buffer[idx] == maxLower )
                {
                  atomicMax(&locLower, idx);
                }

                if ( buffer[idx] == minUpper )
                {
                  atomicMin(&locUpper, idx);
                }
              }
            }

            Fout // TMP Write section values .
            {
              if ( (ite > 0) && (tid == 0) && (bid == 14) )
              {
                __syncthreads();

                for ( int sec = 0; sec < noSections; sec++)
                {
                  int sStart      = MIN(bufferSz*sec,     arrayLength);
                  int sEnd        = MIN(bufferSz*(sec+1), arrayLength);
                  int sLen        = sEnd - sStart;

                  printf("%02i sec\t%02i\tsmid\t%04i\t%22.6f\t%22.6f\t\n", bid,  sec, (int)(sLen/2.0f),  lower[sec], upper[sec] ) ;
                }
              }
            }
          }

          FOLD // Find the index of the median in the buffer  .
          {
            __syncthreads();

            if ( tid == 0 )
            {
              int GMIdx   = arrayLength / 2.0f;
              int SMIdx   = GMIdx - before;

              if      ( (SMIdx <= 0) 	|| (SMIdx <= locLower) )
              {
                medianValue = buffer[locLower];
                //upperVal    = buffer[locLower];
                //medianValue = (upperVal + lowerVal )/2.0;
              }
              else if ( (SMIdx >= len) || (SMIdx >= locUpper) )
              {
                medianValue = buffer[locUpper];
                //lowerVal    = buffer[locUpper];
                //medianValue = (upperVal + lowerVal )/2.0;
              }
              else
              {
                found = true;
              }

              Fout  // TMP  Write lots of output .
              {
                if ( !found )
                {
                  if  ( bid == 12 )
                  {
                    int bid = blockIdx.y  * gridDim.x  + blockIdx.x;        /// Block ID (flat index)
                    printf("\n");
                    printf(" %02i\t%22.6f\t%22.6f\t%22.6f \n", ite,lowerVal, medianValue, upperVal );
                    printf("ERROR: In function %s, In block %02i median not in mid section, median value will be incorrect!  buff %04i | no sec %02i  |  locLower %04i  SMIdx %04i  locUpper %04i \n", __FUNCTION__,  bid, bufferSz, noSections, locLower, SMIdx, locUpper );
                    printf(" ite %02i old rMedian %.6f  new rMedian %.6f \n", ite, rMedian, medianValue );
                    printf(" rMedian %.6f   maxLower %.6f   minUpper %.6f   arrayLength %04i   medIdx %04i    before %04i   len %04i  \n", medianValue, maxLower, minUpper, (int)arrayLength, (int)(arrayLength/2.0), before, len ) ;
                    printf(" %22.6f  %22.6f   %04i  %04i \n", maxLower, minUpper, locLower, locUpper ) ;

                    FOLD // Write section values .
                    {
                      for ( int sec = 0; sec < noSections; sec++)
                      {
                        int sStart      = MIN(bufferSz*sec,     arrayLength);
                        int sEnd        = MIN(bufferSz*(sec+1), arrayLength);
                        int sLen        = sEnd - sStart;

                        printf("sec\t%02i\tsmid\t%04i\t%22.6f\t%22.6f\t\n", sec, (int)(sLen/2.0f),  lower[sec], upper[sec] ) ;
                      }
                    }

                    FOLD // TMP  write medians  .
                    {
                      printf("%02i medians:", noSections );
                      for ( int i = 0; i < noSections; i++ )
                      {
                        printf("\t%.6f", medianA[i] );
                      }
                      printf("\n");
                    }
                  }
                }
              }

              if ( ite > 40 )
              {
                printf("\nERROR: in %s. Block %02i iterated %i times and did not find the correct median.\n", __FUNCTION__, bid, ite+1 );
                found = true;
                SMIdx = MAX(0,SMIdx);
                SMIdx = MIN(len-1,SMIdx);
              }

              if (found)
              {
                FOLD // Median selection  .
                {
                  if ( (arrayLength & 1) )    // odd   .
                  {
                    medianValue = buffer[SMIdx];
                  }
                  else                        // even  .
                  {
                    // mean
                    //medianValue = ( smBuffer[SMIdx-1] + smBuffer[mIdx] ) / 2.0f;

                    // lower
                    medianValue = buffer[SMIdx-1];

                    // upper
                    //medianValue = buffer[SMIdx];
                  }
                }

                //if  ( bid == 12 )
                //  printf("FOUND\n %02i\t%22.6f\t%22.6f\t%22.6f \n", ite, lowerVal, medianValue, upperVal );

                if ( ite > 20 )
                  printf("\nFound median in block %02i after %02i iterations.\n", bid, ite );
              }
            }
          }

          ite++;
          __syncthreads();
        }
      }
    }
    __syncthreads();
  }
  else
  {
    if( tid == 0 )
    {
      printf("\nERROR: in %s number of sections (%.2f) is larger than the max number compiled with (%i).\n", __FUNCTION__, arrayLength/(float)bufferSz, maxSub );
    }
    return 0;
  }

  return medianValue;
}

template< int stride, int BS_MAX, typename stpType>
__global__ void normAndSpread(fcomplexcu* data, stpType lens)
{
  const int bid = blockIdx.y  * gridDim.x  + blockIdx.x;        /// Block ID (flat index)
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;       /// Thread ID in block (flat index)
  const int bSz = blockDim.x  * blockDim.y;                     /// Block size

  int width = lens.val[bid];
  if (width)
  {
    __shared__ float sData[BS_MAX];

    float   medianValue;
    float   factor;

    int batches = ceilf( width / (float) bSz );

    // Stride input data
    data += stride*bid;

    if ( width <= BS_MAX )
    {
      FOLD // Calculate and store powers in shared memory  .
      {
        for ( int batch = 0; batch < batches; batch++)
        {
          int idx = batch*bSz+tid;

          if ( idx < width )
          {
            fcomplexcu val  = data[idx];
            sData[idx]      = val.r*val.r+val.i*val.i;
          }
        }
      }

      medianValue = cuMedianOne(sData, width);
    }
    else
    {
      float* powers = (float*)(&data[stride/2]); // Stride should always be a power of 2

      FOLD // Calculate and store powers in device memory  .
      {
        for ( int batch = 0; batch < batches; batch++)
        {
          fcomplexcu val            = data[batch*bSz+tid];
          powers[batch*bSz  + tid]  = val.r*val.r+val.i*val.i;
        }
      }

      medianValue = cuMedianBySection<BS_MAX>(powers, sData, width);
    }

    // Calculate normalisation factor
    factor = 1.0 / sqrt( medianValue / LN2 );

//    if ( tid == 0 )
//    {
//      float sec = width / (float)BS_MAX ;
//      printf("%02i  batches: %4.2f %4.2f section  median %.6f  factor: %10.10f \n", bid, width / (float) bSz, sec, medianValue, factor );
//    }

    batches = ceil( stride / (float) bSz );

    // Write spread by 2 and normalise
    for ( int batch = batches-1; batch >= 0; batch--)
    {
      // Read all values into registers
      fcomplexcu val = data[batch*bSz+tid];
      __syncthreads();

      int idx = batch*bSz+tid;

      if ( (idx)*2 < stride)
      {
        // Set the value to normalised complex number spread by 2
        if ( idx < width )
        {
          val.i *= factor;
          val.r *= factor;
        }
        else
        {
          val.i = 0;
          val.r = 0;
        }
        data[(idx)*2]     = val;

        // Set every second value to 0
        val.i = 0;
        val.r = 0;
        data[(idx)*2+1]   = val;
      }
    }
  }
}

__global__ void normAndSpread_writeOnly(fcomplexcu* data, int width)
{

  const int bid = blockIdx.y  * gridDim.x  + blockIdx.x;        /// Block ID (flat index)
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;       /// Thread ID in block (flat index)

  const int idx = bid*(blockDim.x * blockDim.y) + tid;                     /// Block size

  if (idx < width )
  {
    fcomplexcu val  = data[idx];
    val.r *= 1;
    val.i *= 1;
    data[idx]       = val;
  }

}

template<int width, int buffer>
__host__ void normAndSpread_b(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t stream, cuFFdotBatch* batch, uint stack )
{
  int harm    = 0;
  int stp     = 0;

  cuFfdotStack* cStack = &batch->stacks[stack];
  harm = cStack->startIdx;

  int noInput = cStack->noInStack * batch->noSteps ;

  if      ( noInput <= 1   )
  {
    int01 iLen;
    for (int si = 0; si < cStack->noInStack; si++)
    {
      for (int step = 0; step < batch->noSteps; step++)
      {
        rVals* rVal = &((*batch->rInput)[step][harm]);

        if ( stp < noInput )
          iLen.val[stp] = rVal->numdata ;

        stp++;
      }
      harm++;
    }

    normAndSpread<width, buffer, int01><<< dimGrid,  dimBlock, 0, stream >>>(cStack->d_iData, iLen);
  }
  else if ( noInput <= 2   )
  {
    int02 iLen;
    for (int si = 0; si < cStack->noInStack; si++)
    {
      for (int step = 0; step < batch->noSteps; step++)
      {
        rVals* rVal = &((*batch->rInput)[step][harm]);

        iLen.val[stp] = rVal->numdata ;
        stp++;
      }
      harm++;
    }

    normAndSpread<width, buffer, int02><<< dimGrid,  dimBlock, 0, stream >>>(cStack->d_iData, iLen);
  }
  else if ( noInput <= 4   )
  {
    int04 iLen;
    for (int si = 0; si < cStack->noInStack; si++)
    {
      for (int step = 0; step < batch->noSteps; step++)
      {
        rVals* rVal = &((*batch->rInput)[step][harm]);

        iLen.val[stp] = rVal->numdata ;
        stp++;
      }
      harm++;
    }

    normAndSpread<width, buffer, int04><<< dimGrid,  dimBlock, 0, stream >>>(cStack->d_iData, iLen);
  }
  else if ( noInput <= 8   )
  {
    int08 iLen;
    for (int si = 0; si < cStack->noInStack; si++)
    {
      for (int step = 0; step < batch->noSteps; step++)
      {
        rVals* rVal = &((*batch->rInput)[step][harm]);

        iLen.val[stp] = rVal->numdata ;
        stp++;
      }
      harm++;
    }

    normAndSpread<width, buffer, int08><<< dimGrid,  dimBlock, 0, stream >>>(cStack->d_iData, iLen);
  }
  else if ( noInput <= 16  )
  {
    int16 iLen;
    for (int si = 0; si < cStack->noInStack; si++)
    {
      for (int step = 0; step < batch->noSteps; step++)
      {
        rVals* rVal = &((*batch->rInput)[step][harm]);

        iLen.val[stp] = rVal->numdata ;
        stp++;
      }
      harm++;
    }

    normAndSpread<width, buffer, int16><<< dimGrid,  dimBlock, 0, stream >>>(cStack->d_iData, iLen);
  }
  else if ( noInput <= 32  )
  {
    int32 iLen;
    for (int si = 0; si < cStack->noInStack; si++)
    {
      for (int step = 0; step < batch->noSteps; step++)
      {
        rVals* rVal = &((*batch->rInput)[step][harm]);

        iLen.val[stp] = rVal->numdata ;
        stp++;
      }
      harm++;
    }

    normAndSpread<width, buffer, int32><<< dimGrid,  dimBlock, 0, stream >>>(cStack->d_iData, iLen);
  }
  else if ( noInput <= 64  )
  {
    int64 iLen;
    for (int si = 0; si < cStack->noInStack; si++)
    {
      for (int step = 0; step < batch->noSteps; step++)
      {
        rVals* rVal = &((*batch->rInput)[step][harm]);

        iLen.val[stp] = rVal->numdata ;
        stp++;
      }
      harm++;
    }

    normAndSpread<width, buffer, int64><<< dimGrid,  dimBlock, 0, stream >>>(cStack->d_iData, iLen);
  }
  else if ( noInput <= 128 )
  {
    int128 iLen;
    for (int si = 0; si < cStack->noInStack; si++)
    {
      for (int step = 0; step < batch->noSteps; step++)
      {
        rVals* rVal = &((*batch->rInput)[step][harm]);

        iLen.val[stp] = rVal->numdata ;
        stp++;
      }
      harm++;
    }

    normAndSpread<width, buffer, int128><<< dimGrid,  dimBlock, 0, stream >>>(cStack->d_iData, iLen);
  }
  else
  {
    fprintf(stderr,"ERROR: %s has not been set up to work with %i input sections.",__FUNCTION__, noInput);
  }
}

template<int width>
__host__ void normAndSpread_w(dim3 dimGrid, dim3 dimBlock, int i1, hipStream_t stream, cuFFdotBatch* batch, uint stack )
{
  cuFfdotStack* cStack  = &batch->stacks[stack];
  int bufWidth          = MIN( globalFloat01, (cStack->width/2.0f)/globalFloat02 );

  bufWidth              = MAX( (cStack->width/2.0f)/32.0, bufWidth );
  bufWidth              = MAX( 256, bufWidth );

  switch ( bufWidth )
  {
    case 256   :
    {
      normAndSpread_b<width,256>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 512   :
    {
      normAndSpread_b<width,512>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 1024  :
    {
      normAndSpread_b<width,1024>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 2048  :
    {
      normAndSpread_b<width,2048>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 4096  :
    {
      normAndSpread_b<width,4096>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 8192  :
    {
      normAndSpread_b<width,8192>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    default    :
    {
      fprintf(stderr, "ERROR: %s has not been templated for sorting with %i elements.\n", __FUNCTION__, bufWidth );
      exit(EXIT_FAILURE);
    }
  }
}

__host__ void normAndSpread_f2(hipStream_t stream, cuFFdotBatch* batch, uint stack )
{
  dim3 dimBlock, dimGrid;
  cuFfdotStack* cStack = &batch->stacks[stack];

  // Blocks of 1024 threads ( the maximum number of threads per block )
  dimBlock.x = NAS_DIMX;
  dimBlock.y = NAS_DIMY;
  dimBlock.z = 1;

  // One block per harmonic, thus we can sort input powers in Shared memory
  dimGrid.x = ceil(cStack->width*cStack->height*batch->noSteps / float(dimBlock.x*dimBlock.y*dimBlock.z));
  dimGrid.y = 1;

  normAndSpread_writeOnly<<<dimGrid, dimBlock, 0, stream>>>(cStack->d_iData, cStack->width*cStack->height*batch->noSteps );
}

__host__ void normAndSpread_f(hipStream_t stream, cuFFdotBatch* batch, uint stack )
{
  //normAndSpread_f2(stream, batch, stack );
  //return;

  dim3 dimBlock, dimGrid;
  int i1 = 0;
  cuFfdotStack* cStack = &batch->stacks[stack];

  // Blocks of 1024 threads ( the maximum number of threads per block )
  dimBlock.x = NAS_DIMX;
  dimBlock.y = NAS_DIMY;
  dimBlock.z = 1;

  // One block per harmonic, thus we can sort input powers in Shared memory
  dimGrid.x = cStack->noInStack * batch->noSteps;
  dimGrid.y = 1;

  switch (cStack->width)
  {
    case 128   :
    {
      normAndSpread_w<128>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 256   :
    {
      normAndSpread_w<256>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 512   :
    {
      normAndSpread_w<512>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 1024  :
    {
      normAndSpread_w<1024>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 2048  :
    {
      normAndSpread_w<2048>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 4096  :
    {
      normAndSpread_w<4096>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 8192  :
    {
      normAndSpread_w<8192>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 16384 :
    {
      normAndSpread_w<16384>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    case 32768 :
    {
      normAndSpread_w<32768>(dimGrid, dimBlock, i1, stream, batch, stack);
      break;
    }
    default    :
    {
      fprintf(stderr, "ERROR: %s has not been templated for %lu steps\n", __FUNCTION__, cStack->width);
      exit(EXIT_FAILURE);
    }
  }

  CUDA_SAFE_CALL(hipGetLastError(), "Calling the normalisation and spreading kernel.");
}
